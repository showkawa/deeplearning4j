#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma (iuriish@yahoo.com), created on 08.11.2018
// @author raver119@gmail.com
//

#include "../scalar_bool.h"
#include <system/op_boilerplate.h>
#include <types/types.h>

#include "../legacy_ops.h"

using namespace simdOps;

////////////////////////////////////////////////////////////////////////
template <typename X, typename Z, typename OpType>
__global__ void scalarAlongDimension(void const* x, Nd4jLong const* xShapeInfo,
                                    void *extraParams,
                                    void *z, Nd4jLong const* zShapeInfo,
                                    void const* scalars,
                                    int *dimension, int dimensionLength,
                                    Nd4jLong const* tadShapeInfo, Nd4jLong const* tadOffsets,
                                    Nd4jLong const* tadShapeInfoZ, Nd4jLong const* tadOffsetsZ) {

    functions::scalar::ScalarBoolTransform<X,Z>::template transformCuda<OpType>(x, xShapeInfo, extraParams, z, zShapeInfo, scalars, dimension, dimensionLength, tadShapeInfo, tadOffsets, tadShapeInfoZ, tadOffsetsZ);
}


////////////////////////////////////////////////////////////////////////
template <typename X, typename Z, typename OpType>
__global__ void scalarSimpleShaped(void const* x, void const* y, Nd4jLong const* xShapeInfo, void *params, void *z, Nd4jLong const* zShapeInfo, int *allocationBuffer) {

    functions::scalar::ScalarBoolTransform<X,Z>::template transformCuda<OpType>(y, x, xShapeInfo, params, z, zShapeInfo, allocationBuffer);
}





// *********************************************************************//
// *********************************************************************//
namespace functions {
namespace scalar    {

////////////////////////////////////////////////////////////////////////
template<typename X, typename Z>
template<typename OpType>
__device__ void  ScalarBoolTransform<X, Z>::transformCuda(void const* vscalar,
                                                        void const* vy, Nd4jLong const* yShapeInfo,
                                                        void *vparams,
                                                        void *vz, Nd4jLong const* zShapeInfo,
                                                        int *allocationBuffer) {
    auto scalar = reinterpret_cast<X const* >(vscalar)[0];
    auto y      = reinterpret_cast<X const* >(vy);
    auto params = reinterpret_cast<X*>(vparams);
    auto z      = reinterpret_cast<Z*>(vz);

    auto yRank   = shape::rank(yShapeInfo);
    auto yEWS    = shape::elementWiseStride(yShapeInfo);
    auto yShape  = shape::shapeOf(yShapeInfo);
    auto yStride = shape::stride(yShapeInfo);

    auto zRank   = shape::rank(zShapeInfo);
    auto zEWS    = shape::elementWiseStride(zShapeInfo);
    auto zShape  = shape::shapeOf(zShapeInfo);
    auto zStride = shape::stride(zShapeInfo);

    int totalThreads = gridDim.x * blockDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ int len;
    if(threadIdx.x == 0)
        len = shape::length(yShapeInfo);
    __syncthreads();

    if(yEWS >= 1 && zEWS >= 1 && shape::order(yShapeInfo) == shape::order(zShapeInfo)) {
            transformCuda<OpType>(len, vscalar, vy, yEWS, vparams, vz, zEWS, allocationBuffer);
    }
    else {
        for (Nd4jLong i = tid; i < len; i+= totalThreads)
            z[shape::getIndexOffset(i, zShapeInfo)] = OpType::op(y[shape::getIndexOffset(i, yShapeInfo)], scalar, params);
    }
}

////////////////////////////////////////////////////////////////////////
template<typename X, typename Z>
template<typename OpType>
__device__ void  ScalarBoolTransform<X, Z>::transformCuda(Nd4jLong len,
                                                          void const* vx,
                                                          void const* vy, Nd4jLong yEWS,
                                                          void *vparams,
                                                          void *vz, Nd4jLong zEWS,
                                                          int *allocationBuffer) {

    auto x = reinterpret_cast<X const*>(vx)[0];
    auto y = reinterpret_cast<X const*>(vy);
    auto z = reinterpret_cast<Z*>(vz);
    auto params = reinterpret_cast<X*>(vparams);

    int totalThreads = gridDim.x * blockDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    Nd4jLong i = tid;
    if(yEWS == 1 && zEWS == 1) {
        for (; i < len; i += totalThreads)
            z[i] = OpType::op(y[i], x, params);
    }
    else {
        for (; i < len; i += totalThreads)
            z[i * zEWS] = OpType::op(y[i * yEWS], x, params);
    }
}


////////////////////////////////////////////////////////////////////////
template<typename X, typename Z>
template<typename OpType>
__device__ void  ScalarBoolTransform<X, Z>::transformCuda(void const* vx, Nd4jLong const* xShapeInfo,
                                                        void *vextraParams,
                                                        void *vz, Nd4jLong const* zShapeInfo,
                                                        void const* vscalars,
                                                        int *dimension, int dimensionLength,
                                                        Nd4jLong const* tadShapeInfo, Nd4jLong const* tadOffsets,
                                                        Nd4jLong const* tadShapeInfoZ, Nd4jLong const* tadOffsetsZ) {
    auto x = reinterpret_cast<X const*>(vx);
    auto scalars = reinterpret_cast<X const*>(vscalars);
    auto z = reinterpret_cast<Z*>(vz);
    auto extraParams = reinterpret_cast<X*>(vextraParams);

    if (tadShapeInfoZ == nullptr) {
        tadShapeInfoZ = tadShapeInfo;
        tadOffsetsZ = tadOffsets;
    }

    // tad preparation
    auto tadEws = shape::elementWiseStride(tadShapeInfo);
    auto zEws = shape::elementWiseStride(tadShapeInfoZ);
    auto tadLength = shape::length(tadShapeInfo);//shape::tadLength(xShapeInfo, dimension, dimensionLength);
    auto numTads =shape::length(xShapeInfo) / tadLength;

    if (tadEws > 0 && zEws > 0 && shape::order(tadShapeInfo) == shape::order(zShapeInfo)) {

        // main loop, rolling over tads
        for (int r = blockIdx.x; r < numTads; r += gridDim.x) {
            Z *oZ = z + tadOffsetsZ[r];
            auto oX = x + tadOffsets[r];

            auto s = scalars[r];

            for (int f = threadIdx.x; f < tadLength; f += blockDim.x)
                oZ[f * zEws] = OpType::op(oX[f * tadEws], s, extraParams);
        }
    } else {
        // main loop, rolling over tads
        for (int r = blockIdx.x; r < numTads; r += gridDim.x) {
            Z *oZ = z + tadOffsetsZ[r];
            auto oX = x + tadOffsets[r];

            auto s = scalars[r];

            for (int f = threadIdx.x; f < tadLength; f += blockDim.x)
                oZ[shape::getIndexOffset(f, tadShapeInfoZ)] = OpType::op(oX[shape::getIndexOffset(f, tadShapeInfo)], s, extraParams);
        }
    }
}


////////////////////////////////////////////////////////////////////////
template<typename X, typename Z>
template <typename OpType>
_CUDA_H void ScalarBoolTransform<X, Z>::intermediateAlongDimension(dim3& launchDims, hipStream_t *stream,
                                                                void const* x, Nd4jLong const* xShapeInfo,
                                                                void *z, Nd4jLong const* zShapeInfo,
                                                                void const* scalars,
                                                                void *extraParams,
                                                                int *dimension, int dimensionLength,
                                                                Nd4jLong const* tadShapeInfo, Nd4jLong const* tadOffsets,
                                                                Nd4jLong const* tadShapeInfoZ, Nd4jLong const* tadOffsetsZ) {

    scalarAlongDimension<X, Z, OpType><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(x, xShapeInfo, extraParams, z, zShapeInfo, scalars, dimension, dimensionLength, tadShapeInfo, tadOffsets, tadShapeInfoZ, tadOffsetsZ);
    sd::DebugHelper::checkErrorCode(stream, "scalarAlongDim(...) failed");
}

////////////////////////////////////////////////////////////////////////
template<typename X, typename Z>
template<typename OpType>
void _CUDA_H ScalarBoolTransform<X,Z>::intermediateShaped(dim3& launchDims, hipStream_t *stream,
                                                            void const* vx, Nd4jLong const* xShapeInfo,
                                                            void *vz, Nd4jLong const* zShapeInfo,
                                                            void const* vscalar,
                                                            void *vextraParams, int *allocPointer){

    scalarSimpleShaped<X, Z, OpType><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(vx, vscalar, xShapeInfo, vextraParams, vz, zShapeInfo, allocPointer);
    sd::DebugHelper::checkErrorCode(stream, "scalarSimpleShaped(...) failed");
}

////////////////////////////////////////////////////////////////////////
template<typename X, typename Y>
void ScalarBoolTransform<X,Y>::executeCudaShaped(dim3& launchDims, hipStream_t *stream,
                                                int opNum,
                                                void const* vx, Nd4jLong const* xShapeInfo,
                                                void *vz, Nd4jLong const* zShapeInfo,
                                                void const* vscalar,
                                                void const* vextraParams) {

    if (sd::Environment::getInstance().isDebugAndVerbose())
        printf("H14 opNum:[%i]\n", opNum);

    DISPATCH_BY_OPNUM_TT(intermediateShaped, PARAMS(launchDims, stream, vx, xShapeInfo, vz, zShapeInfo, vscalar, const_cast<void*>(vextraParams), nullptr), SCALAR_BOOL_OPS);
}

////////////////////////////////////////////////////////////////////////
template<typename X, typename Y>
void ScalarBoolTransform<X,Y>::executeCudaAlongDimension(dim3& launchDims, hipStream_t *stream, int opNum, void const* vx, Nd4jLong const* xShapeInfo, void *vz, Nd4jLong const* zShapeInfo, void const* vscalars, void *vextraParams, int *dimension, int dimensionLength, Nd4jLong const* tadShapeInfo, Nd4jLong const* tadOffsets, Nd4jLong const* tadShapeInfoZ, Nd4jLong const* tadOffsetsZ) {
    DISPATCH_BY_OPNUM_TT(intermediateAlongDimension, PARAMS(launchDims, stream, vx, xShapeInfo, vz, zShapeInfo, vscalars, vextraParams, dimension, dimensionLength, tadShapeInfo, tadOffsets, tadShapeInfoZ, tadOffsetsZ), SCALAR_BOOL_OPS);
}

    BUILD_DOUBLE_TEMPLATE(template class ND4J_EXPORT ScalarBoolTransform, , LIBND4J_TYPES, BOOL_TYPES);
}
}

