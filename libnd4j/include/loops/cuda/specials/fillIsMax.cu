#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
// @author Yurii Shyrma, created on 15.11.2018
//

#include <loops/special_kernels.h>

namespace sd {

////////////////////////////////////////////////////////////////////////
    template <typename T>
    __global__ void execFillIsMax(void *vdZ, const Nd4jLong *xShapeInfo, Nd4jLong length, long idx) {
        auto dz = reinterpret_cast<T*>(vdZ);
        int tid = blockIdx.x * blockDim.x + threadIdx.x;

        for (Nd4jLong i = tid; i < length; i += blockDim.x * gridDim.x)
            dz[shape::getIndexOffset(i, xShapeInfo)] = (i == idx ? (T) 1 : (T) 0);
    }

////////////////////////////////////////////////////////////////////////
    template <typename T>
    __host__ void fillIsMaxGeneric(dim3 &launchDims, hipStream_t *stream, void *dx, const Nd4jLong *xShapeInfo, Nd4jLong length, long idx) {
        execFillIsMax<T><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(dx, xShapeInfo, length, idx);
        sd::DebugHelper::checkErrorCode(stream, "fillIsMax(...) failed");
    }


    BUILD_SINGLE_TEMPLATE(template void ND4J_EXPORT fillIsMaxGeneric, (dim3& launchDims, hipStream_t *stream, void* dz, const Nd4jLong *zShapeInfo, Nd4jLong length, long idx), LIBND4J_TYPES);
}