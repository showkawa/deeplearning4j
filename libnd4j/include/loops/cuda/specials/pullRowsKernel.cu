#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
// @author Yurii Shyrma, created on 15.11.2018
//

#include <loops/special_kernels.h>

namespace sd {

///////////////////////////////////////////////////////////////////////
    template<typename T>
    __device__ void pullRowsKernel(void *vx,
                                   void *vz,
                                   Nd4jLong len,
                                   Nd4jLong *indexes,
                                   Nd4jLong const* tadShapeInfo, Nd4jLong const* tadOffsets,
                                   Nd4jLong const* zTadShapeInfo, Nd4jLong const* zTadOffsets) {

        auto x = reinterpret_cast<T *>(vx);
        auto z = reinterpret_cast<T *>(vz);
        auto xEWS = shape::elementWiseStride(tadShapeInfo);
        auto zEWS = shape::elementWiseStride(zTadShapeInfo);
        auto tadLength = shape::length(tadShapeInfo);

        if (xEWS >= 1 && zEWS >= 1) {
            for (int idx = blockIdx.x; idx < len; idx += gridDim.x) {
                T *rX = x + tadOffsets[indexes[idx]];
                T *rZ = z + zTadOffsets[idx];

                for (int i = threadIdx.x; i < tadLength; i += blockDim.x) {
                    rZ[i * zEWS] = rX[i * xEWS];
                }
            }
        } else {
            for (int idx = blockIdx.x; idx < len; idx += gridDim.x) {
                T *rX = x + tadOffsets[indexes[idx]];
                T *rZ = z + zTadOffsets[idx];

                for (int i = threadIdx.x; i < tadLength; i += blockDim.x) {
                    auto xOffset = shape::getIndexOffset(i, tadShapeInfo);
                    auto zOffset = shape::getIndexOffset(i, zTadShapeInfo);
                    rZ[zOffset] = rX[xOffset];
                }
            }
        }
    }

///////////////////////////////////////////////////////////////////////
    template<typename T>
    __global__ void execPullRowsKernel(void *vx,
                                       void *vz,
                                       Nd4jLong len,
                                       Nd4jLong *indexes,
                                       Nd4jLong const* tadShapeInfo, Nd4jLong const* tadOffsets,
                                       Nd4jLong const* zTadShapeInfo, Nd4jLong const* zTadOffsets) {

        pullRowsKernel<T>(vx, vz, len, indexes, tadShapeInfo, tadOffsets, zTadShapeInfo, zTadOffsets);
    }

///////////////////////////////////////////////////////////////////////
    template<typename T>
    __host__ void pullRowsKernelGeneric(dim3 &launchDims, hipStream_t *stream,
                                        void *vx,
                                        void *vz,
                                        Nd4jLong len,
                                        Nd4jLong *indexes,
                                        Nd4jLong const* tadShapeInfo, Nd4jLong const* tadOffsets,
                                        Nd4jLong const* zTadShapeInfo, Nd4jLong const* zTadOffsets) {

        execPullRowsKernel<T><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(vx, vz, len, indexes, tadShapeInfo, tadOffsets, zTadShapeInfo, zTadOffsets);
        sd::DebugHelper::checkErrorCode(stream, "pullRows(...) failed");
    }

    BUILD_SINGLE_TEMPLATE(template void ND4J_EXPORT pullRowsKernelGeneric, (dim3 & launchDims, hipStream_t * stream, void * vx, void * vz, Nd4jLong len, Nd4jLong * indexes, Nd4jLong const* tadShapeInfo, Nd4jLong const* tadOffsets, Nd4jLong const* zTadShapeInfo, Nd4jLong const* zTadOffsets), LIBND4J_TYPES);
}

