#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
// @author Yurii Shyrma, created on 15.11.2018
//

#include <loops/special_kernels.h>

namespace sd {

////////////////////////////////////////////////////////////////////////
    template<typename T>
    __device__ void
    tearKernel(void *vx, Nd4jLong const* xShapeInfo, Nd4jPointer *targets, Nd4jLong const* zShapeInfo, Nd4jLong const* tadShapeInfo,
               Nd4jLong const* tadOffsets) {



        __shared__         Nd4jLong tadLength;
        __shared__ int tadEWS;
        __shared__ int zEWS;
//        __shared__ int tadRank;
        __shared__         Nd4jLong numTads;
//        __shared__ int zRank;
//        __shared__        Nd4jLong *tadShape;
//        __shared__        Nd4jLong *tadStride;
//        __shared__        Nd4jLong const* zShape;
//        __shared__        Nd4jLong const* zStride;
        __shared__ T* x;
        if (threadIdx.x == 0) {
            tadLength = shape::length(tadShapeInfo);
            tadEWS = shape::elementWiseStride(tadShapeInfo);
            zEWS = shape::elementWiseStride(zShapeInfo);
            numTads = shape::length(xShapeInfo) / tadLength;
            x = static_cast<T *>(vx);
        }
        __syncthreads();

        for (Nd4jLong r = blockIdx.x; r < numTads; r += gridDim.x) {
            T *z = (T *) targets[r];
            T *s = x + tadOffsets[r];

            if (zEWS > 0 && tadEWS > 0) {
                for (Nd4jLong i = threadIdx.x; i < tadLength; i += blockDim.x)
                    z[i * zEWS] = s[i * tadEWS];
            } else {

                for (Nd4jLong j = threadIdx.x; j < tadLength; j += blockDim.x) {
                    auto xOffset = shape::getIndexOffset(j, tadShapeInfo);
                    auto zOffset = shape::getIndexOffset(j, zShapeInfo);

                    z[zOffset] = s[xOffset];
                }
            }
        }
    }


////////////////////////////////////////////////////////////////////////
    template<typename T>
    __global__ void
    execTearKernel(void *vx, Nd4jLong const* xShapeInfo, Nd4jPointer *targets, Nd4jLong const* zShapeInfo, Nd4jLong const* tadShapeInfo,
                   Nd4jLong const* tadOffsets) {

        tearKernel<T>(vx, xShapeInfo, targets, zShapeInfo, tadShapeInfo, tadOffsets);
    }

////////////////////////////////////////////////////////////////////////
    template<typename T>
    __host__ void tearKernelGeneric(dim3 &launchDims, hipStream_t *stream,
                                    void *vx, Nd4jLong const* xShapeInfo,
                                    Nd4jPointer *targets, Nd4jLong const* zShapeInfo,
                                    Nd4jLong const* tadShapeInfo, Nd4jLong const* tadOffsets) {

        execTearKernel<T><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(vx, xShapeInfo, targets, zShapeInfo, tadShapeInfo, tadOffsets);
        sd::DebugHelper::checkErrorCode(stream, "tear(...) failed");
    }

    BUILD_SINGLE_TEMPLATE(template void ND4J_EXPORT tearKernelGeneric, (dim3 & launchDims, hipStream_t * stream, void * vx, Nd4jLong const* xShapeInfo, Nd4jPointer *targets, Nd4jLong const* zShapeInfo, Nd4jLong const* tadShapeInfo, Nd4jLong const* tadOffsets), LIBND4J_TYPES);
}