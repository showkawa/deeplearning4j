#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author raver119@gmail.com
//

#include <system/op_boilerplate.h>
#include <loops/broadcasting_bool.h>
#include <loops/legacy_ops.h>
#include <types/types.h>
#include <system/Environment.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <string>
#include <stdexcept>
#include <helpers/StringUtils.h>

using namespace simdOps;

//////////////////////////////////////////////////////////////////////////
template<typename X, typename Z, typename OpClass>
static __global__ void broadcastBoolSimple(
        void const* x,
        Nd4jLong const* xShapeInfo,
        void const* y,
        Nd4jLong const* yShapeInfo,
        void *z,
        Nd4jLong const* zShapeInfo,
        void *extraParams,
        int *dimension,
        int dimensionLength, Nd4jLong const* tadOnlyShapeInfo, Nd4jLong const* tadOffsets, Nd4jLong const* tadOnlyShapeInfoZ, Nd4jLong const* tadOffsetsZ) {

    functions::broadcast::BroadcastBool<X, Z>::template transformCuda<OpClass>(x,xShapeInfo,y,yShapeInfo,z,zShapeInfo, extraParams, dimension,dimensionLength,tadOnlyShapeInfo,tadOffsets,tadOnlyShapeInfoZ,tadOffsetsZ);
}

//////////////////////////////////////////////////////////////////////////
template<typename X, typename Z, typename OpClass>
static __global__ void broadcastBoolSimple(const void const* x, const Nd4jLong const* xShapeInfo,
                                           const void const* y, const Nd4jLong const* yShapeInfo,
                                                 void *z, const Nd4jLong const* zShapeInfo,
                                                 void *extraParams) {

    functions::broadcast::BroadcastBool<X, Z>::template transformCuda<OpClass>(x, xShapeInfo, y, yShapeInfo, z, zShapeInfo, extraParams);
}
//////////////////////////////////////////////////////////////////////////
template<typename X, typename Z, typename OpClass>
static __global__ void broadcastBoolInverseSimple(
        void const* x,
        Nd4jLong const* xShapeInfo,
        void const* y,
        Nd4jLong const* yShapeInfo,
        void *z,
        Nd4jLong const* zShapeInfo,
        void *extraParams,
        int *dimension,
        int dimensionLength, Nd4jLong const* tadOnlyShapeInfo, Nd4jLong const* tadOffsets, Nd4jLong const* tadOnlyShapeInfoZ, Nd4jLong const* tadOffsetsZ) {

    functions::broadcast::BroadcastBool<X, Z>::template transformInverseCuda<OpClass>(x,xShapeInfo,y,yShapeInfo,z,zShapeInfo,extraParams,dimension,dimensionLength,tadOnlyShapeInfo,tadOffsets,tadOnlyShapeInfoZ,tadOffsetsZ);
}

namespace functions {
namespace broadcast {

//////////////////////////////////////////////////////////////////////////
template<typename X, typename Z>
template <typename OpClass>
__host__ void BroadcastBool<X,Z>::intermediateBroadcast(dim3 launchDims, hipStream_t *stream, void const* x, Nd4jLong const* xShapeInfo, void const* y, Nd4jLong const* yShapeInfo, void* z, Nd4jLong const* zShapeInfo, void *extraParams, int *dimension, int dimensionLength, Nd4jLong const* tadOnlyShapeInfo, Nd4jLong const* tadOffsets, Nd4jLong const* tadOnlyShapeInfoZ, Nd4jLong const* tadOffsetsZ) {
    broadcastBoolSimple<X, Z, OpClass><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(x, xShapeInfo, y, yShapeInfo, z, zShapeInfo, extraParams, dimension, dimensionLength, tadOnlyShapeInfo, tadOffsets, tadOnlyShapeInfoZ, tadOffsetsZ);
    sd::DebugHelper::checkErrorCode(stream, "intermediateBroadcastBool(...) failed");
}

//////////////////////////////////////////////////////////////////////////
template<typename X, typename Z>
template <typename OpClass>
__host__ void BroadcastBool<X,Z>::intermediateBroadcast(dim3 launchDims, hipStream_t *stream,
                                                        const void *x, const Nd4jLong *xShapeInfo,
                                                        const void *y, const Nd4jLong *yShapeInfo,
                                                              void *z, const Nd4jLong *zShapeInfo,
                                                              void *extraParams) {

    broadcastBoolSimple<X, Z, OpClass><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(x, xShapeInfo, y, yShapeInfo, z, zShapeInfo, extraParams);
    sd::DebugHelper::checkErrorCode(stream, "intermediateBroadcastBool(...) failed");
}

//////////////////////////////////////////////////////////////////////////
template<typename X, typename Y>
__host__ void BroadcastBool<X,Y>::execBroadcast(dim3 launchDims, hipStream_t *stream, int opNum, void const* x, Nd4jLong const* xShapeInfo, void const* y, Nd4jLong const* yShapeInfo, void *z, Nd4jLong const* zShapeInfo, void *extraParams, int *dimension, int dimensionLength, Nd4jLong const* tadOnlyShapeInfo, Nd4jLong const* tadOffsets, Nd4jLong const* tadOnlyShapeInfoZ, Nd4jLong const* tadOffsetsZ) {

    DISPATCH_BY_OPNUM_TT(intermediateBroadcast,  PARAMS(launchDims, stream, x, xShapeInfo, y, yShapeInfo, z, zShapeInfo, extraParams, dimension, dimensionLength, tadOnlyShapeInfo, tadOffsets, tadOnlyShapeInfoZ, tadOffsetsZ), OPS_A(BROADCAST_BOOL_OPS))
	DEBUG_KERNEL(stream, opNum);
}

//////////////////////////////////////////////////////////////////////////
template<typename X, typename Y>
__host__ void BroadcastBool<X,Y>::execBroadcast(dim3 launchDims, hipStream_t *stream, const int opNum,
                                                const void *x, const Nd4jLong *xShapeInfo,
                                                const void *y, const Nd4jLong *yShapeInfo,
                                                      void *z, const Nd4jLong *zShapeInfo,
                                                      void *extraParams) {

    DISPATCH_BY_OPNUM_TT(intermediateBroadcast,  PARAMS(launchDims, stream, x, xShapeInfo, y, yShapeInfo, z, zShapeInfo, extraParams), OPS_A(BROADCAST_BOOL_OPS))
    DEBUG_KERNEL(stream, opNum);
}

//////////////////////////////////////////////////////////////////////////
        template<typename X, typename Z>
        template <typename OpClass>
        __host__ void BroadcastBool<X,Z>::intermediateInverseBroadcast(dim3 launchDims, hipStream_t *stream, void const* x, Nd4jLong const* xShapeInfo, void const* y, Nd4jLong const* yShapeInfo, void *z, Nd4jLong const* zShapeInfo, void *extraParams, int *dimension, int dimensionLength, Nd4jLong const* tadOnlyShapeInfo, Nd4jLong const* tadOffsets, Nd4jLong const* tadOnlyShapeInfoZ, Nd4jLong const* tadOffsetsZ) {
            broadcastBoolInverseSimple<X, Z, OpClass><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(x, xShapeInfo, y, yShapeInfo, z, zShapeInfo, extraParams, dimension, dimensionLength, tadOnlyShapeInfo, tadOffsets, tadOnlyShapeInfoZ, tadOffsetsZ);
            sd::DebugHelper::checkErrorCode(stream, "intermediateBroadcastBool(...) failed");
        }

//////////////////////////////////////////////////////////////////////////
        template<typename X, typename Y>
        __host__ void BroadcastBool<X,Y>::execInverseBroadcast(dim3 launchDims, hipStream_t *stream, int opNum, void const* x, Nd4jLong const* xShapeInfo, void const* y, Nd4jLong const* yShapeInfo, void *z, Nd4jLong const* zShapeInfo, void *extraParams, int *dimension, int dimensionLength, Nd4jLong const* tadOnlyShapeInfo, Nd4jLong const* tadOffsets, Nd4jLong const* tadOnlyShapeInfoZ, Nd4jLong const* tadOffsetsZ) {
            DISPATCH_BY_OPNUM_TT(intermediateInverseBroadcast,  PARAMS(launchDims, stream, x, xShapeInfo, y, yShapeInfo, z, zShapeInfo, extraParams, dimension, dimensionLength, tadOnlyShapeInfo, tadOffsets, tadOnlyShapeInfoZ, tadOffsetsZ), OPS_A(BROADCAST_BOOL_OPS))

            DEBUG_KERNEL(stream, opNum);
        }

//////////////////////////////////////////////////////////////////////////
        template<typename X, typename Z>
        template <typename OpType>
        __device__ void BroadcastBool<X,Z>::transformInverseCuda(
                void const* vx, Nd4jLong const* xShapeInfo,
                void const* vy, Nd4jLong const* yShapeInfo,
                void *vz, Nd4jLong const* zShapeInfo,
                void *vextraParams,
                int *dimension, int dimensionLength,
                Nd4jLong const* tadOnlyShapeInfo, Nd4jLong const* tadOffsets, Nd4jLong const* tadOnlyShapeInfoZ, Nd4jLong const* tadOffsetsZ) {

            if (tadOnlyShapeInfoZ == nullptr) {
                tadOnlyShapeInfoZ = tadOnlyShapeInfo;
                tadOffsetsZ = tadOffsets;
            }

            auto x = reinterpret_cast<X const*>(vx);
            auto y = reinterpret_cast<X const*>(vy);
            auto z = reinterpret_cast<Z*>(vz);
            auto extraParams = reinterpret_cast<X*>(vextraParams);

            //decompose in to several sub tads after
            //moving all dimensions (in sorted order)
            //to the back.
            //permuted version of the x shape info for setting up the tad problem
            __shared__ Nd4jLong tadLength;
            __shared__ Nd4jLong tadEWS;
            __shared__ int numTads;
            __shared__ Nd4jLong xEWS;
            __shared__ Nd4jLong zEWS;

            if (threadIdx.x == 0) {
                tadLength = shape::length(tadOnlyShapeInfo);//shape::tadLength(xShapeInfo, dimension, dimensionLength);
                tadEWS = shape::elementWiseStride(tadOnlyShapeInfo);
                numTads = shape::length(yShapeInfo) / tadLength;
                xEWS = shape::elementWiseStride(xShapeInfo);
                zEWS = shape::elementWiseStride(tadOnlyShapeInfoZ);
            }
            __syncthreads();

            for (int r = blockIdx.x; r < numTads; r += gridDim.x) {
                auto rZ = z + tadOffsetsZ[r];
                auto rY = y + tadOffsets[r];

                if(tadEWS > 0 && zEWS > 0 && xEWS > 0 && dimensionLength == 1) {

                    for (int i = threadIdx.x; i < tadLength; i+= blockDim.x)
                        rZ[i * zEWS] = OpType::op(x[i * xEWS], rY[i * tadEWS], extraParams);
                }
                else {
                    // it is expected that x and z tads and y array all have the same length
                    for (Nd4jLong i = threadIdx.x; i < tadLength; i+= blockDim.x) {
                        auto xOffset = shape::getIndexOffset(i, xShapeInfo);
                        auto yOffset = shape::getIndexOffset(i, tadOnlyShapeInfo);
                        auto zOffset = shape::getIndexOffset(i, tadOnlyShapeInfoZ);

                        rZ[zOffset] = OpType::op(x[xOffset], rY[yOffset], extraParams);
                    }
                }
            }
        }

//////////////////////////////////////////////////////////////////////////
        template<typename X, typename Z>
        template <typename OpType>
		__device__ void BroadcastBool<X,Z>::transformCuda(
		                              void const* vx, Nd4jLong const* xShapeInfo,
		                              void const* vy, Nd4jLong const* yShapeInfo,
		                              void *vz, Nd4jLong const* zShapeInfo,
                                      void *vextraParams,
		                              int *dimension, int dimensionLength,
                                      Nd4jLong const* tadOnlyShapeInfo, Nd4jLong const* tadOffsets, Nd4jLong const* tadOnlyShapeInfoZ, Nd4jLong const* tadOffsetsZ) {

            if (tadOnlyShapeInfoZ == nullptr) {
                tadOnlyShapeInfoZ = tadOnlyShapeInfo;
                tadOffsetsZ = tadOffsets;
            }

            auto x = reinterpret_cast<X const*>(vx);
            auto y = reinterpret_cast<X const*>(vy);
            auto z = reinterpret_cast<Z*>(vz);
            auto extraParams = reinterpret_cast<X*>(vextraParams);

            //decompose in to several sub tads after
            //moving all dimensions (in sorted order)
            //to the back.
            //permuted version of the x shape info for setting up the tad problem
            __shared__ Nd4jLong tadLength;
            __shared__ Nd4jLong tadEWS;
            __shared__ int numTads;
            __shared__ Nd4jLong yEWS;
            __shared__ Nd4jLong zEWS;

            if (threadIdx.x == 0) {
   	            tadLength = shape::length(tadOnlyShapeInfo);//shape::tadLength(xShapeInfo, dimension, dimensionLength);
                tadEWS = shape::elementWiseStride(tadOnlyShapeInfo);
                numTads = shape::length(xShapeInfo) / tadLength;
                yEWS = shape::elementWiseStride(yShapeInfo);
                zEWS = shape::elementWiseStride(tadOnlyShapeInfoZ);
            }
            __syncthreads();

            __shared__ Z *rZ;
            __shared__ X const* rX;

		for (int r = blockIdx.x; r < numTads; r += gridDim.x) {

            if (threadIdx.x == 0) {
                rZ = z + tadOffsetsZ[r];
                rX = x + tadOffsets[r];
            }
            __syncthreads();


            if(tadEWS > 0 && zEWS > 0 && yEWS > 0 && dimensionLength == 1) {

                for (int i = threadIdx.x; i < tadLength; i+= blockDim.x)
                    rZ[i * zEWS] = OpType::op(rX[i * tadEWS], y[i * yEWS], extraParams);
            }
            else {
                // it is expected that x and z tads and y array all have the same length
                for (Nd4jLong i = threadIdx.x; i < tadLength; i+= blockDim.x) {
                    auto xOffset = shape::getIndexOffset(i, tadOnlyShapeInfo);
                    auto yOffset = shape::getIndexOffset(i, yShapeInfo);
                    auto zOffset = shape::getIndexOffset(i, tadOnlyShapeInfoZ);

                    rZ[zOffset] = OpType::op(rX[xOffset], y[yOffset], extraParams);
                }
            }
		}
	}

//////////////////////////////////////////////////////////////////////////
template<typename X, typename Z>
template <typename OpType>
__device__ void BroadcastBool<X,Z>::transformCuda(const void *vx, const Nd4jLong *xShapeInfo,
                                                  const void *vy, const Nd4jLong *yShapeInfo,
                                                        void *vz, const Nd4jLong *zShapeInfo,
                                                        void *vextraParams) {

    const X* x = reinterpret_cast<const X*>(vx);
    const X* y = reinterpret_cast<const X*>(vy);
          Z* z = reinterpret_cast<Z*>(vz);

    auto extraParams = reinterpret_cast<X*>(vextraParams);

    __shared__ Nd4jLong zLen;
    __shared__ int rank;
    __shared__ bool xzSameOffsets, yzSameOffsets;

    if (threadIdx.x == 0) {

        zLen  = shape::length(zShapeInfo);
        rank = shape::rank(zShapeInfo);

        xzSameOffsets = shape::haveSameShapeAndStrides(xShapeInfo, zShapeInfo);
        yzSameOffsets = shape::haveSameShapeAndStrides(yShapeInfo, zShapeInfo);
    }
    __syncthreads();


    const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

    int coords[MAX_RANK];

    for (int i = tid; i < zLen; i += blockDim.x * gridDim.x) {

        shape::index2coords(i, zShapeInfo, coords);

        const auto zOffset = shape::getOffset(zShapeInfo, coords);
        const auto xOffset = xzSameOffsets ? zOffset : shape::getOffset(xShapeInfo, coords);
        const auto yOffset = yzSameOffsets ? zOffset : shape::getOffset(yShapeInfo, coords);

        z[zOffset] = OpType::op(x[xOffset], y[yOffset], extraParams);
    }
}


BUILD_DOUBLE_TEMPLATE(template class ND4J_EXPORT BroadcastBool, , LIBND4J_TYPES, BOOL_TYPES);
}
}