#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author raver119@gmail.com
//

#include <system/op_boilerplate.h>
#include <loops/broadcasting_int.h>
#include <loops/legacy_ops.h>
#include <types/types.h>
#include <system/Environment.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <string>
#include <stdexcept>
#include <helpers/StringUtils.h>

using namespace simdOps;

//////////////////////////////////////////////////////////////////////////
template<typename X, typename OpClass>
static __global__ void broadcastIntSimple(
        void const* x,
        Nd4jLong const* xShapeInfo,
        void const* y,
        Nd4jLong const* yShapeInfo,
        void *z,
        Nd4jLong const* zShapeInfo,
        int *dimension,
        int dimensionLength, Nd4jLong const* tadOnlyShapeInfo, Nd4jLong const* tadOffsets, Nd4jLong const* tadOnlyShapeInfoZ, Nd4jLong const* tadOffsetsZ) {

    functions::broadcast::BroadcastInt<X>::template transformCuda<OpClass>(x,xShapeInfo,y,yShapeInfo,z,zShapeInfo,dimension,dimensionLength,tadOnlyShapeInfo,tadOffsets,tadOnlyShapeInfoZ,tadOffsetsZ);
}

//////////////////////////////////////////////////////////////////////////
template<typename X, typename OpClass>
static __global__ void broadcastIntSimple(const void *x, const Nd4jLong const* xShapeInfo,
                                          const void *y, const Nd4jLong const* yShapeInfo,
                                                void *z, const Nd4jLong const* zShapeInfo) {

    functions::broadcast::BroadcastInt<X>::template transformCuda<OpClass>(x, xShapeInfo, y, yShapeInfo, z, zShapeInfo);
}

//////////////////////////////////////////////////////////////////////////
template<typename X, typename OpClass>
static __global__ void broadcastBoolInverseSimple(
        void const* x,
        Nd4jLong const* xShapeInfo,
        void const* y,
        Nd4jLong const* yShapeInfo,
        void *z,
        Nd4jLong const* zShapeInfo,
        int *dimension,
        int dimensionLength, Nd4jLong const* tadOnlyShapeInfo, Nd4jLong const* tadOffsets, Nd4jLong const* tadOnlyShapeInfoZ, Nd4jLong const* tadOffsetsZ) {

    functions::broadcast::BroadcastInt<X>::template transformInverseCuda<OpClass>(x,xShapeInfo,y,yShapeInfo,z,zShapeInfo,dimension,dimensionLength,tadOnlyShapeInfo,tadOffsets,tadOnlyShapeInfoZ,tadOffsetsZ);
}

namespace functions {
namespace broadcast {
//////////////////////////////////////////////////////////////////////////
template<typename X>
template <typename OpClass>
__host__ void BroadcastInt<X>::intermediateBroadcast(dim3 launchDims, hipStream_t *stream, void const* x, Nd4jLong const* xShapeInfo, void const* y, Nd4jLong const* yShapeInfo, void *z, Nd4jLong const* zShapeInfo, int *dimension, int dimensionLength, Nd4jLong const* tadOnlyShapeInfo, Nd4jLong const* tadOffsets, Nd4jLong const* tadOnlyShapeInfoZ, Nd4jLong const* tadOffsetsZ) {
    broadcastIntSimple<X, OpClass><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(x, xShapeInfo, y, yShapeInfo, z, zShapeInfo, dimension, dimensionLength, tadOnlyShapeInfo, tadOffsets, tadOnlyShapeInfoZ, tadOffsetsZ);
}

//////////////////////////////////////////////////////////////////////////
template<typename X>
template <typename OpClass>
__host__ void BroadcastInt<X>::intermediateBroadcast(dim3 launchDims, hipStream_t *stream,
                                                     const void *x, const Nd4jLong *xShapeInfo,
                                                     const void *y, const Nd4jLong *yShapeInfo,
                                                           void *z, const Nd4jLong *zShapeInfo) {

    broadcastIntSimple<X, OpClass><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(x, xShapeInfo, y, yShapeInfo, z, zShapeInfo);
}

//////////////////////////////////////////////////////////////////////////
template<typename X>
__host__ void BroadcastInt<X>::execBroadcast(dim3 launchDims, hipStream_t *stream, int opNum, void const* x, Nd4jLong const* xShapeInfo, void const* y, Nd4jLong const* yShapeInfo, void *z, Nd4jLong const* zShapeInfo, int *dimension, int dimensionLength, Nd4jLong const* tadOnlyShapeInfo, Nd4jLong const* tadOffsets, Nd4jLong const* tadOnlyShapeInfoZ, Nd4jLong const* tadOffsetsZ) {
    DISPATCH_BY_OPNUM_T(intermediateBroadcast,  PARAMS(launchDims, stream, x, xShapeInfo, y, yShapeInfo, z, zShapeInfo, dimension, dimensionLength, tadOnlyShapeInfo, tadOffsets, tadOnlyShapeInfoZ, tadOffsetsZ), OPS_A(BROADCAST_INT_OPS))
}

//////////////////////////////////////////////////////////////////////////
template<typename X>
__host__ void BroadcastInt<X>::execBroadcast(dim3 launchDims, hipStream_t *stream, const int opNum,
                                             const void *x, const Nd4jLong const* xShapeInfo,
                                             const void *y, const Nd4jLong const* yShapeInfo,
                                                   void *z, const Nd4jLong const* zShapeInfo) {

    DISPATCH_BY_OPNUM_T(intermediateBroadcast, PARAMS(launchDims, stream, x, xShapeInfo, y, yShapeInfo, z, zShapeInfo), OPS_A(BROADCAST_INT_OPS))
}

//////////////////////////////////////////////////////////////////////////
        template<typename X>
        template <typename OpClass>
        __host__ void BroadcastInt<X>::intermediateInverseBroadcast(dim3 launchDims, hipStream_t *stream, void const* x, Nd4jLong const* xShapeInfo, void const* y, Nd4jLong const* yShapeInfo, void *z, Nd4jLong const* zShapeInfo, int *dimension, int dimensionLength, Nd4jLong const* tadOnlyShapeInfo, Nd4jLong const* tadOffsets, Nd4jLong const* tadOnlyShapeInfoZ, Nd4jLong const* tadOffsetsZ) {
            broadcastBoolInverseSimple<X, OpClass><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(x, xShapeInfo, y, yShapeInfo, z, zShapeInfo, dimension, dimensionLength, tadOnlyShapeInfo, tadOffsets, tadOnlyShapeInfoZ, tadOffsetsZ);
        }

//////////////////////////////////////////////////////////////////////////
        template<typename X>
        __host__ void BroadcastInt<X>::execInverseBroadcast(dim3 launchDims, hipStream_t *stream, int opNum, void const* x, Nd4jLong const* xShapeInfo, void const* y, Nd4jLong const* yShapeInfo, void *z, Nd4jLong const* zShapeInfo, int *dimension, int dimensionLength, Nd4jLong const* tadOnlyShapeInfo, Nd4jLong const* tadOffsets, Nd4jLong const* tadOnlyShapeInfoZ, Nd4jLong const* tadOffsetsZ) {
            DISPATCH_BY_OPNUM_T(intermediateInverseBroadcast,  PARAMS(launchDims, stream, x, xShapeInfo, y, yShapeInfo, z, zShapeInfo, dimension, dimensionLength, tadOnlyShapeInfo, tadOffsets, tadOnlyShapeInfoZ, tadOffsetsZ), OPS_A(BROADCAST_INT_OPS))
        }

//////////////////////////////////////////////////////////////////////////
        template<typename X>
        template <typename OpType>
        __device__ void BroadcastInt<X>::transformInverseCuda(
                void const* vx, Nd4jLong const* xShapeInfo,
                void const* vy, Nd4jLong const* yShapeInfo,
                void *vz, Nd4jLong const* zShapeInfo,
                int *dimension, int dimensionLength,
                Nd4jLong const* tadOnlyShapeInfo, Nd4jLong const* tadOffsets, Nd4jLong const* tadOnlyShapeInfoZ, Nd4jLong const* tadOffsetsZ) {

            if (tadOnlyShapeInfoZ == nullptr) {
                tadOnlyShapeInfoZ = tadOnlyShapeInfo;
                tadOffsetsZ = tadOffsets;
            }

            auto x = reinterpret_cast<X const*>(vx);
            auto y = reinterpret_cast<X const*>(vy);
            auto z = reinterpret_cast<X*>(vz);

            //decompose in to several sub tads after
            //moving all dimensions (in sorted order)
            //to the back.
            //permuted version of the x shape info for setting up the tad problem
            __shared__ Nd4jLong tadLength;
            __shared__ Nd4jLong tadEWS;
            __shared__ int numTads;
            __shared__ Nd4jLong xEWS;
            __shared__ Nd4jLong zEWS;

            if (threadIdx.x == 0) {
                tadLength = shape::length(tadOnlyShapeInfo);//shape::tadLength(xShapeInfo, dimension, dimensionLength);
                tadEWS = shape::elementWiseStride(tadOnlyShapeInfo);
                numTads = shape::length(yShapeInfo) / tadLength;
                xEWS = shape::elementWiseStride(xShapeInfo);
                zEWS = shape::elementWiseStride(tadOnlyShapeInfoZ);
            }
            __syncthreads();

            for (int r = blockIdx.x; r < numTads; r += gridDim.x) {
                auto rZ = z + tadOffsetsZ[r];
                auto rY = y + tadOffsets[r];

                if(tadEWS > 0 && zEWS > 0 && xEWS > 0 && dimensionLength == 1) {

                    for (int i = threadIdx.x; i < tadLength; i+= blockDim.x)
                        rZ[i * zEWS] = OpType::op(x[i * xEWS], rY[i * tadEWS]);
                }
                else {
                    // it is expected that x and z tads and y array all have the same length
                    for (Nd4jLong i = threadIdx.x; i < tadLength; i+= blockDim.x) {
                        auto xOffset = shape::getIndexOffset(i, xShapeInfo);
                        auto yOffset = shape::getIndexOffset(i, tadOnlyShapeInfo);
                        auto zOffset = shape::getIndexOffset(i, tadOnlyShapeInfoZ);

                        rZ[zOffset] = OpType::op(x[xOffset], rY[yOffset]);
                    }
                }
            }
        }

//////////////////////////////////////////////////////////////////////////
        template<typename X>
        template <typename OpType>
		__device__ void BroadcastInt<X>::transformCuda(
		                              void const* vx, Nd4jLong const* xShapeInfo,
		                              void const* vy, Nd4jLong const* yShapeInfo,
		                              void *vz, Nd4jLong const* zShapeInfo,
		                              int *dimension, int dimensionLength,
                                      Nd4jLong const* tadOnlyShapeInfo, Nd4jLong const* tadOffsets, Nd4jLong const* tadOnlyShapeInfoZ, Nd4jLong const* tadOffsetsZ) {

            if (tadOnlyShapeInfoZ == nullptr) {
                tadOnlyShapeInfoZ = tadOnlyShapeInfo;
                tadOffsetsZ = tadOffsets;
            }

            auto x = reinterpret_cast<X const*>(vx);
            auto y = reinterpret_cast<X const*>(vy);
            auto z = reinterpret_cast<X*>(vz);

            //decompose in to several sub tads after
            //moving all dimensions (in sorted order)
            //to the back.
            //permuted version of the x shape info for setting up the tad problem
            __shared__ Nd4jLong tadLength;
            __shared__ Nd4jLong tadEWS;
            __shared__ int numTads;
            __shared__ Nd4jLong yEWS;
            __shared__ Nd4jLong zEWS;

            if (threadIdx.x == 0) {
   	            tadLength = shape::length(tadOnlyShapeInfo);//shape::tadLength(xShapeInfo, dimension, dimensionLength);
                tadEWS = shape::elementWiseStride(tadOnlyShapeInfo);
                numTads = shape::length(xShapeInfo) / tadLength;
                yEWS = shape::elementWiseStride(yShapeInfo);
                zEWS = shape::elementWiseStride(tadOnlyShapeInfoZ);
            }
            __syncthreads();

            __shared__ X *rZ;
            __shared__ X const* rX;

		for (int r = blockIdx.x; r < numTads; r += gridDim.x) {

            if (threadIdx.x == 0) {
                rZ = z + tadOffsetsZ[r];
                rX = x + tadOffsets[r];
            }
            __syncthreads();


            if(tadEWS > 0 && zEWS > 0 && yEWS > 0 && dimensionLength == 1) {

                for (int i = threadIdx.x; i < tadLength; i+= blockDim.x)
                    rZ[i * zEWS] = OpType::op(rX[i * tadEWS], y[i * yEWS]);
            }
            else {
                // it is expected that x and z tads and y array all have the same length
                for (Nd4jLong i = threadIdx.x; i < tadLength; i+= blockDim.x) {
                    auto xOffset = shape::getIndexOffset(i, tadOnlyShapeInfo);
                    auto yOffset = shape::getIndexOffset(i, yShapeInfo);
                    auto zOffset = shape::getIndexOffset(i, tadOnlyShapeInfoZ);

                    rZ[zOffset] = OpType::op(rX[xOffset], y[yOffset]);
                }
            }
		}
	}

//////////////////////////////////////////////////////////////////////////
template<typename X>
template <typename OpType>
__device__ void BroadcastInt<X>::transformCuda(const void *vx, const Nd4jLong const* xShapeInfo,
                                               const void *vy, const Nd4jLong const* yShapeInfo,
                                                     void *vz, const Nd4jLong const* zShapeInfo) {

    const X* x = reinterpret_cast<const X*>(vx);
    const X* y = reinterpret_cast<const X*>(vy);
          X* z = reinterpret_cast<X*>(vz);

    __shared__ Nd4jLong zLen;
    __shared__ int rank;
    __shared__ bool xzSameOffsets, yzSameOffsets;

    if (threadIdx.x == 0) {

        zLen  = shape::length(zShapeInfo);
        rank = shape::rank(zShapeInfo);

        xzSameOffsets = shape::haveSameShapeAndStrides(xShapeInfo, zShapeInfo);
        yzSameOffsets = shape::haveSameShapeAndStrides(yShapeInfo, zShapeInfo);
    }
    __syncthreads();


    const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

    int coords[MAX_RANK];

    for (int i = tid; i < zLen; i += blockDim.x * gridDim.x) {

        shape::index2coords(i, zShapeInfo, coords);

        const auto zOffset = shape::getOffset(zShapeInfo, coords);
        const auto xOffset = xzSameOffsets ? zOffset : shape::getOffset(xShapeInfo, coords);
        const auto yOffset = yzSameOffsets ? zOffset : shape::getOffset(yShapeInfo, coords);

        z[zOffset] = OpType::op(x[xOffset], y[yOffset]);
    }
}


BUILD_SINGLE_TEMPLATE(template class ND4J_EXPORT BroadcastInt, , INTEGER_TYPES);
}
}