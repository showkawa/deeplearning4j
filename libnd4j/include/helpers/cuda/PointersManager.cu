#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma (iuriish@yahoo.com), created on 06.02.2019
// @author raver119@gmail.com
//

#include <helpers/PointersManager.h>
#include <exceptions/cuda_exception.h>
#include <helpers/StringUtils.h>
#include <helpers/logger.h>
#include <memory/Workspace.h>

namespace sd {

//////////////////////////////////////////////////////////////////////////
PointersManager::PointersManager(const sd::LaunchContext* context, const std::string& funcName)  {
        _context  = const_cast<sd::LaunchContext*>(context);
        _funcName = funcName;
}

//////////////////////////////////////////////////////////////////////////
void* PointersManager::replicatePointer(const void* src, const size_t numberOfBytes) {

	void* dst = nullptr;
	if (_context->getWorkspace() == nullptr) {
        hipError_t cudaResult = hipMalloc(reinterpret_cast<void **>(&dst), numberOfBytes);
        if (cudaResult != 0)
            throw cuda_exception::build(_funcName + ": cannot allocate global memory on device!", cudaResult);
    } else {
	    dst = _context->getWorkspace()->allocateBytes(sd::memory::MemoryType::DEVICE, numberOfBytes);
	}

    if (_context != nullptr)
        hipMemcpyAsync(dst, src, numberOfBytes, hipMemcpyHostToDevice, *_context->getCudaStream());
    else
        hipMemcpy(dst, src, numberOfBytes, hipMemcpyHostToDevice);

    _pOnGlobMem.emplace_back(dst);

    return dst;
}

//////////////////////////////////////////////////////////////////////////
void PointersManager::synchronize() const {
    if (_context != nullptr) {
        hipError_t cudaResult = hipStreamSynchronize(*_context->getCudaStream());
        if (cudaResult != 0)
            throw cuda_exception::build(_funcName + ": cuda stream synchronization failed !", cudaResult);
    } else {
        nd4j_printf("<%s> syncStream isn't possible: no stream set!", _funcName.c_str());
    }
}

//////////////////////////////////////////////////////////////////////////
PointersManager::~PointersManager() {

    for (auto& p :_pOnGlobMem)
        hipFree(p);
}


////////////////////////////////////////////////////////////////////////
template <typename T>
static __global__ void printDevContentOnDev_(const void* pDev, const Nd4jLong len, const int tid) {

    PointersManager::printDevContentOnDev<T>(pDev, len, tid);
}

////////////////////////////////////////////////////////////////////////
template<typename T>
void PointersManager::printDevContentOnDevFromHost(const void* pDev, const Nd4jLong len, const int tid) {
    printDevContentOnDev_<T><<<512, 512, 1024, *sd::LaunchContext ::defaultContext()->getCudaStream()>>>(pDev, len, tid);
    auto res = hipStreamSynchronize(*sd::LaunchContext ::defaultContext()->getCudaStream());
    if (res != 0)
        throw std::runtime_error("PointersManager::printDevContentOnDevFromHost: hipStreamSynchronize failed!");
}
template void PointersManager::printDevContentOnDevFromHost<Nd4jLong>(const void* pDev, const Nd4jLong len, const int tid);
template void PointersManager::printDevContentOnDevFromHost<int>(const void* pDev, const Nd4jLong len, const int tid);
template void PointersManager::printDevContentOnDevFromHost<float>(const void* pDev, const Nd4jLong len, const int tid);
template void PointersManager::printDevContentOnDevFromHost<double>(const void* pDev, const Nd4jLong len, const int tid);

//BUILD_SINGLE_TEMPLATE(template void PointersManager::printDevContentOnDevFromHost, (void* pDev, Nd4jLong len, int tid), LIBND4J_TYPES);

////////////////////////////////////////////////////////////////////////
template<typename T>
void PointersManager::printDevContentOnHost(const void* pDev, const Nd4jLong len) const {
    printf("host print out\n");
    void* pHost = operator new(sizeof(T) * len);

    hipMemcpyAsync(pHost, pDev, sizeof(T) * len, hipMemcpyDeviceToHost, *_context->getCudaStream());
    hipError_t cudaResult = hipStreamSynchronize(*_context->getCudaStream());
    if(cudaResult != 0)
        throw std::runtime_error("PointersManager::printCudaHost: hipStreamSynchronize failed!");

    for(Nd4jLong i = 0; i < len; ++i)
        printf("%f, ", (double)reinterpret_cast<T*>(pHost)[i]);
    printf("\n");

    operator delete(pHost);
}


template void PointersManager::printDevContentOnHost<Nd4jLong>(const void* pDev, const Nd4jLong len) const;
template void PointersManager::printDevContentOnHost<int>(const void* pDev, const Nd4jLong len) const;
template void PointersManager::printDevContentOnHost<float>(const void* pDev, const Nd4jLong len) const;
template void PointersManager::printDevContentOnHost<double>(const void* pDev, const Nd4jLong len) const;


}
