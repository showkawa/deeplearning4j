#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 * Copyright (c) 2019 Konduit K.K.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
// @author Yurii Shyrma (iuriish@yahoo.com)
//
#include <exceptions/cuda_exception.h>
#include <hipblas.h>
#include "../MmulHelper.h"
#include <ops/specials_cuda.h>
#include <helpers/ShapeUtils.h>
#include <helpers/PointersManager.h>
#include <numeric>

namespace sd {

//////////////////////////////////////////////////////////////////////////////
// MXK x KxN = MxN              -> actual sequence of axes doesn't matter
template <typename T1, typename T2, typename T3>
static __global__ void usualCudaGemm(const void* vA, const Nd4jLong* aShapeInfo, const void* vB, const Nd4jLong* bShapeInfo, void* vC, const Nd4jLong* cShapeInfo,
                                     const int aMaxis, const int aKaxis, const int bKaxis, const int bNaxis, const int cMaxis, const int cNaxis,
                                     const double alpha, const double beta) {

    const T1* A = reinterpret_cast<const T1*>(vA);
    const T2* B = reinterpret_cast<const T2*>(vB);
          T3* C = reinterpret_cast<      T3*>(vC);

    __shared__ int K, *coords;
    __shared__ bool betaPresent;
    __shared__ Nd4jLong cLen, totalThreads;
    __shared__ T3 alphaZ, betaZ;

    if (threadIdx.x == 0) {

        extern __shared__ unsigned char shmem[];
        coords = reinterpret_cast<int*>(shmem);
        cLen = shape::length(cShapeInfo);

        K = shape::shapeOf(const_cast<Nd4jLong*>(aShapeInfo))[aKaxis];

        betaPresent = beta;

        totalThreads = gridDim.x * blockDim.x;

        alphaZ = alpha;
        betaZ  = beta;
    }
    __syncthreads();

    auto aCoords = coords + threadIdx.x * 6;    // 6 = (aRank + bRank + cRank)
    auto bCoords = aCoords + 2;
    auto cCoords = bCoords + 2;

    const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (Nd4jLong i = tid; i < cLen; i += totalThreads) {

        // evaluate C coordinates
        shape::index2coords(i, cShapeInfo, cCoords);

        // evaluate A coordinates
        aCoords[aMaxis] = cCoords[cMaxis];
        aCoords[aKaxis] = 0;

        // evaluate B coordinates
        bCoords[bKaxis] = 0;
        bCoords[bNaxis] = cCoords[cNaxis];

        auto aOffset = shape::getOffset(aShapeInfo, aCoords);
        auto bOffset = shape::getOffset(bShapeInfo, bCoords);

        T3 val = A[aOffset] * B[bOffset];                       // first iteration

        for (uint j = 1; j < K; ++j) {                          // rest iterations
            aOffset += shape::stride(aShapeInfo)[aKaxis];
            bOffset += shape::stride(bShapeInfo)[bKaxis];
            val = val + A[aOffset] * B[bOffset];
        }

        auto cOffset = shape::getOffset(cShapeInfo, cCoords);

        if(betaPresent)
            C[cOffset] = alphaZ * val + betaZ * C[cOffset];
        else
            C[cOffset] = alphaZ * val;
    }
}

////////////////////////////////////////////////////////////////////////
template <typename T1, typename T2, typename T3>
__host__ static void usualGemm(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, hipStream_t *stream, const void* vA, const Nd4jLong* aShapeInfo, const void* vB, const Nd4jLong* bShapeInfo, void* vC, const Nd4jLong* cShapeInfo, const int aMaxis, const int aKaxis, const int bKaxis, const int bNaxis, const int cMaxis, const int cNaxis, const double alpha, const double beta) {

    usualCudaGemm<T1,T2,T3><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(vA, aShapeInfo, vB, bShapeInfo, vC, cShapeInfo, aMaxis, aKaxis, bKaxis, bNaxis, cMaxis, cNaxis, alpha, beta);
}

////////////////////////////////////////////////////////////////////////
// MXN x N = M  -> actual sequence of {M,N} axes doesn't matter
template <typename T1, typename T2, typename T3>
static __global__ void usualCudaGemv(const void* vA, const Nd4jLong* aShapeInfo, const void* vX, const Nd4jLong* xShapeInfo, void* vY, const Nd4jLong* yShapeInfo,
                                     const int incx, const int incy, const int aMaxis, const double alpha, const double beta) {

    const T1* A = reinterpret_cast<const T1*>(vA);
    const T2* X = reinterpret_cast<const T2*>(vX);
          T3* Y = reinterpret_cast<      T3*>(vY);

    __shared__ int M, N;
    __shared__ bool betaPresent;
    __shared__ Nd4jLong cLen, totalThreads, aNstride, aMstride;
    __shared__ T3 alphaZ, betaZ;

    if (threadIdx.x == 0) {

        N = shape::length(xShapeInfo);
        M = shape::length(yShapeInfo);

        aMstride = shape::stride(aShapeInfo)[aMaxis];
        aNstride = shape::stride(aShapeInfo)[aMaxis == 0 ? 1 : 0];

        totalThreads = gridDim.x * blockDim.x;

        betaPresent = beta;

        alphaZ = alpha;
        betaZ  = beta;
    }
    __syncthreads();


    const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (Nd4jLong i = tid; i < M; i += totalThreads) {

        // evaluate offsets
        auto aOffset = i * aMstride;
        auto xOffset = 0;

        T3 val = A[aOffset] * X[xOffset];                       // first iteration

        for (uint j = 1; j < N; ++j) {                          // rest iterations
            aOffset += aNstride;
            xOffset += incx;
            val = val + A[aOffset] * X[xOffset];
        }

        auto yOffset = i * incy;

        if(betaPresent)
            Y[yOffset] = alphaZ * val + betaZ * Y[yOffset];
        else
            Y[yOffset] = alphaZ * val;
    }
}

////////////////////////////////////////////////////////////////////////
template <typename T1, typename T2, typename T3>
__host__ static void usualGemv(const int blocksPerGrid, const int threadsPerBlock, hipStream_t *stream, const void* vA, const Nd4jLong* aShapeInfo, const void* vX, const Nd4jLong* xShapeInfo, void* vY, const Nd4jLong* yShapeInfo, const int incx, const int incy, const int aMaxis, const double alpha, const double beta) {

    usualCudaGemv<T1,T2,T3><<<blocksPerGrid, threadsPerBlock, 512, *stream>>>(vA, aShapeInfo, vX, xShapeInfo, vY, yShapeInfo, incx, incy, aMaxis, alpha, beta);
}


//////////////////////////////////////////////////////////////////////////////
template <typename T1, typename T2, typename T3>
static __global__ void usualCudaDot(const Nd4jLong length, const double alpha, const void* vX, const Nd4jLong incx, const void* vY, const Nd4jLong incy, const double beta, void* vZ) {

    T1* X = reinterpret_cast<T1*>(const_cast<void*>(vX));
    T2* Y = reinterpret_cast<T2*>(const_cast<void*>(vY));
    T3* Z = reinterpret_cast<T3*>(vZ);

    extern __shared__ unsigned char shmem[];
    auto pairwiseMul = reinterpret_cast<T3*>(shmem);

    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < length)
        pairwiseMul[tid] = X[tid * incx] * Y[tid * incy];

    __syncthreads();

    if(tid == 0) {
        T3 sum = 0;
        for(Nd4jLong i = 0; i < length; ++i)
            sum = sum + pairwiseMul[i];

        if(beta)
            *Z = (T3)alpha * sum + (T3)beta * *Z;
        else
            *Z = (T3)alpha * sum;
    }
}

////////////////////////////////////////////////////////////////////////
template <typename T1, typename T2, typename T3>
__host__ static void usualDot(const dim3 &blocksPerGrid, const dim3 &threadsPerBlock, hipStream_t *stream, const Nd4jLong length, const double alpha, const void* vX, const Nd4jLong incx, const void* vY, const Nd4jLong incy, const double beta, void* vZ) {

    usualCudaDot<T1,T2,T3><<<blocksPerGrid, threadsPerBlock, length*sizeof(T3) + 128, *stream>>>(length, alpha, vX, incx, vY, incy, beta, vZ);
}

//////////////////////////////////////////////////////////////////////////////
// MXK x KxN = MxN
NDArray* MmulHelper::mmulMxM(const NDArray* A, const NDArray* B, NDArray* C, double alpha, double beta, const char outOrder) {

    if(A->rankOf() != 2)
        throw std::runtime_error("MmulHelper::mmulMxM cuda: rank of A array is not equal 2 !");
    if(B->rankOf() != 2)
        throw std::runtime_error("MmulHelper::mmulMxM cuda: rank of B array is not equal 2 !");

    const auto M = A->sizeAt(0);
    const auto K = A->sizeAt(1);
    const auto N = B->sizeAt(1);

    if(C != nullptr && C->rankOf() != 2)
        throw std::runtime_error("MmulHelper::mmulMxM cuda: rank of C array is not equal 2 !");
    if(B->sizeAt(0) != K)
        throw std::runtime_error("MmulHelper::mmulMxM cuda: B array has wrong number of rows !");
    if(C != nullptr && C->sizeAt(0) != M)
        throw std::runtime_error("MmulHelper::mmulMxM cuda: C array has wrong number of rows !");
    if(C != nullptr && C->sizeAt(1) != N)
        throw std::runtime_error("MmulHelper::mmulMxM cuda: C array has wrong number of columns !");

    if(C == nullptr)
        C = new NDArray(outOrder, {M,N}, DataTypeUtils::pickPairwiseResultType(A->dataType(), B->dataType()), A->getContext());

    if (C->isEmpty())
        return C;

    const int major = Environment::getInstance().capabilities()[AffinityManager::currentDeviceId()].first();

    const auto aType = A->dataType();
    const auto bType = B->dataType();
    const auto cType = C->dataType();

    const bool AB(aType == bType), AC(aType == cType), ABC(AB && AC);

    const bool typeDouble    = ABC && aType == DataType::DOUBLE;
    const bool typeFloat     = ABC && aType == DataType::FLOAT32;
    const bool typeHalf      = ABC && aType == DataType::HALF && major >= 6;
    const bool typeIntFloat  = AB  && aType == DataType::INT8 && cType == DataType::FLOAT32 && major >= 6;
    const bool typeHalfFloat = AB  && aType == DataType::HALF && cType == DataType::FLOAT32  && major >= 6;

    std::lock_guard<std::mutex> lock(*LaunchContext::deviceMutex());

    auto handle = reinterpret_cast<hipblasHandle_t *>(A->getContext()->getCublasHandle());
    auto stream = A->getContext()->getCudaStream();

    auto status = hipblasSetStream(*handle, *stream);
    if (status != HIPBLAS_STATUS_SUCCESS)
        throw cuda_exception::build("MmulHelper::mmulMxM cuda failed !", status);

    if(!typeDouble && !typeFloat && !typeHalf && !typeIntFloat && !typeHalfFloat) {

        const int threadsPerBlock = MAX_NUM_THREADS / 2;
        const int blocksPerGrid = (C->lengthOf() + threadsPerBlock - 1) / threadsPerBlock;
        const int sharedMem = threadsPerBlock * sizeof(int) * 6 + 128;                             // 6 = aRank + bRank + cRank

        NDArray::prepareSpecialUse({C}, {A, B});
        // BUILD_TRIPLE_SELECTOR(aType, bType, cType, usualGemm, (blocksPerGrid, threadsPerBlock, sharedMem, stream, A->specialBuffer(), A->specialShapeInfo(), B->specialBuffer(), B->specialShapeInfo(), C->specialBuffer(), C->special(), 0, 1, 0, 1, 0, 1, alpha, beta), NUMERIC_TYPES, NUMERIC_TYPES, FLOAT_TYPES);
        BUILD_SINGLE_SELECTOR_THRICE(aType, usualGemm, (blocksPerGrid, threadsPerBlock, sharedMem, stream, A->specialBuffer(), A->specialShapeInfo(), B->specialBuffer(), B->specialShapeInfo(), C->specialBuffer(), C->specialShapeInfo(), 0, 1, 0, 1, 0, 1, alpha, beta), NUMERIC_TYPES)
        NDArray::registerSpecialUse({C}, {A, B});

        auto cudaResult = hipStreamSynchronize(*stream);
        if (cudaResult != 0)
            throw cuda_exception::build("MmulHelper::mmulMxM cuda failed !", cudaResult);
    }
    else {

        std::vector<NDArray*> toDelete;

        NDArray *pA(const_cast<NDArray*>(A)), *pB(const_cast<NDArray*>(B)), *pC(const_cast<NDArray*>(C));

        bool aMcont = M == 1 || A->strideAt(0) == 1;
        bool aKcont = K == 1 || A->strideAt(1) == 1;
        bool bKcont = K == 1 || B->strideAt(0) == 1;
        bool bNcont = N == 1 || B->strideAt(1) == 1;
        bool cMcont = M == 1 || C->strideAt(0) == 1;
        bool cNcont = N == 1 || C->strideAt(1) == 1;

        if(!aMcont && !aKcont) {
            pA = new NDArray(A->dup('f'));
            toDelete.push_back(pA);
            aMcont = true;
        }
        if(!bKcont && !bNcont) {
            pB = new NDArray(B->dup('f'));
            toDelete.push_back(pB);
            bKcont = true;
        }
        if(!cMcont) {
            pC = new NDArray(C->dup('f'));
            toDelete.push_back(pC);
            cMcont = true;
        }

        const bool transA = !aMcont;
        const bool transB = !bKcont;

        const int lda = (aMcont && aKcont) ? M : transA ? pA->strideAt(0) : pA->strideAt(1);
        const int ldb = (bKcont && bNcont) ? K : transB ? pB->strideAt(0) : pB->strideAt(1);
        const int ldc = (cMcont && cNcont) ? M : pC->strideAt(1);

        const hipblasOperation_t transAblas = transA ? HIPBLAS_OP_T : HIPBLAS_OP_N;
        const hipblasOperation_t transBblas = transB ? HIPBLAS_OP_T : HIPBLAS_OP_N;

        NDArray::prepareSpecialUse({pC}, {pA, pB});

        // choose appropriate cuda gemm api depending on data types
        if(typeDouble) {
            status = hipblasDgemm(*handle, transAblas, transBblas, M, N, K, &alpha, (double*)pA->specialBuffer(), lda, (double*)pB->specialBuffer(), ldb, &beta, (double*)pC->specialBuffer(), ldc);
        }
        else if(typeFloat) {
            float alphaF(alpha), betaF(beta);
            status = hipblasSgemm(*handle, transAblas, transBblas, M, N, K, &alphaF, (float*)pA->specialBuffer(), lda, (float*)pB->specialBuffer(), ldb, &betaF, (float*)pC->specialBuffer(), ldc);
        }
        else if(typeHalf) {
            float16 alphaH(alpha), betaH(beta);
            status = hipblasHgemm(*handle, transAblas, transBblas, M, N, K, &alphaH.data, (__half*)pA->specialBuffer(), lda, (__half*)pB->specialBuffer(), ldb, &betaH.data, (__half*)pC->specialBuffer(), ldc);
        }
        else if(typeIntFloat) {
               float alphaF(alpha), betaF(beta);
               status = cublasSgemmEx(*handle, transAblas, transBblas, M, N, K, &alphaF, pA->specialBuffer(), HIP_R_8I, lda, pB->specialBuffer(), HIP_R_8I, ldb, &betaF, pC->specialBuffer(), HIP_R_32F, ldc);
        }
        else if(typeHalfFloat) {
            float alphaF(alpha), betaF(beta);
            status = cublasSgemmEx(*handle, transAblas, transBblas, M, N, K, &alphaF, pA->specialBuffer(), HIP_R_16F, lda, pB->specialBuffer(), HIP_R_16F, ldb, &betaF, pC->specialBuffer(), HIP_R_32F, ldc);
        }

        if (status != HIPBLAS_STATUS_SUCCESS)
            throw cuda_exception::build("MmulHelper::mmulMxM cuda failed !", status);

        NDArray::registerSpecialUse({pC}, {pA, pB});

        auto cudaResult = hipStreamSynchronize(*stream);
        if (cudaResult != 0)
            throw cuda_exception::build("MmulHelper::mmulMxM cuda failed !", cudaResult);

        if(C != pC)
            C->assign(pC);

        for(int i = toDelete.size() - 1; i >= 0; --i)
            delete toDelete[i];
    }

    return C;
}

////////////////////////////////////////////////////////////////////////////
// MXN x N = M
NDArray* MmulHelper::mmulMxV(const NDArray* A, const NDArray* X, sd::NDArray* Y, const double alpha, const double beta, const char outOrder) {

    int xLenDim, yLenDim(0);

    if(A->rankOf() != 2)
        throw std::runtime_error("MmulHelper::mmulMxV cuda: rank of A array is not equal 2 !");
    if(!shape::isCommonVector(X->shapeInfo(), xLenDim))
        throw std::runtime_error("MmulHelper::mmulMxV cuda: X array must be vector !");

    const auto M = A->sizeAt(0);
    const auto N = A->sizeAt(1);

    if(Y != nullptr && !shape::isCommonVector(Y->shapeInfo(), yLenDim))
        throw std::runtime_error("MmulHelper::mmulMxV cuda: Y array must be vector !");
    if(X->lengthOf() != N)
        throw std::runtime_error("MmulHelper::mmulMxV cuda: X vector has wrong length !");
    if(Y != nullptr && Y->lengthOf() != M)
        throw std::runtime_error("MmulHelper::mmulMxV cuda: Y array has wrong length !");

    if(Y == nullptr)
        Y = new NDArray(outOrder, {M}, DataTypeUtils::pickPairwiseResultType(A->dataType(), X->dataType()), A->getContext());

    if (Y->isEmpty())
        return Y;

    const int incx = X->strideAt(xLenDim);
    const int incy = Y->strideAt(yLenDim);

    const auto aType = A->dataType();
    const auto xType = X->dataType();
    const auto yType = Y->dataType();

    const bool AX(aType == xType), AY(aType == yType), AXY(AX && AY);

    const bool typeDouble = AXY && aType == DataType::DOUBLE;
    const bool typeFloat  = AXY && aType == DataType::FLOAT32;

    std::lock_guard<std::mutex> lock(*LaunchContext::deviceMutex());

    auto handle = reinterpret_cast<hipblasHandle_t *>(A->getContext()->getCublasHandle());
    auto stream = A->getContext()->getCudaStream();

    auto status = hipblasSetStream(*handle, *stream);
    if (status != HIPBLAS_STATUS_SUCCESS)
        throw cuda_exception::build("MmulHelper::mmulMxV cuda failed !", status);

    if(!typeDouble && !typeFloat) {

        const int threadsPerBlock = MAX_NUM_THREADS;
        const int blocksPerGrid = (M + threadsPerBlock - 1) / threadsPerBlock;

        NDArray::prepareSpecialUse({Y}, {A, X});
        // BUILD_TRIPLE_SELECTOR(aType, xType, yType, usualGemv, (blocksPerGrid, threadsPerBlock, stream, A->specialBuffer(), A->specialShapeInfo(), X->specialBuffer(), X->specialShapeInfo(), Y->specialBuffer(), Y->special(), incx, incy, 0, alpha, beta), NUMERIC_TYPES, NUMERIC_TYPES, FLOAT_TYPES);
        BUILD_SINGLE_SELECTOR_THRICE(xType, usualGemv, (blocksPerGrid, threadsPerBlock, stream, A->specialBuffer(), A->specialShapeInfo(), X->specialBuffer(), X->specialShapeInfo(), Y->specialBuffer(), Y->specialShapeInfo(), incx, incy, 0, alpha, beta), NUMERIC_TYPES)
        NDArray::registerSpecialUse({Y}, {A, X});

        auto cudaResult = hipStreamSynchronize(*stream);
        if (cudaResult != 0)
            throw cuda_exception::build("MmulHelper::mmulMxV cuda failed !", cudaResult);

    }
    else {

        NDArray *pA(const_cast<NDArray*>(A));

        bool aMcont = M == 1 || A->strideAt(0) == 1;
        bool aNcont = N == 1 || A->strideAt(1) == 1;

        if(!aMcont && !aNcont) {
            pA = new NDArray(A->dup('f'));
            aMcont = true;
        }

        const bool transA = !aMcont;

        const int lda = (aMcont && aNcont) ? M : transA ? pA->strideAt(0) : pA->strideAt(1);

        const hipblasOperation_t transAblas = transA ? HIPBLAS_OP_T : HIPBLAS_OP_N;

        NDArray::prepareSpecialUse({Y}, {pA, X});

        // choose appropriate cuda gemm api depending on data types
        if(typeDouble) {
            status = hipblasDgemv(*handle, transAblas, transA ? N : M, transA ? M : N, &alpha, (double*)pA->specialBuffer(), lda, (double*)X->specialBuffer(), incx, &beta, (double*)Y->specialBuffer(), incy);
        }
        else if(typeFloat) {
            float alphaF(alpha), betaF(beta);
            status = hipblasSgemv(*handle, transAblas, transA ? N : M, transA ? M : N, &alphaF, (float*)pA->specialBuffer(), lda, (float*)X->specialBuffer(), incx, &betaF, (float*)Y->specialBuffer(), incy);
        }

        if (status != HIPBLAS_STATUS_SUCCESS)
            throw cuda_exception::build("MmulHelper::mmulMxV cuda failed !", status);

        auto cudaResult = hipStreamSynchronize(*stream);
        if (cudaResult != 0)
            throw cuda_exception::build("MmulHelper::mmulMxV cuda failed !", cudaResult);

        NDArray::registerSpecialUse({Y}, {pA, X});

        if(pA != A)
            delete pA;
    }

    return Y;
}

////////////////////////////////////////////////////////////////////////////
// (X * Y) = Z[0]
NDArray* MmulHelper::dot(const NDArray* X, const NDArray* Y, sd::NDArray* Z, const double alpha, const double beta) {

    int xLenDim(0), yLenDim(0);

    if(!shape::isCommonVector(X->shapeInfo(), xLenDim))
        throw std::runtime_error("MmulHelper::dot cuda: X array must be vector !");
    if(!shape::isCommonVector(Y->shapeInfo(), yLenDim))
        throw std::runtime_error("MmulHelper::dot cuda: Y array must be vector !");
    if(Z != nullptr && !Z->isScalar())
        throw std::runtime_error("MmulHelper::dot cuda: Z array must be scalar !");

    const auto length = X->lengthOf();

    if(Y->lengthOf() != length)
        throw std::runtime_error("MmulHelper::dot cuda: lengths of input vectors are different !");

    if(Z == nullptr)
        Z = new NDArray(DataTypeUtils::pickPairwiseResultType(X->dataType(), Y->dataType()), X->getContext());

    const Nd4jLong incx = X->strideAt(xLenDim);
    const Nd4jLong incy = Y->strideAt(yLenDim);

    const auto xType = X->dataType();
    const auto yType = Y->dataType();
    const auto zType = Z->dataType();

    if(!X->isActualOnDeviceSide())  X->syncToDevice();
    if(!Y->isActualOnDeviceSide())  Y->syncToDevice();
    if(!Z->isActualOnDeviceSide())  Z->syncToDevice();

    hipStream_t* stream = X->getContext()->getCudaStream();

    dim3 threadsPerBlock(512);
    dim3 blocksPerGrid(1);
    if (length > 512)
        threadsPerBlock.x = math::nd4j_ceil<double, int>(static_cast<double>(length) / 512);

    NDArray::prepareSpecialUse({Z}, {X, Y});

    //BUILD_TRIPLE_SELECTOR(xType, yType, zType, usualDot, (blocksPerGrid, threadsPerBlock, stream, length, alpha, X->specialBuffer(), incx, Y->specialBuffer(), incy, beta, Z->specialBuffer()), NUMERIC_TYPES, NUMERIC_TYPES, FLOAT_TYPES);
    BUILD_SINGLE_SELECTOR_THRICE(xType, usualDot, (blocksPerGrid, threadsPerBlock, stream, length, alpha, X->specialBuffer(), incx, Y->specialBuffer(), incy, beta, Z->specialBuffer()), NUMERIC_TYPES)

    auto cudaResult = hipStreamSynchronize(*stream);
    if (cudaResult != 0) throw cuda_exception::build("MmulHelper::dot cuda failed !", cudaResult);

    NDArray::registerSpecialUse({Z}, {X, Y});

    return Z;
}

//////////////////////////////////////////////////////////////////////////////
// [bS,M,K] x [bS,K,N] = [bS,M,N]
// [bS,M,K] x    [K,N] = [bS,M,N]
//    [M,K] x [bS,K,N] = [bS,M,N]
// bS could stand for several axes
template <typename T1, typename T2, typename T3>
static __global__ void batchedCudaGemm(const void* vA, const Nd4jLong* aShapeInfo, const void* vB, const Nd4jLong* bShapeInfo, void* vC, const Nd4jLong* cShapeInfo,
                                       const int* aBatchDims, const int* bBatchDims, const int* cBatchDims,
                                       const int aMaxis, const int aKaxis, const int bKaxis, const int bNaxis, const int cMaxis, const int cNaxis,
                                       const double alpha, const double beta) {

    const T1* A = reinterpret_cast<const T1*>(vA);
    const T2* B = reinterpret_cast<const T2*>(vB);
          T3* C = reinterpret_cast<      T3*>(vC);

    __shared__ bool betaPresent;
    __shared__ int aRank, bRank, cRank, K, *coords;
    __shared__ Nd4jLong cLen, totalThreads;
    __shared__ T3 alphaZ, betaZ;

    if (threadIdx.x == 0) {

        extern __shared__ unsigned char shmem[];
        coords = reinterpret_cast<int*>(shmem);
        cLen = shape::length(cShapeInfo);

        K = shape::shapeOf(const_cast<Nd4jLong*>(aShapeInfo))[aKaxis];

        totalThreads = gridDim.x * blockDim.x;
        aRank = shape::rank(aShapeInfo);
        bRank = shape::rank(bShapeInfo);
        cRank = shape::rank(cShapeInfo);

        betaPresent = beta;

        alphaZ = alpha;
        betaZ  = beta;
    }
    __syncthreads();

    auto aCoords = coords + threadIdx.x * (aRank + bRank + cRank);
    auto bCoords = aCoords + aRank;
    auto cCoords = bCoords + bRank;

    const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (Nd4jLong i = tid; i < cLen; i += totalThreads) {

        // evaluate C coordinates
        shape::index2coords(i, cShapeInfo, cCoords);

        // calculate index of current batch
        Nd4jLong batchInd;
        if(cBatchDims != nullptr)
            batchInd = shape::coords2index(cShapeInfo, cBatchDims, cRank - 2, cCoords);

        // evaluate A coordinates
        if(aBatchDims != nullptr)
            shape::index2coords(batchInd, aShapeInfo, aBatchDims, aRank - 2, aCoords);
        aCoords[aMaxis] = cCoords[cMaxis];
        aCoords[aKaxis] = 0;

        // evaluate B coordinates
        if(bBatchDims != nullptr)
            shape::index2coords(batchInd, bShapeInfo, bBatchDims, bRank - 2, bCoords);
        bCoords[bKaxis] = 0;
        bCoords[bNaxis] = cCoords[cNaxis];

        auto aOffset = shape::getOffset(aShapeInfo, aCoords);
        auto bOffset = shape::getOffset(bShapeInfo, bCoords);

        T3 val = A[aOffset] * B[bOffset];                       // first iteration

        for (uint j = 1; j < K; ++j) {                          // rest iterations
            aOffset += shape::stride(aShapeInfo)[aKaxis];
            bOffset += shape::stride(bShapeInfo)[bKaxis];
            val = val + A[aOffset] * B[bOffset];
        }

        auto cOffset = shape::getOffset(cShapeInfo, cCoords);

        if(betaPresent)
            C[cOffset] = alphaZ * val + betaZ * C[cOffset];
        else
            C[cOffset] = alphaZ * val;
    }
}

////////////////////////////////////////////////////////////////////////
template <typename T1, typename T2, typename T3>
__host__ static void batchedGemm(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, hipStream_t *stream, const void* vA, const Nd4jLong* aShapeInfo, const void* vB, const Nd4jLong* bShapeInfo, void* vC, const Nd4jLong* cShapeInfo, const int* aBatchDims, const int* bBatchDims, const int* cBatchDims, const int aMaxis, const int aKaxis, const int bKaxis, const int bNaxis, const int cMaxis, const int cNaxis, const double alpha, const double beta) {

    batchedCudaGemm<T1,T2,T3><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(vA, aShapeInfo, vB, bShapeInfo, vC, cShapeInfo, aBatchDims, bBatchDims, cBatchDims, aMaxis, aKaxis, bKaxis, bNaxis, cMaxis, cNaxis, alpha, beta);
}

///////////////////////////////////////////////////////////////////
NDArray* MmulHelper::mmulNxN(const NDArray* A, const NDArray* B, NDArray* C, const double alpha, const double beta, const char outOrder) {

    const int aRank = A->rankOf();
    const int bRank = B->rankOf();

    // input ranks validation
    if(aRank > bRank && bRank != 2)
        throw std::runtime_error("MmulHelper::mmulNxN: rank of B array should be equal 2 !");
    else if(bRank > aRank && aRank != 2)
        throw std::runtime_error("MmulHelper::mmulNxN: rank of A array should be equal 2 !");
    else if (aRank == bRank ) {
        for(int i = 0; i < aRank - 2; ++i)
            if(A->sizeAt(i) != B->sizeAt(i))
                throw std::runtime_error("MmulHelper::mmulNxN: shapes of A and B arrays are not suitable for matrix multiplication !");
    }

    if(A->sizeAt(-1) != B->sizeAt(-2))
        throw std::runtime_error("MmulHelper::mmulNxN: shapes of A and B arrays are not suitable for matrix multiplication !");

    // validation of C array
    std::vector<Nd4jLong> cExpectedShape = aRank > bRank ? A->getShapeAsVector() : B->getShapeAsVector();
    cExpectedShape[cExpectedShape.size() - 2] = A->sizeAt(-2);
    cExpectedShape[cExpectedShape.size() - 1] = B->sizeAt(-1);

    if(C != nullptr ) {
        if(!C->isSameShape(cExpectedShape))
            throw std::runtime_error("MmulHelper::mmulNxN: shape of C array is not suitable for AxB matrix multiplication !");
    }
    else
        C = new NDArray(outOrder, cExpectedShape, DataTypeUtils::pickPairwiseResultType(A->dataType(), B->dataType()), A->getContext());

    if (C->isEmpty())
        return C;

    const int cRank = C->rankOf();

    const int aMaxis(aRank-2), aKaxis(aRank-1), bKaxis(bRank-2), bNaxis(bRank-1), cMaxis(cRank-2), cNaxis(cRank-1);

    const int threadsPerBlock = MAX_NUM_THREADS / 8;
    const int blocksPerGrid = (C->lengthOf() + threadsPerBlock - 1) / threadsPerBlock;
    const int sharedMem = threadsPerBlock * sizeof(int) * (aRank + bRank + cRank) + 128;

    PointersManager manager(A->getContext(), "MmulHelper::mmulNxN");

    const int *aBatchDims(nullptr), *bBatchDims(nullptr), *cBatchDims(nullptr);

    if(aRank > 2)
        aBatchDims = reinterpret_cast<int*>(manager.replicatePointer(ShapeUtils::evalDimsToExclude(aRank, {aMaxis, aKaxis}).data(), (aRank - 2) * sizeof(int)));
    if(bRank > 2)
        bBatchDims = reinterpret_cast<int*>(manager.replicatePointer(ShapeUtils::evalDimsToExclude(bRank, {bKaxis, bNaxis}).data(), (bRank - 2) * sizeof(int)));
    if(cRank > 2)
        cBatchDims = reinterpret_cast<int*>(manager.replicatePointer(ShapeUtils::evalDimsToExclude(cRank, {cMaxis, cNaxis}).data(), (cRank - 2) * sizeof(int)));

    NDArray::prepareSpecialUse({C}, {A, B});
    // BUILD_TRIPLE_SELECTOR(A->dataType(), b->dataType(), C->dataType(), batchedGemm, (blocksPerGrid, threadsPerBlock, A->getContext()->getCudaStream(), A->specialBuffer(), A->specialShapeInfo(), B->specialBuffer(), B->specialShapeInfo(), C->specialBuffer(), C->special(), aMaxis, aKaxis, bKaxis, bNaxis, cMaxis, cNaxis, alpha, beta), NUMERIC_TYPES, NUMERIC_TYPES, FLOAT_TYPES);
    BUILD_SINGLE_SELECTOR_THRICE(A->dataType(), batchedGemm, (blocksPerGrid, threadsPerBlock, sharedMem, A->getContext()->getCudaStream(), A->specialBuffer(), A->specialShapeInfo(), B->specialBuffer(), B->specialShapeInfo(), C->specialBuffer(), C->specialShapeInfo(), aBatchDims, bBatchDims, cBatchDims, aMaxis, aKaxis, bKaxis, bNaxis, cMaxis, cNaxis, alpha, beta), NUMERIC_TYPES)
    NDArray::registerSpecialUse({C}, {A, B});

    manager.synchronize();

    return C;
}


/*
//////////////////////////////////////////////////////////////////////////////
// MXN x N = M
template <typename T1, typename T2, typename T3>
static __global__ void usualCudaGemv(const bool transA, const int M, const int N, const double alpha, const void* vA, const int lda, const void* vX, const int incx, const double beta, void* vY, const int incy) {

    T1* A = reinterpret_cast<T1*>(const_cast<void*>(vA));
    T2* X = reinterpret_cast<T2*>(const_cast<void*>(vX));
    T3* Y = reinterpret_cast<T3*>(vY);

    __shared__ T3 alphaZ, betaZ;
    __shared__ Nd4jLong strideArow, strideAcol;

    const int row = blockIdx.x * blockDim.x + threadIdx.x;

    if(row == 0) {

        alphaZ = alpha;
        betaZ  = beta;

        if(transA) { strideArow = lda; strideAcol = 1; } else { strideArow = 1; strideAcol = lda; }
    }

    __syncthreads();

    T3 val = 0;
    if (row < M)
        for (int i = 0; i < N; i++)
            val = val + A[row * strideArow + i * strideAcol] * X[i * incx];

    Y[row * incy] = alphaZ * val + betaZ * Y[row * incy];
}

////////////////////////////////////////////////////////////////////////
template <typename T1, typename T2, typename T3>
__host__ static void usualGemv(const dim3 &blocksPerGrid, const dim3 &threadsPerBlock, hipStream_t *stream, const bool transA, const int M, const int N, const double alpha, const void* vA, const int lda, const void* vX, const int incx, const double beta, void* vY, const int incy) {

    usualCudaGemv<T1,T2,T3><<<blocksPerGrid, threadsPerBlock, 1024, *stream>>>(transA, M, N, alpha, vA, lda, vX, incx, beta, vY, incy);
}
*/
/*
//////////////////////////////////////////////////////////////////////////////
MXK x KxN = MxN
C array must be in f order
template <typename T1, typename T2, typename T3>
static __global__ void usualCudaGemm(const bool transA, const bool transB, const int M, const int N, const int K, const double alpha, const void* vA, const int lda, const void* vB, const int ldb, const double beta, void* vC, const int ldc) {

    T1* A = reinterpret_cast<T1*>(const_cast<void*>(vA));
    T2* B = reinterpret_cast<T2*>(const_cast<void*>(vB));
    T3* C = reinterpret_cast<T3*>(vC);

    __shared__ T3 alphaZ, betaZ;
    __shared__ Nd4jLong strideArow, strideAcol, strideBrow, strideBcol;

    const int row = blockIdx.y * blockDim.y + threadIdx.y;
    const int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row == 0 && col == 0) {

        alphaZ = alpha;
        betaZ  = beta;

        if(transA) { strideArow = lda; strideAcol = 1; } else { strideArow = 1; strideAcol = lda; }
        if(transB) { strideBrow = ldb; strideBcol = 1; } else { strideBrow = 1; strideBcol = ldb; }
    }

    __syncthreads();

    T3 val = 0;
    if (row < M && col < N)
        for (int i = 0; i < K; i++)
            val = val + A[row * strideArow + i * strideAcol] * B[i * strideBrow + col * strideBcol];

    C[row + col * ldc] = alphaZ * val + betaZ * C[row + col * ldc];
}

//////////////////////////////////////////////////////////////////////////////
template <typename T1, typename T2, typename T3>
__host__ static void usualGemm(const dim3 &blocksPerGrid, const dim3 &threadsPerBlock, hipStream_t *stream, const bool transA, const bool transB, const int M, const int N, const int K, const double alpha, const void* vA, const int lda, const void* vB, const int ldb, const double beta, void* vC, const int ldc) {

    usualCudaGemm<T1,T2,T3><<<blocksPerGrid, threadsPerBlock, 1024, *stream>>>(transA, transB, M, N, K, alpha, vA, lda, vB, ldb, beta, vC, ldc);
}
*/
//////////////////////////////////////////////////////////////////////////
/*
NDArray* MmulHelper::mmulNxNold1(const NDArray* A, const NDArray* B, NDArray* C, const double alpha, const double beta, const char outOrder) {

    const int aRank = A->rankOf();
    const int bRank = B->rankOf();

    // input ranks validation
    if(aRank > bRank && bRank != 2)
        throw std::runtime_error("MmulHelper::mmulNxN: rank of B array should be equal 2 !");
    else if(bRank > aRank && aRank != 2)
        throw std::runtime_error("MmulHelper::mmulNxN: rank of A array should be equal 2 !");
    else if (aRank == bRank ) {
        for(int i = 0; i < aRank - 2; ++i)
            if(A->sizeAt(i) != B->sizeAt(i))
                throw std::runtime_error("MmulHelper::mmulNxN: shapes of A and B arrays are not suitable for matrix multiplication !");
    }

    if(A->sizeAt(-1) != B->sizeAt(-2))
        throw std::runtime_error("MmulHelper::mmulNxN: shapes of A and B arrays are not suitable for matrix multiplication !");

    // validation of C array
    std::vector<Nd4jLong> cExpectedShape = aRank > bRank ? A->getShapeAsVector() : B->getShapeAsVector();
    cExpectedShape[cExpectedShape.size() - 2] = A->sizeAt(-2);
    cExpectedShape[cExpectedShape.size() - 1] = B->sizeAt(-1);

    if(C != nullptr ) {
        if(!C->isSameShape(cExpectedShape))
            throw std::runtime_error("MmulHelper::mmulNxN: shape of C array is not suitable for AxB matrix multiplication !");
    }
    else {
        C = new NDArray(outOrder, cExpectedShape, B->dataType());
    }


    // multiplication
    const std::vector<int> dimsToExclude = ShapeUtils::evalDimsToExclude(C->rankOf(), {-2, -1});
    const Nd4jLong numOfSubArrs = ShapeUtils::getNumOfSubArrs(C->shapeInfo(), dimsToExclude);
    std::vector<Nd4jLong> idxRanges(2 * C->rankOf());

// #pragma omp parallel for schedule(guided) firstprivate(idxRanges)
        for(Nd4jLong i = 0; i < numOfSubArrs; ++i) {

            ShapeUtils::evalIdxRangesForSubArr(i, C->shapeInfo(), dimsToExclude, idxRanges.data());
            NDArray cSubArr = (*C)(idxRanges);

            if(aRank > bRank) {
                NDArray aSubArr = (*A)(idxRanges);
                mmulMxM(&aSubArr, B, &cSubArr, 1., 0., outOrder);
            }
            else if(bRank > aRank) {
                NDArray bSubArr = (*B)(idxRanges);
                mmulMxM(A, &bSubArr, &cSubArr, 1., 0, outOrder);
            }
            else {
                NDArray aSubArr = (*A)(idxRanges);
                NDArray bSubArr = (*B)(idxRanges);
                mmulMxM(&aSubArr, &bSubArr, &cSubArr, 1., 0., outOrder);
            }
        }

    return C;
}
*/

//////////////////////////////////////////////////////////////////////////
// [bS,M,K] x [bS,K,N] = [bS,M,N]
// [bS,M,K] x    [K,N] = [bS,M,N]
//    [M,K] x [bS,K,N] = [bS,M,N]
// bS could stand for several axes
/*
NDArray* MmulHelper::mmulNxNold2(const NDArray* A, const NDArray* B, NDArray* C, const double alpha, const double beta, const char outOrder) {

    const int aRank = A->rankOf();
    const int bRank = B->rankOf();

    // input ranks validation
    if(aRank > bRank && bRank != 2)
        throw std::runtime_error("MmulHelper::mmulNxN: rank of B array should be equal 2 !");
    else if(bRank > aRank && aRank != 2)
        throw std::runtime_error("MmulHelper::mmulNxN: rank of A array should be equal 2 !");
    else if (aRank == bRank ) {
        for(int i = 0; i < aRank - 2; ++i)
            if(A->sizeAt(i) != B->sizeAt(i))
                throw std::runtime_error("MmulHelper::mmulNxN: shapes of A and B arrays are not suitable for matrix multiplication !");
    }

    if(A->sizeAt(-1) != B->sizeAt(-2))
        throw std::runtime_error("MmulHelper::mmulNxN: shapes of A and B arrays are not suitable for matrix multiplication !");

    // validation of C array
    std::vector<Nd4jLong> cExpectedShape = aRank > bRank ? A->getShapeAsVector() : B->getShapeAsVector();
    cExpectedShape[cExpectedShape.size() - 2] = A->sizeAt(-2);
    cExpectedShape[cExpectedShape.size() - 1] = B->sizeAt(-1);

    if(C != nullptr ) {
        if(!C->isSameShape(cExpectedShape))
            throw std::runtime_error("MmulHelper::mmulNxN: shape of C array is not suitable for AxB matrix multiplication !");
    }
    else
        C = new NDArray(outOrder, cExpectedShape, B->dataType());

    const int cRank = C->rankOf();

    const auto M = A->sizeAt(-2);
    const auto K = A->sizeAt(-1);
    const auto N = B->sizeAt(-1);

    NDArray *pA(const_cast<NDArray*>(A)), *pB(const_cast<NDArray*>(B)), *pC(const_cast<NDArray*>(C));
    std::vector<NDArray*> toDelete;

    bool aMcont = M == 1 || A->strideAt(-2) == 1;
    bool aKcont = K == 1 || A->strideAt(-1) == 1;
    bool bKcont = K == 1 || B->strideAt(-2) == 1;
    bool bNcont = N == 1 || B->strideAt(-1) == 1;
    bool cMcont = M == 1 || C->strideAt(-2) == 1;
    bool cNcont = N == 1 || C->strideAt(-1) == 1;

    if(!aMcont && !aKcont) {
        pA = new NDArray(A->dup('c'));
        toDelete.push_back(pA);
        aKcont = true;
    }
    if(!bKcont && !bNcont) {
        pB = new NDArray(B->dup('c'));
        toDelete.push_back(pB);
        bNcont = true;
    }
    std::vector<int> permut(cRank);
    if(!cMcont) {
        std::iota(permut.begin(), permut.end(), 0);
        permut[cRank - 2] = cRank - 1;
        permut[cRank - 1] = cRank - 2;  // swap two last dimensions [..., M,N] -> [..., N,M]
        auto Cpermut = C->permute(permut);
        pC = new NDArray('c', Cpermut.getShapeAsVector(), Cpermut.dataType(), A->getContext());
        pC->assign(Cpermut);
        toDelete.push_back(pC);
        cMcont = true;
    }


    const auto aType = pA->dataType();
    const auto bType = pB->dataType();
    const auto cType = pC->dataType();

    const bool AB(aType == bType), AC(aType == cType), ABC(AB && AC);

    bool badTypes = false;
    hipDataType cudaType, cudaAType, cudaBType, cudaCType;

    if(ABC && aType == DataType::HALF) {
        cudaType = cudaAType = cudaBType = cudaCType = HIP_R_16F;
    }
    else if(ABC && aType == DataType::FLOAT32) {
        cudaType = cudaAType = cudaBType = cudaCType = HIP_R_32F;
    }
    else if(ABC && aType == DataType::DOUBLE) {
        cudaType = cudaAType = cudaBType = cudaCType = HIP_R_64F;
    }
    else if(AB && cType == DataType::FLOAT32 && aType == DataType::INT8) {
        cudaType = cudaCType = HIP_R_32F;
        cudaAType = cudaBType = HIP_R_8I;
    }
    else if(AB && cType == DataType::FLOAT32 && aType == DataType::HALF) {
        cudaType = cudaCType = HIP_R_32F;
        cudaAType = cudaBType = HIP_R_16F;
    }
    else
        badTypes = true;

    const int bS = pC->lengthOf() / (M*N);

    const std::vector<int> dimsToExclude = ShapeUtils::evalDimsToExclude(cRank, {-2, -1});

    NDArray::prepareSpecialUse({pC}, {pA, pB});

    if(!badTypes) {

        std::vector<Nd4jLong> subArrOffsets(bS);
        std::vector<Nd4jLong> subArrShapeInfo(shape::shapeInfoLength(2));                         // all sub-arrays have rank = 2

        std::vector<void*> aSubArrs(bS), bSubArrs(bS), cSubArrs(bS);

        if(aRank > 2)
            shape::calcSubArrsShapeInfoAndOffsets(pA->shapeInfo(), bS, dimsToExclude.size(), dimsToExclude.data(), subArrShapeInfo.data(), subArrOffsets.data());
        for (int i = 0; i < bS; ++i)
            aSubArrs[i] = aRank == 2 ? pA->specialBuffer() : pA->specialBuffer() + subArrOffsets[i] * pA->sizeOfT();

        if(bRank > 2)
            shape::calcSubArrsShapeInfoAndOffsets(pB->shapeInfo(), bS, dimsToExclude.size(), dimsToExclude.data(), subArrShapeInfo.data(), subArrOffsets.data());
        for (int i = 0; i < bS; ++i)
            bSubArrs[i] = bRank == 2 ? pB->specialBuffer() : pB->specialBuffer() + subArrOffsets[i] * pB->sizeOfT();

        shape::calcSubArrsShapeInfoAndOffsets(pC->shapeInfo(), bS, dimsToExclude.size(), dimsToExclude.data(), subArrShapeInfo.data(), subArrOffsets.data());
        for (int i = 0; i < bS; ++i)
            cSubArrs[i] = pC->specialBuffer() + subArrOffsets[i] * pC->sizeOfT();

        PointersManager manager(A->getContext(), "mmulNxN");

        const void** aSubArrsCuda = reinterpret_cast<const void **>(manager.replicatePointer(aSubArrs.data(),  aSubArrs.size() * sizeof(void*)));
        const void** bSubArrsCuda = reinterpret_cast<const void **>(manager.replicatePointer(bSubArrs.data(),  bSubArrs.size() * sizeof(void*)));
              void** cSubArrsCuda = reinterpret_cast<      void **>(manager.replicatePointer(cSubArrs.data(),  cSubArrs.size() * sizeof(void*)));

        const bool transA = !aMcont;
        const bool transB = !bKcont;

        const int lda = (aMcont && aKcont) ? M : transA  ? pA->strideAt(-2) : pA->strideAt(-1);
        const int ldb = (bKcont && bNcont) ? K : transB  ? pB->strideAt(-2) : pB->strideAt(-1);
        const int ldc = (cMcont && cNcont) ? M : C != pC ? pC->strideAt(-2) : pC->strideAt(-1);

        const hipblasOperation_t transAblas = transA ? HIPBLAS_OP_T : HIPBLAS_OP_N;
        const hipblasOperation_t transBblas = transB ? HIPBLAS_OP_T : HIPBLAS_OP_N;

        union Coeff {__half _h; float _f; double _d; };
        Coeff uAlpha, uBeta;

        if(cudaType == HIP_R_16F) {
            uAlpha._h = alpha;
            uBeta._h  = beta;
        }
        else if(cudaType == HIP_R_32F) {
            uAlpha._f = alpha;
            uBeta._f  = beta;
        }
        else if(cudaType == HIP_R_64F) {
            uAlpha._d = alpha;
            uBeta._d  = beta;
        }

        auto handle = reinterpret_cast<hipblasHandle_t *>(A->getContext()->getCublasHandle());
        auto stream = A->getContext()->getCudaStream();

        auto status = hipblasSetStream(*handle, *stream);
        if (status != HIPBLAS_STATUS_SUCCESS)
            throw cuda_exception::build("MmulHelper::mmulNxN cuda failed !", status);

        status = hipblasGemmBatchedEx(*handle, transAblas, transBblas, M, N, K, &uAlpha, aSubArrsCuda, cudaAType, lda, bSubArrsCuda, cudaBType, ldb, &uBeta, cSubArrsCuda, cudaCType, ldc, bS, cudaType, HIPBLAS_GEMM_DEFAULT);

        if (status != HIPBLAS_STATUS_SUCCESS)
            throw cuda_exception::build("MmulHelper::mmulNxN cuda failed !", status);

        auto cudaResult = hipStreamSynchronize(*stream);
        if (cudaResult != 0)
            throw cuda_exception::build("MmulHelper::mmulNxN cuda failed !", cudaResult);
    }
    else {

        std::vector<Nd4jLong> idxRanges(2 * pC->rankOf());

        for(Nd4jLong i = 0; i < bS; ++i) {

            ShapeUtils::evalIdxRangesForSubArr(i, pC->shapeInfo(), dimsToExclude, idxRanges.data());
            NDArray cSubArr = (*pC)(idxRanges);

            if(aRank > bRank) {
                NDArray aSubArr = (*pA)(idxRanges);
                mmulMxM(&aSubArr, pB, &cSubArr, 1., 0., pC->ordering());
            }
            else if(bRank > aRank) {
                NDArray bSubArr = (*pB)(idxRanges);
                mmulMxM(pA, &bSubArr, &cSubArr, 1., 0, pC->ordering());
            }
            else {
                NDArray aSubArr = (*pA)(idxRanges);
                NDArray bSubArr = (*pB)(idxRanges);
                mmulMxM(&aSubArr, &bSubArr, &cSubArr, 1., 0., pC->ordering());
            }
        }
    }

    NDArray::registerSpecialUse({pC}, {pA, pB});

    if(C != pC)
        C->assign(pC->permute(permut));

    for(int i = toDelete.size() - 1; i >= 0; --i)
        delete toDelete[i];

    return C;
}
*/

//BUILD_TRIPLE_TEMPLATE(template void usualGemm, (const dim3 &blocksPerGrid, const dim3 &threadsPerBlock, hipStream_t *stream, const bool transA, const bool transB, const int M, const int N, const int K, const double alpha, const void* vA, const int lda, const void* vB, const int ldb, const double beta, void* vC, const int ldc), NUMERIC_TYPES, NUMERIC_TYPES, FLOAT_TYPES);
//BUILD_TRIPLE_TEMPLATE(template void usualGemv, (const dim3 &blocksPerGrid, const dim3 &threadsPerBlock, hipStream_t *stream, const bool transA, const int M, const int N, const double alpha, const void* vA, const int lda, const void* vB, const int incx, const double beta, void* vC, const int incy), NUMERIC_TYPES, NUMERIC_TYPES, FLOAT_TYPES);
//BUILD_TRIPLE_TEMPLATE(template void usualDot,  (const dim3 &blocksPerGrid, const dim3 &threadsPerBlock, hipStream_t *stream, const Nd4jLong length, const double alpha, const void* vX, const Nd4jLong incx, const void* vY, const Nd4jLong incy, const double beta, void* vZ), NUMERIC_TYPES, NUMERIC_TYPES, FLOAT_TYPES);

}