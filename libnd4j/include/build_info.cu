/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

#include <config.h>
#include <build_info.h>

const char* buildInfo() {
	return ""
#if defined(__clang__)
        "Clang: " TOSTRING(__clang_version__)
#elif defined(_MSC_VER)
        "MSVC: " TOSTRING(_MSC_FULL_VER)
#else
        "GCC: " TOSTRING(__VERSION__)
#endif
#if defined(_MSC_VER) && defined(_MSVC_LANG)  
        "\nSTD version: " TOSTRING(_MSVC_LANG)
#elif defined(__cplusplus)
        "\nSTD version: " TOSTRING(__cplusplus)
#endif

#if defined(__HIPCC__)
        "\nCUDA: " TOSTRING(__CUDACC_VER_MAJOR__)
        "."  TOSTRING(__CUDACC_VER_MINOR__)
        "." TOSTRING(__CUDACC_VER_BUILD__)
#endif
#if defined(DEFAULT_ENGINE)
        "\nDEFAULT_ENGINE: " TOSTRING(DEFAULT_ENGINE)
#endif
#if defined(HAVE_FLATBUFFERS)
        "\nHAVE_FLATBUFFERS"
#endif
#if defined(HAVE_MKLDNN)
        "\nHAVE_MKLDNN"
#endif
#if defined(__EXTERNAL_BLAS__)
        "\nHAVE_EXTERNAL_BLAS"
#endif
#if defined(HAVE_OPENBLAS)
        "\nHAVE_OPENBLAS"
#endif
#if defined(HAVE_CUDNN)
        "\nHAVE_CUDNN"
#endif
#if defined(HAVE_ARMCOMPUTE)
        "\nHAVE_ARMCOMPUTE"
#endif
		;
}
