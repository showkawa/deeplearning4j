/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// CUDA workspaces implementation
//
// @author raver119@gmail.com
//

#include <system/op_boilerplate.h>
#include <atomic>
#include <stdio.h>
#include <stdlib.h>
#include "../Workspace.h"
#include <helpers/logger.h>
#include <math/templatemath.h>
#include <cstring>
#include <exceptions/cuda_exception.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

namespace sd {
    namespace memory {
        Workspace::Workspace(ExternalWorkspace *external) {
            if (external->sizeHost() > 0) {
                _ptrHost = (char *) external->pointerHost();
                _ptrDevice = (char *) external->pointerDevice();

                _initialSize = external->sizeDevice();
                _currentSize = external->sizeDevice();
                _initialSizeSecondary = external->sizeHost();
                _currentSizeSecondary = external->sizeHost();
                _offset = 0L;
                _offsetSecondary = 0L;
                this->_cycleAllocations = 0;
                this->_cycleAllocationsSecondary = 0;
                this->_spillsSize = 0;
                this->_spillsSizeSecondary = 0;

                _externalized = true;
            }
        }

        Workspace::Workspace(Nd4jLong primarySize, Nd4jLong secondarySize) {
            if (secondarySize > 0) {
                auto res = hipHostAlloc(reinterpret_cast<void **>(&_ptrHost), secondarySize, hipHostMallocDefault);
                if (res != 0)
                    throw cuda_exception::build("Can't allocate [HOST] memory", res);

                hipMemset(this->_ptrHost, 0, secondarySize);
                this->_allocatedHost = true;
            } else
                this->_allocatedHost = false;

            if (primarySize > 0) {
                auto res = hipMalloc(reinterpret_cast<void **>(&_ptrDevice), primarySize);
                if (res != 0)
                    throw cuda_exception::build("Can't allocate [DEVICE] memory", res);

                hipMemset(this->_ptrDevice, 0, primarySize);
                this->_allocatedDevice = true;
            } else
                this->_allocatedDevice = false;

            this->_initialSize = primarySize;
            this->_initialSizeSecondary = secondarySize;
            this->_currentSize = primarySize;
            this->_currentSizeSecondary = secondarySize;
            this->_offset = 0;
            this->_offsetSecondary = 0;
            this->_cycleAllocations = 0;
            this->_spillsSize = 0;
            this->_spillsSizeSecondary = 0;
        }

        void Workspace::init(Nd4jLong primaryBytes, Nd4jLong secondaryBytes) {
            if (this->_currentSize < primaryBytes) {
                if (this->_allocatedDevice && !_externalized)
                    hipFree((void *)this->_ptrDevice);

                auto res = hipMalloc(reinterpret_cast<void **>(&_ptrDevice), secondaryBytes);
                if (res != 0)
                    throw cuda_exception::build("Can't allocate [DEVICE] memory", res);

                hipMemset(this->_ptrDevice, 0, primaryBytes);
                this->_currentSize = primaryBytes;
                this->_allocatedDevice = true;
            }

            if (this->_currentSizeSecondary < secondaryBytes) {
                if (this->_allocatedHost && !_externalized)
                    hipHostFree((void *)this->_ptrHost);

                auto res = hipHostAlloc(reinterpret_cast<void **>(&_ptrHost), secondaryBytes, hipHostMallocDefault);
                if (res != 0)
                    throw cuda_exception::build("Can't allocate [HOST] memory", res);


                hipMemset(this->_ptrHost, 0, secondaryBytes);
                this->_currentSizeSecondary = secondaryBytes;
                this->_allocatedHost = true;
            }
        }

        void Workspace::expandBy(Nd4jLong numBytes, Nd4jLong secondaryBytes) {
            this->init(_currentSize + numBytes, _currentSizeSecondary + secondaryBytes);
        }

        void Workspace::expandTo(Nd4jLong numBytes, Nd4jLong secondaryBytes) {
            this->init(numBytes, secondaryBytes);
        }

        void Workspace::freeSpills() {
            _spillsSize = 0;
            _spillsSizeSecondary = 0;

            for (auto v:_spills)
                hipFree(v);

            for (auto v:_spillsSecondary)
                hipHostFree(v);

            _spills.clear();
            _spillsSecondary.clear();
        }

        Workspace::~Workspace() {
            if (this->_allocatedHost && !_externalized)
                hipHostFree((void *)this->_ptrHost);

            if (this->_allocatedDevice && !_externalized)
                hipFree((void *)this->_ptrDevice);

            freeSpills();
        }

        Nd4jLong Workspace::getUsedSize() {
            return getCurrentOffset();
        }

        Nd4jLong Workspace::getCurrentSize() {
            return _currentSize;
        }

        Nd4jLong Workspace::getCurrentOffset() {
            return _offset.load();
        }


        void* Workspace::allocateBytes(Nd4jLong numBytes) {
            return allocateBytes(sd::memory::MemoryType::HOST, numBytes);
        }

        Nd4jLong Workspace::getAllocatedSize() {
            return getCurrentSize() + getSpilledSize();
        }

        void Workspace::scopeIn() {
            freeSpills();
            init(_cycleAllocations.load());
            _cycleAllocations = 0;
        }

        void Workspace::scopeOut() {
            _offset = 0;
        }

        Nd4jLong Workspace::getSpilledSize() {
            return _spillsSize.load();
        }

        void* Workspace::allocateBytes(sd::memory::MemoryType type, Nd4jLong numBytes) {
            switch (type) {
                case HOST: {
                        if (numBytes < 1)
                            throw allocation_exception::build("Number of [HOST] bytes for allocation should be positive", numBytes);


                        //numBytes += 32;
                        void* result = nullptr;
                        this->_cycleAllocationsSecondary += numBytes;
                        this->_mutexAllocation.lock();

                        if (_offsetSecondary.load() + numBytes > _currentSizeSecondary) {
                            nd4j_debug("Allocating %lld [HOST] bytes in spills\n", numBytes);
                            this->_mutexAllocation.unlock();

                            Nd4jPointer p;
                            auto res = hipHostAlloc(reinterpret_cast<void **>(&p), numBytes, hipHostMallocDefault);
                            if (res != 0)
                                throw cuda_exception::build("Can't allocate [HOST] memory", res);

                            _mutexSpills.lock();
                            _spillsSecondary.push_back(p);
                            _mutexSpills.unlock();

                            _spillsSizeSecondary += numBytes;

                            return p;
                        }

                        result = (void *)(_ptrHost + _offsetSecondary.load());
                        _offsetSecondary += numBytes;
                        //memset(result, 0, (int) numBytes);

                        nd4j_debug("Allocating %lld bytes from [HOST] workspace; Current PTR: %p; Current offset: %lld\n", numBytes, result, _offset.load());

                        this->_mutexAllocation.unlock();

                        return result;
                    }
                    break;
                case DEVICE: {
                        if (numBytes < 1)
                            throw allocation_exception::build("Number of [DEVICE] bytes for allocation should be positive", numBytes);


                        //numBytes += 32;
                        void* result = nullptr;
                        this->_cycleAllocations += numBytes;
                        this->_mutexAllocation.lock();

                        if (_offset.load() + numBytes > _currentSize) {
                            nd4j_debug("Allocating %lld [DEVICE] bytes in spills\n", numBytes);
                            this->_mutexAllocation.unlock();

                            Nd4jPointer p;
                            auto res = hipMalloc(reinterpret_cast<void **>(&p), numBytes);
                            if (res != 0)
                                throw cuda_exception::build("Can't allocate [DEVICE] memory", res);

                            _mutexSpills.lock();
                            _spills.push_back(p);
                            _mutexSpills.unlock();

                            _spillsSize += numBytes;

                            return p;
                        }

                        result = (void *)(_ptrDevice + _offset.load());
                        _offset += numBytes;
                        //memset(result, 0, (int) numBytes);

                        nd4j_debug("Allocating %lld bytes from [DEVICE] workspace; Current PTR: %p; Current offset: %lld\n", numBytes, result, _offset.load());

                        this->_mutexAllocation.unlock();

                        return result;
                    }
                    break;
                default:
                    throw std::runtime_error("Unknown MemoryType was passed in");
            }
        }

        Workspace* Workspace::clone() {
            // for clone we take whatever is higher: current allocated size, or allocated size of current loop
            return new Workspace(sd::math::nd4j_max<Nd4jLong >(this->getCurrentSize(), this->_cycleAllocations.load()));
        }

        Nd4jLong Workspace::getAllocatedSecondarySize() {
            return getCurrentSecondarySize() + getSpilledSecondarySize();
        }

        Nd4jLong Workspace::getCurrentSecondarySize() {
            return _currentSizeSecondary;
        }

        Nd4jLong Workspace::getCurrentSecondaryOffset() {
            return _offsetSecondary.load();
        }

        Nd4jLong Workspace::getSpilledSecondarySize() {
            return _spillsSizeSecondary;
        }

        Nd4jLong Workspace::getUsedSecondarySize() {
            return getCurrentSecondaryOffset();
        }

    }
}
