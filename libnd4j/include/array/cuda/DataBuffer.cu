/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
// @author Yurii Shyrma (iuriish@yahoo.com)
//

#include "../DataBuffer.h"
#include <array/DataTypeUtils.h>
#include <system/op_boilerplate.h>
#include <exceptions/cuda_exception.h>
#include <execution/AffinityManager.h>
#include <memory/MemoryCounter.h>
#include <exceptions/allocation_exception.h>

namespace sd {
    void DataBuffer::expand(const uint64_t size) {
        if (size > _lenInBytes) {
            // allocate new buffer
            int8_t *newBuffer = nullptr;
            int8_t *newSpecialBuffer = nullptr;
            ALLOCATE_SPECIAL(newSpecialBuffer, _workspace, size, int8_t);

            // copy data from existing buffer
            if (_primaryBuffer != nullptr) {
                // there's non-zero chance that primary buffer doesn't exist yet
                ALLOCATE(newBuffer, _workspace, size, int8_t);
                std::memcpy(newBuffer, _primaryBuffer, _lenInBytes);

                if (_isOwnerPrimary) {
                    auto ipb = reinterpret_cast<int8_t *>(_primaryBuffer);
                    RELEASE(ipb, _workspace);
                }

                _primaryBuffer = newBuffer;
                _isOwnerPrimary = true;
            }

            hipMemcpy(newSpecialBuffer, _specialBuffer, _lenInBytes, hipMemcpyDeviceToDevice);

            if (_isOwnerSpecial) {
                auto isb = reinterpret_cast<int8_t *>(_specialBuffer);
                RELEASE_SPECIAL(isb, _workspace);
            }

            _specialBuffer = newSpecialBuffer;
            _lenInBytes = size;
            _isOwnerSpecial = true;
        }
    }

////////////////////////////////////////////////////////////////////////
void DataBuffer::allocateSpecial() {

    if (_specialBuffer == nullptr && getLenInBytes() > 0) {
        auto deviceId = sd::AffinityManager::currentDeviceId();

        if (_workspace == nullptr)
            if (!sd::memory::MemoryCounter::getInstance().validate(getLenInBytes()))
                throw sd::allocation_exception::build("Requested amount exceeds device limits", sd::memory::MemoryCounter::getInstance().deviceLimit(deviceId), getLenInBytes());


        ALLOCATE_SPECIAL(_specialBuffer, _workspace, getLenInBytes(), int8_t);
        _isOwnerSpecial = true;

        if (_workspace == nullptr) {
            sd::memory::MemoryCounter::getInstance().countIn(deviceId, getLenInBytes());
            sd::memory::MemoryCounter::getInstance().countIn(sd::memory::MemoryType::DEVICE, getLenInBytes());
        }
    }
}

////////////////////////////////////////////////////////////////////////
void DataBuffer::syncToPrimary(const LaunchContext* context, const bool forceSync) {
    if(isPrimaryActual() && !forceSync) {
        return;
    }

    allocatePrimary();

    auto res = hipStreamSynchronize(*context->getCudaStream());
    if (res != 0)
        throw cuda_exception::build("DataBuffer::syncToPrimary failed to to some previous kernel failre", res);

    res = hipMemcpy(_primaryBuffer, _specialBuffer, getLenInBytes(), hipMemcpyDeviceToHost);
    if (res != 0)
        throw cuda_exception::build("DataBuffer::syncToPrimary hipMemcpy failed", res);

    readPrimary();
}


////////////////////////////////////////////////////////////////////////
void DataBuffer::syncToSpecial(const bool forceSync) {
    // in this case there's nothing to do here
    if (_primaryBuffer == nullptr)
        return;

    if(isSpecialActual() && !forceSync) {
        return;
    }

    allocateSpecial();

    auto res = hipMemcpy(_specialBuffer, _primaryBuffer, getLenInBytes(), hipMemcpyHostToDevice);
    if (res != 0)
        throw cuda_exception::build("DataBuffer::syncToSpecial hipMemcpy failed", res);

    readSpecial();
}


////////////////////////////////////////////////////////////////////////
void DataBuffer::deleteSpecial() {

    if(_isOwnerSpecial && _specialBuffer != nullptr && getLenInBytes() != 0) {
        auto p = reinterpret_cast<int8_t*>(_specialBuffer);
        RELEASE_SPECIAL(p, _workspace);
        _specialBuffer = nullptr;
        _isOwnerSpecial = false;

        // count out towards DataBuffer device, only if we're not in workspace
        if (_workspace == nullptr) {
            sd::memory::MemoryCounter::getInstance().countOut(_deviceId, getLenInBytes());
            sd::memory::MemoryCounter::getInstance().countOut(sd::memory::MemoryType::DEVICE, getLenInBytes());
        }
    }
}


////////////////////////////////////////////////////////////////////////
void DataBuffer::setCountersToZero() {

    _counter.store(0L);
    _writePrimary.store(0L);
    _writeSpecial.store(0L);
    _readPrimary.store(0L);
    _readSpecial.store(0L);
}

////////////////////////////////////////////////////////////////////////
void DataBuffer::copyCounters(const DataBuffer& other) {

    _counter.store(other._counter);
    _writePrimary.store(other._readSpecial);
    _writeSpecial.store(other._readPrimary);
    _readPrimary.store(other._writeSpecial);
    _readSpecial.store(other._writePrimary);
}

////////////////////////////////////////////////////////////////////////
void DataBuffer::copyBufferFrom(const DataBuffer& other, size_t sizeToCopyinBytes, const Nd4jLong offsetThis, const Nd4jLong offsetOther) {     // copies only to special buffer

    if(other._primaryBuffer == nullptr && other._specialBuffer == nullptr)
        return;

    if(sizeToCopyinBytes == 0)
        sizeToCopyinBytes = other.getLenInBytes();
    if(sizeToCopyinBytes == 0)
        return;

    if(other.isPrimaryActual()) {
        auto res = hipMemcpy(static_cast<int8_t*>(_specialBuffer) + offsetThis * DataTypeUtils::sizeOfElement(_dataType), static_cast<const int8_t*>(other._primaryBuffer) + offsetOther * DataTypeUtils::sizeOfElement(other._dataType), sizeToCopyinBytes, hipMemcpyHostToDevice);
        if (res != 0)
            throw cuda_exception::build("DataBuffer::copyBufferFrom: cudaMemcpy_cudaMemcpyHostToDevice failed!", res);
        other.readPrimary();
    }
    else {
        auto res = hipMemcpy(static_cast<int8_t*>(_specialBuffer) + offsetThis * DataTypeUtils::sizeOfElement(_dataType), static_cast<const int8_t*>(other._specialBuffer) + offsetOther * DataTypeUtils::sizeOfElement(other._dataType), sizeToCopyinBytes, hipMemcpyDeviceToDevice);
        if (res != 0)
            throw cuda_exception::build("DataBuffer::copyBufferFrom: cudaMemcpy_cudaMemcpyDeviceToDevice failed!", res);
        other.readSpecial();
    }

    writeSpecial();
}

////////////////////////////////////////////////////////////////////////
void DataBuffer::copyBufferFromHost(const void* hostBuffer, size_t sizeToCopyinBytes, const Nd4jLong offsetThis, const Nd4jLong offsetHostBuffer) {     // copies only to special buffer

    if(hostBuffer == nullptr)
        return;

    if(sizeToCopyinBytes == 0)
        sizeToCopyinBytes = getLenInBytes();
    if(sizeToCopyinBytes == 0)
        return;

    auto res = hipMemcpy(static_cast<int8_t*>(_specialBuffer) + offsetThis * DataTypeUtils::sizeOfElement(_dataType), static_cast<const int8_t*>(hostBuffer) + offsetHostBuffer * DataTypeUtils::sizeOfElement(_dataType), sizeToCopyinBytes, hipMemcpyHostToDevice);
    if (res != 0)
        throw cuda_exception::build("DataBuffer::copyBufferFromHost: cudaMemcpy_cudaMemcpyHostToDevice failed!", res);

    writeSpecial();
}

////////////////////////////////////////////////////////////////////////
void DataBuffer::setSpecial(void* special, const bool isOwnerSpecial) {

    deleteSpecial();
    _specialBuffer = special;
    _isOwnerSpecial = isOwnerSpecial;
}


////////////////////////////////////////////////////////////////////////
void DataBuffer::allocateBuffers(const bool allocBoth) {    // always allocate special buffer only (cuda case)

    allocateSpecial();

    if(allocBoth)
        allocatePrimary();
}

////////////////////////////////////////////////////////////////////////
void DataBuffer::setToZeroBuffers(const bool both) {
    hipMemsetAsync(special(), 0, getLenInBytes(), *LaunchContext::defaultContext()->getCudaStream());
    auto res = hipStreamSynchronize(*LaunchContext::defaultContext()->getCudaStream());
    if (res != 0)
        throw cuda_exception::build("DataBuffer::setToZeroBuffers: streamSync failed!", res);

    writeSpecial();

    if(both) {
        memset(primary(), 0, getLenInBytes());
        readPrimary();
    }
}

/////////////////////////
void DataBuffer::memcpy(const DataBuffer &dst, const DataBuffer &src) {
    if (src._lenInBytes > dst._lenInBytes)
        throw std::runtime_error("DataBuffer::memcpy: Source data buffer is larger than destination");


    int res = 0;
    if (src.isSpecialActual()) {
        res = hipMemcpyAsync(dst._specialBuffer, src._specialBuffer, src.getLenInBytes(), hipMemcpyDeviceToDevice, *LaunchContext::defaultContext()->getCudaStream());
    } else if (src.isPrimaryActual()) {
        res = hipMemcpyAsync(dst._specialBuffer, src._primaryBuffer, src.getLenInBytes(), hipMemcpyHostToDevice, *LaunchContext::defaultContext()->getCudaStream());
    }

    if (res != 0)
        throw cuda_exception::build("DataBuffer::memcpy: hipMemcpyAsync failed!", res);

    res = hipStreamSynchronize(*LaunchContext::defaultContext()->getCudaStream());
    if (res != 0)
        throw cuda_exception::build("DataBuffer::memcpy: streamSync failed!", res);

    dst.writeSpecial();
}

////////////////////////////////////////////////////////////////////////
void DataBuffer::migrate() {
    memory::Workspace* newWorkspace = nullptr;
    void* newBuffer;
    ALLOCATE_SPECIAL(newBuffer, newWorkspace, getLenInBytes(), int8_t);
    auto res = hipMemcpy(newBuffer, _specialBuffer, getLenInBytes(), hipMemcpyDeviceToDevice);
    if (res != 0)
        throw cuda_exception::build("DataBuffer::migrate: hipMemcpyAsync failed!", res);

    if (_isOwnerSpecial) {
        // now we're releasing original buffer
        RELEASE_SPECIAL(_specialBuffer, _workspace);
    }

    _isOwnerSpecial = true;
    _specialBuffer = newBuffer;
}

////////////////////////////////////////////////////////////////////////
void DataBuffer::writePrimary() const    {_writePrimary = ++_counter; }
void DataBuffer::writeSpecial() const    { _writeSpecial = ++_counter; }
void DataBuffer::readPrimary()  const    { _readPrimary  = ++_counter; }
void DataBuffer::readSpecial()  const    { _readSpecial  = ++_counter; }
bool DataBuffer::isPrimaryActual() const { return (_writePrimary.load() > _writeSpecial.load() || _readPrimary.load() > _writeSpecial.load()); }
bool DataBuffer::isSpecialActual() const { return (_writeSpecial.load() > _writePrimary.load() || _readSpecial.load() > _writePrimary.load()); }

}
