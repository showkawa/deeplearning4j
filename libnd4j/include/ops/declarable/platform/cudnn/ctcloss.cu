/*******************************************************************************
 *
 * Copyright (c) 2021 Konduit K.K.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/
 //
 // @author AbdelRauf
 //

#include "cudnnUtils.h"
#include <array/NDArrayFactory.h>
#include <vector>


namespace sd   {
namespace ops     {
namespace platforms {



    template<typename Op, typename ...Args>
    void callCudnnIfNoErr(hipdnnStatus_t &err, Op op, Args&&... args){
        if(err==HIPDNN_STATUS_SUCCESS){
            err = op(std::forward<Args>(args)...);
            if(err){
                nd4j_printf("Cudnn error code %s\n",hipdnnGetErrorString(err));
            }
        }
    }

    template <typename T>
    const T* bufferInHost( const NDArray &array)  {
        array.syncToHost();
        return reinterpret_cast<const T*>(array.buffer());
    }

    std::vector<int> getConcatTargets(const NDArray &targetLabels, const NDArray &targetLabelLengths){
                //concatenate target labels
                const int32_t *tlabels = bufferInHost<int32_t>(targetLabels);
                const int32_t *tlens =bufferInHost<int32_t>(targetLabelLengths);
                int32_t nextOffset = targetLabels.strideAt(0);
                int32_t elStride = targetLabels.strideAt(1);
                int32_t batchCount = targetLabelLengths.lengthOf();
                std::vector<int> labels;
                labels.resize(targetLabels.lengthOf());
                int j=0;
                if(targetLabels.ews()){
                    for(int i=0; i<batchCount;i++){
                        int count = tlens[i];
                        for( int k=0;k<count;k++){
                            labels[j] = tlabels[k];
                            j++;
                        }
                        tlabels+=nextOffset;
                    }
                }else{
                    for(int i=0; i<batchCount;i++){
                        int count = tlens[i];
                        for( int k=0;k<count;k++){
                            labels[j] = tlabels[k*elStride];
                            j++;
                        }
                        tlabels+=nextOffset;
                    }
                }
                return labels;
    }

    hipdnnStatus_t cudnnCtcLoss(const LaunchContext  &context, const  NDArray &probs, const int32_t* targetLabelsPtr, const NDArray&  probInputLengthes,
                               const NDArray &targetLabelLengths, NDArray &ctcLosses,  NDArray &grads){
        const int dims[] = {(int)probs.sizeAt(0), (int)probs.sizeAt(1), (int)probs.sizeAt(2)};
        const int strides[] = {(int)probs.strideAt(0), (int)probs.strideAt(1), (int)probs.strideAt(2)};
        auto handle = reinterpret_cast<hipdnnHandle_t *>(context.getCuDnnHandle());
        hipdnnStatus_t err = HIPDNN_STATUS_SUCCESS;
        callCudnnIfNoErr(err, hipdnnSetStream, *handle, *context.getCudaStream());

        cudnnCTCLossDescriptor_t  ctcLossDesc;
        hipdnnTensorDescriptor_t probsDesc = nullptr;
        hipdnnTensorDescriptor_t gradsDesc = nullptr;
        callCudnnIfNoErr(err, cudnnCreateCTCLossDescriptor, &ctcLossDesc);
        callCudnnIfNoErr(err, cudnnSetCTCLossDescriptorEx, ctcLossDesc, HIPDNN_DATA_FLOAT, CUDNN_LOSS_NORMALIZATION_SOFTMAX, HIPDNN_PROPAGATE_NAN);
        callCudnnIfNoErr(err, hipdnnCreateTensorDescriptor, &probsDesc);
        callCudnnIfNoErr(err, hipdnnSetTensorNdDescriptor, probsDesc, cudnnDataType(probs.dataType()), probs.rankOf() , dims, strides);
        if(!grads.isEmpty()){
            const int gradStrides[] = {(int)grads.strideAt(0), (int)grads.strideAt(1), (int)grads.strideAt(2)};
            callCudnnIfNoErr(err, hipdnnCreateTensorDescriptor, &gradsDesc);
            callCudnnIfNoErr(err, hipdnnSetTensorNdDescriptor, gradsDesc, cudnnDataType(grads.dataType()), grads.rankOf() , dims, gradStrides);
        }

        size_t tempWorkSpaceSize=0;
        callCudnnIfNoErr(err,cudnnGetCTCLossWorkspaceSize, *handle,  probsDesc, gradsDesc,
            targetLabelsPtr,
            bufferInHost<int32_t>(targetLabelLengths),
            bufferInHost<int32_t>(probInputLengthes),
            CUDNN_CTC_LOSS_ALGO_DETERMINISTIC,
            ctcLossDesc, &tempWorkSpaceSize);

        // Allocate temp tempWorkspace buffer
        void *tempWorkSpace = nullptr;
        hipMalloc(&tempWorkSpace, tempWorkSpaceSize);

        NDArray::prepareSpecialUse({&ctcLosses, &grads}, {&probs});
        callCudnnIfNoErr(err, cudnnCTCLoss,*handle,
            probsDesc,
            probs.specialBuffer(),
            targetLabelsPtr,
            bufferInHost<int32_t>(targetLabelLengths),
            bufferInHost<int32_t>(probInputLengthes),
            ctcLosses.specialBuffer(),
            gradsDesc,
            grads.specialBuffer(),
            CUDNN_CTC_LOSS_ALGO_DETERMINISTIC,
            ctcLossDesc,
            tempWorkSpace,
            tempWorkSpaceSize);

        NDArray::registerSpecialUse({&ctcLosses, &grads}, {&probs});

        hipFree(tempWorkSpace);
        callCudnnIfNoErr(err, hipdnnDestroyTensorDescriptor,probsDesc);
        if(gradsDesc) callCudnnIfNoErr(err, hipdnnDestroyTensorDescriptor,gradsDesc);
        callCudnnIfNoErr(err, cudnnDestroyCTCLossDescriptor,ctcLossDesc);
        return err;
     }

    PLATFORM_IMPL(ctc_loss, ENGINE_CUDA) {
        auto targetLabels = INPUT_VARIABLE(0);
        auto logitInput = INPUT_VARIABLE(1);
        auto targetLabelLengths = INPUT_VARIABLE(2);
        auto logitInputLengths = INPUT_VARIABLE(3);
        auto outputLosses = OUTPUT_VARIABLE(0);
        auto context = block.launchContext();
        //in Cudnn Batch is in the middle dimension
        logitInput->permutei({1,0,2});
        //in Cudnn targets are concantenated instead of batched as matrix
        auto labels = getConcatTargets(*targetLabels, *targetLabelLengths);
        const int32_t *ldata= labels.data();
        auto emptyGrads= NDArrayFactory::empty<float>();
        auto err = cudnnCtcLoss(*context, *logitInput, ldata, *logitInputLengths, *targetLabelLengths, *outputLosses, emptyGrads);
        if(err!=HIPDNN_STATUS_SUCCESS) throw sd::cuda_exception::build("ctc_loss CUDNN call failure ", err);
        return Status::OK();
    }

    template<typename T>
    bool checkLabelLength(const NDArray &labelLengthArr){
            //check label lengthes
            auto lenBatch = labelLengthArr.lengthOf();
            for(int i=0; i < lenBatch; i++){
                // The labelLengths is greater than 256.
                if(labelLengthArr.e<int32_t>(i)>256) return false;
            }
            return true;
    }

    PLATFORM_CHECK(ctc_loss, ENGINE_CUDA) {
        auto targetLabels = INPUT_VARIABLE(0);
        auto logitInput = INPUT_VARIABLE(1);
        auto targetLabelLengths = INPUT_VARIABLE(2);
        auto logitInputLengths = INPUT_VARIABLE(3);
        auto outputLosses = OUTPUT_VARIABLE(0);
        int blankIndex = INT_ARG(0);

        auto dTypeInput = logitInput->dataType();
        auto intType = targetLabelLengths->dataType();
        auto dTypeOutput = outputLosses->dataType();

        bool is_supported = blankIndex==0 && intType == DataType::INT32  && dTypeInput == DataType::FLOAT32;
        is_supported = is_supported && outputLosses->ews() && targetLabelLengths->ews() && targetLabels->ews() && logitInputLengths->ews();
        is_supported = is_supported && checkLabelLength<int32_t>(*targetLabelLengths);
        return  is_supported;
    }

    PLATFORM_IMPL(ctc_loss_grad, ENGINE_CUDA) {
        auto targetLabels = INPUT_VARIABLE(0);
        auto logitInput = INPUT_VARIABLE(1);
        auto targetLabelLengths = INPUT_VARIABLE(2);
        auto logitInputLengths = INPUT_VARIABLE(3);
        auto outputGradients = OUTPUT_VARIABLE(0);
        auto context = block.launchContext();
        //in Cudnn Batch is in the middle dimension
        logitInput->permutei({1,0,2});
        outputGradients->permutei({1,0,2});
        //in Cudnn targets are concantenated instead of batched as matrix
        auto labels = getConcatTargets(*targetLabels, *targetLabelLengths);
        const int32_t * ldata= labels.data();
        auto tempLosses = NDArrayFactory::create<float>('c', {logitInputLengths->sizeAt(0)});
        auto err = cudnnCtcLoss(*context, *logitInput, ldata, *logitInputLengths, *targetLabelLengths, tempLosses, *outputGradients);
        if(err!=HIPDNN_STATUS_SUCCESS) throw sd::cuda_exception::build("ctc_loss CUDNN call failure ", err);
        //restore grads shape from {T, BATCH, C} -> {BATCHS, T, C}
        outputGradients->permutei({1,0,2});
        //tempLosses.printIndexedBuffer("tempLosses");
        return Status::OK();
    }

    PLATFORM_CHECK(ctc_loss_grad, ENGINE_CUDA) {
        auto targetLabels = INPUT_VARIABLE(0);
        auto logitInput = INPUT_VARIABLE(1);
        auto targetLabelLengths = INPUT_VARIABLE(2);
        auto logitInputLengths = INPUT_VARIABLE(3);
        auto outputGrads = OUTPUT_VARIABLE(0);
        int blankIndex = INT_ARG(0);

        auto dTypeInput = logitInput->dataType();
        auto intType = targetLabelLengths->dataType();
        auto dTypeOutput = outputGrads->dataType();

        bool is_supported = blankIndex==0 && intType == DataType::INT32  && dTypeInput == DataType::FLOAT32;
        is_supported = is_supported && outputGrads->ews() && targetLabelLengths->ews() && targetLabels->ews() && logitInputLengths->ews();
        is_supported = is_supported && checkLabelLength<int32_t>(*targetLabelLengths);
        return  is_supported;
    }

}
}
}