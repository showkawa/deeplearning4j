/*******************************************************************************
 * Copyright (c) 2019 Konduit K.K.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
// @author Yurii Shyrma (iuriish@yahoo.com)
//


#include "cudnnUtils.h"
#include <ops/declarable/helpers/convolutions.h>

namespace sd      {
namespace ops       {
namespace platforms {

//////////////////////////////////////////////////////////////////////////
static void conv3dCUDNN(const LaunchContext* context,
                        const NDArray* input, const NDArray* weights, const NDArray* bias, NDArray* output,
                        const int kD, const int kH, const int kW,
                        const int sD, const int sH, const int sW,
                        const int pD, const int pH, const int pW,
                        const int dD, const int dH, const int dW,
                        const int paddingMode, const bool isNCDHW, const int wFormat) {

    // cudnn support only one format for weights {oC,iC,kD,kH,kW}

    const int numDims = 5;

    int bS, iC, iD, iH, iW, oC, oD, oH, oW;                     // batch size, input channels, input depth/height/width, output channels, output depth/height/width;
    int indIOioC, indIOioD, indWoC, indWiC, indWkD;             // corresponding indexes
    ConvolutionUtils::getSizesAndIndexesConv3d(isNCDHW, wFormat, *input, *output, bS, iC, iD, iH, iW, oC, oD, oH, oW, indIOioC, indIOioD, indWiC, indWoC, indWkD);

    auto handle = reinterpret_cast<hipdnnHandle_t *>(context->getCuDnnHandle());
    hipdnnStatus_t err = hipdnnSetStream(*handle, *context->getCudaStream());
    if (err != 0) throw sd::cuda_exception::build("conv3dCUDNN: can't set stream for cuDNN", err);

    const std::vector<int> pads        = {pD, pH, pW};
    const std::vector<int> filtStrides = {sD, sH, sW};
    const std::vector<int> dilations   = {dD, dH, dW};

    const std::vector<int> xShape   = {bS, iC, iD, iH, iW};
    const std::vector<int> zShape   = {bS, oC, oD, oH, oW};
    const std::vector<int> wShape   = {oC, iC, kD, kH, kW};
    const std::vector<int> bShape   = {1, oC, 1, 1, 1};         // {1, (isNCDHW ? oC : 1), 1, 1, (isNCDHW ? 1 : oC)};

    const std::vector<int> xStrides = {(int)input->strideAt(0), (int)input->strideAt(1), (int)input->strideAt(2), (int)input->strideAt(3), (int)input->strideAt(4)};
    const std::vector<int> zStrides = {(int)output->strideAt(0), (int)output->strideAt(1), (int)output->strideAt(2), (int)output->strideAt(3), (int)output->strideAt(4)};

    hipdnnTensorFormat_t format = isNCDHW ? HIPDNN_TENSOR_NCHW : HIPDNN_TENSOR_NHWC;

    // input descriptor
    hipdnnTensorDescriptor_t x;
    hipdnnCreateTensorDescriptor(&x);
    if(input->ews() == 1)
        err = cudnnSetTensorNdDescriptorEx(x, format, cudnnDataType(input->dataType()), numDims, xShape.data());
    else
        err = hipdnnSetTensorNdDescriptor(x, cudnnDataType(input->dataType()), numDims, xShape.data(), xStrides.data());
    if (err != 0) throw sd::cuda_exception::build("conv3dCUDNN: hipdnnSetTensorNdDescriptor/cudnnSetTensorNdDescriptorEx for input failed", err);

    // weights descriptor
    hipdnnFilterDescriptor_t w;
    hipdnnCreateFilterDescriptor(&w);
    err = hipdnnSetFilterNdDescriptor(w, cudnnDataType(weights->dataType()), HIPDNN_TENSOR_NCHW, numDims, wShape.data());
    if(err != 0) throw sd::cuda_exception::build("conv3dCUDNN: hipdnnSetFilterNdDescriptor failed", err);

    // output descriptor
    hipdnnTensorDescriptor_t z;
    hipdnnCreateTensorDescriptor(&z);
    if(output->ews() == 1)
        err = cudnnSetTensorNdDescriptorEx(z, format, cudnnDataType(output->dataType()), numDims, zShape.data());
    else
        err = hipdnnSetTensorNdDescriptor(z, cudnnDataType(output->dataType()), numDims, zShape.data(), zStrides.data());
    if (err != 0) throw sd::cuda_exception::build("conv3dCUDNN: hipdnnSetTensorNdDescriptor/cudnnSetTensorNdDescriptorEx for output failed", err);

    // description of convolution
    hipdnnConvolutionDescriptor_t conv;
    hipdnnCreateConvolutionDescriptor(&conv);
    err = hipdnnSetConvolutionNdDescriptor(conv, numDims-2, pads.data(), filtStrides.data(), dilations.data(), HIPDNN_CROSS_CORRELATION, cudnnDataType(output->dataType()));
    if (err != 0) throw sd::cuda_exception::build("conv3dCUDNN: hipdnnSetConvolutionNdDescriptor failed", err);

    // algorithm description
    hipdnnConvolutionFwdAlgo_t algo;
    hipdnnConvolutionFwdAlgoPerf_t algoPerf;
    int count = 0;
    //err = hipdnnGetConvolutionForwardAlgorithm(*handle, x, w, conv, z, HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &algo);
    err = hipdnnFindConvolutionForwardAlgorithm(*handle, x, w, conv, z, 1, &count, &algoPerf);
    if (err != 0 || count == 0) throw sd::cuda_exception::build("conv3dCUDNN: hipdnnGetConvolutionForwardAlgorithm failed", err);
    algo = algoPerf.algo;


    // allocate auxiliary device memory, abbreviation ws means workspace
    size_t wsSize;
    err = hipdnnGetConvolutionForwardWorkspaceSize(*handle, x, w, conv, z, algo, &wsSize);
    if (err != 0) throw sd::cuda_exception::build("conv3dCUDNN: hipdnnGetConvolutionForwardWorkspaceSize failed", err);
    void* wsData;
    auto cudaErr = hipMalloc(&wsData, wsSize);
    if (cudaErr != 0) throw sd::cuda_exception::build("conv3dCUDNN: hipMalloc for auxiliary workspace memory failed", cudaErr);

    // provide scaling parameters
    const float  alpha32(1), beta32(0);
    const double alpha64(1), beta64(0);
    const void* alpha = output->sizeOfT() <= 4 ? reinterpret_cast<const void*>(&alpha32) : reinterpret_cast<const void*>(&alpha64);
    const void* beta  = output->sizeOfT() <= 4 ? reinterpret_cast<const void*>(&beta32)  : reinterpret_cast<const void*>(&beta64);

    NDArray::prepareSpecialUse({output}, {input, weights, bias});

    // run calculation
    err = hipdnnConvolutionForward(*handle, alpha, x, input->specialBuffer(), w, weights->specialBuffer(), conv, algo, wsData, wsSize, beta, z, output->specialBuffer());
    if (err != 0) throw sd::cuda_exception::build("conv3dCUDNN: hipdnnConvolutionForward failed", err);

    // add bias if it is present
    if (bias != nullptr) {

        hipdnnTensorDescriptor_t b;
        hipdnnCreateTensorDescriptor(&b);
        err = cudnnSetTensorNdDescriptorEx(b, /*format*/HIPDNN_TENSOR_NCHW, cudnnDataType(bias->dataType()), numDims, bShape.data());
        if (err != 0) throw sd::cuda_exception::build("conv3dCUDNN: hipdnnSetTensorNdDescriptor for bias failed", err);
        err = hipdnnAddTensor(*handle, alpha, b, bias->specialBuffer(), alpha, z, output->specialBuffer());
        if (err != 0) throw sd::cuda_exception::build("conv3dCUDNN: hipdnnAddTensor bias failed", err);
    }

    // cudaErr = hipStreamSynchronize(*context->getCudaStream());
    // if (cudaErr != 0)
    //     throw cuda_exception::build("conv3dCUDNN: hipStreamSynchronize failed !", cudaErr);

    cudaErr = hipFree(wsData);
    if (cudaErr != 0) throw sd::cuda_exception::build("conv3dCUDNN: hipFree for auxiliary workspace memory failed", cudaErr);

    NDArray::registerSpecialUse({output}, {input, weights, bias});
}

//////////////////////////////////////////////////////////////////////////
static void conv3dBpCUDNN(const LaunchContext* context,
                          const NDArray* input, const NDArray* weights, const NDArray* gradO,
                          NDArray* gradI, NDArray* gradW, NDArray* gradB,
                          const int kD, const int kH, const int kW,
                          const int sD, const int sH, const int sW,
                          const int pD, const int pH, const int pW,
                          const int dD, const int dH, const int dW,
                          const int paddingMode, const bool isNCDHW, const int wFormat) {

    // cudnn supports only two formats {oC,iC,kD,kH,kW} and {oC,kD,kH,kW,iC} for weights/gradW

    const int numDims = 5;

    int bS, iC, iD, iH, iW, oC, oD, oH, oW;                     // batch size, input channels, input depth/height/width, output channels, output depth/height/width;
    int indIOioC, indIOioD, indWoC, indWiC, indWkD;             // corresponding indexes
    ConvolutionUtils::getSizesAndIndexesConv3d(isNCDHW, wFormat, *input, *gradO, bS, iC, iD, iH, iW, oC, oD, oH, oW, indIOioC, indIOioD, indWiC, indWoC, indWkD);

    auto handle = reinterpret_cast<hipdnnHandle_t *>(context->getCuDnnHandle());
    hipdnnStatus_t err = hipdnnSetStream(*handle, *context->getCudaStream());
    if (err != 0) throw sd::cuda_exception::build("conv3dBpCUDNN: can't set stream for cuDNN", err);

    const std::vector<int> pads        = {pD, pH, pW};
    const std::vector<int> filtStrides = {sD, sH, sW};
    const std::vector<int> dilations   = {dD, dH, dW};

    const std::vector<int> xShape  = {bS, iC, iD, iH, iW};
    const std::vector<int> dzShape = {bS, oC, oD, oH, oW};
    const std::vector<int> wShape  = {oC, iC, kD, kH, kW};
    const std::vector<int> dbShape = {1, (int)(isNCDHW ? oC : 1), 1, 1, (int)(isNCDHW ? 1 : oC)};

    const std::vector<int> xStrides  = {(int)input->strideAt(0), (int)input->strideAt(1), (int)input->strideAt(2), (int)input->strideAt(3), (int)input->strideAt(4)};
    const std::vector<int> dxStrides = {(int)gradI->strideAt(0), (int)gradI->strideAt(1), (int)gradI->strideAt(2), (int)gradI->strideAt(3), (int)gradI->strideAt(4)};
    const std::vector<int> dzStrides = {(int)gradO->strideAt(0), (int)gradO->strideAt(1), (int)gradO->strideAt(2), (int)gradO->strideAt(3), (int)gradO->strideAt(4)};

    hipdnnTensorFormat_t format = isNCDHW ? HIPDNN_TENSOR_NCHW : HIPDNN_TENSOR_NHWC;
    hipdnnTensorFormat_t formatW = 0 == wFormat ? format : (1 == wFormat ? HIPDNN_TENSOR_NCHW : HIPDNN_TENSOR_NHWC);

    // input descriptor
    hipdnnTensorDescriptor_t x;
    hipdnnCreateTensorDescriptor(&x);
    if(input->ews() == 1)
        err = cudnnSetTensorNdDescriptorEx(x, format, cudnnDataType(input->dataType()), numDims, xShape.data());
    else
        err = hipdnnSetTensorNdDescriptor(x, cudnnDataType(input->dataType()), numDims, xShape.data(), xStrides.data());
    if (err != 0) throw sd::cuda_exception::build("conv3dBpCUDNN: hipdnnSetTensorNdDescriptor/cudnnSetTensorNdDescriptorEx for input failed", err);

    // gradO descriptor
    hipdnnTensorDescriptor_t dz;
    hipdnnCreateTensorDescriptor(&dz);
    if(gradO->ews() == 1)
        err = cudnnSetTensorNdDescriptorEx(dz, format, cudnnDataType(gradO->dataType()), numDims, dzShape.data());
    else
        err = hipdnnSetTensorNdDescriptor(dz, cudnnDataType(gradO->dataType()), numDims, dzShape.data(), dzStrides.data());
    if (err != 0) throw sd::cuda_exception::build("conv3dBpCUDNN: hipdnnSetTensorNdDescriptor/cudnnSetTensorNdDescriptorEx for gradO failed", err);

    // gradI descriptor
    hipdnnTensorDescriptor_t dx;
    hipdnnCreateTensorDescriptor(&dx);
    if(gradI->ews() == 1)
        err = cudnnSetTensorNdDescriptorEx(dx, format, cudnnDataType(gradI->dataType()), numDims, xShape.data());
    else
        err = hipdnnSetTensorNdDescriptor(dx, cudnnDataType(gradI->dataType()), numDims, xShape.data(), dxStrides.data());
    if (err != 0) throw sd::cuda_exception::build("conv3dBpCUDNN: hipdnnSetTensorNdDescriptor/cudnnSetTensorNdDescriptorEx for gradI failed", err);

    // gradW descriptor
    hipdnnFilterDescriptor_t dw;
    hipdnnCreateFilterDescriptor(&dw);
    err = hipdnnSetFilterNdDescriptor(dw, cudnnDataType(gradW->dataType()), formatW, numDims, wShape.data());
    if(err != 0) throw sd::cuda_exception::build("conv3dBpCUDNN: hipdnnSetFilterNdDescriptor failed", err);

    // description of convolution
    hipdnnConvolutionDescriptor_t conv;
    hipdnnCreateConvolutionDescriptor(&conv);
    err = hipdnnSetConvolutionNdDescriptor(conv, numDims-2, pads.data(), filtStrides.data(), dilations.data(), HIPDNN_CROSS_CORRELATION, cudnnDataType(gradO->dataType()));
    if (err != 0) throw sd::cuda_exception::build("conv3dBpCUDNN: hipdnnSetConvolutionNdDescriptor failed", err);

    // gradW algorithm description
    hipdnnConvolutionBwdFilterAlgo_t algoGradW;
    hipdnnConvolutionBwdFilterAlgoPerf_t algoGradWPerf;
    int count = 0;
    //err = hipdnnGetConvolutionBackwardFilterAlgorithm(*handle, x, dz, conv, dw, HIPDNN_CONVOLUTION_BWD_FILTER_PREFER_FASTEST, 0, &algoGradW);
    err = hipdnnFindConvolutionBackwardFilterAlgorithm(*handle, x, dz, conv, dw, 1, &count, &algoGradWPerf);
    if (err != 0 || count == 0) throw sd::cuda_exception::build("conv3dBpCUDNN: hipdnnGetConvolutionBackwardFilterAlgorithm failed", err);
    algoGradW = algoGradWPerf.algo;

    // gradI algorithm description
    hipdnnConvolutionBwdDataAlgo_t algoGradI;
    hipdnnConvolutionBwdDataAlgoPerf_t algoGradIPerf;
    //err = hipdnnGetConvolutionBackwardDataAlgorithm(*handle, dw, dz, conv, x, HIPDNN_CONVOLUTION_BWD_DATA_PREFER_FASTEST, 0, &algoGradI);
    err = hipdnnFindConvolutionBackwardDataAlgorithm(*handle, dw, dz, conv, x, 1, &count, &algoGradIPerf);
    if (err != 0 || count == 0) throw sd::cuda_exception::build("conv3dBpCUDNN: hipdnnGetConvolutionBackwardDataAlgorithm failed", err);
    algoGradI = algoGradIPerf.algo;

    // allocate auxiliary device memory for gradW calculation, abbreviation ws means workspace
    size_t wsGradWSize;
    err = hipdnnGetConvolutionBackwardFilterWorkspaceSize(*handle, x, dz, conv, dw, algoGradW, &wsGradWSize);
    if (err != 0) throw sd::cuda_exception::build("conv3dBpCUDNN: hipdnnGetConvolutionBackwardFilterWorkspaceSize failed", err);
    void* wsGradWData;
    auto cudaErr = hipMalloc(&wsGradWData, wsGradWSize);
    if (cudaErr != 0) throw sd::cuda_exception::build("conv3dBpCUDNN: hipMalloc for auxiliary workspace memory wsGradWData failed", cudaErr);

    // allocate auxiliary device memory for gradI calculation, abbreviation ws means workspace
    size_t wsGradISize;
    err = hipdnnGetConvolutionBackwardDataWorkspaceSize(*handle, dw, dz, conv, dx, algoGradI, &wsGradISize);
    if (err != 0) throw sd::cuda_exception::build("conv3dBpCUDNN: hipdnnGetConvolutionBackwardDataWorkspaceSize failed", err);
    void* wsGradIData;
    cudaErr = hipMalloc(&wsGradIData, wsGradISize);
    if (cudaErr != 0) throw sd::cuda_exception::build("conv3dBpCUDNN: hipMalloc for auxiliary workspace memory wsGradIData failed", cudaErr);

    // provide scaling parameters
    const float  alpha32(1), beta32(0);
    const double alpha64(1), beta64(0);
    const void* alpha = gradO->sizeOfT() <= 4 ? reinterpret_cast<const void*>(&alpha32) : reinterpret_cast<const void*>(&alpha64);
    const void* beta  = gradO->sizeOfT() <= 4 ? reinterpret_cast<const void*>(&beta32)  : reinterpret_cast<const void*>(&beta64);

    NDArray::prepareSpecialUse({gradI, gradW, gradB}, {input, weights, gradO});

    // run calculation for gradB (if not nullptr)
    if(gradB != nullptr) {

        hipdnnTensorDescriptor_t db;
        hipdnnCreateTensorDescriptor(&db);
        err = cudnnSetTensorNdDescriptorEx(db, format, cudnnDataType(gradB->dataType()), numDims, dbShape.data());
        if (err != 0) throw sd::cuda_exception::build("conv3dBpCUDNN: hipdnnSetTensorNdDescriptor for gradB failed", err);

        err = hipdnnConvolutionBackwardBias(*handle, alpha, dz, gradO->specialBuffer(), beta, db, gradB->specialBuffer());
        if (err != 0) throw sd::cuda_exception::build("conv3dBpCUDNN: hipdnnConvolutionBackwardBias failed", err);
    }

    // run calculation for gradW
    err = hipdnnConvolutionBackwardFilter(*handle, alpha, x, input->specialBuffer(), dz, gradO->specialBuffer(), conv, algoGradW, wsGradWData, wsGradWSize, beta, dw, gradW->specialBuffer());
    if (err != 0) throw sd::cuda_exception::build("conv3dBpCUDNN: hipdnnConvolutionBackwardFilter failed", err);

    // run calculation for gradI
    err = hipdnnConvolutionBackwardData(*handle, alpha, dw, weights->specialBuffer(), dz, gradO->specialBuffer(), conv, algoGradI, wsGradIData, wsGradISize, beta, dx, gradI->specialBuffer());
    if (err != 0) throw sd::cuda_exception::build("conv3dBpCUDNN: hipdnnConvolutionBackwardData failed", err);

    // cudaErr = hipStreamSynchronize(*context->getCudaStream());
    // if (cudaErr != 0)
    //     throw cuda_exception::build("conv3dBpCUDNN: hipStreamSynchronize failed !", cudaErr);

    cudaErr = hipFree(wsGradWData);
    if (cudaErr != 0) throw sd::cuda_exception::build("conv3dBpCUDNN: hipFree for auxiliary workspace memory wsGradWData failed", cudaErr);
    cudaErr = hipFree(wsGradIData);
    if (cudaErr != 0) throw sd::cuda_exception::build("conv3dBpCUDNN: hipFree for auxiliary workspace memory wsGradIData failed", cudaErr);

    NDArray::registerSpecialUse({gradI, gradW, gradB}, {input, weights, gradO});
}

//////////////////////////////////////////////////////////////////////////
PLATFORM_IMPL(conv3dnew, ENGINE_CUDA) {

    auto input   = INPUT_VARIABLE(0);                                    // [bS, iD, iH, iW, iC] (NDHWC) or [bS, iC, iD, iH, iW] (NCDHW)
    auto weights = INPUT_VARIABLE(1);                                    // [kD, kH, kW, iC, oC], [oC, iC, kD, kH, kW], [oC, kD, kH, kW, iC]
    auto bias    = block.width() > 2 ? INPUT_VARIABLE(2) : nullptr;      // [oC]
    auto output  = OUTPUT_VARIABLE(0);                                   // [bS, oD, oH, oW, oC] (NDHWC) or [bS, oC, oD, oH, oW] (NCDHW)

    REQUIRE_TRUE(input->rankOf()   == 5, 0, "CONV3D CUDNN OP: rank of input array must be equal to 5, but got %i instead !", input->rankOf());
    REQUIRE_TRUE(weights->rankOf() == 5, 0, "CONV3D CUDNN OP: rank of weights array must be equal to 5, but got %i instead !", weights->rankOf());

    int kD = INT_ARG(0) > 0 ? INT_ARG(0) : static_cast<int>(weights->sizeAt(0));// filter(kernel) depth
    int kH = INT_ARG(1) > 0 ? INT_ARG(1) : static_cast<int>(weights->sizeAt(1));// filter(kernel) height
    int kW = INT_ARG(2) > 0 ? INT_ARG(2) : static_cast<int>(weights->sizeAt(2));// filter(kernel) width
    int sD = INT_ARG(3);                                                        // strides depth
    int sH = INT_ARG(4);                                                        // strides height
    int sW = INT_ARG(5);                                                        // strides width
    int pD = INT_ARG(6);                                                        // paddings depth
    int pH = INT_ARG(7);                                                        // paddings height
    int pW = INT_ARG(8);                                                        // paddings width
    int dD = INT_ARG(9);                                                        // dilations depth
    int dH = INT_ARG(10);                                                       // dilations height
    int dW = INT_ARG(11);                                                       // dilations width
    int paddingMode = INT_ARG(12);                                              // 0-SAME,  1-VALID
    int isNCDHW  = block.getIArguments()->size() > 13 ? !INT_ARG(13) : 1;       // INT_ARG(13): 1-NDHWC, 0-NCDHW
    int wFormat = block.getIArguments()->size() > 14 ? INT_ARG(14) : 0;         // 0-[kD, kH, kW, iC, oC], 1-[oC, iC, kD, kH, kW], 2-[oC, kD, kH, kW, iC]

    REQUIRE_TRUE(paddingMode < 2, 0, "CONV3D CUDNN OP: causal padding mode (paddingMode = 2) is not allowed for this operation !");

    int bS, iC, iD, iH, iW, oC, oD, oH, oW;                     // batch size, input channels, input depth/height/width, output channels, output depth/height/width;
    int indIOioC, indIOioD, indWoC, indWiC, indWkD;             // corresponding indexes
    ConvolutionUtils::getSizesAndIndexesConv3d(isNCDHW, wFormat, *input, *output, bS, iC, iD, iH, iW, oC, oD, oH, oW, indIOioC, indIOioD, indWiC, indWoC, indWkD);

    ConvolutionUtils::calcPadding3D(pD, pH, pW, oD, oH, oW, iD, iH, iW, kD, kH, kW, sD, sH, sW, dD, dH, dW, paddingMode);

    std::vector<Nd4jLong> expectedWeightsShape = ConvolutionUtils::expectWeightsShape(wFormat, kD, kH, kW, iC, oC);
    REQUIRE_TRUE(weights->isSameShape(expectedWeightsShape), 0, "CONV3D CUDNN OP: wrong shape of weights array, expected is %s, but got %s instead !", ShapeUtils::shapeAsString(expectedWeightsShape).c_str(), ShapeUtils::shapeAsString(weights).c_str());
    if (bias)
        REQUIRE_TRUE(bias->rankOf() <= 2 && oC == bias->lengthOf(), 0, "CONV3D CUDNN OP: wrong shape of array with biases, expected rank, length: <=2, %i, but got %i, %i instead !", oC, bias->rankOf(), bias->lengthOf());

    NDArray* newWeights = weights; // cudnn support only one format {oC,iC,kD,kH,kW}
    if(1 != wFormat) {
        newWeights = new NDArray(weights->ordering(), {oC, iC, kD, kH, kW}, weights->dataType(), weights->getContext());
        newWeights->assign(weights->permute(0 == wFormat ? std::vector<int>({4,3,0,1,2}) : std::vector<int>({0,4,1,2,3})));  // kD, kH, kW, iC, oC  --> oC, iC, kD, kH, kW   or oC, kD, kH, kW, iC  --> oC, iC, kD, kH, kW
    }

    NDArray* newInput = input;
    NDArray* newGradI = nullptr;
    if(paddingMode == 1) // in same paddingMode cudnn doesn't support asymmetric left/right top/bottopm paddings
        checkConv3dCUDNNPadAsymmetric(newInput, newGradI, iD, iH, iW, oD, oH, oW, kD, kH, kW, sD, sH, sW, pD, pH, pW, dD, dH, dW, isNCDHW);

    conv3dCUDNN(block.launchContext(), newInput, newWeights, bias, output, kD,kH,kW,sD,sH,sW,pD,pH,pW,dD,dH,dW, paddingMode, isNCDHW, wFormat);

    if(newInput != input)
        delete newInput;

    if(1 != wFormat)
        delete newWeights;

    return Status::OK();
}

//////////////////////////////////////////////////////////////////////////
PLATFORM_CHECK(conv3dnew, ENGINE_CUDA) {

    auto input   = INPUT_VARIABLE(0);                                    // [bS, iD, iH, iW, iC] (NDHWC) or [bS, iC, iD, iH, iW] (NCDHW)
    auto weights = INPUT_VARIABLE(1);                                    // [kD, kH, kW, iC, oC], [oC, iC, kD, kH, kW], [oC, kD, kH, kW, iC]
    auto bias    = block.width() > 2 ? INPUT_VARIABLE(2) : nullptr;      // [oC]

    int paddingMode = INT_ARG(12);                                       // 0-SAME,  1-VALID

    const bool badInputType   = input->dataType()   != DataType::DOUBLE && input->dataType()   != DataType::FLOAT32 && input->dataType()   != DataType::HALF;
    const bool badWeightsType = weights->dataType() != DataType::DOUBLE && weights->dataType() != DataType::FLOAT32 && weights->dataType() != DataType::HALF;
    const bool badBiasType    = bias == nullptr ? false : (bias->dataType() != DataType::DOUBLE && bias->dataType() != DataType::FLOAT32 && bias->dataType() != DataType::HALF);

    return paddingMode != 2 && !badInputType && !badWeightsType && !badBiasType;
}

//////////////////////////////////////////////////////////////////////////
PLATFORM_IMPL(conv3dnew_bp, ENGINE_CUDA) {

    auto input   = INPUT_VARIABLE(0);                                                // [bS, iD, iH, iW, iC] (NDHWC) or [bS, iC, iD, iH, iW] (NCDHW)
    auto weights = INPUT_VARIABLE(1);                                                // [kD, kH, kW, iC, oC], [oC, iC, kD, kH, kW], [oC, kD, kH, kW, iC]
    auto bias    = block.width() > 3 ? INPUT_VARIABLE(2) : nullptr;                  // [oC]
    auto gradO   = block.width() > 3 ? INPUT_VARIABLE(3) : INPUT_VARIABLE(2);        // [bS, oD, oH, oW, oC] (NDHWC) or [bS, oC, oD, oH, oW] (NCDHW), epsilon_next

    auto gradI = OUTPUT_VARIABLE(0);                                                 // [bS, iD, iH, iW, iC] (NDHWC) or [bS, iC, iD, iH, iW] (NCDHW), epsilon
    auto gradW = OUTPUT_VARIABLE(1);                                                 // [kD, kH, kW, iC, oC], [oC, iC, kD, kH, kW], [oC, kD, kH, kW, iC]
    auto gradB = block.width() > 3 ? OUTPUT_VARIABLE(2) : nullptr;                   // [oC]

    REQUIRE_TRUE(input->rankOf()   == 5, 0, "CONV3D_BP CUDNN OP: rank of input array must be equal to 5, but got %i instead !", input->rankOf());
    REQUIRE_TRUE(weights->rankOf() == 5, 0, "CONV3D_BP CUDNN OP: rank of weights array must be equal to 5, but got %i instead !", weights->rankOf());
    REQUIRE_TRUE(gradO->rankOf() == 5, 0,   "CONV3D_BP CUDNN OP: rank of output gradients (next epsilon) array must be equal to 5, but got %i instead !", gradO->rankOf());

    int kD = INT_ARG(0) > 0 ? INT_ARG(0) : static_cast<int>(weights->sizeAt(0));// filter(kernel) depth
    int kH = INT_ARG(1) > 0 ? INT_ARG(1) : static_cast<int>(weights->sizeAt(1));// filter(kernel) height
    int kW = INT_ARG(2) > 0 ? INT_ARG(2) : static_cast<int>(weights->sizeAt(2));// filter(kernel) width
    int sD = INT_ARG(3);                                                        // strides depth
    int sH = INT_ARG(4);                                                        // strides height
    int sW = INT_ARG(5);                                                        // strides width
    int pD = INT_ARG(6);                                                        // paddings depth
    int pH = INT_ARG(7);                                                        // paddings height
    int pW = INT_ARG(8);                                                        // paddings width
    int dD = INT_ARG(9);                                                        // dilations depth
    int dH = INT_ARG(10);                                                       // dilations height
    int dW = INT_ARG(11);                                                       // dilations width
    int paddingMode = INT_ARG(12);                                              // 1-SAME,  0-VALID
    int isNCDHW  = block.getIArguments()->size() > 13 ? !INT_ARG(13) : 1;       // INT_ARG(13): 1-NDHWC, 0-NCDHW
    int wFormat = block.getIArguments()->size() > 14 ? INT_ARG(14) : 0;         // 0-[kD, kH, kW, iC, oC], 1-[oC, iC, kD, kH, kW], 2-[oC, kD, kH, kW, iC]

    int bS, iC, iD, iH, iW, oC, oD, oH, oW;                     // batch size, input channels, input depth/height/width, output channels, output depth/height/width;
    int indIOioC, indIOioD, indWoC, indWiC, indWkD;             // corresponding indexes
    ConvolutionUtils::getSizesAndIndexesConv3d(isNCDHW, wFormat, *input, *gradO, bS, iC, iD, iH, iW, oC, oD, oH, oW, indIOioC, indIOioD, indWiC, indWoC, indWkD);

    int trueoD, trueoH, trueoW;          // true output depth/height/width
    ConvolutionUtils::calcOutSizePool3D(trueoD, trueoH, trueoW, kD, kH, kW, sD, sH, sW, pD, pH, pW, dD, dH, dW, iD, iH, iW, paddingMode);

    REQUIRE_TRUE(paddingMode < 2, 0, "CONV3D_BP CUDNN OP: causal padding mode (paddingMode = 2) is not allowed for this operation !");

    std::vector<Nd4jLong> expectedGradOShape = ShapeUtils::composeShapeUsingDimsAndIdx({bS,oC,trueoD,trueoH,trueoW,  0,indIOioC,indIOioD,indIOioD+1,indIOioD+2});
    std::vector<Nd4jLong> expectedWeightsShape = ConvolutionUtils::expectWeightsShape(wFormat, kD, kH, kW, iC, oC);
    REQUIRE_TRUE(gradO->isSameShape(expectedGradOShape), 0,  "CONV3D_BP CUDNN OP: wrong shape of output gradients (next epsilon) array, expected is %s, but got %s instead !", ShapeUtils::shapeAsString(expectedGradOShape).c_str(), ShapeUtils::shapeAsString(gradO).c_str());
    REQUIRE_TRUE(gradW->isSameShape(expectedWeightsShape), 0, "CONV3D_BP CUDNN OP: wrong shape of weights array, expected is %s, but got %s instead !", ShapeUtils::shapeAsString(expectedWeightsShape).c_str(), ShapeUtils::shapeAsString(weights).c_str());
    if(bias)
        REQUIRE_TRUE(bias->rankOf() <= 2 && oC == bias->lengthOf(), 0, "CONV3D_BP CUDNN OP: wrong shape of array with biases, expected rank, length: <=2, %i, but got %i, %i instead !", oC, bias->rankOf(), bias->lengthOf());

    ConvolutionUtils::calcPadding3D(pD, pH, pW, oD, oH, oW, iD, iH, iW, kD, kH, kW, sD, sH, sW, dD, dH, dW, paddingMode);

    NDArray *newWeights = weights, *newGradW = gradW; // cudnn support only two formats {oC,iC,kD,kH,kW} and {oC,kD,kH,kW,iC}
    if(0 == wFormat) {
        newGradW   = new NDArray(gradW->ordering(),   isNCDHW ? std::vector<Nd4jLong>({oC, iC, kD, kH, kW}) : std::vector<Nd4jLong>({oC, kD, kH, kW, iC}), gradW->dataType(),   gradW->getContext());
        newWeights = new NDArray(weights->ordering(), isNCDHW ? std::vector<Nd4jLong>({oC, iC, kD, kH, kW}) : std::vector<Nd4jLong>({oC, kD, kH, kW, iC}), weights->dataType(), weights->getContext());
        newWeights->assign(weights->permute(isNCDHW ? std::vector<int>({4,3,0,1,2}) : std::vector<int>({4,0,1,2,3}))); // (kD, kH, kW, iC, oC  --> oC, iC, kD, kH, kW) or (kD, kH, kW, iC, oC  --> oC, kD, kH, kW, iC)
    }

    NDArray* newInput = input;
    NDArray* newGradI = gradI;
    if(paddingMode == 1) // in same paddingMode cudnn doesn't support asymmetric left/right top/bottopm paddings
        checkConv3dCUDNNPadAsymmetric(newInput, newGradI, iD, iH, iW, oD, oH, oW, kD, kH, kW, sD, sH, sW, pD, pH, pW, dD, dH, dW, isNCDHW);

    conv3dBpCUDNN(block.launchContext(), newInput, newWeights, gradO,   newGradI, newGradW, gradB, kD,kH,kW,sD,sH,sW,pD,pH,pW,dD,dH,dW,paddingMode,isNCDHW,wFormat);

    if(0 == wFormat) {
        newGradW->permutei(isNCDHW ? std::vector<int>({2,3,4,1,0}) : std::vector<int>({1,2,3,4,0})); // (oC, iC, kD, kH, kW --> kD, kH, kW, iC, oC) or (oC, kD, kH, kW, iC --> kD, kH, kW, iC, oC)
        gradW->assign(newGradW);
    }


    if(newInput != input) {

        if(isNCDHW)
            gradI->assign((*newGradI)({0,0,  0,0,  0,gradI->sizeAt(2),  0,gradI->sizeAt(3),  0,gradI->sizeAt(4)}));
        else
            gradI->assign((*newGradI)({0,0,  0,gradI->sizeAt(1),  0,gradI->sizeAt(2),  0,gradI->sizeAt(3),  0,0}));

        delete newInput;
        delete newGradI;
    }

    if(0 == wFormat) {
        delete newWeights;
        delete newGradW;
    }

    return Status::OK();
}

PLATFORM_CHECK(conv3dnew_bp, ENGINE_CUDA) {

    auto input   = INPUT_VARIABLE(0);                                                // [bS, iD, iH, iW, iC] (NDHWC) or [bS, iC, iD, iH, iW] (NCDHW)
    auto weights = INPUT_VARIABLE(1);                                                // [kD, kH, kW, iC, oC], [oC, iC, kD, kH, kW], [oC, kD, kH, kW, iC]
    auto bias    = block.width() > 3 ? INPUT_VARIABLE(2) : nullptr;                  // [oC]
    auto gradO   = block.width() > 3 ? INPUT_VARIABLE(3) : INPUT_VARIABLE(2);        // [bS, oD, oH, oW, oC] (NDHWC) or [bS, oC, oD, oH, oW] (NCDHW), epsilon_next

    int paddingMode = INT_ARG(12);                                              // 1-SAME,  0-VALID
    int isNCDHW  = block.getIArguments()->size() > 13 ? !INT_ARG(13) : 1;       // INT_ARG(13): 1-NDHWC, 0-NCDHW

    const bool badInputType   = input->dataType()   != DataType::DOUBLE && input->dataType()   != DataType::FLOAT32 && input->dataType()   != DataType::HALF;
    const bool badWeightsType = weights->dataType() != DataType::DOUBLE && weights->dataType() != DataType::FLOAT32 && weights->dataType() != DataType::HALF;
    const bool badGradOType   = gradO->dataType()   != DataType::DOUBLE && gradO->dataType()   != DataType::FLOAT32 && gradO->dataType()   != DataType::HALF;
    const bool badBiasType    = bias == nullptr ? false : (bias->dataType() != DataType::DOUBLE && bias->dataType() != DataType::FLOAT32 && bias->dataType() != DataType::HALF);

    return isNCDHW && paddingMode != 2 && !badInputType && !badWeightsType && !badGradOType && !badBiasType;
}

}
}
}
