/*******************************************************************************
 * Copyright (c) 2019 Konduit K.K.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
// @author Yurii Shyrma (iuriish@yahoo.com)
//


#include "cudnnUtils.h"
#include <ops/declarable/helpers/convolutions.h>

namespace sd      {
namespace ops       {
namespace platforms {

//////////////////////////////////////////////////////////////////////////
static void conv2dCUDNN(const LaunchContext* context,
                        const NDArray* input, const NDArray* weights, const NDArray* bias, NDArray* output,
                        const int kH, const int kW,
                        const int sH, const int sW,
                        const int pH, const int pW,
                        const int dH, const int dW,
                        const int paddingMode, const bool isNCHW, const int wFormat) {

    // cudnn support only two formats for weights {oC,iC,kH,kW} and {oC,kH,kW,iC}

    int bS, iC, iH, iW, oC, oH, oW;                             // batch size, input channels, input height/width, output channels, output height/width;
    int indIOioC, indIiH, indWoC, indWiC, indWkH, indOoH;       // corresponding indexes
    ConvolutionUtils::getSizesAndIndexesConv2d(isNCHW, wFormat, *input, *output, bS, iC, iH, iW, oC, oH, oW, indIOioC, indIiH, indWiC, indWoC, indWkH, indOoH);

    auto handle = reinterpret_cast<hipdnnHandle_t *>(context->getCuDnnHandle());
    hipdnnStatus_t err = hipdnnSetStream(*handle, *context->getCudaStream());
    if (err != 0) throw sd::cuda_exception::build("conv2dCUDNN: can't set stream for cuDNN", err);

    hipdnnTensorFormat_t format  = isNCHW ? HIPDNN_TENSOR_NCHW : HIPDNN_TENSOR_NHWC;
    hipdnnTensorFormat_t formatW = 0 == wFormat ? format : (1 == wFormat ? HIPDNN_TENSOR_NCHW : HIPDNN_TENSOR_NHWC);

    // input descriptor
    hipdnnTensorDescriptor_t x;
    hipdnnCreateTensorDescriptor(&x);
    if(input->ews() == 1 && input->ordering() == 'c')
        err = hipdnnSetTensor4dDescriptor(x, format, cudnnDataType(input->dataType()), bS, iC, iH, iW);
    else
        err = hipdnnSetTensor4dDescriptorEx(x, cudnnDataType(input->dataType()), bS, iC, iH, iW, input->strideAt(0), input->strideAt(indIOioC), input->strideAt(indIiH), input->strideAt(indIiH + 1));
    if (err != 0) throw sd::cuda_exception::build("conv2dCUDNN: hipdnnSetTensor4dDescriptor/hipdnnSetTensor4dDescriptorEx for input failed", err);

    // weights descriptor
    hipdnnFilterDescriptor_t w;
    hipdnnCreateFilterDescriptor(&w);
    err = hipdnnSetFilter4dDescriptor(w, cudnnDataType(weights->dataType()), formatW, oC, iC, kH, kW);
    if(err != 0) throw sd::cuda_exception::build("conv2dCUDNN: hipdnnSetFilter4dDescriptor failed", err);

    // output descriptor
    hipdnnTensorDescriptor_t z;
    hipdnnCreateTensorDescriptor(&z);
    if(output->ews() == 1 && output->ordering() == 'c')
        err = hipdnnSetTensor4dDescriptor(z, format, cudnnDataType(output->dataType()), bS, oC, oH, oW);
    else
        err = hipdnnSetTensor4dDescriptorEx(z, cudnnDataType(output->dataType()), bS, oC, oH, oW, output->strideAt(0), output->strideAt(indIOioC), output->strideAt(indOoH), output->strideAt(indOoH + 1));
    if (err != 0) throw sd::cuda_exception::build("conv2dCUDNN: hipdnnSetTensor4dDescriptor/hipdnnSetTensor4dDescriptorEx for output failed", err);

    // description of convolution
    hipdnnConvolutionDescriptor_t conv;
    hipdnnCreateConvolutionDescriptor(&conv);
    err = hipdnnSetConvolution2dDescriptor(conv, pH, pW, sH, sW, dH, dW, HIPDNN_CROSS_CORRELATION, cudnnDataType(output->dataType()));
    if (err != 0) throw sd::cuda_exception::build("conv2dCUDNN: hipdnnSetConvolution2dDescriptor failed", err);

    // algorithm description
    hipdnnConvolutionFwdAlgo_t algo;
    hipdnnConvolutionFwdAlgoPerf_t algoPerf;
    int count = 0;
    //err = hipdnnGetConvolutionForwardAlgorithm(*handle, x, w, conv, z, HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &algo);
    err = hipdnnFindConvolutionForwardAlgorithm(*handle, x, w, conv, z, 1, &count, &algoPerf);
    if (err != 0 || count == 0) throw sd::cuda_exception::build("conv2dCUDNN: hipdnnGetConvolutionForwardAlgorithm failed", err);
    algo = algoPerf.algo;


    // allocate auxiliary device memory, abbreviation ws means workspace
    size_t wsSize;
    err = hipdnnGetConvolutionForwardWorkspaceSize(*handle, x, w, conv, z, algo, &wsSize);
    if (err != 0) throw sd::cuda_exception::build("conv2dCUDNN: hipdnnGetConvolutionForwardWorkspaceSize failed", err);
    void* wsData;
    auto cudaErr = hipMalloc(&wsData, wsSize);
    if (cudaErr != 0) throw sd::cuda_exception::build("conv2dCUDNN: hipMalloc for auxiliary workspace memory failed", cudaErr);

    // provide scaling parameters
    const float  alpha32(1), beta32(0);
    const double alpha64(1), beta64(0);
    const void* alpha = output->sizeOfT() <= 4 ? reinterpret_cast<const void*>(&alpha32) : reinterpret_cast<const void*>(&alpha64);
    const void* beta  = output->sizeOfT() <= 4 ? reinterpret_cast<const void*>(&beta32)  : reinterpret_cast<const void*>(&beta64);

    NDArray::prepareSpecialUse({output}, {input, weights, bias});

    // run calculation
    err = hipdnnConvolutionForward(*handle, alpha, x, input->specialBuffer(), w, weights->specialBuffer(), conv, algo, wsData, wsSize, beta, z, output->specialBuffer());
    if (err != 0) throw sd::cuda_exception::build("conv2dCUDNN: hipdnnConvolutionForward failed", err);

    // add bias if it is present
    if (bias != nullptr) {
        hipdnnTensorDescriptor_t b;
        hipdnnCreateTensorDescriptor(&b);
        // err = hipdnnSetTensor4dDescriptor(b, format, cudnnDataType(bias->dataType()), 1, isNCHW ? bias->lengthOf() : 1, 1, isNCHW ? 1: bias->lengthOf());
        err = hipdnnSetTensor4dDescriptor(b, HIPDNN_TENSOR_NCHW, cudnnDataType(bias->dataType()), 1, oC, 1, 1);
        if (err != 0) throw sd::cuda_exception::build("conv2dCUDNN: hipdnnSetTensor4dDescriptor for bias failed", err);
        err = hipdnnAddTensor(*handle, alpha, b, bias->specialBuffer(), alpha, z, output->specialBuffer());
        if (err != 0) throw sd::cuda_exception::build("conv2dCUDNN: hipdnnAddTensor bias failed", err);
    }

    // cudaErr = hipStreamSynchronize(*context->getCudaStream());
    // if (cudaErr != 0)
    //     throw cuda_exception::build("conv2dCUDNN: hipStreamSynchronize failed !", cudaErr);

    cudaErr = hipFree(wsData);
    if (cudaErr != 0) throw sd::cuda_exception::build("conv2dCUDNN: hipFree for auxiliary workspace memory failed", cudaErr);

    NDArray::registerSpecialUse({output}, {input, weights, bias});
}

//////////////////////////////////////////////////////////////////////////
static void conv2dBpCUDNN(const LaunchContext* context,
                          const NDArray* input, const NDArray* weights, const NDArray* gradO,
                          NDArray* gradI, NDArray* gradW, NDArray* gradB,
                          const int kH, const int kW,
                          const int sH, const int sW,
                          const int pH, const int pW,
                          const int dH, const int dW,
                          const int paddingMode, const bool isNCHW, const int wFormat) {

    int bS, iC, iH, iW, oC, oH, oW;                             // batch size, input channels, input height/width, output channels, output height/width;
    int indIOioC, indIiH, indWoC, indWiC, indWkH, indOoH;       // corresponding indexes
    ConvolutionUtils::getSizesAndIndexesConv2d(isNCHW, wFormat, *input, *gradO, bS, iC, iH, iW, oC, oH, oW, indIOioC, indIiH, indWiC, indWoC, indWkH, indOoH);

    auto handle = reinterpret_cast<hipdnnHandle_t *>(context->getCuDnnHandle());
    hipdnnStatus_t err = hipdnnSetStream(*handle, *context->getCudaStream());
    if (err != 0) throw sd::cuda_exception::build("conv2dBpCUDNN: can't set stream for cuDNN", err);

    hipdnnTensorFormat_t format  = isNCHW ? HIPDNN_TENSOR_NCHW : HIPDNN_TENSOR_NHWC;
    hipdnnTensorFormat_t formatW = 0 == wFormat ? format : (1 == wFormat ? HIPDNN_TENSOR_NCHW : HIPDNN_TENSOR_NHWC);

    // input descriptor
    hipdnnTensorDescriptor_t x;
    hipdnnCreateTensorDescriptor(&x);
    if(input->ews() == 1 && input->ordering() == 'c')
        err = hipdnnSetTensor4dDescriptor(x, format, cudnnDataType(input->dataType()), bS, iC, iH, iW);
    else
        err = hipdnnSetTensor4dDescriptorEx(x, cudnnDataType(input->dataType()), bS, iC, iH, iW, input->strideAt(0), input->strideAt(indIOioC), input->strideAt(indIiH), input->strideAt(indIiH + 1));
    if (err != 0) throw sd::cuda_exception::build("conv2dBpCUDNN: hipdnnSetTensor4dDescriptor/hipdnnSetTensor4dDescriptorEx for input failed", err);

    // gradO descriptor
    hipdnnTensorDescriptor_t dz;
    hipdnnCreateTensorDescriptor(&dz);
    if(gradO->ews() == 1 && gradO->ordering() == 'c')
        err = hipdnnSetTensor4dDescriptor(dz, format, cudnnDataType(gradO->dataType()), bS, oC, oH, oW);
    else
        err = hipdnnSetTensor4dDescriptorEx(dz, cudnnDataType(gradO->dataType()), bS, oC, oH, oW, gradO->strideAt(0), gradO->strideAt(indIOioC), gradO->strideAt(indOoH), gradO->strideAt(indOoH + 1));
    if (err != 0) throw sd::cuda_exception::build("conv2dBpCUDNN: hipdnnSetTensor4dDescriptor/hipdnnSetTensor4dDescriptorEx for gradO failed", err);

    // gradI descriptor
    hipdnnTensorDescriptor_t dx;
    hipdnnCreateTensorDescriptor(&dx);
    if(gradI->ews() == 1 && gradI->ordering() == 'c')
        err = hipdnnSetTensor4dDescriptor(dx, format, cudnnDataType(gradI->dataType()), bS, iC, iH, iW);
    else
        err = hipdnnSetTensor4dDescriptorEx(dx, cudnnDataType(gradI->dataType()), bS, iC, iH, iW, gradI->strideAt(0), gradI->strideAt(indIOioC), gradI->strideAt(indIiH), gradI->strideAt(indIiH + 1));
    if (err != 0) throw sd::cuda_exception::build("conv2dBpCUDNN: hipdnnSetTensor4dDescriptor/hipdnnSetTensor4dDescriptorEx for gradI failed", err);

    // gradW descriptor
    hipdnnFilterDescriptor_t dw;
    hipdnnCreateFilterDescriptor(&dw);
    err = hipdnnSetFilter4dDescriptor(dw, cudnnDataType(gradW->dataType()), formatW, oC, iC, kH, kW);
    if(err != 0) throw sd::cuda_exception::build("conv2dBpCUDNN: hipdnnSetFilter4dDescriptor gradW failed", err);

    // description of convolution
    hipdnnConvolutionDescriptor_t conv;
    hipdnnCreateConvolutionDescriptor(&conv);
    err = hipdnnSetConvolution2dDescriptor(conv, pH, pW, sH, sW, dH, dW, HIPDNN_CROSS_CORRELATION, cudnnDataType(gradO->dataType()));
    if (err != 0) throw sd::cuda_exception::build("conv2dBpCUDNN: hipdnnSetConvolution2dDescriptor failed", err);

    // gradW algorithm description
    hipdnnConvolutionBwdFilterAlgo_t algoGradW;
    hipdnnConvolutionBwdFilterAlgoPerf_t algoGradWPerf;
    int count = 0;
    //err = hipdnnGetConvolutionBackwardFilterAlgorithm(*handle, x, dz, conv, dw, HIPDNN_CONVOLUTION_BWD_FILTER_PREFER_FASTEST, 0, &algoGradW);
    err = hipdnnFindConvolutionBackwardFilterAlgorithm(*handle, x, dz, conv, dw, 1, &count, &algoGradWPerf);
    if (err != 0 || count == 0) throw sd::cuda_exception::build("conv2dBpCUDNN: hipdnnGetConvolutionBackwardFilterAlgorithm failed", err);
    algoGradW = algoGradWPerf.algo;

    // gradI algorithm description
    hipdnnConvolutionBwdDataAlgo_t algoGradI;
    hipdnnConvolutionBwdDataAlgoPerf_t algoGradIPerf;
    //err = hipdnnGetConvolutionBackwardDataAlgorithm(*handle, dw, dz, conv, x, HIPDNN_CONVOLUTION_BWD_DATA_PREFER_FASTEST, 0, &algoGradI);
    err = hipdnnFindConvolutionBackwardDataAlgorithm(*handle, dw, dz, conv, x, 1, &count, &algoGradIPerf);
    if (err != 0 || count == 0) throw sd::cuda_exception::build("conv2dBpCUDNN: hipdnnGetConvolutionBackwardDataAlgorithm failed", err);
    algoGradI = algoGradIPerf.algo;

    // allocate auxiliary device memory for gradW calculation, abbreviation ws means workspace
    size_t wsGradWSize;
    err = hipdnnGetConvolutionBackwardFilterWorkspaceSize(*handle, x, dz, conv, dw, algoGradW, &wsGradWSize);
    if (err != 0) throw sd::cuda_exception::build("conv2dBpCUDNN: hipdnnGetConvolutionBackwardFilterWorkspaceSize failed", err);
    void* wsGradWData;
    auto cudaErr = hipMalloc(&wsGradWData, wsGradWSize);
    if (cudaErr != 0) throw sd::cuda_exception::build("conv2dBpCUDNN: hipMalloc for auxiliary workspace memory wsGradWData failed", cudaErr);

    // allocate auxiliary device memory for gradI calculation, abbreviation ws means workspace
    size_t wsGradISize;
    err = hipdnnGetConvolutionBackwardDataWorkspaceSize(*handle, dw, dz, conv, dx, algoGradI, &wsGradISize);
    if (err != 0) throw sd::cuda_exception::build("conv2dBpCUDNN: hipdnnGetConvolutionBackwardDataWorkspaceSize failed", err);
    void* wsGradIData;
    cudaErr = hipMalloc(&wsGradIData, wsGradISize);
    if (cudaErr != 0) throw sd::cuda_exception::build("conv2dBpCUDNN: hipMalloc for auxiliary workspace memory wsGradIData failed", cudaErr);

    // provide scaling parameters
    const float  alpha32(1), beta32(0);
    const double alpha64(1), beta64(0);
    const void* alpha = gradO->sizeOfT() <= 4 ? reinterpret_cast<const void*>(&alpha32) : reinterpret_cast<const void*>(&alpha64);
    const void* beta  = gradO->sizeOfT() <= 4 ? reinterpret_cast<const void*>(&beta32)  : reinterpret_cast<const void*>(&beta64);

    NDArray::prepareSpecialUse({gradI, gradW, gradB}, {input, weights, gradO});

    // run calculation for gradB (if not nullptr)
    if(gradB != nullptr) {
        hipdnnTensorDescriptor_t db;
        hipdnnCreateTensorDescriptor(&db);
        // err = hipdnnSetTensor4dDescriptor(db, format, cudnnDataType(gradB->dataType()), 1, isNCHW ? gradB->lengthOf() : 1, 1, isNCHW ? 1: gradB->lengthOf());
        err = hipdnnSetTensor4dDescriptor(db, HIPDNN_TENSOR_NCHW, cudnnDataType(gradB->dataType()), 1, oC, 1, 1);
        if (err != 0) throw sd::cuda_exception::build("conv2dBpCUDNN: hipdnnSetTensor4dDescriptor for gradB failed", err);

        err = hipdnnConvolutionBackwardBias(*handle, alpha, dz, gradO->specialBuffer(), beta, db, gradB->specialBuffer());
        if (err != 0) throw sd::cuda_exception::build("conv2dBpCUDNN: hipdnnConvolutionBackwardBias failed", err);
    }

    // run calculation for gradW
    err = hipdnnConvolutionBackwardFilter(*handle, alpha, x, input->specialBuffer(), dz, gradO->specialBuffer(), conv, algoGradW, wsGradWData, wsGradWSize, beta, dw, gradW->specialBuffer());
    if (err != 0) throw sd::cuda_exception::build("conv2dBpCUDNN: hipdnnConvolutionBackwardFilter failed", err);

    // run calculation for gradI
    err = hipdnnConvolutionBackwardData(*handle, alpha, dw, weights->specialBuffer(), dz, gradO->specialBuffer(), conv, algoGradI, wsGradIData, wsGradISize, beta, dx, gradI->specialBuffer());
    if (err != 0) throw sd::cuda_exception::build("conv2dBpCUDNN: hipdnnConvolutionBackwardData failed", err);

    // cudaErr = hipStreamSynchronize(*context->getCudaStream());
    // if (cudaErr != 0)
    //     throw cuda_exception::build("conv2dBpCUDNN: hipStreamSynchronize failed !", cudaErr);

    cudaErr = hipFree(wsGradWData);
    if (cudaErr != 0) throw sd::cuda_exception::build("conv2dBpCUDNN: hipFree for auxiliary workspace memory wsGradWData failed", cudaErr);
    cudaErr = hipFree(wsGradIData);
    if (cudaErr != 0) throw sd::cuda_exception::build("conv2dBpCUDNN: hipFree for auxiliary workspace memory wsGradIData failed", cudaErr);

    NDArray::registerSpecialUse({gradI, gradW, gradB}, {input, weights, gradO});
}

//////////////////////////////////////////////////////////////////////////
PLATFORM_IMPL(conv2d, ENGINE_CUDA) {

    auto input   = INPUT_VARIABLE(0);                                    // [bS, iH, iW, iC] (NHWC) or [bS, iC, iH, iW] (NCHW)
    auto weights = INPUT_VARIABLE(1);                                    // [kH, kW, iC, oC], [oC, iC, kH, kW], [oC, kH, kW, iC]
    auto bias    = block.width() > 2 ? INPUT_VARIABLE(2) : nullptr;      // [oC]

    auto output  = OUTPUT_VARIABLE(0);                                   // [bS, oH, oW, oC] (NHWC) or [bS, oC, oH, oW] (NCHW)

    int sH = INT_ARG(2);                                                        // strides height
    int sW = INT_ARG(3);                                                        // strides width
    int pH = INT_ARG(4);                                                        // paddings height
    int pW = INT_ARG(5);                                                        // paddings width
    int dH = INT_ARG(6);                                                        // dilations height
    int dW = INT_ARG(7);                                                        // dilations width
    int paddingMode = INT_ARG(8);                                               // 0-VALID, 1-SAME
    bool isNCHW    = block.getIArguments()->size() > 9 ? !INT_ARG(9) : 1;       // INT_ARG(9): 0-NCHW, 1-NHWC
    int wFormat = block.getIArguments()->size() > 10 ? INT_ARG(10) : 0;         // 0 - [kH, kW, iC, oC], 1 - [oC, iC, kH, kW], 2 - [oC, kH, kW, iC]

    int kH = INT_ARG(0) > 0 ? INT_ARG(0) : static_cast<int>(weights->sizeAt(0)); // filter(kernel) height
    int kW = INT_ARG(1) > 0 ? INT_ARG(1) : static_cast<int>(weights->sizeAt(1)); // filter(kernel) width

    REQUIRE_TRUE(input->rankOf()   == 4, 0, "CUSTOM CONV2D CUDNN OP: rank of input array must be equal to 4, but got %i instead !", input->rankOf());
    REQUIRE_TRUE(weights->rankOf() == 4, 0, "CUSTOM CONV2D CUDNN OP: rank of weights array must be equal to 4, but got %i instead !", weights->rankOf());

    int bS, iC, iH, iW, oC, oH, oW;                             // batch size, input channels, input height/width, output channels, output height/width;
    int indIOioC, indIiH, indWoC, indWiC, indWkH, indOoH;       // corresponding indexes
    ConvolutionUtils::getSizesAndIndexesConv2d(isNCHW, wFormat, *input, *output, bS, iC, iH, iW, oC, oH, oW, indIOioC, indIiH, indWiC, indWoC, indWkH, indOoH);

    ConvolutionUtils::calcPadding2D(pH, pW, oH, oW, iH, iW, kH, kW, sH, sW, dH, dW, paddingMode);

    std::vector<Nd4jLong> expectedWeightsShape = ConvolutionUtils::expectWeightsShape(wFormat, kH, kW, iC, oC);
    REQUIRE_TRUE(weights->isSameShape(expectedWeightsShape), 0, "CUSTOM CONV2D CUDNN OP: wrong shape of weights array, expected is %s, but got %s instead !", ShapeUtils::shapeAsString(expectedWeightsShape).c_str(), ShapeUtils::shapeAsString(weights).c_str());
    if (bias) {
        REQUIRE_TRUE(bias->rankOf() <= 2 && oC == bias->lengthOf(), 0, "CUSTOM CONV2D CUDNN OP: wrong shape of array with biases, expected rank, length: <=2, %i, but got %i, %i instead !", oC, bias->rankOf(), bias->lengthOf());
        REQUIRE_TRUE((bias->rankOf() == 1 && bias->strideAt(0) == 1) || (bias->rankOf() == 2 && bias->sizeAt(0) == 1 && bias->strideAt(1) == 1) || (bias->rankOf() == 2 && bias->sizeAt(1) == 1 && bias->strideAt(0) == 1), 0, "CUSTOM CONV2D CUDNN OP: bias array should be contiguous in memory !");
    }

    NDArray* newWeights = weights; // cudnn support only two formats {oC,iC,kH,kW} and {oC,kH,kW,iC}
    if(0 == wFormat) {
        newWeights = new NDArray(weights->ordering(), isNCHW ? std::vector<Nd4jLong>({oC, iC, kH, kW}) : std::vector<Nd4jLong>({oC, kH, kW, iC}), weights->dataType(), weights->getContext());
        newWeights->assign(weights->permute(isNCHW ? std::vector<int>({3,2,0,1}) : std::vector<int>({3,0,1,2}))); // (kH, kW, iC, oC  --> oC, iC, kH, kW) or (kH, kW, iC, oC  --> oC, kH, kW, iC)
    }

    NDArray* newInput = input;
    NDArray* newGradI = nullptr;
    if(paddingMode == 1) // in same paddingMode cudnn doesn't support asymmetric left/right top/bottopm paddings
        checkConv2dCUDNNPadAsymmetric(newInput, newGradI, iH, iW, oH, oW, kH, kW, sH, sW, pH, pW, dH, dW, isNCHW);

    conv2dCUDNN(block.launchContext(), newInput, newWeights, bias, output, kH,kW,sH,sW,pH,pW,dH,dW, paddingMode, isNCHW, wFormat);

    if(newInput != input)
        delete newInput;

    if(0 == wFormat)
        delete newWeights;

    return Status::OK();
}

//////////////////////////////////////////////////////////////////////////
PLATFORM_CHECK(conv2d, ENGINE_CUDA) {

    auto input   = INPUT_VARIABLE(0);                                    // [bS, iH, iW, iC] (NHWC) or [bS, iC, iH, iW] (NCHW)
    auto weights = INPUT_VARIABLE(1);                                    // [kH, kW, iC, oC] always
    auto bias    = block.width() > 2 ? INPUT_VARIABLE(2) : nullptr;      // [oC]

    const int paddingMode = INT_ARG(8);                                  // 0-VALID, 1-SAME, 2-CAUSAL

    const bool badInputType   = input->dataType()   != DataType::DOUBLE && input->dataType()   != DataType::FLOAT32 && input->dataType()   != DataType::HALF;
    const bool badWeightsType = weights->dataType() != DataType::DOUBLE && weights->dataType() != DataType::FLOAT32 && weights->dataType() != DataType::HALF;
    const bool badBiasType    = bias == nullptr ? false : (bias->dataType() != DataType::DOUBLE && bias->dataType() != DataType::FLOAT32 && bias->dataType() != DataType::HALF);

    return paddingMode != 2 && !badInputType && !badWeightsType && !badBiasType;
}

//////////////////////////////////////////////////////////////////////////
PLATFORM_IMPL(conv2d_bp, ENGINE_CUDA) {

    auto input   = INPUT_VARIABLE(0);                                                // [bS, iH, iW, iC] (NHWC) or [bS, iC, iH, iW] (NCHW)
    auto weights = INPUT_VARIABLE(1);                                                // [kH, kW, iC, oC], [oC, iC, kH, kW], [oC, kH, kW, iC]
    auto bias    = block.width() > 3 ? INPUT_VARIABLE(2) : nullptr;                  // [oC]
    auto gradO   = block.width() > 3 ? INPUT_VARIABLE(3) : INPUT_VARIABLE(2);        // [bS, oH, oW, oC] (NHWC) or [bS, oC, oH, oW] (NCHW), epsilon_next

    auto gradI = OUTPUT_VARIABLE(0);                                                 // [bS, iH, iW, iC] (NHWC) or [bS, iC, iH, iW] (NCHW), epsilon
    auto gradW = OUTPUT_VARIABLE(1);                                                 // [kH, kW, iC, oC], [oC, iC, kH, kW], [oC, kH, kW, iC]
    auto gradB = block.width() > 3 ? OUTPUT_VARIABLE(2) : nullptr;                   // [oC]

    int kH = INT_ARG(0);                                                        // filter(kernel) height
    int kW = INT_ARG(1);                                                        // filter(kernel) width
    int sH = INT_ARG(2);                                                        // strides height
    int sW = INT_ARG(3);                                                        // strides width
    int pH = INT_ARG(4);                                                        // paddings height
    int pW = INT_ARG(5);                                                        // paddings width
    int dH = INT_ARG(6);                                                        // dilations height
    int dW = INT_ARG(7);                                                        // dilations width
    int paddingMode = INT_ARG(8);                                               // 0-VALID, 1-SAME
    int isNCHW  = block.getIArguments()->size() > 9 ? !INT_ARG(9) : 1;          // INT_ARG(9): 0-NCHW, 1-NHWC
    int wFormat = block.getIArguments()->size() > 10 ? INT_ARG(10) : 0;         // 0 - [kH, kW, iC, oC], 1 - [oC, iC, kH, kW], 2 - [oC, kH, kW, iC]

    REQUIRE_TRUE(input->rankOf()   == 4, 0, "CUSTOM CONV2D_BP CUDNN OP: rank of input array must be equal to 4, but got %i instead !", input->rankOf());
    REQUIRE_TRUE(weights->rankOf() == 4, 0, "CUSTOM CONV2D_BP CUDNN OP: rank of weights array must be equal to 4, but got %i instead !", weights->rankOf());
    REQUIRE_TRUE(gradO->rankOf() == 4, 0, "CUSTOM CONV2D_BP CUDNN OP: rank of output's gradients (next epsilon) array must be equal to 4, but got %i instead !", gradO->rankOf());

    int bS, iC, iH, iW, oC, oH, oW;                             // batch size, input channels, input height/width, output channels, output height/width;
    int indIOioC, indIiH, indWoC, indWiC, indWkH, indOoH;       // corresponding indexes
    ConvolutionUtils::getSizesAndIndexesConv2d(isNCHW, wFormat, *input, *gradO, bS, iC, iH, iW, oC, oH, oW, indIOioC, indIiH, indWiC, indWoC, indWkH, indOoH);

    int trueoH, trueoW;          // true output height, width
    ConvolutionUtils::calcOutSizePool2D(trueoH, trueoW, kH, kW, sH, sW, pH, pW, dH, dW, iH, iW, paddingMode);

    ConvolutionUtils::calcPadding2D(pH, pW, oH, oW, iH, iW, kH, kW, sH, sW, dH, dW, paddingMode);

    std::vector<Nd4jLong> expectedGradOShape = ShapeUtils::composeShapeUsingDimsAndIdx({bS,oC,trueoH,trueoW,  0,indIOioC,indOoH,indOoH+1});
    std::vector<Nd4jLong> expectedWeightsShape = ConvolutionUtils::expectWeightsShape(wFormat, kH, kW, iC, oC);
    REQUIRE_TRUE(gradO->isSameShape(expectedGradOShape), 0,  "CUSTOM CONV2D_BP CUDNN OP: wrong shape of output gradients (next epsilon) array, expected is %s, but got %s instead !", ShapeUtils::shapeAsString(expectedGradOShape).c_str(), ShapeUtils::shapeAsString(gradO).c_str());
    REQUIRE_TRUE(weights->isSameShape(expectedWeightsShape), 0, "CUSTOM CONV2D_BP CUDNN OP: wrong shape of weights array, expected is %s, but got %s instead !", ShapeUtils::shapeAsString(expectedWeightsShape).c_str(), ShapeUtils::shapeAsString(weights).c_str());
    if(bias)
        REQUIRE_TRUE(bias->rankOf() <= 2 && oC == bias->lengthOf(), 0, "CUSTOM CONV2D_BP CUDNN OP: wrong shape of array with biases, expected rank, length: <=2, %i, but got %i, %i instead !", oC, bias->rankOf(), bias->lengthOf());

    NDArray *newWeights = weights, *newGradW = gradW; // cudnn support only two formats {oC,iC,kH,kW} and {oC,kH,kW,iC}
    if(0 == wFormat) {
        newGradW   = new NDArray(gradW->ordering(),   isNCHW ? std::vector<Nd4jLong>({oC, iC, kH, kW}) : std::vector<Nd4jLong>({oC, kH, kW, iC}), gradW->dataType(),   gradW->getContext());
        newWeights = new NDArray(weights->ordering(), isNCHW ? std::vector<Nd4jLong>({oC, iC, kH, kW}) : std::vector<Nd4jLong>({oC, kH, kW, iC}), weights->dataType(), weights->getContext());
        newWeights->assign(weights->permute(isNCHW ? std::vector<int>({3,2,0,1}) : std::vector<int>({3,0,1,2}))); // (kH, kW, iC, oC  --> oC, iC, kH, kW) or (kH, kW, iC, oC  --> oC, kH, kW, iC)
    }

    NDArray* newInput = input;
    NDArray* newGradI = gradI;
    if(paddingMode == 1) // in same paddingMode cudnn doesn't support asymmetric left/right top/bottopm paddings
        checkConv2dCUDNNPadAsymmetric(newInput, newGradI, iH, iW, oH, oW, kH, kW, sH, sW, pH, pW, dH, dW, isNCHW);

    conv2dBpCUDNN(block.launchContext(), newInput, newWeights, gradO,   newGradI, newGradW, gradB, kH,kW,sH,sW,pH,pW,dH,dW,paddingMode,isNCHW,wFormat);

    if(0 == wFormat) {
        newGradW->permutei(isNCHW ? std::vector<int>({2,3,1,0}) : std::vector<int>({1,2,3,0})); // (oC, iC, kH, kW --> kH, kW, iC, oC) or (oC, kH, kW, iC --> kH, kW, iC, oC)
        gradW->assign(newGradW);
    }

    if(newInput != input) {

        if(isNCHW)
            gradI->assign((*newGradI)({0,0,  0,0,  0,gradI->sizeAt(2),  0,gradI->sizeAt(3)}));
        else
            gradI->assign((*newGradI)({0,0,  0,gradI->sizeAt(1),  0,gradI->sizeAt(2),  0,0}));

        delete newInput;
        delete newGradI;
    }

    if(0 == wFormat) {
        delete newWeights;
        delete newGradW;
    }

    return Status::OK();
}

PLATFORM_CHECK(conv2d_bp, ENGINE_CUDA) {

    auto input   = INPUT_VARIABLE(0);                                           // [bS, iH, iW, iC] (NHWC) or [bS, iC, iH, iW] (NCHW)
    auto weights = INPUT_VARIABLE(1);                                           // [kH, kW, iC, oC] always
    auto bias    = block.width() > 3 ? INPUT_VARIABLE(2) : nullptr;             // [oC]
    auto gradO   = block.width() > 3 ? INPUT_VARIABLE(3) : INPUT_VARIABLE(2);   // [bS, oH, oW, oC] (NHWC) or [bS, oC, oH, oW] (NCHW), epsilon_next

    const int paddingMode = INT_ARG(8);                                             // 0-VALID, 1-SAME, 2-CAUSAL
    const int isNCHW      = block.getIArguments()->size() > 9 ? !INT_ARG(9) : 1;    // INT_ARG(9): 0-NCHW, 1-NHWC

    const bool badInputType   = input->dataType()   != DataType::DOUBLE && input->dataType()   != DataType::FLOAT32 && input->dataType()   != DataType::HALF;
    const bool badWeightsType = weights->dataType() != DataType::DOUBLE && weights->dataType() != DataType::FLOAT32 && weights->dataType() != DataType::HALF;
    const bool badGradOType   = gradO->dataType()   != DataType::DOUBLE && gradO->dataType()   != DataType::FLOAT32 && gradO->dataType()   != DataType::HALF;
    const bool badBiasType    = bias == nullptr ? false : (bias->dataType() != DataType::DOUBLE && bias->dataType() != DataType::FLOAT32 && bias->dataType() != DataType::HALF);

    return isNCHW && paddingMode != 2 && !badInputType && !badWeightsType && !badGradOType && !badBiasType;
}







// PLATFORM_IMPL(conv2d, ENGINE_CUDA) {

//     auto handle = reinterpret_cast<hipdnnHandle_t *>(block.launchContext()->getCuDnnHandle());
//     auto res = hipdnnSetStream(*handle, *block.launchContext()->getCudaStream());
//     if (res != 0)
//         throw sd::cuda_exception::build("Can't set stream for cuDNN", res);

//     auto input   = INPUT_VARIABLE(0);                                    // [bS, iH, iW, iC] (NHWC) or [bS, iC, iH, iW] (NCHW)
//     auto weights = INPUT_VARIABLE(1);                                    // [kH, kW, iC, oC] always
//     auto bias    = block.width() > 2 ? INPUT_VARIABLE(2) : nullptr;      // [oC]

//     auto output  = OUTPUT_VARIABLE(0);                                   // [bS, oH, oW, oC] (NHWC) or [bS, oC, oH, oW] (NCHW)

//     NDArray::prepareSpecialUse({output}, {input, weights, bias});

//     int sH = INT_ARG(2);                                                        // strides height
//     int sW = INT_ARG(3);                                                        // strides width
//     int pH = INT_ARG(4);                                                        // paddings height
//     int pW = INT_ARG(5);                                                        // paddings width
//     int dH = INT_ARG(6);                                                        // dilations height
//     int dW = INT_ARG(7);                                                        // dilations width
//     int isSameMode = INT_ARG(8);                                                // 0-VALID, 1-SAME
//     bool isNCHW    = block.getIArguments()->size() > 9 ? !INT_ARG(9) : 1;       // INT_ARG(9): 0-NCHW,  1-NHWC

//     int kH = INT_ARG(0) > 0 ? INT_ARG(0) : static_cast<int>(weights->sizeAt(0)); // filter(kernel) height
//     int kW = INT_ARG(1) > 0 ? INT_ARG(1) : static_cast<int>(weights->sizeAt(1)); // filter(kernel) width

//     int bS, iC, iH, iW, oC, oH, oW;                             // batch size, input channels, input height/width, output channels, output height/width;
//     int indIOioC, indIiH, indWoC, indWiC, indWkH, indOoH;       // corresponding indexes
//     ConvolutionUtils::getSizesAndIndexesConv2d(isNCHW, *input, *output, bS, iC, iH, iW, oC, oH, oW, indIOioC, indIiH, indWiC, indWoC, indWkH, indOoH);
//     ConvolutionUtils::calcPadding2D(pH, pW, oH, oW, iH, iW, kH, kW, sH, sW, dH, dW, isSameMode);

//     auto dtype = cudnnDataType(input->dataType());


//     hipdnnTensorDescriptor_t src;
//     hipdnnCreateTensorDescriptor(&src);
//     res = hipdnnSetTensor4dDescriptorEx(src, dtype, input->sizeAt(0), input->sizeAt(1), input->sizeAt(2), input->sizeAt(3), input->strideAt(0), input->strideAt(1), input->strideAt(2), input->strideAt(3));
//     if (res != 0)
//         throw sd::cuda_exception::build("hipdnnSetTensor4dDescriptorEx src failed", res);

//     // TODO: we definitely want NHWC here as well
//     hipdnnFilterDescriptor_t wght;
//     hipdnnCreateFilterDescriptor(&wght);
//     res = hipdnnSetFilter4dDescriptor(wght, dtype, HIPDNN_TENSOR_NCHW, oC, iC, kH, kW);
//     if (res != 0)
//         throw sd::cuda_exception::build("hipdnnSetFilter4dDescriptor failed", res);

//     hipdnnConvolutionDescriptor_t cdc;
//     hipdnnCreateConvolutionDescriptor(&cdc);
//     res = hipdnnSetConvolution2dDescriptor(cdc, pH, pW, sH, sW, dH, dW, HIPDNN_CROSS_CORRELATION, dtype);
//     if (res != 0)
//         throw sd::cuda_exception::build("hipdnnSetConvolution2dDescriptor failed", res);

//     hipdnnTensorDescriptor_t dst;
//     hipdnnCreateTensorDescriptor(&dst);
//     res = hipdnnSetTensor4dDescriptorEx(dst, dtype, output->sizeAt(0), output->sizeAt(1), output->sizeAt(2), output->sizeAt(3), output->strideAt(0), output->strideAt(1), output->strideAt(2), output->strideAt(3));
//     if (res != 0)
//         throw sd::cuda_exception::build("hipdnnSetTensor4dDescriptorEx dst failed", res);

//     // TODO: workspace algorithms are supposed to be faster, so we should use it here if we have enough memory
//     hipdnnConvolutionFwdAlgo_t algo;
//     res = hipdnnGetConvolutionForwardAlgorithm(*handle, src, wght, cdc, dst, HIPDNN_CONVOLUTION_FWD_NO_WORKSPACE, 0, &algo);
//     if (res != 0)
//         throw sd::cuda_exception::build("hipdnnGetConvolutionForwardAlgorithm failed", res);

//     // TODO: should be float if dtype is half/float, and double otherwise
//     float alpha = 1.0f;
//     float beta = 0.0f;
//     res = hipdnnConvolutionForward(*handle, &alpha, src, input->specialBuffer(), wght, weights->specialBuffer(), cdc, algo, nullptr, 0, &beta, dst, output->specialBuffer());
//     if (res != 0)
//         throw sd::cuda_exception::build("hipdnnConvolutionForward failed", res);


//     if (bias != nullptr) {
//         hipdnnTensorDescriptor_t bs;
//         hipdnnCreateTensorDescriptor(&bs);
//         if (isNCHW) {
//             res = hipdnnSetTensor4dDescriptor(bs, HIPDNN_TENSOR_NCHW, dtype, 1, bias->lengthOf(), 1, 1);
//             if (res != 0)
//                 throw sd::cuda_exception::build("hipdnnSetTensor4dDescriptorEx bias NHWC failed", res);
//         } else {
//             res = hipdnnSetTensor4dDescriptor(bs, HIPDNN_TENSOR_NHWC, dtype, 1, 1, 1, bias->lengthOf());
//             if (res != 0)
//                 throw sd::cuda_exception::build("hipdnnSetTensor4dDescriptorEx bias NHWC failed", res);
//         }

//         res = hipdnnAddTensor(*handle, &alpha, bs, bias->specialBuffer(), &alpha, dst, output->specialBuffer());
//         if (res != 0)
//             throw sd::cuda_exception::build("hipdnnAddTensor failed", res);
//     }


//     NDArray::registerSpecialUse({output}, {input, weights, bias});

//     return Status::OK();
// }


}
}
}
