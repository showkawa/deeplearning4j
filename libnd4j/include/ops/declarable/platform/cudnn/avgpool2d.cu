/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma (iuriish@yahoo.com)
//


#include "cudnnUtils.h"
#include <ops/declarable/helpers/convolutions.h>

namespace sd      {
namespace ops       {
namespace platforms {


//////////////////////////////////////////////////////////////////////////
PLATFORM_IMPL(avgpool2d, ENGINE_CUDA) {

    auto input = INPUT_VARIABLE(0);
    auto output = OUTPUT_VARIABLE(0);

    // 0,1 - kernel Height/Width; 2,3 - stride Height/Width; 4,5 - pad Height/Width; 6,7 - dilation Height/Width; 8 - same mode;
    const auto kH = INT_ARG(0);
    const auto kW = INT_ARG(1);
    const auto sH = INT_ARG(2);
    const auto sW = INT_ARG(3);
          auto pH = INT_ARG(4);
          auto pW = INT_ARG(5);
    const auto dH = INT_ARG(6);
    const auto dW = INT_ARG(7);
    const auto paddingMode = static_cast<bool>(INT_ARG(8));
    const auto extraParam0 = INT_ARG(9);
    const int isNCHW  = block.getIArguments()->size() > 10 ? !INT_ARG(10) : 1;       // INT_ARG(10): 0-NCHW, 1-NHWC

    REQUIRE_TRUE(input->rankOf() == 4, 0, "AVGPOOL2D CUDNN op: input should have rank of 4, but got %i instead", input->rankOf());
    REQUIRE_TRUE(dH != 0 && dW != 0, 0, "AVGPOOL2D CUDNN op: dilation must not be zero, but got instead {%i, %i}", dH, dW);

    int oH = 0;
    int oW = 0;

    const int iH = static_cast<int>(isNCHW ? input->sizeAt(2) : input->sizeAt(1));
    const int iW = static_cast<int>(isNCHW ? input->sizeAt(3) : input->sizeAt(2));

    ConvolutionUtils::calcOutSizePool2D(oH, oW, kH, kW, sH, sW, pH, pW, dH, dW, iH, iW, paddingMode);

    if (paddingMode)
        ConvolutionUtils::calcPadding2D(pH, pW, oH, oW, iH, iW, kH, kW, sH, sW, dH, dW);

    const hipdnnPoolingMode_t mode = (extraParam0 == 0) ? HIPDNN_POOLING_AVERAGE_COUNT_EXCLUDE_PADDING : HIPDNN_POOLING_AVERAGE_COUNT_INCLUDE_PADDING;

    pooling2dCUDNN(block.launchContext(), input, output, kH, kW, sH, sW, pH, pW, dH, dW, isNCHW, mode);

    return Status::OK();
}

//////////////////////////////////////////////////////////////////////////
PLATFORM_CHECK(avgpool2d, ENGINE_CUDA) {

    auto input = INPUT_VARIABLE(0);
    auto output = OUTPUT_VARIABLE(0);

    const auto goodType  = input->dataType() == DataType::DOUBLE || input->dataType() == DataType::FLOAT32 || input->dataType() == DataType::HALF || input->dataType() == DataType::INT32;

    return goodType && input->dataType() == output->dataType();
}

//////////////////////////////////////////////////////////////////////////
PLATFORM_IMPL(avgpool2d_bp, ENGINE_CUDA) {

    auto input = INPUT_VARIABLE(0);                          // [bS, iH, iW, iC] (NHWC) or [bS, iC, iH, iW] (NCHW)
    auto gradO = INPUT_VARIABLE(1);                          // [bS, oH, oW, oC] (NHWC) or [bS, oC, oH, oW] (NCHW), epsilon_next
    auto gradI = OUTPUT_VARIABLE(0);                         // [bS, iH, iW, iC] (NHWC) or [bS, iC, iH, iW] (NCHW), epsilon

    const auto  kH = INT_ARG(0);                                                        // filter(kernel) height
    const auto  kW = INT_ARG(1);                                                        // filter(kernel) width
    const auto  sH = INT_ARG(2);                                                        // strides height
    const auto  sW = INT_ARG(3);                                                        // strides width
          auto  pH = INT_ARG(4);                                                        // paddings height
          auto  pW = INT_ARG(5);                                                        // paddings width
    const auto  dH = INT_ARG(6);                                                        // dilations height
    const auto  dW = INT_ARG(7);                                                        // dilations width
    const auto  paddingMode = INT_ARG(8);                                               // 0-VALID, 1-SAME
    const auto  extraParam0 = INT_ARG(9);
    const auto  isNCHW = block.getIArguments()->size() > 10 ? !INT_ARG(10) : 1;         // INT_ARG(10): 0-NCHW, 1-NHWC

    REQUIRE_TRUE(input->rankOf() == 4, 0, "AVGPOOL2D_BP CUDNN op: input should have rank of 4, but got %i instead", input->rankOf());
    REQUIRE_TRUE(dH != 0 && dW != 0, 0, "AVGPOOL2D_BP CUDNN op: dilation must not be zero, but got instead {%i, %i}", dH, dW);

    int bS, iC, iH, iW, oC, oH, oW;                             // batch size, input channels, input height/width, output channels, output height/width;
    int indIOioC, indIiH, indWoC, indWiC, indWkH, indOoH;       // corresponding indexes
    ConvolutionUtils::getSizesAndIndexesConv2d(isNCHW, 0, *input, *gradO, bS, iC, iH, iW, oC, oH, oW, indIOioC, indIiH, indWiC, indWoC, indWkH, indOoH);

    std::vector<Nd4jLong>  expectedGradOShape = ShapeUtils::composeShapeUsingDimsAndIdx({bS,iC,oH,oW,  0,indIOioC,indIiH,indIiH+1});
    std::vector<Nd4jLong>  expectedGradIShape = ShapeUtils::composeShapeUsingDimsAndIdx({bS,iC,iH,iW,  0,indIOioC,indIiH,indIiH+1});
    REQUIRE_TRUE(gradO->isSameShape(expectedGradOShape), 0, "AVGPOOL2D_BP CUDNN op: wrong shape of output's gradients array (next epsilon), expected is %s, but got %s instead !", ShapeUtils::shapeAsString(expectedGradOShape).c_str(), ShapeUtils::shapeAsString(gradO).c_str());
    REQUIRE_TRUE(gradI->isSameShape(expectedGradIShape), 0, "AVGPOOL2D_BP CUDNN op: wrong shape of input's gradients array (epsilon), expected is %s, but got %s instead !", ShapeUtils::shapeAsString(expectedGradIShape).c_str(), ShapeUtils::shapeAsString(gradI).c_str());

    if(paddingMode)                       // SAME
        ConvolutionUtils::calcPadding2D(pH, pW, oH, oW, iH, iW, kH, kW, sH, sW, dH, dW);

    const hipdnnPoolingMode_t mode = (extraParam0 == 0) ? HIPDNN_POOLING_AVERAGE_COUNT_EXCLUDE_PADDING : HIPDNN_POOLING_AVERAGE_COUNT_INCLUDE_PADDING;

    pooling2dBpCUDNN(block.launchContext(), input, gradO, gradI, kH, kW, sH, sW, pH, pW, dH, dW, isNCHW, mode);

    return Status::OK();
}

PLATFORM_CHECK(avgpool2d_bp, ENGINE_CUDA) {

    auto input = INPUT_VARIABLE(0);                          // [bS, iH, iW, iC] (NHWC) or [bS, iC, iH, iW] (NCHW)
    auto gradO = INPUT_VARIABLE(1);                          // [bS, oH, oW, oC] (NHWC) or [bS, oC, oH, oW] (NCHW), epsilon_next
    auto gradI = OUTPUT_VARIABLE(0);                         // [bS, iH, iW, iC] (NHWC) or [bS, iC, iH, iW] (NCHW), epsilon

    const auto goodType = input->dataType() == DataType::DOUBLE || input->dataType() == DataType::FLOAT32 || input->dataType() == DataType::HALF || input->dataType() == DataType::INT32;

    return goodType && (input->dataType() == gradO->dataType())
                    && (input->dataType() == gradI->dataType())
                    && shape::haveSameShapeAndStrides(input->shapeInfo(), gradI->shapeInfo());
}


}
}
}
