/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma (iuriish@yahoo.com)
//


#include "cudnnUtils.h"
#include <ops/declarable/helpers/convolutions.h>

namespace sd      {
namespace ops       {
namespace platforms {

//////////////////////////////////////////////////////////////////////////
void checkConv2dCUDNNPadAsymmetric(NDArray* &input, NDArray* &gradI,
                                            const int iH, const int iW,
                                            const int oH, const int oW,
                                            const int kH, const int kW,
                                            const int sH, const int sW,
                                            const int pH, const int pW,
                                            const int dH, const int dW,
                                            const bool isNCHW) {

    const auto pHsum = ((oH - 1) * sH + ((kH - 1) * dH + 1) - iH);
    const auto pWsum = ((oW - 1) * sW + ((kW - 1) * dW + 1) - iW);

    const bool isPHasymm = pH != (pHsum - pH);
    const bool isPWasymm = pW != (pWsum - pW);

    if(!isPHasymm && !isPWasymm)
        return;

    std::vector<Nd4jLong> newShape = input->getShapeAsVector();

    const int iHposition = isNCHW ? 2 : 1;

    if(isPHasymm)
        newShape[iHposition] += 1;
    if(isPWasymm)
        newShape[iHposition + 1] += 1;

    NDArray* newInput = new NDArray(input->ordering(), newShape, input->dataType(), input->getContext());

    if(isNCHW)
        (*newInput)({0,0,  0,0,  0,input->sizeAt(2),  0,input->sizeAt(3)}).assign(input);
    else
        (*newInput)({0,0,  0,input->sizeAt(1),  0,input->sizeAt(2),  0,0}).assign(input);

    input = newInput;

    if(gradI != nullptr)
        gradI = new NDArray(gradI->ordering(), newShape, gradI->dataType(), gradI->getContext());
}


//////////////////////////////////////////////////////////////////////////
void checkConv3dCUDNNPadAsymmetric(NDArray* &input, NDArray* &gradI,
                                            const int iD, const int iH, const int iW,
                                            const int oD, const int oH, const int oW,
                                            const int kD, const int kH, const int kW,
                                            const int sD, const int sH, const int sW,
                                            const int pD, const int pH, const int pW,
                                            const int dD, const int dH, const int dW,
                                            const bool isNCDHW) {

    const auto pDsum = ((oD - 1) * sD + ((kD - 1) * dD + 1) - iD);
    const auto pHsum = ((oH - 1) * sH + ((kH - 1) * dH + 1) - iH);
    const auto pWsum = ((oW - 1) * sW + ((kW - 1) * dW + 1) - iW);

    const bool isPDasymm = pD != (pDsum - pD);
    const bool isPHasymm = pH != (pHsum - pH);
    const bool isPWasymm = pW != (pWsum - pW);

    if(!isPDasymm && !isPHasymm && !isPWasymm)
        return;

    std::vector<Nd4jLong> newShape = input->getShapeAsVector();

    const int iDposition = isNCDHW ? 2 : 1;

    if(isPDasymm)
        newShape[iDposition] += 1;
    if(isPHasymm)
        newShape[iDposition + 1] += 1;
    if(isPWasymm)
        newShape[iDposition + 2] += 1;

    NDArray* newInput = new NDArray(input->ordering(), newShape, input->dataType(), input->getContext());

    if(isNCDHW)
        (*newInput)({0,0,  0,0,  0,input->sizeAt(2),  0,input->sizeAt(3),  0,input->sizeAt(4)}).assign(input);
    else
        (*newInput)({0,0,  0,input->sizeAt(1),  0,input->sizeAt(2),  0,input->sizeAt(3),  0,0}).assign(input);

    input = newInput;

    if(gradI != nullptr)
        gradI = new NDArray(gradI->ordering(), newShape, gradI->dataType(), gradI->getContext());
}

//////////////////////////////////////////////////////////////////////////
void pooling2dCUDNN(const LaunchContext* context,
                    const NDArray* input, NDArray* output,
                    const int kH, const int kW,
                    const int sH, const int sW,
                    const int pH, const int pW,
                    const int dH, const int dW,
                    const bool isNCHW, const hipdnnPoolingMode_t mode) {

    int bS, iC, iH, iW, oC, oH, oW;                             // batch size, input channels, input height/width, output channels, output height/width;
    int indIOioC, indIiH, indWoC, indWiC, indWkH, indOoH;       // corresponding indexes
    ConvolutionUtils::getSizesAndIndexesConv2d(isNCHW, 0, *input, *output, bS, iC, iH, iW, oC, oH, oW, indIOioC, indIiH, indWiC, indWoC, indWkH, indOoH);

    auto handle = reinterpret_cast<hipdnnHandle_t *>(context->getCuDnnHandle());
    hipdnnStatus_t err = hipdnnSetStream(*handle, *context->getCudaStream());
    if (err != 0) throw sd::cuda_exception::build("pooling2dCUDNN: can't set stream for cuDNN", err);

    hipdnnTensorFormat_t format = isNCHW ? HIPDNN_TENSOR_NCHW : HIPDNN_TENSOR_NHWC;

    // input descriptor
    hipdnnTensorDescriptor_t x;
    hipdnnCreateTensorDescriptor(&x);
    if(input->ews() == 1 && input->ordering() == 'c')
        err = hipdnnSetTensor4dDescriptor(x, format, cudnnDataType(input->dataType()), bS, iC, iH, iW);
    else
        err = hipdnnSetTensor4dDescriptorEx(x, cudnnDataType(input->dataType()), bS, iC, iH, iW, input->strideAt(0), input->strideAt(indIOioC), input->strideAt(indIiH), input->strideAt(indIiH + 1));
    if (err != 0) throw sd::cuda_exception::build("pooling2dCUDNN: hipdnnSetTensor4dDescriptor/hipdnnSetTensor4dDescriptorEx for input failed", err);

    // output descriptor
    hipdnnTensorDescriptor_t z;
    hipdnnCreateTensorDescriptor(&z);
    if(output->ews() == 1 && output->ordering() == 'c')
        err = hipdnnSetTensor4dDescriptor(z, format, cudnnDataType(output->dataType()), bS, oC, oH, oW);
    else
        err = hipdnnSetTensor4dDescriptorEx(z, cudnnDataType(output->dataType()), bS, oC, oH, oW, output->strideAt(0), output->strideAt(indIOioC), output->strideAt(indOoH), output->strideAt(indOoH + 1));
    if (err != 0) throw sd::cuda_exception::build("pooling2dCUDNN: hipdnnSetTensor4dDescriptor/hipdnnSetTensor4dDescriptorEx for output failed", err);

    // description of pooling
    hipdnnPoolingDescriptor_t pooling;
    hipdnnCreatePoolingDescriptor(&pooling);
    err = hipdnnSetPooling2dDescriptor(pooling, mode, HIPDNN_PROPAGATE_NAN, kH, kW, pH, pW, sH, sW);
    if (err != 0) throw sd::cuda_exception::build("pooling2dCUDNN: hipdnnSetPooling2dDescriptor failed", err);

    // provide scaling parameters
    const float  alpha32(1), beta32(0);
    const double alpha64(1), beta64(0);
    const void* alpha = output->sizeOfT() <= 4 ? reinterpret_cast<const void*>(&alpha32) : reinterpret_cast<const void*>(&alpha64);
    const void* beta  = output->sizeOfT() <= 4 ? reinterpret_cast<const void*>(&beta32)  : reinterpret_cast<const void*>(&beta64);

    NDArray::prepareSpecialUse({output}, {input});

    // run calculation
    err = hipdnnPoolingForward(*handle, pooling, alpha, x, input->specialBuffer(), beta, z, output->specialBuffer());
    if (err != 0) throw sd::cuda_exception::build("pooling2dCUDNN: hipdnnPoolingForward failed", err);

    auto cudaErr = hipStreamSynchronize(*context->getCudaStream());
    if (cudaErr != 0)
        throw cuda_exception::build("pooling2dCUDNN: hipStreamSynchronize failed !", cudaErr);

    NDArray::registerSpecialUse({output}, {input});
}

//////////////////////////////////////////////////////////////////////////
void pooling2dBpCUDNN(const LaunchContext* context,
                    const NDArray* input, const NDArray* gradO,
                          NDArray* gradI,
                    const int kH, const int kW,
                    const int sH, const int sW,
                    const int pH, const int pW,
                    const int dH, const int dW,
                    const bool isNCHW, const hipdnnPoolingMode_t mode) {

    int bS, iC, iH, iW, oC, oH, oW;                             // batch size, input channels, input height/width, output channels, output height/width;
    int indIOioC, indIiH, indWoC, indWiC, indWkH, indOoH;       // corresponding indexes
    ConvolutionUtils::getSizesAndIndexesConv2d(isNCHW, 0, *input, *gradO, bS, iC, iH, iW, oC, oH, oW, indIOioC, indIiH, indWiC, indWoC, indWkH, indOoH);

    auto handle = reinterpret_cast<hipdnnHandle_t *>(context->getCuDnnHandle());
    hipdnnStatus_t err = hipdnnSetStream(*handle, *context->getCudaStream());
    if (err != 0) throw sd::cuda_exception::build("pooling2dBpCUDNN: can't set stream for cuDNN", err);

    hipdnnTensorFormat_t format = isNCHW ? HIPDNN_TENSOR_NCHW : HIPDNN_TENSOR_NHWC;

    // input and gradI descriptor
    hipdnnTensorDescriptor_t x;
    hipdnnCreateTensorDescriptor(&x);
    if(input->ews() == 1 && input->ordering() == 'c')
        err = hipdnnSetTensor4dDescriptor(x, format, cudnnDataType(input->dataType()), bS, iC, iH, iW);
    else
        err = hipdnnSetTensor4dDescriptorEx(x, cudnnDataType(input->dataType()), bS, iC, iH, iW, input->strideAt(0), input->strideAt(indIOioC), input->strideAt(indIiH), input->strideAt(indIiH + 1));
    if (err != 0) throw sd::cuda_exception::build("pooling2dBpCUDNN: hipdnnSetTensor4dDescriptor/hipdnnSetTensor4dDescriptorEx for input/gradI failed", err);

    // gradO descriptor
    hipdnnTensorDescriptor_t dz;
    hipdnnCreateTensorDescriptor(&dz);
    if(gradO->ews() == 1 && gradO->ordering() == 'c')
        err = hipdnnSetTensor4dDescriptor(dz, format, cudnnDataType(gradO->dataType()), bS, oC, oH, oW);
    else
        err = hipdnnSetTensor4dDescriptorEx(dz, cudnnDataType(gradO->dataType()), bS, oC, oH, oW, gradO->strideAt(0), gradO->strideAt(indIOioC), gradO->strideAt(indOoH), gradO->strideAt(indOoH + 1));
    if (err != 0) throw sd::cuda_exception::build("pooling2dBpCUDNN: hipdnnSetTensor4dDescriptor/hipdnnSetTensor4dDescriptorEx for gradO failed", err);

    // description of pooling
    hipdnnPoolingDescriptor_t pooling;
    hipdnnCreatePoolingDescriptor(&pooling);
    err = hipdnnSetPooling2dDescriptor(pooling, mode, HIPDNN_PROPAGATE_NAN, kH, kW, pH, pW, sH, sW);
    if (err != 0) throw sd::cuda_exception::build("pooling2dBpCUDNN: hipdnnSetPooling2dDescriptor failed", err);

    // provide scaling parameters
    const float  alpha32(1), beta32(0);
    const double alpha64(1), beta64(0);
    const void* alpha = gradO->sizeOfT() <= 4 ? reinterpret_cast<const void*>(&alpha32) : reinterpret_cast<const void*>(&alpha64);
    const void* beta  = gradO->sizeOfT() <= 4 ? reinterpret_cast<const void*>(&beta32)  : reinterpret_cast<const void*>(&beta64);

    NDArray::prepareSpecialUse({gradI}, {input, gradO});

    // run calculation for gradI
    err = hipdnnPoolingBackward(*handle, pooling, alpha, dz, gradO->specialBuffer(), dz, gradO->specialBuffer(), x, input->specialBuffer(), beta, x, gradI->specialBuffer());
    if (err != 0) throw sd::cuda_exception::build("pooling2dBpCUDNN: hipdnnPoolingBackward failed", err);

    auto cudaErr = hipStreamSynchronize(*context->getCudaStream());
    if (cudaErr != 0)
        throw cuda_exception::build("pooling2dBpCUDNN: hipStreamSynchronize failed !", cudaErr);

    NDArray::registerSpecialUse({gradI}, {input, gradO});
}

//////////////////////////////////////////////////////////////////////////
void pooling3dCUDNN(const LaunchContext* context,
                    const NDArray* input, NDArray* output,
                    const int kD, const int kH, const int kW,
                    const int sD, const int sH, const int sW,
                    const int pD, const int pH, const int pW,
                    const int dD, const int dH, const int dW,
                    const bool isNCDHW, const hipdnnPoolingMode_t mode) {

    auto handle = reinterpret_cast<hipdnnHandle_t *>(context->getCuDnnHandle());
    hipdnnStatus_t err = hipdnnSetStream(*handle, *context->getCudaStream());
    if (err != 0) throw sd::cuda_exception::build("pooling3dCUDNN: can't set stream for cuDNN", err);

    const int numDims = 5;

    int bS, iC, iD, iH, iW, oC, oD, oH, oW;                     // batch size, input channels, input depth/height/width, output channels, output depth/height/width;
    int indIOioC, indIOioD, indWoC, indWiC, indWkD;             // corresponding indexes
    ConvolutionUtils::getSizesAndIndexesConv3d(isNCDHW, 0, *input, *output, bS, iC, iD, iH, iW, oC, oD, oH, oW, indIOioC, indIOioD, indWiC, indWoC, indWkD);

    const int pSizes[] = {pD, pH, pW};
    const int sSizes[] = {sD, sH, sW};
    const int kSizes[] = {kD, kH, kW};

    const int xShape[] = {bS, iC, iD, iH, iW};
    const int zShape[] = {bS, oC, oD, oH, oW};

    const int xStrides[] = {(int)input->strideAt(0), (int)input->strideAt(1), (int)input->strideAt(2), (int)input->strideAt(3), (int)input->strideAt(4)};
    const int zStrides[] = {(int)output->strideAt(0), (int)output->strideAt(1), (int)output->strideAt(2), (int)output->strideAt(3), (int)output->strideAt(4)};

    hipdnnTensorFormat_t format = isNCDHW ? HIPDNN_TENSOR_NCHW : HIPDNN_TENSOR_NHWC;

    // input descriptor
    hipdnnTensorDescriptor_t x;
    hipdnnCreateTensorDescriptor(&x);
    if(input->ews() == 1 && input->ordering() == 'c')
        err = cudnnSetTensorNdDescriptorEx(x, format, cudnnDataType(input->dataType()), numDims, xShape);
    else
        err = hipdnnSetTensorNdDescriptor(x, cudnnDataType(input->dataType()), numDims, xShape, xStrides);
    if (err != 0) throw sd::cuda_exception::build("pooling3dCUDNN: hipdnnSetTensorNdDescriptor/cudnnSetTensorNdDescriptorEx for input failed", err);

    // output descriptor
    hipdnnTensorDescriptor_t z;
    hipdnnCreateTensorDescriptor(&z);
    if(output->ews() == 1 && output->ordering() == 'c')
        err = cudnnSetTensorNdDescriptorEx(z, format, cudnnDataType(output->dataType()), numDims, zShape);
    else
        err = hipdnnSetTensorNdDescriptor(z, cudnnDataType(output->dataType()), numDims, zShape, zStrides);
    if (err != 0) throw sd::cuda_exception::build("pooling3dCUDNN: hipdnnSetTensorNdDescriptor/cudnnSetTensorNdDescriptorEx for output failed", err);

    // description of pooling
    hipdnnPoolingDescriptor_t pooling;
    hipdnnCreatePoolingDescriptor(&pooling);
    err = hipdnnSetPoolingNdDescriptor(pooling, mode, HIPDNN_PROPAGATE_NAN, numDims - 2, kSizes, pSizes, sSizes);
    if (err != 0) throw sd::cuda_exception::build("pooling3dCUDNN: hipdnnSetPoolingNdDescriptor failed", err);

    // provide scaling parameters
    const float  alpha32(1), beta32(0);
    const double alpha64(1), beta64(0);
    const void* alpha = output->sizeOfT() <= 4 ? reinterpret_cast<const void*>(&alpha32) : reinterpret_cast<const void*>(&alpha64);
    const void* beta  = output->sizeOfT() <= 4 ? reinterpret_cast<const void*>(&beta32)  : reinterpret_cast<const void*>(&beta64);

    NDArray::prepareSpecialUse({output}, {input});

    // run calculation
    err = hipdnnPoolingForward(*handle, pooling, alpha, x, input->specialBuffer(), beta, z, output->specialBuffer());
    if (err != 0) throw sd::cuda_exception::build("pooling3dCUDNN: hipdnnPoolingForward failed", err);

    auto cudaErr = hipStreamSynchronize(*context->getCudaStream());
    if (cudaErr != 0)
        throw cuda_exception::build("pooling3dCUDNN: hipStreamSynchronize failed !", cudaErr);

    NDArray::registerSpecialUse({output}, {input});
}

//////////////////////////////////////////////////////////////////////////
void pooling3dBpCUDNN(const LaunchContext* context,
                    const NDArray* input, const NDArray* gradO,
                          NDArray* gradI,
                    const int kD, const int kH, const int kW,
                    const int sD, const int sH, const int sW,
                    const int pD, const int pH, const int pW,
                    const int dD, const int dH, const int dW,
                    const bool isNCDHW, const hipdnnPoolingMode_t mode) {

    auto handle = reinterpret_cast<hipdnnHandle_t *>(context->getCuDnnHandle());
    hipdnnStatus_t err = hipdnnSetStream(*handle, *context->getCudaStream());
    if (err != 0) throw sd::cuda_exception::build("pooling3dBpCUDNN: can't set stream for cuDNN", err);

    const int numDims = 5;

    int bS, iC, iD, iH, iW, oC, oD, oH, oW;                // batch size, input channels, input depth/height/width, output channels, output depth/height/width;
    int indIOioC, indIOioD, indWoC, indWiC, indWkD;       // corresponding indexes
    ConvolutionUtils::getSizesAndIndexesConv3d(isNCDHW, 0, *input, *gradO, bS, iC, iD, iH, iW, oC, oD, oH, oW, indIOioC, indIOioD, indWiC, indWoC, indWkD);

    const int pSizes[] = {pD, pH, pW};
    const int sSizes[] = {sD, sH, sW};
    const int kSizes[] = {kD, kH, kW};

    const int xShape[]  = {bS, iC, iD, iH, iW};
    const int dzShape[] = {bS, oC, oD, oH, oW};

    const int xStrides[]  = {(int)input->strideAt(0), (int)input->strideAt(1), (int)input->strideAt(2), (int)input->strideAt(3), (int)input->strideAt(4)};
    const int dzStrides[] = {(int)gradO->strideAt(0), (int)gradO->strideAt(1), (int)gradO->strideAt(2), (int)gradO->strideAt(3), (int)gradO->strideAt(4)};

    hipdnnTensorFormat_t format = isNCDHW ? HIPDNN_TENSOR_NCHW : HIPDNN_TENSOR_NHWC;

    // input and gradI descriptor
    hipdnnTensorDescriptor_t x;
    hipdnnCreateTensorDescriptor(&x);
    if(input->ews() == 1 && input->ordering() == 'c')
        err = cudnnSetTensorNdDescriptorEx(x, format, cudnnDataType(input->dataType()), numDims, xShape);
    else
        err = hipdnnSetTensorNdDescriptor(x, cudnnDataType(input->dataType()), numDims, xShape, xStrides);
    if (err != 0) throw sd::cuda_exception::build("pooling3dBpCUDNN: hipdnnSetTensorNdDescriptor/cudnnSetTensorNdDescriptorEx for input/gradI failed", err);

    // gradO descriptor
    hipdnnTensorDescriptor_t dz;
    hipdnnCreateTensorDescriptor(&dz);
    if(gradO->ews() == 1 && gradO->ordering() == 'c')
        err = cudnnSetTensorNdDescriptorEx(dz, format, cudnnDataType(gradO->dataType()), numDims, dzShape);
    else
        err = hipdnnSetTensorNdDescriptor(dz, cudnnDataType(gradO->dataType()), numDims, dzShape, dzStrides);
    if (err != 0) throw sd::cuda_exception::build("pooling3dBpCUDNN: hipdnnSetTensorNdDescriptor/cudnnSetTensorNdDescriptorEx for gradO failed", err);

    // description of pooling
    hipdnnPoolingDescriptor_t pooling;
    hipdnnCreatePoolingDescriptor(&pooling);
    err = hipdnnSetPoolingNdDescriptor(pooling, mode, HIPDNN_PROPAGATE_NAN, numDims - 2, kSizes, pSizes, sSizes);
    if (err != 0) throw sd::cuda_exception::build("pooling3dBpCUDNN: hipdnnSetPoolingNdDescriptor failed", err);

    // provide scaling parameters
    const float  alpha32(1), beta32(0);
    const double alpha64(1), beta64(0);
    const void* alpha = gradO->sizeOfT() <= 4 ? reinterpret_cast<const void*>(&alpha32) : reinterpret_cast<const void*>(&alpha64);
    const void* beta  = gradO->sizeOfT() <= 4 ? reinterpret_cast<const void*>(&beta32)  : reinterpret_cast<const void*>(&beta64);

    // cudnn maxpool2d_bp api requires ff output as one of input arguments
    if(mode == HIPDNN_POOLING_MAX) {

        NDArray temp(gradO);

        NDArray::prepareSpecialUse({gradI}, {input, gradO, &temp});

        // run ff calculation
        err = hipdnnPoolingForward(*handle, pooling, alpha, x, input->specialBuffer(), beta, dz, temp.specialBuffer());
        if (err != 0) throw sd::cuda_exception::build("pooling3dCUDNN: hipdnnPoolingForward failed", err);

        // run bp calculation for gradI
        err = hipdnnPoolingBackward(*handle, pooling, alpha, dz, temp.specialBuffer(), dz, gradO->specialBuffer(), x, input->specialBuffer(), beta, x, gradI->specialBuffer());
        if (err != 0) throw sd::cuda_exception::build("pooling2dBpCUDNN: hipdnnPoolingBackward failed", err);

        NDArray::registerSpecialUse({gradI}, {input, gradO, &temp});
    }
    else {

        NDArray::prepareSpecialUse({gradI}, {input, gradO});

        // run bp calculation for gradI
        err = hipdnnPoolingBackward(*handle, pooling, alpha, dz, gradO->specialBuffer(), dz, gradO->specialBuffer(), x, input->specialBuffer(), beta, x, gradI->specialBuffer());
        if (err != 0) throw sd::cuda_exception::build("pooling2dBpCUDNN: hipdnnPoolingBackward failed", err);

        NDArray::registerSpecialUse({gradI}, {input, gradO});
    }

    auto cudaErr = hipStreamSynchronize(*context->getCudaStream());
    if (cudaErr != 0)
        throw cuda_exception::build("pooling3dBpCUDNN: hipStreamSynchronize failed !", cudaErr);
}

}
}
}
