/*******************************************************************************
 * Copyright (c) 2019 Konduit K.K.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma (iuriish@yahoo.com)
//


#include "cudnnUtils.h"
#include <ops/declarable/helpers/convolutions.h>

namespace sd      {
namespace ops       {
namespace platforms {


//////////////////////////////////////////////////////////////////////////
static void depthwiseConv2dCUDNN(const LaunchContext* context,
                        const NDArray* input, const NDArray* weights, const NDArray* bias, NDArray* output,
                        const int kH, const int kW,
                        const int sH, const int sW,
                        const int pH, const int pW,
                        const int dH, const int dW,
                        const int paddingMode, const bool isNCHW) {

    // cudnn supports only following case: mC = 1, oC = iC (groupCount == iC)

    // input [bS, iC, iH, iW] nchw or [bS, iH, iW, iC] nhwc
    // weights [iC, mC, kH, kW]
    // bias [oC], may be nullptr
    // output [bS, oC, oH, oW] nchw or [bS, oH, oW, oC] nhwc
    // oC = iC*mC

    int bS, iC, iH, iW, mC, oC, oH, oW;                             // batch size, input channels, input height/width, output channels, output height/width;
    int indIOioC, indIiH, indWmC, indWiC, indWkH, indOoH;           // corresponding indexes
    ConvolutionUtils::getSizesAndIndexesConv2d(isNCHW, 0, *input, *output, bS, iC, iH, iW, oC, oH, oW, indIOioC, indIiH, indWiC, indWmC, indWkH, indOoH);
    mC = weights->sizeAt(1);

    auto handle = reinterpret_cast<hipdnnHandle_t *>(context->getCuDnnHandle());
    hipdnnStatus_t err = hipdnnSetStream(*handle, *context->getCudaStream());
    if (err != 0) throw sd::cuda_exception::build("depthwiseConv2dCUDNN: can't set stream for cuDNN", err);

    hipdnnTensorFormat_t format = isNCHW ? HIPDNN_TENSOR_NCHW : HIPDNN_TENSOR_NHWC;

    // input descriptor
    hipdnnTensorDescriptor_t x;
    hipdnnCreateTensorDescriptor(&x);
    if(input->ews() == 1 && input->ordering() == 'c')
        err = hipdnnSetTensor4dDescriptor(x, format, cudnnDataType(input->dataType()), bS, iC, iH, iW);
    else
        err = hipdnnSetTensor4dDescriptorEx(x, cudnnDataType(input->dataType()), bS, iC, iH, iW, input->strideAt(0), input->strideAt(indIOioC), input->strideAt(indIiH), input->strideAt(indIiH + 1));
    if (err != 0) throw sd::cuda_exception::build("depthwiseConv2dCUDNN: hipdnnSetTensor4dDescriptor/hipdnnSetTensor4dDescriptorEx for input failed", err);

    // weights descriptor
    hipdnnFilterDescriptor_t w;
    hipdnnCreateFilterDescriptor(&w);
    err = hipdnnSetFilter4dDescriptor(w, cudnnDataType(weights->dataType()), HIPDNN_TENSOR_NCHW, iC, mC, kH, kW);
    if(err != 0) throw sd::cuda_exception::build("depthwiseConv2dCUDNN: hipdnnSetFilter4dDescriptor failed", err);

    // output descriptor
    hipdnnTensorDescriptor_t z;
    hipdnnCreateTensorDescriptor(&z);
    if(output->ews() == 1 && output->ordering() == 'c')
        err = hipdnnSetTensor4dDescriptor(z, format, cudnnDataType(output->dataType()), bS, oC, oH, oW);
    else
        err = hipdnnSetTensor4dDescriptorEx(z, cudnnDataType(output->dataType()), bS, oC, oH, oW, output->strideAt(0), output->strideAt(indIOioC), output->strideAt(indOoH), output->strideAt(indOoH + 1));
    if (err != 0) throw sd::cuda_exception::build("depthwiseConv2dCUDNN: hipdnnSetTensor4dDescriptor/hipdnnSetTensor4dDescriptorEx for output failed", err);

    // description of convolution
    hipdnnConvolutionDescriptor_t conv;
    hipdnnCreateConvolutionDescriptor(&conv);
    err = hipdnnSetConvolution2dDescriptor(conv, pH, pW, sH, sW, dH, dW, HIPDNN_CROSS_CORRELATION, cudnnDataType(output->dataType()));
    if (err != 0) throw sd::cuda_exception::build("depthwiseConv2dCUDNN: hipdnnSetConvolution2dDescriptor failed", err);
    err = hipdnnSetConvolutionGroupCount(conv, iC);  // set number of groups (depthwise mode) in description of convolution, groupCount == iC
    if (err != 0) throw sd::cuda_exception::build("depthwiseConv2dCUDNN: hipdnnSetConvolutionGroupCount failed", err);

    // algorithm description
    hipdnnConvolutionFwdAlgo_t algo;
    hipdnnConvolutionFwdAlgoPerf_t algoPerf;
    int count = 0;
    //err = hipdnnGetConvolutionForwardAlgorithm(*handle, x, w, conv, z, HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &algo);
    err = hipdnnFindConvolutionForwardAlgorithm(*handle, x, w, conv, z, 1, &count, &algoPerf);
    if (err != 0 || count == 0) throw sd::cuda_exception::build("depthwiseConv2dCUDNN: hipdnnGetConvolutionForwardAlgorithm failed", err);
    algo = algoPerf.algo;

    // allocate auxiliary device memory, abbreviation ws means workspace
    size_t wsSize;
    err = hipdnnGetConvolutionForwardWorkspaceSize(*handle, x, w, conv, z, algo, &wsSize);
    if (err != 0) throw sd::cuda_exception::build("depthwiseConv2dCUDNN: hipdnnGetConvolutionForwardWorkspaceSize failed", err);
    void* wsData;
    auto cudaErr = hipMalloc(&wsData, wsSize);
    if (cudaErr != 0) throw sd::cuda_exception::build("depthwiseConv2dCUDNN: hipMalloc for auxiliary workspace memory failed", cudaErr);

    // provide scaling parameters
    const float  alpha32(1), beta32(0);
    const double alpha64(1), beta64(0);
    const void* alpha = output->sizeOfT() <= 4 ? reinterpret_cast<const void*>(&alpha32) : reinterpret_cast<const void*>(&alpha64);
    const void* beta  = output->sizeOfT() <= 4 ? reinterpret_cast<const void*>(&beta32)  : reinterpret_cast<const void*>(&beta64);

    NDArray::prepareSpecialUse({output}, {input, weights, bias});

    // run calculation
    err = hipdnnConvolutionForward(*handle, alpha, x, input->specialBuffer(), w, weights->specialBuffer(), conv, algo, wsData, wsSize, beta, z, output->specialBuffer());
    if (err != 0) throw sd::cuda_exception::build("depthwiseConv2dCUDNN: hipdnnConvolutionForward failed", err);

    // add bias if it is present
    if (bias != nullptr) {

        hipdnnTensorDescriptor_t b;
        hipdnnCreateTensorDescriptor(&b);
        // err = hipdnnSetTensor4dDescriptor(b, format, cudnnDataType(bias->dataType()), 1, isNCHW ? bias->lengthOf() : 1, 1, isNCHW ? 1: bias->lengthOf());
        err = hipdnnSetTensor4dDescriptor(b, HIPDNN_TENSOR_NCHW, cudnnDataType(bias->dataType()), 1, oC, 1, 1);
        if (err != 0) throw sd::cuda_exception::build("depthwiseConv2dCUDNN: hipdnnSetTensor4dDescriptor for bias failed", err);
        err = hipdnnAddTensor(*handle, alpha, b, bias->specialBuffer(), alpha, z, output->specialBuffer());
        if (err != 0) throw sd::cuda_exception::build("depthwiseConv2dCUDNN: hipdnnAddTensor bias failed", err);
    }

    // cudaErr = hipStreamSynchronize(*context->getCudaStream());
    // if (cudaErr != 0)
    //     throw cuda_exception::build("depthwiseConv2dCUDNN: hipStreamSynchronize failed !", cudaErr);

    cudaErr = hipFree(wsData);
    if (cudaErr != 0) throw sd::cuda_exception::build("depthwiseConv2dCUDNN: hipFree for auxiliary workspace memory failed", cudaErr);

    NDArray::registerSpecialUse({output}, {input, weights, bias});
}

//////////////////////////////////////////////////////////////////////////
static void depthwiseConv2dBpCUDNN(const LaunchContext* context,
                                const NDArray* input, const NDArray* weights, const NDArray* gradO,
                                NDArray* gradI, NDArray* gradW, NDArray* gradB,
                                const int kH, const int kW,
                                const int sH, const int sW,
                                const int pH, const int pW,
                                const int dH, const int dW,
                                const int paddingMode, const bool isNCHW) {

    // cudnn supports only following case: mC = 1, oC = iC (groupCount == iC)

    // input, gradI [bS, iC, iH, iW] nchw or [bS, iH, iW, iC] nhwc
    // weights, gradW [iC, mC, kH, kW]
    // gradB [oC], may be nullptr
    // gradO [bS, oC, oH, oW] nchw or [bS, oH, oW, oC] nhwc
    // oC = iC*mC

    int bS, iC, iH, iW, mC, oC, oH, oW;                             // batch size, input channels, input height/width, output channels, output height/width;
    int indIOioC, indIiH, indWmC, indWiC, indWkH, indOoH;           // corresponding indexes
    ConvolutionUtils::getSizesAndIndexesConv2d(isNCHW, 0, *input, *gradO, bS, iC, iH, iW, oC, oH, oW, indIOioC, indIiH, indWiC, indWmC, indWkH, indOoH);
    mC = weights->sizeAt(1);

    auto handle = reinterpret_cast<hipdnnHandle_t *>(context->getCuDnnHandle());
    hipdnnStatus_t err = hipdnnSetStream(*handle, *context->getCudaStream());
    if (err != 0) throw sd::cuda_exception::build("depthwiseConv2dBpCUDNN: can't set stream for cuDNN", err);

    hipdnnTensorFormat_t format = isNCHW ? HIPDNN_TENSOR_NCHW : HIPDNN_TENSOR_NHWC;

    // input descriptor
    hipdnnTensorDescriptor_t x;
    hipdnnCreateTensorDescriptor(&x);
    if(input->ews() == 1 && input->ordering() == 'c')
        err = hipdnnSetTensor4dDescriptor(x, format, cudnnDataType(input->dataType()), bS, iC, iH, iW);
    else
        err = hipdnnSetTensor4dDescriptorEx(x, cudnnDataType(input->dataType()), bS, iC, iH, iW, input->strideAt(0), input->strideAt(indIOioC), input->strideAt(indIiH), input->strideAt(indIiH + 1));
    if (err != 0) throw sd::cuda_exception::build("depthwiseConv2dBpCUDNN: hipdnnSetTensor4dDescriptor/hipdnnSetTensor4dDescriptorEx for input failed", err);

    // gradO descriptor
    hipdnnTensorDescriptor_t dz;
    hipdnnCreateTensorDescriptor(&dz);
    if(gradO->ews() == 1 && gradO->ordering() == 'c')
        err = hipdnnSetTensor4dDescriptor(dz, format, cudnnDataType(gradO->dataType()), bS, oC, oH, oW);
    else
        err = hipdnnSetTensor4dDescriptorEx(dz, cudnnDataType(gradO->dataType()), bS, oC, oH, oW, gradO->strideAt(0), gradO->strideAt(indIOioC), gradO->strideAt(indOoH), gradO->strideAt(indOoH + 1));
    if (err != 0) throw sd::cuda_exception::build("depthwiseConv2dBpCUDNN: hipdnnSetTensor4dDescriptor/hipdnnSetTensor4dDescriptorEx for gradO failed", err);

    // gradI descriptor
    hipdnnTensorDescriptor_t dx;
    hipdnnCreateTensorDescriptor(&dx);
    if(gradI->ews() == 1 && gradI->ordering() == 'c')
        err = hipdnnSetTensor4dDescriptor(dx, format, cudnnDataType(gradI->dataType()), bS, iC, iH, iW);
    else
        err = hipdnnSetTensor4dDescriptorEx(dx, cudnnDataType(gradI->dataType()), bS, iC, iH, iW, gradI->strideAt(0), gradI->strideAt(indIOioC), gradI->strideAt(indIiH), gradI->strideAt(indIiH + 1));
    if (err != 0) throw sd::cuda_exception::build("depthwiseConv2dBpCUDNN: hipdnnSetTensor4dDescriptor/hipdnnSetTensor4dDescriptorEx for gradI failed", err);

    // gradW descriptor
    hipdnnFilterDescriptor_t dw;
    hipdnnCreateFilterDescriptor(&dw);
    err = hipdnnSetFilter4dDescriptor(dw, cudnnDataType(gradW->dataType()), HIPDNN_TENSOR_NCHW, iC, mC, kH, kW);
    if(err != 0) throw sd::cuda_exception::build("depthwiseConv2dBpCUDNN: hipdnnSetFilter4dDescriptor gradW failed", err);

    // description of convolution
    hipdnnConvolutionDescriptor_t conv;
    hipdnnCreateConvolutionDescriptor(&conv);
    err = hipdnnSetConvolution2dDescriptor(conv, pH, pW, sH, sW, dH, dW, HIPDNN_CROSS_CORRELATION, cudnnDataType(gradO->dataType()));
    if (err != 0) throw sd::cuda_exception::build("depthwiseConv2dBpCUDNN: hipdnnSetConvolution2dDescriptor failed", err);
    err = hipdnnSetConvolutionGroupCount(conv, iC);  // set number of groups (depthwise mode) in description of convolution, groupCount == iC
    if (err != 0) throw sd::cuda_exception::build("depthwiseConv2dBpCUDNN: hipdnnSetConvolutionGroupCount failed", err);

    // gradW algorithm description
    hipdnnConvolutionBwdFilterAlgo_t algoGradW;
    hipdnnConvolutionBwdFilterAlgoPerf_t algoGradWPerf;
    int count = 0;
    //err = hipdnnGetConvolutionBackwardFilterAlgorithm(*handle, x, dz, conv, dw, HIPDNN_CONVOLUTION_BWD_FILTER_PREFER_FASTEST, 0, &algoGradW);
    err = hipdnnFindConvolutionBackwardFilterAlgorithm(*handle, x, dz, conv, dw, 1, &count, &algoGradWPerf);
    if (err != 0 || count == 0) throw sd::cuda_exception::build("depthwiseConv2dBpCUDNN: hipdnnGetConvolutionBackwardFilterAlgorithm failed", err);
    algoGradW = algoGradWPerf.algo;

    // gradI algorithm description
    hipdnnConvolutionBwdDataAlgo_t algoGradI;
    hipdnnConvolutionBwdDataAlgoPerf_t algoGradIPerf;
    //err = hipdnnGetConvolutionBackwardDataAlgorithm(*handle, dw, dz, conv, x, HIPDNN_CONVOLUTION_BWD_DATA_PREFER_FASTEST, 0, &algoGradI);
    err = hipdnnFindConvolutionBackwardDataAlgorithm(*handle, dw, dz, conv, x, 1, &count, &algoGradIPerf);
    if (err != 0 || count == 0) throw sd::cuda_exception::build("depthwiseConv2dBpCUDNN: hipdnnGetConvolutionBackwardDataAlgorithm failed", err);
    algoGradI = algoGradIPerf.algo;

    // allocate auxiliary device memory for gradW calculation, abbreviation ws means workspace
    size_t wsGradWSize;
    err = hipdnnGetConvolutionBackwardFilterWorkspaceSize(*handle, x, dz, conv, dw, algoGradW, &wsGradWSize);
    if (err != 0) throw sd::cuda_exception::build("depthwiseConv2dBpCUDNN: hipdnnGetConvolutionBackwardFilterWorkspaceSize failed", err);
    void* wsGradWData;
    auto cudaErr = hipMalloc(&wsGradWData, wsGradWSize);
    if (cudaErr != 0) throw sd::cuda_exception::build("depthwiseConv2dBpCUDNN: hipMalloc for auxiliary workspace memory wsGradWData failed", cudaErr);

    // allocate auxiliary device memory for gradI calculation, abbreviation ws means workspace
    size_t wsGradISize;
    err = hipdnnGetConvolutionBackwardDataWorkspaceSize(*handle, dw, dz, conv, dx, algoGradI, &wsGradISize);
    if (err != 0) throw sd::cuda_exception::build("depthwiseConv2dBpCUDNN: hipdnnGetConvolutionBackwardDataWorkspaceSize failed", err);
    void* wsGradIData;
    cudaErr = hipMalloc(&wsGradIData, wsGradISize);
    if (cudaErr != 0) throw sd::cuda_exception::build("depthwiseConv2dBpCUDNN: hipMalloc for auxiliary workspace memory wsGradIData failed", cudaErr);

    // provide scaling parameters
    const float  alpha32(1), beta32(0);
    const double alpha64(1), beta64(0);
    const void* alpha = gradO->sizeOfT() <= 4 ? reinterpret_cast<const void*>(&alpha32) : reinterpret_cast<const void*>(&alpha64);
    const void* beta  = gradO->sizeOfT() <= 4 ? reinterpret_cast<const void*>(&beta32)  : reinterpret_cast<const void*>(&beta64);

    NDArray::prepareSpecialUse({gradI, gradW, gradB}, {input, weights, gradO});

    // run calculation for gradB (if not nullptr)
    if(gradB != nullptr) {
        hipdnnTensorDescriptor_t db;
        hipdnnCreateTensorDescriptor(&db);
        // err = hipdnnSetTensor4dDescriptor(db, format, cudnnDataType(gradB->dataType()), 1, isNCHW ? gradB->lengthOf() : 1, 1, isNCHW ? 1: gradB->lengthOf());
        err = hipdnnSetTensor4dDescriptor(db, HIPDNN_TENSOR_NCHW, cudnnDataType(gradB->dataType()), 1, oC, 1, 1);
        if (err != 0) throw sd::cuda_exception::build("depthwiseConv2dBpCUDNN: hipdnnSetTensor4dDescriptor for gradB failed", err);

        err = hipdnnConvolutionBackwardBias(*handle, alpha, dz, gradO->specialBuffer(), beta, db, gradB->specialBuffer());
        if (err != 0) throw sd::cuda_exception::build("depthwiseConv2dBpCUDNN: hipdnnConvolutionBackwardBias failed", err);
    }

    // run calculation for gradW
    err = hipdnnConvolutionBackwardFilter(*handle, alpha, x, input->specialBuffer(), dz, gradO->specialBuffer(), conv, algoGradW, wsGradWData, wsGradWSize, beta, dw, gradW->specialBuffer());
    if (err != 0) throw sd::cuda_exception::build("depthwiseConv2dBpCUDNN: hipdnnConvolutionBackwardFilter failed", err);

    // run calculation for gradI
    err = hipdnnConvolutionBackwardData(*handle, alpha, dw, weights->specialBuffer(), dz, gradO->specialBuffer(), conv, algoGradI, wsGradIData, wsGradISize, beta, dx, gradI->specialBuffer());
    if (err != 0) throw sd::cuda_exception::build("depthwiseConv2dBpCUDNN: hipdnnConvolutionBackwardData failed", err);

    // cudaErr = hipStreamSynchronize(*context->getCudaStream());
    // if (cudaErr != 0)
    //     throw cuda_exception::build("depthwiseConv2dBpCUDNN: hipStreamSynchronize failed !", cudaErr);

    cudaErr = hipFree(wsGradWData);
    if (cudaErr != 0) throw sd::cuda_exception::build("depthwiseConv2dBpCUDNN: hipFree for auxiliary workspace memory wsGradWData failed", cudaErr);
    cudaErr = hipFree(wsGradIData);
    if (cudaErr != 0) throw sd::cuda_exception::build("depthwiseConv2dBpCUDNN: hipFree for auxiliary workspace memory wsGradIData failed", cudaErr);

    NDArray::registerSpecialUse({gradI, gradW, gradB}, {input, weights, gradO});
}

//////////////////////////////////////////////////////////////////////////
PLATFORM_IMPL(depthwise_conv2d, ENGINE_CUDA) {

    auto input   = INPUT_VARIABLE(0);                                    // [bS, iH, iW, iC] (NHWC) or [bS, iC, iH, iW] (NCHW)
    auto weights = INPUT_VARIABLE(1);                                    // [kH, kW, iC, mC], [mC, iC, kH, kW], [mC, kH, kW, iC]
    auto bias    = block.width() > 2 ? INPUT_VARIABLE(2) : nullptr;      // [oC] = iC*mC

    auto output  = OUTPUT_VARIABLE(0);                                   // [bS, oH, oW, iC*mC] (NHWC) or [bS, iC*mC, oH, oW] (NCHW)

    REQUIRE_TRUE(input->rankOf()   == 4, 0, "DEPTHWISECONV2D CUDNN OP: rank of input array must be equal to 4, but got %i instead !", input->rankOf());
    REQUIRE_TRUE(weights->rankOf() == 4, 0, "DEPTHWISECONV2D CUDNN OP: rank of weights array must be equal to 4, but got %i instead !", weights->rankOf());

    int kH = INT_ARG(0) > 0 ? INT_ARG(0) : static_cast<int>(weights->sizeAt(0));// filter(kernel) height
    int kW = INT_ARG(1) > 0 ? INT_ARG(1) : static_cast<int>(weights->sizeAt(1));// filter(kernel) width
    int sH = INT_ARG(2);                                                        // strides height
    int sW = INT_ARG(3);                                                        // strides width
    int pH = INT_ARG(4);                                                        // paddings height
    int pW = INT_ARG(5);                                                        // paddings width
    int dH = INT_ARG(6);                                                        // dilations height
    int dW = INT_ARG(7);                                                        // dilations width
    int paddingMode = INT_ARG(8);                                               // 0-VALID, 1-SAME
    int isNCHW      = block.getIArguments()->size() > 9 ? !INT_ARG(9) : 1;      // INT_ARG(9): 0-NCHW,  1-NHWC
    int wFormat = block.getIArguments()->size() > 10 ? INT_ARG(10) : 0;         // 0 - [kH, kW, iC, mC], 1 - [mC, iC, kH, kW], 2 - [mC, kH, kW, iC]

    int bS, iC, iH, iW, mC, oC, oH, oW;                     // batch size, input channels, input height/width, channels multiplier(oC = iC*mC), output channels, output height/width
    int indIOioC, indIiH, indWmC, indWiC, indWkH, indOoH;   // corresponding indexes
    ConvolutionUtils::getSizesAndIndexesConv2d(isNCHW, wFormat, *input, *output, bS, iC, iH, iW, oC, oH, oW, indIOioC, indIiH, indWiC, indWmC, indWkH, indOoH);
    mC = weights->sizeAt(indWmC);                           // channels multiplier

    ConvolutionUtils::calcPadding2D(pH, pW, oH, oW, iH, iW, kH, kW, sH, sW, dH, dW, paddingMode);

    std::vector<Nd4jLong> expectedWeightsShape = ConvolutionUtils::expectWeightsShape(wFormat, kH, kW, iC, mC);
    REQUIRE_TRUE(weights->isSameShape(expectedWeightsShape), 0, "DEPTHWISECONV2D CUDNN OP: wrong shape of weights array, expected is %s, but got %s instead !", ShapeUtils::shapeAsString(expectedWeightsShape).c_str(), ShapeUtils::shapeAsString(weights).c_str());
    REQUIRE_TRUE(output->sizeAt(indIOioC) == iC*mC, 0, "DEPTHWISECONV2D CUDNN OP: the output_channels must be equal to input_channels * channels_multiplier = %i !", iC*mC);
    if (bias)
        REQUIRE_TRUE(bias->rankOf() <= 2 && oC == bias->lengthOf(), 0, "DEPTHWISECONV2D CUDNN OP: wrong shape of array with biases, expected rank, length: <=2, %i, but got %i, %i instead !", oC, bias->rankOf(), bias->lengthOf());

    std::vector<int> wPermut;     // cudnn support format {oC, iC/groupCount, kH, kW} only, mC = 1, oC = iC (groupCount == iC) that is {iC, mC, kH, kW} in our case
    if(0 == wFormat)
        wPermut = {2,3,0,1};         // kH, kW, iC, mC -> iC, mC, kH, kW
    else if(1 == wFormat)
        wPermut = {1,0,2,3};         // mC, iC, kH, kW -> iC, mC, kH, kW
    else
        wPermut = {3,0,1,2};         // mC, kH, kW, iC -> iC, mC, kH, kW

    NDArray* newWeights = new NDArray(weights->ordering(), {iC, mC, kH, kW}, weights->dataType(), weights->getContext());
    newWeights->assign(weights->permute(wPermut));

    NDArray* newInput = input;
    NDArray* newGradI = nullptr;
    if(paddingMode == 1) // in same paddingMode cudnn doesn't support asymmetric left/right top/bottopm paddings
        checkConv2dCUDNNPadAsymmetric(newInput, newGradI, iH, iW, oH, oW, kH, kW, sH, sW, pH, pW, dH, dW, isNCHW);

    depthwiseConv2dCUDNN(block.launchContext(), newInput, newWeights, bias, output, kH,kW,sH,sW,pH,pW,dH,dW, paddingMode, isNCHW);

    if(newInput != input)
        delete newInput;

    delete newWeights;

    return Status::OK();
}

//////////////////////////////////////////////////////////////////////////
PLATFORM_CHECK(depthwise_conv2d, ENGINE_CUDA) {

    auto input   = INPUT_VARIABLE(0);                                    // [bS, iH, iW, iC] (NHWC) or [bS, iC, iH, iW] (NCHW)
    auto weights = INPUT_VARIABLE(1);                                    // [kH, kW, iC, mC], [mC, iC, kH, kW], [mC, kH, kW, iC]
    auto bias    = block.width() > 2 ? INPUT_VARIABLE(2) : nullptr;      // [oC] = iC*mC

    const int paddingMode = INT_ARG(8);                                  // 0-VALID, 1-SAME, 2-CAUSAL
    const int wFormat = block.getIArguments()->size() > 10 ? INT_ARG(10) : 0;       // 0 - [kH, kW, iC, mC], 1 - [mC, iC, kH, kW], 2 - [mC, kH, kW, iC]

    const int mC = weights->sizeAt(0 == wFormat ? 3 : 0);

    const bool badInputType   = input->dataType()   != DataType::DOUBLE && input->dataType()   != DataType::FLOAT32 && input->dataType()   != DataType::HALF;
    const bool badWeightsType = weights->dataType() != DataType::DOUBLE && weights->dataType() != DataType::FLOAT32 && weights->dataType() != DataType::HALF;
    const bool badBiasType    = bias == nullptr ? false : (bias->dataType() != DataType::DOUBLE && bias->dataType() != DataType::FLOAT32 && bias->dataType() != DataType::HALF);

    return mC == 1 && paddingMode != 2 && !badInputType && !badWeightsType && !badBiasType;
}

//////////////////////////////////////////////////////////////////////////
PLATFORM_IMPL(depthwise_conv2d_bp, ENGINE_CUDA) {

    auto input   = INPUT_VARIABLE(0);                                                // [bS, iH, iW, iC] (NDHWC) or [bS, iC, iH, iW] (NCDHW)
    auto weights = INPUT_VARIABLE(1);                                                // [kH, kW, iC, mC], [mC, iC, kH, kW], [mC, kH, kW, iC]
    auto bias    = block.width() > 3 ? INPUT_VARIABLE(2) : nullptr;                  // [oC] = [iC*mC]
    auto gradO   = block.width() > 3 ? INPUT_VARIABLE(3) : INPUT_VARIABLE(2);        // [bS, oH, oW, oC] (NDHWC) or [bS, oC, oH, oW] (NCDHW), epsilon_next

    auto gradI = OUTPUT_VARIABLE(0);                                                 // [bS, iH, iW, iC] (NDHWC) or [bS, iC, iH, iW] (NCDHW), epsilon
    auto gradW = OUTPUT_VARIABLE(1);                                                 // [kH, kW, iC, mC], [mC, iC, kH, kW], [mC, kH, kW, iC]
    auto gradB = block.width() > 3 ? OUTPUT_VARIABLE(2) : nullptr;                   // [oC]

    REQUIRE_TRUE(input->rankOf()   == 4, 0, "DEPTHWISECONV2D_BP CUDNN OP: rank of input array must be equal to 4, but got %i instead !", input->rankOf());
    REQUIRE_TRUE(weights->rankOf() == 4, 0, "DEPTHWISECONV2D_BP CUDNN OP: rank of weights array must be equal to 4, but got %i instead !", weights->rankOf());
    REQUIRE_TRUE(gradO->rankOf() == 4, 0,   "DEPTHWISECONV2D_BP CUDNN OP: rank of output gradients (next epsilon) array must be equal to 4, but got %i instead !", gradO->rankOf());

    int kH = INT_ARG(0) > 0 ? INT_ARG(0) : static_cast<int>(weights->sizeAt(0));// filter(kernel) height
    int kW = INT_ARG(1) > 0 ? INT_ARG(1) : static_cast<int>(weights->sizeAt(1));// filter(kernel) width
    int sH = INT_ARG(2);                                                        // strides height
    int sW = INT_ARG(3);                                                        // strides width
    int pH = INT_ARG(4);                                                        // paddings height
    int pW = INT_ARG(5);                                                        // paddings width
    int dH = INT_ARG(6);                                                        // dilations height
    int dW = INT_ARG(7);                                                        // dilations width
    int paddingMode = INT_ARG(8);                                               // 0-VALID, 1-SAME
    int isNCHW  = block.getIArguments()->size() > 9 ? !INT_ARG(9) : 1;          // INT_ARG(9): 1-NHWC, 0-NCHW
    int wFormat = block.getIArguments()->size() > 10 ? INT_ARG(10) : 0;         // 0 - [kH, kW, iC, mC], 1 - [mC, iC, kH, kW], 2 - [mC, kH, kW, iC]

    int bS, iC, iH, iW, mC, oC, oH, oW;                     // batch size, input channels, input height/width, channels multiplier(oC = iC*mC), output channels, output height/width
    int indIOioC, indIiH, indWmC, indWiC, indWkH, indOoH;   // corresponding indexes
    ConvolutionUtils::getSizesAndIndexesConv2d(isNCHW, wFormat, *input, *gradO, bS, iC, iH, iW, oC, oH, oW, indIOioC, indIiH, indWiC, indWmC, indWkH, indOoH);
    mC = weights->sizeAt(indWmC);                           // channels multiplier

    int trueoH, trueoW;          // correct output height, width
    ConvolutionUtils::calcOutSizePool2D(trueoH, trueoW, kH, kW, sH, sW, pH, pW, dH, dW, iH, iW, paddingMode);

    ConvolutionUtils::calcPadding2D(pH, pW, oH, oW, iH, iW, kH, kW, sH, sW, dH, dW, paddingMode);

    std::vector<Nd4jLong> expectedGradOShape   = ShapeUtils::composeShapeUsingDimsAndIdx({bS,oC,trueoH,trueoW,  0,indIOioC,indOoH,indOoH+1});
    std::vector<Nd4jLong> expectedWeightsShape = ConvolutionUtils::expectWeightsShape(wFormat, kH, kW, iC, mC);
    REQUIRE_TRUE(gradO->isSameShape(expectedGradOShape), 0,  "DEPTHWISECONV2D_BP CUDNN OP: wrong shape of output gradients (next epsilon) array, expected is %s, but got %s instead !", ShapeUtils::shapeAsString(expectedGradOShape).c_str(), ShapeUtils::shapeAsString(gradO).c_str());
    REQUIRE_TRUE(weights->isSameShape(expectedWeightsShape), 0, "DEPTHWISECONV2D_BP CUDNN OP: wrong shape of weights array, expected is %s, but got %s instead !", ShapeUtils::shapeAsString(expectedWeightsShape).c_str(), ShapeUtils::shapeAsString(weights).c_str());
    if(bias)
        REQUIRE_TRUE(bias->rankOf() <= 2 && oC == bias->lengthOf(), 0, "DEPTHWISECONV2D_BP CUDNN OP: wrong shape of array with biases, expected rank, length: <=2, %i, but got %i, %i instead !", oC, bias->rankOf(), bias->lengthOf());

    std::vector<int> wPermut, gradWPermut;     // cudnn support format {oC, iC/groupCount, kH, kW} only, mC = 1, oC = iC (groupCount == iC) that is {iC, mC, kH, kW}
    if(0 == wFormat) {
        wPermut = {2,3,0,1};         // kH, kW, iC, mC -> iC, mC, kH, kW
        gradWPermut = {2,3,0,1};     // iC, mC, kH, kW -> kH, kW, iC, mC
    }
    else if(1 == wFormat) {
        wPermut = {1,0,2,3};         // mC, iC, kH, kW -> iC, mC, kH, kW
        gradWPermut = {1,0,2,3};     // iC, mC, kH, kW -> mC, iC, kH, kW
    }
    else {
        wPermut = {3,0,1,2};         // mC, kH, kW, iC -> iC, mC, kH, kW
        gradWPermut = {1,2,3,0};     // iC, mC, kH, kW -> mC, kH, kW, iC
    }

    NDArray* newGradW   = new NDArray(gradW->ordering(),   {iC, mC, kH, kW}, gradW->dataType(),   gradW->getContext());
    NDArray* newWeights = new NDArray(weights->ordering(), {iC, mC, kH, kW}, weights->dataType(), weights->getContext());

    newWeights->assign(weights->permute(wPermut));

    NDArray* newInput = input;
    NDArray* newGradI = gradI;
    if(paddingMode == 1) // in same paddingMode cudnn doesn't support asymmetric left/right top/bottopm paddings
        checkConv2dCUDNNPadAsymmetric(newInput, newGradI, iH, iW, oH, oW, kH, kW, sH, sW, pH, pW, dH, dW, isNCHW);

    depthwiseConv2dBpCUDNN(block.launchContext(), newInput, newWeights, gradO,   newGradI, newGradW, gradB, kH,kW,sH,sW,pH,pW,dH,dW,paddingMode,isNCHW);

    newGradW->permutei(gradWPermut);
    gradW->assign(newGradW);

    if(newInput != input) {

        if(isNCHW)
            gradI->assign((*newGradI)({0,0,  0,0,  0,gradI->sizeAt(2),  0,gradI->sizeAt(3)}));
        else
            gradI->assign((*newGradI)({0,0,  0,gradI->sizeAt(1),  0,gradI->sizeAt(2),  0,0}));

        delete newInput;
        delete newGradI;
    }

    delete newWeights;
    delete newGradW;

    return Status::OK();
}

PLATFORM_CHECK(depthwise_conv2d_bp, ENGINE_CUDA) {

    auto input   = INPUT_VARIABLE(0);                                                // [bS, iH, iW, iC] (NDHWC) or [bS, iC, iH, iW] (NCDHW)
    auto weights = INPUT_VARIABLE(1);                                                // [kH, kW, iC, mC], [mC, iC, kH, kW], [mC, kH, kW, iC]
    auto bias    = block.width() > 3 ? INPUT_VARIABLE(2) : nullptr;                  // [oC] = [iC*mC]
    auto gradO   = block.width() > 3 ? INPUT_VARIABLE(3) : INPUT_VARIABLE(2);        // [bS, oH, oW, oC] (NDHWC) or [bS, oC, oH, oW] (NCDHW), epsilon_next

    const int paddingMode = INT_ARG(8);                                             // 0-VALID, 1-SAME, 2-CAUSAL
    const int isNCHW      = block.getIArguments()->size() > 9 ? !INT_ARG(9) : 1;    // INT_ARG(9): 0-NCHW, 1-NHWC
    const int wFormat = block.getIArguments()->size() > 10 ? INT_ARG(10) : 0;       // 0 - [kH, kW, iC, mC], 1 - [mC, iC, kH, kW], 2 - [mC, kH, kW, iC]

    const int mC = weights->sizeAt(0 == wFormat ? 3 : 0);

    const bool badInputType   = input->dataType()   != DataType::DOUBLE && input->dataType()   != DataType::FLOAT32 && input->dataType()   != DataType::HALF;
    const bool badWeightsType = weights->dataType() != DataType::DOUBLE && weights->dataType() != DataType::FLOAT32 && weights->dataType() != DataType::HALF;
    const bool badGradOType   = gradO->dataType()   != DataType::DOUBLE && gradO->dataType()   != DataType::FLOAT32 && gradO->dataType()   != DataType::HALF;
    const bool badBiasType    = bias == nullptr ? false : (bias->dataType() != DataType::DOUBLE && bias->dataType() != DataType::FLOAT32 && bias->dataType() != DataType::HALF);

    return mC == 1 && isNCHW && paddingMode != 2 && !badInputType && !badWeightsType && !badGradOType && !badBiasType;
}


}
}
}
