/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma (iuriish@yahoo.com)
//


#include "cudnnUtils.h"
#include <ops/declarable/helpers/convolutions.h>

namespace sd      {
namespace ops       {
namespace platforms {

//////////////////////////////////////////////////////////////////////////
static void batchnormCUDNN(const LaunchContext* context,
                            const NDArray* input, const NDArray* mean, const NDArray* variance,
                            const NDArray* gamma, const NDArray* beta,
                                  NDArray* output,
                            const double epsilon, const bool isSpatialMode) {


    // input, output -> 4D:nchw, 5D:ncdhw
    // mean, variance, gamma, beta -> 1xCx1x1 for 4D and 1xCx1x1x1 for 5D for BATCHNORM_MODE_SPATIAL mode
    //                             -> 1xCxHxW for 4D and 1xCxDxHxW for 5D for BATCHNORM_MODE_PER_ACTIVATION mode

    const hipdnnDataType_t dataType = cudnnDataType(input->dataType());

    const int xRank = input->rankOf();

    auto handle = reinterpret_cast<hipdnnHandle_t *>(context->getCuDnnHandle());
    hipdnnStatus_t err = hipdnnSetStream(*handle, *context->getCudaStream());
    if (err != 0) throw sd::cuda_exception::build("conv2dCUDNN: can't set stream for cuDNN", err);

    const std::vector<int> xShape = input->getShapeAsVectorInt();               // input and output have same shapes

    std::vector<int> paramsShape, paramsStrides;                                 // mean, variance, gamma and beta have same shapes
    if(isSpatialMode) { // 1xCx1x1
        const int iC = mean->lengthOf();
        const int stride0 = mean->strideAt(0);
        paramsShape   = xRank == 4 ? std::vector<int>({1, iC, 1, 1}) : std::vector<int>({1, iC, 1, 1, 1});
        paramsStrides = xRank == 4 ? std::vector<int>({iC*stride0, stride0, 1, 1}) : std::vector<int>({iC*stride0, stride0, 1, 1, 1});
    }
    else {
        paramsShape = mean->getShapeAsVectorInt();
        paramsStrides = xRank == 4 ? std::vector<int>({(int)mean->strideAt(0), (int)mean->strideAt(1), (int)mean->strideAt(2), (int)mean->strideAt(3)}) : std::vector<int>({(int)mean->strideAt(0), (int)mean->strideAt(1), (int)mean->strideAt(2), (int)mean->strideAt(3), (int)mean->strideAt(4)});
    }

    std::vector<int> xStrides = {(int)input->strideAt(0),  (int)input->strideAt(1),  (int)input->strideAt(2),  (int)input->strideAt(3)};
    std::vector<int> zStrides = {(int)output->strideAt(0), (int)output->strideAt(1), (int)output->strideAt(2), (int)output->strideAt(3)};

    if(xRank > 4) { // 5D
        xStrides.push_back((int)input->strideAt(4));
        zStrides.push_back((int)output->strideAt(4));
    }

    hipdnnTensorFormat_t format = HIPDNN_TENSOR_NCHW;

     // input descriptor
    hipdnnTensorDescriptor_t x;
    hipdnnCreateTensorDescriptor(&x);
    if(input->ews() == 1)
        err = cudnnSetTensorNdDescriptorEx(x, format, dataType, xRank, xShape.data());
    else
        err = hipdnnSetTensorNdDescriptor(x, dataType, xRank, xShape.data(), xStrides.data());
    if (err != 0) throw sd::cuda_exception::build("batchnormCUDNN: hipdnnSetTensorNdDescriptor/cudnnSetTensorNdDescriptorEx for input failed", err);

    // output descriptor
    hipdnnTensorDescriptor_t z;
    hipdnnCreateTensorDescriptor(&z);
    if(output->ews() == 1)
        err = cudnnSetTensorNdDescriptorEx(z, format, dataType, xRank, xShape.data());
    else
        err = hipdnnSetTensorNdDescriptor(z, dataType, xRank, xShape.data(), zStrides.data());
    if (err != 0) throw sd::cuda_exception::build("batchnormCUDNN: hipdnnSetTensorNdDescriptor/cudnnSetTensorNdDescriptorEx for output failed", err);

    // mean, variance, gamma and beta descriptor, the same descriptor for all of them
    hipdnnTensorDescriptor_t params;
    hipdnnCreateTensorDescriptor(&params);
    if(mean->ews() == 1)
        err = cudnnSetTensorNdDescriptorEx(params, format, dataType, xRank, paramsShape.data());
    else
        err = hipdnnSetTensorNdDescriptor(params, dataType, xRank, paramsShape.data(), paramsStrides.data());
    if (err != 0) throw sd::cuda_exception::build("batchnormCUDNN: hipdnnSetTensorNdDescriptor/cudnnSetTensorNdDescriptorEx for mean/variance/gamma/beta failed", err);

    // provide scaling parameters
    const float  alpha32(1), beta32(0);
    const double alpha64(1), beta64(0);
    const void* ptrAlpha = output->sizeOfT() <= 4 ? reinterpret_cast<const void*>(&alpha32) : reinterpret_cast<const void*>(&alpha64);
    const void* ptrBeta  = output->sizeOfT() <= 4 ? reinterpret_cast<const void*>(&beta32)  : reinterpret_cast<const void*>(&beta64);

    NDArray::prepareSpecialUse({output}, {input, mean, variance, gamma, beta});

    // calculations
    err = hipdnnBatchNormalizationForwardInference(*handle, isSpatialMode ? HIPDNN_BATCHNORM_SPATIAL : HIPDNN_BATCHNORM_PER_ACTIVATION,
                                                 ptrAlpha, ptrBeta,
                                                 x, input->specialBuffer(),
                                                 z, output->specialBuffer(),
                                                 params,
                                                 gamma->specialBuffer(), beta->specialBuffer(),
                                                 mean->specialBuffer(), variance->specialBuffer(), epsilon);

    if (err != 0) throw sd::cuda_exception::build("batchnormCUDNN: hipdnnBatchNormalizationForwardInference failed", err);

    auto cudaErr = hipStreamSynchronize(*context->getCudaStream());
    if (cudaErr != 0)
        throw cuda_exception::build("batchnormCUDNN: hipStreamSynchronize failed !", cudaErr);

    NDArray::registerSpecialUse({output}, {input, mean, variance, gamma, beta});
}

//////////////////////////////////////////////////////////////////////////
static void batchnormBpCUDNN(const LaunchContext* context,
                            const NDArray* input, const NDArray* mean, const NDArray* variance, const NDArray* gamma, const NDArray* gradO,
                                  NDArray* gradI, NDArray* gradG, NDArray* gradB,
                            const double epsilon, const bool isSpatialMode) {

    // input, gradO, gradI -> 4D:nchw, 5D:ncdhw
    // mean, variance, gamma, beta, gradM, gradV, gradG, gradB -> 1xCx1x1 for 4D and 1xCx1x1x1 for 5D for BATCHNORM_MODE_SPATIAL mode
    //                                                         -> 1xCxHxW for 4D and 1xCxDxHxW for 5D for BATCHNORM_MODE_PER_ACTIVATION mode

    const hipdnnDataType_t dataType = cudnnDataType(input->dataType());

    const int xRank = input->rankOf();

    auto handle = reinterpret_cast<hipdnnHandle_t *>(context->getCuDnnHandle());
    hipdnnStatus_t err = hipdnnSetStream(*handle, *context->getCudaStream());
    if (err != 0) throw sd::cuda_exception::build("batchnormBpCUDNN: can't set stream for cuDNN", err);

    const std::vector<int> xShape = input->getShapeAsVectorInt();               // input and output have same shapes

    std::vector<int> paramsShape, paramsStrides;                                 // mean, variance, gamma and beta have same shapes
    if(isSpatialMode) { // 1xCx1x1
        const int iC = mean->lengthOf();
        const int stride0 = mean->strideAt(0);
        paramsShape   = xRank == 4 ? std::vector<int>({1, iC, 1, 1}) : std::vector<int>({1, iC, 1, 1, 1});
        paramsStrides = xRank == 4 ? std::vector<int>({iC*stride0, stride0, 1, 1}) : std::vector<int>({iC*stride0, stride0, 1, 1, 1});
    }
    else {
        paramsShape = mean->getShapeAsVectorInt();
        paramsStrides = xRank == 4 ? std::vector<int>({(int)mean->strideAt(0), (int)mean->strideAt(1), (int)mean->strideAt(2), (int)mean->strideAt(3)}) : std::vector<int>({(int)mean->strideAt(0), (int)mean->strideAt(1), (int)mean->strideAt(2), (int)mean->strideAt(3), (int)mean->strideAt(4)});
    }

    std::vector<int> xStrides = {(int)input->strideAt(0),  (int)input->strideAt(1),  (int)input->strideAt(2),  (int)input->strideAt(3)};
    std::vector<int> dxStrides = {(int)gradI->strideAt(0),  (int)gradI->strideAt(1),  (int)gradI->strideAt(2),  (int)gradI->strideAt(3)};
    std::vector<int> dzStrides = {(int)gradO->strideAt(0), (int)gradO->strideAt(1), (int)gradO->strideAt(2), (int)gradO->strideAt(3)};

    if(xRank > 4) { // 5D
        xStrides.push_back((int)input->strideAt(4));
        dxStrides.push_back((int)gradI->strideAt(4));
        dzStrides.push_back((int)gradO->strideAt(4));
    }

    hipdnnTensorFormat_t format = HIPDNN_TENSOR_NCHW;

     // input descriptor
    hipdnnTensorDescriptor_t x;
    hipdnnCreateTensorDescriptor(&x);
    if(input->ews() == 1)
        err = cudnnSetTensorNdDescriptorEx(x, format, dataType, xRank, xShape.data());
    else
        err = hipdnnSetTensorNdDescriptor(x, dataType, xRank, xShape.data(), xStrides.data());
    if (err != 0) throw sd::cuda_exception::build("batchnormBpCUDNN: hipdnnSetTensorNdDescriptor/cudnnSetTensorNdDescriptorEx for input failed", err);

    // gradO descriptor
    hipdnnTensorDescriptor_t dz;
    hipdnnCreateTensorDescriptor(&dz);
    if(gradO->ews() == 1)
        err = cudnnSetTensorNdDescriptorEx(dz, format, dataType, xRank, xShape.data());
    else
        err = hipdnnSetTensorNdDescriptor(dz, dataType, xRank, xShape.data(), dzStrides.data());
    if (err != 0) throw sd::cuda_exception::build("batchnormBpCUDNN: hipdnnSetTensorNdDescriptor/cudnnSetTensorNdDescriptorEx for gradO failed", err);

    // gradI descriptor
    hipdnnTensorDescriptor_t dx;
    hipdnnCreateTensorDescriptor(&dx);
    if(input->ews() == 1)
        err = cudnnSetTensorNdDescriptorEx(dx, format, dataType, xRank, xShape.data());
    else
        err = hipdnnSetTensorNdDescriptor(dx, dataType, xRank, xShape.data(), dxStrides.data());
    if (err != 0) throw sd::cuda_exception::build("batchnormBpCUDNN: hipdnnSetTensorNdDescriptor/cudnnSetTensorNdDescriptorEx for gradI failed", err);

    // mean, variance, gamma, gradG and gradB descriptor, the same descriptor for all of them
    hipdnnTensorDescriptor_t params;
    hipdnnCreateTensorDescriptor(&params);
    if(mean->ews() == 1)
        err = cudnnSetTensorNdDescriptorEx(params, format, dataType, xRank, paramsShape.data());
    else
        err = hipdnnSetTensorNdDescriptor(params, dataType, xRank, paramsShape.data(), paramsStrides.data());
    if (err != 0) throw sd::cuda_exception::build("batchnormBpCUDNN: hipdnnSetTensorNdDescriptor/cudnnSetTensorNdDescriptorEx for mean/variance/gamma/gradG/gradB failed", err);

    // provide scaling parameters
    const float  alpha32(1), beta32(0);
     double alpha64(1), beta64(0);
    const void* ptrAlpha = input->sizeOfT() <= 4 ? reinterpret_cast<const void*>(&alpha32) : reinterpret_cast<const void*>(&alpha64);
    const void* ptrBeta  = input->sizeOfT() <= 4 ? reinterpret_cast<const void*>(&beta32)  : reinterpret_cast<const void*>(&beta64);

    NDArray::prepareSpecialUse({gradI, gradG, gradB}, {input, mean, variance, gamma, gradO});

    // calculations
    // TODO: we can use cache here
    err = hipdnnBatchNormalizationBackward(*handle, isSpatialMode ? HIPDNN_BATCHNORM_SPATIAL : HIPDNN_BATCHNORM_PER_ACTIVATION,
                                            ptrAlpha, ptrBeta, ptrAlpha, ptrBeta,
                                            x, input->specialBuffer(),
                                            dz, gradO->specialBuffer(),
                                            dx, gradI->specialBuffer(),
                                            params,
                                            gamma->specialBuffer(), gradG->specialBuffer(), gradB->specialBuffer(),
                                            epsilon,
                                            nullptr/*mean->specialBuffer()*/, nullptr/*variance->specialBuffer()*/);

    if (err != 0) throw sd::cuda_exception::build("batchnormBpCUDNN: hipdnnBatchNormalizationBackward failed", err);

    auto cudaErr = hipStreamSynchronize(*context->getCudaStream());
    if (cudaErr != 0)
        throw cuda_exception::build("batchnormBpCUDNN: hipStreamSynchronize failed !", cudaErr);

    NDArray::registerSpecialUse({gradI, gradG, gradB}, {input, mean, variance, gamma, gradO});
}


//////////////////////////////////////////////////////////////////////////
PLATFORM_IMPL(batchnorm, ENGINE_CUDA) {

    auto input    = INPUT_VARIABLE(0);
    auto mean     = INPUT_VARIABLE(1);
    auto variance = INPUT_VARIABLE(2);
    NDArray* gamma    = nullptr;
    NDArray* beta     = nullptr;

    auto output   = OUTPUT_VARIABLE(0);

    const bool   applyScale  = (bool)INT_ARG(0);
    const bool   applyOffset = (bool)INT_ARG(1);
    const double epsilon     = T_ARG(0);

    if(applyScale)
        gamma = INPUT_VARIABLE(3);
    if(applyOffset)
        beta = INPUT_VARIABLE(3 + (int)applyScale);

    const int numOfIntArgs = block.getIArguments()->size();
    const int inRank = input->rankOf();

    // get axes args to normalize input array over
    std::vector<int> axes;
    if(numOfIntArgs > 2)
        for(int i = 2; i < numOfIntArgs; ++i)
            axes.push_back(INT_ARG(i));
    else
        axes.push_back(inRank-1);               // default dimension to reduce along is last dimension

    const int numOfAxes = axes.size();
    REQUIRE_TRUE(numOfAxes <= inRank, 0, "BATCHNORM CUDNN op: too big number of input axes to normalize over, expected number should be less or equal to rank of input array, but got %i and %i correspondingly !", numOfAxes, inRank);

    // evaluate expected shape for mean, variance and gamma. These 3 arrays should have identical shapes
    // for example if input shape is {2,3,4,5,6} and axes = {1,3}, then expected shape would be {1,3,1,5,1}, and if axes = {3}, then expected shape would be {5}
    std::vector<Nd4jLong> expShape;
    if(numOfAxes == 1)
        expShape.push_back(input->sizeAt(axes[0]));
    else {      // get, for example, something like {1, inputDim1, 1, inputDim3, 1} if axes = {1, 3}
        expShape = std::vector<Nd4jLong>(inRank, 1);
        for(uint i = 0; i < numOfAxes; ++i)
            expShape[axes[i]] = input->sizeAt(axes[i]);
    }

    REQUIRE_TRUE(mean->isSameShape(expShape) , 0, "BATCHNORM CUDNN op: wrong shape of mean array, expected is %s, but got %s instead !", ShapeUtils::shapeAsString(expShape).c_str(), ShapeUtils::shapeAsString(mean).c_str());
    REQUIRE_TRUE(variance->isSameShape(expShape), 0, "BATCHNORM CUDNN op: wrong shape of variance array, expected is %s, but got %s instead !", ShapeUtils::shapeAsString(expShape).c_str(), ShapeUtils::shapeAsString(variance).c_str());
    if(gamma)
        REQUIRE_TRUE(gamma->isSameShape(expShape), 0, "BATCHNORM CUDNN op: wrong shape of gamma array, expected is %s, but got %s instead !", ShapeUtils::shapeAsString(expShape).c_str(), ShapeUtils::shapeAsString(gamma).c_str());
    if(beta)
        REQUIRE_TRUE(beta->isSameShape(expShape), 0, "BATCHNORM CUDNN op: wrong shape of beta array, expected is %s, but got %s instead !", ShapeUtils::shapeAsString(expShape).c_str(), ShapeUtils::shapeAsString(beta).c_str());

    // types of all input arrays should be the same
    for(int i = 1; i < block.width(); ++i)
        REQUIRE_TRUE(INPUT_VARIABLE(0)->dataType() == INPUT_VARIABLE(i)->dataType(), 0, "BATCHNORM CUDNN op: types of all input arrays should be the same !");

    // cudnn supports NCHW format only
    const bool needPermut = axes.size() == 1 && mean->lengthOf() == input->sizeAt(-1);

    if(needPermut) {    // if NHWC
        std::vector<int> perm = inRank == 4 ? std::vector<int>({0, 3, 1, 2}) : std::vector<int>({0, 4, 1, 2, 3});           // NHWC -> NCHW
        input  = new NDArray(input->permute(perm));
        output = new NDArray(output->permute(perm));
    }

    // cudnn requires gamma and beta to be non-nullptr
    if(!applyScale) {
        gamma = new NDArray(mean);
        *gamma = 1;
    }
    if(!applyOffset) {
        beta = new NDArray(mean);
        *beta = 0;
    }

    // calculations
    batchnormCUDNN(block.launchContext(), input, mean, variance, gamma, beta, output, epsilon, axes.size() == 1);

    if(needPermut) {
        delete input;
        delete output;
    }

    if(!applyScale)
        delete gamma;

    if(!applyOffset)
        delete beta;

    return Status::OK();
}

//////////////////////////////////////////////////////////////////////////
PLATFORM_CHECK(batchnorm, ENGINE_CUDA) {

    const bool   applyScale  = (bool)INT_ARG(0);
    const bool   applyOffset = (bool)INT_ARG(1);

    NDArray* input     = INPUT_VARIABLE(0);
    NDArray* mean      = INPUT_VARIABLE(1);
    NDArray* variance  = INPUT_VARIABLE(2);
    NDArray* gamma     = applyScale  ? INPUT_VARIABLE(3) : nullptr;
    NDArray* beta      = applyOffset ? INPUT_VARIABLE(3 + (int)applyScale) : nullptr;

    const int numOfIntArgs = block.getIArguments()->size();
    const int xRank = input->rankOf();

    // *********************************** //
    if(xRank != 4 && xRank != 5)
        return false;

    // *********************************** //
    const bool badType = input->dataType() != DataType::DOUBLE && input->dataType() != DataType::FLOAT32 && input->dataType() != DataType::HALF;
    if(badType)
        return false;

    // *********************************** //
    // get axes args to normalize input array over
    std::vector<int> axes;
    if(numOfIntArgs > 2)
        for(int i = 2; i < numOfIntArgs; ++i)
            axes.push_back(INT_ARG(i));
    else
        axes.push_back(xRank-1);               // default dimension to reduce along is last dimension

    if(axes.size() != 1 && axes.size() != 3 && axes.size() != 4)
        return false;

    // *********************************** //
    bool allParamsHaveSameShapeAndStrides = shape::haveSameShapeAndStrides(mean->shapeInfo(), variance->shapeInfo());
    if(gamma)
        allParamsHaveSameShapeAndStrides &= shape::haveSameShapeAndStrides(mean->shapeInfo(), gamma->shapeInfo());
    if(beta)
        allParamsHaveSameShapeAndStrides &= shape::haveSameShapeAndStrides(mean->shapeInfo(), beta->shapeInfo());

    if(!allParamsHaveSameShapeAndStrides)
        return false;

    // *********************************** //
    bool isFormatGood = false;
    if(axes.size() == 1)
        isFormatGood = mean->lengthOf() == input->sizeAt(1) || mean->lengthOf() == input->sizeAt(-1);   // mean [C]
    else {
        auto inputShapeModif = input->getShapeAsVector();     // [dim0,dim1,dim2,dim3] 4D or [dim0,dim1,dim2,dim3,dim4]
        inputShapeModif[0] = 1;
        isFormatGood = mean->isSameShape(inputShapeModif);    // mean [1,dim1,dim2,dim3] 4D or [1,dim1,dim2,dim3,dim4]
    }
    if(!isFormatGood)
        return false;

    return true;
}

//////////////////////////////////////////////////////////////////////////
PLATFORM_IMPL(batchnorm_bp, ENGINE_CUDA) {

    NDArray* input    = INPUT_VARIABLE(0);
    NDArray* mean     = INPUT_VARIABLE(1);
    NDArray* variance = INPUT_VARIABLE(2);
    NDArray* gamma    = nullptr;
    NDArray* beta     = nullptr;
    NDArray* gradO     = INPUT_VARIABLE(block.width() - 1);    // next epsilon

    NDArray* gradI = OUTPUT_VARIABLE(0);
    NDArray* gradM = OUTPUT_VARIABLE(1);
    NDArray* gradV = OUTPUT_VARIABLE(2);
    NDArray* gradG = nullptr;
    NDArray* gradB = nullptr;

    const bool   applyScale  = (bool)INT_ARG(0);
    const bool   applyOffset = (bool)INT_ARG(1);
    const float  epsilon     = T_ARG(0);

    if(applyScale) {
        gamma = INPUT_VARIABLE(3);
        gradG  = OUTPUT_VARIABLE(3);
    }
    if(applyOffset) {
        beta = INPUT_VARIABLE(3 + (int)applyScale);
        gradB = OUTPUT_VARIABLE(3 + (int)applyScale);
    }

    const int numOfIntArgs = block.getIArguments()->size();
    const int inRank = input->rankOf();

    // get axes args to normalize input array over
    std::vector<int> axes;
    if(numOfIntArgs > 2)
        for(int i = 2; i < numOfIntArgs; ++i)
            axes.push_back(INT_ARG(i));
    else
        axes.push_back(inRank-1);               // default dimension to reduce along is last dimension

    const int numOfAxes = axes.size();
    REQUIRE_TRUE(numOfAxes <= inRank, 0, "BATCHNORM_BP CUDNN op: too big number of input axes to normalize over, expected number should be less or equal to rank of input array, but got %i and %i correspondingly !", numOfAxes, inRank);

    // evaluate expected shape for mean, variance and gamma. These 3 arrays should have identical shapes
    // for example if input shape is {2,3,4,5,6} and axes = {1,3}, then expected shape would be {1,3,1,5,1}, and if axes = {3}, then expected shape would be {5}
    std::vector<Nd4jLong> expShape;
    if(numOfAxes == 1)
        expShape.push_back(input->sizeAt(axes[0]));
    else {      // get, for example, something like {1, inputDim1, 1, inputDim3, 1} if axes = {1, 3}
        expShape = std::vector<Nd4jLong>(inRank, 1);
        for(uint i = 0; i < numOfAxes; ++i)
            expShape[axes[i]] = input->sizeAt(axes[i]);
    }

    REQUIRE_TRUE(mean->isSameShape(expShape), 0, "BATCHNORM_BP CUDNN op: wrong shape of mean array, expected is %s, but got %s instead !", ShapeUtils::shapeAsString(expShape).c_str(), ShapeUtils::shapeAsString(mean).c_str());
    REQUIRE_TRUE(variance->isSameShape(expShape), 0, "BATCHNORM_BP CUDNN op: wrong shape of variance array, expected is %s, but got %s instead !", ShapeUtils::shapeAsString(expShape).c_str(), ShapeUtils::shapeAsString(variance).c_str());
    if(gamma)
        REQUIRE_TRUE(gamma->isSameShape(expShape), 0, "BATCHNORM_BP CUDNN op: wrong shape of gamma array, expected is %s, but got %s instead !", ShapeUtils::shapeAsString(expShape).c_str(), ShapeUtils::shapeAsString(gamma).c_str());
    if(beta)
        REQUIRE_TRUE(beta->isSameShape(expShape), 0, "BATCHNORM_BP CUDNN op: wrong shape of beta array, expected is %s, but got %s instead !", ShapeUtils::shapeAsString(expShape).c_str(), ShapeUtils::shapeAsString(beta).c_str());

    REQUIRE_TRUE(input->isSameShape(gradO), 0, "BATCHNORM_BP CUDNN op: wrong shape of output gradients array, expected is %s, but got %s instead !", ShapeUtils::shapeAsString(input).c_str(), ShapeUtils::shapeAsString(gradO).c_str());

    // types of all input arrays should be the same (except gradO)
    for(int i = 1; i < block.width() - 2; ++i)
        REQUIRE_TRUE(INPUT_VARIABLE(0)->dataType() == INPUT_VARIABLE(i)->dataType(), 0, "BATCHNORM_BP CUDNN op: types of arrays (input, mean, variance, gamma, beta) should be the same !");

    // cudnn supports NCHW format only
    const bool needPermut = axes.size() == 1 && mean->lengthOf() != input->sizeAt(1);

    if(needPermut) {    // if NHWC
        std::vector<int> perm = inRank == 4 ? std::vector<int>({0, 3, 1, 2}) : std::vector<int>({0, 4, 1, 2, 3});           // NHWC -> NCHW
        input = new NDArray(input->permute(perm));
        gradO = new NDArray(gradO->permute(perm));
        gradI = new NDArray(gradI->permute(perm));
    }

    // cudnn requires gamma, gradG, gradB to be non-nullptr
    if(!applyScale) {
        gamma = new NDArray(mean);
        gradG = new NDArray(mean);
        *gamma = 1;
    }
    if(!applyOffset)
        gradB = new NDArray(mean);

    // calculations
    batchnormBpCUDNN(block.launchContext(), input, mean, variance, gamma, gradO,   gradI, gradG, gradB, epsilon, axes.size() == 1);

    *gradM = 0;      // put zeros so far
    *gradV = 0;      // put zeros so far

    if(needPermut) {
        delete input;
        delete gradO;
        delete gradI;
    }

    if(!applyScale) {
        delete gamma;
        delete gradG;
    }

    if(!applyOffset)
        delete gradB;

    return Status::OK();

}

PLATFORM_CHECK(batchnorm_bp, ENGINE_CUDA) {

    NDArray* input    = INPUT_VARIABLE(0);
    NDArray* mean     = INPUT_VARIABLE(1);
    NDArray* variance = INPUT_VARIABLE(2);
    NDArray* gamma    = nullptr;
    NDArray* beta     = nullptr;
    NDArray* gradO    = INPUT_VARIABLE(block.width() - 1);    // next epsilon

    NDArray* gradI = OUTPUT_VARIABLE(0);
    NDArray* gradM = OUTPUT_VARIABLE(1);
    NDArray* gradV = OUTPUT_VARIABLE(2);
    NDArray* gradG = nullptr;
    NDArray* gradB = nullptr;

    const int numOfIntArgs = block.getIArguments()->size();
    const int xRank = input->rankOf();

    // *********************************** //
    if(xRank != 4 && xRank != 5)
        return false;

    // *********************************** //
    const bool badType = input->dataType() != DataType::DOUBLE && input->dataType() != DataType::FLOAT32 && input->dataType() != DataType::HALF;
    if(badType)
        return false;

    // *********************************** //
    // get axes args to normalize input array over
    std::vector<int> axes;
    if(numOfIntArgs > 2)
        for(int i = 2; i < numOfIntArgs; ++i)
            axes.push_back(INT_ARG(i));
    else
        axes.push_back(xRank-1);               // default dimension to reduce along is last dimension

    if(axes.size() != 1 && axes.size() != 3 && axes.size() != 4)
        return false;

    // *********************************** //
    bool allParamsHaveSameShapeAndStrides = shape::haveSameShapeAndStrides(mean->shapeInfo(), variance->shapeInfo());
    if(gamma)
        allParamsHaveSameShapeAndStrides &= shape::haveSameShapeAndStrides(mean->shapeInfo(), gamma->shapeInfo());
    if(gradG)
        allParamsHaveSameShapeAndStrides &= shape::haveSameShapeAndStrides(mean->shapeInfo(), gradG->shapeInfo());
    if(gradB)
        allParamsHaveSameShapeAndStrides &= shape::haveSameShapeAndStrides(mean->shapeInfo(), gradB->shapeInfo());

    if(!allParamsHaveSameShapeAndStrides)
        return false;

    // *********************************** //
    bool isFormatGood = false;
    if(axes.size() == 1)
        isFormatGood = mean->lengthOf() == input->sizeAt(1) || mean->lengthOf() == input->sizeAt(-1);   // mean [C]
    else {
        auto inputShapeModif = input->getShapeAsVector();     // [dim0,dim1,dim2,dim3] 4D or [dim0,dim1,dim2,dim3,dim4]
        inputShapeModif[0] = 1;
        isFormatGood = mean->isSameShape(inputShapeModif);    // mean [1,dim1,dim2,dim3] 4D or [1,dim1,dim2,dim3,dim4]
    }
    if(!isFormatGood)
        return false;

    return true;
}


}
}
}
