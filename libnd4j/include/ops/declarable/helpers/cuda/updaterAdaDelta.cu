#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Oleh Semeniv (oleg.semeniv@gmail.com)
//

#include <system/op_boilerplate.h>
#include <ops/declarable/helpers/updatersHelpers.h>
#include <helpers/PointersManager.h>
#include <math/platformmath.h>
#include <math/templatemath.h>

namespace sd    {
namespace ops     {
namespace helpers {

///////////////////////////////////////////////////////////////////
template<typename T>
__global__ void adaDeltaUpdaterCuda(const void* vx, const Nd4jLong* xShapeInfo, const void* vinMsg, const Nd4jLong* inMsgShapeInfo, 
    const void* vinMsdx, const Nd4jLong* inMsdxShapeInfo, void* vz, const Nd4jLong* zShapeInfo, void* vstMsg, 
    const Nd4jLong* stMsgShapeInfo, void* vstMsdx, const Nd4jLong* stMsdxShapeInfo, const T rho, const T epsilon) {

    const auto grad = reinterpret_cast<const T*>(vx);
    const auto initMsg= reinterpret_cast<const T*>(vinMsg);
    const auto initMsdx = reinterpret_cast<const T*>(vinMsdx);
   
    auto up = reinterpret_cast<T*>(vz);
    auto stMsg = reinterpret_cast<T*>(vstMsg);
    auto stMsdx = reinterpret_cast<T*>(vstMsdx);

    __shared__ Nd4jLong xLen;
    __shared__ T rhoT;
    __shared__ bool bEWS, bOrdering, bXZsame, bXInMsgSame, bXStMsgSame, bXInMsdxSame, bXStMsdxSame;

    if (threadIdx.x == 0) {
        xLen = shape::length(xShapeInfo);
        
        rhoT = (1 - rho);

        bEWS =  1 == shape::elementWiseStride(xShapeInfo) && 1 == shape::elementWiseStride(zShapeInfo) &&
                1 == shape::elementWiseStride(stMsgShapeInfo) && 1 == shape::elementWiseStride(inMsgShapeInfo) &&
                1 == shape::elementWiseStride(stMsdxShapeInfo) && 1 == shape::elementWiseStride(inMsdxShapeInfo);
        bOrdering = shape::order(xShapeInfo) == shape::order(zShapeInfo) && shape::order(zShapeInfo) == shape::order(stMsgShapeInfo) && 
                    shape::order(stMsgShapeInfo) == shape::order(inMsgShapeInfo) && shape::order(inMsgShapeInfo) == shape::order(stMsdxShapeInfo) &&
                    shape::order(stMsdxShapeInfo) == shape::order(inMsdxShapeInfo);

        bXZsame = shape::haveSameShapeAndStrides(xShapeInfo, zShapeInfo);
        bXInMsgSame = shape::haveSameShapeAndStrides(xShapeInfo, inMsgShapeInfo);
        bXStMsgSame = shape::haveSameShapeAndStrides(xShapeInfo, stMsgShapeInfo);
        bXInMsdxSame = shape::haveSameShapeAndStrides(xShapeInfo, inMsdxShapeInfo);
        bXStMsdxSame = shape::haveSameShapeAndStrides(xShapeInfo, stMsdxShapeInfo);
    }
    __syncthreads();

    int coords[MAX_RANK];

    for (Nd4jLong i = blockIdx.x * blockDim.x + threadIdx.x; i < xLen; i += gridDim.x * blockDim.x) {

        auto xOffset = i, zOffset = i, initMsgOffset = i, initMsdxOffset = i, stMsgOffset = i, stMsdxOffset = i;

        if (!bEWS || !bOrdering){

            shape::index2coords(i, xShapeInfo, coords);
            xOffset  = shape::getOffset(xShapeInfo, coords);
            zOffset  = bXZsame ? xOffset : shape::getOffset(zShapeInfo, coords);
            initMsgOffset = bXInMsgSame ? xOffset : shape::getOffset(inMsgShapeInfo, coords);
            stMsgOffset = bXStMsgSame ? xOffset : shape::getOffset(stMsgShapeInfo, coords);
            initMsdxOffset = bXInMsdxSame ? xOffset : shape::getOffset(inMsdxShapeInfo, coords);
            stMsdxOffset = bXStMsdxSame ? xOffset : shape::getOffset(stMsdxShapeInfo, coords);
        }

        stMsg[stMsgOffset] = rho * initMsg[initMsgOffset] + grad[xOffset] * grad[xOffset] * rhoT;

        up[zOffset] = grad[xOffset] * (sd::math::nd4j_sqrt<T, T>(initMsdx[initMsdxOffset] + epsilon) / sd::math::nd4j_sqrt<T, T>(stMsg[stMsgOffset] + epsilon));

        stMsdx[stMsdxOffset] = rho * initMsdx[initMsdxOffset] + up[zOffset] * up[zOffset] * rhoT;
    }
}

///////////////////////////////////////////////////////////////////
template<typename T>
linkage void adaDeltaUpdaterCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const hipStream_t* stream, const void* vx, const Nd4jLong* xShapeInfo,
    const void* vinMsg, const Nd4jLong* inMsgShapeInfo, const void* vinMsdx, const Nd4jLong* inMsdxShapeInfo,
    void* vz, const Nd4jLong* zShapeInfo, void* vstMsg, const Nd4jLong* stMsgShapeInfo, 
    void* vstMsdx, const Nd4jLong* stMsdxShapeInfo, const double dRho, const double dEpsilon) {

    const T rho = static_cast<T>(dRho);
    const T epsilon = static_cast<T>(dEpsilon);

    adaDeltaUpdaterCuda<T><<<blocksPerGrid, threadsPerBlock, 256, * stream>>>(vx, xShapeInfo, vinMsg, inMsgShapeInfo,
        vinMsdx, inMsdxShapeInfo, vz, zShapeInfo, vstMsg, stMsgShapeInfo, vstMsdx, stMsdxShapeInfo, rho, epsilon);
}

///////////////////////////////////////////////////////////////////
void updaterAdaDelta(sd::LaunchContext* context, const NDArray& gradient, const NDArray& initStateMsg, const NDArray& initStateMsdx, 
                    NDArray& update, NDArray& stateMsg, NDArray& stateMsdx, const double dRho, const double dEpsilon) {

    PointersManager manager(context, "adaDeltaUpdater");

    const int threadsPerBlock = MAX_NUM_THREADS / 4;
    const int blocksPerGrid = (gradient.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;

    NDArray::prepareSpecialUse({ &update, &stateMsg, &stateMsdx }, { &gradient, &initStateMsg, &initStateMsdx });
    BUILD_SINGLE_SELECTOR(gradient.dataType(), adaDeltaUpdaterCudaLauncher, (blocksPerGrid, threadsPerBlock, context->getCudaStream(), gradient.specialBuffer(), gradient.specialShapeInfo(),
        initStateMsg.specialBuffer(), initStateMsg.specialShapeInfo(), initStateMsdx.specialBuffer(), initStateMsdx.specialShapeInfo(),
        update.specialBuffer(), update.specialShapeInfo(),stateMsg.specialBuffer(), stateMsg.specialShapeInfo(),
        stateMsdx.specialBuffer(), stateMsdx.specialShapeInfo(), dRho, dEpsilon), FLOAT_TYPES);
    NDArray::registerSpecialUse({ &update, &stateMsg, &stateMsdx }, { &gradient, &initStateMsg, &initStateMsdx });

    manager.synchronize();
}

}
}
}
