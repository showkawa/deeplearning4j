#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author George A. Shulinok <sgazeos@gmail.com>
//
#include <ops/declarable/helpers/qr.h>
#include <array/NDArrayFactory.h>
#include <helpers/MmulHelper.h>

namespace sd {
namespace ops {
namespace helpers {

    template <typename T>
    static __global__ void matrixMinorKernel(T* outBuffer, Nd4jLong* outShape, T* inBuffer, Nd4jLong* inShape, Nd4jLong column, Nd4jLong rows, Nd4jLong columns) {
//        auto tid = threadIdx.x + blockDim.x * blockIdx.x;
//        auto step = blockDim.x * gridDim.x;
//        if (threadIdx.x == 0) {
//            for (auto i = tid; i < column; i += step) {
//                Nd4jLong diagPos[] = {i, i};
//                auto zIndex = shape::getOffset(outShape, diagPos);
//                outBuffer[zIndex] = T(1.f);
//            }
//        }
//        __syncthreads();

        for (auto i = blockIdx.x; i < rows; i += gridDim.x)
            for (auto j = threadIdx.x; j < columns; j += blockDim.x) {
                Nd4jLong pos[] = {i,j};
                auto zIndex = shape::getOffset(outShape, pos);
                auto xIndex = shape::getOffset(inShape, pos);
                if (i < column || j < column) {
                    outBuffer[zIndex] = i != j?T(0.f):T(1.f);
                }
                else
                    outBuffer[zIndex] = inBuffer[xIndex]; //m.t<T>(i,j) = in.t<T>(i,j);
            }


    }

    template <typename T>
    NDArray matrixMinor(LaunchContext* context, NDArray& in, Nd4jLong col) {
        NDArray m = in.ulike();
        m.setIdentity();
        m({col, m.rows(), col, m.columns()}).assign(in({col, m.rows(), col, m.columns()}));

//        auto stream = context->getCudaStream();
//        matrixMinorKernel<T><<<128, 128, 256, *stream>>>(m.dataBuffer()->specialAsT<T>(), m.special(),
//        matrixMinorKernel<T><<<128, 128, 256, *stream>>>(m.dataBuffer()->specialAsT<T>(), m.special(),
//                reinterpret_cast<T*>(in.specialBuffer()), in.special(), col, in.rows(), in.columns());
//
        m.tickWriteDevice();
        return m;
    }

/* m = I - v v^T */
    template <typename T>
    static __global__ void vmulKernel(T* resBuf, const Nd4jLong* resShape, T const* vBuff, Nd4jLong const* vShape, Nd4jLong n) {
        for (auto i = blockIdx.x; i < n; i += gridDim.x)
            for (auto j = threadIdx.x; j < n; j += blockDim.x) {
                Nd4jLong posR[] = {i, j};
                auto indexR = shape::getOffset(resShape, posR);
                auto indexX = shape::getIndexOffset(i, vShape);
                auto indexY = shape::getIndexOffset(j, vShape);

                resBuf[indexR] = T(-2.f) * vBuff[indexX] * vBuff[indexY] + (i != j?T(0.f):T(1.f));
            }
    }

    template <typename T>
    NDArray vmul(LaunchContext* context, NDArray const& v, int n)
    {
        NDArray res('c', {n,n}, v.dataType(), context); // x = matrix_new(n, n);

        auto stream = context->getCudaStream();
        vmulKernel<T><<<128, 128, 128, *stream>>>(res.dataBuffer()->specialAsT<T>(), res.specialShapeInfo(),
                reinterpret_cast<T const*>(v.specialBuffer()), v.specialShapeInfo(), n);
        return res;
    }

    template <typename T>
    static bool diagonalIsPositive(NDArray* matrix, Nd4jLong k) {
        T hVal;
        Nd4jLong pos[] = {k, k};
        auto shift = shape::getOffset(matrix->shapeInfo(), pos);
        hipMemcpy(&hVal, matrix->specialBuffer(), sizeof(T), hipMemcpyDeviceToHost);
        return hVal > T(0.f);
    }

    template <typename T>
    void qrSingle(LaunchContext* context, NDArray* matrix, NDArray* Q, NDArray* R, bool const fullMatricies) {
        Nd4jLong M = matrix->sizeAt(0);
        Nd4jLong N = matrix->sizeAt(1);
        auto resQ = fullMatricies?Q->ulike():NDArrayFactory::create<T>(matrix->ordering(), {M,M}, Q->getContext());
        auto resR = fullMatricies?R->ulike():matrix->ulike();
        std::vector<NDArray> q(M);
        NDArray z = *matrix;
        NDArray e('c', {M}, DataTypeUtils::fromT<T>(), context); // two internal buffers and scalar for squared norm
        for (auto k = 0; k < N && k < M - 1; k++) { // loop for columns, but not further then row number
            e.nullify();
            z = matrixMinor<T>(context, z, k); // minor computing for current column with given matrix z (initally is a input matrix)

            auto currentColumn = z({0, 0, k, k + 1}); // retrieve k column from z to x buffer
            auto norm = currentColumn.reduceAlongDimension(reduce::Norm2, {0});
            if (diagonalIsPositive<T>(matrix, k)) //matrix->t<T>(k,k) > T(0.f)) // negate on positive matrix diagonal element
                norm.applyTransform(transform::Neg, norm); // *= -1.f;//-norm.t<T>(0);

            e.p(k, norm); // e - is filled by 0 vector except diagonal element (filled by 1)
            e += currentColumn; // e[i] = x[i] + a * e[i] for each i from 0 to n - 1
            auto normE = e.reduceAlongDimension(reduce::Norm2, {0});
            e /= normE;
            q[k] = vmul<T>(context, e, M);
            auto qQ = z.ulike();
            MmulHelper::matmul(&q[k], &z, &qQ, false, false);
            z = std::move(qQ);
        }
        resQ.assign(q[0]); //
//        MmulHelper::matmul(&q[0], matrix, &resR, false, false);
        for (int i = 1; i < N && i < M - 1; i++) {
            auto tempResQ = resQ;
            MmulHelper::matmul(&q[i], &resQ, &tempResQ, false, false);
            resQ = std::move(tempResQ);
        }
        MmulHelper::matmul(&resQ, matrix, &resR, false, false);
        // resR *= -1.f;
        resQ.transposei();

        if (fullMatricies) {
            Q->assign(resQ);
            R->assign(resR);
        }
        else {
            Q->assign(resQ({0, 0, 0, N}));
            R->assign(resR({0, N, 0, 0}));
        }
    }

    template <typename T>
    void qr_(LaunchContext* context, NDArray const* input, NDArray* outputQ, NDArray* outputR, bool const fullMatricies) {
        Nd4jLong lastDim = input->rankOf() - 1;
        Nd4jLong preLastDim = input->rankOf() - 2;

        NDArray::prepareSpecialUse({outputQ, outputR}, {input});
        ResultSet listOutQ(outputQ->allTensorsAlongDimension({(int)preLastDim, (int)lastDim}));
        ResultSet listOutR(outputR->allTensorsAlongDimension({(int)preLastDim, (int)lastDim}));
        ResultSet listInput(input->allTensorsAlongDimension({(int)preLastDim, (int)lastDim}));
        auto start = 0;
        auto stop = listInput.size();
        auto increment = 1;

        for (auto batch = start; batch < stop; batch += increment) {
            //qr here
            qrSingle<T>(context, listInput.at(batch), listOutQ.at(batch), listOutR.at(batch), fullMatricies);
        }
        NDArray::registerSpecialUse({outputQ, outputR}, {input});
    }

    void qr(sd::LaunchContext* context, NDArray const* input, NDArray* outputQ, NDArray* outputR, bool const fullMatricies) {
        BUILD_SINGLE_SELECTOR(input->dataType(), qr_, (context, input, outputQ, outputR, fullMatricies), FLOAT_TYPES);
    }

}
}
}
