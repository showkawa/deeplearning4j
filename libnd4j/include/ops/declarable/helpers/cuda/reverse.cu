#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma, created on 16.04.2018
//

#include <ops/declarable/helpers/reverse.h>
#include <helpers/ShapeUtils.h>
#include <array/ResultSet.h>
#include <helpers/TAD.h>
#include <helpers/PointersManager.h>
#include <helpers/ConstantTadHelper.h>


namespace sd    {
namespace ops     {
namespace helpers {

    template <typename T>
    static __global__ void reverseTadKernel(const void* vinput, const Nd4jLong *inputShape, void* voutput, const Nd4jLong *outputShape, const Nd4jLong *inputTadShape, const Nd4jLong *inputTadOffsets, const Nd4jLong *outputTadShape, const Nd4jLong *outputTadOffsets, uint64_t limit, uint64_t numOfElemsToReverse, uint64_t numTads) {
        auto input = reinterpret_cast<const T*>(vinput);
        auto output = reinterpret_cast<T*>(voutput);
        const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
        const auto step = gridDim.x * blockDim.x;

        // this means that we'll have additional cycle, to move middle element
        auto div = numOfElemsToReverse / 2;
        auto odd = numOfElemsToReverse % 2 != 0;
        auto rlimit = odd ? limit / 2 + 1 : limit / 2;

        // all threads operate in the same input/output space
        for (uint64_t e = tid; e < rlimit; e += step) {
            // finding out the TAD we're going to process
            auto tadId = e / div;

            if (tadId >= numTads)
                continue;

            // now finding out element within tad
            auto idx = e % div;

            //printf("TID: %i; numTads: %lld; tadLength: %lld; tadId: %i, idx: %lld\n", tid, numTads, numOfElemsToReverse, tadId, idx);

            auto tadInput = input + inputTadOffsets[tadId];
            auto tadOutput = output + outputTadOffsets[tadId];

            // we're calculating offsets within input TAD
            auto fOffset = shape::getIndexOffset(idx, inputTadShape);
            auto lOffset = shape::getIndexOffset(numOfElemsToReverse - idx - 1, inputTadShape);

            // now we're storing input values
            auto v1 = tadInput[fOffset];
            auto v2 = tadInput[lOffset];

            // now we're calculating offsets within output TAD
            auto zfOffset = shape::getIndexOffset(idx, outputTadShape);
            auto zlOffset = shape::getIndexOffset(numOfElemsToReverse - idx - 1, outputTadShape);

            // and saving values to output arrays
            tadOutput[zfOffset] = v2;
            tadOutput[zlOffset] = v1;
        }

        // moving odd element in blocks
        if (odd && threadIdx.x == 0) {
            for (uint64_t e = blockIdx.x; e < numTads; e += gridDim.x) {
                auto tadInput = input + inputTadOffsets[e];
                auto tadOutput = output + outputTadOffsets[e];

                auto xOffset = shape::getIndexOffset(numOfElemsToReverse / 2, inputTadShape);
                auto zOffset = shape::getIndexOffset(numOfElemsToReverse / 2, outputTadShape);

                tadOutput[zOffset] = tadInput[xOffset];
            }
        }

    }


    template <typename T>
    static __global__ void reverseArrayKernel(const void* input, const Nd4jLong *inputShape, void* output, const Nd4jLong *outputShape, Nd4jLong numOfElemsToReverse) {
        const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
        const auto step = gridDim.x * blockDim.x;
        __shared__ int linearStatus;
        __shared__ const T* inputArr;
        __shared__ T* outputArr;
        __shared__ char inputOrder, outputOrder;

        if (threadIdx.x == 0) {
            linearStatus = (shape::elementWiseStride(inputShape) == shape::elementWiseStride(outputShape)) && (inputOrder == outputOrder)? shape::elementWiseStride(inputShape):0;

            char inputOrder = shape::order(inputShape);
            char outputOrder = shape::order(outputShape);
            inputArr = reinterpret_cast<const T*>(input);
            outputArr = reinterpret_cast<T*>(output);
        }
        __syncthreads();

        auto odd = numOfElemsToReverse % 2 != 0;
        auto limit = numOfElemsToReverse / 2;

        for (uint64_t e = tid; e < limit; e += step) {
            // we're calculating offsets within input array
            auto fOffset = shape::getIndexOffset(e, inputShape);
            auto lOffset = shape::getIndexOffset(numOfElemsToReverse - e - 1, inputShape);

            // now we're storing input values
            auto v1 = inputArr[fOffset];
            auto v2 = inputArr[lOffset];

            // now we're calculating offsets within output array
            auto zfOffset = shape::getIndexOffset(e, outputShape);
            auto zlOffset = shape::getIndexOffset(numOfElemsToReverse - e - 1, outputShape);

            // and saving values to output arrays
            outputArr[zfOffset] = v2;
            outputArr[zlOffset] = v1;
        }

        // in case of odd array we'll have to move middle value
        if (odd && tid == 0) {
            auto xOffset = shape::getIndexOffset(limit, inputShape);
            auto zOffset = shape::getIndexOffset(limit, outputShape);

            outputArr[zOffset] = inputArr[xOffset];
        }
    }

    template<typename T>
    static void reverseTad(sd::LaunchContext * context, const NDArray* input, NDArray* output, const Nd4jLong *inputTadShape, const Nd4jLong *inputTadOffsets, const Nd4jLong *outputTadShape, const Nd4jLong *outputTadOffsets, uint64_t tadLength) {
        auto stream = context->getCudaStream();
        reverseTadKernel<T><<<256, 512, 8192, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo(), inputTadShape, inputTadOffsets, outputTadShape, outputTadOffsets, input->lengthOf(), tadLength, input->lengthOf() / tadLength);
    }

    template<typename T>
    static void reverseArray(sd::LaunchContext * context, const NDArray* input, NDArray* output, Nd4jLong numOfElemsToReverse) {
        auto stream = context->getCudaStream();
        Nd4jLong numOfReverse = numOfElemsToReverse;
        if (numOfElemsToReverse == 0)
            numOfReverse = input->lengthOf();

        reverseArrayKernel<T><<<256, 512, 8192, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo(), numOfReverse);
    }


    ///////////////////////////////////////////////////////////////////
    template <typename T>
    static void reverseSequence_(sd::LaunchContext * context, const NDArray* input, const NDArray* seqLengths, NDArray* output, int seqDim, const int batchDim){
        int posOfNonUnityDim = -1;
        seqLengths->syncToHost();
        auto stream = context->getCudaStream();

        if(input->isVector() || shape::isLikeVector(input->shapeInfo(), posOfNonUnityDim) || seqLengths->lengthOf() == 1) {
            int numOfElemsToReverse = seqLengths->e<int>(0);
            if((seqDim == 0 && input->sizeAt(0) == 1) || (batchDim == posOfNonUnityDim))
                output->assign(input);
            else
                reverseArrayKernel<T><<<256, 512, 8192, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo(), numOfElemsToReverse);//helpers::reverseArray<T>(context, const_cast<NDArray*>(input), output, numOfElemsToReverse);
        }
        else {

            if(seqDim > batchDim)
                --seqDim;

            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {batchDim});

            auto inSubArrsSet  = input->allTensorsAlongDimension(dimensions);
            auto outSubArrsSet = output->allTensorsAlongDimension(dimensions);

            for(int i = 0; i < inSubArrsSet.size(); ++i) {

                int numOfElemsToReverse = seqLengths->e<int>(i);

                if(numOfElemsToReverse == 0 || numOfElemsToReverse == 1) {
                    outSubArrsSet.at(i)->assign(inSubArrsSet.at(i));
                }
                else {
                    auto inInnerSet  = inSubArrsSet.at(i)->allTensorsAlongDimension({seqDim});
                    auto outInnerSet = outSubArrsSet.at(i)->allTensorsAlongDimension({seqDim});
                    for(int j = 0; j < inInnerSet.size(); ++j)
                        reverseArray<T>(context, inInnerSet.at(j), outInnerSet.at(j), numOfElemsToReverse);
                }
            }
        }
    }

    void reverseSequence(sd::LaunchContext * context, const NDArray* input, const NDArray* seqLengths, NDArray* output, int seqDim, const int batchDim) {
        NDArray::prepareSpecialUse({output}, {input, seqLengths});

        // if op isn't inplace - copy original data into output array
        if (output->specialBuffer() != input->specialBuffer())
            output->assign(input);

        BUILD_SINGLE_SELECTOR(input->dataType(), reverseSequence_, (context, input, seqLengths, output, seqDim, batchDim), LIBND4J_TYPES);
        NDArray::registerSpecialUse({output}, {input, seqLengths});
    }

    //////////////////////////////////////////////////////////////////////////
    void reverse(sd::LaunchContext * context, const NDArray* input, NDArray* output, const std::vector<int>* intArgs) {

        auto packX = sd::ConstantTadHelper::getInstance().tadForDimensions(input->shapeInfo(), *intArgs);
        auto packZ = sd::ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(), *intArgs);

        NDArray::prepareSpecialUse({output}, {input});

        if (packX.numberOfTads() == 1) {
            BUILD_SINGLE_SELECTOR(input->dataType(), reverseArray, (context, input, output, 0),  LIBND4J_TYPES);
        } else {
            BUILD_SINGLE_SELECTOR(input->dataType(), reverseTad, (context, input, output, packX.platformShapeInfo(), packX.platformOffsets(), packZ.platformShapeInfo(), packZ.platformOffsets(), (uint64_t) (input->lengthOf() / packX.numberOfTads())),  LIBND4J_TYPES);
        }

        NDArray::registerSpecialUse({output}, {input});
    }
}
}
}

