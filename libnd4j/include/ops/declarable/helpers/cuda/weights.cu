#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author sgazeos@gmail.com
//

#include <ops/declarable/helpers/weights.h>

namespace sd {
namespace ops {
namespace helpers {



    template <typename T>
    static __device__ void adjustWeightsKernelD(void* inputBuffer,   Nd4jLong const*  inputShape,
                                               void* weightsBuffer, Nd4jLong const*  weightsShape,
                                               void* outputBuffer,  Nd4jLong inputLength,
                                               Nd4jLong outputLength, int val) {
    //    typedef Nd4jLong T;
        auto tid = threadIdx.x;
        //int threadCount = gridDim.x * blockDim.x;
        __shared__ T* outputPart;
        __shared__ Nd4jLong offset;
        //for (int e = 0; e < inputLength; e++) {
        for (Nd4jLong e = tid; e < inputLength; e += blockDim.x) {

            Nd4jLong xOffset = shape::getIndexOffset(e, inputShape);
            int current = *(reinterpret_cast<int*>(inputBuffer) + xOffset);
            if (current == val) {
                //printf("%lld\n", xOffset);
                //Nd4jLong zOffset = shape::getIndexOffset(val, outputShape);
                if (weightsBuffer != nullptr) {
                    Nd4jLong yOffset = shape::getIndexOffset(e, weightsShape);
                    //atomicAdd();
                    //*reinterpret_cast<int *>(outputBuffer) +=  reinterpret_cast<int *>(weightsBuffer)[yOffset];
                    sd::math::atomics::nd4j_atomicAdd(reinterpret_cast<T *>(outputBuffer), reinterpret_cast<T *>(weightsBuffer)[yOffset]); //output->p(val, output->e<T>(val) + 1);
//                    atomicAdd(reinterpret_cast<int *>(outputBuffer), reinterpret_cast<int *>(weightsBuffer)[yOffset]); //output->p(val, output->e<T>(val) + 1);
                }
                else {
                    //*reinterpret_cast<int *>(outputBuffer) += int(1);
                    //printf("outputBuffer[0] = %d\n", static_cast<int>(*(reinterpret_cast<T *>(outputBuffer))));
                    sd::math::atomics::nd4j_atomicAdd(reinterpret_cast<T *>(outputBuffer), T(1)); //output->p(val, output->e<T>(val) + 1);
//                    atomicAdd(reinterpret_cast<int *>(outputBuffer), int(1)); //output->p(val, output->e<T>(val) + 1);
                    //            printf("outputBuffer[%ld] = %d\n", zOffset, static_cast<int>(*(reinterpret_cast<T *>(outputBuffer) + zOffset)));
                }
                //printf("xOffset is %ld, zOffset is %ld\n", xOffset, zOffset);
            }
        }
//        if (threadIdx.x + offset < outputLength)
//            reinterpret_cast<T *>(outputBuffer)[threadIdx.x + offset] = outputPart[threadIdx.x];
    }

        template <typename T>
    static __global__ void adjustWeightsKernel(void* inputBuffer,   Nd4jLong const*  inputShape,
                                               void* weightsBuffer, Nd4jLong const*  weightsShape,
                                               void* outputBuffer,  Nd4jLong const*  outputShape,
                                               int minLength, int maxLength) {

        //auto tid = blockIdx.x * blockDim.x + threadIdx.x; // * blockDim.x; // + threadIdx.x;
        int threadCount = gridDim.x * blockDim.x;
        Nd4jLong inputLength = shape::length(inputShape);

        Nd4jLong outputLength = shape::length(outputShape);
        Nd4jLong borderLen = 1;

        for (Nd4jLong e = blockIdx.x; e < outputLength; e += threadCount) {
        //if (blockIdx.x < outputLength) {
            //if (e + threadCount < outputLength) {
            Nd4jLong zOffset = shape::getIndexOffset(e, outputShape);
            //printf("%d %d %d\n", blockIdx.x, blockDim.x, threadIdx.x);
            //Nd4jLong borderLen = 1;
            T* outputBufferZ = reinterpret_cast<T*>(outputBuffer) + zOffset;
            adjustWeightsKernelD<T>(inputBuffer, inputShape, weightsBuffer, weightsShape, (void*)outputBufferZ,
                                 inputLength, outputLength, (int)zOffset);

        }
    }

    template <typename T>
    static void adjustWeights_(sd::LaunchContext * context, NDArray* input, NDArray* weights, NDArray* output, int minLength, int maxLength) {
//        for (int e = 0; e < input->lengthOf(); e++) {
//            int val = input->e<int>(e);
//            if (val < maxLength) {
//                if (weights != nullptr)
//                    output->p(val, output->e<T>(val) + weights->e<T>(e));
//                else
//                    output->p(val, output->e<T>(val) + 1);
//            }
//        }
        dim3 launchDims(256, 512, 8192);
        auto stream = context->getCudaStream();
        adjustWeightsKernel<T><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(input->specialBuffer(),
                input->specialShapeInfo(), weights?weights->specialBuffer():nullptr, weights?weights->specialShapeInfo():nullptr,
                output->specialBuffer(), output->specialShapeInfo(), minLength, maxLength);
    }

    void adjustWeights(sd::LaunchContext * context, NDArray* input, NDArray* weights, NDArray* output, int minLength, int maxLength) {
        BUILD_SINGLE_SELECTOR(output->dataType(), adjustWeights_, (context, input, weights, output, minLength, maxLength), GENERIC_NUMERIC_TYPES);
    }

    BUILD_SINGLE_TEMPLATE(template void adjustWeights_, (sd::LaunchContext * context, NDArray* input, NDArray* weights, NDArray* output, int minLength, int maxLength), GENERIC_NUMERIC_TYPES);
}
}
}