#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author GS <sgazeos@gmail.com>
//

#include <ops/declarable/helpers/segment.h>
#include <ops/declarable/helpers/segment_common.h>
#include <array/NDArrayFactory.h>
#include <helpers/ShapeUtils.h>
#include <helpers/TAD.h>
#include <exceptions/cuda_exception.h>
#include <helpers/PointersManager.h>
#include <helpers/ConstantTadHelper.h>

namespace sd {
namespace ops {
namespace helpers {
    // -------------------------------------------------------------------------------------------------------------- //
    // Segment ops linear kernels
    // -------------------------------------------------------------------------------------------------------------- //
    template <typename T, typename I>
    static __global__ void segmentMeanLinearKernel(void* input, Nd4jLong const*  inputShape, int* starts, int* lengths, Nd4jLong numOfClasses, void* output, Nd4jLong const*  outputShape) {
        __shared__ T* val;
        __shared__ Nd4jLong xLen, zLen, segment, zIndex;
        __shared__ T* x;
        __shared__ T* z;
        __shared__ int threadsPerSegment, start, finish;

        if (threadIdx.x == 0) {
            threadsPerSegment = (gridDim.x + numOfClasses - 1) / numOfClasses;
            segment = blockIdx.x / threadsPerSegment;
            x = reinterpret_cast<T*>(input);
            z = reinterpret_cast<T*>(output);
//            extern __shared__ unsigned char shmem[];
//            val = reinterpret_cast<T*>(shmem);
            xLen = shape::length(inputShape);
            zLen = shape::length(outputShape);

            //[zIndex] =
            if (segment < numOfClasses) {
                zIndex = shape::getIndexOffset(segment, outputShape);
                start = starts[segment];
                finish = start + lengths[segment];
                //val[segment] = ;
                z[zIndex] = T(x[shape::getIndexOffset(start, inputShape)] / lengths[segment]);
//                val[segment] = z[zIndex];
            }

        }
        __syncthreads();

        for (auto e = start + threadIdx.x + 1; e < finish; e += blockDim.x) {
            auto xIndex = shape::getIndexOffset(e, inputShape);
            if (lengths[segment])
                sd::math::atomics::nd4j_atomicAdd(&z[zIndex], T(x[xIndex] / lengths[segment]));
        }
    }
    // -------------------------------------------------------------------------------------------------------------- //
    template <typename T, typename I>
    static __global__ void unsortedSegmentMeanLinearKernel(void* input, Nd4jLong const*  inputShape, void* indices, Nd4jLong const*  indicesShape, int* starts, int* lengths, Nd4jLong numOfClasses, void* output, Nd4jLong const*  outputShape) {
        __shared__ T* val;
        __shared__ Nd4jLong xLen, zLen, zIndex;
        __shared__ T* x;
        __shared__ T* z;
        __shared__ I* y; //int threadsPerSegment, start, finish;
        auto segment = blockIdx.x;// /
        if (threadIdx.x == 0) {
//            threadsPerSegment = (gridDim.x + numOfClasses - 1) / numOfClasses;
//            threadsPerSegment;
            x = reinterpret_cast<T*>(input);
            z = reinterpret_cast<T*>(output);
            y = reinterpret_cast<I*>(indices);
//            extern __shared__ unsigned char shmem[];
//            val = reinterpret_cast<T*>(shmem);
            xLen = shape::length(inputShape);
            zLen = shape::length(outputShape);

//            if (segment < numOfClasses) {
            zIndex = shape::getIndexOffset(segment, outputShape);
            //start = starts[segment];
            //finish = start + lengths[segment];
            if (lengths[segment] > 0)
                z[zIndex] = T(x[shape::getIndexOffset(starts[segment], inputShape)] / T(lengths[segment]));
            else
                z[zIndex] = 0; //DataTypeUtils::max<T>();
//                val[segment] = z[zIndex];
//            }

        }
        __syncthreads();
        if (lengths[segment] > 0)
            for (auto e = threadIdx.x; e < xLen; e += blockDim.x) {
                auto xIndex = shape::getIndexOffset(e, inputShape);
                auto yIndex = shape::getIndexOffset(e, indicesShape);
                if (y[yIndex] == segment && e != starts[segment]) {
                    sd::math::atomics::nd4j_atomicAdd(&z[zIndex], T(x[xIndex]/T(lengths[segment])));
                }
            }
    }
    // -------------------------------------------------------------------------------------------------------------- //
    // SegmentMean kernel
    template <typename T, typename I>
    static __global__ void segmentMeanTadKernel(void* inputBuf, Nd4jLong const*  inputShape, Nd4jLong const*  inputTads, Nd4jLong const*  inputTadOffsets, I* indices, int* starts, int* lengths, Nd4jLong numOfClasses, void* outputBuf, Nd4jLong const*  outputShape, Nd4jLong const*  outputTads, Nd4jLong const*  outputTadOffsets) {
        __shared__ T* val;
        __shared__ Nd4jLong len, zIndex, total;
        __shared__ T* z;
        __shared__ int threadsPerSegment, start, finish;
        auto segment = indices[blockIdx.x]; // / threadsPerSegment;

        if (threadIdx.x == 0) {
            z = reinterpret_cast<T*>(outputBuf) + outputTadOffsets[segment];
            len = shape::length(inputTads);
            start = starts[segment];
            finish = start + lengths[segment];
            total = shape::sizeAt(inputShape, 0);

        }
        __syncthreads();

        auto idx = blockIdx.x;
        if (blockIdx.x <= total) {
            auto x = reinterpret_cast<T *>(inputBuf) + inputTadOffsets[idx];
            if (blockIdx.x == start) {
                for (auto e = threadIdx.x; e < len; e += blockDim.x) {
                    auto xIndex = shape::getIndexOffset(e, inputTads);
                    auto zIndex = shape::getIndexOffset(e, outputTads);
                    sd::math::atomics::nd4j_atomicAdd(&z[zIndex], T(x[xIndex]/lengths[segment]));
                }
            }
            else {
                for (auto e = threadIdx.x; e < len; e += blockDim.x) {
                    auto xIndex = shape::getIndexOffset(e, inputTads);
                    auto zIndex = shape::getIndexOffset(e, outputTads);
                    if (lengths[segment])
                        sd::math::atomics::nd4j_atomicAdd(&z[zIndex], T(x[xIndex]/lengths[segment]));
                }
            }
        }
    }
    // -------------------------------------------------------------------------------------------------------------- //
    // segmen mean
    template <typename T, typename I>
    static void segmentMeanFunctor_(LaunchContext* context, NDArray* input, NDArray* indices, NDArray* output) {
        auto stream = context->getCudaStream();
        Nd4jLong numClasses = indices->e<Nd4jLong>(indices->lengthOf() - 1) + 1;
        NDArray classesRangesLens = NDArrayFactory::create<int>('c', {numClasses}, context);
        NDArray classesRangesBegs = NDArrayFactory::create<int>('c', {numClasses}, context);

        classesRangesBegs.assign(indices->lengthOf());
        classesRangesLens.assign(0);
        NDArray::prepareSpecialUse({output}, {input, indices});
        dim3 dims(numClasses, indices->lengthOf(), numClasses * 32 + 32);
        int* begins = reinterpret_cast<int*>(classesRangesBegs.specialBuffer());
        int* lengths = reinterpret_cast<int*>(classesRangesLens.specialBuffer());
        fillUpSegments(indices, numClasses, classesRangesBegs, classesRangesLens);

        if (input->isVector()) {
            segmentMeanLinearKernel<T,I><<<numClasses, input->lengthOf(), numClasses * 32 + 32, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), begins, lengths, numClasses, output->specialBuffer(), output->specialShapeInfo());
        }
        else {
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = sd::ConstantTadHelper::getInstance().tadForDimensions(input->shapeInfo(), dimensions);
            auto packZ = sd::ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(), dimensions);
            auto inputTads = packX.specialShapeInfo();
            auto inputTadOffsets = packX.specialOffsets();
            auto outputTads = packZ.specialShapeInfo();
            auto outputTadOffsets = packZ.specialOffsets();
            segmentMeanTadKernel<T,I><<<input->sizeAt(0), 512, 2048, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), inputTads, inputTadOffsets, reinterpret_cast<I*>(indices->specialBuffer()), begins, lengths, numClasses, output->specialBuffer(), output->specialShapeInfo(), outputTads, outputTadOffsets);
        }
        NDArray::registerSpecialUse({output}, {input, indices});

    }
    // -------------------------------------------------------------------------------------------------------------- //
    void segmentMeanFunctor(sd::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* output) {
        NDArray::prepareSpecialUse({output}, {input, indices});
        BUILD_DOUBLE_SELECTOR(output->dataType(), indices->dataType(), segmentMeanFunctor_, (context, input, indices, output), NUMERIC_TYPES, INDEXING_TYPES);
        NDArray::registerSpecialUse({output}, {input, indices});
    }

    // -------------------------------------------------------------------------------------------------------------- //
    template <typename T, typename I>
    static void unsortedSegmentMeanFunctor_(sd::LaunchContext* context, NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {
        auto stream = context->getCudaStream();
//        NDArray classes = NDArrayFactory::create<int>('c', {numOfClasses, 2});

        NDArray classesRangesBegs = NDArrayFactory::create<int>('c', {numOfClasses}, context);
        NDArray classesRangesLens = NDArrayFactory::create<int>('c', {numOfClasses}, context);
//        NDArray row = NDArrayFactory::create<int>('c', {1, 2}, {(int)indices->lengthOf(), (int)0});
//        classes.applyTrueBroadcast(sd::BroadcastOpsTuple::Assign(), &row, &classes);
        classesRangesBegs.assign(indices->lengthOf());
        classesRangesLens.assign(0);
        dim3 dims(numOfClasses, indices->lengthOf(), numOfClasses * 32 + 32);
//        int* classesBuf = reinterpret_cast<int*>(classes.specialBuffer());
        fillUpSegments(indices, numOfClasses, classesRangesBegs, classesRangesLens);
        int* begins = reinterpret_cast<int*>(classesRangesBegs.specialBuffer());
        int* lengths = reinterpret_cast<int*>(classesRangesLens.specialBuffer());

        if (input->isVector()) {
            unsortedSegmentMeanLinearKernel<T,I><<<dims.x, dims.y, dims.z, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), indices->specialBuffer(), indices->specialShapeInfo(), begins, lengths, numOfClasses, output->specialBuffer(), output->specialShapeInfo());
        }
        else {
            output->assign(0);
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = sd::ConstantTadHelper::getInstance().tadForDimensions(input->shapeInfo(), dimensions);
            auto packZ = sd::ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(), dimensions);
            Nd4jLong const*  inputTads = packX.specialShapeInfo();
            Nd4jLong const*  inputTadOffsets = packX.specialOffsets();
            Nd4jLong const*  outputTads = packZ.specialShapeInfo();
            Nd4jLong const*  outputTadOffsets = packZ.specialOffsets();
            dims.x = input->sizeAt(0);
            segmentMeanTadKernel<T,I><<<dims.x, dims.y, dims.z, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), inputTads, inputTadOffsets, reinterpret_cast<I*>(indices->specialBuffer()), begins, lengths, numOfClasses, output->specialBuffer(), output->specialShapeInfo(), outputTads, outputTadOffsets);
        }

    }
    // -------------------------------------------------------------------------------------------------------------- //
    void unsortedSegmentMeanFunctor(sd::LaunchContext* context , NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {
        NDArray::prepareSpecialUse({output}, {input, indices});
        BUILD_DOUBLE_SELECTOR(input->dataType(), indices->dataType(), unsortedSegmentMeanFunctor_, (context, input, indices, numOfClasses, output),
                              NUMERIC_TYPES, INDEXING_TYPES);
        NDArray::registerSpecialUse({output}, {input, indices});
    }

    // -------------------------------------------------------------------------------------------------------------- //
    template <typename T, typename I>
    static __global__ void segmentMeanBPLinearKernel(void* inputBuf, Nd4jLong const*  inputShape, void* eps, Nd4jLong const*  epsShape, void* indicesBuf, Nd4jLong const*  indicesShape,
                                                     int* lengths, void* outputBuf, Nd4jLong const*  outputShape) {
        __shared__ T* x;
        __shared__ T* gradIn;
        __shared__ T* gradOut;
        __shared__ I* y;
        __shared__ T* z;
        __shared__ Nd4jLong xLen, gradLen;

        if (threadIdx.x == 0) {
            xLen = shape::length(inputShape);
            x = reinterpret_cast<T*>(inputBuf);
            y = reinterpret_cast<I*>(indicesBuf);
            z = reinterpret_cast<T*>(outputBuf);
            gradOut = reinterpret_cast<T*>(eps);
            gradLen = shape::length(epsShape);
        }
        __syncthreads();

        auto start = blockIdx.x * blockDim.x + threadIdx.x;
        auto step = gridDim.x * blockDim.x;

        for (auto e = start; e < xLen; e += step) {

            auto zOffset = shape::getIndexOffset(e, outputShape);
            auto xOffset = shape::getIndexOffset(e, inputShape);
            auto yOffset = shape::getIndexOffset(e, indicesShape);
            auto classIndex = y[yOffset];
            auto gradOffsetO = shape::getIndexOffset(classIndex, epsShape);

            z[zOffset] = T(gradOut[gradOffsetO] / float(lengths[classIndex]));
        }
    }
    // -------------------------------------------------------------------------------------------------------------- //
    template <typename T, typename I>
    static __global__ void segmentMeanBPTadKernel(void* inputBuf, Nd4jLong const*  inputShape, void* eps, Nd4jLong const*  epsShape,
                                                  void* indicesBuf, Nd4jLong const*  indicesShape, int* lengths, void* outputBuf, Nd4jLong const*  outputShape,Nd4jLong const*  inputTad,
                                                  Nd4jLong const*  inputOffsets, Nd4jLong const*  gradOutTad, Nd4jLong const*  gradOutOffsets, Nd4jLong const*  outTad, Nd4jLong const*  outOffsets) {
        __shared__ T* x;
        __shared__ T* gradOut;
        __shared__ I* y;
        __shared__ T* z;
        __shared__ Nd4jLong xLen, yLen, gradLen, currentLen;

        if (threadIdx.x == 0) {
            xLen = shape::length(inputShape);
            x = reinterpret_cast<T*>(inputBuf);
            y = reinterpret_cast<I*>(indicesBuf);
            z = reinterpret_cast<T*>(outputBuf);
            yLen = shape::length(indicesShape);
            gradOut = reinterpret_cast<T*>(eps);
            gradLen = shape::length(epsShape);
            currentLen = shape::length(outTad);
        }
        __syncthreads();

        for (auto i = blockIdx.x; i < yLen; i += gridDim.x) {
//            auto yIndex = shape::getIndexOffset(i, indicesShape);
            auto segment = y[i]; //yIndex];
            T* currentOut = z + outOffsets[i];
            T* outGrad = gradOut + gradOutOffsets[segment];

            for (auto e = threadIdx.x; e < currentLen; e += blockDim.x) {
                auto zIndex = shape::getIndexOffset(e, outTad);
                auto gradIndex = shape::getIndexOffset(e, gradOutTad);
                if (lengths[segment] > 0)
                    currentOut[zIndex] = T(outGrad[gradIndex] / float(lengths[segment]));
            }
        }
    }
    // -------------------------------------------------------------------------------------------------------------- //
    // backrop for mean
    template <typename T, typename I>
    int segmentMeanFunctorBP_(sd::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output) {
        auto stream = context->getCudaStream();
        NDArray::prepareSpecialUse({output}, {input, indices, gradOut});
        auto numClasses = indices->e<int>(indices->lengthOf() - 1) + 1;
        NDArray classesRangesLens = NDArrayFactory::create<int>('c', {numClasses}, context);
        NDArray classesRangesBegs = NDArrayFactory::create<int>('c', {numClasses}, context);

        classesRangesBegs.assign(indices->lengthOf());
        classesRangesLens.assign(0);
        dim3 dims(numClasses, indices->lengthOf(), numClasses * 32 + 32);
        fillUpSegments(indices, numClasses, classesRangesBegs, classesRangesLens);
        int* begins = reinterpret_cast<int*>(classesRangesBegs.specialBuffer());
        int* lengths = reinterpret_cast<int*>(classesRangesLens.specialBuffer());

        if (input->isVector()) {
            Nd4jLong loop_size = input->lengthOf();
            auto numOfClasses = gradOut->lengthOf(); //indices->e<Nd4jLong>(loop_size - 1);
            segmentMeanBPLinearKernel<T,I><<<gradOut->lengthOf(), input->lengthOf(), 256, *stream>>>(input->specialBuffer(),
                    input->specialShapeInfo(), gradOut->specialBuffer(), gradOut->specialShapeInfo(),
                    indices->specialBuffer(), indices->specialShapeInfo(), lengths, output->specialBuffer(), output->specialShapeInfo());
        }
        else {
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = sd::ConstantTadHelper::getInstance().tadForDimensions(input->shapeInfo(), dimensions);
            auto packZ = sd::ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(), dimensions);
//            auto packGradIn = sd::ConstantTadHelper::getInstance().tadForDimensions(tempRes.shapeInfo(), dimensions);
            auto packGradOut = sd::ConstantTadHelper::getInstance().tadForDimensions(gradOut->shapeInfo(), dimensions);
            Nd4jLong const*  inputTads = packX.specialShapeInfo();
            Nd4jLong const*  inputTadOffsets = packX.specialOffsets();
            Nd4jLong const*  outputTads = packZ.specialShapeInfo();
            Nd4jLong const*  outputTadOffsets = packZ.specialOffsets();
            Nd4jLong const*  gradOutTads = packGradOut.specialShapeInfo();
            Nd4jLong const*  gradOutTadOffsets = packGradOut.specialOffsets();

            segmentMeanBPTadKernel<T,I><<<indices->lengthOf(), input->lengthOf(), 256, *stream>>>(input->specialBuffer(), input->specialShapeInfo(),
                    gradOut->specialBuffer(), gradOut->specialShapeInfo(), indices->specialBuffer(), indices->specialShapeInfo(), lengths,
                    output->specialBuffer(), output->specialShapeInfo(), inputTads, inputTadOffsets, gradOutTads, gradOutTadOffsets,
                    outputTads, outputTadOffsets);
        }
        NDArray::registerSpecialUse({output}, {input, indices, gradOut});
        return Status::OK();
    }
    // -------------------------------------------------------------------------------------------------------------- //
    // segmen mean bp main
    int segmentMeanFunctorBP(sd::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output) {
        NDArray::prepareSpecialUse({output}, {input, indices, gradOut});
        BUILD_DOUBLE_SELECTOR(output->dataType(), indices->dataType(), return segmentMeanFunctorBP_, (context, input,
                indices, gradOut, output), FLOAT_TYPES, INDEXING_TYPES);
        NDArray::registerSpecialUse({output}, {input, indices, gradOut});
    }
    // -------------------------------------------------------------------------------------------------------------- //

    template <typename T, typename I>
    static int unsortedSegmentMeanFunctorBP_(sd::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        auto stream = context->getCudaStream();
        NDArray::prepareSpecialUse({output}, {input, indices, gradOut});
        auto numClasses = indices->e<int>(indices->lengthOf() - 1) + 1;
        NDArray classesRangesLens = NDArrayFactory::create<int>('c', {numClasses}, context);
        NDArray classesRangesBegs = NDArrayFactory::create<int>('c', {numClasses}, context);

        classesRangesBegs.assign(indices->lengthOf());
        classesRangesLens.assign(0);
        dim3 dims(numClasses, indices->lengthOf(), numClasses * 32 + 32);
        fillUpSegments(indices, numClasses, classesRangesBegs, classesRangesLens);
        int* begins = reinterpret_cast<int*>(classesRangesBegs.specialBuffer());
        int* lengths = reinterpret_cast<int*>(classesRangesLens.specialBuffer());

        if (input->isVector()) {
            Nd4jLong loop_size = input->lengthOf();
            auto numOfClasses = gradOut->lengthOf(); //indices->e<Nd4jLong>(loop_size - 1);
            segmentMeanBPLinearKernel<T,I><<<gradOut->lengthOf(), input->lengthOf(), 256, *stream>>>(input->specialBuffer(),
                    input->specialShapeInfo(), gradOut->specialBuffer(), gradOut->specialShapeInfo(),
                    indices->specialBuffer(), indices->specialShapeInfo(), lengths, output->specialBuffer(), output->specialShapeInfo());
        }
        else {
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = sd::ConstantTadHelper::getInstance().tadForDimensions(input->shapeInfo(), dimensions);
            auto packZ = sd::ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(), dimensions);
//            auto packGradIn = sd::ConstantTadHelper::getInstance().tadForDimensions(tempRes.shapeInfo(), dimensions);
            auto packGradOut = sd::ConstantTadHelper::getInstance().tadForDimensions(gradOut->shapeInfo(), dimensions);
            Nd4jLong const*  inputTads = packX.specialShapeInfo();
            Nd4jLong const*  inputTadOffsets = packX.specialOffsets();
            Nd4jLong const*  outputTads = packZ.specialShapeInfo();
            Nd4jLong const*  outputTadOffsets = packZ.specialOffsets();
            Nd4jLong const*  gradOutTads = packGradOut.specialShapeInfo();
            Nd4jLong const*  gradOutTadOffsets = packGradOut.specialOffsets();

            segmentMeanBPTadKernel<T,I><<<indices->lengthOf(), input->lengthOf(), 256, *stream>>>(input->specialBuffer(), input->specialShapeInfo(),
                    gradOut->specialBuffer(), gradOut->specialShapeInfo(), indices->specialBuffer(), indices->specialShapeInfo(), lengths,
                    output->specialBuffer(), output->specialShapeInfo(), inputTads, inputTadOffsets, gradOutTads, gradOutTadOffsets,
                    outputTads, outputTadOffsets);
        }
        NDArray::registerSpecialUse({output}, {input, indices, gradOut});
        return Status::OK();
    }
    // -------------------------------------------------------------------------------------------------------------- //
    int unsortedSegmentMeanFunctorBP(sd::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        NDArray::prepareSpecialUse({output}, {input, indices, gradOut});
        BUILD_DOUBLE_SELECTOR(output->dataType(), indices->dataType(), return unsortedSegmentMeanFunctorBP_, (context, input, indices, gradOut, numOfClasses, output), FLOAT_TYPES, INDEXING_TYPES);
        NDArray::registerSpecialUse({output}, {input, indices, gradOut});
    }

}
}
}