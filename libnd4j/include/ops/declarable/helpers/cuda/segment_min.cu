#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author GS <sgazeos@gmail.com>
//

#include <ops/declarable/helpers/segment.h>
#include <ops/declarable/helpers/segment_common.h>
#include <array/NDArrayFactory.h>
#include <helpers/ShapeUtils.h>
#include <helpers/TAD.h>
#include <exceptions/cuda_exception.h>
#include <helpers/PointersManager.h>
#include <helpers/ConstantTadHelper.h>

namespace sd {
namespace ops {
namespace helpers {
    // -------------------------------------------------------------------------------------------------------------- //
    // Segment ops linear kernels
    // -------------------------------------------------------------------------------------------------------------- //

    template<typename T, typename I>
    static __global__ void
    segmentMinLinearKernel(const void *input, const Nd4jLong *inputShape, int *starts, int *lengths, Nd4jLong numOfClasses,
                           void *output, const Nd4jLong *outputShape) {
        __shared__        T *val;
        __shared__        Nd4jLong xLen, zLen, zIndex;
        __shared__        const T *x;
        __shared__        T *z;
        __shared__ int threadsPerSegment, start, finish;

        auto segment = blockIdx.x;
        if (threadIdx.x == 0) {
//            threadsPerSegment = (gridDim.x + numOfClasses - 1) / numOfClasses;
//            segment = blockIdx.x / threadsPerSegment;
            x = reinterpret_cast<const T *>(input);
            z = reinterpret_cast<T *>(output);
            extern __shared__ unsigned char shmem[];
            val = reinterpret_cast<T *>(shmem);
            xLen = shape::length(inputShape);
            zLen = shape::length(outputShape);

            if (segment < numOfClasses) {
                zIndex = shape::getIndexOffset(segment, outputShape);
                start = starts[segment];
                finish = start + lengths[segment];
                z[zIndex] = x[shape::getIndexOffset(start, inputShape)];
                val[segment] = z[zIndex];
            }

        }
        __syncthreads();

        for (auto e = start + threadIdx.x + 1; e < finish; e += blockDim.x) {
            auto xIndex = shape::getIndexOffset(e, inputShape);
            sd::math::atomics::nd4j_atomicMin(&z[zIndex], x[xIndex]);
        }

    }
    // -------------------------------------------------------------------------------------------------------------- //

    template<typename T, typename I>
    static __global__ void
    unsortedSegmentMinLinearKernel(const void *input, const Nd4jLong *inputShape, const void *indices, const Nd4jLong *indicesShape,
                                   int *starts, int *lengths, Nd4jLong numOfClasses, void *output,
                                   const Nd4jLong *outputShape) {
        __shared__
        T *val;
        __shared__
        Nd4jLong xLen, zLen, segment, zIndex;
        __shared__
        const T *x;
        __shared__
        T *z;
        __shared__
        const I *y; //int threadsPerSegment, start, finish;

        if (threadIdx.x == 0) {
            segment = blockIdx.x;
            x = reinterpret_cast<const T *>(input);
            z = reinterpret_cast<T *>(output);
            y = reinterpret_cast<const I *>(indices);
            xLen = shape::length(inputShape);
            zLen = shape::length(outputShape);

            zIndex = shape::getIndexOffset(segment, outputShape);
            if (lengths[segment] > 0)
                z[zIndex] = x[shape::getIndexOffset(starts[segment], inputShape)];
            else
                z[zIndex] = DataTypeUtils::max<T>();

        }
        __syncthreads();

        if (lengths[segment] > 0)
            for (auto e = threadIdx.x + 1; e < xLen; e += blockDim.x) {
                auto xIndex = shape::getIndexOffset(e, inputShape);
                auto yIndex = shape::getIndexOffset(e, indicesShape);
                if (y[yIndex] == segment) {
                    sd::math::atomics::nd4j_atomicMin(&z[zIndex], x[xIndex]);
                }
            }
    }
    // -------------------------------------------------------------------------------------------------------------- //
// SegmentMin kernel
    template <typename T, typename I>
    static __global__ void segmentMinTadKernel(const void* inputBuf, const Nd4jLong* inputShape, const Nd4jLong* inputTads, const Nd4jLong* inputTadOffsets, I* indices, int* starts, int* lengths, Nd4jLong numOfClasses, void* outputBuf, const Nd4jLong* outputShape, const Nd4jLong* outputTads, const Nd4jLong* outputTadOffsets) {
        __shared__ T* val;
        __shared__ Nd4jLong len, zIndex, total;
        __shared__ T* z;
        __shared__ int threadsPerSegment, start, finish;

        auto segment = indices[blockIdx.x]; // / threadsPerSegment;
        if (threadIdx.x == 0) {
            z = reinterpret_cast<T*>(outputBuf) + outputTadOffsets[segment];
            len = shape::length(inputTads);
            start = starts[segment];
            finish = start + lengths[segment];
            total = shape::sizeAt(inputShape, 0);

        }
        __syncthreads();

        auto idx = blockIdx.x;
        if (blockIdx.x <= total) {
            auto x = reinterpret_cast<const T *>(inputBuf) + inputTadOffsets[idx];
            if (blockIdx.x == start) {
                for (auto e = threadIdx.x; e < len; e += blockDim.x) {
                    auto xIndex = shape::getIndexOffset(e, inputTads);
                    auto zIndex = shape::getIndexOffset(e, outputTads);
                    sd::math::atomics::nd4j_atomicMin(&z[zIndex], x[xIndex]);
                }
            }
            else {
                for (auto e = threadIdx.x; e < len; e += blockDim.x) {
                    auto xIndex = shape::getIndexOffset(e, inputTads);
                    auto zIndex = shape::getIndexOffset(e, outputTads);
//                    if (lengths[indices[idx]])
                        sd::math::atomics::nd4j_atomicMin(&z[zIndex], x[xIndex]);
                }
            }
        }
    }
    // -------------------------------------------------------------------------------------------------------------- //
    // segmen min
    template <typename T, typename I>
    static void segmentMinFunctor_(LaunchContext* context, NDArray* input, NDArray* indices, NDArray* output) {
        auto stream = context->getCudaStream();
        Nd4jLong numClasses = indices->e<Nd4jLong>(indices->lengthOf() - 1) + 1;
        auto classesRangesLens = NDArrayFactory::create<int>('c', {numClasses}, context);
        auto classesRangesBegs = NDArrayFactory::create<int>('c', {numClasses}, context);
        output->assign(DataTypeUtils::infOrMax<T>());
        classesRangesBegs.assign(indices->lengthOf());
        classesRangesLens.assign(0);

        fillUpSegments(indices, numClasses, classesRangesBegs, classesRangesLens);
        NDArray::prepareSpecialUse({output}, {input, indices, &classesRangesBegs, &classesRangesLens});
        int* begins = reinterpret_cast<int*>(classesRangesBegs.specialBuffer());
        int* lengths = reinterpret_cast<int*>(classesRangesLens.specialBuffer());
        if (input->isVector()) {
            segmentMinLinearKernel<T,I><<<numClasses, input->lengthOf(), numClasses * 32 + 32, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), begins, lengths, numClasses, output->specialBuffer(), output->specialShapeInfo());
        }
        else {
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = sd::ConstantTadHelper::getInstance().tadForDimensions(input->shapeInfo(), dimensions);
            auto packZ = sd::ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(), dimensions);
            auto inputTads = packX.specialShapeInfo();
            auto inputTadOffsets = packX.specialOffsets();
            auto outputTads = packZ.specialShapeInfo();
            auto outputTadOffsets = packZ.specialOffsets();
            segmentMinTadKernel<T,I><<<input->sizeAt(0), 512, 2048, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), inputTads, inputTadOffsets, reinterpret_cast<I*>(indices->specialBuffer()), begins, lengths, numClasses, output->specialBuffer(), output->specialShapeInfo(), outputTads, outputTadOffsets);

        }
        NDArray::registerSpecialUse({output}, {input, indices, &classesRangesBegs, &classesRangesLens});

    }
    // -------------------------------------------------------------------------------------------------------------- //
    void segmentMinFunctor(sd::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* output) {
        NDArray::prepareSpecialUse({output}, {input, indices});
        output->nullify();
        BUILD_DOUBLE_SELECTOR(input->dataType(), indices->dataType(), segmentMinFunctor_, (context, input, indices, output), NUMERIC_TYPES, INDEXING_TYPES);
        NDArray::registerSpecialUse({output}, {input, indices});
    }

    // -------------------------------------------------------------------------------------------------------------- //

    template <typename T, typename I>
    static void unsortedSegmentMinFunctor_(sd::LaunchContext* context, NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {
        auto stream = context->getCudaStream();
//        NDArray classes = NDArrayFactory::create<int>('c', {numOfClasses, 2});
        NDArray classesRangesBegs = NDArrayFactory::create<int>('c', {numOfClasses}, context);
        NDArray classesRangesLens = NDArrayFactory::create<int>('c', {numOfClasses}, context);
//        NDArray row = NDArrayFactory::create<int>('c', {1, 2}, {(int)indices->lengthOf(), (int)0});
//        classes.applyTrueBroadcast(sd::BroadcastOpsTuple::Assign(), &row, &classes);
        output->assign(DataTypeUtils::infOrMax<T>());
        classesRangesBegs.assign(indices->lengthOf());
        classesRangesLens.assign(0);
        dim3 dims(numOfClasses, indices->lengthOf(), numOfClasses * 32 + 32);
//        int* classesBuf = reinterpret_cast<int*>(classes.specialBuffer());
        fillUpSegments(indices, numOfClasses, classesRangesBegs, classesRangesLens);
        int* begins = reinterpret_cast<int*>(classesRangesBegs.specialBuffer());
        int* lengths = reinterpret_cast<int*>(classesRangesLens.specialBuffer());
        NDArray::prepareSpecialUse({output}, {input, indices});
        if (input->isVector()) {
            unsortedSegmentMinLinearKernel<T,I><<<dims.x, dims.y, dims.z, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), indices->specialBuffer(), indices->specialShapeInfo(), begins, lengths, numOfClasses, output->specialBuffer(), output->specialShapeInfo());
        }
        else {
            output->assign(DataTypeUtils::max<T>());
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = sd::ConstantTadHelper::getInstance().tadForDimensions(input->shapeInfo(), dimensions);
            auto packZ = sd::ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(), dimensions);
            auto inputTads = packX.specialShapeInfo();
            auto inputTadOffsets = packX.specialOffsets();
            auto outputTads = packZ.specialShapeInfo();
            auto outputTadOffsets = packZ.specialOffsets();
            dims.x = input->sizeAt(0);
            segmentMinTadKernel<T,I><<<dims.x, dims.y, dims.z, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), inputTads, inputTadOffsets, reinterpret_cast<I*>(indices->specialBuffer()), begins, lengths, numOfClasses, output->specialBuffer(), output->specialShapeInfo(), outputTads, outputTadOffsets);
        }
        NDArray::registerSpecialUse({output}, {input, indices});

    }
    // -------------------------------------------------------------------------------------------------------------- //
    void unsortedSegmentMinFunctor(sd::LaunchContext* context , NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {
        NDArray::prepareSpecialUse({output}, {input, indices});
        output->nullify();
        BUILD_DOUBLE_SELECTOR(input->dataType(), indices->dataType(), unsortedSegmentMinFunctor_, (context, input, indices, numOfClasses, output),
                              NUMERIC_TYPES, INDEXING_TYPES);
        NDArray::registerSpecialUse({output}, {input, indices});
    }

    template <typename T, typename I>
    static __global__ void segmentMinBPLinearKernel(const void* inputBuf, const Nd4jLong* inputShape, void* forwardOutput,
                                                    const Nd4jLong* forwardShape, void* eps, const Nd4jLong* epsShape, const void* indicesBuf, const Nd4jLong* indicesShape,
                                                    void* outputBuf, const Nd4jLong* outputShape) {
        __shared__ const T* x;
        __shared__ T* gradIn;
        __shared__ T* gradOut;
        __shared__ const I* y;
        __shared__ T* z;
        __shared__ Nd4jLong xLen, gradLen;

        if (threadIdx.x == 0) {
            xLen = shape::length(inputShape);
            x = reinterpret_cast<const T*>(inputBuf);
            y = reinterpret_cast<const I*>(indicesBuf);
            z = reinterpret_cast<T*>(outputBuf);
            gradIn = reinterpret_cast<T*>(forwardOutput);
            gradOut = reinterpret_cast<T*>(eps);
            gradLen = shape::length(epsShape);
        }
        __syncthreads();

        auto start = blockIdx.x * blockDim.x + threadIdx.x;
        auto step = gridDim.x * blockDim.x;

        for (auto e = start; e < xLen; e += step) {

            auto zOffset = shape::getIndexOffset(e, outputShape);
            auto xOffset = shape::getIndexOffset(e, inputShape);
            auto yOffset = shape::getIndexOffset(e, indicesShape);
            auto classIndex = y[yOffset];
            auto gradOffsetI = shape::getIndexOffset(classIndex, forwardShape);
            auto gradOffsetO = shape::getIndexOffset(classIndex, epsShape);

            if (sd::math::nd4j_abs(gradIn[gradOffsetI] - x[xOffset]) <= T(1.e-6)) {
                z[zOffset] = gradOut[gradOffsetO];
            }
        }
    }

    // -------------------------------------------------------------------------------------------------------------- //
    template <typename T, typename I>
    static __global__ void segmentMinBPTadKernel(const void* inputBuf, const Nd4jLong* inputShape, void* forwardOutput,
                                                 const Nd4jLong* forwardShape, void* eps, const Nd4jLong* epsShape,
                                                 const void* indicesBuf, const Nd4jLong* indicesShape,
                                                 void* outputBuf, const Nd4jLong* outputShape,
                                                 const Nd4jLong* inputTad, const Nd4jLong* inputOffsets,
                                                 const Nd4jLong* gradInTad, const Nd4jLong* gradInOffsets,
                                                 const Nd4jLong* gradOutTad, const Nd4jLong* gradOutOffsets,
                                                 const Nd4jLong* outTad, const Nd4jLong* outOffsets) {
        __shared__ const T* x;
        __shared__ T* gradIn;
        __shared__ T* gradOut;
        __shared__ const I* y;
        __shared__ T* z;
        __shared__ Nd4jLong xLen, yLen, gradLen, currentLen;

        if (threadIdx.x == 0) {
            xLen = shape::length(inputShape);
            x = reinterpret_cast<const T*>(inputBuf);
            y = reinterpret_cast<const I*>(indicesBuf);
            z = reinterpret_cast<T*>(outputBuf);
            yLen = shape::length(indicesShape);
            gradOut = reinterpret_cast<T*>(eps);
            gradIn = reinterpret_cast<T*>(forwardOutput);
            gradLen = shape::length(epsShape);
            currentLen = shape::length(outTad);
        }
        __syncthreads();

        for (auto i = blockIdx.x; i < yLen; i += gridDim.x) {
            auto yIndex = shape::getIndexOffset(i, indicesShape);
            auto segment = y[yIndex];
            auto current = x + inputOffsets[i];
            auto currentOut = z + outOffsets[i];
            auto in = gradIn + gradInOffsets[segment];
            auto outGrad = gradOut + gradOutOffsets[segment];

            for (auto e = threadIdx.x; e < currentLen; e += blockDim.x) {
                if (sd::math::nd4j_abs(in[e] - current[e]) <= T(1.e-6))
                    currentOut[e] = outGrad[e];
            }
        }
    }

    // -------------------------------------------------------------------------------------------------------------- //
    template <typename T, typename I>
    int segmentMinFunctorBP_(sd::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output) {
        //int numOfClasses = gradOut->sizeAt(0);
        // if input is a vector: (as if in doc sample)
        auto stream = context->getCudaStream();
        NDArray tempRes(gradOut->ordering(), gradOut->getShapeAsVector(), DataTypeUtils::fromT<T>(), context);//->shapeInfo(), context);
        segmentMinFunctor_<T, I>(context, input, indices, &tempRes);
        NDArray::prepareSpecialUse({output}, {input, indices, gradOut, &tempRes});
        if (input->isVector()) {
            Nd4jLong loop_size = input->lengthOf();
            auto numOfClasses = gradOut->lengthOf(); //indices->e<Nd4jLong>(loop_size - 1);

            segmentMinBPLinearKernel<T,I><<<gradOut->lengthOf(), input->lengthOf(), 256, *stream>>>(input->specialBuffer(), input->specialShapeInfo(),
                    tempRes.specialBuffer(), tempRes.specialShapeInfo(), gradOut->specialBuffer(), gradOut->specialShapeInfo(),
                    indices->specialBuffer(), indices->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo());
        }
        else {
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = sd::ConstantTadHelper::getInstance().tadForDimensions(input->shapeInfo(), dimensions);
            auto packZ = sd::ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(), dimensions);
            auto packGradIn = sd::ConstantTadHelper::getInstance().tadForDimensions(tempRes.shapeInfo(), dimensions);
            auto packGradOut = sd::ConstantTadHelper::getInstance().tadForDimensions(gradOut->shapeInfo(), dimensions);
            auto inputTads = packX.specialShapeInfo();
            auto inputTadOffsets = packX.specialOffsets();
            auto outputTads = packZ.specialShapeInfo();
            auto outputTadOffsets = packZ.specialOffsets();
            auto gradInTads = packGradIn.specialShapeInfo();
            auto gradInTadOffsets = packGradIn.specialOffsets();
            auto gradOutTads = packGradOut.specialShapeInfo();
            auto gradOutTadOffsets = packGradOut.specialOffsets();

            segmentMinBPTadKernel<T,I><<<gradOut->lengthOf(), input->lengthOf(), 256, *stream>>>(input->specialBuffer(), input->specialShapeInfo(),
                    tempRes.specialBuffer(), tempRes.specialShapeInfo(), gradOut->specialBuffer(), gradOut->specialShapeInfo(),
                    indices->specialBuffer(), indices->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo(),
                    inputTads, inputTadOffsets, gradInTads, gradInTadOffsets, gradOutTads, gradOutTadOffsets,
                    outputTads, outputTadOffsets);
        }
        NDArray::registerSpecialUse({output}, {input, indices, gradOut, &tempRes});
        return Status::OK();
    }
    // -------------------------------------------------------------------------------------------------------------- //
    // segmen min
    int segmentMinFunctorBP(sd::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output) {
        NDArray::prepareSpecialUse({output}, {input, indices, gradOut});
        BUILD_DOUBLE_SELECTOR(output->dataType(), indices->dataType(), return segmentMinFunctorBP_, (context, input,
                indices, gradOut, output), FLOAT_TYPES, INDEXING_TYPES);
        NDArray::registerSpecialUse({output}, {input, indices, gradOut});
    }

    template <typename T, typename I>
    static int unsortedSegmentMinFunctorBP_(sd::LaunchContext* context, NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        //int numOfClasses = gradOut->sizeAt(0);
        // if input is a vector: (as if in doc sample)
        auto stream = context->getCudaStream();
        NDArray tempRes(gradOut->ordering(), gradOut->getShapeAsVector(), DataTypeUtils::fromT<T>(), context);//->shapeInfo(), context);
        unsortedSegmentMinFunctor_<T, I>(context, input, indices, numOfClasses, &tempRes);
        NDArray::prepareSpecialUse({output}, {input, indices, gradOut, &tempRes});
        if (input->isVector()) {
            Nd4jLong loop_size = input->lengthOf();
            auto numOfClasses = gradOut->lengthOf(); //indices->e<Nd4jLong>(loop_size - 1);
            segmentMinBPLinearKernel<T,I><<<gradOut->lengthOf(), input->lengthOf(), 256, *stream>>>(input->specialBuffer(), input->specialShapeInfo(),
                    tempRes.specialBuffer(), tempRes.specialShapeInfo(), gradOut->specialBuffer(), gradOut->specialShapeInfo(),
                    indices->specialBuffer(), indices->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo());
        }
        else {
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = sd::ConstantTadHelper::getInstance().tadForDimensions(input->shapeInfo(), dimensions);
            auto packZ = sd::ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(), dimensions);
            auto packGradIn = sd::ConstantTadHelper::getInstance().tadForDimensions(tempRes.shapeInfo(), dimensions);
            auto packGradOut = sd::ConstantTadHelper::getInstance().tadForDimensions(gradOut->shapeInfo(), dimensions);
            auto inputTads = packX.specialShapeInfo();
            auto inputTadOffsets = packX.specialOffsets();
            auto outputTads = packZ.specialShapeInfo();
            auto outputTadOffsets = packZ.specialOffsets();
            auto gradInTads = packGradIn.specialShapeInfo();
            auto gradInTadOffsets = packGradIn.specialOffsets();
            auto gradOutTads = packGradOut.specialShapeInfo();
            auto gradOutTadOffsets = packGradOut.specialOffsets();

            segmentMinBPTadKernel<T,I><<<gradOut->lengthOf(), input->lengthOf(), 256, *stream>>>(input->specialBuffer(), input->specialShapeInfo(),
                    tempRes.specialBuffer(), tempRes.specialShapeInfo(), gradOut->specialBuffer(), gradOut->specialShapeInfo(),
                    indices->specialBuffer(), indices->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo(),
                    inputTads, inputTadOffsets, gradInTads, gradInTadOffsets, gradOutTads, gradOutTadOffsets,
                    outputTads, outputTadOffsets);
        }
        NDArray::registerSpecialUse({output}, {input, indices, gradOut, &tempRes});
        return Status::OK();
    }
    // -------------------------------------------------------------------------------------------------------------- //
    int unsortedSegmentMinFunctorBP(sd::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        NDArray::prepareSpecialUse({output}, {input, indices, gradOut});
        BUILD_DOUBLE_SELECTOR(output->dataType(), indices->dataType(), return unsortedSegmentMinFunctorBP_, (context, input, indices, gradOut, numOfClasses, output), FLOAT_TYPES, INDEXING_TYPES);
        NDArray::registerSpecialUse({output}, {input, indices, gradOut});
    }
}
}
}