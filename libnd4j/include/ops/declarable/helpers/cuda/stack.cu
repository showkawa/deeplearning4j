#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// Created by Yurii Shyrma on 02.01.2018
//

#include <ops/declarable/helpers/stack.h>
#include <helpers/ShapeUtils.h>
#include <array/ResultSet.h>
#include <exceptions/cuda_exception.h>
#include <helpers/TAD.h>
#include <helpers/PointersManager.h>
#include <helpers/ConstantTadHelper.h>

namespace sd {
namespace ops {
namespace helpers {


///////////////////////////////////////////////////////////////////
template <typename T>
static __global__ void stackScalarsCuda(void* pVx, void* vz, const Nd4jLong* zShapeInfo) {

    T* z = reinterpret_cast<T*>(vz);

    __shared__ Nd4jLong zLen, totalThreads;

    if (threadIdx.x == 0) {
        zLen  = shape::length(zShapeInfo);
        totalThreads = gridDim.x * blockDim.x;
    }
    __syncthreads();

    const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (Nd4jLong i = tid; i < zLen; i += totalThreads) {

        const T *x = reinterpret_cast<const T*>(reinterpret_cast<void**>(pVx)[i]);
        z[shape::getIndexOffset(i, zShapeInfo)] = *x;
    }
}


///////////////////////////////////////////////////////////////////
template<typename T>
__host__ static void stackScalarsCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const hipStream_t *stream,
                                             void* pVx, void* vz, const Nd4jLong* zShapeInfo) {

    stackScalarsCuda<T><<<blocksPerGrid, threadsPerBlock, 256, *stream>>>(pVx, vz, zShapeInfo);
}

///////////////////////////////////////////////////////////////////
template <typename T>
static void stack_(sd::LaunchContext* context, const std::vector<const NDArray*>& inArrs, NDArray& output, const int dim) {

    const int numOfSubArrs = inArrs.size();

    NDArray::prepareSpecialUse({&output}, inArrs);

    if(inArrs[0]->rankOf() == 0) {

        std::vector<void const*> hInBuffers(numOfSubArrs);

        for(int i = 0; i < numOfSubArrs; ++i)
            hInBuffers[i] = inArrs[i]->specialBuffer();

        PointersManager manager(context, "helpers::stack cuda");

        void* dInBuffers = manager.replicatePointer(hInBuffers.data(), hInBuffers.size() * sizeof(void*));

        const int threadsPerBlock = MAX_NUM_THREADS / 2;
        const int blocksPerGrid = (output.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;

        stackScalarsCudaLauncher<T>(blocksPerGrid, threadsPerBlock, context->getCudaStream(), dInBuffers, output.specialBuffer(), output.specialShapeInfo());

        manager.synchronize();
    }
    else {

        auto zTadPack = ConstantTadHelper::getInstance().tadForDimensions(output.shapeInfo(), ShapeUtils::evalDimsToExclude(output.rankOf(), {dim}));
        auto zTadShapeInfo  = zTadPack.primaryShapeInfo();

        for (uint i = 0; i < numOfSubArrs; ++i) {

            void* zBuff = output.specialBufferWithOffset(zTadPack.primaryOffsets()[i]);

            NativeOpExecutioner::execTransformAny(context, transform::Assign,
                                                 nullptr, inArrs[i]->shapeInfo(), inArrs[i]->specialBuffer(), inArrs[i]->specialShapeInfo(),
                                                 nullptr, zTadShapeInfo,             zBuff,                         zTadPack.specialShapeInfo(),
                                                 nullptr, nullptr, nullptr, false/*allowParallelism*/);
        }
    }

   NDArray::registerSpecialUse({&output}, inArrs);
}

////////////////////////////////////////////////////////////////////////
void stack(sd::LaunchContext* context, const std::vector<const NDArray*>& inArrs, NDArray& output, const int dim) {
    BUILD_SINGLE_SELECTOR(output.dataType(), stack_, (context, inArrs, output, dim), LIBND4J_TYPES);
}
BUILD_SINGLE_TEMPLATE(template void stack_ , (sd::LaunchContext* context, const std::vector<const NDArray*>& inArrs, NDArray& output, const int dim), LIBND4J_TYPES);


///////////////////////////////////////////////////////////////////
template <typename T>
static __global__ void unstackScalarsCuda(const void* vx, const Nd4jLong* xShapeInfo, void* pVz) {

    const T* x = reinterpret_cast<const T*>(vx);

    __shared__ Nd4jLong xLen, totalThreads;

    if (threadIdx.x == 0) {
        xLen  = shape::length(xShapeInfo);
        totalThreads = gridDim.x * blockDim.x;
    }
    __syncthreads();

    const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (Nd4jLong i = tid; i < xLen; i += totalThreads) {

        T* z = reinterpret_cast<T*>(reinterpret_cast<void**>(pVz)[i]);
        *z = x[shape::getIndexOffset(i, xShapeInfo)];
    }
}


///////////////////////////////////////////////////////////////////
template<typename T>
__host__ static void unstackScalarsCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const hipStream_t *stream,
                                                const void* vx, const Nd4jLong* xShapeInfo, void* pVz) {

    unstackScalarsCuda<T><<<blocksPerGrid, threadsPerBlock, 256, *stream>>>(vx, xShapeInfo, pVz);
}

///////////////////////////////////////////////////////////////////
template <typename T>
static void unstack_(sd::LaunchContext* context, const NDArray& input, const std::vector<NDArray*>& outArrs, const int dim) {

    const int numOfSubArrs = outArrs.size();

    // NDArray::prepareSpecialUse(outArrs, {&input});
    input.syncToDevice();
    for (const auto a : outArrs)
        a->getDataBuffer()->allocateSpecial();


    if(outArrs[0]->rankOf() == 0) {

        std::vector<void*> hOutBuffers(numOfSubArrs);

        for(int i = 0; i < numOfSubArrs; ++i)
            hOutBuffers[i] = outArrs[i]->specialBuffer();

        PointersManager manager(context, "helpers::unstack cuda");

        void* dOutBuffers = manager.replicatePointer(hOutBuffers.data(), hOutBuffers.size() * sizeof(void*));

        const int threadsPerBlock = MAX_NUM_THREADS / 2;
        const int blocksPerGrid = (input.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;

        unstackScalarsCudaLauncher<T>(blocksPerGrid, threadsPerBlock, context->getCudaStream(), input.specialBuffer(), input.specialShapeInfo(), dOutBuffers);

        manager.synchronize();
    }
    else {

        auto xTadPack = ConstantTadHelper::getInstance().tadForDimensions(input.shapeInfo(), ShapeUtils::evalDimsToExclude(input.rankOf(), {dim}));
        auto xTadShapeInfo  = xTadPack.primaryShapeInfo();

        for (uint i = 0; i < numOfSubArrs; ++i) {

            auto xBuff = input.specialBufferWithOffset(xTadPack.primaryOffsets()[i]);

            NativeOpExecutioner::execTransformAny(input.getContext(), transform::Assign,
                                                 nullptr, xTadShapeInfo,              xBuff,                       xTadPack.specialShapeInfo(),
                                                 nullptr, outArrs[i]->shapeInfo(), outArrs[i]->specialBuffer(), outArrs[i]->specialShapeInfo(),
                                                 nullptr, nullptr, nullptr, false/*allowParallelism*/);
        }
    }

    // NDArray::registerSpecialUse(outArrs, {&input});
    input.tickReadDevice();
    for (const auto p : outArrs)
        p->tickWriteDevice();
}

////////////////////////////////////////////////////////////////////////
void unstack(sd::LaunchContext* context, const NDArray& input, const std::vector<NDArray*>& outArrs, const int dim) {
    BUILD_SINGLE_SELECTOR(input.dataType(), unstack_, (context, input, outArrs, dim), LIBND4J_TYPES);
}
BUILD_SINGLE_TEMPLATE(template void unstack_, (sd::LaunchContext* context, const NDArray& input, const std::vector<NDArray*>& outArrs, const int dim), LIBND4J_TYPES);

///////////////////////////////////////////////////////////////////
// template <typename T>
// static __global__ void unstackCuda(const void* vx, const Nd4jLong* xShapeInfo, void* pVz, const Nd4jLong* zTadShapeInfo, const int axis) {

// 	const T* x = reinterpret_cast<const T*>(vx);
//     __shared__ Nd4jLong xLen, totalThreads;
//     __shared__ int xRank;

//     if (threadIdx.x == 0) {
//         xLen  = shape::length(xShapeInfo);
//         xRank = shape::rank(xShapeInfo);
//         totalThreads = gridDim.x * blockDim.x;
//     }
//     __syncthreads();

//     const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

//     Nd4jLong coords[MAX_RANK];

//     for (uint64_t i = tid; i < xLen; i += totalThreads) {

//         shape::index2coords(i, xShapeInfo, coords);

//         const auto xOffset = shape::getOffset(xShapeInfo, coords);

//         T *z = reinterpret_cast<T*>(reinterpret_cast<void **>(pVz)[coords[axis]]);

//         for (uint j = axis; j < xRank - 1; ++j)	// shift coords staring from axis position
//         	coords[j] = coords[j + 1];

//         const auto zOffset = shape::getOffset(zTadShapeInfo, coords);

//         z[zOffset] = x[xOffset];
//     }
// }

// ///////////////////////////////////////////////////////////////////
// template<typename T>
// __host__ static void unstackCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const hipStream_t *stream,
// 										 const void* vx, const Nd4jLong* xShapeInfo, void* pVz, const Nd4jLong* zTadShapeInfo, const int axis) {

//     unstackCuda<T><<<blocksPerGrid, threadsPerBlock, 256, *stream>>>(vx, xShapeInfo, pVz, zTadShapeInfo, axis);
// }
// BUILD_SINGLE_TEMPLATE(template void unstackCudaLauncher, (const int blocksPerGrid, const int threadsPerBlock, const hipStream_t *stream, const void* vx, const Nd4jLong* xShapeInfo, void* pVz, const Nd4jLong* zTadShapeInfo, const int axis), LIBND4J_TYPES);


// ///////////////////////////////////////////////////////////////////
// void unstack(sd::LaunchContext* context, const NDArray& input, const std::vector<const NDArray*>& outArrs, const int axis) {

// 	const int threadsPerBlock = MAX_NUM_THREADS / 2;
// 	const int blocksPerGrid = (input.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;

// 	const int numOfSubArrs = outArrs.size();

//     std::vector<void*> hOutBuffers(numOfSubArrs);

//     for(int i = 0; i < numOfSubArrs; ++i)
//         hOutBuffers[i] = outArrs[i]->specialBuffer();

//     PointersManager manager(context, "helpers::unstack");

//     void* dOutBuffers = manager.replicatePointer(hOutBuffers.data(), hOutBuffers.size() * sizeof(void*));

//     for(uint i = 0; i < numOfSubArrs; ++i)
// 		outArrs[i]->syncToDevice();
//     input.syncToDevice();

//     BUILD_SINGLE_SELECTOR(input.dataType(), unstackCudaLauncher, (blocksPerGrid, threadsPerBlock, context->getCudaStream(), input.specialBuffer(), input.specialShapeInfo(), dOutBuffers, outArrs[0]->special(), axis), LIBND4J_TYPES);

//     manager.synchronize();

//     for(uint i = 0; i < numOfSubArrs; ++i)
//         outArrs[i]->tickReadDevice();
//     input.tickWriteDevice();
// }


// ///////////////////////////////////////////////////////////////////
// template <typename T>
// static __global__ void stackCuda(void* pVx, const Nd4jLong* xTadShapeInfo, void* vz, const Nd4jLong* zShapeInfo, const int axis) {

// 	T* z = reinterpret_cast<T*>(vz);

//     __shared__ Nd4jLong zLen, totalThreads;
//     __shared__ int zRank;

//     if (threadIdx.x == 0) {
//         zLen  = shape::length(zShapeInfo);
//         zRank = shape::rank(zShapeInfo);
//         totalThreads = gridDim.x * blockDim.x;
//     }
//     __syncthreads();

//     const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

//     Nd4jLong coords[MAX_RANK];

//     for (uint64_t i = tid; i < zLen; i += totalThreads) {

//         shape::index2coords(i, zShapeInfo, coords);

//         const auto zOffset = shape::getOffset(zShapeInfo, coords);

//         const T *x = reinterpret_cast<const T*>(reinterpret_cast<void**>(pVx)[coords[axis]]);

//         for (uint j = axis; j < zRank - 1; ++j)	// shift coords staring from axis position
//         	coords[j] = coords[j + 1];

//         const auto xOffset = shape::getOffset(xTadShapeInfo, coords);

//         z[zOffset] = x[xOffset];
//     }
// }

// ///////////////////////////////////////////////////////////////////
// template<typename T>
// __host__ static void stackCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const hipStream_t *stream,
// 					 				   void* pVx, const Nd4jLong* xTadShapeInfo, void* vz, const Nd4jLong* zShapeInfo, const int axis) {

//     stackCuda<T><<<blocksPerGrid, threadsPerBlock, 256, *stream>>>(pVx, xTadShapeInfo, vz, zShapeInfo, axis);
// }
// BUILD_SINGLE_TEMPLATE(template void stackCudaLauncher, (const int blocksPerGrid, const int threadsPerBlock, const hipStream_t *stream, void* pVx, const Nd4jLong* xTadShapeInfo, void* vz, const Nd4jLong* zShapeInfo, const int axis), LIBND4J_TYPES);


// ///////////////////////////////////////////////////////////////////
// void stack(sd::LaunchContext* context, const std::vector<const NDArray*>& inArrs, NDArray& output, const int axis) {

// 	const int threadsPerBlock = MAX_NUM_THREADS / 2;
// 	const int blocksPerGrid = (output.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;

// 	const int numOfSubArrs = inArrs.size();

//     std::vector<void*> hInBuffers(numOfSubArrs);

//     for(int i = 0; i < numOfSubArrs; ++i)
//         hInBuffers[i] = inArrs[i]->specialBuffer();

//     PointersManager manager(context, "helpers::stack");

//     void* dInBuffers = manager.replicatePointer(hInBuffers.data(), hInBuffers.size() * sizeof(void*));

//     for(uint i = 0; i < numOfSubArrs; ++i)
// 		inArrs[i]->syncToDevice();
//     output.syncToDevice();

//     BUILD_SINGLE_SELECTOR(output.dataType(), stackCudaLauncher, (blocksPerGrid, threadsPerBlock, context->getCudaStream(), dInBuffers, inArrs[0]->specialShapeInfo(), output.specialBuffer(), output.special(), axis), LIBND4J_TYPES);

//     manager.synchronize();

//     for(uint i = 0; i < numOfSubArrs; ++i)
//         inArrs[i]->tickReadDevice();
//     output.tickWriteDevice();
// }

}
}
}

