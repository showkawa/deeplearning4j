#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

 //
 // @author Yurii Shyrma (iuriish@yahoo.com), created on 20.04.2018
 //


#include<ops/declarable/helpers/transforms.h>
#include <array/ResultSet.h>
#include <helpers/ShapeUtils.h>
#include <numeric>
#include <array/NDArrayFactory.h>
#include <helpers/TAD.h>
#include <exceptions/cuda_exception.h>
#include <helpers/PointersManager.h>
#include <helpers/ConstantTadHelper.h>

namespace sd {
    namespace ops {
        namespace helpers {
            //////////////////////////////////////////////////////////////////////////
            template <typename T, typename Z>
            static __global__ void mergeMaxIndexCudaLauncher(void** inArrs, void** inShapes, const int numArrays, void* voutput, const Nd4jLong* outputShape, Nd4jLong length) {
                auto output = reinterpret_cast<Z*>(voutput);

                const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
                const auto step = gridDim.x * blockDim.x;

                for (Nd4jLong e = tid; e < length; e += step) {
                    T mVal = -DataTypeUtils::max<T>();
                    Z mIdx(0);

                    for (int i = 0; i < numArrays; i++) {
                        auto x = reinterpret_cast<T*>(inArrs[i]);
                        auto xShape = reinterpret_cast<Nd4jLong*>(inShapes[i]);
                        auto val = x[shape::getIndexOffset(e, xShape)];;
                        if (mVal < val) {
                            mIdx = static_cast<Z>(i);
                            mVal = val;
                        }
                    }
                    
                    output[shape::getIndexOffset(e, outputShape)] = mIdx;
                }
            }

            template <typename T, typename Z>
            static void mergeMaxIndex_(sd::LaunchContext* context, const std::vector<const NDArray*>& inArrs, NDArray& output) {
                
                int nArrSize = static_cast<int>(inArrs.size());
                std::vector<const void*> inBuffers(nArrSize), inShapes(nArrSize);

                for (int e = 0; e < nArrSize; e++) {
                    inBuffers[e] = inArrs[e]->specialBuffer();
                    inShapes[e] = inArrs[e]->specialShapeInfo();
                }

                PointersManager manager(context, "mergeMaxIndex");

                auto pInBuffers = reinterpret_cast<void**>(manager.replicatePointer(inBuffers.data(), inBuffers.size() * sizeof(void*)));
                auto pInShapes = reinterpret_cast<void**>(manager.replicatePointer(inShapes.data(), inShapes.size() * sizeof(void*)));
                auto length = output.lengthOf();

                const int threadsPerBlock = MAX_NUM_THREADS / 2;
                const int blocksPerGrid = (length + threadsPerBlock - 1) / threadsPerBlock;

                mergeMaxIndexCudaLauncher<T, Z><<<blocksPerGrid, threadsPerBlock, 512, *context->getCudaStream()>>>(pInBuffers, pInShapes, nArrSize, output.specialBuffer(), output.specialShapeInfo(), length);

                manager.synchronize();
            }

            void mergeMaxIndex(sd::LaunchContext* context, const std::vector<const NDArray*>& inArrs, NDArray& output) {
                
                NDArray::prepareSpecialUse({ &output }, inArrs);
                
                BUILD_DOUBLE_SELECTOR(inArrs[0]->dataType(), output.dataType(), mergeMaxIndex_, (context, inArrs, output), LIBND4J_TYPES, INDEXING_TYPES);

                NDArray::registerSpecialUse({ &output }, inArrs);
            }


            //////////////////////////////////////////////////////////////////////////
            template <typename T>
            static __global__ void mergeMaxCudaLauncher(void** inArrs, void** inShapes, const int numArrays, void* voutput, const Nd4jLong* outputShape, Nd4jLong length) {
                auto output = reinterpret_cast<T*>(voutput);

                const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
                const auto step = gridDim.x * blockDim.x;

                for (Nd4jLong e = tid; e < length; e += step) {
                    T mVal = -DataTypeUtils::max<T>();

                    for (int i = 0; i < numArrays; i++) {
                        auto x = reinterpret_cast<const T*>(inArrs[i]);
                        auto xShape = reinterpret_cast<const Nd4jLong*>(inShapes[i]);
                        auto val = x[shape::getIndexOffset(e, xShape)];;
                        if (mVal < val)
                            mVal = val;
                    }

                    output[shape::getIndexOffset(e, outputShape)] = mVal;
                }
            }

            template<typename T>
            static void mergeMax_(sd::LaunchContext* context, const std::vector<const NDArray*>& inArrs, NDArray& output) {
                
                int nArrsSize = static_cast<int>(inArrs.size());

                std::vector<const void*> inBuffers(nArrsSize), inShapes(nArrsSize);

                for (int e = 0; e < nArrsSize; e++) {
                    inBuffers[e] = inArrs[e]->specialBuffer();
                    inShapes[e] = inArrs[e]->specialShapeInfo();
                }

                PointersManager manager(context, "mergeMax");

                auto pInBuffers = reinterpret_cast<void**>(manager.replicatePointer(inBuffers.data(), inBuffers.size() * sizeof(void*)));
                auto pInShapes = reinterpret_cast<void**>(manager.replicatePointer(inShapes.data(), inShapes.size() * sizeof(void*)));
                auto length = output.lengthOf();

                const int threadsPerBlock = MAX_NUM_THREADS / 2;
                const int blocksPerGrid = (length + threadsPerBlock - 1) / threadsPerBlock;

                mergeMaxCudaLauncher<T><<<blocksPerGrid, threadsPerBlock, 512, *context->getCudaStream()>>>(pInBuffers, pInShapes, nArrsSize, output.specialBuffer(), output.specialShapeInfo(), length);

                manager.synchronize();
            }

            void mergeMax(sd::LaunchContext* context, const std::vector<const NDArray*>& inArrs, NDArray& output) {
                
                NDArray::prepareSpecialUse({ &output }, inArrs);
                
                BUILD_SINGLE_SELECTOR(output.dataType(), mergeMax_, (context, inArrs, output), LIBND4J_TYPES);
                
                NDArray::registerSpecialUse({ &output }, inArrs);
            }

            //////////////////////////////////////////////////////////////////////////
            template <typename T>
            static __global__ void mergeMaxBpCudaLauncher(
                    void** inArrs, void** inShapes,
                    const void* vgradient, const Nd4jLong* gradientShape,
                    const int numArrays,
                    void** outArrs, void** outShapes,
                    Nd4jLong length,
                    bool bSameOrderAndEws1) {

                auto grad = reinterpret_cast<const T*>(vgradient);

                const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
                const auto step = gridDim.x * blockDim.x;

                int coords[MAX_RANK];

                for (Nd4jLong e = tid; e < length; e += step) {

                    T mVal = -DataTypeUtils::max<T>();
                    int nMaxIndex = 0;
                    auto xOffset = e, zOffset = e, gradOffset = e;

                    if (!bSameOrderAndEws1) {
                        shape::index2coords(e, gradientShape, coords);
                        gradOffset = shape::getOffset(gradientShape, coords);
                    }

                    for (int i = 0; i < numArrays; i++) {
                        auto x = reinterpret_cast<T*>(inArrs[i]);

                        if (!bSameOrderAndEws1) {
                            auto xShape = reinterpret_cast<Nd4jLong*>(inShapes[i]);
                            xOffset = shape::getOffset(xShape, coords);
                        }

                        auto val = x[xOffset];
                        if (mVal < val) {
                            mVal = val;
                            nMaxIndex = i;
                        }
                    }
                  
                    // outputs have to be pre-nullify                 
                    if (!bSameOrderAndEws1) {
                        auto outShape = reinterpret_cast<Nd4jLong*>(outShapes[nMaxIndex]);
                        zOffset = shape::getOffset(outShape, coords);
                    }

                    auto output = reinterpret_cast<T*>(outArrs[nMaxIndex]);

                    output[zOffset] = grad[gradOffset];
                }
            }

            template<typename T>
            static void mergeMaxBp_(sd::LaunchContext* context, const std::vector<const NDArray*>& inArrs, std::vector<NDArray*>& outArrs, int nArrSize, bool bSameOrderAndEws1) {

                std::vector<const void*> inBuffers(nArrSize), inShapes(nArrSize), outBuffers(nArrSize), outShapes(nArrSize);

                for (int e = 0; e < nArrSize; e++) {
                    inBuffers[e] = inArrs[e]->specialBuffer();
                    inShapes[e] = inArrs[e]->specialShapeInfo();
                    outBuffers[e] = outArrs[e]->specialBuffer();
                    outShapes[e] = outArrs[e]->specialShapeInfo();
                }

                PointersManager manager(context, "mergeMaxBp");

                auto pInBuffers = reinterpret_cast<void**>(manager.replicatePointer(inBuffers.data(), inBuffers.size() * sizeof(void*)));
                auto pInShapes = reinterpret_cast<void**>(manager.replicatePointer(inShapes.data(), inShapes.size() * sizeof(void*)));

                auto pOutBuffers = reinterpret_cast<void**>(manager.replicatePointer(outBuffers.data(), outBuffers.size() * sizeof(void*)));
                auto pOutShapes = reinterpret_cast<void**>(manager.replicatePointer(outShapes.data(), outShapes.size() * sizeof(void*)));

                auto length = inArrs[nArrSize]->lengthOf();

                const int threadsPerBlock = MAX_NUM_THREADS / 2;
                const int blocksPerGrid = (length + threadsPerBlock - 1) / threadsPerBlock;

                mergeMaxBpCudaLauncher<T><<<blocksPerGrid, threadsPerBlock, 512, *context->getCudaStream()>>>(pInBuffers, pInShapes, inArrs[nArrSize]->specialBuffer(),
                    inArrs[nArrSize]->specialShapeInfo(), nArrSize, pOutBuffers, pOutShapes,
                    length, bSameOrderAndEws1);
                
                manager.synchronize();
            }

            void mergeMaxBp(sd::LaunchContext* context, const std::vector<const NDArray*>& inArrs, std::vector<NDArray*>& outArrs) {

                // not use gradient
                int nArrSize = static_cast<int>(inArrs.size() - 1);
                
                const std::vector<const NDArray*>& out = reinterpret_cast<const std::vector<const NDArray*>&>(outArrs);

                NDArray::prepareSpecialUse(out, inArrs);

                bool bSameOrderAndEws1 = (1 == inArrs[nArrSize]->ews());
                auto ordering = inArrs[nArrSize]->ordering();
                  
                for (int i = 0; i < nArrSize; ++i) {
                    bSameOrderAndEws1 &= (ordering == inArrs[i]->ordering());
                    bSameOrderAndEws1 &= (1 == inArrs[i]->ews());
                   
                    bSameOrderAndEws1 &= (ordering == outArrs[i]->ordering());
                    bSameOrderAndEws1 &= (1 == outArrs[i]->ews());
                }

                BUILD_SINGLE_SELECTOR(inArrs[nArrSize]->dataType(), mergeMaxBp_, (context, inArrs, outArrs, nArrSize, bSameOrderAndEws1), LIBND4J_TYPES);

                NDArray::registerSpecialUse( out, inArrs );
            }


            //////////////////////////////////////////////////////////////////////////
            template <typename T>
            static __global__ void mergeAvgCudaLauncher(void** inArrs, void** inShapes, const int numArrays, void* voutput, const Nd4jLong* outputShape, Nd4jLong length) {
                auto output = reinterpret_cast<T*>(voutput);

                const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
                const auto step = gridDim.x * blockDim.x;

                for (Nd4jLong e = tid; e < length; e += step) {
                    T sum(0.0f);

                    for (int i = 0; i < numArrays; i++) {
                        auto x = reinterpret_cast<T*>(inArrs[i]);
                        auto xShape = reinterpret_cast<Nd4jLong*>(inShapes[i]);

                        sum += x[shape::getIndexOffset(e, xShape)];
                    }

                    output[shape::getIndexOffset(e, outputShape)] = sum / numArrays;
                }
            }

            template<typename T>
            static void mergeAvg_(sd::LaunchContext* context, const std::vector<const NDArray*>& inArrs, NDArray& output) {
                
                std::vector<const void*> inBuffers(inArrs.size()), inShapes(inArrs.size());

                for (int e = 0; e < inArrs.size(); e++) {
                    inBuffers[e] = inArrs[e]->specialBuffer();
                    inShapes[e] = inArrs[e]->specialShapeInfo();
                }

                PointersManager manager(context, "mergeAvg");

                auto pInBuffers = reinterpret_cast<void**>(manager.replicatePointer(inBuffers.data(), inBuffers.size() * sizeof(void*)));
                auto pInShapes = reinterpret_cast<void**>(manager.replicatePointer(inShapes.data(), inShapes.size() * sizeof(void*)));
                auto length = output.lengthOf();

                const int threadsPerBlock = MAX_NUM_THREADS / 2;
                const int blocksPerGrid = (length + threadsPerBlock - 1) / threadsPerBlock;

                mergeAvgCudaLauncher<T><<<blocksPerGrid, threadsPerBlock, 512, *context->getCudaStream()>>>(pInBuffers, pInShapes, (int)inArrs.size(), output.specialBuffer(), output.specialShapeInfo(), length);

                manager.synchronize();
            }

            void mergeAvg(sd::LaunchContext* context, const std::vector<const NDArray*>& inArrs, NDArray& output) {
                
                NDArray::prepareSpecialUse({ &output }, inArrs);

                BUILD_SINGLE_SELECTOR(output.dataType(), mergeAvg_, (context, inArrs, output), FLOAT_TYPES);

                NDArray::registerSpecialUse({ &output }, inArrs);
            }
            //////////////////////////////////////////////////////////////////////////
            template <typename T>
            static __global__ void mergeAvgBpCudaLauncher(
                    const void* vgradient, const Nd4jLong* gradientShape,
                    void** outArrs, void** outShapes,
                    const int numArrays,
                    Nd4jLong length,
                    bool bSameOrderAndEws1) {

                auto grad = reinterpret_cast<const T*>(vgradient);

                const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
                const auto step = gridDim.x * blockDim.x;

                int coords[MAX_RANK];

                for (Nd4jLong e = tid; e < length; e += step) {

                    auto zOffset = e, gradOffset = e;
                    if (!bSameOrderAndEws1) {
                        shape::index2coords(e, gradientShape, coords);
                        gradOffset = shape::getOffset(gradientShape, coords);
                    }

                    for (int i = 0; i < numArrays; i++) {

                        if (!bSameOrderAndEws1) {
                            auto outShape = reinterpret_cast<Nd4jLong*>(outShapes[i]);
                            zOffset = shape::getOffset(outShape, coords);
                        }

                        auto output = reinterpret_cast<T*>(outArrs[i]);

                        output[zOffset] = grad[gradOffset] / numArrays;
                    }
                }
            }

            template<typename T>
            static void mergeAvgBp_(sd::LaunchContext* context, const NDArray& gradient, std::vector<NDArray*>& outArrs, bool bSameOrderAndEws1) {

                int nArrSize = static_cast<int>(outArrs.size());

                std::vector<const void*> outBuffers(nArrSize), outShapes(nArrSize);

                for (int e = 0; e < nArrSize; e++) {
                    outBuffers[e] = outArrs[e]->specialBuffer();
                    outShapes[e] = outArrs[e]->specialShapeInfo();
                }

                PointersManager manager(context, "mergeAvgBp");

                auto pOutBuffers = reinterpret_cast<void**>(manager.replicatePointer(outBuffers.data(), outBuffers.size() * sizeof(void*)));
                auto pOutShapes = reinterpret_cast<void**>(manager.replicatePointer(outShapes.data(), outShapes.size() * sizeof(void*)));

                auto length = gradient.lengthOf();
                
                const int threadsPerBlock = MAX_NUM_THREADS / 2;
                const int blocksPerGrid = (length + threadsPerBlock - 1) / threadsPerBlock;

                mergeAvgBpCudaLauncher<T><<<blocksPerGrid, threadsPerBlock, 512, *context->getCudaStream()>>>(gradient.specialBuffer(), gradient.specialShapeInfo(),
                    pOutBuffers, pOutShapes, nArrSize, length, bSameOrderAndEws1);

                manager.synchronize();
            }

            void mergeAvgBp(sd::LaunchContext* context, const NDArray& gradient, std::vector<NDArray*>& outArrs) {

                const std::vector<const NDArray*>& out = reinterpret_cast<const std::vector<const NDArray*>&>(outArrs);

                NDArray::prepareSpecialUse( out, { &gradient });

                bool bSameOrderAndEws1 = (1 == gradient.ews());
                auto ordering = gradient.ordering();

                for (const auto& v : outArrs) {
                    bSameOrderAndEws1 &= (ordering == v->ordering());
                    bSameOrderAndEws1 &= (1 == v->ews());
                }

                BUILD_SINGLE_SELECTOR(gradient.dataType(), mergeAvgBp_, (context, gradient, outArrs, bSameOrderAndEws1), LIBND4J_TYPES);

                NDArray::prepareSpecialUse(out, { &gradient });
            }

            //////////////////////////////////////////////////////////////////////////
            template <typename T>
            static __global__ void mergeAddCudaLauncher(void** inArrs, void** inShapes, const int numArrays, void* voutput, const Nd4jLong* outputShape, Nd4jLong length) {
                
                auto output = reinterpret_cast<T*>(voutput);

                const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
                const auto step = gridDim.x * blockDim.x;

                for (Nd4jLong e = tid; e < length; e += step) {
                    T sum(0.0f);

                    for (int i = 0; i < numArrays; i++) {
                        auto x = reinterpret_cast<T*>(inArrs[i]);
                        auto xShape = reinterpret_cast<Nd4jLong*>(inShapes[i]);

                        sum += x[shape::getIndexOffset(e, xShape)];
                    }

                    output[shape::getIndexOffset(e, outputShape)] = sum;
                }
            }

            template<typename T>
            static void mergeAdd_(sd::LaunchContext* context, const std::vector<const NDArray*>& inArrs, NDArray& output) {
                
                int nArrSize = static_cast<int>(inArrs.size());
                std::vector<const void*> inBuffers(nArrSize), inShapes(nArrSize);

                for (int e = 0; e < nArrSize; e++) {
                    inBuffers[e] = inArrs[e]->specialBuffer();
                    inShapes[e] = inArrs[e]->specialShapeInfo();
                }

                PointersManager manager(context, "mergeAdd");

                auto pInBuffers = reinterpret_cast<void**>(manager.replicatePointer(inBuffers.data(), inBuffers.size() * sizeof(void*)));
                auto pInShapes = reinterpret_cast<void**>(manager.replicatePointer(inShapes.data(), inShapes.size() * sizeof(void*)));
                auto length = output.lengthOf();

                const int threadsPerBlock = MAX_NUM_THREADS / 2;
                const int blocksPerGrid = (length + threadsPerBlock - 1) / threadsPerBlock;

                mergeAddCudaLauncher<T><<<blocksPerGrid, threadsPerBlock, 512, *context->getCudaStream()>>>(pInBuffers, pInShapes, nArrSize, output.specialBuffer(), output.specialShapeInfo(), length);

                manager.synchronize();
            }
            BUILD_SINGLE_TEMPLATE(template void mergeAdd_, (sd::LaunchContext* context, const std::vector<const NDArray*>& inArrs, NDArray& output), NUMERIC_TYPES);

            void mergeAdd(sd::LaunchContext* context, const std::vector<const NDArray*>& inArrs, NDArray& output) {
                
                NDArray::prepareSpecialUse({ &output }, inArrs);
                
                BUILD_SINGLE_SELECTOR(output.dataType(), mergeAdd_, (context, inArrs, output), NUMERIC_TYPES);

                NDArray::registerSpecialUse({ &output }, inArrs);
            }

            //////////////////////////////////////////////////////////////////////////
            template <typename T>
            static __global__ void mergeAddBpCudaLauncher(const void* vgradient, const Nd4jLong* gradientShape, void** outArrs, void** outShapes,
                const int numArrays, Nd4jLong length, bool bSameOrderAndEws1) {

                auto grad = reinterpret_cast<const T*>(vgradient);

                const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
                const auto step = gridDim.x * blockDim.x;

                int coords[MAX_RANK];

                for (Nd4jLong e = tid; e < length; e += step) {

                    auto zOffset = e, gradOffset = e;
                    if (!bSameOrderAndEws1) {
                        shape::index2coords(e, gradientShape, coords);
                        gradOffset = shape::getOffset(gradientShape, coords);
                    }

                    for (int i = 0; i < numArrays; i++) {
                        
                        if (!bSameOrderAndEws1) {
                            auto outShape = reinterpret_cast<Nd4jLong*>(outShapes[i]);
                            zOffset = shape::getOffset(outShape, coords);
                        }

                        auto output = reinterpret_cast<T*>(outArrs[i]);

                        output[zOffset] = grad[gradOffset];
                    }
                }
            }

            template<typename T>
            static void mergeAddBp_(sd::LaunchContext* context, const NDArray& gradient, std::vector<NDArray*>& outArrs, bool bSameOrderAndEws1) {

                int nArrSize = static_cast<int>(outArrs.size());

                std::vector<const void*> outBuffers(nArrSize), outShapes(nArrSize);

                for (int e = 0; e < nArrSize; e++) {
                    outBuffers[e] = outArrs[e]->specialBuffer();
                    outShapes[e] = outArrs[e]->specialShapeInfo();
                }

                PointersManager manager(context, "mergeAddBp");

                auto pOutBuffers = reinterpret_cast<void**>(manager.replicatePointer(outBuffers.data(), outBuffers.size() * sizeof(void*)));
                auto pOutShapes = reinterpret_cast<void**>(manager.replicatePointer(outShapes.data(), outShapes.size() * sizeof(void*)));

                auto length = gradient.lengthOf();

                const int threadsPerBlock = MAX_NUM_THREADS / 2;
                const int blocksPerGrid = (length + threadsPerBlock - 1) / threadsPerBlock;

                mergeAddBpCudaLauncher<T><<<blocksPerGrid, threadsPerBlock, 512, *context->getCudaStream()>>>(gradient.specialBuffer(), gradient.specialShapeInfo(),
                    pOutBuffers, pOutShapes, nArrSize, length, bSameOrderAndEws1);

                manager.synchronize();
            }

            void mergeAddBp(sd::LaunchContext* context, const NDArray& gradient, std::vector<NDArray*>& outArrs) {

                const std::vector<const NDArray*>& out = reinterpret_cast<const std::vector<const NDArray*>& >(outArrs);
                NDArray::prepareSpecialUse( out, { &gradient });

                bool bSameOrderAndEws1 = (1 == gradient.ews());
                auto ordering = gradient.ordering();

                for (const auto& v : outArrs) {
                    bSameOrderAndEws1 &= (ordering == v->ordering());
                    bSameOrderAndEws1 &= (1 == v->ews());
                }

                BUILD_SINGLE_SELECTOR(gradient.dataType(), mergeAddBp_, (context, gradient, outArrs, bSameOrderAndEws1), LIBND4J_TYPES);

                NDArray::prepareSpecialUse( out, { &gradient });
            }

        }
    }
}
