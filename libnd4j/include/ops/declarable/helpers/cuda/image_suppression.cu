#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author sgazeos@gmail.com
//

#include <ops/declarable/helpers/image_suppression.h>
#include <array/NDArrayFactory.h>
#include <legacy/NativeOps.h>
#include <exceptions/cuda_exception.h>
#include <queue>

namespace sd {
namespace ops {
namespace helpers {
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// needToSuppressWithThreshold - predicate for suppression
//      boxes - boxes tensor buffer
//      boxesShape boxes tensor shape
//      previousIndex - index for current pos value
//      nextIndex - index for neighbor pos value
//      threshold - threashold value to suppress
//
//      return value: true, if threshold is overcome, false otherwise
//
    template <typename T>
    static __device__ bool needToSuppressWithThreshold(T* boxes, Nd4jLong const* boxesShape, int previousIndex, int nextIndex, T threshold) {
        Nd4jLong previous0[] = {previousIndex, 0};
        Nd4jLong previous1[] = {previousIndex, 1};
        Nd4jLong previous2[] = {previousIndex, 2};
        Nd4jLong previous3[] = {previousIndex, 3};
        Nd4jLong next0[] = {nextIndex, 0};
        Nd4jLong next1[] = {nextIndex, 1};
        Nd4jLong next2[] = {nextIndex, 2};
        Nd4jLong next3[] = {nextIndex, 3};

        // we have rectangle with given max values. Compute vexes of rectangle first

        T minYPrev = sd::math::nd4j_min(boxes[shape::getOffset(boxesShape, previous0)], boxes[shape::getOffset(boxesShape, previous2)]);
        T minXPrev = sd::math::nd4j_min(boxes[shape::getOffset(boxesShape, previous1)], boxes[shape::getOffset(boxesShape, previous3)]);
        T maxYPrev = sd::math::nd4j_max(boxes[shape::getOffset(boxesShape, previous0)], boxes[shape::getOffset(boxesShape, previous2)]);
        T maxXPrev = sd::math::nd4j_max(boxes[shape::getOffset(boxesShape, previous1)], boxes[shape::getOffset(boxesShape, previous3)]);
        T minYNext = sd::math::nd4j_min(boxes[shape::getOffset(boxesShape, next0)],     boxes[shape::getOffset(boxesShape, next2)]);
        T minXNext = sd::math::nd4j_min(boxes[shape::getOffset(boxesShape, next1)],     boxes[shape::getOffset(boxesShape, next3)]);
        T maxYNext = sd::math::nd4j_max(boxes[shape::getOffset(boxesShape, next0)],     boxes[shape::getOffset(boxesShape, next2)]);
        T maxXNext = sd::math::nd4j_max(boxes[shape::getOffset(boxesShape, next1)],     boxes[shape::getOffset(boxesShape, next3)]);

        // compute areas for comparation
        T areaPrev = (maxYPrev - minYPrev) * (maxXPrev - minXPrev);
        T areaNext = (maxYNext - minYNext) * (maxXNext - minXNext);

        // of course, areas should be positive
        if (areaNext <= T(0.f) || areaPrev <= T(0.f)) return false;

        // compute intersection of rectangles
        T minIntersectionY = sd::math::nd4j_max(minYPrev, minYNext);
        T minIntersectionX = sd::math::nd4j_max(minXPrev, minXNext);
        T maxIntersectionY = sd::math::nd4j_min(maxYPrev, maxYNext);
        T maxIntersectionX = sd::math::nd4j_min(maxXPrev, maxXNext);
        T intersectionArea =
                sd::math::nd4j_max(T(maxIntersectionY - minIntersectionY), T(0.0f)) *
                sd::math::nd4j_max(T(maxIntersectionX - minIntersectionX), T(0.0f));
        T intersectionValue = intersectionArea / (areaPrev + areaNext - intersectionArea);
        // final check
        return intersectionValue > threshold;
    }

    template <typename T>
    static __device__ T similirityV3(T* boxes, Nd4jLong const* boxesShape, int previousIndex, int nextIndex) {
        Nd4jLong previous0[] = {previousIndex, 0};
        Nd4jLong previous1[] = {previousIndex, 1};
        Nd4jLong previous2[] = {previousIndex, 2};
        Nd4jLong previous3[] = {previousIndex, 3};
        Nd4jLong next0[] = {nextIndex, 0};
        Nd4jLong next1[] = {nextIndex, 1};
        Nd4jLong next2[] = {nextIndex, 2};
        Nd4jLong next3[] = {nextIndex, 3};

        // we have rectangle with given max values. Compute vexes of rectangle first

        T minYPrev = sd::math::nd4j_min(boxes[shape::getOffset(boxesShape, previous0)], boxes[shape::getOffset(boxesShape, previous2)]);
        T minXPrev = sd::math::nd4j_min(boxes[shape::getOffset(boxesShape, previous1)], boxes[shape::getOffset(boxesShape, previous3)]);
        T maxYPrev = sd::math::nd4j_max(boxes[shape::getOffset(boxesShape, previous0)], boxes[shape::getOffset(boxesShape, previous2)]);
        T maxXPrev = sd::math::nd4j_max(boxes[shape::getOffset(boxesShape, previous1)], boxes[shape::getOffset(boxesShape, previous3)]);
        T minYNext = sd::math::nd4j_min(boxes[shape::getOffset(boxesShape, next0)],     boxes[shape::getOffset(boxesShape, next2)]);
        T minXNext = sd::math::nd4j_min(boxes[shape::getOffset(boxesShape, next1)],     boxes[shape::getOffset(boxesShape, next3)]);
        T maxYNext = sd::math::nd4j_max(boxes[shape::getOffset(boxesShape, next0)],     boxes[shape::getOffset(boxesShape, next2)]);
        T maxXNext = sd::math::nd4j_max(boxes[shape::getOffset(boxesShape, next1)],     boxes[shape::getOffset(boxesShape, next3)]);

        // compute areas for comparation
        T areaPrev = (maxYPrev - minYPrev) * (maxXPrev - minXPrev);
        T areaNext = (maxYNext - minYNext) * (maxXNext - minXNext);

        // of course, areas should be positive
        if (areaNext <= T(0.f) || areaPrev <= T(0.f)) return false;

        // compute intersection of rectangles
        T minIntersectionY = sd::math::nd4j_max(minYPrev, minYNext);
        T minIntersectionX = sd::math::nd4j_max(minXPrev, minXNext);
        T maxIntersectionY = sd::math::nd4j_min(maxYPrev, maxYNext);
        T maxIntersectionX = sd::math::nd4j_min(maxXPrev, maxXNext);
        T intersectionArea =
                sd::math::nd4j_max(T(maxIntersectionY - minIntersectionY), T(0.0f)) *
                sd::math::nd4j_max(T(maxIntersectionX - minIntersectionX), T(0.0f));
        T intersectionValue = intersectionArea / (areaPrev + areaNext - intersectionArea);
        // final check
        return intersectionValue;
    }

    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// shouldSelectKernel - compute status for all selected rectangles (boxes)
//
// we compute boolean flag as shared uint32 and return it on final only for the first thread
//
    template <typename T, typename I>
    static __global__ void shouldSelectKernel(T* boxesBuf, Nd4jLong const* boxesShape, I* indexBuf, I* selectedIndicesData, double threshold, int numSelected, int i, bool* shouldSelect) {
        auto tid = blockIdx.x * blockDim.x + threadIdx.x;
        auto step = gridDim.x * blockDim.x;
        __shared__ unsigned int shouldSelectShared;
        if (threadIdx.x == 0) {
            shouldSelectShared = (unsigned int)shouldSelect[0];
        }
        __syncthreads();
        for (int j = numSelected - 1 - tid; j >= 0; j -= step) {
            if (shouldSelectShared) {
                if (needToSuppressWithThreshold(boxesBuf, boxesShape, indexBuf[i],
                                                                  indexBuf[selectedIndicesData[j]], T(threshold)))
                    atomicCAS(&shouldSelectShared, 1, 0); // exchange only when need to suppress
            }
        }
        __syncthreads();

        // final move: collect result
        if (threadIdx.x == 0) {
            *shouldSelect = shouldSelectShared > 0;
        }
    }

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// indices - type depended, indicesLong - type defined (only 64bit integers)
//
    template <typename I>
    static __global__ void copyIndices(void* indices,  void* indicesLong, Nd4jLong len) {
        I* indexBuf = reinterpret_cast<I*>(indices);
        Nd4jLong* srcBuf = reinterpret_cast<Nd4jLong*>(indicesLong);;

        auto tid = threadIdx.x + blockIdx.x * blockDim.x;
        auto step = blockDim.x * gridDim.x;

        for (auto i = tid; i < len; i += step)
            indexBuf[i] = (I)srcBuf[i];
    }

    template <typename T, typename I>
    static __global__ void suppressScores(T* scores, I* indices, Nd4jLong length, T scoreThreshold) {
        auto start = blockIdx.x * blockDim.x;
        auto step = gridDim.x * blockDim.x;

        for (auto e = start + threadIdx.x; e < (int)length; e += step) {
            if (scores[e] < scoreThreshold) {
                scores[e] = scoreThreshold;
                indices[e] = -1;
            }
            else {
                indices[e] = I(e);
            }
        }
    }

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// nonMaxSuppressionV2 algorithm - given from TF NonMaxSuppressionV2 implementation
//
    template <typename T, typename I>
    static void nonMaxSuppressionV2_(sd::LaunchContext* context, NDArray* boxes, NDArray* scales, int maxSize, double threshold, double scoreThreshold, NDArray* output) {
        auto stream = context->getCudaStream();
        NDArray::prepareSpecialUse({output}, {boxes, scales});
        std::unique_ptr<NDArray> indices(NDArrayFactory::create_<I>('c', {scales->lengthOf()}, context)); // - 1, scales->lengthOf()); //, scales->getContext());

        NDArray scores(*scales);
        Nd4jPointer extras[2] = {nullptr, stream};
        auto indexBuf = indices->dataBuffer()->specialAsT<I>();///reinterpret_cast<I*>(indices->specialBuffer());
        auto scoreBuf = scores.dataBuffer()->specialAsT<T>();
        suppressScores<T,I><<<128, 128, 128, *stream>>>(scoreBuf, indexBuf, scores.lengthOf(), T(scoreThreshold));
        indices->tickWriteDevice();
        sortByValue(extras, indices->buffer(), indices->shapeInfo(), indices->specialBuffer(), indices->specialShapeInfo(), scores.buffer(), scores.shapeInfo(), scores.specialBuffer(), scores.specialShapeInfo(), true);
        indices->tickWriteDevice();
        NDArray selectedIndices = NDArrayFactory::create<I>('c', {output->lengthOf()}, context);
        int numSelected = 0;
        int numBoxes = boxes->sizeAt(0);
        auto boxesBuf = reinterpret_cast<T*>(boxes->specialBuffer());

        auto selectedIndicesData = reinterpret_cast<I*>(selectedIndices.specialBuffer());
        auto outputBuf = reinterpret_cast<I*>(output->specialBuffer());

        bool* shouldSelectD;
        auto err = hipMalloc(&shouldSelectD, sizeof(bool));
        if (err) {
            throw cuda_exception::build("helpers::nonMaxSuppressionV2: Cannot allocate memory for bool flag", err);
        }
        for (I i = 0; i < boxes->sizeAt(0); ++i) {
            bool shouldSelect = numSelected < output->lengthOf();
            if (shouldSelect) {
                err = hipMemcpy(shouldSelectD, &shouldSelect, sizeof(bool), hipMemcpyHostToDevice);
                if (err) {
                    throw cuda_exception::build("helpers::nonMaxSuppressionV2: Cannot set up bool flag to device", err);
                }

                shouldSelectKernel<T,I><<<128, 256, 1024, *stream>>>(boxesBuf, boxes->specialShapeInfo(), indexBuf, selectedIndicesData, threshold, numSelected, i, shouldSelectD);
                err = hipMemcpy(&shouldSelect, shouldSelectD, sizeof(bool), hipMemcpyDeviceToHost);
                if (err) {
                    throw cuda_exception::build("helpers::nonMaxSuppressionV2: Cannot set up bool flag to host", err);
                }
            }

            if (shouldSelect) {
                hipMemcpy(reinterpret_cast<I*>(output->specialBuffer()) + numSelected, indexBuf + i, sizeof(I), hipMemcpyDeviceToDevice);
                hipMemcpy(selectedIndicesData + numSelected, &i, sizeof(I), hipMemcpyHostToDevice);
                numSelected++;
            }
        }

        err = hipFree(shouldSelectD);
        if (err) {
            throw cuda_exception::build("helpers::nonMaxSuppressionV2: Cannot deallocate memory for bool flag", err);
        }

    }

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    template <typename T, typename I>
    static __device__ bool checkOverlapBoxes(T* boxes, Nd4jLong const* shape, T* scores, I* indices, I* selectedIndices, I* startIndices, I selectedSize, I nextCandidateIndex, T overlapThreshold, T scoreThreshold, bool simple) {
        bool shouldHardSuppress = false;
        T& nextCandidateScore = scores[nextCandidateIndex];
        I selectedIndex = indices[nextCandidateIndex];
        I finish = startIndices[nextCandidateIndex];

        for (int j = selectedSize; j > finish; --j) {
            T boxVal;
            if (simple) {
                Nd4jLong xPos[] = {selectedIndex, selectedIndices[j - 1]};
                auto xShift = shape::getOffset(shape, xPos, 0);
                boxVal = boxes[xShift];
            }
            else {
                boxVal = similirityV3(boxes, shape, selectedIndex, selectedIndices[j - 1]);
            }
            if (boxVal > static_cast<T>(overlapThreshold))
                nextCandidateScore = static_cast<T>(0.f);

            // First decide whether to perform hard suppression
            if (boxVal >= overlapThreshold) {
                shouldHardSuppress = true;
                break;
            }

            // If nextCandidate survives hard suppression, apply soft suppression
            if (nextCandidateScore <= static_cast<T>(scoreThreshold)) break;
        }

        return shouldHardSuppress;
    }
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    template <typename T, typename I>
    static __global__ void
    suppressNonMaxOverlapKernel(T* boxes, Nd4jLong const* boxesShape, T* scoresData, I* indices, I* startIndices, Nd4jLong length, I maxOutputLen,
    T overlapThreshold, T scoreThreshold, I* output, Nd4jLong const* outputShape, I* outputLength, bool simple) {

        __shared__ I selectedSize;
        __shared__ I* tempOutput;

        if (threadIdx.x == 0) {
            selectedSize = outputLength?*outputLength:maxOutputLen;
            extern __shared__ unsigned char shmem[];
            tempOutput = (I*)shmem;
        }
        __syncthreads();

        auto start = blockIdx.x * blockDim.x;
        auto step = blockDim.x * gridDim.x;

        for (I nextCandidateIndex = start + threadIdx.x; selectedSize < maxOutputLen && nextCandidateIndex < (I)length; ) {
            auto originalScore = scoresData[nextCandidateIndex];//nextCandidate._score;
            I nextCandidateBoxIndex = indices[nextCandidateIndex];
            auto selectedSizeMark = selectedSize;

            // skip for cases when index is less than 0 (under score threshold)
            if (nextCandidateBoxIndex < 0) {
                nextCandidateIndex += step;
                continue;
            }
            // check for overlaps
            bool shouldHardSuppress = checkOverlapBoxes(boxes, boxesShape, scoresData, indices, tempOutput, startIndices, selectedSize,
                    nextCandidateIndex, overlapThreshold, scoreThreshold, simple);//false;
            T nextCandidateScore = scoresData[nextCandidateIndex];

            startIndices[nextCandidateIndex] = selectedSize;
            if (!shouldHardSuppress) {
                if (nextCandidateScore == originalScore) {
                    // Suppression has not occurred, so select nextCandidate
                    if (output)
                        output[selectedSize] = nextCandidateBoxIndex;
                    tempOutput[selectedSize] = nextCandidateBoxIndex;
                    math::atomics::nd4j_atomicAdd(&selectedSize, (I)1);
                }

                if (nextCandidateScore > scoreThreshold) {
                    // Soft suppression has occurred and current score is still greater than
                    // scoreThreshold; add nextCandidate back onto priority queue.
                    continue;  // in some cases, this index not 0
                }
            }
            nextCandidateIndex += step;
        }

        if (threadIdx.x == 0) {
            if (outputLength)
                *outputLength = selectedSize;
        }
    }

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    template <typename T, typename I>
    static Nd4jLong
    nonMaxSuppressionGeneric_(sd::LaunchContext* context, NDArray* boxes, NDArray* scores, int outputSize,
                              double overlapThreshold, double scoreThreshold, NDArray* output, bool simple) {
        auto stream = context->getCudaStream();
        if (output)
            NDArray::prepareSpecialUse({output}, {boxes, scores});
        else {
            if (!boxes->isActualOnDeviceSide())
                boxes->syncToDevice();
            if (!scores->isActualOnDeviceSide())
                scores->syncToDevice();
        }

        NDArray indices = NDArrayFactory::create<I>('c', {scores->lengthOf()}, context); // - 1, scales->lengthOf()); //, scales->getContext());
        NDArray startPositions = NDArrayFactory::create<I>('c', {scores->lengthOf()}, context);
        NDArray selectedScores(*scores);
        Nd4jPointer extras[2] = {nullptr, stream};
        auto indexBuf = indices.dataBuffer()->specialAsT<I>();///reinterpret_cast<I*>(indices->specialBuffer());

        suppressScores<<<128, 128, 128, *stream>>>(selectedScores.dataBuffer()->specialAsT<T>(), indexBuf, selectedScores.lengthOf(), T(scoreThreshold));

        sortByValue(extras, indices.buffer(), indices.shapeInfo(), indices.specialBuffer(), indices.specialShapeInfo(), selectedScores.buffer(), selectedScores.shapeInfo(), selectedScores.specialBuffer(), selectedScores.specialShapeInfo(), true);
        indices.tickWriteDevice();
        selectedScores.tickWriteDevice();

        auto scoresData = selectedScores.dataBuffer()->specialAsT<T>();//, numBoxes, scoresData.begin());

        auto startIndices = startPositions.dataBuffer()->specialAsT<I>();
        I selectedSize = 0;
        Nd4jLong res = 0;
        if (output) { // this part used when output shape already calculated to fill up values on output
            DataBuffer selectedSizeBuf(&selectedSize, sizeof(I), DataTypeUtils::fromT<I>());
            suppressNonMaxOverlapKernel<<<1, 1, 1024, *stream >>> (boxes->dataBuffer()->specialAsT<T>(),
                    boxes->specialShapeInfo(), scoresData, indexBuf, startIndices, scores->lengthOf(), (I) outputSize,
                    T(overlapThreshold), T(scoreThreshold), output->dataBuffer()->specialAsT<I>(), output->specialShapeInfo(),
                    selectedSizeBuf.specialAsT<I>(), simple);
        }
        else { // this case used on calculation of output shape. Output and output shape shoulde be nullptr.
            DataBuffer selectedSizeBuf(&selectedSize, sizeof(I), DataTypeUtils::fromT<I>());
            suppressNonMaxOverlapKernel<<<1, 1, 1024, *stream >>> (boxes->dataBuffer()->specialAsT<T>(),
                    boxes->specialShapeInfo(), scoresData, indexBuf, startIndices, scores->lengthOf(), (I)outputSize,
                    T(overlapThreshold), T(scoreThreshold), (I*)nullptr, (Nd4jLong*) nullptr, selectedSizeBuf.specialAsT<I>(), simple);
            selectedSizeBuf.syncToPrimary(context, true);
            res = *selectedSizeBuf.primaryAsT<I>();
        }

        if (output)
            NDArray::registerSpecialUse({output}, {boxes, scores});

        return res;
    }
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    void nonMaxSuppression(sd::LaunchContext * context, NDArray* boxes, NDArray* scales, int maxSize, double threshold, double scoreThreshold, NDArray* output) {
        BUILD_DOUBLE_SELECTOR(boxes->dataType(), output->dataType(), nonMaxSuppressionV2_,
                (context, boxes, scales, maxSize, threshold, scoreThreshold, output),
                FLOAT_TYPES, INDEXING_TYPES);
    }
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

    Nd4jLong nonMaxSuppressionGeneric(sd::LaunchContext * context, NDArray* boxes, NDArray* scales, int maxSize, double threshold, double scoreThreshold, NDArray* output) {
        BUILD_DOUBLE_SELECTOR(boxes->dataType(), output ? output->dataType():DataType::INT32, return nonMaxSuppressionGeneric_,
                              (context, boxes, scales, maxSize, threshold, scoreThreshold, output, true),
                              FLOAT_TYPES, INDEXING_TYPES);
        return boxes->sizeAt(0);
    }

    Nd4jLong
    nonMaxSuppressionV3(sd::LaunchContext* context, NDArray* boxes, NDArray* scores, int maxSize,
                             double overlapThreshold, double scoreThreshold, NDArray* output) {
        BUILD_DOUBLE_SELECTOR(boxes->dataType(), output ? output->dataType():DataType::INT32, return nonMaxSuppressionGeneric_,
                              (context, boxes, scores, maxSize, overlapThreshold, scoreThreshold, output, false),
                              FLOAT_TYPES, INDEXING_TYPES);
        return boxes->sizeAt(0);
    }

}
}
}
