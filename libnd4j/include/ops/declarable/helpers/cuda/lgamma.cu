#include "hip/hip_runtime.h"
/*
 *  ******************************************************************************
 *  *
 *  *
 *  * This program and the accompanying materials are made available under the
 *  * terms of the Apache License, Version 2.0 which is available at
 *  * https://www.apache.org/licenses/LICENSE-2.0.
 *  *
 *  * See the NOTICE file distributed with this work for additional
 *  * information regarding copyright ownership.
 *  * Unless required by applicable law or agreed to in writing, software
 *  * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 *  * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 *  * License for the specific language governing permissions and limitations
 *  * under the License.
 *  *
 *  * SPDX-License-Identifier: Apache-2.0
 *  *****************************************************************************
 */

//
// @author George A. Shulinok <sgazeos@gmail.com>
//

#include<ops/declarable/helpers/lgamma.h>
//#include <execution/Threads.h>
//#include <hip/hip_vector_types.h>

namespace sd {
namespace ops {
namespace helpers {

//////////////////////////////////////////////////////////////////////////
// calculate digamma function for array elements
template <typename T>
void lgamma_(NDArray& x, NDArray& z) {
    //auto dtype = x.dataType();
    auto lgammaProc = LAMBDA_T(x_, dtype) {
        return T(DataTypeUtils::fromT<T>() == DataType::DOUBLE?::lgamma(x_): ::lgammaf(x_)); //math::nd4j_log<T,T>(math::nd4j_gamma<T,T>(x));
    };

    x.applyLambda(lgammaProc, z);
}

void lgamma(sd::LaunchContext* context, NDArray& x, NDArray& z) {

	BUILD_SINGLE_SELECTOR(x.dataType(), lgamma_, (x, z), FLOAT_TYPES);
}

BUILD_SINGLE_TEMPLATE(template void lgamma_, (NDArray& x, NDArray& z), FLOAT_TYPES);



}
}
}

