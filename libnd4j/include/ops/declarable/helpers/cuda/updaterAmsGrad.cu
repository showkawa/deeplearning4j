#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Oleh Semeniv (oleg.semeniv@gmail.com)
//

#include <system/op_boilerplate.h>
#include <ops/declarable/helpers/updatersHelpers.h>
#include <helpers/PointersManager.h>
#include <math/platformmath.h>
#include <math/templatemath.h>

namespace sd    {
namespace ops     {
namespace helpers {

///////////////////////////////////////////////////////////////////
template<typename T>
__global__ void amsGradUpdaterCuda(const void* vx, const Nd4jLong* xShapeInfo, const void* vinv, const Nd4jLong* invShapeInfo, 
                                  const void* vinm, const Nd4jLong* inmShapeInfo, const void* vinh, const Nd4jLong* inhShapeInfo,
                                  void* vz, const Nd4jLong* zShapeInfo,  void* vstV, const Nd4jLong* stvShapeInfo, void* vstM, 
                                  const Nd4jLong* stmShapeInfo, void* vstH, const Nd4jLong* sthShapeInfo,
                                  const T lr, const T beta1, const T beta2, const T epsilon, const T iteration) {

    const auto grad = reinterpret_cast<const T*>(vx);
    const auto initV = reinterpret_cast<const T*>(vinv);
    const auto initM = reinterpret_cast<const T*>(vinm);
    const auto initH = reinterpret_cast<const T*>(vinh);
    
    auto up = reinterpret_cast<T*>(vz);
    auto stV = reinterpret_cast<T*>(vstV);
    auto stM = reinterpret_cast<T*>(vstM);
    auto stH = reinterpret_cast<T*>(vstH);

    __shared__ Nd4jLong xLen;
    __shared__ T mbeta1, mbeta2, epsilonT;
    __shared__ bool bEWS, bOrdering, bXZsame, bXInUSame, bXStUSame, bXInMSame, bXStMSame, bXInHSame, bXStHSame;

    if (threadIdx.x == 0) {
        xLen = shape::length(xShapeInfo);
        
        epsilonT = lr * sd::math::nd4j_sqrt<T, T>(1.0 - sd::math::nd4j_pow<T, T, T>(beta2, (iteration + 1))) / (1.0 - sd::math::nd4j_pow<T, T, T>(beta1, (iteration + 1)));

        if (sd::math::nd4j_isnan(epsilonT) || 0 == epsilonT || sd::math::nd4j_isinf(epsilonT))
            epsilonT = epsilon;

        mbeta1 = (1 - beta1);
        mbeta2 = (1 - beta2);

        bEWS = 1 == shape::elementWiseStride(xShapeInfo) && 1 == shape::elementWiseStride(zShapeInfo) &&
               1 == shape::elementWiseStride(stmShapeInfo) && 1 == shape::elementWiseStride(inmShapeInfo) &&
               1 == shape::elementWiseStride(stvShapeInfo) && 1 == shape::elementWiseStride(invShapeInfo) &&
               1 == shape::elementWiseStride(sthShapeInfo) && 1 == shape::elementWiseStride(inhShapeInfo);

        bOrdering = shape::order(xShapeInfo) == shape::order(zShapeInfo) && shape::order(zShapeInfo) == shape::order(stmShapeInfo) &&
                    shape::order(stmShapeInfo) == shape::order(inmShapeInfo) && shape::order(inmShapeInfo) == shape::order(stvShapeInfo) &&
                    shape::order(stvShapeInfo) == shape::order(invShapeInfo) && shape::order(invShapeInfo) == shape::order(sthShapeInfo) &&
                    shape::order(sthShapeInfo) == shape::order(inhShapeInfo);

        bXZsame = shape::haveSameShapeAndStrides(xShapeInfo, zShapeInfo);
        bXInUSame = shape::haveSameShapeAndStrides(xShapeInfo, invShapeInfo);
        bXStUSame = shape::haveSameShapeAndStrides(xShapeInfo, stvShapeInfo);
        bXInMSame = shape::haveSameShapeAndStrides(xShapeInfo, inmShapeInfo);
        bXStMSame = shape::haveSameShapeAndStrides(xShapeInfo, stmShapeInfo);
        bXInHSame = shape::haveSameShapeAndStrides(xShapeInfo, inhShapeInfo);
        bXStHSame = shape::haveSameShapeAndStrides(xShapeInfo, sthShapeInfo);
    }
    __syncthreads();

    int coords[MAX_RANK];

    for (Nd4jLong i = blockIdx.x * blockDim.x + threadIdx.x; i < xLen; i += gridDim.x * blockDim.x) {

        auto xOffset = i, zOffset = i, initMOffset = i, initVOffset = i, initHOffset = i, stMOffset = i, stVOffset = i, stHOffset = i;

        if (!bEWS || !bOrdering){

            shape::index2coords(i, xShapeInfo, coords);
            xOffset  = shape::getOffset(xShapeInfo, coords);
            zOffset  = bXZsame ? xOffset : shape::getOffset(zShapeInfo, coords);
            initMOffset = bXInMSame ? xOffset : shape::getOffset(inmShapeInfo, coords);
            stMOffset = bXStMSame ? xOffset : shape::getOffset(stmShapeInfo, coords);
            initVOffset = bXInUSame ? xOffset : shape::getOffset(invShapeInfo, coords);
            stVOffset = bXStUSame ? xOffset : shape::getOffset(stvShapeInfo, coords);
            initHOffset = bXInHSame ? xOffset : shape::getOffset(inhShapeInfo, coords);
            stHOffset = bXStHSame ? xOffset : shape::getOffset(sthShapeInfo, coords);
        }

        stM[stMOffset] = beta1 * initM[initMOffset] + grad[xOffset] * mbeta1;
        stV[stVOffset] = beta2 * initV[initVOffset] + grad[xOffset] * grad[xOffset] * mbeta2;
        stH[stHOffset] = sd::math::nd4j_max(initH[initHOffset], stV[stVOffset]);

        up[zOffset] = epsilonT * stM[stMOffset] / (sd::math::nd4j_sqrt<T, T>(stH[stHOffset]) + epsilon);
    }
}

///////////////////////////////////////////////////////////////////
template<typename T>
linkage void amsGradUpdaterCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const hipStream_t* stream, const void* vx, const Nd4jLong* xShapeInfo,
    const void* vinv, const Nd4jLong* invShapeInfo, const void* vinm, const Nd4jLong* inmShapeInfo,
    const void* vinh, const Nd4jLong* inhShapeInfo,  void* vz, const Nd4jLong* zShapeInfo,
    void* vstV, const Nd4jLong* stvShapeInfo, void* vstM, const Nd4jLong* stmShapeInfo, 
    void* vstH, const Nd4jLong* sthShapeInfo, const double dLr, const double dBeta1, const double dBeta2, const double dEpsilon, const int nIteration) {
    
    const T lr = static_cast<T>(dLr);
    const T beta1 = static_cast<T>(dBeta1);
    const T beta2 = static_cast<T>(dBeta2);
    const T epsilon = static_cast<T>(dEpsilon);
    const T iteration = static_cast<T>(nIteration);

    amsGradUpdaterCuda<T><<<blocksPerGrid, threadsPerBlock, 256, * stream>>>(vx, xShapeInfo, vinv, invShapeInfo, vinm, inmShapeInfo,
        vinh, inhShapeInfo, vz, zShapeInfo, vstV, stvShapeInfo, vstM, stmShapeInfo, vstH, sthShapeInfo, lr, beta1, beta2, epsilon, iteration);
}

///////////////////////////////////////////////////////////////////
void updaterAmsGrad(sd::LaunchContext* context, const NDArray& gradient, const NDArray& initStateV, const NDArray& initStateM, const NDArray& initStateH, 
     NDArray& update, NDArray& stateV, NDArray& stateM, NDArray& stateH, const double dLr, const double dBeta1, const double dBeta2, const double dEpsilon, const int nIteration) {

    PointersManager manager(context, "amsGradUpdater");

    const int threadsPerBlock = MAX_NUM_THREADS / 4;
    const int blocksPerGrid = (gradient.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;

    NDArray::prepareSpecialUse({ &update, &stateV, &stateM, &stateH }, { &gradient, &initStateV, &initStateM, &initStateH });
    BUILD_SINGLE_SELECTOR(gradient.dataType(), amsGradUpdaterCudaLauncher, (blocksPerGrid, threadsPerBlock, context->getCudaStream(), gradient.specialBuffer(), gradient.specialShapeInfo(),
        initStateV.specialBuffer(), initStateV.specialShapeInfo(), initStateM.specialBuffer(), initStateM.specialShapeInfo(),
        initStateH.specialBuffer(), initStateH.specialShapeInfo(), update.specialBuffer(), update.specialShapeInfo(),
        stateV.specialBuffer(), stateV.specialShapeInfo(), stateM.specialBuffer(), stateM.specialShapeInfo(),
        stateH.specialBuffer(), stateH.specialShapeInfo(), dLr, dBeta1, dBeta2, dEpsilon, nIteration), FLOAT_TYPES);
    NDArray::registerSpecialUse({ &update, &stateV, &stateM , &stateH }, { &gradient, &initStateV, &initStateM, &initStateH });

    manager.synchronize();
}


}
}
}
