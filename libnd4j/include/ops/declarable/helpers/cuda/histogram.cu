#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
//

#include <ops/declarable/helpers/histogram.h>
#include <array/NDArrayFactory.h>

namespace sd {
    namespace ops {
        namespace helpers {
            template <typename X, typename Z>
            void _CUDA_G histogramKernel(void *xBuffer, const Nd4jLong *xShapeInfo, void *zBuffer, const Nd4jLong *zShapeInfo, void *allocationPointer, void *reductionPointer, Nd4jLong numBins, X* min_val, X* max_val) {
                int tid = blockIdx.x * blockDim.x + threadIdx.x;
                auto dx = reinterpret_cast<X*>(xBuffer);
                auto result = reinterpret_cast<Z*>(zBuffer);

                __shared__ Z *bins;
                __shared__ int length;
                __shared__ Z *reductor;
                if (threadIdx.x == 0) {
                    extern __shared__ unsigned char shmem[];
                    bins = (Z *) shmem;
                    reductor = ((Z *) allocationPointer) + (numBins * blockIdx.x);

                    length = shape::length(xShapeInfo);
                }
                __syncthreads();

                X binSize = X((*max_val - *min_val) / numBins);

                // nullify bins
                for (int e = threadIdx.x; e < numBins; e += blockDim.x) {
                    bins[e] = (Z) 0;
                }
                __syncthreads();

                for (int e = tid; e < length; e += blockDim.x * gridDim.x) {
                    int idx = int((dx[e] - *min_val) / binSize);
                    idx = math::nd4j_max(idx, 0); //atomicMax(&idx, 0);//atomicMax(&idx, 0);
                    idx = math::nd4j_min(idx, int(numBins - 1)); //atomicMin(&idx, int(numBins - 1));
                    sd::math::atomics::nd4j_atomicAdd<Z>(&bins[idx], (Z)1);
                }
                __syncthreads();
                // at this point all bins in shared memory are calculated, so we aggregate them now via threadfence trick

                // transfer shared memory to reduction memory
                if (gridDim.x > 1) {
                    unsigned int *tc = (unsigned int *)reductionPointer;
                    __shared__ bool amLast;

                    for (int e = threadIdx.x; e < numBins; e += blockDim.x) {
                        reductor[e] = bins[e];
                    }
                    __threadfence();
                    __syncthreads();

                    if (threadIdx.x == 0) {
                        unsigned int ticket = atomicInc(&tc[16384], gridDim.x);
                        amLast = (ticket == gridDim.x - 1);
                    }
                    __syncthreads();

                    if (amLast) {
                        tc[16384] = 0;

                        // nullify shared memory for future accumulation
                        for (int e = threadIdx.x; e < numBins; e += blockDim.x) {
                            bins[e] = (Z) 0;
                        }

                        // accumulate reduced bins
                        for (int r = 0; r < gridDim.x; r++) {
                            Z *ptrBuf = ((Z *)allocationPointer) + (r * numBins);

                            for (int e = threadIdx.x; e < numBins; e += blockDim.x) {
                                math::atomics::nd4j_atomicAdd(&bins[e], ptrBuf[e]);
                            }
                        }
                        __syncthreads();

                        // write them out to Z
                        for (int e = threadIdx.x; e < numBins; e += blockDim.x) {
                            result[e] = bins[e];
                        }
                    }
                } else {
                    // if there's only 1 block - just write away data
                    for (int e = threadIdx.x; e < numBins; e += blockDim.x) {
                        result[e] = bins[e];
                    }
                }
            }

            template <typename X, typename Z>
            static void histogram_(sd::LaunchContext *context, void *xBuffer, const Nd4jLong *xShapeInfo, const Nd4jLong *dxShapeInfo, void *zBuffer, const Nd4jLong *zShapeInfo, Nd4jLong numBins, void* min_val, void* max_val) {
                int numThreads = 256;
                int numBlocks = sd::math::nd4j_max<int>(256, sd::math::nd4j_min<int>(1, shape::length(xShapeInfo) / numThreads));
                int workspaceSize = numBlocks * numBins;
                auto tmp = NDArrayFactory::create<Z>('c', {workspaceSize}, context);

                histogramKernel<X, Z><<<numBlocks, numThreads, 32768, *context->getCudaStream()>>>(xBuffer, dxShapeInfo, zBuffer, zShapeInfo, tmp.specialBuffer(), context->getReductionPointer(), numBins, reinterpret_cast<X*>(min_val), reinterpret_cast<X*>(max_val));

                hipStreamSynchronize(*context->getCudaStream());
            }

            void histogramHelper(sd::LaunchContext *context, NDArray &input, NDArray &output) {
                Nd4jLong numBins = output.lengthOf();
                NDArray::registerSpecialUse({&output}, {&input});

                auto min_val = input.reduceNumber(reduce::SameOps::Min);
                auto max_val = input.reduceNumber(reduce::SameOps::Max);
//                min_val.printIndexedBuffer("MIN");
//                max_val.printIndexedBuffer("MAX");
                BUILD_DOUBLE_SELECTOR(input.dataType(), output.dataType(), histogram_, (context, input.specialBuffer(), input.shapeInfo(), input.specialShapeInfo(), output.specialBuffer(), output.specialShapeInfo(), numBins, min_val.specialBuffer(), max_val.specialBuffer()), LIBND4J_TYPES, INTEGER_TYPES);
                NDArray::registerSpecialUse({&output}, {&input});
            }
        }
    }
}