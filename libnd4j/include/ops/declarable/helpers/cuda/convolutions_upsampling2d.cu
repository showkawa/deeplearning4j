#include "hip/hip_runtime.h"
/*
 *  ******************************************************************************
 *  *
 *  *
 *  * This program and the accompanying materials are made available under the
 *  * terms of the Apache License, Version 2.0 which is available at
 *  * https://www.apache.org/licenses/LICENSE-2.0.
 *  *
 *  * See the NOTICE file distributed with this work for additional
 *  * information regarding copyright ownership.
 *  * Unless required by applicable law or agreed to in writing, software
 *  * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 *  * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 *  * License for the specific language governing permissions and limitations
 *  * under the License.
 *  *
 *  * SPDX-License-Identifier: Apache-2.0
 *  *****************************************************************************
 */

//
// @author Yurii Shyrma (iuriish@yahoo.com)
//

#include <ops/declarable/helpers/convolutions.h>
#include <helpers/PointersManager.h>

namespace sd {
namespace ops  {

//////////////////////////////////////////////////////////////////////////
template <typename T>
__global__ static void upsampling2dCuda(const void* vx, const Nd4jLong* xShapeInfo, void* vz, const Nd4jLong* zShapeInfo, const int factorH, const int factorW, const bool isNCHW) {

    // x has shape [bS, iC, iH, iW] (NCHW) or [bS, iH, iW, iC] (NHWC)
    // z has shape [bS, iC, factorH*iH, factorW*iW ] (NCHW) or [bS, factorH*iH, factorW*iW, iC] (NHWC)

    const T* x = reinterpret_cast<const T*>(vx);
          T* z = reinterpret_cast<T*>(vz);

    __shared__ int rank, dimIH;
    __shared__ Nd4jLong zLen, *sharedMem;

    if (threadIdx.x == 0) {
        extern __shared__ unsigned char shmem[];
        sharedMem = reinterpret_cast<Nd4jLong*>(shmem);

        dimIH  = isNCHW ? 2 : 1;
        zLen   = shape::length(zShapeInfo);
        rank   = 4;
    }
    __syncthreads();

    const auto zInd = threadIdx.x + blockIdx.x * blockDim.x;

    if(zInd >= zLen)
        return;

    auto coords = sharedMem + threadIdx.x * rank;

    shape::index2coords(zInd, zShapeInfo, coords);

    const auto zOffset = shape::getOffset(zShapeInfo, coords);

    coords[dimIH]     /= factorH;
    coords[dimIH + 1] /= factorW;

    const auto xOffset = shape::getOffset(xShapeInfo, coords);

    z[zOffset] = x[xOffset];
}

//////////////////////////////////////////////////////////////////////////
template <typename T>
static void upsampling2dCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream,
                                     const void* vx, const Nd4jLong* xShapeInfo,
                                           void* vz, const Nd4jLong* zShapeInfo,
                                     const int factorH, const int factorW, const bool isNCHW) {

    upsampling2dCuda<T><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(vx, xShapeInfo, vz, zShapeInfo, factorH, factorW, isNCHW);
}

//////////////////////////////////////////////////////////////////////////
void ConvolutionUtils::upsampling2d(sd::graph::Context& block, const NDArray& input, NDArray& output, const int factorH, const int factorW, const bool isNCHW) {

    PointersManager manager(block.launchContext(), "upsampling2d");

    const int threadsPerBlock = MAX_NUM_THREADS / 2;
    const int blocksPerGrid = (output.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;
    const int sharedMem = output.rankOf() * sizeof(Nd4jLong) * threadsPerBlock + 128;

    NDArray::prepareSpecialUse({&output}, {&input});
    BUILD_SINGLE_SELECTOR(input.dataType(), upsampling2dCudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, block.launchContext()->getCudaStream(), input.specialBuffer(), input.specialShapeInfo(), output.specialBuffer(), output.specialShapeInfo(), factorH, factorW, isNCHW), FLOAT_TYPES);
    NDArray::registerSpecialUse({&output}, {&input});

    manager.synchronize();
}

}
}
