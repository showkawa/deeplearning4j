#include "hip/hip_runtime.h"
/*
 *  ******************************************************************************
 *  *
 *  *
 *  * This program and the accompanying materials are made available under the
 *  * terms of the Apache License, Version 2.0 which is available at
 *  * https://www.apache.org/licenses/LICENSE-2.0.
 *  *
 *  * See the NOTICE file distributed with this work for additional
 *  * information regarding copyright ownership.
 *  * Unless required by applicable law or agreed to in writing, software
 *  * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 *  * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 *  * License for the specific language governing permissions and limitations
 *  * under the License.
 *  *
 *  * SPDX-License-Identifier: Apache-2.0
 *  *****************************************************************************
 */

//
// @author Yurii Shyrma (iuriish@yahoo.com)
//

#include <ops/declarable/helpers/convolutions.h>
#include <helpers/PointersManager.h>
#include <math/templatemath.h>

namespace sd {
namespace ops  {

//////////////////////////////////////////////////////////////////////////
template <typename T>
__global__ static void pooling3dBPCuda(const void* vx, const Nd4jLong* xShapeInfo, const void* vy, const Nd4jLong* yShapeInfo, void* vz, const Nd4jLong* zShapeInfo, const int kD, const int kH, const int kW, const int sD, const int sH, const int sW, const int pD, const int pH, const int pW, const int dD, const int dH, const int dW, const int poolingMode, const int extraParam0) {

    // x: input [bS, iC, iD, iH, iW]
    // y: gradO [bS, iC, oD, oH, oW]
    // z: gradI [bS, iC, iD, iH, iW] -> gradI is output in this function


    const T* x = reinterpret_cast<const T*>(vx);
    const T* y = reinterpret_cast<const T*>(vy);
          T* z = reinterpret_cast<T*>(vz);

    Nd4jLong coord2, coord3, coord4;
    __shared__ int rank, kDeff, kHeff, kWeff, iD, iH, iW, kProd;
    __shared__ Nd4jLong yLen, *sharedMem;

    if (threadIdx.x == 0) {
        extern __shared__ unsigned char shmem[];
        sharedMem = reinterpret_cast<Nd4jLong*>(shmem);

        yLen = shape::length(yShapeInfo);
        rank = 5;

        kDeff = kD + (kD - 1) * (dD - 1);
        kHeff = kH + (kH - 1) * (dH - 1);
        kWeff = kW + (kW - 1) * (dW - 1);

        iD = xShapeInfo[3];
        iH = xShapeInfo[4];
        iW = xShapeInfo[5];

        kProd = kD * kH * kW;
    }
    __syncthreads();

    const auto yInd = threadIdx.x + blockIdx.x * blockDim.x;

    if(yInd >= yLen)
        return;

    auto coords = sharedMem + threadIdx.x * rank;

    shape::index2coords(yInd, yShapeInfo, coords);

    const auto yOffset = shape::getOffset(yShapeInfo, coords);

    int dstart = coords[2] * sD - pD;
    int hstart = coords[3] * sH - pH;
    int wstart = coords[4] * sW - pW;
    int dend = dstart + kDeff;
    int hend = hstart + kHeff;
    int wend = wstart + kWeff;

    if(dstart < 0)
        dstart += dD * ((-dstart + dD - 1) / dD);
    if(hstart < 0)
        hstart += dH * ((-hstart + dH - 1) / dH);
    if(wstart < 0)
        wstart += dW * ((-wstart + dW - 1) / dW);
    if(dend > iD)
        dend -= dD * ((dend - iD + dD - 1) / dD);
    if(hend > iH)
        hend -= dH * ((hend - iH + dH - 1) / dH);
    if(wend > iW)
        wend -= dW * ((wend - iW + dW - 1) / dW);


    switch (poolingMode) {

        /*** max ***/
        case 0: {

            T max = -DataTypeUtils::max<T>();
            for (coords[2] = dstart; coords[2] < dend; coords[2] += dD) {
                for (coords[3] = hstart; coords[3] < hend; coords[3] += dH){
                    for (coords[4] = wstart; coords[4] < wend; coords[4] += dW) {
                        T val = x[shape::getOffset(xShapeInfo, coords)];
                        if (val > max) {
                            max = val;
                            coord2 = coords[2];
                            coord3 = coords[3];
                            coord4 = coords[4];
                        }
                    }
                }
            }
            coords[2] = coord2;
            coords[3] = coord3;
            coords[4] = coord4;
            sd::math::atomics::nd4j_atomicAdd<T>(&z[shape::getOffset(zShapeInfo, coords)], y[yOffset]);
        }
        break;

        /*** avg ***/
        case 1: {

            T val = y[yOffset];

            if (extraParam0 == 0)         //Exclude padding
                val /= sd::math::nd4j_ceil<double,T>(static_cast<double>(dend - dstart) / static_cast<double>(dD))  * sd::math::nd4j_ceil<double,T>(static_cast<double>(hend - hstart) / static_cast<double>(dH))     * sd::math::nd4j_ceil<double,T>(static_cast<double>(wend - wstart)    / static_cast<double>(dW));   //Accounts for dilation
            else if (extraParam0 == 1)    //Include padding
                val /= kProd;

            for (coords[2] = dstart; coords[2] < dend; coords[2] += dD)
                for (coords[3] = hstart; coords[3] < hend; coords[3] += dH)
                    for (coords[4] = wstart; coords[4] < wend; coords[4] += dW)
                        sd::math::atomics::nd4j_atomicAdd<T>(&z[shape::getOffset(zShapeInfo, coords)], val);
        }
        break;

        /*** pnorm ***/
        case 2: {

            T sum = static_cast<T>(0.);
            T val = y[yOffset];

            for (coords[2] = dstart; coords[2] < dend; coords[2] += dD)
                for (coords[3] = hstart; coords[3] < hend; coords[3] += dH)
                    for (coords[4] = wstart; coords[4] < wend; coords[4] += dW)
                        sum += sd::math::nd4j_pow<T,T,T>(sd::math::nd4j_abs<T>(x[shape::getOffset(xShapeInfo, coords)]), extraParam0);

            val *= sd::math::nd4j_pow<T,T,T>(sum, ((T)1.f - extraParam0) / extraParam0);

            for (coords[2] = dstart; coords[2] < dend; coords[2] += dD) {
                for (coords[3] = hstart; coords[3] < hend; coords[3] += dH) {
                    for (coords[4] = wstart; coords[4] < wend; coords[4] += dW) {
                        const auto xOffset = shape::getOffset(xShapeInfo, coords);
                        const auto zOffset = shape::getOffset(zShapeInfo, coords);
                        sd::math::atomics::nd4j_atomicAdd<T>(&z[zOffset], val * sd::math::nd4j_pow<T,T,T>(sd::math::nd4j_abs<T>(x[xOffset]), extraParam0 - 1.f) * sd::math::nd4j_sgn<T,T>(x[xOffset]));
                    }
                }
            }
        }
        break;
    }
}

//////////////////////////////////////////////////////////////////////////
template <typename T>
static void pooling3dBPCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream,
                                    const void* vx, const Nd4jLong* xShapeInfo,
                                    const void* vy, const Nd4jLong* yShapeInfo,
                                          void* vz, const Nd4jLong* zShapeInfo,
                                    const int kD, const int kH, const int kW,
                                    const int sD, const int sH, const int sW,
                                    const int pD, const int pH, const int pW,
                                    const int dD, const int dH, const int dW,
                                    const int poolingMode, const int extraParam0) {

    pooling3dBPCuda<T><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(vx, xShapeInfo, vy, yShapeInfo, vz, zShapeInfo, kD, kH, kW, sD, sH, sW, pD, pH, pW, dD, dH, dW, poolingMode, extraParam0);
}

//////////////////////////////////////////////////////////////////////////
void ConvolutionUtils::pooling3dBP(sd::graph::Context& block, const NDArray& input, const NDArray& gradO, NDArray& gradI, const int kD, const int kH, const int kW, const int sD, const int sH, const int sW, const int pD, const int pH, const int pW, const int dD, const int dH, const int dW, const int poolingMode, const int extraParam0) {

    // initial zeroing of gradI
    gradI.nullify();

    PointersManager manager(block.launchContext(), "pooling3dBP");

    const int threadsPerBlock = MAX_NUM_THREADS / 2;
    const int blocksPerGrid = (gradO.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;
    const int sharedMem = gradO.rankOf() * sizeof(Nd4jLong) * threadsPerBlock + 128;

    NDArray::prepareSpecialUse({&gradI}, {&input, &gradO});
    BUILD_SINGLE_SELECTOR(input.dataType(), pooling3dBPCudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, block.launchContext()->getCudaStream(), input.specialBuffer(), input.specialShapeInfo(), gradO.specialBuffer(), gradO.specialShapeInfo(), gradI.specialBuffer(), gradI.specialShapeInfo(), kD, kH, kW, sD, sH, sW, pD, pH, pW, dD, dH, dW, poolingMode, extraParam0), FLOAT_TYPES);
    NDArray::registerSpecialUse({&gradI}, {&input, &gradO});

    manager.synchronize();
}

}
}
