#include "hip/hip_runtime.h"
/*
 *  ******************************************************************************
 *  *
 *  *
 *  * This program and the accompanying materials are made available under the
 *  * terms of the Apache License, Version 2.0 which is available at
 *  * https://www.apache.org/licenses/LICENSE-2.0.
 *  *
 *  * See the NOTICE file distributed with this work for additional
 *  * information regarding copyright ownership.
 *  * Unless required by applicable law or agreed to in writing, software
 *  * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 *  * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 *  * License for the specific language governing permissions and limitations
 *  * under the License.
 *  *
 *  * SPDX-License-Identifier: Apache-2.0
 *  *****************************************************************************
 */

//
// @author Yurii Shyrma (iuriish@yahoo.com)
//

#include <ops/declarable/helpers/convolutions.h>
#include <helpers/PointersManager.h>

namespace sd {
namespace ops  {

//////////////////////////////////////////////////////////////////////////
// vol [bS, iC, iD, iH, iW] is convoluted to col [bS, iC, kD, kH, kW, oD, oH, oW]
template <typename T>
static __global__ void vol2colCuda(const void* volume, const Nd4jLong* volShapeInfo, void* columns, const Nd4jLong* colShapeInfo,  const int sD, const int sH, const int sW, const int pD, const int pH, const int pW, const int dD, const int dH, const int dW) {

    const T* vol = reinterpret_cast<const T*>(volume);
          T* col = reinterpret_cast<T*>(columns);

    __shared__ int colRank, volRank;
    __shared__ Nd4jLong colLen, iD, iH, iW, *sharedMem;

    if (threadIdx.x == 0) {
        extern __shared__ unsigned char shmem[];
        sharedMem = reinterpret_cast<Nd4jLong*>(shmem);

        volRank = 5;
        colRank = 8;

        colLen = shape::length(colShapeInfo);

        iD = volShapeInfo[3];
        iH = volShapeInfo[4];
        iW = volShapeInfo[5];
    }
    __syncthreads();

    const auto colInd = threadIdx.x + blockIdx.x * blockDim.x;

    if(colInd >= colLen)
        return;

    auto coords = sharedMem + threadIdx.x * colRank;

    shape::index2coords(colInd, colShapeInfo, coords);

    // const auto colW = coords[7];
    // const auto colH = coords[6];
    // const auto colD = coords[5];
    // const auto kCol = coords[4];
    // const auto kRow = coords[3];
    // const auto kDep = coords[2];
    // const auto c    = coords[1];
    // const auto b    = coords[0];

    const auto colOffset = shape::getOffset(colShapeInfo, coords);

    coords[2] = -pD + coords[2] * dD + coords[5] * sD;     // const auto volDep = (-pD + kDep * dD) + colD * sD;
    coords[3] = -pH + coords[3] * dH + coords[6] * sH;     // const auto volRow = (-pH + kRow * dH) + colH * sH;
    coords[4] = -pW + coords[4] * dW + coords[7] * sW;     // const auto volCol = (-pW + kCol * dW) + colW * sW;

    if (static_cast<unsigned>(coords[2]) >= static_cast<unsigned>(iD) || static_cast<unsigned>(coords[3]) >= static_cast<unsigned>(iH) || static_cast<unsigned>(coords[4]) >= static_cast<unsigned>(iW))
        col[colOffset] = static_cast<T>(0.);
    else
        col[colOffset] = vol[shape::getOffset(volShapeInfo, coords)];
}

//////////////////////////////////////////////////////////////////////////
template <typename T>
static void vol2colCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream,
                                const void* volume, const Nd4jLong* volShapeInfo,
                                      void* columns, const Nd4jLong* colShapeInfo,
                                const int sD, const int sH, const int sW, const int pD, const int pH, const int pW, const int dD, const int dH, const int dW) {

    vol2colCuda<T><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(volume, volShapeInfo, columns, colShapeInfo,  sD, sH, sW, pD, pH, pW, dD, dH, dW);
}

//////////////////////////////////////////////////////////////////////////
void ConvolutionUtils::vol2col(sd::graph::Context& block, const NDArray& vol, NDArray& col, const int sD, const int sH, const int sW, const int pD, const int pH, const int pW, const int dD, const int dH, const int dW) {

    PointersManager manager(block.launchContext(), "vol2col");

    const int threadsPerBlock = MAX_NUM_THREADS / 4;
    const int blocksPerGrid = (col.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;
    const int sharedMem = col.rankOf() * sizeof(Nd4jLong) * threadsPerBlock  + 128;

    NDArray::prepareSpecialUse({&col}, {&vol});
    BUILD_SINGLE_SELECTOR(vol.dataType(), vol2colCudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, block.launchContext()->getCudaStream(), vol.specialBuffer(), vol.specialShapeInfo(), col.specialBuffer(), col.specialShapeInfo(), sD, sH, sW, pD, pH, pW, dD, dH, dW), FLOAT_TYPES);
    NDArray::registerSpecialUse({&col}, {&vol});

    manager.synchronize();
}

}
}