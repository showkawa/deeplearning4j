#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author GS <sgazeos@gmail.com>
//

#include <ops/declarable/helpers/segment.h>
#include <ops/declarable/helpers/segment_common.h>
#include <array/NDArrayFactory.h>
#include <helpers/ShapeUtils.h>
#include <helpers/TAD.h>
#include <exceptions/cuda_exception.h>
#include <helpers/PointersManager.h>
#include <helpers/ConstantTadHelper.h>

namespace sd {
namespace ops {
namespace helpers {
    // -------------------------------------------------------------------------------------------------------------- //
    // Segment Prod ops linear kernels
    // -------------------------------------------------------------------------------------------------------------- //

    template <typename T, typename I>
    static __global__ void segmentProdLinearKernel(void* input, Nd4jLong const* inputShape, int* starts, int* lengths,
            Nd4jLong numOfClasses, void* output, Nd4jLong const* outputShape) {

        __shared__ Nd4jLong xLen, zLen;
        __shared__ T* x;
        __shared__ T* z;

        if (threadIdx.x == 0) {
            x = reinterpret_cast<T*>(input);
            z = reinterpret_cast<T*>(output);
            xLen = shape::length(inputShape);
            zLen = shape::length(outputShape);
        }
        __syncthreads();

        for(auto segment = blockIdx.x; segment < numOfClasses; segment += gridDim.x) {
            auto zIndex = shape::getIndexOffset(segment, outputShape);
            auto start = starts[segment];
            auto finish = start + lengths[segment];
            if (lengths[segment] == 0) {
                continue;
            }
            for (auto e = start + threadIdx.x; e < finish; e += blockDim.x) {
                auto xIndex = shape::getIndexOffset(e, inputShape);
                sd::math::atomics::nd4j_atomicMul(&z[segment], x[xIndex]);
            }
        }

    }
    // -------------------------------------------------------------------------------------------------------------- //
    template <typename T, typename I>
    static __global__ void unsortedSegmentProdLinearKernel(T* input, Nd4jLong const* inputShape, I* indices, Nd4jLong const* indicesShape, int* starts, int* lengths, Nd4jLong numOfClasses, T* output, Nd4jLong const* outputShape) {
        __shared__ Nd4jLong xLen, zLen;

        if (threadIdx.x == 0) {
            xLen = shape::length(inputShape);
            zLen = shape::length(outputShape);
        }
        __syncthreads();
        auto start = threadIdx.x + blockIdx.x * blockDim.x;
        auto step = blockDim.x * gridDim.x;
        for (auto idx = start; idx < xLen; idx += step) {
            auto xIndex = shape::getIndexOffset(idx, inputShape);
            auto yIndex = shape::getIndexOffset(idx, indicesShape);
            auto segment = indices[yIndex];
            auto zIndex = shape::getIndexOffset(segment, outputShape);
            if (lengths[segment] == 0) {
                continue;
            }
            sd::math::atomics::nd4j_atomicMul(&output[zIndex], input[xIndex]);
        }
    }
    // -------------------------------------------------------------------------------------------------------------- //
    // SegmentProd kernel
    template <typename T, typename I>
    static __global__ void segmentProdTadKernel(void* inputBuf, Nd4jLong const* inputShape, Nd4jLong const* inputTads,
            Nd4jLong const* inputTadOffsets, I* indices, int* starts, int* lengths, Nd4jLong numOfClasses, void* outputBuf,
            Nd4jLong const* outputShape, Nd4jLong const* outputTads, Nd4jLong const* outputTadOffsets) {

        __shared__ Nd4jLong len, total;

        if (threadIdx.x == 0) {
            total = shape::sizeAt(inputShape, 0);
            len = shape::length(inputTads);
        }
        __syncthreads();

        for (auto idx = blockIdx.x; idx < total; idx += gridDim.x) {
            auto x = reinterpret_cast<T *>(inputBuf) + inputTadOffsets[idx];
            auto segment = indices[idx]; // / threadsPerSegment;
            auto z = reinterpret_cast<T *>(outputBuf) + outputTadOffsets[segment];
            auto start = starts[segment];
            auto finish = start + lengths[segment];
            if (lengths[segment] == 0) continue;
            for (auto e = threadIdx.x; e < len; e += blockDim.x) {
                auto xIndex = shape::getIndexOffset(e, inputTads);
                auto zIndex = shape::getIndexOffset(e, outputTads);
                sd::math::atomics::nd4j_atomicMul(&z[zIndex], x[xIndex]);
            }
        }
    }
    // -------------------------------------------------------------------------------------------------------------- //

    template <typename T, typename I>
    static void segmentProdFunctor_(sd::LaunchContext* context, NDArray* input, NDArray* indices, NDArray* output) {
        auto stream = context->getCudaStream();
        Nd4jLong numClasses = indices->e<Nd4jLong>(indices->lengthOf() - 1) + 1;
        NDArray classesRangesLens = NDArrayFactory::create<int>('c', {numClasses}, context);
        NDArray classesRangesBegs = NDArrayFactory::create<int>('c', {numClasses}, context);
        output->assign(1);
        classesRangesBegs.assign(indices->lengthOf());
        classesRangesLens.assign(0);

        dim3 dims(numClasses, indices->lengthOf(), numClasses * 32 + 32);
        fillUpSegments(indices, numClasses, classesRangesBegs, classesRangesLens);
        int* begins = reinterpret_cast<int*>(classesRangesBegs.specialBuffer());
        int* lengths = reinterpret_cast<int*>(classesRangesLens.specialBuffer());

        if (input->isVector()) {
            segmentProdLinearKernel<T,I><<<128, 256, 128, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), begins, lengths, numClasses, output->specialBuffer(), output->specialShapeInfo());
        }
        else {
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = sd::ConstantTadHelper::getInstance().tadForDimensions(input->shapeInfo(), dimensions);
            auto packZ = sd::ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(), dimensions);
            auto inputTads = packX.specialShapeInfo();
            auto inputTadOffsets = packX.specialOffsets();
            auto outputTads = packZ.specialShapeInfo();
            auto outputTadOffsets = packZ.specialOffsets();
            segmentProdTadKernel<T,I><<<128, 512, 2048, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), inputTads, inputTadOffsets, reinterpret_cast<I*>(indices->specialBuffer()), begins, lengths, numClasses, output->specialBuffer(), output->specialShapeInfo(), outputTads, outputTadOffsets);
        }

    }
    // -------------------------------------------------------------------------------------------------------------- //
    void segmentProdFunctor(sd::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* output) {
        NDArray::prepareSpecialUse({output}, {input, indices});
        BUILD_DOUBLE_SELECTOR(output->dataType(), indices->dataType(), segmentProdFunctor_, (context, input, indices, output), NUMERIC_TYPES, INDEXING_TYPES);
        NDArray::registerSpecialUse({output}, {input, indices});
    }

    // -------------------------------------------------------------------------------------------------------------- //
    template <typename T, typename I>
    static void unsortedSegmentProdFunctor_(sd::LaunchContext* context, NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {
        auto stream = context->getCudaStream();
//        NDArray classes = NDArrayFactory::create<int>('c', {numOfClasses, 2});
        NDArray classesRangesBegs = NDArrayFactory::create<int>('c', {numOfClasses}, context);
        NDArray classesRangesLens = NDArrayFactory::create<int>('c', {numOfClasses}, context);
//        NDArray row = NDArrayFactory::create<int>('c', {1, 2}, {(int)indices->lengthOf(), (int)0});
//        classes.applyTrueBroadcast(sd::BroadcastOpsTuple::Assign(), &row, &classes);
        classesRangesBegs.assign(indices->lengthOf());
        classesRangesLens.assign(0);
        dim3 dims(numOfClasses, indices->lengthOf(), numOfClasses * 32 + 32);
//        int* classesBuf = reinterpret_cast<int*>(classes.specialBuffer());
        fillUpSegments(indices, numOfClasses, classesRangesBegs, classesRangesLens);
        int* begins = reinterpret_cast<int*>(classesRangesBegs.specialBuffer());
        int* lengths = reinterpret_cast<int*>(classesRangesLens.specialBuffer());
        output->assign(1);

        if (input->isVector()) {
            unsortedSegmentProdLinearKernel<T,I><<<128, 256, 256, *stream>>>(
                    input->dataBuffer()->specialAsT<T>(), input->specialShapeInfo(),
                    indices->dataBuffer()->specialAsT<I>(), indices->specialShapeInfo(), begins, lengths, numOfClasses,
                    output->dataBuffer()->specialAsT<T>(), output->specialShapeInfo());
        }
        else {
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = sd::ConstantTadHelper::getInstance().tadForDimensions(input->shapeInfo(), dimensions);
            auto packZ = sd::ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(), dimensions);
            auto inputTads = packX.specialShapeInfo();
            auto inputTadOffsets = packX.specialOffsets();
            auto outputTads = packZ.specialShapeInfo();
            auto outputTadOffsets = packZ.specialOffsets();
            dims.x = input->sizeAt(0);
            segmentProdTadKernel<T,I><<<128, 256, 256, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), inputTads, inputTadOffsets, reinterpret_cast<I*>(indices->specialBuffer()), begins, lengths, numOfClasses, output->specialBuffer(), output->specialShapeInfo(), outputTads, outputTadOffsets);
        }

    }
    // -------------------------------------------------------------------------------------------------------------- //
    void unsortedSegmentProdFunctor(sd::LaunchContext* context , NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {
        NDArray::prepareSpecialUse({output}, {input, indices});
        BUILD_DOUBLE_SELECTOR(input->dataType(), indices->dataType(), unsortedSegmentProdFunctor_, (context, input, indices, numOfClasses, output),
                              NUMERIC_TYPES, INDEXING_TYPES);
        NDArray::registerSpecialUse({output}, {input, indices});
    }

    // -------------------------------------------------------------------------------------------------------------- //
    template <typename T, typename I>
    static __global__ void segmentProdBPLinearKernel(void* inputBuf, Nd4jLong const* inputShape, void* forwardOutput,
                                                     Nd4jLong const* forwardShape, void* eps, Nd4jLong const* epsShape, void* indicesBuf, Nd4jLong const* indicesShape,
                                                     void* outputBuf, Nd4jLong const* outputShape) {
        __shared__ T* x;
        __shared__ T* gradIn;
        __shared__ T* gradOut;
        __shared__ I* y;
        __shared__ T* z;
        __shared__ Nd4jLong xLen, gradLen;

        if (threadIdx.x == 0) {
            xLen = shape::length(inputShape);
            x = reinterpret_cast<T*>(inputBuf);
            y = reinterpret_cast<I*>(indicesBuf);
            z = reinterpret_cast<T*>(outputBuf);
            gradIn = reinterpret_cast<T*>(forwardOutput);
            gradOut = reinterpret_cast<T*>(eps);
            gradLen = shape::length(epsShape);
        }
        __syncthreads();

        auto start = blockIdx.x * blockDim.x + threadIdx.x;
        auto step = gridDim.x * blockDim.x;

        for (auto e = start; e < xLen; e += step) {

            auto zOffset = shape::getIndexOffset(e, outputShape);
            auto xOffset = shape::getIndexOffset(e, inputShape);
            auto yOffset = shape::getIndexOffset(e, indicesShape);
            auto classIndex = y[yOffset];
            auto gradOffsetI = shape::getIndexOffset(classIndex, forwardShape);
            auto gradOffsetO = shape::getIndexOffset(classIndex, epsShape);

            z[zOffset] = gradOut[gradOffsetO]  * gradIn[gradOffsetI] / x[xOffset];
        }
    }
    // -------------------------------------------------------------------------------------------------------------- //
    template <typename T, typename I>
    static __global__ void segmentProdBPTadKernel(void* inputBuf, Nd4jLong const* inputShape, void* forwardOutput,
                                                  Nd4jLong const* forwardShape, void* eps, Nd4jLong const* epsShape, void* indicesBuf, Nd4jLong const* indicesShape,
                                                  void* outputBuf, Nd4jLong const* outputShape, Nd4jLong const* inputTad,
                                                  Nd4jLong const* inputOffsets, Nd4jLong const* gradInTad, Nd4jLong const* gradInOffsets,
                                                  Nd4jLong const* gradOutTad, Nd4jLong const* gradOutOffsets, Nd4jLong const* outTad,
                                                  Nd4jLong const* outOffsets) {
        __shared__ T* x;
        __shared__ T* gradIn;
        __shared__ T* gradOut;
        __shared__ I* y;
        __shared__ T* z;
        __shared__ Nd4jLong xLen, yLen, gradLen, currentLen;

        if (threadIdx.x == 0) {
            xLen = shape::length(inputShape);
            x = reinterpret_cast<T*>(inputBuf);
            y = reinterpret_cast<I*>(indicesBuf);
            z = reinterpret_cast<T*>(outputBuf);
            yLen = shape::length(indicesShape);
            gradOut = reinterpret_cast<T*>(eps);
            gradIn = reinterpret_cast<T*>(forwardOutput);
            gradLen = shape::length(epsShape);
            currentLen = shape::length(outTad);
        }
        __syncthreads();

        for (auto i = blockIdx.x; i < yLen; i += gridDim.x) {
            auto yIndex = shape::getIndexOffset(i, indicesShape);
            auto segment = y[yIndex];
            T* current = x + inputOffsets[i];
            T* currentOut = z + outOffsets[i];
            T* in = gradIn + gradInOffsets[segment];
            T* outGrad = gradOut + gradOutOffsets[segment];

            for (auto e = threadIdx.x; e < currentLen; e += blockDim.x) {
                currentOut[e] = outGrad[e] * in[e] / current[e];
            }
        }
    }

    // -------------------------------------------------------------------------------------------------------------- //
    template <typename T, typename I>
    int segmentProdFunctorBP_(sd::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output) {
        auto stream = context->getCudaStream();
        NDArray tempRes(gradOut->ordering(), gradOut->getShapeAsVector(), DataTypeUtils::fromT<T>(), context);//->shapeInfo(), context);
        segmentProdFunctor_<T, I>(context, input, indices, &tempRes);
        NDArray::prepareSpecialUse({output}, {input, indices, gradOut});
        if (input->isVector()) {
            Nd4jLong loopSize = input->lengthOf();
            auto numOfClasses = gradOut->lengthOf(); //indices->e<Nd4jLong>(loop_size - 1);
            segmentProdBPLinearKernel<T,I><<<gradOut->lengthOf(), loopSize, 256, *stream>>>(input->specialBuffer(), input->specialShapeInfo(),
                    tempRes.specialBuffer(), tempRes.specialShapeInfo(), gradOut->specialBuffer(), gradOut->specialShapeInfo(),
                    indices->specialBuffer(), indices->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo());
        }
        else {
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = sd::ConstantTadHelper::getInstance().tadForDimensions(input->shapeInfo(), dimensions);
            auto packZ = sd::ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(), dimensions);
            auto packGradIn = sd::ConstantTadHelper::getInstance().tadForDimensions(tempRes.shapeInfo(), dimensions);
            auto packGradOut = sd::ConstantTadHelper::getInstance().tadForDimensions(gradOut->shapeInfo(), dimensions);
            auto inputTads = packX.specialShapeInfo();
            auto inputTadOffsets = packX.specialOffsets();
            auto outputTads = packZ.specialShapeInfo();
            auto outputTadOffsets = packZ.specialOffsets();
            auto gradInTads = packGradIn.specialShapeInfo();
            auto gradInTadOffsets = packGradIn.specialOffsets();
            auto gradOutTads = packGradOut.specialShapeInfo();
            auto gradOutTadOffsets = packGradOut.specialOffsets();

            segmentProdBPTadKernel<T,I><<<gradOut->lengthOf(), input->lengthOf(), 256, *stream>>>(input->specialBuffer(), input->specialShapeInfo(),
                    tempRes.specialBuffer(), tempRes.specialShapeInfo(), gradOut->specialBuffer(), gradOut->specialShapeInfo(),
                    indices->specialBuffer(), indices->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo(),
                    inputTads, inputTadOffsets, gradInTads, gradInTadOffsets, gradOutTads, gradOutTadOffsets,
                    outputTads, outputTadOffsets);
        }
        NDArray::registerSpecialUse({output}, {input, indices, gradOut});
        return Status::OK();
    }

    // -------------------------------------------------------------------------------------------------------------- //

    int segmentProdFunctorBP(sd::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output) {
        NDArray::prepareSpecialUse({output}, {input, indices, gradOut});
        BUILD_DOUBLE_SELECTOR(output->dataType(), indices->dataType(), return segmentProdFunctorBP_, (context, input,
                indices, gradOut, output), FLOAT_TYPES, INDEXING_TYPES);
        NDArray::registerSpecialUse({output}, {input, indices, gradOut});
    }

    // -------------------------------------------------------------------------------------------------------------- //

    template <typename T, typename I>
    static int unsortedSegmentProdFunctorBP_(sd::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        auto stream = context->getCudaStream();

        NDArray tempRes(gradOut->ordering(), gradOut->getShapeAsVector(), DataTypeUtils::fromT<T>(), context);//->shapeInfo(), context);
        unsortedSegmentProdFunctor_<T, I>(context, input, indices, numOfClasses, &tempRes);
        NDArray::prepareSpecialUse({output}, {input, indices, gradOut});
        if (input->isVector()) {
            Nd4jLong loopSize = input->lengthOf();
            auto numOfClasses = gradOut->lengthOf(); //indices->e<Nd4jLong>(loop_size - 1);
            segmentProdBPLinearKernel<T,I><<<gradOut->lengthOf(), loopSize, 256, *stream>>>(input->specialBuffer(), input->specialShapeInfo(),
                    tempRes.specialBuffer(), tempRes.specialShapeInfo(), gradOut->specialBuffer(), gradOut->specialShapeInfo(),
                    indices->specialBuffer(), indices->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo());
        }
        else {
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = sd::ConstantTadHelper::getInstance().tadForDimensions(input->shapeInfo(), dimensions);
            auto packZ = sd::ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(), dimensions);
            auto packGradIn = sd::ConstantTadHelper::getInstance().tadForDimensions(tempRes.shapeInfo(), dimensions);
            auto packGradOut = sd::ConstantTadHelper::getInstance().tadForDimensions(gradOut->shapeInfo(), dimensions);
            auto inputTads = packX.specialShapeInfo();
            auto inputTadOffsets = packX.specialOffsets();
            auto outputTads = packZ.specialShapeInfo();
            auto outputTadOffsets = packZ.specialOffsets();
            auto gradInTads = packGradIn.specialShapeInfo();
            auto gradInTadOffsets = packGradIn.specialOffsets();
            auto gradOutTads = packGradOut.specialShapeInfo();
            auto gradOutTadOffsets = packGradOut.specialOffsets();

            segmentProdBPTadKernel<T,I><<<indices->lengthOf(), input->lengthOf(), 256, *stream>>>(input->specialBuffer(), input->specialShapeInfo(),
                    tempRes.specialBuffer(), tempRes.specialShapeInfo(), gradOut->specialBuffer(), gradOut->specialShapeInfo(),
                    indices->specialBuffer(), indices->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo(),
                    inputTads, inputTadOffsets, gradInTads, gradInTadOffsets, gradOutTads, gradOutTadOffsets,
                    outputTads, outputTadOffsets);
        }
        NDArray::registerSpecialUse({output}, {input, indices, gradOut});
        return Status::OK();
    }

    // -------------------------------------------------------------------------------------------------------------- //
    int unsortedSegmentProdFunctorBP(sd::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        NDArray::prepareSpecialUse({output}, {input, indices, gradOut});
        BUILD_DOUBLE_SELECTOR(output->dataType(), indices->dataType(), return unsortedSegmentProdFunctorBP_, (context, input, indices, gradOut, numOfClasses, output), FLOAT_TYPES, INDEXING_TYPES);
        NDArray::registerSpecialUse({output}, {input, indices, gradOut});
    }

    // -------------------------------------------------------------------------------------------------------------- //

}
}
}
