#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author GS <sgazeos@gmail.com>
//

#include <ops/declarable/helpers/segment.h>
#include <ops/declarable/helpers/segment_common.h>

#include <array/NDArrayFactory.h>
#include <helpers/ShapeUtils.h>
#include <helpers/TAD.h>
#include <exceptions/cuda_exception.h>
#include <helpers/PointersManager.h>
#include <helpers/ConstantTadHelper.h>

namespace sd {
    namespace ops {
        namespace helpers {

            // -------------------------------------------------------------------------------------------------------------- //
            // Segment ops linear kernels
            // -------------------------------------------------------------------------------------------------------------- //

            template<typename T, typename I>
            static __global__ void
            segmentMaxLinearKernel(void *input, Nd4jLong const* inputShape, int *starts, int *lengths, Nd4jLong numOfClasses,
                                   void *output, Nd4jLong const* outputShape) {
                __shared__                 T *val;
                __shared__                Nd4jLong xLen, zLen, zIndex;
                __shared__                T *x;
                __shared__                T *z;
                __shared__ int threadsPerSegment, start, finish;

                auto segment = blockIdx.x;
                if (threadIdx.x == 0) {
//                    threadsPerSegment = (gridDim.x + numOfClasses - 1) / numOfClasses;
//                    segment = blockIdx.x / threadsPerSegment;
                    x = reinterpret_cast<T *>(input);
                    z = reinterpret_cast<T *>(output);
                    extern __shared__ unsigned char shmem[];
                    val = reinterpret_cast<T *>(shmem);
                    xLen = shape::length(inputShape);
                    zLen = shape::length(outputShape);

                    if (segment < numOfClasses) {
                        zIndex = shape::getIndexOffset(segment, outputShape);
                        start = starts[segment];
                        finish = start + lengths[segment];
                        z[zIndex] = x[shape::getIndexOffset(start, inputShape)];
                        val[segment] = z[zIndex];
                    }

                }
                __syncthreads();

                for (auto e = start + threadIdx.x + 1; e < finish; e += blockDim.x) {
                    auto xIndex = shape::getIndexOffset(e, inputShape);
                    sd::math::atomics::nd4j_atomicMax(&z[zIndex], x[xIndex]);
                }
            }
            // -------------------------------------------------------------------------------------------------------------- //

            template<typename T, typename I>
            static __global__ void
            unsortedSegmentMaxLinearKernel(void *input, Nd4jLong const* inputShape, void *indices, Nd4jLong const* indicesShape,
                                           int *starts, int *lengths, Nd4jLong numOfClasses, void *output,
                                           Nd4jLong const* outputShape) {
                __shared__                 T *val;
                __shared__                Nd4jLong xLen, zLen, zIndex;
                __shared__                T *x;
                __shared__                T *z;
                __shared__                I *y; //int threadsPerSegment, start, finish;
                auto segment = blockIdx.x;

                if (threadIdx.x == 0) {
                    x = reinterpret_cast<T *>(input);
                    z = reinterpret_cast<T *>(output);
                    y = reinterpret_cast<I *>(indices);
                    xLen = shape::length(inputShape);
                    zLen = shape::length(outputShape);

                    zIndex = shape::getIndexOffset(segment, outputShape);
                    //start = starts[segment];
                    //finish = start + lengths[segment];
                    if (lengths[segment] > 0)
                        z[zIndex] = x[shape::getIndexOffset(starts[segment], inputShape)];
                    else
                        z[zIndex] = -DataTypeUtils::max<T>();
                }
                __syncthreads();
                if (lengths[segment] > 0)
                    for (auto e = threadIdx.x + 1; e < xLen; e += blockDim.x) {
                        auto xIndex = shape::getIndexOffset(e, inputShape);
                        auto yIndex = shape::getIndexOffset(e, indicesShape);
                        if (y[yIndex] == segment) {
                            sd::math::atomics::nd4j_atomicMax(&z[zIndex], x[xIndex]);
                        }
                    }
            }
            // -------------------------------------------------------------------------------------------------------------- //
            template <typename T, typename I>
            static __global__ void segmentMaxTadKernel(void* inputBuf, Nd4jLong const* inputShape, Nd4jLong const* inputTads,
                                                       Nd4jLong const* inputTadOffsets, I* indices, int* starts, int* lengths, Nd4jLong numOfClasses, void* outputBuf,
                                                       Nd4jLong const* outputShape, Nd4jLong const* outputTads, Nd4jLong const* outputTadOffsets, T filler = 0) {

                __shared__ T* val;
                __shared__ Nd4jLong len, zIndex, total;
                __shared__ T* z;
                __shared__ int start, finish;
                __shared__ I segment;

                if (threadIdx.x == 0) {
                    segment = indices[blockIdx.x]; // / threadsPerSegment;
                    z = reinterpret_cast<T*>(outputBuf) + outputTadOffsets[segment];
                    len = shape::length(inputTads);

                    start = starts[segment];
                    finish = start + lengths[segment];
                    total = shape::sizeAt(inputShape, 0);
                }
                __syncthreads();

                auto idx = blockIdx.x;
                if (idx <= total) {
                    auto x = reinterpret_cast<T *>(inputBuf) + inputTadOffsets[idx];
                    if (blockIdx.x == start) {
                        for (auto e = threadIdx.x; e < len; e += blockDim.x) {
                            auto xIndex = shape::getIndexOffset(e, inputTads);
                            auto zIndex = shape::getIndexOffset(e, outputTads);
                            sd::math::atomics::nd4j_atomicMax(&z[zIndex], x[xIndex]);
                            //z[zIndex] = x[xIndex];
                        }
                    }
                    else {
                        for (auto e = threadIdx.x; e < len; e += blockDim.x) {
                            auto xIndex = shape::getIndexOffset(e, inputTads);
                            auto zIndex = shape::getIndexOffset(e, outputTads);
                            if (lengths[segment])
                                sd::math::atomics::nd4j_atomicMax(&z[zIndex], x[xIndex]);
                        }
                    }
                }
            }
            // -------------------------------------------------------------------------------------------------------------- //

            template <typename T, typename I>
            static void segmentMaxFunctor_(LaunchContext* context, NDArray* input, NDArray* indices, NDArray* output) {
                //int numClasses = output->sizeAt(0);
                // if input is a vector: (as if in doc sample)
                //Nd4jLong idx = indices->e<Nd4jLong>(0);
                output->assign(-DataTypeUtils::infOrMax<T>());
                auto stream = context->getCudaStream();
                indices->syncToHost();
                Nd4jLong numOfClasses = indices->e<Nd4jLong>(indices->lengthOf() - 1) + 1;
                NDArray classesRangesLens = NDArrayFactory::create<int>('c', {numOfClasses}, context);
                NDArray classesRangesBegs = NDArrayFactory::create<int>('c', {numOfClasses}, context);

                classesRangesBegs.assign(indices->lengthOf());
                classesRangesLens.assign(0);
                dim3 dims(256, 512, 256);
                int* begins = reinterpret_cast<int*>(classesRangesBegs.specialBuffer());
                int* lengths = reinterpret_cast<int*>(classesRangesLens.specialBuffer());
                fillUpSegments(indices, numOfClasses, classesRangesBegs, classesRangesLens);

                NDArray::prepareSpecialUse({output}, {input, indices, &classesRangesBegs, &classesRangesLens});

                if (input->isVector()) {

                    segmentMaxLinearKernel<T,I><<<numOfClasses, input->lengthOf(), numOfClasses * 32 + 32, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), begins, lengths, numOfClasses, output->specialBuffer(), output->specialShapeInfo());
                }
                else {
                    std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
                    auto packX = sd::ConstantTadHelper::getInstance().tadForDimensions(input->shapeInfo(), dimensions);
                    auto packZ = sd::ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(), dimensions);
                    auto inputTads = packX.specialShapeInfo();
                    auto inputTadOffsets = packX.specialOffsets();
                    auto outputTads = packZ.specialShapeInfo();
                    auto outputTadOffsets = packZ.specialOffsets();
                    segmentMaxTadKernel<T,I><<<packX.numberOfTads(), 512, 2048, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), inputTads, inputTadOffsets, reinterpret_cast<I*>(indices->specialBuffer()), begins, lengths, numOfClasses, output->specialBuffer(), output->specialShapeInfo(), outputTads, outputTadOffsets);
                }
                NDArray::registerSpecialUse({output}, {input, indices, &classesRangesBegs, &classesRangesLens});
            }
            // -------------------------------------------------------------------------------------------------------------- //
            void segmentMaxFunctor(sd::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* output) {
                NDArray::prepareSpecialUse({output}, {input, indices});
                BUILD_DOUBLE_SELECTOR(input->dataType(), indices->dataType(), segmentMaxFunctor_, (context, input, indices, output), NUMERIC_TYPES, INDEXING_TYPES);
                NDArray::registerSpecialUse({output}, {input, indices});
            }
            // -------------------------------------------------------------------------------------------------------------- //

            template <typename T, typename I>
            static void unsortedSegmentMaxFunctor_(sd::LaunchContext* context, NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {
                auto stream = context->getCudaStream();
//        NDArray classes = NDArrayFactory::create<int>('c', {numOfClasses, 2});
                output->assign(DataTypeUtils::infOrMax<T>());

                NDArray classesRangesBegs = NDArrayFactory::create<int>('c', {numOfClasses}, context);
                NDArray classesRangesLens = NDArrayFactory::create<int>('c', {numOfClasses}, context);
//        NDArray row = NDArrayFactory::create<int>('c', {1, 2}, {(int)indices->lengthOf(), (int)0});
//        classes.applyTrueBroadcast(sd::BroadcastOpsTuple::Assign(), row, classes);
                classesRangesBegs.assign(indices->lengthOf());
                classesRangesLens.assign(0);
                dim3 dims(numOfClasses, indices->lengthOf(), numOfClasses * 32 + 32);
//        int* classesBuf = reinterpret_cast<int*>(classes.specialBuffer());
                fillUpSegments(indices, numOfClasses, classesRangesBegs, classesRangesLens);
                int* begins = reinterpret_cast<int*>(classesRangesBegs.specialBuffer());
                int* lengths = reinterpret_cast<int*>(classesRangesLens.specialBuffer());

                if (input->isVector()) {
                    unsortedSegmentMaxLinearKernel<T,I><<<dims.x, dims.y, dims.z, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), indices->specialBuffer(), indices->specialShapeInfo(), begins, lengths, numOfClasses, output->specialBuffer(), output->specialShapeInfo());
                }
                else {
                    std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
                    auto packX = sd::ConstantTadHelper::getInstance().tadForDimensions(input->shapeInfo(), dimensions);
                    auto packZ = sd::ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(), dimensions);
                    auto inputTads = packX.specialShapeInfo();
                    auto inputTadOffsets = packX.specialOffsets();
                    auto outputTads = packZ.specialShapeInfo();
                    auto outputTadOffsets = packZ.specialOffsets();
                    dims.x = input->sizeAt(0);
                    output->assign(-DataTypeUtils::max<T>());
                    segmentMaxTadKernel<T,I><<<dims.x, dims.y, dims.z, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), inputTads, inputTadOffsets, reinterpret_cast<I*>(indices->specialBuffer()), begins, lengths, numOfClasses, output->specialBuffer(), output->specialShapeInfo(), outputTads, outputTadOffsets);
                }

            }
            // -------------------------------------------------------------------------------------------------------------- //
            void unsortedSegmentMaxFunctor(sd::LaunchContext* context, NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {
                NDArray::prepareSpecialUse({output}, {input, indices});
                output->nullify();
                BUILD_DOUBLE_SELECTOR(input->dataType(), indices->dataType(), unsortedSegmentMaxFunctor_, (context, input, indices, numOfClasses, output), NUMERIC_TYPES, INDEXING_TYPES);
                NDArray::registerSpecialUse({output}, {input, indices});
            }

            // -------------------------------------------------------------------------------------------------------------- //
            // segment max
            // -------------------------------------------------------------------------------------------------------------- //
            template <typename T, typename I>
            static __global__ void segmentMaxBPLinearKernel(void* inputBuf, Nd4jLong const*  inputShape, void* forwardOutput,
                                                            Nd4jLong const*  forwardShape, void* eps, Nd4jLong const*  epsShape, void* indicesBuf, Nd4jLong const*  indicesShape,
                                                            void* outputBuf, Nd4jLong const*  outputShape) {
                __shared__ T* x;
                __shared__ T* gradIn;
                __shared__ T* gradOut;
                __shared__ I* y;
                __shared__ T* z;
                __shared__ Nd4jLong xLen, gradLen;

                if (threadIdx.x == 0) {
                    xLen = shape::length(inputShape);
                    x = reinterpret_cast<T*>(inputBuf);
                    y = reinterpret_cast<I*>(indicesBuf);
                    z = reinterpret_cast<T*>(outputBuf);
                    gradIn = reinterpret_cast<T*>(forwardOutput);
                    gradOut = reinterpret_cast<T*>(eps);
                    gradLen = shape::length(epsShape);
                }
                __syncthreads();

                auto start = blockIdx.x * blockDim.x + threadIdx.x;
                auto step = gridDim.x * blockDim.x;

                for (auto e = start; e < xLen; e += step) {

                    auto zOffset = shape::getIndexOffset(e, outputShape);
                    auto xOffset = shape::getIndexOffset(e, inputShape);
                    auto yOffset = shape::getIndexOffset(e, indicesShape);
                    auto classIndex = y[yOffset];
                    auto gradOffsetI = shape::getIndexOffset(classIndex, forwardShape);
                    auto gradOffsetO = shape::getIndexOffset(classIndex, epsShape);

                    if (sd::math::nd4j_abs(gradIn[gradOffsetI] - x[xOffset]) <= T(1.e-6)) {
                        z[zOffset] = gradOut[gradOffsetO];
                    }
                }
            }

            // -------------------------------------------------------------------------------------------------------------- //
            template <typename T, typename I>
            static __global__ void segmentMaxBPTadKernel(void* inputBuf, Nd4jLong const*  inputShape, void* forwardOutput,
                                                         Nd4jLong const*  forwardShape, void* eps, Nd4jLong const*  epsShape, void* indicesBuf, Nd4jLong const*  indicesShape,
                                                         void* outputBuf, Nd4jLong const*  outputShape,Nd4jLong const*  inputTad,
                                                         Nd4jLong const*  inputOffsets, Nd4jLong const*  gradInTad, Nd4jLong const*  gradInOffsets,
                                                         Nd4jLong const*  gradOutTad, Nd4jLong const*  gradOutOffsets, Nd4jLong const*  outTad,
                                                         Nd4jLong const*  outOffsets) {
                __shared__ T* x;
                __shared__ T* gradIn;
                __shared__ T* gradOut;
                __shared__ I* y;
                __shared__ T* z;
                __shared__ Nd4jLong xLen, yLen, gradLen, currentLen;

                if (threadIdx.x == 0) {
                    xLen = shape::length(inputShape);
                    x = reinterpret_cast<T*>(inputBuf);
                    y = reinterpret_cast<I*>(indicesBuf);
                    z = reinterpret_cast<T*>(outputBuf);
                    yLen = shape::length(indicesShape);
                    gradOut = reinterpret_cast<T*>(eps);
                    gradIn = reinterpret_cast<T*>(forwardOutput);
                    gradLen = shape::length(epsShape);
                    currentLen = shape::length(outTad);
                }
                __syncthreads();

                for (auto i = blockIdx.x; i < yLen; i += gridDim.x) {
                    auto yIndex = shape::getIndexOffset(i, indicesShape);
                    auto segment = y[yIndex];
                    T* current = x + inputOffsets[i];
                    T* currentOut = z + outOffsets[i];
                    T* in = gradIn + gradInOffsets[segment];
                    T* outGrad = gradOut + gradOutOffsets[segment];

                    for (auto e = threadIdx.x; e < currentLen; e += blockDim.x) {
                        if (sd::math::nd4j_abs(in[e] - current[e]) <= T(1.e-6))
                            currentOut[e] = outGrad[e];
                    }
                }
            }
            // -------------------------------------------------------------------------------------------------------------- //
            template <typename T, typename I>
            int segmentMaxFunctorBP_(sd::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output) {
                //int numOfClasses = gradOut->sizeAt(0);
                // if input is a vector: (as if in doc sample)
                auto stream = context->getCudaStream();
                NDArray tempRes(gradOut->ordering(), gradOut->getShapeAsVector(), DataTypeUtils::fromT<T>(), context);//->shapeInfo(), context);
                segmentMaxFunctor_<T, I>(context, input, indices, &tempRes);
                NDArray::prepareSpecialUse({output}, {input, indices, gradOut, &tempRes});
                if (input->isVector()) {
                    Nd4jLong loop_size = input->lengthOf();
                    auto numOfClasses = gradOut->lengthOf(); //indices->e<Nd4jLong>(loop_size - 1);
                    segmentMaxBPLinearKernel<T,I><<<1 + gradOut->lengthOf(), input->lengthOf(), 256, *stream>>>(input->specialBuffer(), input->specialShapeInfo(),
                            tempRes.specialBuffer(), tempRes.specialShapeInfo(), gradOut->specialBuffer(), gradOut->specialShapeInfo(),
                            indices->specialBuffer(), indices->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo());
                }
                else {
                    std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
                    auto packX = sd::ConstantTadHelper::getInstance().tadForDimensions(input->shapeInfo(), dimensions);
                    auto packZ = sd::ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(), dimensions);
                    auto packGradIn = sd::ConstantTadHelper::getInstance().tadForDimensions(tempRes.shapeInfo(), dimensions);
                    auto packGradOut = sd::ConstantTadHelper::getInstance().tadForDimensions(gradOut->shapeInfo(), dimensions);
                    Nd4jLong const*  inputTads = packX.specialShapeInfo();
                    Nd4jLong const*  inputTadOffsets = packX.specialOffsets();
                    Nd4jLong const*  outputTads = packZ.specialShapeInfo();
                    Nd4jLong const*  outputTadOffsets = packZ.specialOffsets();
                    Nd4jLong const*  gradInTads = packGradIn.specialShapeInfo();
                    Nd4jLong const*  gradInTadOffsets = packGradIn.specialOffsets();
                    Nd4jLong const*  gradOutTads = packGradOut.specialShapeInfo();
                    Nd4jLong const*  gradOutTadOffsets = packGradOut.specialOffsets();

                    segmentMaxBPTadKernel<T,I><<<gradOut->lengthOf(), input->lengthOf(), 256, *stream>>>(input->specialBuffer(), input->specialShapeInfo(),
                            tempRes.specialBuffer(), tempRes.specialShapeInfo(), gradOut->specialBuffer(), gradOut->specialShapeInfo(),
                            indices->specialBuffer(), indices->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo(),
                            inputTads, inputTadOffsets, gradInTads, gradInTadOffsets, gradOutTads, gradOutTadOffsets,
                            outputTads, outputTadOffsets);
                }
                NDArray::registerSpecialUse({output}, {input, indices, gradOut, &tempRes});
                return Status::OK();
            }
            // -------------------------------------------------------------------------------------------------------------- //
            int segmentMaxFunctorBP(sd::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output) {
                NDArray::prepareSpecialUse({output}, {input, indices, gradOut});
                BUILD_DOUBLE_SELECTOR(output->dataType(), indices->dataType(), return segmentMaxFunctorBP_, (context, input,
                        indices, gradOut, output), FLOAT_TYPES, INDEXING_TYPES);
                NDArray::registerSpecialUse({output}, {input, indices, gradOut});
            }

            // -------------------------------------------------------------------------------------------------------------- //
            template <typename T, typename I>
            static int unsortedSegmentMaxFunctorBP_(sd::LaunchContext* context, NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
                //int numOfClasses = gradOut->sizeAt(0);
                // if input is a vector: (as if in doc sample)
                auto stream = context->getCudaStream();
                NDArray tempRes(gradOut->ordering(), gradOut->getShapeAsVector(), DataTypeUtils::fromT<T>(), context);//->shapeInfo(), context);
                unsortedSegmentMaxFunctor_<T, I>(context, input, indices, numOfClasses, &tempRes);
                NDArray::prepareSpecialUse({output}, {input, indices, gradOut, &tempRes});
                if (input->isVector()) {
                    Nd4jLong loop_size = input->lengthOf();
                    auto numOfClasses = gradOut->lengthOf(); //indices->e<Nd4jLong>(loop_size - 1);
                    segmentMaxBPLinearKernel<T,I><<<gradOut->lengthOf(), input->lengthOf(), 256, *stream>>>(input->specialBuffer(), input->specialShapeInfo(),
                            tempRes.specialBuffer(), tempRes.specialShapeInfo(), gradOut->specialBuffer(), gradOut->specialShapeInfo(),
                            indices->specialBuffer(), indices->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo());
                }
                else {
                    std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
                    auto packX = sd::ConstantTadHelper::getInstance().tadForDimensions(input->shapeInfo(), dimensions);
                    auto packZ = sd::ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(), dimensions);
                    auto packGradIn = sd::ConstantTadHelper::getInstance().tadForDimensions(tempRes.shapeInfo(), dimensions);
                    auto packGradOut = sd::ConstantTadHelper::getInstance().tadForDimensions(gradOut->shapeInfo(), dimensions);
                    Nd4jLong const*  inputTads = packX.specialShapeInfo();
                    Nd4jLong const*  inputTadOffsets = packX.specialOffsets();
                    Nd4jLong const*  outputTads = packZ.specialShapeInfo();
                    Nd4jLong const*  outputTadOffsets = packZ.specialOffsets();
                    Nd4jLong const*  gradInTads = packGradIn.specialShapeInfo();
                    Nd4jLong const*  gradInTadOffsets = packGradIn.specialOffsets();
                    Nd4jLong const*  gradOutTads = packGradOut.specialShapeInfo();
                    Nd4jLong const*  gradOutTadOffsets = packGradOut.specialOffsets();

                    segmentMaxBPTadKernel<T,I><<<gradOut->lengthOf(), input->lengthOf(), 256, *stream>>>(input->specialBuffer(), input->specialShapeInfo(),
                            tempRes.specialBuffer(), tempRes.specialShapeInfo(), gradOut->specialBuffer(), gradOut->specialShapeInfo(),
                            indices->specialBuffer(), indices->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo(),
                            inputTads, inputTadOffsets, gradInTads, gradInTadOffsets, gradOutTads, gradOutTadOffsets,
                            outputTads, outputTadOffsets);
                }
                NDArray::registerSpecialUse({output}, {input, indices, gradOut, &tempRes});
                return Status::OK();
            }
            // -------------------------------------------------------------------------------------------------------------- //
            int unsortedSegmentMaxFunctorBP(sd::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
                NDArray::prepareSpecialUse({output}, {input, indices, gradOut});
                BUILD_DOUBLE_SELECTOR(output->dataType(), indices->dataType(), return unsortedSegmentMaxFunctorBP_, (context, input, indices, gradOut, numOfClasses, output), FLOAT_TYPES, INDEXING_TYPES);
                NDArray::registerSpecialUse({output}, {input, indices, gradOut});
            }
        }
    }
}