#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 * Copyright (c) 2019 Konduit K.K.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma (iuriish@yahoo.com), created on 20.04.2018
//


#include<ops/declarable/helpers/transforms.h>
#include <array/ResultSet.h>
#include <helpers/ShapeUtils.h>
#include <numeric>
#include <array/NDArrayFactory.h>
#include <helpers/TAD.h>
#include <exceptions/cuda_exception.h>
#include <helpers/PointersManager.h>
#include <helpers/ConstantTadHelper.h>

namespace sd 	  {
namespace ops 	  {
namespace helpers {

///////////////////////////////////////////////////////////////////
template<typename T>
__global__ static void invertPermutationCuda(const void* vx, const Nd4jLong* xShapeInfo, void* vz, const Nd4jLong* zShapeInfo) {

    const T* x = reinterpret_cast<const T*>(vx);
          T* z = reinterpret_cast<T*>(vz);

    __shared__ Nd4jLong len, totalThreads;

    if (threadIdx.x == 0) {

        len  = shape::length(xShapeInfo);
        totalThreads = gridDim.x * blockDim.x;
    }

    __syncthreads();

    const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (Nd4jLong i = tid; i < len; i += totalThreads) {

        const auto xOffset = shape::getIndexOffset(i, xShapeInfo);
        const Nd4jLong index = x[xOffset];
        const auto zOffset = shape::getIndexOffset(index, zShapeInfo);
        z[zOffset] = i;
    }
}

///////////////////////////////////////////////////////////////////
template<typename T>
__host__ static void invertPermutationCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const hipStream_t *stream,
                                                   const void* vx, const Nd4jLong* xShapeInfo, void* vz, const Nd4jLong* zShapeInfo) {

    invertPermutationCuda<T><<<blocksPerGrid, threadsPerBlock, 1024, *stream>>>(vx, xShapeInfo, vz, zShapeInfo);
}

////////////////////////////////////////////////////////////////////////
void invertPermutation(sd::LaunchContext* context, const NDArray& input, NDArray& output) {

    const int threadsPerBlock = MAX_NUM_THREADS;
    const int blocksPerGrid = (input.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;

    PointersManager manager(context, "invertPermutation");

    NDArray::prepareSpecialUse({&output}, {&input});
    BUILD_SINGLE_SELECTOR(input.dataType(), invertPermutationCudaLauncher, (blocksPerGrid, threadsPerBlock, context->getCudaStream(), input.specialBuffer(), input.specialShapeInfo(), output.specialBuffer(), output.specialShapeInfo()), LIBND4J_TYPES);
    NDArray::registerSpecialUse({&output}, {&input});

    manager.synchronize();
}

//////////////////////////////////////////////////////////////////////////
template<typename T>
__global__ static void traceCuda(const void* vx, const Nd4jLong* xShapeInfo, void* vz, const Nd4jLong* zShapeInfo, const uint diagLen) {

    const auto x = reinterpret_cast<const T*>(vx);
          auto z = reinterpret_cast<T*>(vz);

    __shared__ T sharedMem[CUDA_BLOCK_SIZE];
    __shared__ int xRank, zRank;        // xRank = zRank + 2
    __shared__ Nd4jLong xLen, zLen;

    if (threadIdx.x == 0) {
        xRank = shape::rank(xShapeInfo);
        zRank = shape::rank(zShapeInfo);
        xLen = shape::length(xShapeInfo);
        zLen = shape::length(zShapeInfo);   // corresponds to number of matrices

    }
    __syncthreads();

    Nd4jLong coords[MAX_RANK];

    for (uint m = blockIdx.x; m < zLen; m += gridDim.x) {   // one block per each element of z, that is per each matrix

        shape::index2coords(m, zShapeInfo, coords);
        const auto zOffset = shape::getOffset(zShapeInfo, coords);

        sharedMem[threadIdx.x] = 0;

          for (uint i = threadIdx.x; i < diagLen; i += blockDim.x) {

            coords[zRank] = coords[zRank + 1] = i;
            const auto xOffset = shape::getOffset(xShapeInfo, coords);
            sharedMem[threadIdx.x] += x[xOffset];
          }

          __syncthreads();

        // aggregate sum
        for (Nd4jLong activeThreads = blockDim.x / 2; activeThreads > 0; activeThreads /= 2) {
            if (threadIdx.x < activeThreads)
                sharedMem[threadIdx.x] += sharedMem[threadIdx.x + activeThreads];
            __syncthreads();
        }

        if (threadIdx.x == 0)
            z[zOffset] = *sharedMem;
        __syncthreads();
    }

}

///////////////////////////////////////////////////////////////////
template<typename T>
static void traceCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream,
                              const void *vx, const Nd4jLong *xShapeInfo,
                                    void *vz, const Nd4jLong *zShapeInfo,
                                    const uint diagLen) {

    traceCuda<T><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(vx, xShapeInfo, vz, zShapeInfo, diagLen);
}


///////////////////////////////////////////////////////////////////
void trace(sd::LaunchContext* context, const NDArray& input, NDArray& output) {

    PointersManager manager(context, "trace");

    const uint diagLen = input.sizeAt(-1) < input.sizeAt(-2) ? input.sizeAt(-1) : input.sizeAt(-2);
    const int threadsPerBlock = CUDA_BLOCK_SIZE;
    const int blocksPerGrid = (output.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;
    const int sharedMem = 1024;

    NDArray::prepareSpecialUse({&output}, {&input});
    BUILD_SINGLE_SELECTOR(input.dataType(), traceCudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, context->getCudaStream(), input.specialBuffer(), input.specialShapeInfo(), output.specialBuffer(), output.specialShapeInfo(), diagLen), LIBND4J_TYPES);
    NDArray::registerSpecialUse({&output}, {&input});

    manager.synchronize();
}

///////////////////////////////////////////////////////////////////
template<typename T>
__global__ static void triuBPCuda(const void* vx, const Nd4jLong* xShapeInfo, void* vz, const Nd4jLong* zShapeInfo, const int diag) {

    // x and z have same shapes
    const auto x = reinterpret_cast<const T*>(vx);  // gradO
          auto z = reinterpret_cast<T*>(vz);        // gradI

    __shared__ int rank, areSameOffsets;
    __shared__ Nd4jLong len, totalThreads;  // xLen = zLen

    if (threadIdx.x == 0) {
        areSameOffsets = shape::haveSameShapeAndStrides(xShapeInfo, zShapeInfo);
        rank = shape::rank(xShapeInfo);
        len  = shape::length(zShapeInfo);
        totalThreads = gridDim.x * blockDim.x;
    }

    __syncthreads();

    Nd4jLong coords[MAX_RANK];

    const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (Nd4jLong i = tid; i < len; i += totalThreads) {

        shape::index2coords(i, zShapeInfo, coords);

        const auto zOffset = shape::getOffset(zShapeInfo, coords);

        if((coords[rank - 2] + diag > coords[rank - 1]))    // row + diag > col
            z[zOffset] = 0;
        else
            z[zOffset] = x[areSameOffsets ? zOffset : shape::getOffset(xShapeInfo, coords)];
    }
}

///////////////////////////////////////////////////////////////////
template<typename T>
static void triuBPCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream,  const void* vx, const Nd4jLong* xShapeInfo, void* vz, const Nd4jLong* zShapeInfo, const int diag) {

    triuBPCuda<T><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(vx, xShapeInfo, vz, zShapeInfo, diag);
}

///////////////////////////////////////////////////////////////////
void triuBP(sd::LaunchContext* context, const NDArray& input, const NDArray& gradO, NDArray& gradI, const int diagonal) {

    const int threadsPerBlock = MAX_NUM_THREADS / 4;
    const int blocksPerGrid = (gradO.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;
    const int sharedMem = threadsPerBlock * sizeof(int) * gradO.rankOf() + 128;

    PointersManager manager(context, "triuBP");

    NDArray::prepareSpecialUse({&gradI}, {&gradO});
    BUILD_SINGLE_SELECTOR(gradI.dataType(), triuBPCudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, context->getCudaStream(), gradO.specialBuffer(), gradO.specialShapeInfo(), gradI.specialBuffer(), gradI.specialShapeInfo(), diagonal), LIBND4J_TYPES);
    NDArray::registerSpecialUse({&gradI}, {&gradO});

    manager.synchronize();
}

///////////////////////////////////////////////////////////////////
template<typename T>
__global__ static void tileBPCuda(const void* vx, const Nd4jLong* xShapeInfo, void* vz, const Nd4jLong* zShapeInfo, Nd4jLong* globMem) {

    // x and z have same shapes
    const auto x = reinterpret_cast<const T*>(vx);  // gradO
          auto z = reinterpret_cast<T*>(vz);        // gradI

    __shared__ int xRank, zRank;                // xRank >= zRank
    __shared__ Nd4jLong numOfXOffsets, zLen, totalThreads;  // xLen >= zLen

    if (threadIdx.x == 0) {
        xRank = shape::rank(zShapeInfo);
        zLen  = shape::length(zShapeInfo);
        numOfXOffsets = shape::length(xShapeInfo) / zLen;

        totalThreads = gridDim.x * blockDim.x;
    }

    __syncthreads();

    const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

    int memBuff[MAX_RANK * 2];
    auto xOffsets = globMem + tid * numOfXOffsets;

    for (Nd4jLong i = tid; i < zLen; i += totalThreads) {

        const auto zOffset = shape::getIndexOffset(i, zShapeInfo);

        shape::outerArrayOffsets(xOffsets, i, xShapeInfo, zShapeInfo, memBuff);

        z[zOffset] = x[xOffsets[0]];                    // first offset
        for (Nd4jLong j = 1; j < numOfXOffsets; ++j)    // rest offsets
            z[zOffset] += x[xOffsets[j]];
    }
}

///////////////////////////////////////////////////////////////////
template<typename T>
static void tileBPCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream,  const void* vx, const Nd4jLong* xShapeInfo, void* vz, const Nd4jLong* zShapeInfo, Nd4jLong* globMem) {

    tileBPCuda<T><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(vx, xShapeInfo, vz, zShapeInfo, globMem);
}


//////////////////////////////////////////////////////////////////////////
void tileBP(sd::LaunchContext * context, const NDArray& gradO /*input*/, NDArray& gradI /*output*/, const std::vector<Nd4jLong> reps) {

    NDArray memBuff('c', gradO.getShapeAsVector(), sd::DataType::INT64, context);        // empty auxiliary array for storing device memory which will be used in kernel calculations

    const int threadsPerBlock = MAX_NUM_THREADS / 4;
    const int blocksPerGrid = (gradI.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;
    const int sharedMem = threadsPerBlock * sizeof(int) * 2 * gradO.rankOf() + 128;

    PointersManager manager(context, "tileBP");

    NDArray::prepareSpecialUse({&gradI}, {&gradO, &memBuff});
    BUILD_SINGLE_SELECTOR(gradI.dataType(), tileBPCudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, context->getCudaStream(), gradO.specialBuffer(), gradO.specialShapeInfo(), gradI.specialBuffer(), gradI.specialShapeInfo(), reinterpret_cast<Nd4jLong*>(memBuff.specialBuffer())), FLOAT_TYPES);
    NDArray::registerSpecialUse({&gradI}, {&gradO, &memBuff});

    manager.synchronize();
}

    //////////////////////////////////////////////////////////////////////////
    void eye(sd::LaunchContext * context, NDArray& output) {

        output.setIdentity();
    }

}
}
}

