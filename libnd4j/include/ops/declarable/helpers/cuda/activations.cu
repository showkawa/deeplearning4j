#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma (iuriish@yahoo.com), created on 19.04.2018
// @author raver119@gmail.com
//

#include <system/op_boilerplate.h>
#include <ops/declarable/helpers/activations.h>
#include <helpers/ShapeUtils.h>
#include <numeric>
#include <helpers/PointersManager.h>
#include <helpers/ConstantTadHelper.h>

namespace sd    {
namespace ops     {
namespace helpers {

///////////////////////////////////////////////////////////////////
template<typename X, typename Y>
__global__ void preluCuda(const void *vx, const Nd4jLong *xShapeInfo,
		   			 	  const void *vy, const Nd4jLong *yShapeInfo,
						        void *vz) {

	const auto x = reinterpret_cast<const X*>(vx);
	const auto y = reinterpret_cast<const Y*>(vy);
		  auto z = reinterpret_cast<X*>(vz);

	__shared__ Nd4jLong xzLen;
	__shared__ int xzRank, yRank;

	if (threadIdx.x == 0) {
		xzLen = shape::length(xShapeInfo);

		xzRank = shape::rank(xShapeInfo);
		yRank  = shape::rank(yShapeInfo);
	}
	__syncthreads();

	const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
	int coords[MAX_RANK];

	for (int i = tid; i < xzLen; i += blockDim.x * gridDim.x) {
    	shape::index2coords(i, xShapeInfo, coords);

		const auto xzOffset = shape::getOffset(xShapeInfo, coords);
		const auto xVal = x[xzOffset];

		if(xVal < 0) {
			for (uint j = 0; j < yRank; ++j)
				if(yShapeInfo[j + 1] == 1)
					coords[j + 1] = 0;

			z[xzOffset] = xVal * y[shape::getOffset(yShapeInfo, coords + 1)];
		}
		else
			z[xzOffset] = xVal;
	}
}

///////////////////////////////////////////////////////////////////
template<typename X, typename Y>
linkage void preluCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream, const void *vx, const Nd4jLong *xShapeInfo, const void *vy, const Nd4jLong *yShapeInfo, void *vz) {
	preluCuda<X, Y><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(vx, xShapeInfo, vy, yShapeInfo, vz);
}

///////////////////////////////////////////////////////////////////
void prelu(sd::LaunchContext * context, const NDArray& input, const NDArray& alpha, NDArray& output) {

	PointersManager manager(context, "prelu");

    const int threadsPerBlock = 256;
    const int blocksPerGrid = 512;
    const int sharedMem = 512;

	const auto xType = input.dataType();
	const auto yType = alpha.dataType();

	NDArray::prepareSpecialUse({&output}, {&input, &alpha});
	BUILD_SINGLE_SELECTOR_TWICE(xType, preluCudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, context->getCudaStream(), input.specialBuffer(), input.specialShapeInfo(), alpha.specialBuffer(), alpha.specialShapeInfo(), output.specialBuffer()), FLOAT_TYPES);
	NDArray::registerSpecialUse({&output}, {&input, &alpha});

	manager.synchronize();
}

///////////////////////////////////////////////////////////////////
template<typename X, typename Y>
__global__ linkage void preluBPCuda(const void *vIn,    const Nd4jLong *inShapeInfo,
								   const void *vAlpha, const Nd4jLong *alphaShapeInfo,
								   const void *vdLdO,  const Nd4jLong *dLdOShapeInfo,
										 void *vdLdI,  const Nd4jLong *dLdIShapeInfo,
										 void *vdLdA,  const Nd4jLong *dLdAShapeInfo) {

	const auto in    = reinterpret_cast<const X*>(vIn);
	const auto alpha = reinterpret_cast<const Y*>(vAlpha);
	const auto dLdO  = reinterpret_cast<const Y*>(vdLdO);
		  auto dLdI  = reinterpret_cast<Y*>(vdLdI);
		  auto dLdA  = reinterpret_cast<Y*>(vdLdA);

	__shared__ Nd4jLong inLen, totalThreads;
	__shared__ int inRank, alphaRank;

	if (threadIdx.x == 0) {
		inLen = shape::length(inShapeInfo);
		totalThreads = gridDim.x * blockDim.x;

		inRank     = shape::rank(inShapeInfo);
		alphaRank  = shape::rank(alphaShapeInfo);
	}
	__syncthreads();

	const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
	int coords[MAX_RANK];

	for (int i = tid; i < inLen; i += totalThreads) {
    	shape::index2coords(i, inShapeInfo, coords);

		const auto inOffset   = shape::getOffset(inShapeInfo, coords);
		const auto dLdOOffset = shape::getOffset(dLdOShapeInfo, coords);
		const auto dLdIOffset = shape::getOffset(dLdIShapeInfo, coords);

		const auto xVal = in[inOffset];
		const auto grO  = dLdO[dLdOOffset];

		if(xVal < 0) {

			for (uint j = 0; j < alphaRank; ++j)
				if(alphaShapeInfo[j + 1] == 1)
					coords[j + 1] = 0;

			const auto alphaOffset = shape::getOffset(alphaShapeInfo, coords + 1);
			const auto dLdAOffset  = shape::getOffset(dLdAShapeInfo, coords + 1);

			dLdI[dLdIOffset] =  grO * alpha[alphaOffset];

			sd::math::atomics::nd4j_atomicAdd<Y>(&dLdA[dLdAOffset], static_cast<Y>(grO * xVal));
		}
		else
			dLdI[dLdIOffset] = grO;
	}
}

//////////////////////////////////////////////////////////////////////////
template<typename X, typename Y>
__host__ linkage void preluBPCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream, const void *vIn, const Nd4jLong *inShapeInfo, const void *vAlpha, const Nd4jLong *alphaShapeInfo, const void *vdLdO,  const Nd4jLong *dLdOShapeInfo, void *vdLdI,  const Nd4jLong *dLdIShapeInfo, void *vdLdA,  const Nd4jLong *dLdAShapeInfo) {

	preluBPCuda<X, Y><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(vIn, inShapeInfo, vAlpha, alphaShapeInfo, vdLdO, dLdOShapeInfo, vdLdI, dLdIShapeInfo, vdLdA, dLdAShapeInfo);
}

//////////////////////////////////////////////////////////////////////////
void preluBP(sd::LaunchContext* context, const NDArray& input, const NDArray& alpha, const NDArray& dLdO, NDArray& dLdI, NDArray& dLdA) {
    dLdA.nullify();

	PointersManager manager(context, "preluBP");

    const int threadsPerBlock = 256;
    const int blocksPerGrid = 512;
    const int sharedMem = 512;

	const auto xType = input.dataType();
	const auto zType = alpha.dataType();

	NDArray::prepareSpecialUse({&dLdI, &dLdA}, {&input, &alpha, &dLdO});
	BUILD_SINGLE_SELECTOR_TWICE(xType, preluBPCudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, context->getCudaStream(), input.specialBuffer(), input.specialShapeInfo(), alpha.specialBuffer(), alpha.specialShapeInfo(), dLdO.specialBuffer(),  dLdO.specialShapeInfo(), dLdI.specialBuffer(), dLdI.specialShapeInfo(), dLdA.specialBuffer(), dLdA.specialShapeInfo()), FLOAT_TYPES);
	NDArray::registerSpecialUse({&dLdI, &dLdA}, {&input, &alpha, &dLdO});

	manager.synchronize();
}


///////////////////////////////////////////////////////////////////
template<typename T>
__device__ void softMaxForVectorCuda(const void *vx, const Nd4jLong *xShapeInfo, void *vz, const Nd4jLong *zShapeInfo) {

	// logic of this kernel is based on assumption gridDim = 1

	const auto x = reinterpret_cast<const T*>(vx);
		  auto z = reinterpret_cast<T*>(vz);

	__shared__ Nd4jLong  len;
	__shared__ int numOfIters;
	__shared__ T shmem[CUDA_BLOCK_SIZE];

	if (threadIdx.x == 0) {
		len = shape::length(xShapeInfo);
		numOfIters = (len + blockDim.x - 1) / blockDim.x;   // ceil (len / blockDim.x)
	}
	__syncthreads();

	T temp = -DataTypeUtils::max<T>();	// set start value to compare with at first iteration, FIXME: what if T is unsigned ??

	// ************ evaluate max element in input array x ************ //
	for (int i = 0; i < numOfIters; ++i) {

		const Nd4jLong elemIdx = i * blockDim.x + threadIdx.x;
		if(elemIdx < len) {
			const Nd4jLong xOffset = shape::getIndexOffset(elemIdx, xShapeInfo);
			shmem[threadIdx.x] = (threadIdx.x != 0) ? x[xOffset] : sd::math::nd4j_max<T>(x[xOffset], temp);	// take into account max element evaluated on previous iteration and stored in temp
		}
		else
			shmem[threadIdx.x] = -DataTypeUtils::max<T>();	// FIXME: what if T is unsigned ??

		__syncthreads();

		for (int s = blockDim.x / 2; s > 0; s /= 2) {
			if(threadIdx.x < s)
				shmem[threadIdx.x] = sd::math::nd4j_max<T>(shmem[threadIdx.x], shmem[threadIdx.x + s]);
			__syncthreads();
		}

		temp = shmem[0];	// save max value calculated at current iteration
	}

	const T max = temp;
	temp = 0;

	// ************ evaluate value of exp(x[offset] - max) per each element, store it to shared memory shmem ************ //
	// at the same evaluate sum of exponents, sum will be stored in shmem[0]
	for (int i = 0; i < numOfIters; ++i) {

		const Nd4jLong elemIdx = i * blockDim.x + threadIdx.x;
		if(elemIdx < len) {
			const Nd4jLong xOffset = shape::getIndexOffset(elemIdx, xShapeInfo);
			const Nd4jLong zOffset = shape::getIndexOffset(elemIdx, zShapeInfo);
			z[zOffset] = sd::math::nd4j_exp<T, T>(x[xOffset] - max);
			shmem[threadIdx.x] = (threadIdx.x != 0) ? z[zOffset] : (z[zOffset] + temp); // take into account sum element evaluated on previous iteration and stored in temp
		}
		else
			shmem[threadIdx.x] = 0;

		__syncthreads();

		for (int s = blockDim.x / 2; s > 0; s /= 2) {
			if(threadIdx.x < s)
				shmem[threadIdx.x] += shmem[threadIdx.x + s];
			__syncthreads();
		}

		temp = shmem[0];	// save sum calculated at current iteration
	}

	// ************ evaluate z[offset] / sum  ************ //
	for (int i = 0; i < numOfIters; ++i) {
		const Nd4jLong elemIdx = i * blockDim.x + threadIdx.x;
		if(elemIdx >= len) continue;
		const Nd4jLong zOffset = shape::getIndexOffset(elemIdx, zShapeInfo);
		z[zOffset] /= shmem[0];
	}
}

template<typename T>
__global__ void softMaxForVectorCudaGlobal(const void *vx, const Nd4jLong *xShapeInfo, void *vz, const Nd4jLong *zShapeInfo) {

	softMaxForVectorCuda<T>(vx, xShapeInfo, vz, zShapeInfo);
}

///////////////////////////////////////////////////////////////////
template <typename T>
linkage void softMaxForVectorCudaLauncher(const hipStream_t* stream, const void *vx, const Nd4jLong *xShapeInfo, void *vz, const Nd4jLong *zShapeInfo) {

	softMaxForVectorCudaGlobal<T><<<1, CUDA_BLOCK_SIZE, 1024, *stream>>>(vx, xShapeInfo, vz, zShapeInfo);
}

///////////////////////////////////////////////////////////////////
template<typename T>
__global__ static void softMaxCuda(const void* vx, const Nd4jLong *xTadShapeInfo, const Nd4jLong *xOffsets,
                                         void* vz, const Nd4jLong *zTadShapeInfo, const Nd4jLong *zOffsets) {

    const auto x = reinterpret_cast<const T*>(vx);
          auto z = reinterpret_cast<T*>(vz);

    const auto* xTad = x + xOffsets[blockIdx.x];
          auto* zTad = z + zOffsets[blockIdx.x];

    softMaxForVectorCuda<T>(xTad, xTadShapeInfo, zTad, zTadShapeInfo);
}

///////////////////////////////////////////////////////////////////
template<typename T>
static void softMaxCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream,
                                const void* vx, const Nd4jLong *xTadShapeInfo, const Nd4jLong *xOffsets,
                                	  void* vz, const Nd4jLong *zTadShapeInfo, const Nd4jLong *zOffsets) {

    softMaxCuda<T><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(vx, xTadShapeInfo, xOffsets, vz, zTadShapeInfo, zOffsets);
}


//////////////////////////////////////////////////////////////////////////
void softmax(sd::LaunchContext * context, const NDArray& input, NDArray& output, const int dimension) {

	if(!input.isActualOnDeviceSide()) input.syncToDevice();
	const int rank = input.rankOf();

	PointersManager manager(context, "helpers::softmax");

	if(input.isVector()) {

		if(rank == 1 || input.sizeAt(dimension) != 1) {
			NDArray::prepareSpecialUse({&output}, {&input});
			BUILD_SINGLE_SELECTOR(input.dataType(), softMaxForVectorCudaLauncher, (context->getCudaStream(), input.specialBuffer(), input.specialShapeInfo(), output.specialBuffer(), output.specialShapeInfo()), FLOAT_TYPES);
			NDArray::registerSpecialUse({&output}, {&input});
		}
		else
			output = 1.;
	}
	else {

		auto packX = sd::ConstantTadHelper::getInstance().tadForDimensions(input.shapeInfo(), {dimension});
        auto packZ = sd::ConstantTadHelper::getInstance().tadForDimensions(output.shapeInfo(), {dimension});

        const int threadsPerBlock = CUDA_BLOCK_SIZE;
        const int blocksPerGrid = packZ.numberOfTads();
        const int sharedMem = 1024;

        NDArray::prepareSpecialUse({&output}, {&input});
    	BUILD_SINGLE_SELECTOR(input.dataType(), softMaxCudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, context->getCudaStream(), input.specialBuffer(), packX.specialShapeInfo(), packX.specialOffsets(), output.specialBuffer(), packZ.specialShapeInfo(), packZ.specialOffsets()), FLOAT_TYPES);
    	NDArray::registerSpecialUse({&output}, {&input});

		// auto maxAlongDim = const_cast<NDArray&>(input).reduceAlongDimension(reduce::Max, {dimension}, true);
		// (input - maxAlongDim).applyTransform(transform::Exp, &output); // output contains exponents temporarily
		// auto sumAlongDim = output.reduceAlongDimension(reduce::Sum, {dimension}, true);
		// output /= sumAlongDim;
		// input.tickReadDevice();
	}


	manager.synchronize();

	output.tickWriteDevice();
}

///////////////////////////////////////////////////////////////////
template<typename T>
__global__  void logSoftMaxForVectorCuda(const void *vx, const Nd4jLong *xzShapeInfo, void *vz) {

	// logic of this kernel is based on assumption gridDim = 1

	const auto x = reinterpret_cast<const T*>(vx);
		  auto z = reinterpret_cast<T*>(vz);

	__shared__ Nd4jLong  len;
	__shared__ int numOfIters;
	__shared__ T shmem[CUDA_BLOCK_SIZE];

	if (threadIdx.x == 0) {
		len = shape::length(xzShapeInfo);
		numOfIters = (len + blockDim.x - 1) / blockDim.x;   // ceil (len / blockDim.x)
	}
	__syncthreads();

	T temp = -DataTypeUtils::max<T>();	// set start value to compare with at first iteration, FIXME: what if T is unsigned ??

	// ************ evaluate max element in input array x ************ //
	for (int i = 0; i < numOfIters; ++i) {

		const Nd4jLong elemIdx = i * blockDim.x + threadIdx.x;
		if(elemIdx < len) {
			const Nd4jLong offset = shape::getIndexOffset(elemIdx, xzShapeInfo);
			shmem[threadIdx.x] = (threadIdx.x != 0) ? x[offset] : sd::math::nd4j_max<T>(x[offset], temp);	// take into account max element evaluated on previous iteration and stored in temp
		}
		else
			shmem[threadIdx.x] = -DataTypeUtils::max<T>();	// FIXME: what if T is unsigned ??

		__syncthreads();

		for (int s = blockDim.x / 2; s > 0; s /= 2) {
			if(threadIdx.x < s)
				shmem[threadIdx.x] = sd::math::nd4j_max<T>(shmem[threadIdx.x], shmem[threadIdx.x + s]);
			__syncthreads();
		}

		temp = shmem[0];	// save max value calculated at current iteration
	}

	const T max = temp;
	temp = 0;

	// ************ evaluate value of exp(x[offset] - max) per each element, store it to shared memory shmem ************ //
	// at the same time evaluate sum of exponents, sum will be stored in shmem[0]
	for (int i = 0; i < numOfIters; ++i) {

		const Nd4jLong elemIdx = i * blockDim.x + threadIdx.x;
		if(elemIdx < len) {
			const Nd4jLong offset = shape::getIndexOffset(elemIdx, xzShapeInfo);
			z[offset] = sd::math::nd4j_exp<T, T>(x[offset] - max);
			shmem[threadIdx.x] = (threadIdx.x != 0) ? z[offset] : (z[offset] + temp); // take into account sum element evaluated on previous iteration and stored in temp
		}
		else
			shmem[threadIdx.x] = 0;

		__syncthreads();

		for (int s = blockDim.x / 2; s > 0; s /= 2) {
			if(threadIdx.x < s)
				shmem[threadIdx.x] += shmem[threadIdx.x + s];
			__syncthreads();
		}

		temp = shmem[0];	// save sum calculated at current iteration
	}

	// ************ evaluate log(z[offset] / sum)  ************ //
	for (int i = 0; i < numOfIters; ++i) {
		const Nd4jLong elemIdx = i * blockDim.x + threadIdx.x;
		if(elemIdx >= len) continue;
		const Nd4jLong offset = shape::getIndexOffset(elemIdx, xzShapeInfo);
		z[offset] = sd::math::nd4j_log<T,T>(z[offset] / shmem[0]);
	}
}

///////////////////////////////////////////////////////////////////
template <typename T>
linkage void logSoftMaxForVectorCudaLauncher(const hipStream_t* stream, const void *vx, const Nd4jLong *xzShapeInfo, void *vz) {

	logSoftMaxForVectorCuda<T><<<1, CUDA_BLOCK_SIZE, 1024, *stream>>>(vx, xzShapeInfo, vz);
}

//////////////////////////////////////////////////////////////////////////
void logSoftmax(sd::LaunchContext * context, const NDArray& input, NDArray& output, const int dimension) {

	if(!input.isActualOnDeviceSide()) input.syncToDevice();
	const int rank = input.rankOf();

	if(input.isVector()) {

		if(rank == 1 || input.sizeAt(dimension) != 1) {
			BUILD_SINGLE_SELECTOR(input.dataType(), logSoftMaxForVectorCudaLauncher, (context->getCudaStream(), input.specialBuffer(), input.specialShapeInfo(), output.specialBuffer()), FLOAT_TYPES);
			input.tickReadDevice();
		}
		else
			output = 0.;
	}
	else {

		auto maxAlongDim = const_cast<NDArray&>(input).reduceAlongDimension(reduce::Max, {dimension}, true);
		(input - maxAlongDim).applyTransform(transform::Exp, output); // output contains exponents temporarily
		auto sumAlongDim = output.reduceAlongDimension(reduce::Sum, {dimension}, true);
		output /= sumAlongDim;
		output.applyTransform(transform::Log, output);
		input.tickReadDevice();
	}

	PointersManager manager(context, "helpers::logSoftmax");
	manager.synchronize();

	output.tickWriteDevice();
}

///////////////////////////////////////////////////////////////////
template<typename T>
__global__ linkage void softMaxDerivForVectorCuda(const void *vx, const Nd4jLong *xzShapeInfo, void *vz) {

	// logic of this kernel is based on assumption gridDim = 1

	const auto x = reinterpret_cast<const T*>(vx);
		  auto z = reinterpret_cast<T*>(vz);

	__shared__ Nd4jLong  len;
	__shared__ int numOfIters;
	__shared__ T shmem[CUDA_BLOCK_SIZE];

	if (threadIdx.x == 0) {
		len = shape::length(xzShapeInfo);
		numOfIters = (len + blockDim.x - 1) / blockDim.x;   // ceil (len / blockDim.x)
	}
	__syncthreads();

	T temp = -DataTypeUtils::max<T>();	// set start value to compare with at first iteration, FIXME: what if T is unsigned ??

	// ************ evaluate max element in input array x ************ //
	for (int i = 0; i < numOfIters; ++i) {

		const Nd4jLong elemIdx = i * blockDim.x + threadIdx.x;
		if(elemIdx < len) {
			const Nd4jLong offset = shape::getIndexOffset(elemIdx, xzShapeInfo);
			shmem[threadIdx.x] = (threadIdx.x != 0) ? x[offset] : sd::math::nd4j_max<T>(x[offset], temp);	// take into account max element evaluated on previous iteration and stored in temp
		}
		else
			shmem[threadIdx.x] = -DataTypeUtils::max<T>();	// FIXME: what if T is unsigned ??

		__syncthreads();

		for (int s = blockDim.x / 2; s > 0; s /= 2) {
			if(threadIdx.x < s)
				shmem[threadIdx.x] = sd::math::nd4j_max<T>(shmem[threadIdx.x], shmem[threadIdx.x + s]);
			__syncthreads();
		}

		temp = shmem[0];	// save max value calculated at current iteration
	}

	const T max = temp;
	temp = 0;

	// ************ evaluate value of exp(x[offset] - max) per each element, store it to shared memory shmem ************ //
	// at the same evaluate sum of exponents, sum will be stored in shmem[0]
	for (int i = 0; i < numOfIters; ++i) {

		const Nd4jLong elemIdx = i * blockDim.x + threadIdx.x;
		if(elemIdx < len) {
			const Nd4jLong offset = shape::getIndexOffset(elemIdx, xzShapeInfo);
			z[offset] = sd::math::nd4j_exp<T, T>(x[offset] - max);
			shmem[threadIdx.x] = (threadIdx.x != 0) ? z[offset] : (z[offset] + temp); // take into account sum element evaluated on previous iteration and stored in temp
		}
		else
			shmem[threadIdx.x] = 0;

		__syncthreads();

		for (int s = blockDim.x / 2; s > 0; s /= 2) {
			if(threadIdx.x < s)
				shmem[threadIdx.x] += shmem[threadIdx.x + s];
			__syncthreads();
		}

		temp = shmem[0];	// save sum calculated at current iteration
	}

	// ************ evaluate (z[offset] / sum) and derivative z[offset] = z[offset] * (1 - z[offset]) ************ //
	for (int i = 0; i < numOfIters; ++i) {
		const Nd4jLong elemIdx = i * blockDim.x + threadIdx.x;
		if(elemIdx >= len) continue;
		const Nd4jLong offset = shape::getIndexOffset(elemIdx, xzShapeInfo);
		z[offset] /= shmem[0];
		z[offset] *= (1.f - z[offset]);		// derivative
	}
}

///////////////////////////////////////////////////////////////////
template <typename T>
linkage void softMaxDerivForVectorCudaLauncher(const hipStream_t* stream, const void *vx, const Nd4jLong *xzShapeInfo, void *vz) {

	softMaxDerivForVectorCuda<T><<<1, CUDA_BLOCK_SIZE, 1024, *stream>>>(vx, xzShapeInfo, vz);
}

///////////////////////////////////////////////////////////////////
void softmaxDerivative(sd::LaunchContext * context, const NDArray& input, NDArray& output, const int dimension) {

	if(!input.isActualOnDeviceSide()) input.syncToDevice();
	const int rank = input.rankOf();
	int temp;

	if(shape::isCommonVector(input.shapeInfo(), temp)) {

		BUILD_SINGLE_SELECTOR(input.dataType(), softMaxDerivForVectorCudaLauncher, (context->getCudaStream(), input.specialBuffer(), input.specialShapeInfo(), output.specialBuffer()), FLOAT_TYPES);
		input.tickReadDevice();
	}
	else {

		auto maxAlongDim = const_cast<NDArray&>(input).reduceAlongDimension(reduce::Max, {dimension}, true);
		(input - maxAlongDim).applyTransform(transform::Exp, output); // output contains exponents temporarily
		auto sumAlongDim = output.reduceAlongDimension(reduce::Sum, {dimension}, true);
		output /= sumAlongDim;
		output *= (1.f - output);	// derivative
		input.tickReadDevice();
	}

	PointersManager manager(context, "helpers::softmaxDerivative");
	manager.synchronize();

	output.tickWriteDevice();
}


	template <typename T>
	linkage void thresholdRelu_(NDArray const& input, double threshold, NDArray& output) {
		auto routine = LAMBDA_T(_x, threshold) {
			return _x > (T)threshold ? _x: (T)0.f;
		};
		const_cast<NDArray&>(input).applyLambda(routine, output);
	}

	void thresholdRelu(sd::LaunchContext * context, NDArray const& input, double threshold, NDArray& output) {
		BUILD_SINGLE_SELECTOR(input.dataType(), thresholdRelu_, (input, threshold, output), FLOAT_TYPES);
	}

	template <typename T>
	linkage void thresholdReluDerivative_(NDArray* input, double theta, NDArray* dLdO, NDArray* output) {
        auto derivative = LAMBDA_TT(_x, grO, theta) {if (_x > theta) return grO; else return static_cast<T>(0); };

        input->applyPairwiseLambda(*dLdO, derivative, *output);
	}

	void thresholdReluDerivative(sd::LaunchContext * context, NDArray* input, double threshold, NDArray* dLdO, NDArray* output) {
		BUILD_SINGLE_SELECTOR(input->dataType(), thresholdReluDerivative_, (input, threshold, dLdO, output), FLOAT_TYPES);
	}

}
}
}

