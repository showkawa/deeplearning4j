#include "hip/hip_runtime.h"
/*
 *  ******************************************************************************
 *  *
 *  *
 *  * This program and the accompanying materials are made available under the
 *  * terms of the Apache License, Version 2.0 which is available at
 *  * https://www.apache.org/licenses/LICENSE-2.0.
 *  *
 *  * See the NOTICE file distributed with this work for additional
 *  * information regarding copyright ownership.
 *  * Unless required by applicable law or agreed to in writing, software
 *  * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 *  * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 *  * License for the specific language governing permissions and limitations
 *  * under the License.
 *  *
 *  * SPDX-License-Identifier: Apache-2.0
 *  *****************************************************************************
 */

//
// @author Yurii Shyrma (iuriish@yahoo.com)
//

#include <ops/declarable/helpers/convolutions.h>
#include <helpers/PointersManager.h>
#include <math/templatemath.h>

namespace sd {
namespace ops  {

//////////////////////////////////////////////////////////////////////////
// columns [bS, iC, kD, kH, kW, oD, oH, oW] to be de-convoluted to volume [bS, iC, iD, iH, iW]
template <typename T>
static __global__ void col2volCuda(const void* columns, const Nd4jLong* colShapeInfo, void* volume, const Nd4jLong* volShapeInfo,  const int sD, const int sH, const int sW, const int pD, const int pH, const int pW, const int dD, const int dH, const int dW) {

    const T* col = reinterpret_cast<const T*>(columns);
          T* vol = reinterpret_cast<T*>(volume);

    __shared__ uint kD, kH, kW, oD, oH, oW, *sharedMem;
    __shared__ Nd4jLong volLen;

    if (threadIdx.x == 0) {
        extern __shared__ unsigned char shmem[];
        sharedMem = reinterpret_cast<uint*>(shmem);

        oD = colShapeInfo[6];
        oH = colShapeInfo[7];
        oW = colShapeInfo[8];

        kD = dD * (colShapeInfo[3] - 1) + 1;
        kH = dH * (colShapeInfo[4] - 1) + 1;
        kW = dW * (colShapeInfo[5] - 1) + 1;

        volLen  = shape::length(volShapeInfo);
    }
    __syncthreads();

    auto coords = sharedMem + threadIdx.x * 8;

    const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (Nd4jLong i = tid; i < volLen; i += gridDim.x * blockDim.x) {

        shape::index2coords(i, volShapeInfo, coords);

        const auto volOffset = shape::getOffset(volShapeInfo, coords);

        const auto bSiCoffset = coords[0] * colShapeInfo[9] + coords[1] * colShapeInfo[10];

        const uint imD = coords[2] + pD;
        const uint imH = coords[3] + pH;
        const uint imW = coords[4] + pW;

        const uint colDstart = (imD < kD) ? 0 : (imD - kD) / sD + 1;
        const uint colHstart = (imH < kH) ? 0 : (imH - kH) / sH + 1;
        const uint colWstart = (imW < kW) ? 0 : (imW - kW) / sW + 1;

        const uint colDend = sd::math::nd4j_min<uint>(imD / sD + 1, oD);
        const uint colHend = sd::math::nd4j_min<uint>(imH / sH + 1, oH);
        const uint colWend = sd::math::nd4j_min<uint>(imW / sW + 1, oW);

        T val = 0;

        for(uint colD = colDstart; colD < colDend; ++colD) {
            coords[2] = imD - colD * sD;
            if(coords[2] % dD != 0) continue;

            for(uint colH = colHstart; colH < colHend; ++colH) {
                coords[3] = imH - colH * sH;
                if(coords[3] % dH != 0) continue;

                for(uint colW = colWstart; colW < colWend; ++colW) {
                    coords[4] = imW - colW * sW;
                    if(coords[4] % dW != 0) continue;

                    val += col[bSiCoffset + (coords[2]/dD)*colShapeInfo[11] + (coords[3]/dH)*colShapeInfo[12] + (coords[4]/dW)*colShapeInfo[13] + colD*colShapeInfo[14] + colH*colShapeInfo[15] + colW*colShapeInfo[16]];

                }
            }
        }

        vol[volOffset] = val;
    }
}

//////////////////////////////////////////////////////////////////////////
template <typename T>
static void col2volCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream,
                                const void* columns, const Nd4jLong* colShapeInfo,
                                      void* volume, const Nd4jLong* volShapeInfo,
                                const int sD, const int sH, const int sW, const int pD, const int pH, const int pW, const int dD, const int dH, const int dW) {

    col2volCuda<T><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(columns, colShapeInfo, volume, volShapeInfo, sD, sH, sW, pD, pH, pW, dD, dH, dW);
}

//////////////////////////////////////////////////////////////////////////
void ConvolutionUtils::col2vol(sd::graph::Context& block, const NDArray& col, NDArray& vol, const int sD, const int sH, const int sW, const int pD, const int pH, const int pW, const int dD, const int dH, const int dW) {

    PointersManager manager(block.launchContext(), "col2vol");

    const int threadsPerBlock = MAX_NUM_THREADS / 4;
    const int blocksPerGrid = (vol.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;
    const int sharedMem = col.rankOf() * sizeof(uint) * threadsPerBlock  + 256;

    NDArray::prepareSpecialUse({&vol}, {&col});
    BUILD_SINGLE_SELECTOR(vol.dataType(), col2volCudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, block.launchContext()->getCudaStream(), col.specialBuffer(), col.specialShapeInfo(), vol.specialBuffer(), vol.specialShapeInfo(), sD, sH, sW, pD, pH, pW, dD, dH, dW), FLOAT_TYPES);
    NDArray::registerSpecialUse({&vol}, {&col});

    manager.synchronize();
}

}
}
