#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Oleh Semeniv (oleg.semeniv@gmail.com)
//

#include <system/op_boilerplate.h>
#include <ops/declarable/helpers/updatersHelpers.h>
#include <helpers/PointersManager.h>
#include <math/platformmath.h>
#include <math/templatemath.h>

namespace sd    {
namespace ops     {
namespace helpers {

///////////////////////////////////////////////////////////////////
template<typename T>
__global__ void nadamUpdaterCuda(const void* vx, const Nd4jLong* xShapeInfo, const void* vinv, const Nd4jLong* invShapeInfo,
                                 const void* vinm, const Nd4jLong* inmShapeInfo, void* vz, const Nd4jLong* zShapeInfo, 
                                 void* vstV, const Nd4jLong* stvShapeInfo, void* vstM, const Nd4jLong* stmShapeInfo,
                                const T lr, const T beta1, const T beta2, const T epsilon, const T iteration) {

    const auto grad = reinterpret_cast<const T*>(vx);
    const auto initV = reinterpret_cast<const T*>(vinv);
    const auto initM = reinterpret_cast<const T*>(vinm);

    auto up = reinterpret_cast<T*>(vz);
    auto stV = reinterpret_cast<T*>(vstV);
    auto stM = reinterpret_cast<T*>(vstM);

    __shared__ Nd4jLong xLen;
    __shared__ T mbeta1T, mbeta1, mbeta2;
    __shared__ bool bEWS, bOrdering, bXZsame, bXInUSame, bXStUSame, bXInMSame, bXStMSame;

    if (threadIdx.x == 0) {
        xLen = shape::length(xShapeInfo);
        
        mbeta1T = 1.0 - sd::math::nd4j_pow<T, T, T>(beta1, (iteration + 1));
        mbeta1 = (1 - beta1);
        mbeta2 = (1 - beta2);

        bEWS =  1 == shape::elementWiseStride(xShapeInfo) && 1 == shape::elementWiseStride(zShapeInfo) &&
                1 == shape::elementWiseStride(stmShapeInfo) && 1 == shape::elementWiseStride(inmShapeInfo) &&
                1 == shape::elementWiseStride(stvShapeInfo) && 1 == shape::elementWiseStride(invShapeInfo);
        bOrdering = shape::order(xShapeInfo) == shape::order(zShapeInfo) && shape::order(zShapeInfo) == shape::order(stmShapeInfo) &&
                    shape::order(stmShapeInfo) == shape::order(inmShapeInfo) && shape::order(inmShapeInfo) == shape::order(stvShapeInfo) &&
                    shape::order(stvShapeInfo) == shape::order(invShapeInfo);
        
        bXZsame = shape::haveSameShapeAndStrides(xShapeInfo, zShapeInfo);
        bXInUSame = shape::haveSameShapeAndStrides(xShapeInfo, invShapeInfo);
        bXStUSame = shape::haveSameShapeAndStrides(xShapeInfo, stvShapeInfo);
        bXInMSame = shape::haveSameShapeAndStrides(xShapeInfo, inmShapeInfo);
        bXStMSame = shape::haveSameShapeAndStrides(xShapeInfo, stmShapeInfo);
    }
    __syncthreads();

    int coords[MAX_RANK];

    for (Nd4jLong i = blockIdx.x * blockDim.x + threadIdx.x; i < xLen; i += gridDim.x * blockDim.x) {

        auto xOffset = i, zOffset = i, initMOffset = i, initUOffset = i, stMOffset = i, stUOffset = i;

        if (!bEWS || !bOrdering){

            shape::index2coords(i, xShapeInfo, coords);
            xOffset  = shape::getOffset(xShapeInfo, coords);
            zOffset  = bXZsame ? xOffset : shape::getOffset(zShapeInfo, coords);
            initUOffset = bXInUSame ? xOffset : shape::getOffset(invShapeInfo, coords);
            stUOffset = bXStUSame ? xOffset : shape::getOffset(stvShapeInfo, coords);
            initMOffset = bXInMSame ? xOffset : shape::getOffset(inmShapeInfo, coords);
            stMOffset = bXStMSame ? xOffset : shape::getOffset(stmShapeInfo, coords);
        }

        auto oneMinusBeta1Grad = grad[xOffset] * mbeta1;

        stM[stMOffset] = beta1 * initM[initMOffset] + oneMinusBeta1Grad;
        stV[stUOffset] = beta2 * initV[initUOffset] + grad[xOffset] * grad[xOffset] * mbeta2;

        up[zOffset] = (lr * ((stM[stMOffset] * beta1 + oneMinusBeta1Grad) / mbeta1T)) / (sd::math::nd4j_sqrt<T, T>(stV[stUOffset]) + epsilon);
    }
}

///////////////////////////////////////////////////////////////////
template<typename T>
linkage void nadamUpdaterCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const hipStream_t* stream, const void* vx, const Nd4jLong* xShapeInfo,
    const void* vinv, const Nd4jLong* invShapeInfo, const void* vinm, const Nd4jLong* inmShapeInfo,
    void* vz, const Nd4jLong* zShapeInfo, void* vstV, const Nd4jLong* stvShapeInfo, void* vstM, 
    const Nd4jLong* stmShapeInfo, const double dLr, const double dBeta1, const double dBeta2, const double dEpsilon, const int nIteration) {

    const T lr = static_cast<T>(dLr);
    const T beta1 = static_cast<T>(dBeta1);
    const T beta2 = static_cast<T>(dBeta2);
    const T epsilon = static_cast<T>(dEpsilon);
    const T iteration = static_cast<T>(nIteration);

    nadamUpdaterCuda<T><<<blocksPerGrid, threadsPerBlock, 256, * stream>>>(vx, xShapeInfo, vinv, invShapeInfo, vinm, inmShapeInfo,
             vz, zShapeInfo, vstV, stvShapeInfo, vstM, stmShapeInfo, lr, beta1, beta2, epsilon, iteration);
}

///////////////////////////////////////////////////////////////////
void updaterNadam(sd::LaunchContext* context, const NDArray& gradient, const NDArray& initStateV, const NDArray& initStateM, 
     NDArray& update, NDArray& stateV, NDArray& stateM, const double dLr, const double dBeta1, const double dBeta2, const double dEpsilon, const int nIteration) {

    PointersManager manager(context, "nadamUpdater");

    const int threadsPerBlock = MAX_NUM_THREADS / 4;
    const int blocksPerGrid = (gradient.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;

    NDArray::prepareSpecialUse({ &update, &stateV, &stateM }, { &gradient, &initStateV, &initStateM });
    BUILD_SINGLE_SELECTOR(gradient.dataType(), nadamUpdaterCudaLauncher, (blocksPerGrid, threadsPerBlock, context->getCudaStream(), gradient.specialBuffer(), gradient.specialShapeInfo(),
        initStateV.specialBuffer(), initStateV.specialShapeInfo(), initStateM.specialBuffer(), initStateM.specialShapeInfo(),
        update.specialBuffer(), update.specialShapeInfo(), stateV.specialBuffer(), stateV.specialShapeInfo(),
        stateM.specialBuffer(), stateM.specialShapeInfo(), dLr, dBeta1, dBeta2, dEpsilon, nIteration), FLOAT_TYPES);
    NDArray::registerSpecialUse({ &update, &stateV, &stateM }, { &gradient, &initStateV, &initStateM });

    manager.synchronize();
}


}
}
}
