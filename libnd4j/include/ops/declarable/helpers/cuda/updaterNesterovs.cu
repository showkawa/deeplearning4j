#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Oleh Semeniv (oleg.semeniv@gmail.com)
//

#include <system/op_boilerplate.h>
#include <ops/declarable/helpers/updatersHelpers.h>
#include <helpers/PointersManager.h>
#include <math/platformmath.h>
#include <math/templatemath.h>

namespace sd    {
namespace ops     {
namespace helpers {


///////////////////////////////////////////////////////////////////
template<typename T>
__global__ void nesterovsUpdaterCuda(const void* vx, const Nd4jLong* xShapeInfo, const void* vin, const Nd4jLong* inShapeInfo, 
                                     void* vz, const Nd4jLong* zShapeInfo, void* vst, const Nd4jLong* stShapeInfo, const T lr, const T momentum) {

    const auto grad = reinterpret_cast<const T*>(vx);
    const auto init = reinterpret_cast<const T*>(vin);
    auto up = reinterpret_cast<T*>(vz);
    auto st = reinterpret_cast<T*>(vst);

    __shared__ Nd4jLong xLen;
    __shared__ T momentumT;
    __shared__ bool bEWS, bOrdering, bXZsame, bXInSame, bXStSame;

    if (threadIdx.x == 0) {
        xLen = shape::length(xShapeInfo);
        momentumT = (-momentum - 1);

        bEWS =  1 == shape::elementWiseStride(xShapeInfo) && 1 == shape::elementWiseStride(zShapeInfo) &&
                1 == shape::elementWiseStride(stShapeInfo) && 1 == shape::elementWiseStride(inShapeInfo);
        bOrdering = shape::order(xShapeInfo) == shape::order(zShapeInfo) && shape::order(xShapeInfo) == shape::order(inShapeInfo) &&
                    shape::order(xShapeInfo) == shape::order(stShapeInfo);

        bXZsame = shape::haveSameShapeAndStrides(xShapeInfo, zShapeInfo);
        bXInSame = shape::haveSameShapeAndStrides(xShapeInfo, inShapeInfo);
        bXStSame = shape::haveSameShapeAndStrides(xShapeInfo, stShapeInfo);
    }
    __syncthreads();

    int coords[MAX_RANK];
    
    for (Nd4jLong i = blockIdx.x * blockDim.x + threadIdx.x; i < xLen; i += gridDim.x * blockDim.x) {

        auto xOffset = i, zOffset = i, initOffset = i, stOffset = i;

        if (!bEWS || !bOrdering) {

            shape::index2coords(i, xShapeInfo, coords);
            xOffset  = shape::getOffset(xShapeInfo, coords);
            zOffset  = bXZsame ? xOffset : shape::getOffset(zShapeInfo, coords);
            initOffset = bXInSame ? xOffset : shape::getOffset(inShapeInfo, coords);
            stOffset = bXStSame ? xOffset : shape::getOffset(stShapeInfo, coords);
        }

        T prevState =  momentum * init[initOffset];
        st[stOffset] = prevState - lr * grad[xOffset];
        up[zOffset] = prevState + momentumT * st[stOffset];
    }
}

///////////////////////////////////////////////////////////////////
template<typename T>
linkage void nesterovsUpdaterCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const hipStream_t* stream, 
                                          const void* vx, const Nd4jLong* xShapeInfo, const void* vin, const Nd4jLong* inShapeInfo, 
                                          void* vz, const Nd4jLong* zShapeInfo, void* vst, const Nd4jLong* stShapeInfo,
                                          const double dLr, const double dMomentum) {
    
     const T lr = static_cast<T>(dLr);
     const T momentum = static_cast<T>(dMomentum);
     nesterovsUpdaterCuda<T><<<blocksPerGrid, threadsPerBlock, 256, * stream>>>(vx, xShapeInfo, vin, inShapeInfo,
                                             vz, zShapeInfo, vst, stShapeInfo, lr, momentum);
}

///////////////////////////////////////////////////////////////////
void updaterNesterovs(sd::LaunchContext* context, const NDArray& gradient, const NDArray& initState, 
                      NDArray& update, NDArray& stateV, const double dLr, const double dMomentum) {

    PointersManager manager(context, "nesterovsUpdater");

    const int threadsPerBlock = MAX_NUM_THREADS / 4;
    const int blocksPerGrid = (gradient.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;

    NDArray::prepareSpecialUse({ &update, &stateV }, { &gradient, &initState });
    BUILD_SINGLE_SELECTOR(gradient.dataType(), nesterovsUpdaterCudaLauncher, (blocksPerGrid, threadsPerBlock, 
        context->getCudaStream(), gradient.specialBuffer(), gradient.specialShapeInfo(),
        initState.specialBuffer(), initState.specialShapeInfo(),
        update.specialBuffer(), update.specialShapeInfo(),
        stateV.specialBuffer(), stateV.specialShapeInfo(), dLr, dMomentum), FLOAT_TYPES);
    NDArray::registerSpecialUse({ &update, &stateV }, { &gradient, &initState });

    manager.synchronize();
}

}
}
}
