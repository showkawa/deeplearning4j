#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma (iuriish@yahoo.com)
// @author sgazeos@gmail.com
// @author raver119@gmail.com
//


#include <ops/declarable/helpers/transforms.h>
#include <helpers/ShapeUtils.h>
#include <helpers/PointersManager.h>
#include <helpers/ConstantTadHelper.h>

namespace sd 	  {
namespace ops 	  {
namespace helpers {

//////////////////////////////////////////////////////////////////////////
template<typename T>
__global__ static void clipByNormCuda(const void* vClipNorm, const void* vNorm, const Nd4jLong* normShapeInfo, void* vz, const Nd4jLong* zShapeInfo, const int* dimensions, const int dimsLen, const bool useAverage) {

    const T clipNorm = *reinterpret_cast<const T*>(vClipNorm);
    const T* norm    = reinterpret_cast<const T*>(vNorm);
          T* z       = reinterpret_cast<T*>(vz);

    __shared__ Nd4jLong zLen, tadLen, totalThreads;

    if (threadIdx.x == 0) {

        zLen   = shape::length(zShapeInfo);
        tadLen = zLen / shape::length(normShapeInfo);
        totalThreads = gridDim.x * blockDim.x;
    }

    __syncthreads();

    int zCoords[MAX_RANK], normCoords[MAX_RANK];

    const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (Nd4jLong i = tid; i < zLen; i += totalThreads) {

        shape::index2coords(i, zShapeInfo, zCoords);

        // deduce norm coords
        for (int j = 0; j < dimsLen; ++j)
            normCoords[j] = zCoords[dimensions[j]];

        const T actualNorm = useAverage ? norm[shape::getOffset(normShapeInfo, normCoords)] / tadLen : norm[shape::getOffset(normShapeInfo, normCoords)];

        if(actualNorm > clipNorm)
            z[shape::getOffset(zShapeInfo, zCoords)] *= clipNorm / actualNorm;
    }
}

//////////////////////////////////////////////////////////////////////////
template<typename T>
__host__ static void clipByNormCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const hipStream_t *stream,
                                            const void* vClipNorm, const void* vNorm, const Nd4jLong* normShapeInfo, void* vz, const Nd4jLong* zShapeInfo,
                                            const int* dimensions, const int dimsLen, const bool useAverage) {

    clipByNormCuda<T><<<blocksPerGrid, threadsPerBlock, 512, *stream>>>(vClipNorm, vNorm, normShapeInfo, vz, zShapeInfo, dimensions, dimsLen, useAverage);
}

//////////////////////////////////////////////////////////////////////////
void clipByNorm(sd::LaunchContext* context, NDArray& input, NDArray& output, const std::vector<int>& dims, const NDArray& clipNorm, const bool isInplace, const bool useAverage) {

    NDArray* z = nullptr;

    if(isInplace) {
        z = &input;
    }
    else {
        output.assign(input);
        z = &output;
    }

    if(dims.empty()) {

        const NDArray actualNorm = useAverage ? z->reduceAlongDimension(reduce::Norm2, {}) / z->lengthOf() : z->reduceAlongDimension(reduce::Norm2, {});

        if(actualNorm.e<float>(0) > clipNorm.e<float>(0))
            *z *= clipNorm / actualNorm;
    }
    else {

        const NDArray actualNorms = z->reduceAlongDimension(reduce::Norm2, dims);

        std::vector<int> dimsToExclude = ShapeUtils::evalDimsToExclude(z->rankOf(), dims);

        const int threadsPerBlock = MAX_NUM_THREADS / 2;
        const int blocksPerGrid = (z->lengthOf() + threadsPerBlock - 1) / threadsPerBlock;

        PointersManager manager(context, "clipByNorm");

        const int* dimensions = reinterpret_cast<const int*>(manager.replicatePointer(dimsToExclude.data(), dimsToExclude.size() * sizeof(int)));

        NDArray::prepareSpecialUse({z}, {z, &actualNorms, &clipNorm});
        BUILD_SINGLE_SELECTOR(z->dataType(), clipByNormCudaLauncher, (blocksPerGrid, threadsPerBlock, context->getCudaStream(), clipNorm.specialBuffer(), actualNorms.specialBuffer(), actualNorms.specialShapeInfo(), z->specialBuffer(), z->specialShapeInfo(), dimensions, (int)dimsToExclude.size(), useAverage), FLOAT_TYPES);
        NDArray::registerSpecialUse({z}, {z, &actualNorms, &clipNorm});

        manager.synchronize();
    }
}

//////////////////////////////////////////////////////////////////////////
template<typename T>
__global__ static void clipByNormBpCuda(const void* vClipNorm,
                                        const void* vx, const Nd4jLong* xShapeInfo,         // input
                                        const void* vy, const Nd4jLong* yShapeInfo,         // gradO
                                        const void* vNorm, const Nd4jLong* normShapeInfo,
                                        const void* vSum, const Nd4jLong* sumShapeInfo,
                                        void* vz, const Nd4jLong* zShapeInfo,               // gradI
                                        const int* dimensions, const int dimsLen, const bool useAverage) {

    const T clipNorm = *reinterpret_cast<const T*>(vClipNorm);
    const T* norm    = reinterpret_cast<const T*>(vNorm);
    const T* sum     = reinterpret_cast<const T*>(vSum);
    const T* x       = reinterpret_cast<const T*>(vx);
    const T* y       = reinterpret_cast<const T*>(vy);
          T* z       = reinterpret_cast<T*>(vz);

    __shared__ Nd4jLong zLen, tadLen, totalThreads;
    __shared__ bool sameOffsets;

    if (threadIdx.x == 0) {

        zLen   = shape::length(zShapeInfo);
        tadLen = zLen / shape::length(normShapeInfo);
        totalThreads = gridDim.x * blockDim.x;

        sameOffsets = shape::haveSameShapeAndStrides(xShapeInfo, yShapeInfo, zShapeInfo);
    }

    __syncthreads();

    int zCoords[MAX_RANK], normCoords[MAX_RANK];

    const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (Nd4jLong i = tid; i < zLen; i += totalThreads) {

        shape::index2coords(i, zShapeInfo, zCoords);

        const auto zOffset = shape::getOffset(zShapeInfo, zCoords);
        const auto yOffset = sameOffsets ? zOffset : shape::getOffset(yShapeInfo, zCoords);

        // deduce norm coords
        for (int j = 0; j < dimsLen; ++j)
            normCoords[j] = zCoords[dimensions[j]];

        const T actualNorm = useAverage ? norm[shape::getOffset(normShapeInfo, normCoords)] / tadLen : norm[shape::getOffset(normShapeInfo, normCoords)];

        if(actualNorm > clipNorm) {

            const T sumVal =  sum[shape::getOffset(sumShapeInfo, normCoords)];
            const auto xOffset = sameOffsets ? zOffset : shape::getOffset(xShapeInfo, zCoords);

            z[zOffset] = (clipNorm / actualNorm) * y[yOffset] * (static_cast<T>(1.f) - (x[xOffset] * sumVal) / (actualNorm * actualNorm));
        }
        else
            z[zOffset] = y[yOffset];
    }
}

//////////////////////////////////////////////////////////////////////////
template<typename T>
void clipByNormBp_(sd::LaunchContext* context, const NDArray& input, const NDArray& gradO, NDArray& gradI, const std::vector<int>& dims, const NDArray& clipNorm, const bool useAverage) {

    const int rank = input.rankOf();

    auto actualNorms = input.reduceAlongDimension(reduce::Norm2, dims);

    if(actualNorms.lengthOf() == 1) {

        const T norm = useAverage ? actualNorms.e<T>(0) / static_cast<T>(input.lengthOf()) : actualNorms.e<T>(0);

        auto clipVal = clipNorm.e<T>(0);

        if(norm > clipVal) {

            const T sum = input.reduceNumber(reduce::Sum).e<T>(0);    // reduce to scalar
            const T factor1 =  clipVal / norm;
            const T factor2 = static_cast<T>(1.f) / (norm * norm);                                            // 1 / (norm*norm*norm)

            auto lambda = LAMBDA_TT(x, y, sum, factor1, factor2) {
                return factor1 * y * (static_cast<T>(1.f) - factor2 * x * sum);
            };

            const_cast<NDArray&>(input).applyPairwiseLambda(const_cast<NDArray&>(gradO), lambda, gradI);
        }
        else
            gradI.assign(gradO);
    }
    else {

        const NDArray actualNorms = input.reduceAlongDimension(reduce::Norm2, dims);
        const NDArray sums        = input.reduceAlongDimension(reduce::Sum, dims);

        std::vector<int> dimsToExclude = ShapeUtils::evalDimsToExclude(gradI.rankOf(), dims);

        const int threadsPerBlock = MAX_NUM_THREADS / 2;
        const int blocksPerGrid = (gradI.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;

        PointersManager manager(context, "clipByNormBp");

        const int* dimensions = reinterpret_cast<const int*>(manager.replicatePointer(dimsToExclude.data(), dimsToExclude.size() * sizeof(int)));

        NDArray::prepareSpecialUse({&gradI}, {&actualNorms, &sums, &clipNorm, &input, &gradO});
        clipByNormBpCuda<T><<<blocksPerGrid, threadsPerBlock, 512, *context->getCudaStream()>>>(clipNorm.specialBuffer(), input.specialBuffer(), input.specialShapeInfo(), gradO.specialBuffer(), gradO.specialShapeInfo(), actualNorms.specialBuffer(), actualNorms.specialShapeInfo(), sums.specialBuffer(), sums.specialShapeInfo(), gradI.specialBuffer(), gradI.specialShapeInfo(), dimensions, (int)dimsToExclude.size(), useAverage);
        NDArray::registerSpecialUse({&gradI}, {&actualNorms, &sums, &clipNorm, &input, &gradO});

        manager.synchronize();
    }
}
BUILD_SINGLE_TEMPLATE(template void clipByNormBp_, (sd::LaunchContext* context, const NDArray& input, const NDArray& gradO, NDArray& gradI, const std::vector<int>& dimensions, const NDArray& clipNorm, const bool useAverage), FLOAT_TYPES);

//////////////////////////////////////////////////////////////////////////
void clipByNormBp(sd::LaunchContext* context, const NDArray& input, const NDArray& gradO, NDArray& gradI, const std::vector<int>& dimensions, const NDArray& clipNorm, const bool useAverage) {

    const NDArray& castedInput = gradI.dataType() == input.dataType() ? input : input.cast(gradI.dataType());
    BUILD_SINGLE_SELECTOR(gradI.dataType(), clipByNormBp_, (context, castedInput, gradO, gradI, dimensions, clipNorm, useAverage), FLOAT_TYPES);
}






        template <typename T>
    void clipByGlobalNorm_(sd::LaunchContext * context, std::vector<NDArray*> const& inputs, double clipNorm, sd::memory::Workspace* workspace, std::vector<NDArray*>& outputs, bool isInplace) {
        NDArray globalNorm = NDArrayFactory::create<T>(0, inputs[0]->getContext()); //sqrt(sum([l2norm(t)**2 for t in t_list]))

        for (auto i = 0; i < inputs.size(); i++) {
            auto input = inputs[i];
            auto l2norm = input->reduceNumber(reduce::Norm2);
            globalNorm += l2norm * l2norm;
        }

        globalNorm.applyTransform(transform::Sqrt, globalNorm);     // = sd::math::nd4j_sqrt(globalNorm);
        outputs[inputs.size()]->p(0, globalNorm);
        globalNorm.syncToHost();
        const T factor = static_cast<T>(clipNorm) / globalNorm.e<T>(0);

        for (size_t e = 0; e < inputs.size(); e++) {
            // all-reduce
            auto input = inputs[e];
            auto output = outputs[e];

            if (globalNorm.e<double>(0) <= clipNorm) {
                output->assign(input);
            }
            else {

                auto lambda = LAMBDA_T(_x, factor) { return _x * factor; };
                input->applyLambda(lambda, *output);
            }
        }
    }

    void clipByGlobalNorm(sd::LaunchContext * context, std::vector<NDArray*> const& inputs, double clipNorm, sd::memory::Workspace* workspace, std::vector<NDArray*>& outputs, bool isInplace) {
        BUILD_SINGLE_SELECTOR(outputs[0]->dataType(), clipByGlobalNorm_, (context, inputs, clipNorm, workspace, outputs, isInplace), FLOAT_TYPES);
    }

    BUILD_SINGLE_TEMPLATE(template void clipByGlobalNorm_, (sd::LaunchContext * context, std::vector<NDArray*> const& inputs, double clipNorm, sd::memory::Workspace* workspace, std::vector<NDArray*>& outputs, bool isInplace), FLOAT_TYPES);


    template <typename T>
    static void __global__ clipByValueKernel(void* input, const Nd4jLong* inputShape, void* output, const Nd4jLong* outputShape, double leftBound, double rightBound) {
        __shared__ T* outputBuf;
        __shared__ T* inputBuf;
        __shared__ Nd4jLong length;
        __shared__ bool linearBuffers;
        if (threadIdx.x == 0) {
            outputBuf = reinterpret_cast<T *>(output);
            inputBuf = reinterpret_cast<T *>(input);
            length = shape::length(inputShape);
            linearBuffers = shape::elementWiseStride(inputShape) == shape::elementWiseStride(outputShape) && shape::elementWiseStride(inputShape) == 1;
        }
        __syncthreads();
        const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
        const auto step = gridDim.x * blockDim.x;

        for (Nd4jLong e = tid; e < length; e += step) {
            if (linearBuffers) {
                if (inputBuf[e] > rightBound) outputBuf[e] = (T) rightBound;
                else if (inputBuf[e] < leftBound) outputBuf[e] = (T) leftBound;
                else outputBuf[e] = inputBuf[e];
            }
            else {
                auto inputOffset = shape::getIndexOffset(e, inputShape);
                auto outputOffset = shape::getIndexOffset(e, outputShape);
                if (inputBuf[inputOffset] > rightBound) outputBuf[outputOffset] = (T) rightBound;
                else if (inputBuf[inputOffset] < leftBound) outputBuf[outputOffset] = (T) leftBound;
                else outputBuf[outputOffset] = inputBuf[outputOffset];
            }
        }
    }

    template <typename T>
    static void clipByValue_(sd::LaunchContext * context, NDArray& input, double leftBound, double rightBound, NDArray& output) {
        auto stream = context->getCudaStream();
        if (!input.isActualOnDeviceSide())
            input.syncToDevice();
        NDArray::prepareSpecialUse({&output}, {&input});
        clipByValueKernel<T><<<256, 512, 8192, *stream>>>(input.specialBuffer(), input.specialShapeInfo(), output.specialBuffer(), output.specialShapeInfo(), leftBound, rightBound);
        NDArray::registerSpecialUse({&output}, {&input});
    }

    void clipByValue(sd::LaunchContext * context, NDArray& input, double leftBound, double rightBound, NDArray& output) {
        BUILD_SINGLE_SELECTOR(input.dataType(), clipByValue_, (context, input, leftBound, rightBound, output), FLOAT_TYPES);
    }

    BUILD_SINGLE_TEMPLATE(template void clipByValue_, (sd::LaunchContext * context, NDArray& input, double leftBound, double rightBound, NDArray& output);, FLOAT_TYPES);

}
}
}

