#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author GS <sgazeos@gmail.com>
//

#include <ops/declarable/helpers/segment.h>
#include <ops/declarable/helpers/segment_common.h>
#include <array/NDArrayFactory.h>
#include <helpers/ShapeUtils.h>
#include <helpers/TAD.h>
#include <exceptions/cuda_exception.h>
#include <helpers/PointersManager.h>
#include <helpers/ConstantTadHelper.h>

namespace sd {
namespace ops {
namespace helpers {
    // -------------------------------------------------------------------------------------------------------------- //
    template <typename T, typename I>
    static __global__ void unsortedSegmentSqrtNLinearKernel(T* input, Nd4jLong const* inputShape, I* indices, Nd4jLong const* indicesShape, int* starts, int* lengths, Nd4jLong numOfClasses, T* output, Nd4jLong const* outputShape) {
        __shared__ Nd4jLong xLen, zLen;

        if (threadIdx.x == 0) {
            xLen = shape::length(inputShape);
            zLen = shape::length(outputShape);
        }
        __syncthreads();

        auto start = threadIdx.x + blockIdx.x * blockDim.x;
        auto step = blockDim.x * gridDim.x;

        for  (auto idx = start; idx < xLen; idx += step) {
            auto yIndex = shape::getIndexOffset(idx, indicesShape);
            auto segment = indices[yIndex];
            auto zIndex = shape::getIndexOffset(segment, outputShape);
            if (lengths[segment] == 0) continue;
            auto xIndex = shape::getIndexOffset(idx, inputShape);

            sd::math::atomics::nd4j_atomicAdd(&output[zIndex],  input[xIndex] / sd::math::nd4j_sqrt<int, T>(lengths[segment]));
        }
    }
    // -------------------------------------------------------------------------------------------------------------- //
    // SegmentSqrtN kernel
    template <typename T, typename I>
    static __global__ void segmentSqrtNTadKernel(T* inputBuf, Nd4jLong const* inputShape, Nd4jLong const* inputTads, Nd4jLong const* inputTadOffsets, I* indices, int* starts, int* lengths, Nd4jLong numOfClasses, void* outputBuf, Nd4jLong const* outputShape, Nd4jLong const* outputTads, Nd4jLong const* outputTadOffsets) {

        __shared__ Nd4jLong len, total;

        if (threadIdx.x == 0) {
            total = shape::sizeAt(inputShape, 0);
            len = shape::length(inputTads);
        }
        __syncthreads();

        for (auto idx = blockIdx.x; idx  < total; idx += gridDim.x) {
            auto segment = indices[idx];
            auto x = inputBuf + inputTadOffsets[idx];
            auto z = reinterpret_cast<T *>(outputBuf) + outputTadOffsets[segment];
            auto start = starts[segment];
            auto finish = start + lengths[segment];

            for (auto e = threadIdx.x; e < len; e += blockDim.x) {
                auto xIndex = shape::getIndexOffset(e, inputTads);
                auto zIndex = shape::getIndexOffset(e, outputTads);
                sd::math::atomics::nd4j_atomicAdd(&z[zIndex], x[xIndex] / sd::math::nd4j_sqrt<int, T>(lengths[segment]));
            }
        }
    }
    // -------------------------------------------------------------------------------------------------------------- //
    template <typename T, typename I>
    static void unsortedSegmentSqrtNFunctor_(sd::LaunchContext* context, NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {
        auto stream = context->getCudaStream();
//        NDArray classes = NDArrayFactory::create<int>('c', {numOfClasses, 2});
        NDArray classesRangesBegs = NDArrayFactory::create<int>('c', {numOfClasses}, context);
        NDArray classesRangesLens = NDArrayFactory::create<int>('c', {numOfClasses}, context);
//        NDArray row = NDArrayFactory::create<int>('c', {1, 2}, {(int)indices->lengthOf(), (int)0});
//        classes.applyTrueBroadcast(sd::BroadcastOpsTuple::Assign(), &row, &classes);
        classesRangesBegs.assign(indices->lengthOf());
        classesRangesLens.assign(0);
//        dim3 dims(numOfClasses, indices->lengthOf(), numOfClasses * 32 + 32);
        dim3 dims(128, 256, 256);
//        int* classesBuf = reinterpret_cast<int*>(classes.specialBuffer());
        fillUpSegments(indices, numOfClasses, classesRangesBegs, classesRangesLens);
        int* begins = reinterpret_cast<int*>(classesRangesBegs.specialBuffer());
        int* lengths = reinterpret_cast<int*>(classesRangesLens.specialBuffer());
        output->nullify();
        if (input->isVector()) {
            unsortedSegmentSqrtNLinearKernel<T,I><<<dims.x, dims.y, dims.z, *stream>>>(
                    input->dataBuffer()->specialAsT<T>(), input->specialShapeInfo(),
                    indices->dataBuffer()->specialAsT<I>(), indices->specialShapeInfo(), begins, lengths, numOfClasses,
                    output->dataBuffer()->specialAsT<T>(), output->specialShapeInfo());
        }
        else {
            output->nullify();
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = sd::ConstantTadHelper::getInstance().tadForDimensions(input->shapeInfo(), dimensions);
            auto packZ = sd::ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(), dimensions);
            auto inputTads = packX.specialShapeInfo();
            auto inputTadOffsets = packX.specialOffsets();
            auto outputTads = packZ.specialShapeInfo();
            auto outputTadOffsets = packZ.specialOffsets();
            dims.x = input->sizeAt(0);
            segmentSqrtNTadKernel<T,I><<<dims.x, dims.y, dims.z, *stream>>>(
                    input->dataBuffer()->specialAsT<T>(), input->specialShapeInfo(), inputTads, inputTadOffsets, indices->dataBuffer()->specialAsT<I>(),
                    begins, lengths, numOfClasses, output->specialBuffer(), output->specialShapeInfo(), outputTads,  outputTadOffsets);
        }
    }
    // -------------------------------------------------------------------------------------------------------------- //
    void unsortedSegmentSqrtNFunctor(sd::LaunchContext* context , NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {
        NDArray::prepareSpecialUse({output}, {input, indices});
        BUILD_DOUBLE_SELECTOR(input->dataType(), indices->dataType(), unsortedSegmentSqrtNFunctor_, (context, input, indices, numOfClasses, output),
                              FLOAT_TYPES, INDEXING_TYPES);
        NDArray::registerSpecialUse({output}, {input, indices});
    }
    // -------------------------------------------------------------------------------------------------------------- //
    template <typename T, typename I>
    static __global__ void segmentSqrtNBPLinearKernel(void* inputBuf, Nd4jLong const*  inputShape, void* eps, Nd4jLong const*  epsShape, void* indicesBuf, Nd4jLong const*  indicesShape,
                                                      int* lengths, void* outputBuf, Nd4jLong const*  outputShape) {
        __shared__ T* x;
        __shared__ T* gradIn;
        __shared__ T* gradOut;
        __shared__ I* y;
        __shared__ T* z;
        __shared__ Nd4jLong xLen, gradLen;

        if (threadIdx.x == 0) {
            xLen = shape::length(inputShape);
            x = reinterpret_cast<T*>(inputBuf);
            y = reinterpret_cast<I*>(indicesBuf);
            z = reinterpret_cast<T*>(outputBuf);
            gradOut = reinterpret_cast<T*>(eps);
            gradLen = shape::length(epsShape);
        }
        __syncthreads();

        auto start = blockIdx.x * blockDim.x + threadIdx.x;
        auto step = gridDim.x * blockDim.x;

        for (auto e = start; e < xLen; e += step) {

            auto zOffset = shape::getIndexOffset(e, outputShape);
            auto xOffset = shape::getIndexOffset(e, inputShape);
            auto yOffset = shape::getIndexOffset(e, indicesShape);
            auto classIndex = y[yOffset];
            auto gradOffsetO = shape::getIndexOffset(classIndex, epsShape);

            z[zOffset] = T(gradOut[gradOffsetO] / math::nd4j_sqrt<int, float>(lengths[classIndex]));
        }
    }
    // -------------------------------------------------------------------------------------------------------------- //

    template <typename T, typename I>
    static __global__ void segmentSqrtNBPTadKernel(void* inputBuf, Nd4jLong const*  inputShape, void* eps, Nd4jLong const*  epsShape,
                                                   void* indicesBuf, Nd4jLong const*  indicesShape, int* lengths, void* outputBuf, Nd4jLong const*  outputShape,Nd4jLong const*  inputTad,
                                                   Nd4jLong const*  inputOffsets, Nd4jLong const*  gradOutTad, Nd4jLong const*  gradOutOffsets, Nd4jLong const*  outTad, Nd4jLong const*  outOffsets) {
        __shared__ T* x;
        __shared__ T* gradOut;
        __shared__ I* y;
        __shared__ T* z;
        __shared__ Nd4jLong xLen, yLen, gradLen, currentLen;

        if (threadIdx.x == 0) {
            xLen = shape::length(inputShape);
            x = reinterpret_cast<T*>(inputBuf);
            y = reinterpret_cast<I*>(indicesBuf);
            z = reinterpret_cast<T*>(outputBuf);
            yLen = shape::length(indicesShape);
            gradOut = reinterpret_cast<T*>(eps);
            gradLen = shape::length(epsShape);
            currentLen = shape::length(outTad);
        }
        __syncthreads();

        for (auto i = blockIdx.x; i < yLen; i += gridDim.x) {
//            auto yIndex = shape::getIndexOffset(i, indicesShape);
            auto segment = y[i]; //yIndex];
            T* currentOut = z + outOffsets[i];
            T* outGrad = gradOut + gradOutOffsets[segment];

            for (auto e = threadIdx.x; e < currentLen; e += blockDim.x) {
                auto zIndex = shape::getIndexOffset(e, outTad);
                auto gradIndex = shape::getIndexOffset(e, gradOutTad);
                if (lengths[segment] > 0)
                    currentOut[zIndex] = T(outGrad[gradIndex] / math::nd4j_sqrt<int, float>(lengths[segment]));
            }
        }
    }
    // -------------------------------------------------------------------------------------------------------------- //

    template <typename T, typename I>
    static int unsortedSegmentSqrtNFunctorBP_(sd::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        auto stream = context->getCudaStream();
        NDArray::prepareSpecialUse({output}, {input, indices, gradOut});
        auto numClasses = indices->e<int>(indices->lengthOf() - 1) + 1;
        NDArray classesRangesLens = NDArrayFactory::create<int>('c', {numClasses}, context);
        NDArray classesRangesBegs = NDArrayFactory::create<int>('c', {numClasses}, context);

        classesRangesBegs.assign(indices->lengthOf());
        classesRangesLens.assign(0);
        dim3 dims(numClasses, indices->lengthOf(), numClasses * 32 + 32);
        fillUpSegments(indices, numClasses, classesRangesBegs, classesRangesLens);
        int* begins = reinterpret_cast<int*>(classesRangesBegs.specialBuffer());
        int* lengths = reinterpret_cast<int*>(classesRangesLens.specialBuffer());

        if (input->isVector()) {
            Nd4jLong loop_size = input->lengthOf();
            auto numOfClasses = gradOut->lengthOf(); //indices->e<Nd4jLong>(loop_size - 1);
            segmentSqrtNBPLinearKernel<T,I><<<gradOut->lengthOf(), input->lengthOf(), 256, *stream>>>(input->specialBuffer(),
                    input->specialShapeInfo(), gradOut->specialBuffer(), gradOut->specialShapeInfo(),
                    indices->specialBuffer(), indices->specialShapeInfo(), lengths, output->specialBuffer(), output->specialShapeInfo());
        }
        else {
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = sd::ConstantTadHelper::getInstance().tadForDimensions(input->shapeInfo(), dimensions);
            auto packZ = sd::ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(), dimensions);
//            auto packGradIn = sd::ConstantTadHelper::getInstance().tadForDimensions(tempRes.shapeInfo(), dimensions);
            auto packGradOut = sd::ConstantTadHelper::getInstance().tadForDimensions(gradOut->shapeInfo(), dimensions);
            auto inputTads = packX.specialShapeInfo();
            auto inputTadOffsets = packX.specialOffsets();
            auto outputTads = packZ.specialShapeInfo();
            auto outputTadOffsets = packZ.specialOffsets();
            auto gradOutTads = packGradOut.specialShapeInfo();
            auto gradOutTadOffsets = packGradOut.specialOffsets();

            segmentSqrtNBPTadKernel<T,I><<<indices->lengthOf(), input->lengthOf(), 256, *stream>>>(input->specialBuffer(), input->specialShapeInfo(),
                    gradOut->specialBuffer(), gradOut->specialShapeInfo(), indices->specialBuffer(), indices->specialShapeInfo(), lengths,
                    output->specialBuffer(), output->specialShapeInfo(), inputTads, inputTadOffsets, gradOutTads, gradOutTadOffsets,
                    outputTads, outputTadOffsets);
        }
        NDArray::registerSpecialUse({output}, {input, indices, gradOut});

        return Status::OK();
    }
    // -------------------------------------------------------------------------------------------------------------- //
    int unsortedSegmentSqrtNFunctorBP(sd::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        NDArray::prepareSpecialUse({output}, {input, indices, gradOut});
        BUILD_DOUBLE_SELECTOR(output->dataType(), indices->dataType(), return unsortedSegmentSqrtNFunctorBP_, (context, input, indices, gradOut, numOfClasses, output), FLOAT_TYPES, INDEXING_TYPES);
        NDArray::registerSpecialUse({output}, {input, indices, gradOut});
    }
}
}
}