#include "hip/hip_runtime.h"
/*
 *  ******************************************************************************
 *  *
 *  *
 *  * This program and the accompanying materials are made available under the
 *  * terms of the Apache License, Version 2.0 which is available at
 *  * https://www.apache.org/licenses/LICENSE-2.0.
 *  *
 *  * See the NOTICE file distributed with this work for additional
 *  * information regarding copyright ownership.
 *  * Unless required by applicable law or agreed to in writing, software
 *  * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 *  * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 *  * License for the specific language governing permissions and limitations
 *  * under the License.
 *  *
 *  * SPDX-License-Identifier: Apache-2.0
 *  *****************************************************************************
 */

//
// @author Yurii Shyrma (iuriish@yahoo.com)
//

#include<ops/declarable/helpers/gammaMathFunc.h>
#include <array/NDArrayFactory.h>

namespace sd {
namespace ops {
namespace helpers {

///////////////////////////////////////////////////////////////////
template<typename T>
__global__ static void diGammaCuda(const void *vx, const Nd4jLong *xShapeInfo,
                                     	 void *vz, const Nd4jLong *zShapeInfo) {

    const auto x = reinterpret_cast<const T*>(vx);
          auto z = reinterpret_cast<T*>(vz);

    __shared__ Nd4jLong len;
    __shared__ bool sameOffset;

    if (threadIdx.x == 0) {
        len = shape::length(xShapeInfo);
        sameOffset = shape::haveSameShapeAndStrides(xShapeInfo, zShapeInfo);
    }
    __syncthreads();

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < len; i += gridDim.x * blockDim.x) {

        const auto xOffset = shape::getIndexOffset(i, xShapeInfo);
        const auto zOffset = sameOffset ? xOffset : shape::getIndexOffset(i, zShapeInfo);

        z[zOffset] = diGammaScalar<T>(x[xOffset]);
    }
}

///////////////////////////////////////////////////////////////////
template<typename T>
static void diGammaCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const hipStream_t *stream, const void *vx, const Nd4jLong *xShapeInfo, void *vz, const Nd4jLong *zShapeInfo) {

    diGammaCuda<T><<<blocksPerGrid, threadsPerBlock, 1024, *stream>>>(vx, xShapeInfo, vz, zShapeInfo);
}

///////////////////////////////////////////////////////////////////
void diGamma(sd::LaunchContext* context, const NDArray& x, NDArray& z) {

    int threadsPerBlock = MAX_NUM_THREADS / 2;
    int blocksPerGrid = (z.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;

    NDArray::prepareSpecialUse({&z}, {&x});
    BUILD_SINGLE_SELECTOR(x.dataType(), diGammaCudaLauncher, (blocksPerGrid, threadsPerBlock, context->getCudaStream(), x.specialBuffer(), x.specialShapeInfo(), z.specialBuffer(), z.specialShapeInfo()), FLOAT_TYPES);
    NDArray::registerSpecialUse({&z}, {&x});
}

BUILD_SINGLE_TEMPLATE(template void diGammaCudaLauncher, (const int blocksPerGrid, const int threadsPerBlock, const hipStream_t *stream, const void *vx, const Nd4jLong *xShapeInfo, void *vz, const Nd4jLong *zShapeInfo), FLOAT_TYPES);

}
}
}

