#include "hip/hip_runtime.h"
/*
 *  ******************************************************************************
 *  *
 *  *
 *  * This program and the accompanying materials are made available under the
 *  * terms of the Apache License, Version 2.0 which is available at
 *  * https://www.apache.org/licenses/LICENSE-2.0.
 *  *
 *  * See the NOTICE file distributed with this work for additional
 *  * information regarding copyright ownership.
 *  * Unless required by applicable law or agreed to in writing, software
 *  * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 *  * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 *  * License for the specific language governing permissions and limitations
 *  * under the License.
 *  *
 *  * SPDX-License-Identifier: Apache-2.0
 *  *****************************************************************************
 */

//
// @author Yurii Shyrma (iuriish@yahoo.com)
//

#include <ops/declarable/helpers/convolutions.h>
#include <helpers/PointersManager.h>

namespace sd {
namespace ops  {

//////////////////////////////////////////////////////////////////////////
template <typename T>
__global__ static void upsampling3dCuda(const void* vx, const Nd4jLong* xShapeInfo, void* vz, const Nd4jLong* zShapeInfo, const int factorD, const int factorH, const int factorW, const bool isNCDHW) {

    // x has shape [bS, iC, iD, iH, iW] (NCDHW) or [bS, iD, iH, iW, iC] (NDHWC)
    // z has shape [bS, iC, factorD*iD, factorH*iH, factorW*iW ] (NCDHW) or [bS, factorD*iD, factorH*iH, factorW*iW, iC] (NDHWC)

    const T* x = reinterpret_cast<const T*>(vx);
          T* z = reinterpret_cast<T*>(vz);

    __shared__ int rank, dimID;
    __shared__ Nd4jLong zLen, *sharedMem;

    if (threadIdx.x == 0) {
        extern __shared__ unsigned char shmem[];
        sharedMem = reinterpret_cast<Nd4jLong*>(shmem);

        dimID = isNCDHW ? 2 : 1;
        zLen  = shape::length(zShapeInfo);
        rank  = 5;
    }
    __syncthreads();

    const auto zInd = threadIdx.x + blockIdx.x * blockDim.x;

    if(zInd >= zLen)
        return;

    auto coords = sharedMem + threadIdx.x * rank;

    shape::index2coords(zInd, zShapeInfo, coords);

    const auto zOffset = shape::getOffset(zShapeInfo, coords);

    coords[dimID]     /= factorD;
    coords[dimID + 1] /= factorH;
    coords[dimID + 2] /= factorW;

    const auto xOffset = shape::getOffset(xShapeInfo, coords);

    z[zOffset] = x[xOffset];
}

//////////////////////////////////////////////////////////////////////////
template <typename T>
static void upsampling3dCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream,
                                     const void* vx, const Nd4jLong* xShapeInfo,
                                           void* vz, const Nd4jLong* zShapeInfo,
                                     const int factorD, const int factorH, const int factorW, const bool isNCDHW) {

    upsampling3dCuda<T><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(vx, xShapeInfo, vz, zShapeInfo, factorD, factorH, factorW, isNCDHW);
}

//////////////////////////////////////////////////////////////////////////
void ConvolutionUtils::upsampling3d(sd::graph::Context& block, const NDArray& input, NDArray& output, const int factorD, const int factorH, const int factorW, const bool isNCDHW) {

    PointersManager manager(block.launchContext(), "upsampling3d");

    const int threadsPerBlock = MAX_NUM_THREADS / 2;
    const int blocksPerGrid = (output.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;
    const int sharedMem = output.rankOf() * sizeof(Nd4jLong) * threadsPerBlock + 128;

    NDArray::prepareSpecialUse({&output}, {&input});
    BUILD_SINGLE_SELECTOR(input.dataType(), upsampling3dCudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, block.launchContext()->getCudaStream(), input.specialBuffer(), input.specialShapeInfo(), output.specialBuffer(), output.specialShapeInfo(), factorD, factorH, factorW, isNCDHW), FLOAT_TYPES);
    NDArray::registerSpecialUse({&output}, {&input});

    manager.synchronize();
}

}
}
