#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author sgazeos@gmail.com
//

#include <ops/declarable/helpers/fake_quantization.h>
#include <array/NDArrayFactory.h>

namespace sd {
namespace ops {
namespace helpers {
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// fakeQuantWithMinMaxVars_
// input - input tensor
// min - min scalar tensor
// max - max scalar tensor
// numBits - (default 16bit)
// narrowed - shrink is true
// output - output tensor
//
    template <typename T>
    static __host__ __device__ void
    nudge(T min, T max, int quantMin, int quantMax, T* scale, T* nudgedMin, T* nudgedMax) {
        T quantMaxF = static_cast<T>(quantMax);
        T quantMinF = static_cast<T>(quantMin);
        *scale = (max - min) / (quantMaxF - quantMinF);
        auto zeroPointFromMin = quantMinF - min / *scale;
        uint16_t const nudgedZeroPoint = [zeroPointFromMin, quantMin, quantMax, quantMaxF, quantMinF] {
            if (zeroPointFromMin < quantMinF) {
                return static_cast<uint16_t>(quantMin);
            }
            if (zeroPointFromMin > quantMaxF) {
                return static_cast<uint16_t>(quantMax);
            }
            return sd::math::nd4j_round<T,uint16_t>(zeroPointFromMin);
        }();
        *nudgedMax = (quantMaxF - static_cast<T>(nudgedZeroPoint)) * (*scale);
        *nudgedMin = (quantMinF - static_cast<T>(nudgedZeroPoint)) * (*scale);
    }

    template <typename T>
    void fakeQuantWithMinMaxVars_(NDArray* input, NDArray* min, NDArray* max, int numBits, bool narrowed, NDArray* output) {
        int lowIntBound = narrowed?1:0;
        int upperIntBound = (1 << numBits) - 1;
        min->syncToHost(); // these are scalars, so nothing much happened
        max->syncToHost();
        T scale, nudgedMin, nudgedMax;
        nudge(min->t<T>(0), max->t<T>(0), lowIntBound, upperIntBound, &scale, &nudgedMin, &nudgedMax);

        auto wiseMinMaxAndSoOn = LAMBDA_T(x, nudgedMin, nudgedMax, scale) {
            T val = x;
            if (x < nudgedMin) {
                val = nudgedMin;
            }
            else if (x > nudgedMax) {
                val = nudgedMax;
            }
            else
                val = x;
            return (math::nd4j_floor<T,T>((val - nudgedMin) / scale + T(0.5)) * scale + nudgedMin);
        };

        input->applyLambda(wiseMinMaxAndSoOn, *output);
    }

    template <typename T>
    static __global__ void fakeQuantWithMinMaxKernel(const T* input, const Nd4jLong* inputShape,
                                                     T* min, T* max,
                                                     int lowIntBound, int upperIntBound, Nd4jLong channels,
                                                     T* output, const Nd4jLong* outputShape,
                                                     Nd4jLong length) {
        __shared__ int block;
        if (threadIdx.x == 0) {
            block = length / channels; // to loop with last dimension as block
        }
        __syncthreads();

        for (auto i = blockIdx.x; i < (int)channels; i += gridDim.x) {
            T scale, nudgedMin, nudgedMax;
            nudge(min[i], max[i], lowIntBound, upperIntBound, &scale, &nudgedMin, &nudgedMax);
            // loop over blocks to quantization between nudged min and max
            for (auto b = threadIdx.x; b < block; b += blockDim.x) {
                T val = input[shape::getIndexOffset(b * channels + i, inputShape)];
                if (val < nudgedMin) {
                    val = nudgedMin;
                } else if (val > nudgedMax) {
                    val = nudgedMax;
                }
                output[shape::getIndexOffset(b * channels + i, outputShape)] =
                        (math::nd4j_floor<T, T>((val - nudgedMin) / scale + T(0.5f)) * scale + nudgedMin);
            };
        }
    }

    template <typename T>
    void fakeQuantWithMinMaxVarsPerChannel_(LaunchContext* context, NDArray* input, NDArray* min, NDArray* max, int numBits, bool narrowed, NDArray* output) {
        int lowIntBound = narrowed?1:0;
        int upperIntBound = (1 << numBits) - 1;
        auto channels = min->lengthOf();
        auto length = input->lengthOf();
        NDArray::prepareSpecialUse({output}, {min, max, input});
        auto stream = context->getCudaStream();
        T* inputBuf = input->dataBuffer()->specialAsT<T>();
        T* outputBuf = output->dataBuffer()->specialAsT<T>();
        T* minBuf = min->dataBuffer()->specialAsT<T>();
        T* maxBuf = max->dataBuffer()->specialAsT<T>();
        fakeQuantWithMinMaxKernel<<<128, 256, 256, *stream>>>(inputBuf, input->specialShapeInfo(),
                minBuf, maxBuf, lowIntBound, upperIntBound, channels, outputBuf, output->specialShapeInfo(), length);
        NDArray::registerSpecialUse({output}, {min, max, input});

    }

    void fakeQuantWithMinMaxVars(NDArray* input, NDArray* min, NDArray* max, int numBits, bool narrowed, NDArray* output) {
        BUILD_SINGLE_SELECTOR(input->dataType(), fakeQuantWithMinMaxVars_, (input, min, max, numBits, narrowed, output), FLOAT_TYPES);
    }
    void fakeQuantWithMinMaxVarsPerChannel(LaunchContext* context, NDArray* input, NDArray* min, NDArray* max, int numBits, bool narrowed, NDArray* output) {
        BUILD_SINGLE_SELECTOR(input->dataType(), fakeQuantWithMinMaxVarsPerChannel_, (context, input, min, max, numBits, narrowed, output), FLOAT_TYPES);
    }
}
}
}
