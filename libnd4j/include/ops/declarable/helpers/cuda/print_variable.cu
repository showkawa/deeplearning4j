/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author raver119@gmail.com
//

#include <ops/declarable/helpers/print_variable.h>
#include <helpers/PointersManager.h>

namespace sd {
    namespace ops {
        namespace helpers {
            template <typename T>
            static _CUDA_G void print_device(const void *special, const Nd4jLong *shapeInfo) {
                auto length = shape::length(shapeInfo);
                auto x = reinterpret_cast<const T*>(special);

                // TODO: add formatting here
                printf("[");

                for (uint64_t e = 0; e < length; e++) {
                    printf("%f", (float) x[shape::getIndexOffset(e, shapeInfo)]);

                    if (e < length - 1)
                        printf(", ");
                }

                printf("]\n");
            }

            template <typename T>
            static _CUDA_H void exec_print_device(LaunchContext &ctx, const void *special, const Nd4jLong *shapeInfo) {
                print_device<T><<<1, 1, 1024, *ctx.getCudaStream()>>>(special, shapeInfo);
            }

            void print_special(LaunchContext &ctx, const NDArray &array, const std::string &message) {
                NDArray::prepareSpecialUse({}, {&array});

                PointersManager pm(&ctx, "print_device");
                BUILD_SINGLE_SELECTOR(array.dataType(), exec_print_device, (ctx, array.specialBuffer(), array.specialShapeInfo()), LIBND4J_TYPES)
                pm.synchronize();

                NDArray::registerSpecialUse({}, {&array});
            }
        }
    }
}
