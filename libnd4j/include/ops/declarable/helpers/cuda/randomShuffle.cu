#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma (iuriish@yahoo.com)
// implemented algorithm is GPU adaptation of algorithm described in following article:
// "MergeShuffle: A Very Fast, Parallel Random Permutation Algorithm", https://arxiv.org/abs/1508.03167
//

#include<ops/declarable/helpers/transforms.h>
#include <array/ResultSet.h>
#include <numeric>
#include <execution/Threads.h>
#include <helpers/ShapeUtils.h>
#include <helpers/PointersManager.h>

namespace sd    {
namespace ops     {
namespace helpers {

//////////////////////////////////////////////////////////////////////////
template <typename T>
static __global__ void fisherYatesCuda(sd::graph::RandomGenerator* rng, void* vx, const Nd4jLong ews, const Nd4jLong len, const int power) {

    T* x = reinterpret_cast<T*>(vx);

    __shared__ T* shmem, temp;
    __shared__ Nd4jLong ind, blockOffset, lenPerBlock;

    if (threadIdx.x == 0) {
        extern __shared__ unsigned char sharedMemory[];
        shmem = reinterpret_cast<T*>(sharedMemory);

        blockOffset = (len * blockIdx.x) >> power;
        lenPerBlock = ((len * (blockIdx.x + 1)) >> power) - blockOffset;
        ind = blockOffset;
    }
    __syncthreads();

    // copy from global memory to shared memory
    if(threadIdx.x < lenPerBlock)
        shmem[threadIdx.x] = x[(blockOffset + threadIdx.x) * ews];
    __syncthreads();

    // *** apply Fisher-Yates shuffle to lenPerBlock number of elements
    if (threadIdx.x == 0) {
        for(Nd4jLong i = lenPerBlock - 1; i > 0; --i) {
           const Nd4jLong j = rng->relativeLong(ind++) % (i + 1);
            if(i != j) {
                temp = shmem[i];
                shmem[i] = shmem[j];
                shmem[j] = temp;
            }
        }
    }
    __syncthreads();

    // copy from shared memory to global memory
    if(threadIdx.x < lenPerBlock)
        x[(blockOffset + threadIdx.x) * ews] = shmem[threadIdx.x];
}

template <typename T>
static __global__ void mergeShuffleCuda(sd::graph::RandomGenerator* rng, void* vx, const Nd4jLong ews, const Nd4jLong len, const int power, const Nd4jLong iterNum) {


    T* x = reinterpret_cast<T*>(vx);

    __shared__ Nd4jLong ind, blockOffset, factor, beg, mid, totLen, iterExp;

    // *** apply mergeShuffle algorithm
    if(threadIdx.x == 0) {

        factor = blockIdx.x << iterNum;
        iterExp = 1 << (iterNum - 1);
        blockOffset = (len * factor) >> power;
        mid         = ((len * (factor + iterExp)) >> power) - blockOffset;                // middle
        totLen      = ((len * (factor + 2*iterExp)) >> power) - blockOffset;
        ind         = iterNum * len + blockOffset;
        beg = 0;               // beginning

        // printf("m %lld, blockIdx.x %lld, factor %lld, blockOffset %lld, mid %lld, totLen %lld \n", m,k,factor,blockOffset,mid,totLen);

        while (true) {
            if(rng->relativeLong(ind++) % 2) {
                if(mid == totLen)
                    break;
                math::nd4j_swap<T>(x[(blockOffset + beg) * ews], x[(blockOffset + mid++) * ews]);
            } else {
                if(beg == mid)
                    break;
            }
            ++beg;
        }

        // Fisher-Yates
        while (beg < totLen) {
            const Nd4jLong e = rng->relativeLong(ind++) % (beg + 1);
            if(beg != e)
                math::nd4j_swap<T>(x[(blockOffset + beg) * ews], x[(blockOffset + e) * ews]);
            ++beg;
        }
    }
}


//////////////////////////////////////////////////////////////////////////
// Fisher-Yates shuffle
template <typename T>
static void fisherYates(sd::graph::RandomGenerator& rng, T* buff, const Nd4jLong& len, const Nd4jLong& ews, Nd4jLong ind) {

    for(Nd4jLong i = len-1; i > 0; --i) {
        const Nd4jLong j = rng.relativeLong(ind++) % (i + 1);
        if(i != j)
            math::nd4j_swap<T>(buff[i*ews], buff[j*ews]);
    }
}

//////////////////////////////////////////////////////////////////////////
template <typename T>
static void randomShuffle_(sd::LaunchContext* context, NDArray& input, NDArray& output, sd::graph::RandomGenerator& rng, const bool isInplace) {

    const int firstDim = input.sizeAt(0);
    int temp;

    if(input.lengthOf() == 1 || firstDim == 1) {

        if(!isInplace)
            output.assign(input);
    }
    else if (shape::isCommonVector(input.shapeInfo(), temp)) {

        NDArray* arr = &input;

        if (!isInplace) {
            output.assign(input);
            arr = &output;
        }

        const Nd4jLong len = arr->lengthOf();

        const int threadsPerBlock = MAX_NUM_THREADS;

        int power = 0;
        while ((len >> power) > threadsPerBlock)
            ++power;

        const int blocksPerGrid = 1 << power;
        const int sharedMem = threadsPerBlock * input.sizeOfT() + 256;

        PointersManager manager(context, "NDArray::randomShuffle cuda");

        sd::graph::RandomGenerator* pRng = reinterpret_cast<sd::graph::RandomGenerator*>(manager.replicatePointer(&rng, sizeof(sd::graph::RandomGenerator)));

        NDArray::prepareSpecialUse({arr}, {arr});
        fisherYatesCuda<T><<<blocksPerGrid, threadsPerBlock, sharedMem, *context->getCudaStream()>>>(pRng, arr->specialBuffer(), arr->ews(), len, power);
        for (Nd4jLong j = 1, i = 1; j < blocksPerGrid; j += j, ++i)
            mergeShuffleCuda<T><<<blocksPerGrid/(2*j), threadsPerBlock, 256, *context->getCudaStream()>>>(pRng, arr->specialBuffer(), arr->ews(), len, power, i);
        NDArray::registerSpecialUse({arr}, {arr});

        manager.synchronize();

        rng.rewindH((len + 1) * power);
    }
    else {

        auto dimsToExclude = ShapeUtils::evalDimsToExclude(input.rankOf(), {0});

        if(isInplace) {

            auto subArrsList = input.allTensorsAlongDimension(dimsToExclude);

            // Fisher-Yates shuffle
            for(int i = firstDim - 1; i > 0; --i) {
                const int j = rng.relativeInt(i) % (i + 1);
                if(i != j)
                    subArrsList.at(i)->swapUnsafe(*subArrsList.at(j));
            }
        }
        else {

            auto subArrsListIn  = input.allTensorsAlongDimension(dimsToExclude);
            auto subArrsListOut = output.allTensorsAlongDimension(dimsToExclude);

            std::vector<int> indices(firstDim);
            std::iota(indices.begin(), indices.end(), 0);   // 0,1,2,3, ... firstDim-1

            // shuffle indices
            fisherYates<int>(rng, indices.data(), firstDim, 1, 0);

            auto func = PRAGMA_THREADS_FOR {

                for (auto i = start; i < stop; ++i)
                    subArrsListOut.at(i)->assign(subArrsListIn.at(indices[i]));
            };

            samediff::Threads::parallel_for(func, 0, firstDim);
        }

        rng.rewindH(firstDim-1);
    }
}

/////////////////////////////////////////////////////////////////////////
void randomShuffle(sd::LaunchContext * context, NDArray& input, NDArray& output, sd::graph::RandomGenerator& rng, const bool isInplace) {
    BUILD_SINGLE_SELECTOR(input.dataType(), randomShuffle_, (context, input, output, rng, isInplace), LIBND4J_TYPES);
}

// BUILD_SINGLE_TEMPLATE(template void randomShuffle_, (sd::LaunchContext* context, NDArray& input, NDArray& output, sd::graph::RandomGenerator& rng, const bool isInplace), LIBND4J_TYPES);



}
}
}