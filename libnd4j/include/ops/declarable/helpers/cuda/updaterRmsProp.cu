#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Oleh Semeniv (oleg.semeniv@gmail.com)
//

#include <system/op_boilerplate.h>
#include <ops/declarable/helpers/updatersHelpers.h>
#include <helpers/PointersManager.h>
#include <math/platformmath.h>
#include <math/templatemath.h>

namespace sd    {
namespace ops     {
namespace helpers {

///////////////////////////////////////////////////////////////////
template<typename T>
__global__ void rmsPropUpdaterCuda(const void *vx, const Nd4jLong *xShapeInfo, const void *vin, const Nd4jLong *inShapeInfo, 
                                   void *vz, const Nd4jLong *zShapeInfo, void* vst, const Nd4jLong* stShapeInfo,
                                   const T lr, const T rmsDecay, const T epsilon) {

    const auto x = reinterpret_cast<const T*>(vx);
    const auto init = reinterpret_cast<const T*>(vin);
    
          auto up = reinterpret_cast<T*>(vz);
          auto st = reinterpret_cast<T*>(vst);

    __shared__ Nd4jLong xLen;   
    __shared__ bool bEWS, bOrdering, bXZsame, bXInSame, bXStSame;

    if (threadIdx.x == 0) {

        xLen = shape::length(xShapeInfo);
        
        bEWS = 1 == shape::elementWiseStride(xShapeInfo) && 1 == shape::elementWiseStride(zShapeInfo) &&
               1 == shape::elementWiseStride(stShapeInfo) && 1 == shape::elementWiseStride(inShapeInfo);
        
        bOrdering = shape::order(zShapeInfo) == shape::order(xShapeInfo) && shape::order(xShapeInfo) == shape::order(stShapeInfo) &&
            shape::order(xShapeInfo) == shape::order(inShapeInfo);
        bXZsame = shape::haveSameShapeAndStrides(xShapeInfo, zShapeInfo);
        bXInSame = shape::haveSameShapeAndStrides(xShapeInfo, inShapeInfo); 
        bXStSame = shape::haveSameShapeAndStrides(xShapeInfo, stShapeInfo);
    }
    __syncthreads();
    
    int coords[MAX_RANK];

    for (Nd4jLong i = blockIdx.x * blockDim.x + threadIdx.x; i < xLen; i +=  gridDim.x * blockDim.x) {
        
        auto xOffset = i, zOffset = i, initOffset = i, stOffset = i;

        if (!bEWS || !bOrdering) {

            shape::index2coords(i, xShapeInfo, coords);
            xOffset  = shape::getOffset(xShapeInfo, coords);
            zOffset  = bXZsame ? xOffset : shape::getOffset(zShapeInfo, coords);
            initOffset = bXInSame ? xOffset : shape::getOffset(inShapeInfo, coords);
            stOffset = bXStSame ? xOffset : shape::getOffset(stShapeInfo, coords);
        }

        st[stOffset] = init[initOffset] * rmsDecay + x[xOffset] * x[xOffset] * (1 - rmsDecay) ;
        up[zOffset] = (lr * x[xOffset]) / (  math::nd4j_sqrt<T, T>(st[stOffset]) + epsilon);
    }
}

///////////////////////////////////////////////////////////////////
template<typename T>
linkage void rmsPropUpdaterCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const hipStream_t *stream, 
                                        const void *vx, const Nd4jLong *xShapeInfo, const void *vin, const Nd4jLong *inShapeInfo, 
                                        void *vz, const Nd4jLong *zShapeInfo, void* vst, const Nd4jLong* stShapeInfo,
                                        const double dLr, const double dRmsDecay, const double dEpsilon) {
    
    const T lr = static_cast<T>(dLr);
    const T rmsDecay = static_cast<T>(dRmsDecay);
    const T epsilon = static_cast<T>(dEpsilon);

    rmsPropUpdaterCuda<T><<<blocksPerGrid, threadsPerBlock, 256, *stream>>>(vx, xShapeInfo, vin, inShapeInfo, 
                             vz, zShapeInfo, vst, stShapeInfo, lr, rmsDecay, epsilon);
}

///////////////////////////////////////////////////////////////////
void updaterRmsProp(sd::LaunchContext* context, const NDArray& gradient, const NDArray& initState, NDArray& update, NDArray& stateG, 
                    const double dLr, const double dRmsDecay, const double dEpsilon) {

    PointersManager manager(context, "rmsPropUpdater");

    const int threadsPerBlock = MAX_NUM_THREADS / 4;
    const int blocksPerGrid = (gradient.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;

    NDArray::prepareSpecialUse({&update, &stateG}, {&gradient, &initState });

    BUILD_SINGLE_SELECTOR(gradient.dataType(), rmsPropUpdaterCudaLauncher, (blocksPerGrid, threadsPerBlock, 
                          context->getCudaStream(), gradient.specialBuffer(), gradient.specialShapeInfo(),
                          initState.specialBuffer(), initState.specialShapeInfo(),
                          update.specialBuffer(), update.specialShapeInfo(),
                          stateG.specialBuffer(), stateG.specialShapeInfo(),
                          dLr, dRmsDecay, dEpsilon ), FLOAT_TYPES);

    NDArray::registerSpecialUse({&update, &stateG}, {&gradient, &initState});

    manager.synchronize();
}

}
}
}
