#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author raver119@gmail.com
//

#include <ops/declarable/helpers/top_k.h>
#include <helpers/MmulHelper.h>
#include <array/NDArrayFactory.h>
#include <graph/Status.h>
#include <helpers/ConstantTadHelper.h>
#include <helpers/ShapeUtils.h>
//#include <ops/declarable/generic/helpers/BroadcastHelper.h>

#include <hipsolver.h>
#include <exceptions/cuda_exception.h>

namespace sd {
namespace ops {
namespace helpers {

// ------------------------------------------------------------------------------------------------------------------ //
//  invert the second diagonal for lower diagonal matrix
    template<typename T>
    static __global__ void
    invertKernelLow(void *invertedBuf, const Nd4jLong *invertedShape, const void *inputBuf, const Nd4jLong *inputShape, Nd4jLong n) {
        auto inverted = reinterpret_cast<T *>(invertedBuf);
        auto input = reinterpret_cast<const T*>(inputBuf);

        auto start = threadIdx.x + blockIdx.x * blockDim.x;
        auto step = blockDim.x * gridDim.x;

        for (int i = start + 1; i < n; i += step) {
            Nd4jLong pos[] = {i, i - 1};
            Nd4jLong posX[] = {i, i};
            Nd4jLong posY[] = {i - 1, i - 1};
            auto xIndex = shape::getOffset(inputShape, pos);
            auto dxIndex = shape::getOffset(inputShape, posX);
            auto dyIndex = shape::getOffset(inputShape, posY);
            auto zIndex = shape::getOffset(invertedShape, pos);
            // invert lower triangular matrix
            inverted[zIndex] = -input[xIndex] / (input[dxIndex] * input[dyIndex]);
//            math::atomics::nd4j_atomicAdd(&inverted[zIndex], - input[xIndex] * inverted[iIndex] / input[dIndex]);
        }
    }
// ------------------------------------------------------------------------------------------------------------------ //
// invert diagonal vals to upper diagonal matrix
    template<typename T>
    static __global__ void
    upvertKernel(void *invertedBuf, const Nd4jLong *invertedShape, const void *inputBuf, const Nd4jLong *inputShape, Nd4jLong n) {
        auto inverted = reinterpret_cast<T *>(invertedBuf);
        auto input = reinterpret_cast<const T *>(inputBuf);

        auto start = threadIdx.x + blockIdx.x * blockDim.x;
        auto step = blockDim.x * gridDim.x;

        for (int i = start; i < n; i += step) {
            Nd4jLong pos[] = {i, i};
            auto xIndex = shape::getOffset(inputShape, pos);
            auto zIndex = shape::getOffset(invertedShape, pos);

            // invert diagonal elements
            inverted[zIndex] /= input[xIndex];
        }
    }

// ------------------------------------------------------------------------------------------------------------------ //
//  invert upper second diagonal
    template<typename T>
    static __global__ void
    upvertKernelUp(void *invertedBuf, const Nd4jLong *invertedShape, const void *inputBuf, const Nd4jLong *inputShape, Nd4jLong n) {

        __shared__ T* inverted;
        __shared__ const T* input;
        if (threadIdx.x == 0) {
            inverted = reinterpret_cast<T *>(invertedBuf);
            input = reinterpret_cast<const T *>(inputBuf);
        }
        __syncthreads();

        auto start = threadIdx.x + blockIdx.x * blockDim.x;
        auto step = blockDim.x * gridDim.x;

        for (int i = start; i < n - 1; i += step) {
            Nd4jLong pos[] = {i, i + 1};
            Nd4jLong posX[] = {i + 1, i + 1};
            auto xIndex = shape::getOffset(inputShape, pos);
            auto iIndex = shape::getOffset(invertedShape, posX);
            auto zIndex = shape::getOffset(invertedShape, pos);
            // invert upper matrix
            math::atomics::nd4j_atomicAdd(&inverted[zIndex], -input[xIndex] * inverted[iIndex]); // / input[yIndex]);
            //inputMatrix->t<T>(i, i + 1) * invertedMatrix->t<T>(i + 1, i + 1) / inputMatrix->t<T>(i, i)
        }
    }

// ------------------------------------------------------------------------------------------------------------------ //
    template<typename T>
    static __global__ void
    invertLowKernel(void *invertedBuf, const Nd4jLong *invertedShape, const void *inputBuf, const Nd4jLong *inputShape, Nd4jLong n) {

        auto input = reinterpret_cast<const T *>(inputBuf);
        auto inverted = reinterpret_cast<T *>(invertedBuf);


        auto tid = blockIdx.x * blockDim.x + threadIdx.x;
        auto step = gridDim.x * blockDim.x;

        for (int i = tid + 2; i < n; i += step) {
            for (int j = i - 2; j >= 0; --j)
                for (int k = 0; k < i; k++) {
                    Nd4jLong posZ[] = {i, j};
                    Nd4jLong posY[] = {k, j};
                    Nd4jLong posX[] = {i, k};
                    Nd4jLong posD[] = {i, i};

                    auto xIndex = shape::getOffset(inputShape, posX);
                    auto yIndex = shape::getOffset(invertedShape, posY);
                    auto dIndex = shape::getOffset(inputShape, posD);
                    auto zIndex = shape::getOffset(invertedShape, posZ);
                    // invert non-diagonal elements
                    math::atomics::nd4j_atomicAdd(&inverted[zIndex], -inverted[yIndex] * input[xIndex] / input[dIndex]);
                }
        }
    }

// ------------------------------------------------------------------------------------------------------------------ //
// Invertion of upper triangular matrix non-diagonal elements when main and second diagonals already processed
    template<typename T>
    static __global__ void
    invertUpKernel(
            void *invertedBuf, const Nd4jLong *invertedShape,
            const void *inputBuf, const Nd4jLong *inputShape,
            Nd4jLong n) {

        auto inverted = reinterpret_cast<T *>(invertedBuf);;
        auto input = reinterpret_cast<const T *>(inputBuf);

        auto tid = blockIdx.x * blockDim.x + threadIdx.x;
        auto step = blockDim.x * gridDim.x;

        for (int i = (int)n - tid - 2; i >= 0; i -= step) {
            for (int j = i + 2; j < (int)n; j++)
                for (int k = i; k < (int)n; k++) {
                    Nd4jLong posZ[] = {i, j};
                    Nd4jLong posY[] = {k, j};
                    Nd4jLong posX[] = {i, k};
                    // inversion with Joardan Gauss transformation
                    auto xIndex = shape::getOffset(inputShape, posX);
                    auto yIndex = shape::getOffset(invertedShape, posY);
                    auto zIndex = shape::getOffset(invertedShape, posZ);
                    // invert upper non-diagonal elements
                    math::atomics::nd4j_atomicAdd(&inverted[zIndex], -inverted[yIndex] * input[xIndex]);
                }
        }
    }

// ------------------------------------------------------------------------------------------------------------------ //
// procedure to invert lower-triangular matrix.
// In current case lower triangular matrix has main diagonal with general values
//
    template<typename T>
    static void invertLowerMatrix_(LaunchContext *context, NDArray *inputMatrix, NDArray *invertedMatrix) {
        int n = inputMatrix->rows();
        invertedMatrix->setIdentity();

        if (inputMatrix->isIdentityMatrix()) return;

        auto stream = context->getCudaStream();

        // invert lower matrix
        // invert main diagonal
        upvertKernel<T><<<1, n, 512, *stream>>>(invertedMatrix->specialBuffer(), invertedMatrix->specialShapeInfo(), inputMatrix->specialBuffer(), inputMatrix->specialShapeInfo(), n);
        // invert the second diagonal
        invertKernelLow<T><<<1, n, 512, *stream>>>(invertedMatrix->specialBuffer(), invertedMatrix->specialShapeInfo(), inputMatrix->specialBuffer(), inputMatrix->specialShapeInfo(), n);
        // invert non-diagonal elements
        invertLowKernel<T><<<n, n, 512, *stream>>>(invertedMatrix->specialBuffer(), invertedMatrix->specialShapeInfo(), inputMatrix->specialBuffer(), inputMatrix->specialShapeInfo(), n);
    }

// ------------------------------------------------------------------------------------------------------------------ //
// caller for invert lower matrix routine
    void invertLowerMatrix(LaunchContext *context, NDArray *inputMatrix, NDArray *invertedMatrix) {
        NDArray::prepareSpecialUse({invertedMatrix}, {inputMatrix});
        BUILD_SINGLE_SELECTOR(inputMatrix->dataType(), invertLowerMatrix_, (context, inputMatrix, invertedMatrix), FLOAT_NATIVE);
        NDArray::registerSpecialUse({invertedMatrix}, {inputMatrix});
    }

// ------------------------------------------------------------------------------------------------------------------ //
// procedure to invert upper-triangular matrix.
// In current case upper triangular matrix has main diagonal with all ones on it.
    template<typename T>
    static void invertUpperMatrix_(LaunchContext *context, NDArray* inputMatrix, NDArray* invertedMatrix) {
        int n = inputMatrix->rows();
        invertedMatrix->setIdentity();
        auto stream = context->getCudaStream();
        if (inputMatrix->isIdentityMatrix()) { // the inverse for I is I
            return;
        }

        // invert upper matrix
        // invert the second diagonal
        upvertKernelUp<T><<<1, n, 512, *stream >>>(invertedMatrix->specialBuffer(), invertedMatrix->specialShapeInfo(),
                inputMatrix->specialBuffer(), inputMatrix->specialShapeInfo(), n);

        // invert other elements
        invertUpKernel<T><<<n, n, 512, *stream >>>(invertedMatrix->specialBuffer(), invertedMatrix->specialShapeInfo(),inputMatrix->specialBuffer(), inputMatrix->specialShapeInfo(), n);
    }

// ------------------------------------------------------------------------------------------------------------------ //
//  invertion of upper triangular matrix - runner routine
    void invertUpperMatrix(LaunchContext *context, NDArray *inputMatrix, NDArray *invertedMatrix) {
        NDArray::prepareSpecialUse({invertedMatrix}, {inputMatrix});
        BUILD_SINGLE_SELECTOR(invertedMatrix->dataType(), invertUpperMatrix_, (context, inputMatrix, invertedMatrix), FLOAT_NATIVE);
        NDArray::prepareSpecialUse({invertedMatrix}, {inputMatrix});
    }

// ------------------------------------------------------------------------------------------------------------------ //
    // determinant kernel - accumulation product of all values on the main diagonal
    template<typename T>
    static __global__ void determinantKernel(T *compound, T *result, Nd4jLong len) {
        auto start = blockIdx.x * blockDim.x + threadIdx.x;
        auto step = blockDim.x * gridDim.x;
        for (auto i = start; i < len; i += step) {
            auto pos = i * len + i; //shape::getOffset(0, shape::shapeOf(shape), shape::stride(shape), di, 2);
            // multiply all diagonal elements
            math::atomics::nd4j_atomicMul(&result[0], compound[pos]);
        }
    }

// ------------------------------------------------------------------------------------------------------------------ //
    // determinant logarithm - accumulation sum of all logarithm values on the main diagonal. All in logarithic values
    // should be positive
    template<typename T>
    static __global__ void determinantLogKernel(T *compound, T *result, Nd4jLong len) {
        auto start = blockIdx.x * blockDim.x + threadIdx.x;
        auto step = blockDim.x * gridDim.x;
        for (auto i = start; i < len; i += step) {
            auto pos = i * len + i; //shape::getOffset(0, shape::shapeOf(shape), shape::stride(shape), di, 2);
            // sum logs of all diagonal elements
            math::atomics::nd4j_atomicAdd(result, math::nd4j_log<T,T>(math::nd4j_abs(compound[pos])));
        }
    }

// ------------------------------------------------------------------------------------------------------------------ //
    // kernel to copy matrix with given shape to compound tensor with given pos
    // output - a N-D tensor buffer with rank not less than 2, input - 2D square n x n matrix with n = rowLen
    template<typename T, typename F>
    static __global__ void
    fillMatrix(void *output, const Nd4jLong *outShape, const void *input, const Nd4jLong *inputShape, Nd4jLong pos, Nd4jLong rowLen) {
        __shared__ F *matrix;
        __shared__ const T *inputBuf;
        __shared__ Nd4jLong inputLen;
        __shared__ Nd4jLong n2;

        if (threadIdx.x == 0) {
            matrix = reinterpret_cast<F*>(output);
            inputBuf = reinterpret_cast<const T*>(input);
            inputLen = shape::length(inputShape);
            n2 = rowLen * rowLen;
        }
        __syncthreads();

        auto start = blockIdx.x * blockDim.x + threadIdx.x;
        auto step = blockDim.x * gridDim.x;

        for (int k = pos + start, j = start; j < n2; k += step, j += step) {
            auto xIndex = shape::getIndexOffset(k, inputShape);
            matrix[j] = (F) inputBuf[xIndex];
        }
    }

// ------------------------------------------------------------------------------------------------------------------ //
// same as above, but without type conversion
    template<typename T>
    static __global__ void
    returnMatrix(void *output, const Nd4jLong *outputShape, const void *input, const Nd4jLong *inputShape, Nd4jLong pos, Nd4jLong rowLen) {
        __shared__ Nd4jLong outputLen;
        __shared__ Nd4jLong n2;
        auto matrix = reinterpret_cast<const T *>(input);
        auto outputBuf = reinterpret_cast<T *>(output);

        if (threadIdx.x == 0) {

            outputLen = shape::length(inputShape);
            n2 = rowLen * rowLen;
        }
        __syncthreads();
        auto start = blockIdx.x * blockDim.x + threadIdx.x;
        auto step = blockDim.x * gridDim.x;

        for (int k = pos + start, j = start; j < n2; k += step, j += step) {
            auto zIndex = shape::getIndexOffset(k, outputShape);
            outputBuf[zIndex] = matrix[j];
        }
    }

// ------------------------------------------------------------------------------------------------------------------ //
    // fill up permutaion matrix kernel. Permutation matrix filled with zeros and ones
    template<typename F>
    static __global__ void fillUpPermutation(void *output, const Nd4jLong *shape, int *source, int rowNum) {
        F *permutation = reinterpret_cast<F *>(output);

        auto start = blockIdx.x * blockDim.x + threadIdx.x;
        auto step = blockDim.x * gridDim.x;
        for (auto i = start; i < rowNum; i += step) {
            int val = source[i] - 1;
            Nd4jLong posF[] = {i, val};
            auto pos = shape::getOffset(shape, posF);
            permutation[pos] = F(1.f);
        }
    }

// ------------------------------------------------------------------------------------------------------------------ //
    // LUP decomposition runner - using CUBLAS SOLVER
    // if permutation is given, then using LUP decomposition, LU decomposition otherwise
    // L - lower triangular, U - upper triangular, P - permutation matricies
    // PA = LU
    //
    // input - A matrix nxn
    // compound - C matrix L + U - I, or main diagonal and lower - L matrix, from the 2nd diagonal - U matrix
    template<typename T, typename I>
    static void lup_(LaunchContext *context, NDArray *input, NDArray *compound, NDArray *permutation) {
        auto stream = context->getCudaStream();
        auto n = input->rows();
        std::lock_guard<std::mutex> lock(*LaunchContext::deviceMutex());

        hipsolverHandle_t* cusolverH = (hipsolverHandle_t*)context->getCusolverHandle(); //nullptr;
        // create solver handle
        hipsolverStatus_t status; //hipsolverDnCreate(&cusolverH);
//        if (HIPSOLVER_STATUS_SUCCESS != status) {
//            throw cuda_exception::build("Cannot create cuSolver handle", status);
//        }
        // set solver stream
        status = hipsolverSetStream(*cusolverH, *stream);
        if (HIPSOLVER_STATUS_SUCCESS != status) {
            throw cuda_exception::build("Cannot set up stream for cuda solver", status);
        }
        int lwork = 0;
        int *d_info = nullptr;
        // allocate memory for permutation vector
        auto err = hipMalloc((void **) &d_info, sizeof(int));
        if (err) {
            throw cuda_exception::build("helpers::lup_: Cannot allocate memory for solver info buffer", err);
        }

        DataType dtype = input->dataType();
        switch (dtype) { // there are two implementations with cublas for LUP decomposition - double and float

            case DataType::DOUBLE: {
                double *d_work = nullptr;
                // compute internal buffer size
                double *matrix = reinterpret_cast<double *>(input->specialBuffer());
                status = hipsolverDnDgetrf_bufferSize(
                        *cusolverH,
                        n,
                        n,
                        matrix,
                        n,
                        &lwork);
                if (HIPSOLVER_STATUS_SUCCESS != status) {
                    throw cuda_exception::build("helpers::lup_: Cannot create cuSolver handle", status);
                }

                err = hipMalloc((void **) &d_work, sizeof(float) * lwork);
                if (err) {
                    throw cuda_exception::build("helpers::lup_: Cannot allocate memory for solver data buffer",
                                                err);
                }

                if (permutation == nullptr) {
                    status = hipsolverDnDgetrf(
                            *cusolverH,
                            n,
                            n,
                            matrix,
                            n,
                            d_work,
                            nullptr,
                            d_info);

                    if (status != HIPSOLVER_STATUS_SUCCESS) {
                        throw cuda_exception::build("helpers::lup_: LU factorization is failed due ",
                                                    status);
                    }
                }
                else {
                    NDArray permutVector('c', {n}, sd::DataType::INT32, context);
                    int* permutationBuf = permutVector.dataBuffer()->specialAsT<int>();
                    status = hipsolverDnDgetrf(
                            *cusolverH,
                            n,
                            n,
                            matrix,
                            n,
                            d_work,
                            permutationBuf,
                            d_info);
                    if (status != HIPSOLVER_STATUS_SUCCESS) {
                        throw cuda_exception::build("helpers::lup_: LU factorization is failed due ",
                                                    status);
                    }

                    if (permutation->rankOf() == 2) {
                        fillUpPermutation<double> <<< n, n, 1024, *stream >>>
                                                                  (permutation->specialBuffer(), permutation->specialShapeInfo(), permutationBuf, n);
                    }
                    else {
                        permutVector.tickWriteDevice();
                        input->tickWriteDevice();
                        compound->assign(input);
                        permutation->assign(permutVector);
                    }
                }
                err = hipFree(d_work);
                if (err) {
                    throw cuda_exception::build("helpers::lup_: Cannot deallocate memory for solver data buffer",
                                                err);
                }
            }
                break;
            case DataType::FLOAT32: {
                float *matrix = reinterpret_cast<float*>(input->specialBuffer());
                float *d_work = nullptr;

                status = hipsolverDnSgetrf_bufferSize(
                        *cusolverH,
                        n,
                        n,
                        matrix,
                        n,
                        &lwork);
                if (HIPSOLVER_STATUS_SUCCESS != status) {
                    throw cuda_exception::build("helpers::lup_: Cannot create cuSolver handle", status);
                }

                err = hipMalloc((void **) &d_work, sizeof(float) * lwork);
                if (err) {
                    throw cuda_exception::build("helpers::lup_: Cannot allocate memory for solver data buffer",
                                                err);
                }

                if (permutation == nullptr)
                    status = hipsolverDnSgetrf(
                            *cusolverH,
                            n,
                            n,
                            matrix,
                            n,
                            d_work,
                            nullptr,
                            d_info);
                else {
                    NDArray permutVector('c', {n}, DataType::INT32, context);
                    int *permutationBuf = reinterpret_cast<int *>(permutVector.specialBuffer());
                    status = hipsolverDnSgetrf(
                            *cusolverH,
                            n,
                            n,
                            matrix,
                            n,
                            d_work,
                            permutationBuf,
                            d_info);
                    if (permutation->rankOf() == 2) {
                        fillUpPermutation<I> <<< n, n, 128, *stream >>>
                                                             (permutation->specialBuffer(), permutation->specialShapeInfo(), permutationBuf, n);
                        permutation->tickWriteDevice();
                    }
                    else {
                        input->tickWriteDevice();
                        compound->assign(input);
                        permutation->assign(permutVector);
                    }
                }
                err = hipFree(d_work);
                if (err) {
                    throw cuda_exception::build("helpers::lup_: Cannot deallocate memory for solver data buffer",
                                                err);
                }

            }
        }
        if (HIPSOLVER_STATUS_SUCCESS != status) {
            throw cuda_exception::build("helpers::lup_: Cannot make LU decomposition", status);
        }
        err = hipFree(d_info);
        if (err) {
            throw cuda_exception::build("helpers::lup_: Cannot deallocate memory for solver info buffer", err);
        }
//        hipsolverDnDestroy(cusolverH);
//        NDArray::registerSpecialUse({input}, {input});
        input->tickWriteDevice();
    }
// ------------------------------------------------------------------------------------------------------------------ //

    BUILD_DOUBLE_TEMPLATE(template void lup_,(LaunchContext * context, NDArray * input, NDArray * output, NDArray * permutation), FLOAT_NATIVE, INDEXING_TYPES);

    template <typename T>
    static __device__ void  swapRows(T* matrix, const Nd4jLong* shape, Nd4jLong theFirst, Nd4jLong theSecond, Nd4jLong n) {
        if (theFirst != theSecond) {
            for (auto i = 0; i < n; i++) {
                Nd4jLong theFirstPos[] = {theFirst, i};
                Nd4jLong theSecondPos[] = {theSecond, i};
                auto theFirstIndex = shape::getOffset(shape, theFirstPos, 0);
                auto theSecondIndex = shape::getOffset(shape, theSecondPos, 0);
                math::nd4j_swap(matrix[theFirstIndex], matrix[theSecondIndex]);
            }
        }
    }

    template <typename T>
    static __device__ void processColumns(Nd4jLong currentRow, Nd4jLong rowNum, T* compoundBuf, const Nd4jLong* compoundShape) {
        Nd4jLong xDiag[] = {currentRow, currentRow};
        auto diagIndex = shape::getOffset(compoundShape, xDiag, 0);
        for (auto j = currentRow + 1; j < rowNum; j++) {
            Nd4jLong xRow[] = {j, currentRow};
            auto rowIndex = shape::getOffset(compoundShape, xRow, 0);
            compoundBuf[rowIndex] /= compoundBuf[diagIndex]; //output->t<T>(i, i);
            for (auto k = currentRow + 1; k < rowNum; k++) {
                Nd4jLong yRow[] = {j, k};
                Nd4jLong yCol[] = {currentRow, k};
                auto rowIndexY = shape::getOffset(compoundShape, yRow, 0);
                auto colIndex = shape::getOffset(compoundShape, yCol, 0);
                compoundBuf[rowIndexY] -= compoundBuf[rowIndex] * compoundBuf[colIndex];
            }
        }
    }

    template <typename T>
    __device__ Nd4jLong argmaxCol(Nd4jLong column, T* compoundBuffer, const Nd4jLong* compoundShape) {
        auto rowNum = shape::sizeAt(compoundShape, 0);
        Nd4jLong xInitial[] = {column, column};
        auto xInitialIndex = shape::getOffset(compoundShape, xInitial, 0);
        auto maxValue = T(0); //sd::math::nd4j_abs(compoundBuffer[xInitialIndex]);
        auto result = -1LL;

        for (auto rowCounter = column; rowCounter < rowNum; rowCounter++) {
            Nd4jLong xPos[] = {rowCounter, column};
            auto xIndex = shape::getOffset(compoundShape, xPos, 0);
            if (sd::math::nd4j_abs(compoundBuffer[xIndex]) > maxValue) {
                maxValue = sd::math::nd4j_max(maxValue, sd::math::nd4j_abs(compoundBuffer[xIndex]));
                result = rowCounter;
            }
        }
        return result;
    }

        template <typename T, typename I>
    static __device__ int  luNN(T* matrix, const Nd4jLong* shape, I* permutation, const Nd4jLong* permuShape, Nd4jLong n) {

        for (auto i = 0; i < n - 1; i++) {
            auto pivotIndex = argmaxCol(i, matrix, shape);
            if (pivotIndex < 0) {
                return -1;//throw std::runtime_error("helpers::luNN_: input matrix is singular.");
            }
            math::nd4j_swap(permutation[shape::getIndexOffset(i, permuShape)], permutation[shape::getIndexOffset(pivotIndex, permuShape)]);
            swapRows(matrix, shape, (Nd4jLong)i, pivotIndex, n);

            processColumns(i, n, matrix, shape);
        }
        return 0;
    }

    template <typename T, typename I>
    static __global__ void luBatchedKernel(
            T* outputBuf, const Nd4jLong* outputShape,
            I* permutations, const Nd4jLong* permuShape,
            const Nd4jLong* outputTadShape, const Nd4jLong* outputTadOffsets,
            const Nd4jLong* permuTadShape, const Nd4jLong* permuTadOffsets,
            Nd4jLong batchNum) {

        auto start = blockIdx.x * blockDim.x + threadIdx.x;
        auto step = blockDim.x * gridDim.x;

        for (auto b = start; b < batchNum; b += step) {
            T* matrix = outputBuf + outputTadOffsets[b];
            I* permutation = permutations + permuTadOffsets[b];

            if (0 != luNN(matrix, outputTadShape, permutation, permuTadShape, shape::length(permuTadShape))) break;
        }
    }

    template <typename T, typename I>
    static void lu_(LaunchContext * context, NDArray* input, NDArray* output, NDArray* permutationVectors) {
        auto n = input->sizeAt(-1);
        auto stream = context->getCudaStream();
        NDArray iota('c', {n}, permutationVectors->dataType(), context);// = NDArrayFactory::create(); // <int>('c', {n});
        iota.linspace(0); iota.syncToDevice();

        output->assign(input); // fill up output tensor with zeros
//        output->tickWriteDevice();
        permutationVectors->applyTrueBroadcast(sd::BroadcastOpsTuple::Assign(), iota, *permutationVectors, true, nullptr);
//        permutationVectors->tickWriteDevice();
        auto tads = ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(), {-2, -1});
        auto permutaionTads = ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(), {-1});
        auto batchNum = tads.numberOfTads();
        luBatchedKernel<T,I><<<batchNum, 256, 1024, *stream>>>(reinterpret_cast<T*>(output->platformBuffer()),
                output->specialShapeInfo(), reinterpret_cast<I*>(permutationVectors->platformBuffer()),
                permutationVectors->specialShapeInfo(), tads.specialShapeInfo(), tads.specialOffsets(),
                permutaionTads.specialShapeInfo(), permutaionTads.specialOffsets(), batchNum);
    }

    void lu(LaunchContext* context, NDArray* input, NDArray* output, NDArray* permutations) {
        NDArray::prepareSpecialUse({output, permutations}, {input});
        BUILD_DOUBLE_SELECTOR(input->dataType(), permutations->dataType(), lu_, (context, input, output, permutations), FLOAT_NATIVE, INDEXING_TYPES);
        NDArray::registerSpecialUse({output, permutations}, {input});
    }
// ------------------------------------------------------------------------------------------------------------------ //
    template<typename T>
    static int determinant_(sd::LaunchContext *context, NDArray *input, NDArray *output) {
        Nd4jLong n = input->sizeAt(-1);
        Nd4jLong n2 = n * n;
        std::vector<int> dims();
        auto packX = ConstantTadHelper::getInstance().tadForDimensions(input->shapeInfo(), {input->rankOf() - 2, input->rankOf() - 1});
        //auto packZ = ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(), {output->rankOf() - 1});
//        DataType dtype = input->dataType();
//        if (dtype != DataType::DOUBLE)
//            dtype = DataType::FLOAT32;
        auto matrix = NDArrayFactory::create(input->ordering(), {n, n}, DataTypeUtils::fromT<T>(), context); //, block.getWorkspace());
        auto det = NDArrayFactory::create<T>(1, context);
        auto stream = context->getCudaStream();
        NDArray::prepareSpecialUse({output}, {input});
        dim3 launchDims(256, 256, 1024);
        output->assign(1.f);
        for (int e = 0; e < output->lengthOf(); e++) {
            Nd4jLong pos = e * n2;
//            if (matrix.dataType() == input->dataType())
            fillMatrix<T, T><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(matrix.specialBuffer(), matrix.specialShapeInfo(), input->specialBuffer(), input->specialShapeInfo(), pos, n);
//            else
//                fillMatrix<T, float><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(matrix.specialBuffer(), matrix.specialShapeInfo(), input->specialBuffer(), input->special(), pos, n);
            lup_<T, int>(context, &matrix, nullptr, nullptr);
//            else
//                lup_<float>(context, &matrix, nullptr, nullptr);
            auto offset = shape::getIndexOffset(e, output->shapeInfo());
            auto inputBuf = reinterpret_cast<T *>(matrix.specialBuffer());
            auto outputBuf = reinterpret_cast<T *>(output->specialBuffer()) + offset;
//            if (matrix.dataType() == input->dataType())
            determinantKernel<T><<< launchDims.x, launchDims.y, launchDims.z, *stream>>>(inputBuf, outputBuf, n);
//            else
//                determinantKernel<T, float><<<launchDims.x, launchDims.y, launchDims.z, *stream >>> (inputBuf, outputBuf, n);
        }
        NDArray::registerSpecialUse({output}, {input});

        return Status::OK();
    }

        int determinant(sd::LaunchContext *context, NDArray *input, NDArray *output) {
            NDArray::prepareSpecialUse({output}, {input});
            BUILD_SINGLE_SELECTOR(input->dataType(), return determinant_, (context, input, output), FLOAT_NATIVE);
            NDArray::registerSpecialUse({output}, {input});
        }

        template<typename T>
        int logAbsDeterminant_(LaunchContext *context, NDArray *input, NDArray *output) {
            Nd4jLong n = input->sizeAt(-1);
            Nd4jLong n2 = n * n;
            std::vector<int> dims();
            auto packX = ConstantTadHelper::getInstance().tadForDimensions(input->shapeInfo(), {input->rankOf() - 2, input->rankOf() - 1});
            //auto packZ = ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(), {output->rankOf() - 1});
            DataType dtype = input->dataType();
            if (dtype != DataType::DOUBLE)
                dtype = DataType::FLOAT32;

            auto matrix = NDArrayFactory::create(input->ordering(), {n, n}, dtype, context); //, block.getWorkspace());
            auto det = NDArrayFactory::create<T>(1, context);
            auto stream = context->getCudaStream();
            NDArray::prepareSpecialUse({output}, {input});
            dim3 launchDims(256, 256, 1024);
            output->assign(0.f);
            for (int e = 0; e < output->lengthOf(); e++) {
                Nd4jLong pos = e * n2;
//            if (matrix.dataType() == input->dataType())
                fillMatrix<T, T><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(matrix.specialBuffer(), matrix.specialShapeInfo(), input->specialBuffer(), input->specialShapeInfo(), pos, n);
//            else
//                fillMatrix<T, float><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(matrix.specialBuffer(), matrix.specialShapeInfo(), input->specialBuffer(), input->special(), pos, n);

//            if (matrix.dataType() == input->dataType())
                lup_<T, int>(context, &matrix, nullptr, nullptr);
//            else
//                lup_<float>(context, &matrix, nullptr, nullptr);
                auto offset = shape::getIndexOffset(e, output->shapeInfo());
                auto inputBuf = reinterpret_cast<T *>(matrix.specialBuffer());
                auto outputBuf = reinterpret_cast<T *>(output->specialBuffer()) + offset;
//            if (matrix.dataType() == input->dataType())
                determinantLogKernel<T><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(inputBuf, outputBuf, n);
//            else
//                determinantLogKernel<T, float><<<launchDims.x, launchDims.y, launchDims.z, *stream >>> (inputBuf, outputBuf, n);
            }
            NDArray::registerSpecialUse({output}, {input});

            return Status::OK();

            return ND4J_STATUS_OK;
        }

        int logAbsDeterminant(sd::LaunchContext *context, NDArray *input, NDArray *output) {
            NDArray::prepareSpecialUse({output}, {input});
            BUILD_SINGLE_SELECTOR(input->dataType(), return logAbsDeterminant_, (context, input, output), FLOAT_NATIVE);
            NDArray::registerSpecialUse({output}, {input});
        }

        template<typename T>
        static __global__ void
        fillLowerUpperKernel(
                void *lowerBuf, const Nd4jLong *lowerShape,
                void *upperBuf, const Nd4jLong *upperShape,
                void *matrixBuf, const Nd4jLong *matrixShape,
                Nd4jLong n) {

            __shared__ T *lowerMatrix;
            __shared__ T *upperMatrix;
            __shared__ T *matrix;

            if (threadIdx.x == 0) {
                lowerMatrix = reinterpret_cast<T *>(lowerBuf);
                upperMatrix = reinterpret_cast<T *>(upperBuf);
                matrix = reinterpret_cast<T *>(matrixBuf);
            }
            __syncthreads();

            for (int k = blockIdx.x; k < n; k += gridDim.x) {  // and then put all values under main diagonal on to it
                for (int j = threadIdx.x; j < n; j += blockDim.x) {
                    Nd4jLong posX[] = {k, j};
                    Nd4jLong posD[] = {j, j};
                    auto xPos = shape::getOffset(lowerShape, posX);
                    auto yPos = shape::getOffset(upperShape, posX);
                    auto iPos = shape::getOffset(matrixShape, posX);
                    auto dPos = shape::getOffset(matrixShape, posD);
                    if (k >= j)
                        lowerMatrix[xPos] = matrix[iPos];//(k, j);
                    else
                        upperMatrix[yPos] = matrix[iPos]; //k, j);
                }
            }
        }

        template<typename T>
        static int inverse_(sd::LaunchContext *context, NDArray *input, NDArray *output) {
            auto n = input->sizeAt(-1);
            auto n2 = n * n;
            auto dtype = DataTypeUtils::fromT<T>(); //input->dataType();
//            if (dtype != DataType::DOUBLE)
//                dtype = DataType::FLOAT32;
            NDArray matrix = NDArrayFactory::create('c', {n, n}, dtype, context);
            NDArray upper = NDArrayFactory::create('c', {n, n}, dtype, context);
            NDArray lower = NDArrayFactory::create('c', {n, n}, dtype, context);
            NDArray compound = NDArrayFactory::create('c', {n, n}, dtype, context);
            NDArray permutation = NDArrayFactory::create('c', {n, n}, dtype, context);
            auto packX = sd::ConstantTadHelper::getInstance().tadForDimensions(input->shapeInfo(),
                                                                                  {input->rankOf() - 2,
                                                                                   input->rankOf() - 1});
            auto packZ = sd::ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(),
                                                                                  {output->rankOf() - 2,
                                                                                   output->rankOf() - 1});
            auto stream = context->getCudaStream();

            for (auto i = 0LL; i < packX.numberOfTads(); i++) {
                fillMatrix<T, T><<<1, n2, 1024, *stream>>>(matrix.specialBuffer(), matrix.specialShapeInfo(), input->specialBuffer(), input->specialShapeInfo(), i * n2, n);
                matrix.tickWriteDevice();
                //compound.assign(matrix);
//            if (matrix.dataType() == input->dataType())
                lup_<T, int>(context, &matrix, nullptr, nullptr);
                fillLowerUpperKernel<T><<<n, n, 1024, *stream>>>(lower.specialBuffer(), lower.specialShapeInfo(), upper.specialBuffer(), upper.specialShapeInfo(), matrix.specialBuffer(), matrix.specialShapeInfo(), n);
                lower.tickWriteDevice();
                upper.tickWriteDevice();
//                lower.printIndexedBuffer("LOWER");
//                upper.printIndexedBuffer("UPPER");
                matrix.assign(0);
                invertUpperMatrix(context, &upper, &matrix); // U^{-1}
                matrix.tickWriteDevice();
//                matrix.printIndexedBuffer("Upper Inverted");
                compound.assign(0);
                invertLowerMatrix(context, &lower, &compound); // L{-1}
                compound.tickWriteDevice();
//                compound.printIndexedBuffer("Lower Inverted");
//                matrix.tickWriteDevice();
//                compound.tickWriteDevice();
                sd::MmulHelper::mmul(&matrix, &compound, &upper, 1.0, 0.0);
                upper.tickWriteDevice();
//                upper.printIndexedBuffer("Full inverted");
                returnMatrix<T><<<1, n2, 1024, *stream>>>(output->specialBuffer(), output->specialShapeInfo(), upper.specialBuffer(), upper.specialShapeInfo(), i * n2, n);
            }
            return Status::OK();
        }

        int inverse(sd::LaunchContext *context, NDArray *input, NDArray *output) {
            NDArray::prepareSpecialUse({output}, {input});
            BUILD_SINGLE_SELECTOR(input->dataType(), return inverse_, (context, input, output), FLOAT_NATIVE);
            NDArray::registerSpecialUse({output}, {input});
        }

        bool checkCholeskyInput(sd::LaunchContext *context, NDArray const *input) {
            return true;
        }

        template<typename F>
        __global__ void fillBatchKernel(F **dArrayBatch, F *buf, const Nd4jLong *offsets, Nd4jLong batchSize) {
            auto start = blockIdx.x * blockDim.x + threadIdx.x;
            auto step = blockDim.x * gridDim.x;

            for (auto i = start; i < batchSize; i += step) {
                dArrayBatch[i] = buf + offsets[i];
            }
        }

        template<typename F>
        __global__ void
        adjustResultsKernel(F *dArray, const Nd4jLong *shape, const Nd4jLong *offsets, Nd4jLong batchSize, Nd4jLong n) {
            //auto i = blockIdx.x * blockDim.x + threadIdx.x;
            Nd4jLong *shapeOf = shape::shapeOf(shape);
            Nd4jLong *strideOf = shape::stride(shape);

            for (auto i = blockIdx.x; i < batchSize; i += gridDim.x) {
                auto current = dArray + offsets[i];
                for (auto r = threadIdx.x; r < n; r += blockDim.x) {
                    for (auto c = r + 1; c < n; c++) {
                        Nd4jLong posRC[] = {r, c};
                        auto pos = r * n + c; //shape::getOffset(0, shapeOf, strideOf, posRC, 2);
                        current[pos] = 0.;
                    }
                }
            }
        }

        template<typename F>
        int cholesky__(LaunchContext *context, NDArray *input, NDArray *output, bool inplace) {
            if (!inplace)
                output->assign(input);
            auto tempOutput =output->dup();
            hipsolverHandle_t handle = nullptr;
            auto n = input->sizeAt(-1);
            auto n2 = n * n;
            NDArray::prepareSpecialUse({output}, {input});
            auto status = hipsolverDnCreate(&handle);
            if (HIPSOLVER_STATUS_SUCCESS != status) {
                throw cuda_exception::build("helpers::cholesky_: Cannot create solver handle", status);
            }
            F **dArrayBatch = nullptr;
            auto packX = sd::ConstantTadHelper::getInstance().tadForDimensions(tempOutput.shapeInfo(),
                                                                                  {tempOutput.rankOf() - 2,
                                                                                   tempOutput.rankOf() - 1});
            const Nd4jLong batchSize = packX.numberOfTads();
            int *dInfoArray = nullptr;
            auto err = hipMalloc((void **) &dArrayBatch, sizeof(F *) * batchSize);
            if (err) {
                throw cuda_exception::build("helpers::cholesky_: Cannot allocate memory for solver batch data buffer",
                                            err);
            }
            err = hipMalloc((void **) &dInfoArray, sizeof(int) * batchSize);
            if (err) {
                throw cuda_exception::build("helpers::cholesky_: Cannot allocate memory for solver errors buffer", err);
            }
            auto stream = context->getCudaStream();
            fillBatchKernel<F><<<1, batchSize, 128, *stream>>>(dArrayBatch, reinterpret_cast<F *>(tempOutput.specialBuffer()), packX.specialOffsets(), batchSize);

            status = hipsolverSetStream(handle, *stream);
            if (HIPSOLVER_STATUS_SUCCESS != status) {
                throw cuda_exception::build("helpers::cholesky_: Cannot set stream to solver handle", status);
            }
            const hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_UPPER;
            if (input->dataType() == DataType::DOUBLE)
                status = hipsolverDnDpotrfBatched(
                        handle,
                        uplo,
                        n,
                        (double **) dArrayBatch,
                        n,
                        dInfoArray,
                        batchSize);
            else
                status = hipsolverDnSpotrfBatched(
                        handle,
                        uplo,
                        n,
                        (float **) dArrayBatch,
                        n,
                        dInfoArray,
                        batchSize);

            if (HIPSOLVER_STATUS_SUCCESS != status) {
                throw cuda_exception::build("helpers::cholesky_: Cholesky factorization failed for batch", status);
            }
            adjustResultsKernel<F><<<batchSize, n2, 128, *stream>>>(reinterpret_cast<F *>(tempOutput.specialBuffer()), packX.specialShapeInfo(), packX.specialOffsets(), batchSize, n);

            err = hipFree(dArrayBatch);
            if (err) {
                throw cuda_exception::build("helpers::cholesky_: Cannot deallocate memory for solver batch data buffer",
                                            err);
            }
            err = hipFree(dInfoArray);
            if (err) {
                throw cuda_exception::build("helpers::cholesky_: Cannot allocate memory for solver errors buffer", err);
            }

            if (!inplace)
                output->assign(tempOutput);
            else
                input->assign(tempOutput);

            NDArray::registerSpecialUse({output}, {input});
            return Status::OK();
        }

//    template <typename T>
        int cholesky_(LaunchContext *context, NDArray *input, NDArray *output, bool inplace) {
            NDArray::prepareSpecialUse({output}, {input});
            if (input->dataType() == DataType::DOUBLE)
                cholesky__<double>(context, input, output, inplace);
            else if (input->dataType() == DataType::FLOAT32)
                cholesky__<float>(context, input, output, inplace);
            else {
                std::unique_ptr<NDArray> tempOutput(
                        NDArrayFactory::create_('c', input->getShapeAsVector(), DataType::FLOAT32, context));
                tempOutput->assign(input);
                cholesky__<float>(context, tempOutput.get(), tempOutput.get(), true);
                output->assign(tempOutput.get());
            }
            NDArray::registerSpecialUse({output}, {input});
            return Status::OK();
        }

        int cholesky(sd::LaunchContext *context, NDArray *input, NDArray *output, bool inplace) {
//        BUILD_SINGLE_SELECTOR(input->dataType(), return cholesky_, (context, input, output, inplace), FLOAT_TYPES);
            return cholesky_(context, input, output, inplace);
        }
//    BUILD_SINGLE_TEMPLATE(template int cholesky_, (LaunchContext* context, NDArray* input, NDArray* output, bool inplace), FLOAT_TYPES);
        BUILD_SINGLE_TEMPLATE(template int inverse_, (sd::LaunchContext * context, NDArray * input, NDArray * output),
                              FLOAT_NATIVE);

        template<typename T>
        __global__ void logDetKernel(
                const T *inputBuf, const Nd4jLong *inputShape,
                Nd4jLong batchNum,
                const Nd4jLong *tadShape, const Nd4jLong *tadOffsets,
                T *outputBuf, const Nd4jLong *outputShape) {

            __shared__ int n;
            if (threadIdx.x == 0) {
                n = shape::sizeAt(inputShape, -1); // * shape::sizeAt(inputShape, -1);
            }
            __syncthreads();

            auto output = outputBuf;
            auto input = inputBuf;

            for (auto i = blockIdx.x; i < batchNum; i += gridDim.x) {
                auto current = input + tadOffsets[i];

                auto zIndex = shape::getIndexOffset(i, outputShape);
                for (auto e = threadIdx.x; e < n; e += blockDim.x) {
                    Nd4jLong diag[] = {e, e};
                    auto xIndex = shape::getOffset(tadShape, diag);
                    math::atomics::nd4j_atomicAdd(&output[zIndex],math::nd4j_log<T, T>(current[xIndex] * current[xIndex]));
                }
            }
        }

        template<typename T>
        int logdetFunctor_(sd::LaunchContext *context, NDArray *input, NDArray *output) {
            NDArray::prepareSpecialUse({output}, {input});
            auto n2 = input->sizeAt(-1) * input->sizeAt(-2);
            auto stream = context->getCudaStream();
            NDArray tempOutput(*input);

            cholesky(context, input, &tempOutput, false);

            auto outputBuf = output->dataBuffer()->specialAsT<T>(); //reinterpret_cast<T*>(output->specialBuffer()); // + e * n2; // + e * n2;
            auto inputBuf = tempOutput.dataBuffer()->specialAsT<T>(); //reinterpret_cast<T*>(tempOutput.specialBuffer());
            output->nullify();
            auto packX = sd::ConstantTadHelper::getInstance().tadForDimensions(tempOutput.shapeInfo(),
                                                                                  {tempOutput.rankOf() - 2,
                                                                                   tempOutput.rankOf() - 1});
            logDetKernel<T><<<128, 512, 256, *stream>>>(inputBuf, tempOutput.specialShapeInfo(),
                    packX.numberOfTads(), packX.specialShapeInfo(),
                    packX.specialOffsets(), outputBuf, output->specialShapeInfo());
            output->tickWriteDevice();
            NDArray::registerSpecialUse({output}, {input});
            return Status::OK();
        }

        int logdetFunctor(sd::LaunchContext *context, NDArray *input, NDArray *output) {
            BUILD_SINGLE_SELECTOR(output->dataType(), return logdetFunctor_, (context, input, output), FLOAT_NATIVE);
        }

        /*
         * lup - batched input, batched outputs
         * */
        int lup(LaunchContext *context, NDArray *input, NDArray *compound, NDArray *permutation) {
            BUILD_DOUBLE_SELECTOR(input->dataType(), permutation->dataType(), lup_,(context, input, compound, permutation), FLOAT_NATIVE, INDEXING_TYPES);
            return Status::OK();
        }

//        BUILD_SINGLE_TEMPLATE(template int logdetFunctor_,
//                              (sd::LaunchContext * context, NDArray * input, NDArray * output), FLOAT_NATIVE);
    }
}
}
