#include "hip/hip_runtime.h"
/*
 *  ******************************************************************************
 *  *
 *  *
 *  * This program and the accompanying materials are made available under the
 *  * terms of the Apache License, Version 2.0 which is available at
 *  * https://www.apache.org/licenses/LICENSE-2.0.
 *  *
 *  * See the NOTICE file distributed with this work for additional
 *  * information regarding copyright ownership.
 *  * Unless required by applicable law or agreed to in writing, software
 *  * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 *  * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 *  * License for the specific language governing permissions and limitations
 *  * under the License.
 *  *
 *  * SPDX-License-Identifier: Apache-2.0
 *  *****************************************************************************
 */

//
// @author Yurii Shyrma (iuriish@yahoo.com)
//

#include <ops/declarable/helpers/convolutions.h>
#include <exceptions/cuda_exception.h>
#include <helpers/PointersManager.h>
#include <math/templatemath.h>

namespace sd {
namespace ops  {


//////////////////////////////////////////////////////////////////////////
template <typename X, typename Z>
static __global__ void avgPooling2dCuda(const void *vx, const Nd4jLong *xShapeInfo, void *vz, const Nd4jLong *zShapeInfo, const int kH, const int kW, const int sH, const int sW, const int pH, const int pW, const int dH, const int dW, const int extraParam0) {

    // input is  [bS, iC, iH, iW]
    // output is [bS, iC, oH, oW]

    const auto x = reinterpret_cast<const X*>(vx);
          auto z = reinterpret_cast<Z*>(vz);

    __shared__ int bS, iC, oH, oW, iH, iW, strideB, strideC, strideY, strideX, strideOB, strideOC, strideOY, strideOX, length, kHEff, kWEff;

    if (threadIdx.x == 0) {
        bS = shape::sizeAt(xShapeInfo, 0);
        iC = shape::sizeAt(xShapeInfo, 1);
        oH = shape::sizeAt(zShapeInfo, 2);
        oW = shape::sizeAt(zShapeInfo, 3);
        iH = shape::sizeAt(xShapeInfo, 2);
        iW = shape::sizeAt(xShapeInfo, 3);

        strideB = shape::stride(xShapeInfo)[0];
        strideC = shape::stride(xShapeInfo)[1];
        strideY = shape::stride(xShapeInfo)[2];
        strideX = shape::stride(xShapeInfo)[3];

        strideOB = shape::stride(zShapeInfo)[0];
        strideOC = shape::stride(zShapeInfo)[1];
        strideOY = shape::stride(zShapeInfo)[2];
        strideOX = shape::stride(zShapeInfo)[3];

        length = shape::length(zShapeInfo);

        //Replace kernel H/W with *effective* kernel H/W accounting for dilatyon
        kHEff = kH + (kH-1)*(dH-1);
        kWEff = kW + (kW-1)*(dW-1);
    }
    __syncthreads();

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (int index = tid; index < length; index += blockDim.x * gridDim.x) {

        const int pw = index % oW;
        const int ph = (index / oW) % oH;
        const int c = (index / oW / oH) % iC;
        const int n = index / oW / oH / iC;

        int hstart = sH * ph - pH;
        int wstart = sW * pw - pW;
        int hend = hstart + kHEff;
        int wend = wstart + kWEff;

        if(hstart < 0){
            int f = sd::math::nd4j_ceil<Z,int>((Z) -hstart / (Z)dH);
            hstart += f * dH;
        }
        if(wstart < 0){
            int f = sd::math::nd4j_ceil<Z,int>((Z) -wstart / (Z) dW);
            wstart += f * dW;
        }
        if(hend > iH){
            int f = sd::math::nd4j_ceil<Z,int>((Z) (hend-iH) / (Z) dH);
            hend -= f * dH;
        }
        if(wend > iW){
            int f = sd::math::nd4j_ceil<Z,int>((Z) (wend-iW) / (Z) dW);
            wend -= f * dW;
        }

        //Accounts for dilation
        int pool_size = sd::math::nd4j_ceil<double,int>((double) (hend-hstart) / (double) dH) * sd::math::nd4j_ceil<double,int>((double) (wend-wstart) / (double) dW);

        Z sum = 0.0f;

        const X *inSlice = x + (n * strideB + c * strideC);

        for (int h = hstart; h < hend; h += dH)
            for (int w = wstart; w < wend; w += dW)
                sum += static_cast<Z>(inSlice[h * strideY + w * strideX]);

        int divide_factor = pool_size;  //Case 0: exclude padding
        if (extraParam0 == 1)     //Case 1: include padding
            divide_factor = kH * kW;

        z[n * strideOB + c * strideOC + pw * strideOX + ph * strideOY] = sum / static_cast<Z>(divide_factor);
    }
}

//////////////////////////////////////////////////////////////////////////
template <typename X, typename Z>
static void avgPooling2dCudaLauncher(sd::LaunchContext & block, const void *vx, const Nd4jLong *vxShapeInfo, void *vz, const Nd4jLong *vzShapeInfo, const int kH, const int kW, const int sH, const int sW, const int pH, const int pW, const int dH, const int dW, const int extraParam0) {
    avgPooling2dCuda<X, Z><<<512, 512, 4192, *block.getCudaStream()>>>(vx, vxShapeInfo, vz, vzShapeInfo, kH, kW, sH, sW, pH, pW, dH, dW, extraParam0);
}

//////////////////////////////////////////////////////////////////////////
template <typename X, typename Z>
static __global__ void pnormPooling2dCuda(const void *vx, const Nd4jLong *xShapeInfo, void *vz, const Nd4jLong *zShapeInfo, const int kH, const int kW, const int sH, const int sW, const int pH, const int pW, const int dH, const int dW, const int extraParam0) {

    // input is  [bS, iC, iH, iW]
    // output is [bS, iC, oH, oW]

    const auto x = reinterpret_cast<const X*>(vx);
          auto z = reinterpret_cast<Z*>(vz);

    __shared__ int bS, iC, oH, oW, iH, iW, strideB, strideC, strideY, strideX, strideOB, strideOC, strideOY, strideOX, length, kHEff, kWEff;
    __shared__ bool fOrder;

    if (threadIdx.x == 0) {
        bS = shape::sizeAt(xShapeInfo, 0);
        iC = shape::sizeAt(xShapeInfo, 1);
        oH = shape::sizeAt(zShapeInfo, 2);
        oW = shape::sizeAt(zShapeInfo, 3);
        iH = shape::sizeAt(xShapeInfo, 2);
        iW = shape::sizeAt(xShapeInfo, 3);

        strideB = shape::stride(xShapeInfo)[0];
        strideC = shape::stride(xShapeInfo)[1];
        strideY = shape::stride(xShapeInfo)[2];
        strideX = shape::stride(xShapeInfo)[3];

        strideOB = shape::stride(zShapeInfo)[0];
        strideOC = shape::stride(zShapeInfo)[1];
        strideOY = shape::stride(zShapeInfo)[2];
        strideOX = shape::stride(zShapeInfo)[3];

        length = shape::length(zShapeInfo);

        //Replace kernel H/W with *effective* kernel H/W accounting for dilatyon
        kHEff = kH + (kH-1)*(dH-1);
        kWEff = kW + (kW-1)*(dW-1);
    }
    __syncthreads();

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (int index = tid; index < length; index += blockDim.x * gridDim.x) {

        const int pw = index % oW;
        const int ph = (index / oW) % oH;
        const int c = (index / oW / oH) % iC;
        const int n = index / oW / oH / iC;

        int hstart = sH * ph - pH;
        int wstart = sW * pw - pW;
        int hend = hstart + kHEff;
        int wend = wstart + kWEff;

        if (hstart < 0) {
            int f = sd::math::nd4j_ceil<Z, int>((Z) -hstart / (Z) dH);
            hstart += f * dH;
        }
        if (wstart < 0) {
            int f = sd::math::nd4j_ceil<Z, int>((Z) -wstart / (Z) dW);
            wstart += f * dW;
        }
        if (hend > iH) {
            int f = sd::math::nd4j_ceil<Z, int>((Z) (hend - iH) / (Z) dH);
            hend -= f * dH;
        }
        if (wend > iW) {
            int f = sd::math::nd4j_ceil<Z, int>((Z) (wend - iW) / (Z) dW);
            wend -= f * dW;
        }
        //Accounts for dilation
        int pool_size = sd::math::nd4j_ceil<double, int>((double) (hend - hstart) / (double) dH) *
                        sd::math::nd4j_ceil<double, int>((double) (wend - wstart) / (double) dW);

        Z sum = 0.f;

        const X *inSlice = x + (n * strideB + c * strideC);

        for (int h = hstart; h < hend; h += dH)
            for (int w = wstart; w < wend; w += dW)
                sum += sd::math::nd4j_pow<Z, Z, Z>(static_cast<Z>(sd::math::nd4j_abs<X>(inSlice[h * strideY + w * strideX])), extraParam0);

        z[n * strideOB + c * strideOC + pw * strideOX + ph * strideOY] = sd::math::nd4j_pow<Z, Z, Z>(sum, (Z) 1.0f / extraParam0);
    }
}

//////////////////////////////////////////////////////////////////////////
template <typename X, typename Z>
static void pnormPooling2dCudaLauncher(sd::LaunchContext & block, const void *vx, const Nd4jLong *vxShapeInfo, void *vz, const Nd4jLong *vzShapeInfo, const int kH, const int kW, const int sH, const int sW, const int pH, const int pW, const int dH, const int dW, const int extraParam0) {
    pnormPooling2dCuda<X, Z><<<512, 512, 4192, *block.getCudaStream()>>>(vx, vxShapeInfo, vz, vzShapeInfo, kH, kW, sH, sW, pH, pW, dH, dW, extraParam0);
}

//////////////////////////////////////////////////////////////////////////
template <typename X, typename Z>
static __global__ void maxPooling2dCuda(const void *vx, const Nd4jLong *xShapeInfo, void *vz, const Nd4jLong *zShapeInfo, const int kH, const int kW, const int sH, const int sW, const int pH, const int pW, const int dH, const int dW, const int extraParam0) {

    // input is  [bS, iC, iH, iW]
    // output is [bS, iC, oH, oW]

    const auto x = reinterpret_cast<const X*>(vx);
          auto z = reinterpret_cast<Z*>(vz);

    __shared__ int bS, iC, oH, oW, iH, iW, strideB, strideC, strideY, strideX, strideOB, strideOC, strideOY, strideOX, length, kHEff, kWEff;
    __shared__ bool fOrder;

    if (threadIdx.x == 0) {
        bS = shape::sizeAt(xShapeInfo, 0);
        iC = shape::sizeAt(xShapeInfo, 1);
        oH = shape::sizeAt(zShapeInfo, 2);
        oW = shape::sizeAt(zShapeInfo, 3);
        iH = shape::sizeAt(xShapeInfo, 2);
        iW = shape::sizeAt(xShapeInfo, 3);

        strideB = shape::stride(xShapeInfo)[0];
        strideC = shape::stride(xShapeInfo)[1];
        strideY = shape::stride(xShapeInfo)[2];
        strideX = shape::stride(xShapeInfo)[3];

        strideOB = shape::stride(zShapeInfo)[0];
        strideOC = shape::stride(zShapeInfo)[1];
        strideOY = shape::stride(zShapeInfo)[2];
        strideOX = shape::stride(zShapeInfo)[3];

        length = shape::length(zShapeInfo);

        //Replace kernel H/W with *effective* kernel H/W accounting for dilatyon
        kHEff = kH + (kH-1)*(dH-1);
        kWEff = kW + (kW-1)*(dW-1);
    }
    __syncthreads();

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (int index = tid; index < length; index += blockDim.x * gridDim.x) {

        const int pw = index % oW;
        const int ph = (index / oW) % oH;
        const int c = (index / oW / oH) % iC;
        const int n = index / oW / oH / iC;

        int hstart = sH * ph - pH;
        int wstart = sW * pw - pW;
        int hend = hstart + kHEff;
        int wend = wstart + kWEff;

        if(hstart < 0){
            int f = sd::math::nd4j_ceil<Z,int>((Z) -hstart / (Z)dH);
            hstart += f * dH;
        }
        if(wstart < 0){
            int f = sd::math::nd4j_ceil<Z,int>((Z) -wstart / (Z) dW);
            wstart += f * dW;
        }
        if(hend > iH){
            int f = sd::math::nd4j_ceil<Z,int>((Z) (hend-iH) / (Z) dH);
            hend -= f * dH;
        }
        if(wend > iW){
            int f = sd::math::nd4j_ceil<Z,int>((Z) (wend-iW) / (Z) dW);
            wend -= f * dW;
        }
        //Accounts for dilation
        int pool_size = sd::math::nd4j_ceil<double,int>((double) (hend-hstart) / (double) dH) * sd::math::nd4j_ceil<double,int>((double) (wend-wstart) / (double) dW);

        Z max = -sd::DataTypeUtils::max<Z>();

        const X *inSlice = x + (n * strideB + c * strideC);

        for (int h = hstart; h < hend; h += dH) {
            for (int w = wstart; w < wend; w += dW) {
                Z v = static_cast<Z>(inSlice[h * strideY + w * strideX]);
                if (v > max)
                    max = v;
            }
        }

        z[n * strideOB + c * strideOC + pw * strideOX + ph * strideOY] = max;
    }
}

//////////////////////////////////////////////////////////////////////////
template <typename X, typename Z>
static void maxPooling2dCudaLauncher(sd::LaunchContext & block, const void *vx, const Nd4jLong *vxShapeInfo, void *vz, const Nd4jLong *vzShapeInfo, const int kH, const int kW, const int sH, const int sW, const int pH, const int pW, const int dH, const int dW, const int extraParam0) {
    maxPooling2dCuda<X,Z><<<512, 512, 4192, *block.getCudaStream()>>>(vx, vxShapeInfo, vz, vzShapeInfo, kH, kW, sH, sW, pH, pW, dH, dW, extraParam0);
}

//////////////////////////////////////////////////////////////////////////
void ConvolutionUtils::pooling2d(sd::graph::Context& block, const NDArray& input, NDArray& output, const int kH, const int kW, const int sH, const int sW, const int pH, const int pW, const int dH, const int dW, const PoolingType poolingMode, const int extraParam0) {

    if(!input.isActualOnDeviceSide()) input.syncToDevice();

    switch (poolingMode) {

        case MAX_POOL: {
                BUILD_SINGLE_SELECTOR_TWICE(input.dataType(), maxPooling2dCudaLauncher, (*block.launchContext(), input.specialBuffer(), input.specialShapeInfo(), output.specialBuffer(), output.specialShapeInfo(), kH, kW, sH, sW, pH, pW, dH, dW, extraParam0), NUMERIC_TYPES);
            }
            break;
        case AVG_POOL: {
                BUILD_SINGLE_SELECTOR_TWICE(input.dataType(), avgPooling2dCudaLauncher, (*block.launchContext(), input.specialBuffer(), input.specialShapeInfo(), output.specialBuffer(), output.specialShapeInfo(), kH, kW, sH, sW, pH, pW, dH, dW, extraParam0), NUMERIC_TYPES);
            }
            break;
        case PNORM_POOL: {
                BUILD_SINGLE_SELECTOR_TWICE(input.dataType(), pnormPooling2dCudaLauncher, (*block.launchContext(), input.specialBuffer(), input.specialShapeInfo(), output.specialBuffer(), output.specialShapeInfo(), kH, kW, sH, sW, pH, pW, dH, dW, extraParam0), FLOAT_TYPES);
            }
            break;
        default:
            throw std::runtime_error("Pooling2D: Unknown PoolingType used");
    }

    output.tickWriteDevice();
    input.tickReadDevice();

    auto result = hipStreamSynchronize(*block.launchContext()->getCudaStream());
    if (result != 0)
        throw cuda_exception::build("Pooling2D failed", result);
}

}
}
