#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author GS <sgazeos@gmail.com>
//

#include <ops/declarable/helpers/segment.h>
#include <ops/declarable/helpers/segment_common.h>
#include <array/NDArrayFactory.h>
#include <helpers/ShapeUtils.h>
#include <helpers/TAD.h>
#include <exceptions/cuda_exception.h>
#include <helpers/PointersManager.h>
#include <helpers/ConstantTadHelper.h>

namespace sd {
namespace ops {
namespace helpers {
    // -------------------------------------------------------------------------------------------------------------- //
    // Segment ops linear kernels
    // -------------------------------------------------------------------------------------------------------------- //
    template<typename T, typename I>
    static __global__ void
    segmentSumLinearKernel(
            const void *input, const Nd4jLong *inputShape,
            int *starts, int *lengths, Nd4jLong numOfClasses,
            void *output, const Nd4jLong *outputShape) {
        __shared__
        T *val;
        __shared__
        Nd4jLong xLen, zLen, segment, zIndex;
        __shared__
        const T *x;
        __shared__
        T *z;
        __shared__ int threadsPerSegment, start, finish;

        if (threadIdx.x == 0) {
            threadsPerSegment = (gridDim.x + numOfClasses - 1) / numOfClasses;
            segment = blockIdx.x / threadsPerSegment;
            x = reinterpret_cast<const T *>(input);
            z = reinterpret_cast<T *>(output);

            xLen = shape::length(inputShape);
            zLen = shape::length(outputShape);


            if (segment < numOfClasses) {
                zIndex = shape::getIndexOffset(segment, outputShape);
                start = starts[segment];
                finish = start + lengths[segment];
                //val[segment] = ;
                z[zIndex] = x[shape::getIndexOffset(start, inputShape)];
            }

        }
        __syncthreads();

        for (auto e = start + threadIdx.x + 1; e < finish; e += blockDim.x) {
            auto xIndex = shape::getIndexOffset(e, inputShape);
            sd::math::atomics::nd4j_atomicAdd(&z[zIndex], x[xIndex]);
        }
    }
    // -------------------------------------------------------------------------------------------------------------- //

    template<typename T, typename I>
    static __global__ void
    unsortedSegmentSumLinearKernel(
            const void *input, const Nd4jLong *inputShape,
            const void *indices, const Nd4jLong *indicesShape,
            int *starts, int *lengths, Nd4jLong numOfClasses,
            void *output, const Nd4jLong *outputShape) {
        __shared__
        T *val;
        __shared__
        Nd4jLong xLen, zLen, segment, zIndex;
        __shared__
        const T *x;
        __shared__
        T *z;
        __shared__
        const I *y; //int threadsPerSegment, start, finish;

        if (threadIdx.x == 0) {
            segment = blockIdx.x;
            x = reinterpret_cast<const T *>(input);
            z = reinterpret_cast<T *>(output);
            y = reinterpret_cast<const I *>(indices);
            xLen = shape::length(inputShape);
            zLen = shape::length(outputShape);

            zIndex = shape::getIndexOffset(segment, outputShape);
            if (lengths[segment] > 0)
                z[zIndex] = x[shape::getIndexOffset(starts[segment], inputShape)];
            else
                z[zIndex] = 0; //DataTypeUtils::max<T>();
        }
        __syncthreads();

        if (lengths[segment] > 0)
            for (auto e = threadIdx.x; e < xLen; e += blockDim.x) {
                auto xIndex = shape::getIndexOffset(e, inputShape);
                auto yIndex = shape::getIndexOffset(e, indicesShape);
                if (y[yIndex] == segment && e != starts[segment]) {
                    sd::math::atomics::nd4j_atomicAdd(&z[zIndex], x[xIndex]);
                }
            }
    }
    // -------------------------------------------------------------------------------------------------------------- //
    // SegmentSum kernel
    template <typename T, typename I>
    static __global__ void segmentSumTadKernel(
            const void* inputBuf, const Nd4jLong* inputShape, const Nd4jLong* inputTads, const Nd4jLong* inputTadOffsets,
            const I* indices,
            int* starts, int* lengths, Nd4jLong numOfClasses,
            void* outputBuf, const Nd4jLong* outputShape, const Nd4jLong* outputTads, const Nd4jLong* outputTadOffsets) {
        __shared__ T* val;
        __shared__ Nd4jLong len, zIndex, total;
        __shared__ T* z;
        __shared__ int start, finish;

        if (threadIdx.x == 0) {
            auto segment = indices[blockIdx.x]; // / threadsPerSegment;
            z = reinterpret_cast<T*>(outputBuf) + outputTadOffsets[segment];
            len = shape::length(inputTads);
            start = starts[segment];
            finish = start + lengths[segment];
            total = shape::sizeAt(inputShape, 0);

        }
        __syncthreads();

        auto idx = blockIdx.x;
        if (blockIdx.x <= total) {
            auto x = reinterpret_cast<const T *>(inputBuf) + inputTadOffsets[idx];
            if (blockIdx.x == start) {
                for (auto e = threadIdx.x; e < len; e += blockDim.x) {
                    auto xIndex = shape::getIndexOffset(e, inputTads);
                    auto zIndex = shape::getIndexOffset(e, outputTads);
                    sd::math::atomics::nd4j_atomicAdd(&z[zIndex], x[xIndex]);
                }
            }
            else {
                for (auto e = threadIdx.x; e < len; e += blockDim.x) {
                    auto xIndex = shape::getIndexOffset(e, inputTads);
                    auto zIndex = shape::getIndexOffset(e, outputTads);
                    if (lengths[indices[idx]])
                        sd::math::atomics::nd4j_atomicAdd(&z[zIndex], x[xIndex]);
                }
            }
        }
    }
    // -------------------------------------------------------------------------------------------------------------- //

    template <typename T, typename I>
    static void segmentSumFunctor_(sd::LaunchContext* context, NDArray* input, NDArray* indices, NDArray* output) {
        auto stream = context->getCudaStream();
        Nd4jLong numClasses = indices->e<Nd4jLong>(indices->lengthOf() - 1) + 1;
        NDArray classesRangesLens = NDArrayFactory::create<int>('c', {numClasses}, context);
        NDArray classesRangesBegs = NDArrayFactory::create<int>('c', {numClasses}, context);

        classesRangesBegs.assign(indices->lengthOf());
        classesRangesLens.assign(0);

        dim3 dims(numClasses, indices->lengthOf(), numClasses * 32 + 32);
        fillUpSegments(indices, numClasses, classesRangesBegs, classesRangesLens);
        int* begins = reinterpret_cast<int*>(classesRangesBegs.specialBuffer());
        int* lengths = reinterpret_cast<int*>(classesRangesLens.specialBuffer());

        if (input->isVector()) {
            segmentSumLinearKernel<T,I><<<numClasses, input->lengthOf(), numClasses * 32 + 32, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), begins, lengths, numClasses, output->specialBuffer(), output->specialShapeInfo());
        }
        else {
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = sd::ConstantTadHelper::getInstance().tadForDimensions(input->shapeInfo(), dimensions);
            auto packZ = sd::ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(), dimensions);
            auto inputTads = packX.specialShapeInfo();
            auto inputTadOffsets = packX.specialOffsets();
            auto outputTads = packZ.specialShapeInfo();
            auto outputTadOffsets = packZ.specialOffsets();
            segmentSumTadKernel<T,I><<<input->sizeAt(0), 512, 2048, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), inputTads, inputTadOffsets, reinterpret_cast<I*>(indices->specialBuffer()), begins, lengths, numClasses, output->specialBuffer(), output->specialShapeInfo(), outputTads, outputTadOffsets);
        }

    }
    // -------------------------------------------------------------------------------------------------------------- //
    void segmentSumFunctor(sd::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* output) {
        NDArray::prepareSpecialUse({output}, {input, indices});
        output->nullify();
        BUILD_DOUBLE_SELECTOR(input->dataType(), indices->dataType(), segmentSumFunctor_, (context, input, indices, output), NUMERIC_TYPES, INDEXING_TYPES);
        NDArray::registerSpecialUse({output}, {input, indices});
    }

    // -------------------------------------------------------------------------------------------------------------- //
    template <typename T, typename I>
    static void unsortedSegmentSumFunctor_(sd::LaunchContext* context, NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {
        auto stream = context->getCudaStream();
//        NDArray classes = NDArrayFactory::create<int>('c', {numOfClasses, 2});
        NDArray classesRangesBegs = NDArrayFactory::create<int>('c', {numOfClasses}, context);
        NDArray classesRangesLens = NDArrayFactory::create<int>('c', {numOfClasses}, context);
//        NDArray row = NDArrayFactory::create<int>('c', {1, 2}, {(int)indices->lengthOf(), (int)0});
//        classes.applyTrueBroadcast(sd::BroadcastOpsTuple::Assign(), &row, &classes);
        classesRangesBegs.assign(indices->lengthOf());
        classesRangesLens.assign(0);
        dim3 dims(numOfClasses, indices->lengthOf(), (numOfClasses + 1) * 64);
//        int* classesBuf = reinterpret_cast<int*>(classes.specialBuffer());
        fillUpSegments(indices, numOfClasses, classesRangesBegs, classesRangesLens);
        int* begins = reinterpret_cast<int*>(classesRangesBegs.specialBuffer());
        int* lengths = reinterpret_cast<int*>(classesRangesLens.specialBuffer());

        if (input->isVector()) {
            unsortedSegmentSumLinearKernel<T,I><<<dims.x, dims.y, dims.z, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), indices->specialBuffer(), indices->specialShapeInfo(), begins, lengths, numOfClasses, output->specialBuffer(), output->specialShapeInfo());
        }
        else {
            output->assign(0);
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = sd::ConstantTadHelper::getInstance().tadForDimensions(input->shapeInfo(), dimensions);
            auto packZ = sd::ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(), dimensions);
            auto inputTads = packX.specialShapeInfo();
            auto inputTadOffsets = packX.specialOffsets();
            auto outputTads = packZ.specialShapeInfo();
            auto outputTadOffsets = packZ.specialOffsets();
            dims.x = input->sizeAt(0);
            segmentSumTadKernel<T,I><<<dims.x, dims.y, dims.z, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), inputTads, inputTadOffsets, reinterpret_cast<I*>(indices->specialBuffer()), begins, lengths, numOfClasses, output->specialBuffer(), output->specialShapeInfo(), outputTads, outputTadOffsets);
        }

    }
    // -------------------------------------------------------------------------------------------------------------- //
    void unsortedSegmentSumFunctor(sd::LaunchContext* context , NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {
        NDArray::prepareSpecialUse({output}, {input, indices});
        output->nullify();
        BUILD_DOUBLE_SELECTOR(input->dataType(), indices->dataType(), unsortedSegmentSumFunctor_, (context, input, indices, numOfClasses, output),
                              NUMERIC_TYPES, INDEXING_TYPES);
        NDArray::registerSpecialUse({output}, {input, indices});

    }

    // -------------------------------------------------------------------------------------------------------------- //
    // Backpropagate ops
    // -------------------------------------------------------------------------------------------------------------- //
    // Sorted sum backpropagate
    template <typename T, typename I>
    static __global__ void segmentSumBPLinearKernel(
            const void* inputBuf, const Nd4jLong* inputShape,
            const void* eps, const Nd4jLong* epsShape,
            const void* indicesBuf, const Nd4jLong* indicesShape,
            void* outputBuf, const Nd4jLong* outputShape) {
        auto x = reinterpret_cast<const T*>(inputBuf);
        auto y = reinterpret_cast<const I*>(indicesBuf);
        auto z = reinterpret_cast<T*>(outputBuf);
        auto gradOut = reinterpret_cast<const T*>(eps);
        __shared__ Nd4jLong xLen, gradLen;

        if (threadIdx.x == 0) {
            xLen = shape::length(inputShape);
            gradLen = shape::length(epsShape);
        }
        __syncthreads();

        auto start = blockIdx.x * blockDim.x + threadIdx.x;
        auto step = gridDim.x * blockDim.x;

        for (auto e = start; e < xLen; e += step) {

            auto zOffset = shape::getIndexOffset(e, outputShape);
            auto xOffset = shape::getIndexOffset(e, inputShape);
            auto yOffset = shape::getIndexOffset(e, indicesShape);
            auto classIndex = y[yOffset];
            auto gradOffsetO = shape::getIndexOffset(classIndex, epsShape);

            z[zOffset] = gradOut[gradOffsetO];
        }
    }
    // -------------------------------------------------------------------------------------------------------------- //
    template <typename T, typename I>
    static __global__ void segmentSumBPTadKernel(
            const void* inputBuf, const Nd4jLong* inputShape,
            const void* eps, const Nd4jLong* epsShape,
            const void* indicesBuf, const Nd4jLong* indicesShape,
            void* outputBuf, const Nd4jLong* outputShape,
            const Nd4jLong* inputTad, const Nd4jLong* inputOffsets,
            const Nd4jLong* gradOutTad, const Nd4jLong* gradOutOffsets,
            const Nd4jLong* outTad, const Nd4jLong* outOffsets) {
        __shared__ const T* x;
        __shared__ const T* gradOut;
        __shared__ const I* y;
        __shared__ T* z;
        __shared__ Nd4jLong xLen, yLen, gradLen, currentLen;

        if (threadIdx.x == 0) {
            xLen = shape::length(inputShape);
            x = reinterpret_cast<const T*>(inputBuf);
            y = reinterpret_cast<const I*>(indicesBuf);
            z = reinterpret_cast<T*>(outputBuf);
            yLen = shape::length(indicesShape);
            gradOut = reinterpret_cast<const T*>(eps);
            gradLen = shape::length(epsShape);
            currentLen = shape::length(outTad);
        }
        __syncthreads();

        for (auto i = blockIdx.x; i < yLen; i += gridDim.x) {
            auto yIndex = shape::getIndexOffset(i, indicesShape);
            auto segment = y[yIndex];
            auto currentOut = z + outOffsets[i];
            auto outGrad = gradOut + gradOutOffsets[segment];

            for (auto e = threadIdx.x; e < currentLen; e += blockDim.x) {
                currentOut[e] = outGrad[e];
            }
        }

    }
    // -------------------------------------------------------------------------------------------------------------- //
    template <typename T, typename I>
    int segmentSumFunctorBP_(sd::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output) {
        auto stream = context->getCudaStream();
        NDArray::prepareSpecialUse({output}, {input, indices, gradOut});
        if (input->isVector()) {
            Nd4jLong loop_size = input->lengthOf();
            auto numOfClasses = gradOut->lengthOf(); //indices->e<Nd4jLong>(loop_size - 1);
            segmentSumBPLinearKernel<T,I><<<gradOut->lengthOf(), input->lengthOf(), 256, *stream>>>(input->specialBuffer(),
                    input->specialShapeInfo(), gradOut->specialBuffer(), gradOut->specialShapeInfo(),
                    indices->specialBuffer(), indices->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo());
        }
        else {
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = sd::ConstantTadHelper::getInstance().tadForDimensions(input->shapeInfo(), dimensions);
            auto packZ = sd::ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(), dimensions);
            auto packGradOut = sd::ConstantTadHelper::getInstance().tadForDimensions(gradOut->shapeInfo(), dimensions);
            auto inputTads = packX.specialShapeInfo();
            auto inputTadOffsets = packX.specialOffsets();
            auto outputTads = packZ.specialShapeInfo();
            auto outputTadOffsets = packZ.specialOffsets();
            auto gradOutTads = packGradOut.specialShapeInfo();
            auto gradOutTadOffsets = packGradOut.specialOffsets();

            segmentSumBPTadKernel<T,I><<<gradOut->lengthOf(), input->lengthOf(), 256, *stream>>>(input->specialBuffer(), input->specialShapeInfo(),
                    gradOut->specialBuffer(), gradOut->specialShapeInfo(),
                    indices->specialBuffer(), indices->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo(),
                    inputTads, inputTadOffsets, gradOutTads, gradOutTadOffsets,
                    outputTads, outputTadOffsets);
        }
        NDArray::registerSpecialUse({output}, {input, indices, gradOut});
        return Status::OK();
    }
    // -------------------------------------------------------------------------------------------------------------- //

    int segmentSumFunctorBP(sd::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output) {
        NDArray::prepareSpecialUse({output}, {input, indices, gradOut});
        BUILD_DOUBLE_SELECTOR(output->dataType(), indices->dataType(), return segmentSumFunctorBP_, (context, input,
                indices, gradOut, output), FLOAT_TYPES, INDEXING_TYPES);
        NDArray::registerSpecialUse({output}, {input, indices, gradOut});
    }

    template <typename T, typename I>
    static int unsortedSegmentSumFunctorBP_(sd::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        auto stream = context->getCudaStream();
        NDArray::prepareSpecialUse({output}, {input, indices, gradOut});
        if (input->isVector()) {
            Nd4jLong loop_size = input->lengthOf();
            auto numOfClasses = gradOut->lengthOf(); //indices->e<Nd4jLong>(loop_size - 1);
            segmentSumBPLinearKernel<T,I><<<gradOut->lengthOf(), input->lengthOf(), 256, *stream>>>(input->specialBuffer(),
                    input->specialShapeInfo(), gradOut->specialBuffer(), gradOut->specialShapeInfo(),
                    indices->specialBuffer(), indices->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo());
        }
        else {
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = sd::ConstantTadHelper::getInstance().tadForDimensions(input->shapeInfo(), dimensions);
            auto packZ = sd::ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(), dimensions);
            auto packGradOut = sd::ConstantTadHelper::getInstance().tadForDimensions(gradOut->shapeInfo(), dimensions);
            auto inputTads = packX.specialShapeInfo();
            auto inputTadOffsets = packX.specialOffsets();
            auto outputTads = packZ.specialShapeInfo();
            auto outputTadOffsets = packZ.specialOffsets();
            auto gradOutTads = packGradOut.specialShapeInfo();
            auto gradOutTadOffsets = packGradOut.specialOffsets();

            segmentSumBPTadKernel<T,I><<<gradOut->lengthOf(), input->lengthOf(), 256, *stream>>>(input->specialBuffer(), input->specialShapeInfo(),
                    gradOut->specialBuffer(), gradOut->specialShapeInfo(),
                    indices->specialBuffer(), indices->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo(),
                    inputTads, inputTadOffsets, gradOutTads, gradOutTadOffsets,
                    outputTads, outputTadOffsets);
        }
        NDArray::registerSpecialUse({output}, {input, indices, gradOut});
        return Status::OK();
    }
    // -------------------------------------------------------------------------------------------------------------- //
    int unsortedSegmentSumFunctorBP(sd::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        NDArray::prepareSpecialUse({output}, {input, indices, gradOut});
        BUILD_DOUBLE_SELECTOR(output->dataType(), indices->dataType(), return unsortedSegmentSumFunctorBP_, (context, input, indices, gradOut, numOfClasses, output), FLOAT_TYPES, INDEXING_TYPES);
        NDArray::registerSpecialUse({output}, {input, indices, gradOut});
    }

}
}
}