#include "hip/hip_runtime.h"
/*
 *  ******************************************************************************
 *  *
 *  *
 *  * This program and the accompanying materials are made available under the
 *  * terms of the Apache License, Version 2.0 which is available at
 *  * https://www.apache.org/licenses/LICENSE-2.0.
 *  *
 *  * See the NOTICE file distributed with this work for additional
 *  * information regarding copyright ownership.
 *  * Unless required by applicable law or agreed to in writing, software
 *  * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 *  * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 *  * License for the specific language governing permissions and limitations
 *  * under the License.
 *  *
 *  * SPDX-License-Identifier: Apache-2.0
 *  *****************************************************************************
 */

//
// @author Yurii Shyrma (iuriish@yahoo.com)
//


#include<ops/declarable/helpers/transforms.h>
#include <array/ResultSet.h>
#include <helpers/ShapeUtils.h>
#include <numeric>
#include <array/NDArrayFactory.h>
#include <helpers/TAD.h>
#include <exceptions/cuda_exception.h>
#include <helpers/PointersManager.h>
#include <helpers/ConstantTadHelper.h>

namespace sd    {
namespace ops     {
namespace helpers {


///////////////////////////////////////////////////////////////////
template<typename T>
__global__ static void splitCuda(const void* vx, const Nd4jLong* xShapeInfo, void* pVz, const Nd4jLong* zTadShapeInfo, const int axis) {

    const T* x = reinterpret_cast<const T*>(vx);

    __shared__ Nd4jLong xLen, totalThreads;
    __shared__ int xRank, zDim;

    if (threadIdx.x == 0) {
        xLen  = shape::length(xShapeInfo);
        xRank = shape::rank(xShapeInfo);
        zDim  = shape::shapeOf(zTadShapeInfo)[axis];          // same for all input arrays
        totalThreads = gridDim.x * blockDim.x;
    }
    __syncthreads();

    const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

    int coords[MAX_RANK];

    for (uint64_t i = tid; i < xLen; i += totalThreads) {

        shape::index2coords(i, xShapeInfo, coords);

        const auto xOffset = shape::getOffset(xShapeInfo, coords);

        auto *z = reinterpret_cast<T*>(reinterpret_cast<void **>(pVz)[coords[axis] / zDim]);

        coords[axis] %= zDim;

        const auto zOffset = shape::getOffset(zTadShapeInfo, coords);

        z[zOffset] = x[xOffset];
    }
}

///////////////////////////////////////////////////////////////////
template<typename T>
__host__ static void splitCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const hipStream_t *stream,
                                       const void* vx, const Nd4jLong* xShapeInfo, void* pVz, const Nd4jLong* zTadShapeInfo, const int axis) {

    splitCuda<T><<<blocksPerGrid, threadsPerBlock, 256, *stream>>>(vx, xShapeInfo, pVz, zTadShapeInfo, axis);
}
BUILD_SINGLE_TEMPLATE(template void splitCudaLauncher, (const int blocksPerGrid, const int threadsPerBlock, const hipStream_t *stream, const void* vx, const Nd4jLong* xShapeInfo, void* pVz, const Nd4jLong* zTadShapeInfo, const int axis), LIBND4J_TYPES);

//////////////////////////////////////////////////////////////////////////
void split(sd::LaunchContext* context, const NDArray& input, std::vector<NDArray*>& outArrs, const int axis) {

    const int numOfSubArrs = outArrs.size();
    const auto sizeofT     = input.sizeOfT();

    for(int i = 0; i < numOfSubArrs; ++i)
        outArrs[i]->syncToDevice();
    input.syncToDevice();

    bool luckCase1 = ((axis == 0 && input.ordering() == 'c') || (axis == input.rankOf() - 1 && input.ordering() == 'f')) && input.ews() == 1;

    if(luckCase1) {
        for (uint i = 0; i < numOfSubArrs; ++i) {
            luckCase1 &= outArrs[i]->ordering() == input.ordering() && outArrs[i]->ews() == 1;
            if(!luckCase1)
                break;
        }
    }

    if(luckCase1) {     // for example {1,10} + {2,10} + {3,10} = {6, 10} order c; or {10,1} + {10,2} + {10,3} = {10, 6} order f

        auto x = static_cast<const int8_t*>(input.specialBuffer());

        for (uint i = 0; i < numOfSubArrs; ++i) {
            const auto memAmountToCopy = outArrs[i]->lengthOf() * sizeofT;
            hipMemcpyAsync(static_cast<int8_t*>(outArrs[i]->specialBuffer()), x, memAmountToCopy, hipMemcpyDeviceToDevice, *context->getCudaStream());
            x = static_cast<const int8_t*>(x) + memAmountToCopy;
        }

        if(hipStreamSynchronize(*context->getCudaStream()) != 0)
            throw std::runtime_error("split cuda: luckCase1 failed!");

        for(int i = 0; i < numOfSubArrs; ++i)
            outArrs[i]->tickWriteDevice();
        input.tickReadDevice();

        return;
    }

    // const bool isXcontin = input.strideAt(axis) == 1;
    // bool areOutputsContin = true;
    // bool allSameOrder    = true;
    // std::vector<Nd4jLong> strideOfContigStride(outArrs.size());

    // if(isXcontin) {

    //     for (uint i = 0; i < outArrs.size(); ++i) {

    //         areOutputsContin &= outArrs[i]->strideAt(axis) == 1;
    //         allSameOrder     &= input.ordering() == outArrs[i]->ordering();
    //         if(!areOutputsContin || !allSameOrder)
    //             break;

    //         strideOfContigStride[i] = shape::strideOverContigAxis(axis, outArrs[i]->shapeInfo());
    //     }
    // }

    // const bool luckCase2 = isXcontin && areOutputsContin && allSameOrder;

    // if(luckCase2) {     // for example {2,1,3} + {2,5,3} + {2,10,3} = {2,16,3}, here axis 1 shoud have stride = 1 for all inputs arrays and input array

    //     const auto xStep = shape::strideOverContigAxis(axis, input.shapeInfo());
    //     const auto zDim = outArrs[0]->sizeAt(axis);     // same for all outArrs

    //     for (uint i = 0; i < input.lengthOf() / input.sizeAt(axis); ++i) {

    //         const auto iShift = i * sizeofT;
    //         void* x = static_cast<int8_t*>(input.specialBuffer()) + xStep * iShift;

    //         for (uint j = 0; j < numOfSubArrs; ++j) {
    //             void* z = static_cast<int8_t*>(outArrs[j]->specialBuffer()) + strideOfContigStride[j] * iShift;
    //             const auto memSizeToCopy = zDim * sizeofT;
    //             hipMemcpyAsync(z, x, memSizeToCopy, hipMemcpyDeviceToDevice, *context->getCudaStream());
    //             x = static_cast<int8_t*>(x) + memSizeToCopy;
    //         }
    //     }

    //     if(hipStreamSynchronize(*context->getCudaStream()) != 0)
    //         throw std::runtime_error("split cuda: luckCase2 failed!");
    // }
    // else {      // general (slower) case

        const int threadsPerBlock = MAX_NUM_THREADS / 2;
        const int blocksPerGrid = (input.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;

        // prepare arrays of pointers on buffers and shapes
        std::vector<void*> hOutBuffers(numOfSubArrs);

        for(int i = 0; i < numOfSubArrs; ++i)
            hOutBuffers[i]   = outArrs[i]->specialBuffer();

        PointersManager manager(context, "helpers::split");

        void* dOutBuffers = manager.replicatePointer(hOutBuffers.data(), hOutBuffers.size() * sizeof(void*));

        BUILD_SINGLE_SELECTOR(input.dataType(), splitCudaLauncher, (blocksPerGrid, threadsPerBlock, context->getCudaStream(), input.specialBuffer(), input.specialShapeInfo(), dOutBuffers, outArrs[0]->specialShapeInfo(), axis), LIBND4J_TYPES);

        manager.synchronize();
    // }

    for(int i = 0; i < numOfSubArrs; ++i)
        outArrs[i]->tickWriteDevice();
    input.tickReadDevice();
}

}
}
}