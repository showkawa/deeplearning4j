#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author George A. Shulinok <sgazeos@gmail.com>
//
#include <ops/declarable/helpers/matrix_band.h>
#include <helpers/TAD.h>
#include <exceptions/cuda_exception.h>
#include <helpers/ShapeUtils.h>
#include <helpers/ConstantTadHelper.h>

namespace sd {
namespace ops {
namespace helpers {
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// matrix band kernel
//
// inputBuffer - buffer of input tensor
// inputShape - shape of input tensor
// outputBuffer - buffer of output tensor
// outputShape - shape of output tensor
// lowerBand - lower band of matrix
// upperBand - upper band of matrix
// tadOnlyInputShapeInfo - TAD shape for input
// tadInputOffsets - TAD offsets for input
// tadOnlyOutputShapeInfo - TAD output shape
// tadOutputOffsets - TAD output offsets
// numTads - number of subarrays
// inputLength - input subarray length
//
    template <typename T>
    static __global__ void matrixBandKernel(const void* inputBuffer, const Nd4jLong* inputShape,
                                            void* outputBuffer, const Nd4jLong* outputShape,
                                            Nd4jLong lowerBand, Nd4jLong upperBand,
                                            const Nd4jLong* tadOnlyInputShapeInfo,  const Nd4jLong* tadInputOffsets,
                                            const Nd4jLong* tadOnlyOutputShapeInfo, const Nd4jLong* tadOutputOffsets,
                                            Nd4jLong numTads,
                                            Nd4jLong inputLength) {
        int totalThreads = blockDim.x;
        Nd4jLong rows = shape::sizeAt(inputShape, -2);
        Nd4jLong cols = shape::sizeAt(inputShape, -1);
        for (Nd4jLong e = blockIdx.x; e < numTads; e += gridDim.x) {
            auto yOffset = tadInputOffsets[e];
            auto xOffset = tadOutputOffsets[e];
            for (Nd4jLong i = blockIdx.y; i < rows; i += gridDim.y) {
                for (Nd4jLong j = threadIdx.x; j < cols; j += totalThreads) {
                    Nd4jLong coords[2] = {i, j};
                    Nd4jLong tadOffsetOut = shape::getOffset(tadOnlyOutputShapeInfo, coords);
                    Nd4jLong tadOffsetIn = shape::getOffset(tadOnlyInputShapeInfo, coords);

                    if (i >= j) { // check lower diagonals
                        if (lowerBand > 0) {
                            if ((i - j) > lowerBand)
                                *(reinterpret_cast<T *>(outputBuffer) + xOffset + tadOffsetOut) = T(0);
                            else
                                *(reinterpret_cast<T *>(outputBuffer) + xOffset + tadOffsetOut) = *(
                                        reinterpret_cast<T const *>(inputBuffer) + yOffset + tadOffsetIn);
                        }
                    } else if (j > i) {
                        if (upperBand > 0)
                            if ((j - i) > upperBand)
                                *(reinterpret_cast<T *>(outputBuffer) + xOffset + tadOffsetOut) = T(0);
                            else
                                *(reinterpret_cast<T *>(outputBuffer) + xOffset + tadOffsetOut) = *(
                                        reinterpret_cast<T const *>(inputBuffer) + yOffset + tadOffsetIn);
                    }
                }
            }
        }

    }
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// matrixBandPart_ - main algorithm caller
//
    template <typename T>
    void matrixBandPart_(sd::LaunchContext * context, NDArray* input, NDArray* output, Nd4jLong lowerBand, Nd4jLong upperBand) {
        dim3 launchDims(256, 512, 8192);
        auto stream = context->getCudaStream();

        std::vector<int> lastDims({input->rankOf() - 2, input->rankOf() - 1});
        std::vector<int> dimsToExclude = ShapeUtils::evalDimsToExclude(input->rankOf(), lastDims);

        auto packX = sd::ConstantTadHelper::getInstance().tadForDimensions(input->shapeInfo(), lastDims);
        auto packZ = sd::ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(), lastDims);

        const Nd4jLong numTads = packX.numberOfTads();

        NDArray::prepareSpecialUse({output}, {input});
        matrixBandKernel<T><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(input->specialBuffer(),
                input->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo(),
                lowerBand, upperBand, packX.specialShapeInfo(), packX.specialOffsets(), packZ.specialShapeInfo(), packZ.specialOffsets(), numTads, input->lengthOf());
        NDArray::registerSpecialUse({output}, {input});
    }

    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    void matrixBandPart(sd::LaunchContext * context, NDArray* input, NDArray* output, Nd4jLong lowerBand, Nd4jLong upperBand) {
        BUILD_SINGLE_SELECTOR(input->dataType(), matrixBandPart_, (context, input, output, lowerBand, upperBand), FLOAT_TYPES);
    }
}
}
}

