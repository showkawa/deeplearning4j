#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma (iuriish@yahoo.com)
// @author Oleh Semeniv (oleg.semeniv@gmail.com)
//

#include <system/op_boilerplate.h>
#include <ops/declarable/helpers/imagesHelpers.h>
#include <helpers/ConstantTadHelper.h>
#include <ops/declarable/helpers/adjust_hue.h>
#include <helpers/PointersManager.h>


namespace sd    {
namespace ops     {
namespace helpers {


///////////////////////////////////////////////////////////////////
template<typename T>
__global__ void rgbToYuvCuda(const void* vx, const Nd4jLong* xShapeInfo, const Nd4jLong* xTadOffsets, void* vz, const Nd4jLong *zShapeInfo, const Nd4jLong* zTadOffsets, const Nd4jLong numOfTads, const int dimC) {

    const T* x = reinterpret_cast<const T*>(vx);
    T* z = reinterpret_cast<T*>(vz);

    __shared__ int rank;
    __shared__ Nd4jLong xDimCstride, zDimCstride;

    if (threadIdx.x == 0) {
        rank = shape::rank(xShapeInfo);
        xDimCstride = shape::stride(xShapeInfo)[dimC];
        zDimCstride = shape::stride(zShapeInfo)[dimC];
    }
    __syncthreads();

    const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (Nd4jLong i = tid; i < numOfTads; i += gridDim.x * blockDim.x) {
        const T* xTad = x + xTadOffsets[i];
        T* zTad = z + zTadOffsets[i];

        rgbYuv<T>(xTad[0], xTad[xDimCstride], xTad[2 * xDimCstride], zTad[0], zTad[zDimCstride], zTad[2 * zDimCstride]);
    }

}

///////////////////////////////////////////////////////////////////
template<typename T>
linkage void rgbToYuvCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const hipStream_t* stream, const void* vx, const Nd4jLong* xShapeInfo, const Nd4jLong* xTadOffsets, void* vz, const Nd4jLong* zShapeInfo, const Nd4jLong* zTadOffsets, const Nd4jLong numOfTads, const int dimC) {

    rgbToYuvCuda<T> << <blocksPerGrid, threadsPerBlock, 256, * stream >> > (vx, xShapeInfo, xTadOffsets, vz, zShapeInfo, zTadOffsets, numOfTads, dimC);
}

///////////////////////////////////////////////////////////////////
void transformRgbYuv(sd::LaunchContext* context, const NDArray& input, NDArray& output, const int dimC) {

    auto packX = sd::ConstantTadHelper::getInstance().tadForDimensions(input.shapeInfo(), { dimC });
    auto packZ = sd::ConstantTadHelper::getInstance().tadForDimensions(output.shapeInfo(), { dimC });

    const Nd4jLong numOfTads = packX.numberOfTads();

    const int threadsPerBlock = MAX_NUM_THREADS / 2;
    const int blocksPerGrid = (numOfTads + threadsPerBlock - 1) / threadsPerBlock;

    PointersManager manager(context, "yuv_to_rgb");

    NDArray::prepareSpecialUse({ &output }, { &input });
    BUILD_SINGLE_SELECTOR(input.dataType(), rgbToYuvCudaLauncher, (blocksPerGrid, threadsPerBlock, context->getCudaStream(), input.specialBuffer(), input.specialShapeInfo(), packX.platformOffsets(), output.specialBuffer(), output.specialShapeInfo(), packZ.platformOffsets(), numOfTads, dimC), FLOAT_TYPES);
    NDArray::registerSpecialUse({ &output }, { &input });

    manager.synchronize();
}

///////////////////////////////////////////////////////////////////
template<typename T>
__global__ void yuvToRgbCuda(const void* vx, const Nd4jLong* xShapeInfo, const Nd4jLong* xTadOffsets, void* vz, const Nd4jLong *zShapeInfo, const Nd4jLong* zTadOffsets, const Nd4jLong numOfTads, const int dimC) {

    const T* x = reinterpret_cast<const T*>(vx);
    T* z = reinterpret_cast<T*>(vz);

    __shared__ int rank;
    __shared__ Nd4jLong xDimCstride, zDimCstride;

    if (threadIdx.x == 0) {
        rank = shape::rank(xShapeInfo);
        xDimCstride = shape::stride(xShapeInfo)[dimC];
        zDimCstride = shape::stride(zShapeInfo)[dimC];
    }
    __syncthreads();

    const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (Nd4jLong i = tid; i < numOfTads; i += gridDim.x * blockDim.x) {
        const T* xTad = x + xTadOffsets[i];
        T* zTad = z + zTadOffsets[i];

        yuvRgb<T>(xTad[0], xTad[xDimCstride], xTad[2 * xDimCstride], zTad[0], zTad[zDimCstride], zTad[2 * zDimCstride]);
    }

}

///////////////////////////////////////////////////////////////////
template<typename T>
linkage void yuvToRgbCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const hipStream_t* stream, const void* vx, const Nd4jLong* xShapeInfo, const Nd4jLong* xTadOffsets, void* vz, const Nd4jLong* zShapeInfo, const Nd4jLong* zTadOffsets, const Nd4jLong numOfTads, const int dimC) {

    yuvToRgbCuda<T> << <blocksPerGrid, threadsPerBlock, 256, * stream >> > (vx, xShapeInfo, xTadOffsets, vz, zShapeInfo, zTadOffsets, numOfTads, dimC);
}

///////////////////////////////////////////////////////////////////
void transformYuvRgb(sd::LaunchContext* context, const NDArray& input, NDArray& output, const int dimC) {

    auto packX = sd::ConstantTadHelper::getInstance().tadForDimensions(input.shapeInfo(), { dimC });
    auto packZ = sd::ConstantTadHelper::getInstance().tadForDimensions(output.shapeInfo(), { dimC });

    const Nd4jLong numOfTads = packX.numberOfTads();

    const int threadsPerBlock = MAX_NUM_THREADS / 2;
    const int blocksPerGrid = (numOfTads + threadsPerBlock - 1) / threadsPerBlock;

    PointersManager manager(context, "yuv_to_rgb");

    NDArray::prepareSpecialUse({ &output }, { &input });
    BUILD_SINGLE_SELECTOR(input.dataType(), yuvToRgbCudaLauncher, (blocksPerGrid, threadsPerBlock, context->getCudaStream(), input.specialBuffer(), input.specialShapeInfo(), packX.platformOffsets(), output.specialBuffer(), output.specialShapeInfo(), packZ.platformOffsets(), numOfTads, dimC), FLOAT_TYPES);
    NDArray::registerSpecialUse({ &output }, { &input });

    manager.synchronize();
}

///////////////////////////////////////////////////////////////////
// for example xShapeInfo = {2,3,4}, zShapeInfo = {2,1,4}
template<typename T>
__global__ void rgbToGrsCuda(const void *vx, const Nd4jLong *xShapeInfo, void *vz, const Nd4jLong *zShapeInfo, const int dimC) {

	const auto x = reinterpret_cast<const T*>(vx);
		  auto z = reinterpret_cast<T*>(vz);

	__shared__ Nd4jLong zLen;
	__shared__ int rank, *sharedMem;	// xRank == zRank

	if (threadIdx.x == 0) {
		extern __shared__ unsigned char shmem[];
        sharedMem = reinterpret_cast<int*>(shmem);

		zLen = shape::length(zShapeInfo);
		rank = shape::rank(zShapeInfo);
	}
	__syncthreads();

	auto coords = sharedMem + threadIdx.x * rank;

	for (Nd4jLong i = blockIdx.x * blockDim.x + threadIdx.x; i < zLen; i +=  gridDim.x * blockDim.x) {

		if (dimC == (rank - 1) && 'c' == shape::order(xShapeInfo) && 1 == shape::elementWiseStride(xShapeInfo) && 'c' == shape::order(zShapeInfo) && 1 == shape::elementWiseStride(zShapeInfo)) {
			const auto xStep = i*3;
            z[i] = 0.2989f * x[xStep] + 0.5870f * x[xStep + 1] + 0.1140f * x[xStep + 2];
		}
		else {

	    	shape::index2coords(i, zShapeInfo, coords);

            const auto zOffset  = shape::getOffset(zShapeInfo, coords);
            const auto xOffset0 = shape::getOffset(xShapeInfo, coords);
            const auto xOffset1 = xOffset0 + shape::stride(xShapeInfo)[dimC];
            const auto xOffset2 = xOffset1 + shape::stride(xShapeInfo)[dimC];

            z[zOffset] = 0.2989f * x[xOffset0] + 0.5870f * x[xOffset1] + 0.1140f * x[xOffset2];
		}
	}
}

///////////////////////////////////////////////////////////////////
template<typename T>
linkage void rgbToGrsCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream, const void *vx, const Nd4jLong *xShapeInfo, void *vz, const Nd4jLong *zShapeInfo, const int dimC) {

	rgbToGrsCuda<T><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(vx, xShapeInfo, vz, zShapeInfo, dimC);
}

///////////////////////////////////////////////////////////////////
void transformRgbGrs(sd::LaunchContext* context, const NDArray& input, NDArray& output, const int dimC) {

	PointersManager manager(context, "rgbToGrs");

    const int threadsPerBlock = MAX_NUM_THREADS / 4;
    const int blocksPerGrid = (input.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;
    const int sharedMem = input.rankOf() * sizeof(int) * threadsPerBlock + 128;

	NDArray::prepareSpecialUse({&output}, {&input});
	BUILD_SINGLE_SELECTOR(input.dataType(), rgbToGrsCudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, context->getCudaStream(), input.specialBuffer(), input.specialShapeInfo(), output.specialBuffer(), output.specialShapeInfo(), dimC), NUMERIC_TYPES);
	NDArray::registerSpecialUse({&output}, {&input});

	manager.synchronize();
}


///////////////////////////////////////////////////////////////////
template <typename T>
static void _CUDA_G rgbToHsvCuda(const void* vx, const Nd4jLong* xShapeInfo, const Nd4jLong* xTadOffsets,
                                  void* vz, const Nd4jLong *zShapeInfo, const Nd4jLong* zTadOffsets,
                                  const Nd4jLong numOfTads, const int dimC) {

    const T* x = reinterpret_cast<const T*>(vx);
    T* z = reinterpret_cast<T*>(vz);

    __shared__ int rank;
    __shared__ Nd4jLong xDimCstride, zDimCstride;

    if (threadIdx.x == 0) {
        rank = shape::rank(xShapeInfo);
        xDimCstride = shape::stride(xShapeInfo)[dimC];
        zDimCstride = shape::stride(zShapeInfo)[dimC];
    }
    __syncthreads();

    const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (Nd4jLong i = tid; i < numOfTads; i += gridDim.x * blockDim.x) {
        const T* xTad = x + xTadOffsets[i];
        T* zTad = z + zTadOffsets[i];

        rgbToHsv<T>(xTad[0], xTad[xDimCstride], xTad[2 * xDimCstride], zTad[0], zTad[zDimCstride], zTad[2 * zDimCstride]);
    }
}

///////////////////////////////////////////////////////////////////
template <typename T>
static void _CUDA_G hsvToRgbCuda(const void* vx, const Nd4jLong* xShapeInfo, const Nd4jLong* xTadOffsets,
                                 void* vz, const Nd4jLong *zShapeInfo, const Nd4jLong* zTadOffsets,
                                 const Nd4jLong numOfTads, const int dimC) {

    const T* x = reinterpret_cast<const T*>(vx);
    T* z = reinterpret_cast<T*>(vz);

    __shared__ int rank;
    __shared__ Nd4jLong xDimCstride, zDimCstride;

    if (threadIdx.x == 0) {
        rank = shape::rank(xShapeInfo);
        xDimCstride = shape::stride(xShapeInfo)[dimC];
        zDimCstride = shape::stride(zShapeInfo)[dimC];
    }
    __syncthreads();

    const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (Nd4jLong i = tid; i < numOfTads; i += gridDim.x * blockDim.x) {
        const T* xTad = x + xTadOffsets[i];
        T* zTad = z + zTadOffsets[i];

        hsvToRgb<T>(xTad[0], xTad[xDimCstride], xTad[2 * xDimCstride], zTad[0], zTad[zDimCstride], zTad[2 * zDimCstride]);
    }
}

///////////////////////////////////////////////////////////////////
template<typename T>
static _CUDA_H void hsvToRgbCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const hipStream_t *stream,
                                          const void* vx, const Nd4jLong* xShapeInfo, const Nd4jLong* xTadOffsets,
                                          void* vz, const Nd4jLong* zShapeInfo, const Nd4jLong* zTadOffsets,
                                          const Nd4jLong numOfTads, const int dimC) {

    hsvToRgbCuda<T><<<blocksPerGrid, threadsPerBlock, 256, *stream>>>(vx, xShapeInfo, xTadOffsets, vz, zShapeInfo, zTadOffsets, numOfTads, dimC);
}

template<typename T>
static _CUDA_H void rgbToHsvCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const hipStream_t *stream,
                                         const void* vx, const Nd4jLong* xShapeInfo, const Nd4jLong* xTadOffsets,
                                         void* vz, const Nd4jLong* zShapeInfo, const Nd4jLong* zTadOffsets,
                                         const Nd4jLong numOfTads, const int dimC) {

    rgbToHsvCuda<T><<<blocksPerGrid, threadsPerBlock, 256, *stream>>>(vx, xShapeInfo, xTadOffsets, vz, zShapeInfo, zTadOffsets, numOfTads, dimC);
}

///////////////////////////////////////////////////////////////////
void transformHsvRgb(sd::LaunchContext* context, const NDArray* input, NDArray* output, const int dimC) {

    auto packX = sd::ConstantTadHelper::getInstance().tadForDimensions(input->shapeInfo(),  {dimC});
    auto packZ = sd::ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(), {dimC});

    const Nd4jLong numOfTads = packX.numberOfTads();

    const int threadsPerBlock = MAX_NUM_THREADS / 2;
    const int blocksPerGrid = (numOfTads + threadsPerBlock - 1) / threadsPerBlock;

    PointersManager manager(context, "hsv_to_rgb");

    NDArray::prepareSpecialUse({output}, {input});
    BUILD_SINGLE_SELECTOR(input->dataType(), hsvToRgbCudaLauncher, (blocksPerGrid, threadsPerBlock, context->getCudaStream(), input->specialBuffer(), input->specialShapeInfo(), packX.platformOffsets(), output->specialBuffer(), output->specialShapeInfo(), packZ.platformOffsets(), numOfTads, dimC), FLOAT_TYPES);
    NDArray::registerSpecialUse({output}, {input});

    manager.synchronize();
}

///////////////////////////////////////////////////////////////////
void transformRgbHsv(sd::LaunchContext* context, const NDArray* input, NDArray* output, const int dimC) {
    auto packX = sd::ConstantTadHelper::getInstance().tadForDimensions(input->shapeInfo(),  {dimC});
    auto packZ = sd::ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(), {dimC});

    const Nd4jLong numOfTads = packX.numberOfTads();

    const int threadsPerBlock = MAX_NUM_THREADS / 2;
    const int blocksPerGrid = (numOfTads + threadsPerBlock - 1) / threadsPerBlock;

    PointersManager manager(context, "rgb_to_hsv");

    NDArray::prepareSpecialUse({output}, {input});
    BUILD_SINGLE_SELECTOR(input->dataType(), rgbToHsvCudaLauncher, (blocksPerGrid, threadsPerBlock, context->getCudaStream(), input->specialBuffer(), input->specialShapeInfo(), packX.platformOffsets(), output->specialBuffer(), output->specialShapeInfo(), packZ.platformOffsets(), numOfTads, dimC), FLOAT_TYPES);
    NDArray::registerSpecialUse({output}, {input});

    manager.synchronize();
}

template<typename T>
__global__ void tripleTransformerCuda(const void *vx, const Nd4jLong *xShapeInfo, const Nd4jLong *xTadShapeInfo, const Nd4jLong *xOffsets, void *vz, const Nd4jLong *zShapeInfo, const Nd4jLong *zTadShapeInfo, const Nd4jLong *zOffsets, const int dimC, int mode, uint64_t numTads) {
    const auto x = reinterpret_cast<const T*>(vx);
    auto z = reinterpret_cast<T*>(vz);

    __shared__ Nd4jLong zLen, *sharedMem;
    __shared__ int rank;	// xRank == zRank

    float yiqarr[3][3] = {
            { 0.299f,  0.59590059f,  0.2115f },
            { 0.587f, -0.27455667f,  -0.52273617f },
            { 0.114f, -0.32134392f,  0.31119955f }
    };

    float rgbarr[3][3] = {
            { 1.f,  1.f,  1.f },
            { 0.95598634f, -0.27201283f, -1.10674021f },
            { 0.6208248f, -0.64720424f, 1.70423049f }
    };

    auto tr = mode == 1? yiqarr : rgbarr;

    if (threadIdx.x == 0) {
        extern __shared__ unsigned char shmem[];
        sharedMem = reinterpret_cast<Nd4jLong*>(shmem);

        zLen = shape::length(zShapeInfo);
        rank = shape::rank(zShapeInfo);
    }
    __syncthreads();

    Nd4jLong* coords = sharedMem + threadIdx.x * rank;

    if (dimC == (rank - 1) && 'c' == shape::order(xShapeInfo) && 1 == shape::elementWiseStride(xShapeInfo) && 'c' == shape::order(zShapeInfo) && 1 == shape::elementWiseStride(zShapeInfo)) {
        for (uint64_t f = blockIdx.x * blockDim.x + threadIdx.x; f < zLen / 3; f +=  gridDim.x * blockDim.x) {
            auto i = f * 3;

            auto xi0 = x[i];
            auto xi1 = x[i+1];
            auto xi2 = x[i+2];

            for (int e = 0; e < 3; e++)
                z[i + e] = xi0 * tr[0][e] + xi1 * tr[1][e] + xi2 * tr[2][e];
        }
    } else {
        // TAD based case
        const Nd4jLong xDimCstride = shape::stride(xShapeInfo)[dimC];
        const Nd4jLong zDimCstride = shape::stride(zShapeInfo)[dimC];

        for (uint64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < numTads; i += blockDim.x * gridDim.x) {
            const T* xTad = x + xOffsets[i];
            T* zTad = z + zOffsets[i];

            auto xi0 = xTad[0];
            auto xi1 = xTad[xDimCstride];
            auto xi2 = xTad[xDimCstride * 2];

            for (int e = 0; e < 3; e++)
                zTad[zDimCstride * e] = xi0 * tr[0][e] + xi1 * tr[1][e] + xi2 * tr[2][e];
        }
    }
}


template <typename T>
static void rgbYiq(sd::LaunchContext* context, const NDArray* input, NDArray* output, const int dimC) {
    auto packX = sd::ConstantTadHelper::getInstance().tadForDimensions(input->shapeInfo(), dimC);
    auto packZ = sd::ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(), dimC);

    NDArray::prepareSpecialUse({output}, {input});
    return tripleTransformerCuda<T><<<256, 256, 8192, *context->getCudaStream()>>>(input->specialBuffer(), input->specialShapeInfo(), packX.platformShapeInfo(), packX.platformOffsets(), output->specialBuffer(), output->specialShapeInfo(), packZ.platformShapeInfo(), packZ.platformOffsets(), dimC, 1, packZ.numberOfTads());
    NDArray::registerSpecialUse({output}, {input});
}

template <typename T>
FORCEINLINE static void yiqRgb(sd::LaunchContext* context, const NDArray* input, NDArray* output, const int dimC) {
    auto packX = sd::ConstantTadHelper::getInstance().tadForDimensions(input->shapeInfo(), dimC);
    auto packZ = sd::ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(), dimC);

    NDArray::prepareSpecialUse({output}, {input});
    return tripleTransformerCuda<T><<<256, 256, 8192, *context->getCudaStream()>>>(input->specialBuffer(), input->specialShapeInfo(), packX.platformShapeInfo(), packX.platformOffsets(), output->specialBuffer(), output->specialShapeInfo(), packZ.platformShapeInfo(), packZ.platformOffsets(), dimC, 2, packZ.numberOfTads());
    NDArray::registerSpecialUse({output}, {input});
}

void transformYiqRgb(sd::LaunchContext* context, const NDArray* input, NDArray* output, const int dimC) {
    BUILD_SINGLE_SELECTOR(input->dataType(), yiqRgb, (context, input, output, dimC), FLOAT_TYPES);
}

void transformRgbYiq(sd::LaunchContext* context, const NDArray* input, NDArray* output, const int dimC) {
    BUILD_SINGLE_SELECTOR(input->dataType(), rgbYiq, (context, input, output, dimC), FLOAT_TYPES);
}





}
}
}

