/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//
//

#include <ops/declarable/helpers/d_t_s.h>

namespace sd {
namespace ops {
namespace helpers {

    template <typename T>
    static _CUDA_G void depthToSpaceKernel(const void *vx, const Nd4jLong *xShapeInfo, void *vz, const Nd4jLong *zShapeInfo, const int block_size, const bool isNHWC) {
        auto input_ptr = reinterpret_cast<const T *>(vx);
        auto output_ptr = reinterpret_cast<T *>(vz);

        const int batch_size = shape::sizeAt(xShapeInfo, 0);
        const int input_depth = isNHWC ? shape::sizeAt(xShapeInfo, 3) : shape::sizeAt(xShapeInfo, 1);
        const int input_height = isNHWC ? shape::sizeAt(xShapeInfo, 1) : shape::sizeAt(xShapeInfo, 2);
        const int input_width = isNHWC ? shape::sizeAt(xShapeInfo, 2) : shape::sizeAt(xShapeInfo, 3);

        const int output_depth = isNHWC ? shape::sizeAt(zShapeInfo, 3) : shape::sizeAt(zShapeInfo, 1);
        const int output_height = isNHWC ? shape::sizeAt(zShapeInfo, 1) : shape::sizeAt(zShapeInfo, 2);
        const int output_width = isNHWC ? shape::sizeAt(zShapeInfo, 2) : shape::sizeAt(zShapeInfo, 3);

        const int input_area = input_width * input_height;
        const int input_depth_by_input_area = input_depth * input_area;
        const int output_depth_by_input_height = output_depth * input_height;

        auto tid = threadIdx.x + blockIdx.x * blockDim.x;

        if (isNHWC) {
            const int total_count = batch_size * output_height * output_width * output_depth;
            for (int out_idx = tid; out_idx < total_count; out_idx += blockDim.x * gridDim.x) {
                const int d = out_idx % output_depth;
                const int out_idx2 = out_idx / output_depth;
                const int w = out_idx2 % output_width;
                const int out_idx3 = out_idx2 / output_width;
                const int h = out_idx3 % output_height;
                const int b = out_idx3 / output_height;

                const int in_h = h / block_size;
                const int offset_h = h % block_size;
                const int in_w = w / block_size;
                const int offset_w = w % block_size;
                const int offset_d = (offset_h * block_size + offset_w) * output_depth;
                const int in_d = d + offset_d;
                const int inp_idx = in_d + input_depth * (in_w + input_width * (in_h + input_height * b));
                (output_ptr + out_idx)[0] = (input_ptr + inp_idx)[0];
            }
        } else {
            const int total_count = batch_size * input_depth_by_input_area;

            for (int input_idx = tid; input_idx < total_count; input_idx += blockDim.x * gridDim.x) {
                const int n_bY_bX_oC_iY = input_idx / input_width;
                const int iX = input_idx - n_bY_bX_oC_iY * input_width;

                const int n_bY_bX = n_bY_bX_oC_iY / output_depth_by_input_height;
                const int oC_iY = n_bY_bX_oC_iY - n_bY_bX * output_depth_by_input_height;

                const int n_bY = n_bY_bX / block_size;
                const int bX = n_bY_bX - n_bY * block_size;

                const int n = n_bY / block_size;
                const int bY = n_bY - n * block_size;

                const int output_idx = bX + block_size * (iX + input_width * (bY + block_size * (oC_iY + n * output_depth_by_input_height)));

                (output_ptr + output_idx)[0] = (input_ptr + input_idx)[0];
            }
        }
    }


    template <typename T>
    static void __depthToSpace(sd::LaunchContext * context, const NDArray &input, NDArray *output, int block_size, bool isNHWC) {
        depthToSpaceKernel<T><<<512, 512, 1024, *context->getCudaStream()>>>(input.specialBuffer(), input.specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo(), block_size, isNHWC);
    }

    void _depthToSpace(sd::LaunchContext * context, const NDArray &input, NDArray *output, int block_size, bool isNHWC) {
        auto xType = input.dataType();

        NDArray::prepareSpecialUse({output}, {&input});

        BUILD_SINGLE_SELECTOR(xType, __depthToSpace, (context, input, output, block_size, isNHWC), LIBND4J_TYPES);
        NDArray::registerSpecialUse({output}, {&input});
    }
}
}
}