#include "hip/hip_runtime.h"
/*
 *  ******************************************************************************
 *  *
 *  *
 *  * This program and the accompanying materials are made available under the
 *  * terms of the Apache License, Version 2.0 which is available at
 *  * https://www.apache.org/licenses/LICENSE-2.0.
 *  *
 *  * See the NOTICE file distributed with this work for additional
 *  * information regarding copyright ownership.
 *  * Unless required by applicable law or agreed to in writing, software
 *  * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 *  * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 *  * License for the specific language governing permissions and limitations
 *  * under the License.
 *  *
 *  * SPDX-License-Identifier: Apache-2.0
 *  *****************************************************************************
 */

//
//  @author GS <sgazeos@gmail.com>
//

#include <system/op_boilerplate.h>
#include <array/NDArray.h>
#include <array/NDArrayFactory.h>
#include <helpers/MmulHelper.h>

#include <execution/Threads.h>
#include <helpers/ConstantTadHelper.h>
#include "../triangular_solve.h"
#include "../lup.h"
#include "../solve.h"

namespace sd {
    namespace ops {
        namespace helpers {
    
            template <typename T>
            static __global__ void oneOnDiagonalKernel(T* ioBuf, Nd4jLong const*  ioShape, Nd4jLong const*  tadShape, Nd4jLong const*  tadOffsets, Nd4jLong batchNum, Nd4jLong rowNum) {
                for (auto i = blockIdx.x; i < batchNum; i += gridDim.x) {
                    auto matrixPart = ioBuf + tadOffsets[i];
                    for (auto j = threadIdx.x; j < rowNum; j += blockDim.x) {
                        Nd4jLong pos[] = {j, j};
                        auto offset = shape::getOffset(tadShape, pos);

                        matrixPart[offset] = T(1.f);
                    }
                }
            }

            template <typename T>
            static __global__ void restorePermutationsKernel(T* PBuf, Nd4jLong const*  PShapeInfo, int const* permutationsBuf,
            Nd4jLong const*  PTadShapeInfo, Nd4jLong const*  PTadSOffsets, Nd4jLong const*  permutationsTadShapeInfo, Nd4jLong const*  permutationsTadOffsets, Nd4jLong batchNum, Nd4jLong rowNum) {
                for (auto batch = blockIdx.x; batch < batchNum; batch += gridDim.x) {
                    auto permutations = permutationsBuf + permutationsTadOffsets[batch];
                    auto P = PBuf + PTadSOffsets[batch];

                    for (auto row = threadIdx.x; row < rowNum; row += blockDim.x) {
                        //auto posX[] = {row};
                        Nd4jLong posZ[] = {row, permutations[row]};
                        auto zOffset = shape::getOffset(PTadShapeInfo, posZ);
                        P[zOffset] = T(1.f);
                    }
                }
            }

            template <typename T>
            static int solveFunctor_(sd::LaunchContext * context, NDArray* leftInput, NDArray* rightInput,
                                     bool adjoint, NDArray* output) {
                NDArray::prepareSpecialUse({output}, {leftInput, rightInput});
                // stage 1: LU decomposition batched
                auto leftOutput = leftInput->ulike();
                auto permuShape = rightInput->getShapeAsVector(); permuShape.pop_back();
                auto permutations = NDArrayFactory::create<int>('c', permuShape, context);
                helpers::lu(context, leftInput, &leftOutput, &permutations);
                auto leftLower = leftOutput.dup();
                auto rightOutput = rightInput->ulike();
                auto leftLowerTad = ConstantTadHelper::getInstance().tadForDimensions(leftLower.shapeInfo(), {-2, -1});
                auto stream = context->getCudaStream();
                oneOnDiagonalKernel<T><<<128, 256, 256, *stream>>>(leftLower.dataBuffer()->specialAsT<T>(), leftLower.specialShapeInfo(), leftLowerTad.specialShapeInfo(), leftLowerTad.specialOffsets(), leftLowerTad.numberOfTads(), leftLower.sizeAt(-1));
                auto P = leftOutput.ulike(); P.nullify();
                auto PTad = ConstantTadHelper::getInstance().tadForDimensions(P.shapeInfo(), {-2, -1});
                auto permutationsTad = ConstantTadHelper::getInstance().tadForDimensions(permutations.shapeInfo(), {-1});
                restorePermutationsKernel<T><<<128, 256, 256, *stream>>>(P.dataBuffer()->specialAsT<T>(), P.specialShapeInfo(), permutations.dataBuffer()->specialAsT<int>(),
                        PTad.specialShapeInfo(), PTad.specialOffsets(), permutationsTad.specialShapeInfo(), permutationsTad.specialOffsets(), permutationsTad.numberOfTads(), permutations.sizeAt(-1));
                P.tickWriteDevice();
                auto rightPart = rightInput->ulike();
                MmulHelper::matmul(&P, rightInput, &rightPart, 0, 0);
                
                // stage 2: triangularSolveFunctor for Lower with given b
                helpers::triangularSolveFunctor(context, &leftLower, &rightPart, true, false, &rightOutput);
                // stage 3: triangularSolveFunctor for Upper with output of previous stage
                helpers::triangularSolveFunctor(context, &leftOutput, &rightOutput, false, false, output);
                NDArray::registerSpecialUse({output}, {leftInput, rightInput});

                return Status::OK();
            }

            int solveFunctor(sd::LaunchContext * context, NDArray* leftInput, NDArray* rightInput, bool adjoint, NDArray* output) {
                BUILD_SINGLE_SELECTOR(leftInput->dataType(), return solveFunctor_, (context, leftInput, rightInput, adjoint, output), FLOAT_TYPES);
            }

            template <typename T>
            static __global__ void adjointKernel(T* output, Nd4jLong batchSize, Nd4jLong rows, Nd4jLong columns, Nd4jLong const*  outputTads, 
                                                 Nd4jLong const*  outputOffsets) {

                for (auto b = blockIdx.x; b < batchSize; b += gridDim.x) {
                    auto outputPart = output + outputOffsets[b];
                    for (auto r = threadIdx.x; r < rows; r += blockDim.x) {
                        for (auto c = threadIdx.y; c < r; c += blockDim.y) {
                            Nd4jLong zPos[] = {r, c};
                            Nd4jLong xPos[] = {c, r};
                            auto zIndex = shape::getOffset(outputTads, zPos);
                            auto xIndex = shape::getOffset(outputTads, xPos);
                            math::nd4j_swap(outputPart[zIndex], outputPart[xIndex]);
                        }
                    }
                }

            }

            template <typename T>
            static void adjointMatrix_(sd::LaunchContext* context, NDArray const* input, NDArray* output) {
                NDArray::prepareSpecialUse({output}, {input});
                auto inputTads = ConstantTadHelper::getInstance().tadForDimensions(input->shapeInfo(), {-2, -1});
                auto outputTads = ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(), {-2, -1});
                auto stream = context->getCudaStream();
                auto outputBuf = reinterpret_cast<T*>(output->specialBuffer());
                auto rows = input->sizeAt(-2);
                auto columns = input->sizeAt(-1);
                output->assign(input);
                adjointKernel<T><<<128, 256, 256, *stream>>>(outputBuf, outputTads.numberOfTads(), rows, columns, outputTads.specialShapeInfo(), outputTads.specialOffsets());
                NDArray::registerSpecialUse({output}, {input});
            }

            void adjointMatrix(sd::LaunchContext* context, NDArray const* input, NDArray* output) {
                BUILD_SINGLE_SELECTOR(input->dataType(), adjointMatrix_, (context, input, output), FLOAT_TYPES);
            }

        }
    }
}
