#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
// @author Yurii Shyrma (iuriish@yahoo.com)
//

#include <ops/declarable/helpers/scatter.h>
#include <numeric>
#include <helpers/ShapeUtils.h>
#include <helpers/TAD.h>
#include <helpers/ConstantShapeHelper.h>
#include <helpers/ConstantTadHelper.h>
#include <helpers/PointersManager.h>

namespace sd    {
namespace ops     {
namespace helpers {


///////////////////////////////////////////////////////////////////
// x - indices, y - contains number of bad indices, z - input/output
template<typename X>
__global__ static void checkIndicesCuda(const void *vx, const Nd4jLong *xShapeInfo, Nd4jLong* y, const Nd4jLong *zShapeInfo, const int axis) {

    const auto x = reinterpret_cast<const X*>(vx);

    __shared__ int xRank, *coords, xLastDim;
    __shared__ Nd4jLong xLen, numOfBadIndxPerBlock;

    if (threadIdx.x == 0) {
        extern __shared__ unsigned char shmem[];
        coords = reinterpret_cast<int*>(shmem);

        xRank = shape::rank(xShapeInfo);
        xLen  = shape::length(xShapeInfo);

        numOfBadIndxPerBlock = 0;
    }
    __syncthreads();

    auto xCoords = coords + threadIdx.x * xRank;

    for (Nd4jLong i = blockIdx.x * blockDim.x + threadIdx.x; i < xLen; i += gridDim.x * blockDim.x) {

        shape::index2coords(i, xShapeInfo, xCoords);

        const Nd4jLong currentInd = x[shape::getOffset(xShapeInfo, xCoords)];

        if(currentInd >= shape::sizeAt(zShapeInfo, axis == -1 ? xCoords[xRank-1] : axis)) {
            printf("checkIndices cuda: out of range element %lld at index %lld \n", currentInd,  i);
            sd::math::atomics::nd4j_atomicAdd<Nd4jLong>(&numOfBadIndxPerBlock, 1);
        }
    }
    __syncthreads();

    if (threadIdx.x == 0 && numOfBadIndxPerBlock != 0)
        sd::math::atomics::nd4j_atomicAdd<Nd4jLong>(y, numOfBadIndxPerBlock);
}

///////////////////////////////////////////////////////////////////
template<typename X>
static void checkIndicesCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream,
                                              const void *vx, const Nd4jLong *xShapeInfo, Nd4jLong* y, const Nd4jLong *zShapeInfo, const int axis) {

    checkIndicesCuda<X><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(vx, xShapeInfo, y, zShapeInfo, axis);
}


///////////////////////////////////////////////////////////////////
Nd4jLong checkIndices(sd::LaunchContext *context, const NDArray& indices, const NDArray& output, const int axis) {

    const int threadsPerBlock = MAX_NUM_THREADS / 2;
    const int blocksPerGrid = (indices.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;
    const int sharedMem = threadsPerBlock * sizeof(int) * indices.rankOf() + 256;

    const auto xType = indices.dataType();

    PointersManager manager(context, "scatterNDcheckIndices");

    // scalar, initial value = 0
    NDArray numOfBadIndx(sd::DataType::INT64, context, true);

    NDArray::prepareSpecialUse({&numOfBadIndx}, {&indices});
    BUILD_SINGLE_SELECTOR(xType, checkIndicesCudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, context->getCudaStream(), indices.specialBuffer(), indices.specialShapeInfo(), reinterpret_cast<Nd4jLong*>(numOfBadIndx.specialBuffer()), output.specialShapeInfo(), axis), INDEXING_TYPES);
    NDArray::registerSpecialUse({&numOfBadIndx}, {&indices});

    manager.synchronize();

    return numOfBadIndx.t<Nd4jLong>(0);
}

///////////////////////////////////////////////////////////////////
// x - indices, y - updates, z - input/output
template<typename X, typename Y>
__global__ static void scatterLockCuda(const int opCode,
                                        const void *vx, const Nd4jLong *xShapeInfo,
                                        const void *vy, const Nd4jLong *yShapeInfo,
                                              void *vz, const Nd4jLong *zShapeInfo) {

    const auto x = reinterpret_cast<const X*>(vx);
    const auto y = reinterpret_cast<const Y*>(vy);
          auto z = reinterpret_cast<Y*>(vz);

    __shared__ int xRank, yRank, zRank, xNonUnitDim, yNonUnitDim, zNonUnitDim, *coords;
    __shared__ Nd4jLong xLen, zLen;
    __shared__ bool is1Dcase, xySameStride;

    if (threadIdx.x == 0) {
        extern __shared__ unsigned char shmem[];
        coords = reinterpret_cast<int*>(shmem);

        xLen = shape::length(xShapeInfo);
        zLen = shape::length(zShapeInfo);

        xRank = shape::rank(xShapeInfo);
        yRank = shape::rank(yShapeInfo);
        zRank = shape::rank(zShapeInfo);

        xNonUnitDim = yNonUnitDim = zNonUnitDim = 0;

        is1Dcase = (shape::isCommonVector(zShapeInfo, zNonUnitDim) || shape::isScalar(zShapeInfo)) && (shape::isCommonVector(yShapeInfo, yNonUnitDim) || shape::isScalar(yShapeInfo)) && (shape::isCommonVector(xShapeInfo, xNonUnitDim) || shape::isScalar(xShapeInfo));

        if(is1Dcase)
            xySameStride = shape::stride(xShapeInfo)[xNonUnitDim] = shape::stride(yShapeInfo)[yNonUnitDim];
    }
    __syncthreads();


    Nd4jLong yOffset, zOffset;
    int zFirstCoord, *yCoords, *zCoords;

    for (Nd4jLong i = blockIdx.x * blockDim.x + threadIdx.x; i < zLen; i += gridDim.x * blockDim.x) {

        if(!is1Dcase) {

            yCoords = coords + threadIdx.x * (yRank + zRank);
            zCoords = yCoords + yRank;
            shape::index2coords(i, zShapeInfo, zCoords);
        }

        for (Nd4jLong j = 0; j < xLen; ++j) {

            if(is1Dcase) {

                yOffset = j * shape::stride(yShapeInfo)[yNonUnitDim];
                zFirstCoord = x[xySameStride ? yOffset : j * shape::stride(xShapeInfo)[xNonUnitDim]];

                if(i != zFirstCoord)
                    continue;

                zOffset = i * shape::stride(zShapeInfo)[zNonUnitDim];
            }

            else {

                shape::index2coords(j, xShapeInfo, yCoords);                 // first xRank coordinates in yCoords are the same for y and x

                zFirstCoord = x[shape::getOffset(xShapeInfo, yCoords)];

                if(zCoords[0] != zFirstCoord)
                    continue;

                for (uint k = 0; k < yRank - xRank; ++k)
                    yCoords[xRank + k] = zCoords[k + 1];

                yOffset = shape::getOffset(yShapeInfo, yCoords);
                zOffset = shape::getOffset(zShapeInfo, zCoords);
            }

            switch (opCode) {
                case pairwise::Add:
                    z[zOffset] += y[yOffset];
                    break;
                case pairwise::Subtract:
                    z[zOffset] -= y[yOffset];
                    break;
                case pairwise::Multiply:
                    z[zOffset] *= y[yOffset];
                    break;
                case pairwise::Divide:
                    z[zOffset] /= y[yOffset];
                    break;
                case pairwise::ReverseSubtract:
                    z[zOffset] = y[yOffset] - z[zOffset];
                    break;
                case pairwise::ReverseDivide:
                    z[zOffset] = y[yOffset] / z[zOffset];
                    break;
                case pairwise::CopyPws:
                    z[zOffset] = y[yOffset];
                    break;
                case pairwise::MaxPairwise:
                    if(z[zOffset] < y[yOffset]) z[zOffset] = y[yOffset];
                    break;
                case pairwise::MinPairwise:
                    if(z[zOffset] > y[yOffset]) z[zOffset] = y[yOffset];
                    break;
                default:
                    continue;
            }
        }
    }
}


///////////////////////////////////////////////////////////////////
// x - indices, y - updates, z - input/output
template<typename X, typename Y>
__global__ static void scatterCuda(const int opCode,
                                   const void *vx, const Nd4jLong *xShapeInfo,
                                   const void *vy, const Nd4jLong *yShapeInfo,
                                         void *vz, const Nd4jLong *zShapeInfo) {

    const auto x = reinterpret_cast<const X*>(vx);
    const auto y = reinterpret_cast<const Y*>(vy);
          auto z = reinterpret_cast<Y*>(vz);

    __shared__ int xRank, yRank, zRank, xNonUnitDim, yNonUnitDim, zNonUnitDim, *coords;
    __shared__ Nd4jLong yLen;
    __shared__ bool is1Dcase, xySameStride;

    if (threadIdx.x == 0) {
        extern __shared__ unsigned char shmem[];
        coords = reinterpret_cast<int*>(shmem);

        yLen = shape::length(yShapeInfo);

        xRank = shape::rank(xShapeInfo);
        yRank = shape::rank(yShapeInfo);
        zRank = shape::rank(zShapeInfo);

        xNonUnitDim = yNonUnitDim = zNonUnitDim = 0;

        is1Dcase = (shape::isCommonVector(zShapeInfo, zNonUnitDim) || shape::isScalar(zShapeInfo)) && (shape::isCommonVector(yShapeInfo, yNonUnitDim) || shape::isScalar(yShapeInfo)) && (shape::isCommonVector(xShapeInfo, xNonUnitDim) || shape::isScalar(xShapeInfo));

        if(is1Dcase)
            xySameStride = shape::stride(xShapeInfo)[xNonUnitDim] = shape::stride(yShapeInfo)[yNonUnitDim];
    }
    __syncthreads();


    Nd4jLong xOffset, yOffset, zOffset;
    int *yCoords, *zCoords;

    if(!is1Dcase) {
        yCoords = coords + threadIdx.x * (yRank + zRank);
        zCoords = yCoords + yRank;
    }

    for (Nd4jLong i = blockIdx.x * blockDim.x + threadIdx.x; i < yLen; i += gridDim.x * blockDim.x) {

        if(is1Dcase) {

            yOffset = i * shape::stride(yShapeInfo)[yNonUnitDim];
            zOffset = x[xySameStride ? yOffset : i * shape::stride(xShapeInfo)[xNonUnitDim]] * shape::stride(zShapeInfo)[zNonUnitDim];
        }
        else {
            shape::index2coords(i, yShapeInfo, yCoords);

            yOffset = shape::getOffset(yShapeInfo, yCoords);
            xOffset = shape::getOffset(xShapeInfo, yCoords);                // first xRank coordinates in yCoords are the same for y and x -> for (uint j = 0; j < xRank; ++j) xCoords[j] = yCoords[j];

            zCoords[0] = x[xOffset];

            for (uint j = 0; j < yRank - xRank; ++j)
                zCoords[j + 1] = yCoords[xRank + j];

            zOffset = shape::getOffset(zShapeInfo, zCoords);
        }

        switch (opCode) {
            case pairwise::Add:
                z[zOffset] += y[yOffset];
                break;
            case pairwise::Subtract:
                z[zOffset] -= y[yOffset];
                break;
            case pairwise::Multiply:
                z[zOffset] *= y[yOffset];
                break;
            case pairwise::Divide:
                z[zOffset] /= y[yOffset];
                break;
            case pairwise::ReverseSubtract:
                z[zOffset] = y[yOffset] - z[zOffset];
                break;
            case pairwise::ReverseDivide:
                z[zOffset] = y[yOffset] / z[zOffset];
                break;
            case pairwise::CopyPws:
                z[zOffset] = y[yOffset];
                break;
            case pairwise::MaxPairwise:
                if(z[zOffset] < y[yOffset]) z[zOffset] = y[yOffset];
                break;
            case pairwise::MinPairwise:
                if(z[zOffset] > y[yOffset]) z[zOffset] = y[yOffset];
                break;
            default:
                continue;
        }
    }
}

///////////////////////////////////////////////////////////////////
template<typename X, typename Y>
static void scatterCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream,
                                const int opCode,
                                const void *vx, const Nd4jLong *xShapeInfo,
                                const void *vy, const Nd4jLong *yShapeInfo,
                                      void *vz, const Nd4jLong *zShapeInfo,
                                const bool lock) {

    if(lock)
        scatterLockCuda<X,Y><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(opCode, vx, xShapeInfo, vy, yShapeInfo, vz, zShapeInfo);
    else
        scatterCuda<X,Y><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(opCode, vx, xShapeInfo, vy, yShapeInfo, vz, zShapeInfo);
}


///////////////////////////////////////////////////////////////////
void scatter(sd::LaunchContext  *context, pairwise::Ops op, const NDArray& indices, const NDArray& updates, NDArray& output, const bool lock) {

    const auto xType = indices.dataType();
    const auto yType = updates.dataType();

    const int threadsPerBlock = MAX_NUM_THREADS / 4;
    const int blocksPerGrid = ((lock ? output.lengthOf() : updates.lengthOf()) + threadsPerBlock - 1) / threadsPerBlock;
    const int sharedMem = sizeof(int) * threadsPerBlock * (updates.rankOf() + output.rankOf()) + 256;

    PointersManager manager(context, "scatter");

    NDArray::prepareSpecialUse({&output}, {&updates, &indices});
    BUILD_DOUBLE_SELECTOR(xType, yType, scatterCudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, context->getCudaStream(), op, indices.specialBuffer(), indices.specialShapeInfo(), updates.specialBuffer(), updates.specialShapeInfo(), output.specialBuffer(), output.specialShapeInfo(), lock), INDEXING_TYPES, GENERIC_NUMERIC_TYPES);
    NDArray::registerSpecialUse({&output}, {&updates, &indices});

    manager.synchronize();
}

///////////////////////////////////////////////////////////////////
// x - indices, y - updates, z - output
template<typename X, typename Y>
__global__ static void scatterNDLockCuda(const int opCode,
                                        const void *vx, const Nd4jLong *xShapeInfo,
                                        const void *vy, const Nd4jLong *yShapeInfo,
                                              void *vz, const Nd4jLong *zShapeInfo) {

    const auto x = reinterpret_cast<const X*>(vx);
    const auto y = reinterpret_cast<const Y*>(vy);
          auto z = reinterpret_cast<Y*>(vz);

    __shared__ int xRank, yRank, zRank, biggerXYRank, xLastDim, *coords, xNonUnitDim, yNonUnitDim, zNonUnitDim;
    __shared__ Nd4jLong zLen, len;
    __shared__ bool is1Dcase;

    if (threadIdx.x == 0) {
        extern __shared__ unsigned char shmem[];
        coords = reinterpret_cast<int*>(shmem);

        xRank = shape::rank(xShapeInfo);
        yRank = shape::rank(yShapeInfo);
        zRank = shape::rank(zShapeInfo);
        xLastDim = shape::sizeAt(xShapeInfo, -1);

        biggerXYRank = xRank > yRank ? xRank : yRank;

        xNonUnitDim = yNonUnitDim = zNonUnitDim = 0;

        is1Dcase = (shape::isCommonVector(zShapeInfo, zNonUnitDim) || shape::isScalar(zShapeInfo)) && (shape::isCommonVector(yShapeInfo, yNonUnitDim) || shape::isScalar(yShapeInfo)) && (shape::isCommonVector(xShapeInfo, xNonUnitDim) || shape::isScalar(xShapeInfo));

        len  = is1Dcase ?  shape::length(xShapeInfo) : shape::length(xShapeInfo) / xLastDim;
        zLen = shape::length(zShapeInfo);
    }
    __syncthreads();

    Nd4jLong yOffset, zOffset, xOffset;
    int *yCoords, *zCoords;

    if(!is1Dcase) {
        yCoords = coords + threadIdx.x * (biggerXYRank + zRank);
        zCoords = yCoords + biggerXYRank;
    }

    for (Nd4jLong i = blockIdx.x * blockDim.x + threadIdx.x; i < zLen; i += gridDim.x * blockDim.x) {

        if(!is1Dcase)
            shape::index2coords(i, zShapeInfo, zCoords);

        for (Nd4jLong j = 0; j < len; ++j) {        // if !is1Dcase then we loop through first xRank-1 dimensions of x, that is we exclude last x dimension

            if(is1Dcase) {

                if(x[j * shape::stride(xShapeInfo)[xNonUnitDim]] != i)
                    continue;

                yOffset = j * shape::stride(yShapeInfo)[yNonUnitDim];
                zOffset = i * shape::stride(zShapeInfo)[zNonUnitDim];
            }
            else {

                shape::index2coords(j, xRank-1, shape::shapeOf(const_cast<Nd4jLong*>(xShapeInfo)), yCoords);        // first xRank-1 coordinates in yCoords are the same for y and x

                // first iteration
                yCoords[xRank - 1] = 0;
                xOffset = shape::getOffset(xShapeInfo, yCoords);
                if(zCoords[0] != x[xOffset])
                    continue;

                // rest iterations
                bool matched = true;
                for (uint k = 1; k < xLastDim; ++k) {
                    yCoords[xRank - 1] = k;
                    xOffset += shape::stride(xShapeInfo)[xRank-1];
                    if(zCoords[k] != x[xOffset]) {
                        matched = false;
                        break;
                    }
                }

                if(!matched)
                    continue;

                for (uint k = xLastDim; k < zRank; ++k)
                    yCoords[yRank - zRank + k] = zCoords[k];

                yOffset = shape::getOffset(yShapeInfo, yCoords);
                zOffset = shape::getOffset(zShapeInfo, zCoords);
            }

            switch (opCode) {
                case pairwise::Add:
                    z[zOffset] += y[yOffset];
                    break;
                case pairwise::Subtract:
                    z[zOffset] -= y[yOffset];
                    break;
                case pairwise::Multiply:
                    z[zOffset] *= y[yOffset];
                    break;
                case pairwise::Divide:
                    z[zOffset] /= y[yOffset];
                    break;
                case pairwise::ReverseSubtract:
                    z[zOffset] = y[yOffset] - z[zOffset];
                    break;
                case pairwise::ReverseDivide:
                    z[zOffset] = y[yOffset] / z[zOffset];
                    break;
                case pairwise::CopyPws:
                    z[zOffset] = y[yOffset];
                    break;
                case pairwise::MaxPairwise:
                    if(z[zOffset] < y[yOffset]) z[zOffset] = y[yOffset];
                    break;
                case pairwise::MinPairwise:
                    if(z[zOffset] > y[yOffset]) z[zOffset] = y[yOffset];
                    break;
                default:
                    continue;
            }
        }
    }
}

///////////////////////////////////////////////////////////////////
// x - indices, y - updates, z - output
template<typename X, typename Y>
__global__ static void scatterNDCuda(const int opCode,
                                     const void *vx, const Nd4jLong *xShapeInfo,
                                     const void *vy, const Nd4jLong *yShapeInfo,
                                           void *vz, const Nd4jLong *zShapeInfo) {

    const auto x = reinterpret_cast<const X*>(vx);
    const auto y = reinterpret_cast<const Y*>(vy);
          auto z = reinterpret_cast<Y*>(vz);

    __shared__ int xRank, yRank, zRank, biggerXYRank, xLastDim, *coords, xNonUnitDim, yNonUnitDim, zNonUnitDim;
    __shared__ Nd4jLong yLen;
    __shared__ bool is1Dcase;

    if (threadIdx.x == 0) {
        extern __shared__ unsigned char shmem[];
        coords = reinterpret_cast<int*>(shmem);

        yLen  = shape::length(yShapeInfo);
        xRank = shape::rank(xShapeInfo);
        yRank = shape::rank(yShapeInfo);
        zRank = shape::rank(zShapeInfo);
        xLastDim = shape::sizeAt(xShapeInfo, -1);

        biggerXYRank = xRank > yRank ? xRank : yRank;

        xNonUnitDim = yNonUnitDim = zNonUnitDim = 0;

        is1Dcase = (shape::isCommonVector(zShapeInfo, zNonUnitDim) || shape::isScalar(zShapeInfo)) && (shape::isCommonVector(yShapeInfo, yNonUnitDim) || shape::isScalar(yShapeInfo)) && (shape::isCommonVector(xShapeInfo, xNonUnitDim) || shape::isScalar(xShapeInfo));
    }
    __syncthreads();

    Nd4jLong yOffset, zOffset;
    int *yCoords, *zCoords;

    if(!is1Dcase) {
        yCoords = coords + threadIdx.x * (biggerXYRank + zRank);
        zCoords = yCoords + biggerXYRank;
    }

    for (Nd4jLong i = blockIdx.x * blockDim.x + threadIdx.x; i < yLen; i += gridDim.x * blockDim.x) {

        if(is1Dcase) {

            yOffset = i * shape::stride(yShapeInfo)[zNonUnitDim];
            zOffset = x[i * shape::stride(xShapeInfo)[xNonUnitDim]] * shape::stride(zShapeInfo)[zNonUnitDim];
        }
        else {

            shape::index2coords(i, yShapeInfo, yCoords);

            yOffset = shape::getOffset(yShapeInfo, yCoords);

            if(yRank >= xRank)
                zCoords[xLastDim] = yCoords[xRank - 1];                // saving y coordinate, since it might be changed in next instructions

            for (uint j = 0; j < xLastDim; ++j) {                      // first xRank-1 coordinates in yCoords are the same for y and x
                yCoords[xRank - 1] = j;
                zCoords[j] = x[shape::getOffset(xShapeInfo, yCoords)];
            }

            for (uint j = xLastDim + 1; j < zRank; ++j)
                zCoords[j] = yCoords[yRank - zRank + j];

            zOffset = shape::getOffset(zShapeInfo, zCoords);
        }

        switch (opCode) {
            case pairwise::Add:
                z[zOffset] += y[yOffset];
                break;
            case pairwise::Subtract:
                z[zOffset] -= y[yOffset];
                break;
            case pairwise::Multiply:
                z[zOffset] *= y[yOffset];
                break;
            case pairwise::Divide:
                z[zOffset] /= y[yOffset];
                break;
            case pairwise::ReverseSubtract:
                z[zOffset] = y[yOffset] - z[zOffset];
                break;
            case pairwise::ReverseDivide:
                z[zOffset] = y[yOffset] / z[zOffset];
                break;
            case pairwise::CopyPws:
                z[zOffset] = y[yOffset];
                break;
            case pairwise::MaxPairwise:
                if(z[zOffset] < y[yOffset]) z[zOffset] = y[yOffset];
                break;
            case pairwise::MinPairwise:
                if(z[zOffset] > y[yOffset]) z[zOffset] = y[yOffset];
                break;
            default:
                continue;
        }
    }
}

///////////////////////////////////////////////////////////////////
template<typename X, typename Y>
static void scatterNDCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream,
                                  const int opCode,
                                  const void *vx, const Nd4jLong *xShapeInfo,
                                  const void *vy, const Nd4jLong *yShapeInfo,
                                        void *vz, const Nd4jLong *zShapeInfo,
                                  const bool lock) {

    if(lock)
        scatterNDLockCuda<X,Y><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(opCode, vx, xShapeInfo, vy, yShapeInfo, vz, zShapeInfo);
    else
        scatterNDCuda<X,Y><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(opCode, vx, xShapeInfo, vy, yShapeInfo, vz, zShapeInfo);
}

///////////////////////////////////////////////////////////////////
void scatterND(sd::LaunchContext  *context, pairwise::Ops op, const NDArray& indices, const NDArray& updates, NDArray& output, const bool lock) {

    const int xRank = indices.rankOf();
    const int yRank = updates.rankOf();
    const int zRank = output.rankOf();

    const int threadsPerBlock = MAX_NUM_THREADS / 4;
    const int blocksPerGrid = ((lock ? output.lengthOf() : updates.lengthOf()) + threadsPerBlock - 1) / threadsPerBlock;
    const int sharedMem = threadsPerBlock * sizeof(int) * ((yRank > xRank ? yRank : xRank) + zRank) + 256;

    const auto xType = indices.dataType();
    const auto yType = updates.dataType();

    PointersManager manager(context, "scatterND");

    NDArray::prepareSpecialUse({&output}, {&updates, &indices});
    BUILD_DOUBLE_SELECTOR(xType, yType, scatterNDCudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, context->getCudaStream(), op, indices.specialBuffer(), indices.specialShapeInfo(), updates.specialBuffer(), updates.specialShapeInfo(), output.specialBuffer(), output.specialShapeInfo(), lock), INDEXING_TYPES, GENERIC_NUMERIC_TYPES);
    NDArray::registerSpecialUse({&output}, {&updates, &indices});

    manager.synchronize();
}

///////////////////////////////////////////////////////////////////
template<typename X, typename Z>
__global__ void scatterForLossCuda(const void *vx, const Nd4jLong *xShapeInfo,
                                         void *vy, const Nd4jLong *yShapeInfo,
                                         void *vz, const Nd4jLong *zShapeInfo) {

    const auto x = reinterpret_cast<const X*>(vx);
          auto y = reinterpret_cast<Z*>(vy);
          auto z = reinterpret_cast<Z*>(vz);

    __shared__ Nd4jLong xLen;
    __shared__ int xRank, *sharedMem;   // xRank = zRank, yRank = xRank + 1

    if (threadIdx.x == 0) {
        extern __shared__ unsigned char shmem[];
        sharedMem = reinterpret_cast<int*>(shmem);

        xLen  = shape::length(xShapeInfo);
        xRank = shape::rank(xShapeInfo);
    }
    __syncthreads();

    const auto xInd = threadIdx.x + blockIdx.x * blockDim.x;

    if(xInd >= xLen)
        return;

    auto coords = sharedMem + threadIdx.x * (xRank + 1);

    shape::index2coords(xInd, xShapeInfo, coords);

    // y last coordinate
    coords[xRank] = x[shape::getOffset(xShapeInfo, coords)];

    const auto yOffset = shape::getOffset(yShapeInfo, coords);

    if(z == nullptr) { // gradient calculation
        y[yOffset] -= 1.f;
    }
    else {
        z[shape::getOffset(zShapeInfo, coords)] = y[yOffset];
    }
}

///////////////////////////////////////////////////////////////////
template<typename X, typename Z>
static void scatterForLossCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream, const void *vx, const Nd4jLong* xShapeInfo, void *vy, const Nd4jLong* yShapeInfo, void *vz, const Nd4jLong* zShapeInfo) {

    scatterForLossCuda<X, Z><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(vx, xShapeInfo, vy, yShapeInfo, vz, zShapeInfo);
}

///////////////////////////////////////////////////////////////////
void scatterForLoss(sd::LaunchContext* context, const NDArray& indices, NDArray& updates, NDArray& output, const bool calcGrad) {
    // shapes of indices and output must be the same
    // shape of indices should be the same as updates shape with last dimension excluded, for example if updates is {a,b,c} then indices should be {a,b}

    PointersManager manager(context, "scatterForLoss");

    const int threadsPerBlock = MAX_NUM_THREADS / 2;
    const int blocksPerGrid = (indices.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;
    const int sharedMem = updates.rankOf() * sizeof(int) * threadsPerBlock  + 128;

    if(calcGrad) {
        NDArray::prepareSpecialUse({&updates}, {&indices});
        BUILD_DOUBLE_SELECTOR(indices.dataType(), updates.dataType(), scatterForLossCudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, context->getCudaStream(), indices.specialBuffer(), indices.specialShapeInfo(), updates.specialBuffer(), updates.specialShapeInfo(), nullptr, nullptr), INDEXING_TYPES, FLOAT_TYPES);
        NDArray::registerSpecialUse({&updates}, {&indices});
    }
    else {
        NDArray::prepareSpecialUse({&output}, {&indices, &updates});
        BUILD_DOUBLE_SELECTOR(indices.dataType(), updates.dataType(), scatterForLossCudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, context->getCudaStream(), indices.specialBuffer(), indices.specialShapeInfo(), updates.specialBuffer(), updates.specialShapeInfo(), output.specialBuffer(), output.specialShapeInfo()), INDEXING_TYPES, FLOAT_TYPES);
        NDArray::registerSpecialUse({&output}, {&indices, &updates});
    }

    manager.synchronize();
}

}
}
}


/*

///////////////////////////////////////////////////////////////////
template<typename X, typename Y>
static void scatterLockCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream,
                                    const int opCode,
                                    const void* vx, const Nd4jLong *xShapeInfo,
                                    const void* vy, const Nd4jLong *yTadShapeInfo, const Nd4jLong *yOffsets,
                                          void* vz, const Nd4jLong *zTadShapeInfo, const Nd4jLong *zOffsets,
                                    const Nd4jLong xLen, const Nd4jLong yTadLen, const Nd4jLong zTadLen) {

    scatterLockCuda<X,Y><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(opCode, vx, xShapeInfo, vy, yTadShapeInfo, yOffsets, vz, zTadShapeInfo, zOffsets, xLen, yTadLen, zTadLen);
}


///////////////////////////////////////////////////////////////////
// x - indices, y - updates, z - input/output
template<typename X, typename Y>
__global__ static void scatterLockCuda(const int opCode,
                                       const void* vx, const Nd4jLong *xShapeInfo,
                                       const void* vy, const Nd4jLong *yTadShapeInfo, const Nd4jLong *yOffsets,
                                             void* vz, const Nd4jLong *zTadShapeInfo, const Nd4jLong *zOffsets,
                                       const Nd4jLong xLen, const Nd4jLong yTadLen, const Nd4jLong zTadLen) {



 const int xRank = indices.rankOf();

        std::vector<int> zTadDims = ShapeUtils::evalDimsToExclude(output.rankOf(), {0});

        int sizeOfUpdDims = xRank;
        if(output.rankOf() == updates.rankOf() && indices.isVector())
            sizeOfUpdDims = 1;

        std::vector<int> yTadDims(sizeOfUpdDims);
        std::iota(yTadDims.begin(), yTadDims.end(), 0);

        auto packY = sd::ConstantTadHelper::getInstance().tadForDimensions(updates.shapeInfo(), ShapeUtils::evalDimsToExclude(updates.rankOf(), yTadDims));
        auto packZ = sd::ConstantTadHelper::getInstance().tadForDimensions(output.shapeInfo(), zTadDims);

        const Nd4jLong zTadLen = shape::length(packZ.primaryShapeInfo());
        const Nd4jLong yTadLen = shape::length(packY.primaryShapeInfo());

        const auto threadsPerBlock = sd::math::nd4j_max<int>(32, sd::math::nd4j_min<int>(zTadLen, 1024));
        const auto blocksPerGrid = indices.lengthOf();

        const auto xType = indices.dataType();
        const auto yType = updates.dataType();

        BUILD_DOUBLE_SELECTOR(xType, yType, scatterLockCudaLauncher, (blocksPerGrid, threadsPerBlock, 1024, context->getCudaStream(), op, indices.specialBuffer(), indices.specialShapeInfo(), updates.specialBuffer(), packY.specialShapeInfo(), packY.specialOffsets(), output.specialBuffer(), packZ.specialShapeInfo(), packZ.specialOffsets(), indices.lengthOf(), yTadLen, zTadLen), INDEXING_TYPES, GENERIC_NUMERIC_TYPES);



    const auto x = reinterpret_cast<const X*>(vx);
    const auto y = reinterpret_cast<const Y*>(vy);
          auto z = reinterpret_cast<Y*>(vz);

    __shared__ bool vectorCase;
    if(threadIdx.x == 0)
        vectorCase = yTadLen == xLen && shape::rank(xShapeInfo) <= 1;
    __syncthreads();

    for (int e = 0; e < xLen; e++) {

        const Nd4jLong zIndex = x[shape::getIndexOffset(e, xShapeInfo)];
        const bool isOwner = zIndex < gridDim.x ? blockIdx.x == zIndex : blockIdx.x == zIndex % gridDim.x;

        if (!isOwner)
            continue;

        if(vectorCase) { // means z_rank = 1 and might be yTadLen != zTadLen in this case

            if(threadIdx.x != 0)
                continue;

            const auto yOffset = shape::getIndexOffset(e,      yTadShapeInfo);
            const auto zOffset = shape::getIndexOffset(zIndex, zTadShapeInfo);

            switch (opCode) {
                case pairwise::Add:
                    z[zOffset] += y[yOffset];
                    break;
                case pairwise::Subtract:
                    z[zOffset] -= y[yOffset];
                    break;
                case pairwise::Multiply:
                    z[zOffset] *= y[yOffset];
                    break;
                case pairwise::Divide:
                    z[zOffset] /= y[yOffset];
                    break;
                case pairwise::ReverseSubtract:
                    z[zOffset] = y[yOffset] - z[zOffset];
                    break;
                case pairwise::ReverseDivide:
                    z[zOffset] = y[yOffset] / z[zOffset];
                    break;
                case pairwise::CopyPws:
                    z[zOffset] = y[yOffset];
                    break;
                case pairwise::MaxPairwise:
                    if(z[zOffset] < y[yOffset]) z[zOffset] = y[yOffset];
                    break;
                case pairwise::MinPairwise:
                    if(z[zOffset] > y[yOffset]) z[zOffset] = y[yOffset];
                    break;
                default:
                    continue;
            }
        }
        else {      // yTadLen == zTadLen in this case

            const Y* yTad = y + yOffsets[e];
                  Y* zTad = z + zOffsets[zIndex];

            for (Nd4jLong i = threadIdx.x; i < zTadLen; i += blockDim.x) {

                const auto yOffset = shape::getIndexOffset(i, yTadShapeInfo);
                const auto zOffset = shape::getIndexOffset(i, zTadShapeInfo);

                switch (opCode) {
                    case pairwise::Add:
                        zTad[zOffset] += yTad[yOffset];
                        break;
                    case pairwise::Subtract:
                        zTad[zOffset] -= yTad[yOffset];
                        break;
                    case pairwise::Multiply:
                        zTad[zOffset] *= yTad[yOffset];
                        break;
                    case pairwise::Divide:
                        zTad[zOffset] /= yTad[yOffset];
                        break;
                    case pairwise::ReverseSubtract:
                        zTad[zOffset] = yTad[yOffset] - zTad[zOffset];
                        break;
                    case pairwise::ReverseDivide:
                        zTad[zOffset] = yTad[yOffset] / zTad[zOffset];
                        break;
                    case pairwise::CopyPws:
                        zTad[zOffset] = yTad[yOffset];
                        break;
                    case pairwise::MaxPairwise:
                        if(zTad[zOffset] < yTad[yOffset]) zTad[zOffset] = yTad[yOffset];
                        break;
                    case pairwise::MinPairwise:
                        if(zTad[zOffset] > yTad[yOffset]) zTad[zOffset] = yTad[yOffset];
                        break;
                    default:
                        continue;
                }
            }
        }
    }
}

            template<typename T, bool locking>
            __global__ static void scatterCuda(const int opCode, const int numOfSubArrs,
                                                     void* vx, const Nd4jLong *xShapeInfo, const Nd4jLong *xOffsets,
                                                     void* vy, const Nd4jLong *yShapeInfo, const Nd4jLong *yOffsets,
                                                     const int* indexes, unsigned int arrLenX, unsigned int arrLenY) {

                __shared__ T *x, *y;

                if (locking) {

                    for (int e = 0; e < numOfSubArrs; e++) {

                        const auto xIndex = indexes[e];
                        const bool isOwner = xIndex < gridDim.x ? blockIdx.x == xIndex : blockIdx.x == xIndex % gridDim.x;

                        if (!isOwner)
                            continue;

                        if (threadIdx.x == 0) {
                            x = reinterpret_cast<T *>(vx) + xOffsets[xIndex];
                            y = reinterpret_cast<T *>(vy) + yOffsets[e];
                        }
                        __syncthreads();

                        for (Nd4jLong i = threadIdx.x; i < arrLenX; i += blockDim.x) {

                            const auto xOffset = shape::getIndexOffset(i, xShapeInfo);
                            const auto yOffset = shape::getIndexOffset(i, yShapeInfo);

                            switch (opCode) {
                                case pairwise::Add:
                                    x[xOffset] += y[yOffset];
                                    break;
                                case pairwise::Subtract:
                                    x[xOffset] -= y[yOffset];
                                    break;
                                case pairwise::Multiply:
                                    x[xOffset] *= y[yOffset];
                                    break;
                                case pairwise::Divide:
                                    x[xOffset] /= y[yOffset];
                                    break;
                                case pairwise::ReverseSubtract:
                                    x[xOffset] = y[yOffset] - x[xOffset];
                                    break;
                                case pairwise::ReverseDivide:
                                    x[xOffset] = y[yOffset] / x[xOffset];
                                    break;
                                case pairwise::CopyPws:
                                    x[xOffset] = y[yOffset];
                                    break;
                                default:
                                    continue;
                            }
                        }
                        __syncthreads();
                    }
                } else {
                    for (int e = blockIdx.x; e < numOfSubArrs; e+= gridDim.x) {

                        if (threadIdx.x == 0) {
                            const auto xIndex = indexes[e];
                            x = reinterpret_cast<T *>(vx) + xOffsets[xIndex];
                            y = reinterpret_cast<T *>(vy) + yOffsets[e];
                        }
                        __syncthreads();

                        for (Nd4jLong i = threadIdx.x; i < arrLenX; i += blockDim.x) {
                            const auto xOffset = shape::getIndexOffset(i, xShapeInfo);
                            const auto yOffset = shape::getIndexOffset(i, yShapeInfo);

                            switch (opCode) {
                                case pairwise::Add:
                                    x[xOffset] += y[yOffset];
                                    break;
                                case pairwise::Subtract:
                                    x[xOffset] -= y[yOffset];
                                    break;
                                case pairwise::Multiply:
                                    x[xOffset] *= y[yOffset];
                                    break;
                                case pairwise::Divide:
                                    x[xOffset] /= y[yOffset];
                                    break;
                                case pairwise::ReverseSubtract:
                                    x[xOffset] = y[yOffset] - x[xOffset];
                                    break;
                                case pairwise::ReverseDivide:
                                    x[xOffset] = y[yOffset] / x[xOffset];
                                    break;
                                case pairwise::CopyPws:
                                    x[xOffset] = y[yOffset];
                                    break;
                                default:
                                    continue;
                            }
                        }
                        __syncthreads();
                    }
                }
            }


            template <typename T>
            void scatter_(sd::LaunchContext  *context, pairwise::Ops op, const NDArray& indices, const NDArray& updates, NDArray& output, const bool lock) {
                std::vector<int> dims = {0};
                auto inverted = ShapeUtils::evalDimsToExclude(output.rankOf(), dims);

                auto packX = sd::ConstantTadHelper::getInstance().tadForDimensions(output.shapeInfo(), inverted);
                auto packY = sd::ConstantTadHelper::getInstance().tadForDimensions(updates.shapeInfo(), inverted);

                auto psX = packX.specialShapeInfo();
                auto psY = packY.special();

                PointersManager manager(context, "scatter");

                auto poX = packX.specialOffsets();
                auto poY = packY.special();

                NDArray::prepareSpecialUse({&output}, {&updates, &indices});

                unsigned int tadLengthX = shape::length(packX.primaryShapeInfo());
                unsigned int tadLengthY = shape::length(packY.primary());
                if (tadLengthX != tadLengthY)
                    throw std::runtime_error("scatter: Lengths of TADs must be equal");

                auto blockSize = sd::math::nd4j_max<int>(32, sd::math::nd4j_min<int>(tadLengthX, 1024));

                if (lock)
                    scatterCuda<T, true><<<512, blockSize, 1024, *context->getCudaStream()>>>(op, indices.lengthOf(), output.specialBuffer(), psX, poX, updates.specialBuffer(), psY, poY, reinterpret_cast<int *>(indices.specialBuffer()), tadLengthX, tadLengthY);
                else
                    scatterCuda<T, false><<<512, blockSize, 1024, *context->getCudaStream()>>>(op, indices.lengthOf(), output.specialBuffer(), psX, poX, updates.specialBuffer(), psY, poY, reinterpret_cast<int *>(indices.specialBuffer()), tadLengthX, tadLengthY);

                 NDArray::registerSpecialUse({&output}, {&updates, &indices});
                manager.synchronize();
            }



///////////////////////////////////////////////////////////////////
// x - indices, y - updates, z - output
template<typename X, typename Y>
__global__ static void scatterNDLockCuda(const int opCode,
                                         const void* vx, const Nd4jLong *xTadShapeInfo, const Nd4jLong *xOffsets,
                                         const void* vy, const Nd4jLong *yTadShapeInfo, const Nd4jLong *yOffsets,
                                               void* vz, const Nd4jLong *zTadShapeInfo, const Nd4jLong *zOffsets,
                                         const Nd4jLong *zShapeInfo,
                                         const Nd4jLong numOfXTads, const Nd4jLong numOfZTads, const Nd4jLong yTadLen) {



---------------------------------------------------------------------------
const int xLastDim = indices.sizeAt(-1);

        // y_tad and z_tad have the same shape
        std::vector<int> yTadDims(zRank - xLastDim), zTadDims(zRank - xLastDim);
        for (int j = 0, i = zTadDims.size() - 1; i >=0 ; --i, ++j) {
            yTadDims[i] = yRank - 1 - j;
            zTadDims[i] = zRank - 1 - j;
        }

        auto packX = sd::ConstantTadHelper::getInstance().tadForDimensions(indices.shapeInfo(), {xRank - 1});
        auto packY = sd::ConstantTadHelper::getInstance().tadForDimensions(updates.shapeInfo(), yTadDims);
        auto packZ = sd::ConstantTadHelper::getInstance().tadForDimensions(output.shapeInfo(), zTadDims);

        const int threadsPerBlock = MAX_NUM_THREADS / 4;
        const int blocksPerGrid = packZ.numberOfTads();
        const int sharedMem = 8 * threadsPerBlock * xLastDim + 128;
---------------------------------------------------------------------------

    // zTadLen == yTadLen if numOfZTads > 1, in opposite case z and y are vectors
    // numOfXTads == numOfYTads if numOfZTads > 1, in opposite case z and y are vectors

    const auto x = reinterpret_cast<const X*>(vx);
    const auto y = reinterpret_cast<const Y*>(vy);
          auto z = reinterpret_cast<Y*>(vz);

    __shared__ Nd4jLong *zTadCoords;
    __shared__ int xLastDim;

    if (threadIdx.x == 0) {
        extern __shared__ unsigned char shmem[];
        zTadCoords = reinterpret_cast<Nd4jLong*>(shmem);
        xLastDim = xTadShapeInfo[1];   // xTad has rank = 1 always
    }
    __syncthreads();

    Nd4jLong* zTadCoordsPerThread = zTadCoords + threadIdx.x * xLastDim;

    for (Nd4jLong i = 0; i < numOfXTads; ++i) {

        const X* xTad = x + xOffsets[i];

        for (uint k = 0; k < xLastDim; ++k)
            zTadCoordsPerThread[k] = xTad[shape::getIndexOffset(k, xTadShapeInfo)];

        const auto zTadIndex = shape::coords2index(xLastDim, zShapeInfo + 1, zTadCoordsPerThread);

        const bool isOwner = zTadIndex < gridDim.x ? blockIdx.x == zTadIndex : blockIdx.x == zTadIndex % gridDim.x;

        if(!isOwner)
            continue;

        if(numOfZTads == 1) {     // yTadLen == numOfXTads in this case

            if(threadIdx.x != 0)
                continue;

            const auto yOffset = shape::getIndexOffset(i,         yTadShapeInfo);
            const auto zOffset = shape::getIndexOffset(zTadIndex, zTadShapeInfo);

            switch (opCode) {
                case pairwise::Add:
                    z[zOffset] += y[yOffset];
                    break;
                case pairwise::Subtract:
                    z[zOffset] -= y[yOffset];
                    break;
                case pairwise::Multiply:
                    z[zOffset] *= y[yOffset];
                    break;
                case pairwise::Divide:
                    z[zOffset] /= y[yOffset];
                    break;
                case pairwise::ReverseSubtract:
                    z[zOffset] = y[yOffset] - z[zOffset];
                    break;
                case pairwise::ReverseDivide:
                    z[zOffset] = y[yOffset] / z[zOffset];
                    break;
                case pairwise::CopyPws:
                    z[zOffset] = y[yOffset];
                    break;
                case pairwise::MaxPairwise:
                    if(z[zOffset] < y[yOffset]) z[zOffset] = y[yOffset];
                    break;
                case pairwise::MinPairwise:
                    if(z[zOffset] > y[yOffset]) z[zOffset] = y[yOffset];
                    break;
                default:
                    continue;
            }
        }
        else {
            const auto yTad = y + yOffsets[i];
            const auto zTad = z + zOffsets[zTadIndex];

            for (Nd4jLong j = threadIdx.x; j < yTadLen; j += blockDim.x) {

                const auto yOffset = shape::getIndexOffset(j, yTadShapeInfo);
                const auto zOffset = shape::getIndexOffset(j, zTadShapeInfo);

                switch (opCode) {
                    case pairwise::Add:
                        zTad[zOffset] += yTad[yOffset];
                        break;
                    case pairwise::Subtract:
                        zTad[zOffset] -= yTad[yOffset];
                        break;
                    case pairwise::Multiply:
                        zTad[zOffset] *= yTad[yOffset];
                        break;
                    case pairwise::Divide:
                        zTad[zOffset] /= yTad[yOffset];
                        break;
                    case pairwise::ReverseSubtract:
                        zTad[zOffset] = yTad[yOffset] - zTad[zOffset];
                        break;
                    case pairwise::ReverseDivide:
                        zTad[zOffset] = yTad[yOffset] / zTad[zOffset];
                        break;
                    case pairwise::CopyPws:
                        zTad[zOffset] = yTad[yOffset];
                        break;
                    case pairwise::MaxPairwise:
                        if(zTad[zOffset] < yTad[yOffset]) zTad[zOffset] = yTad[yOffset];
                        break;
                    case pairwise::MinPairwise:
                        if(zTad[zOffset] > yTad[yOffset]) zTad[zOffset] = yTad[yOffset];
                        break;
                    default:
                        continue;
                }
            }
        }
    }
}

*/
        // PointersManager manager(&context, "NativeOps::concat");
        // PointersManager::printDevContentOnDev<int>(vx, 2);
        // PointersManager::printDevContentOnDev<Nd4jLong>(xShapeInfo, 8);
        // PointersManager::printDevContentOnDev<float>(vy, 8);
        // PointersManager::printDevContentOnDev<Nd4jLong>(yShapeInfo, 8);
        // PointersManager::printDevContentOnDev<Nd4jLong>(zShapeInfo, 8);

        // manager.printDevContentOnHost<int>(indices.specialBuffer(), indices.lengthOf());
        // manager.printDevContentOnHost<Nd4jLong>(indices.special(), shape::shapeInfoLength(indices.rankOf()));
        // manager.printDevContentOnHost<float>(updates.specialBuffer(), updates.lengthOf());
        // manager.printDevContentOnHost<Nd4jLong>(updates.special(), shape::shapeInfoLength(updates.rankOf()));
        // manager.printDevContentOnHost<Nd4jLong>(output.special(), shape::shapeInfoLength(output.rankOf()));
        // printf("!!!!!!!\n");
        // manager.printDevContentOnHost<Nd4jLong>(packX.special(), 2*shape::rank(packX.primary()) + 4);
        // manager.printDevContentOnHost<Nd4jLong>(packX.special(), packX.numberOfTads());
        // manager.printDevContentOnHost<Nd4jLong>(packY.special(), 2*shape::rank(packY.primary()) + 4);
        // manager.printDevContentOnHost<Nd4jLong>(packY.special(), packY.numberOfTads());
        // manager.printDevContentOnHost<Nd4jLong>(packZ.special(), 2*shape::rank(packZ.primary()) + 4);
        // manager.printDevContentOnHost<Nd4jLong>(packZ.special(), packZ.numberOfTads());
        // printf("dddddddd\n");
        // shape::printShapeInfoLinear(packY.primary());