#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma (iuriish@yahoo.com)
//


#include<ops/declarable/helpers/addBias.h>
#include <helpers/PointersManager.h>

namespace sd    {
namespace ops     {
namespace helpers {

//////////////////////////////////////////////////////////////////////
template<typename X, typename Y>
__global__ static void addBiasCuda( const void* vx, const Nd4jLong* xShapeInfo,
                                    const void* vy, const Nd4jLong* yShapeInfo,
                                          void* vz, const Nd4jLong* zShapeInfo,
                                    const bool isNCHW) {

    // bias [oC]

    // if(input_rank == 4)
        // input and output have same shapes: [bS, oH, oW, oC] (NHWC) or [bS, oC, oH, oW] (NCHW)
    // if(input_rank == 5)
        // input and output have same shapes: [bS, oD, oH, oW, oC] (NHWC) or [bS, oD, oC, oH, oW] (NCHW)

    const X* x = reinterpret_cast<const X*>(vx);
    const Y* y = reinterpret_cast<const Y*>(vy);
          X* z = reinterpret_cast<X*>(vz);

    __shared__ int rank, channelPosition, posOfNonUnityDim;
    __shared__ Nd4jLong len, *sharedMem;
    __shared__ bool xzSameOffsets, xzAreSame;

    if (threadIdx.x == 0) {

        extern __shared__ unsigned char shmem[];
        sharedMem = reinterpret_cast<Nd4jLong*>(shmem);

        rank = shape::rank(xShapeInfo);     // xRank == zRank
        xzSameOffsets = shape::haveSameShapeAndStrides(xShapeInfo, zShapeInfo);
        len = shape::length(xShapeInfo);
        channelPosition = isNCHW ? 1 : rank - 1;        // second or last
        xzAreSame = x == z;

        shape::isCommonVector(yShapeInfo, posOfNonUnityDim);
    }
    __syncthreads();

    auto coords = sharedMem + threadIdx.x * rank;

    for (Nd4jLong i = blockIdx.x * blockDim.x + threadIdx.x; i < len; i += blockDim.x * gridDim.x) {

        shape::index2coords(i, xShapeInfo, coords);

        const auto xOffsets = shape::getOffset(xShapeInfo, coords);
        const auto zOffsets = xzSameOffsets ? xOffsets : shape::getOffset(zShapeInfo, coords);
        const auto yOffsets = coords[channelPosition] * shape::stride(yShapeInfo)[posOfNonUnityDim];

        if(xzAreSame)
            z[zOffsets] += static_cast<X>(y[yOffsets]);
        else
            z[zOffsets] = x[xOffsets] + static_cast<X>(y[yOffsets]);
    }
}

//////////////////////////////////////////////////////////////////////////
template<typename X, typename Y>
static void addBiasCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream,
                                         const void* vx, const Nd4jLong* xShapeInfo,
                                         const void* vy, const Nd4jLong* yShapeInfo,
                                               void* vz, const Nd4jLong* zShapeInfo,
                                         const bool isNCHW) {

    addBiasCuda<X,Y><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(vx, xShapeInfo, vy, yShapeInfo, vz, zShapeInfo, isNCHW);
}

template<typename X, typename Y>
__global__ static void addBias2DCuda( const void* vx,
                                        const void* vy,
                                        void* vz,
                                        uint32_t blocks, uint32_t length) {

    auto y = reinterpret_cast<const Y*>(vy);

    for (uint32_t b = blockIdx.x; b < blocks; b += gridDim.x) {
        auto x = reinterpret_cast<const X*>(vx) + length * b;
        auto z = reinterpret_cast<X*>(vz) + length * b;

        for (uint32_t e = threadIdx.x; e < length; e += blockDim.x) {
            z[e] = x[e] + y[e];
        }
    }
}

template<typename X, typename Y>
static void addBias2DCudaLauncher(const hipStream_t *stream, const void* vx,
                                  const void* vy,
                                  void* vz,
                                  uint32_t blocks, uint32_t length) {

    addBias2DCuda<X,Y><<<256, 1024, 128, *stream>>>(vx, vy, vz, blocks, length);
}

//////////////////////////////////////////////////////////////////////////
void addBias(sd::graph::Context& block, const NDArray& input, const NDArray& bias, NDArray& output, const bool isNCHW) {

    PointersManager manager(block.launchContext(), "addBias");
    NDArray::prepareSpecialUse({&output}, {&input, &bias});

    if (input.rankOf() == 2 && bias.rankOf() == 1 && input.ordering() == 'c' && output.ordering() == 'c' && input.ews() == 1 && bias.ews() == 1 && input.sizeAt(1) == bias.sizeAt(0)) {
        BUILD_DOUBLE_SELECTOR(input.dataType(), bias.dataType(), addBias2DCudaLauncher,
                              (block.launchContext()->getCudaStream(), input.specialBuffer(), bias.specialBuffer(), output.specialBuffer(), input.sizeAt(0), bias.sizeAt(0)),
                              FLOAT_TYPES, FLOAT_TYPES);
    } else {
        // default case
        const int threadsPerBlock = MAX_NUM_THREADS / 4;
        const int blocksPerGrid = (input.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;
        const int sharedMem = input.rankOf() * sizeof(Nd4jLong) * threadsPerBlock + 128;


        BUILD_DOUBLE_SELECTOR(input.dataType(), bias.dataType(), addBiasCudaLauncher,
                              (blocksPerGrid, threadsPerBlock, sharedMem, block.launchContext()->getCudaStream(), input.specialBuffer(), input.specialShapeInfo(), bias.specialBuffer(), bias.specialShapeInfo(), output.specialBuffer(), output.specialShapeInfo(), isNCHW),
                              FLOAT_TYPES, FLOAT_TYPES);
    }
    NDArray::registerSpecialUse({&output}, {&input, &bias});
    manager.synchronize();
}

}
}
}