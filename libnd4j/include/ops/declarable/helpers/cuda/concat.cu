#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma (iuriish@yahoo.com), created on 20.04.2018
//


#include<ops/declarable/helpers/transforms.h>
#include <array/ResultSet.h>
#include <helpers/ShapeUtils.h>
#include <numeric>
#include <array/NDArrayFactory.h>
#include <helpers/TAD.h>
#include <exceptions/cuda_exception.h>
#include <helpers/PointersManager.h>
#include <helpers/ConstantTadHelper.h>

namespace sd    {
namespace ops     {
namespace helpers {


///////////////////////////////////////////////////////////////////
template<typename T>
__global__ static void concatCuda(void* pVx,  void* pxShapeInfo, void* vz, const Nd4jLong* zShapeInfo, const int axis) {

    T* z = reinterpret_cast<T*>(vz);
    __shared__ Nd4jLong zLen, totalThreads;
    __shared__ int rank;

    if (threadIdx.x == 0) {
        zLen = shape::length(zShapeInfo);
        rank = shape::rank(zShapeInfo);
        totalThreads = gridDim.x * blockDim.x;
    }
    __syncthreads();

    const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

    int coords[MAX_RANK];

    for (Nd4jLong i = tid; i < zLen; i += totalThreads) {
        shape::index2coords(i, zShapeInfo, coords);

        const auto zOffset = shape::getOffset(zShapeInfo, coords);

        int inArrIdx = 0;
        Nd4jLong *xShapeInfo = reinterpret_cast<Nd4jLong **>(pxShapeInfo)[inArrIdx];

        while (coords[axis] >= xShapeInfo[axis + 1]) {
            coords[axis] -= xShapeInfo[axis + 1];
            xShapeInfo = reinterpret_cast<Nd4jLong **>(pxShapeInfo)[++inArrIdx];
        }

        const auto *x = reinterpret_cast<T *>(reinterpret_cast<void **>(pVx)[inArrIdx]);
        const auto xOffset = shape::getOffset(xShapeInfo, coords);

        z[zOffset] = x[xOffset];
    }
}

///////////////////////////////////////////////////////////////////
template<typename T>
__host__ static void concatCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream,
                                        void* pVx, void* pxShapeInfo, void* vz, const Nd4jLong* zShapeInfo, const int axis) {

    concatCuda<T><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(pVx, pxShapeInfo, vz, zShapeInfo, axis);
}

//////////////////////////////////////////////////////////////////////////
void concat(sd::LaunchContext * context, const std::vector<const NDArray*>& inArrs, NDArray& output, const int axis) {

    const int numOfInArrs = inArrs.size();
    const auto sizeofT    = output.sizeOfT();

    NDArray::prepareSpecialUse({&output}, inArrs);

    bool luckCase1 = ((axis == 0 && output.ordering() == 'c') || (axis == output.rankOf() - 1 && output.ordering() == 'f')) && output.ews() == 1;

    if(luckCase1) {
        for (uint i = 0; i < numOfInArrs; ++i) {
            luckCase1 &= inArrs[i]->ordering() == output.ordering() && inArrs[i]->ews() == 1;
            if(!luckCase1)
                break;
        }
    }

    if(luckCase1) {     // for example {1,10} + {2,10} + {3,10} = {6, 10} order c; or {10,1} + {10,2} + {10,3} = {10, 6} order f

        void* z = static_cast<int8_t*>(output.specialBuffer());

        for (uint i = 0; i < numOfInArrs; ++i) {
            const auto memAmountToCopy = inArrs[i]->lengthOf() * sizeofT;
            hipMemcpyAsync(z, reinterpret_cast<const int8_t*>(inArrs[i]->specialBuffer()), memAmountToCopy, hipMemcpyDeviceToDevice, *context->getCudaStream());
            z = static_cast<int8_t*>(z) + memAmountToCopy;
        }

        if(hipStreamSynchronize(*context->getCudaStream()) != 0)
            throw std::runtime_error("concat cuda: luckCase1 failed!");

        for(int i = 0; i < numOfInArrs; ++i)
            inArrs[i]->tickReadDevice();
        output.tickWriteDevice();

        return;
    }

    // const bool isZcontin = output.strideAt(axis) == 1;
    // bool areInputsContin = true;
    // bool allSameOrder    = true;
    // std::vector<Nd4jLong> strideOfContigStride(numOfInArrs);

    // if(isZcontin) {

    //     for (uint i = 0; i < inArrs.size(); ++i) {

    //         areInputsContin &= inArrs[i]->strideAt(axis) == 1;
    //         allSameOrder    &= output.ordering() == inArrs[i]->ordering();
    //         if(!areInputsContin || !allSameOrder)
    //             break;

    //         strideOfContigStride[i] = shape::strideOverContigAxis(axis, inArrs[i]->shapeInfo());
    //     }
    // }

    // const bool luckCase2 = isZcontin && areInputsContin && allSameOrder;

    // if(luckCase2) {     // for example {2,1,3} + {2,5,3} + {2,10,3} = {2,16,3}, here axis 1 shoud have stride = 1 for all inputs arrays and output array

    //     const auto zStep = shape::strideOverContigAxis(axis, output.shapeInfo());

    //     for (uint i = 0; i < output.lengthOf() / output.sizeAt(axis); ++i) {

    //         const auto iShift = i * sizeofT;
    //         void* z = static_cast<int8_t*>(output.specialBuffer()) + zStep * iShift;

    //         for (uint j = 0; j < numOfInArrs; ++j) {
    //             const auto xDim = inArrs[j]->sizeAt(axis);
    //             void* x = static_cast<int8_t*>(inArrs[j]->specialBuffer()) + strideOfContigStride[j] * iShift;
    //             const auto memSizeToCopy = xDim * sizeofT;
    //             hipMemcpyAsync(z, x, memSizeToCopy, hipMemcpyDeviceToDevice, *context->getCudaStream());
    //             z = static_cast<int8_t*>(z) + memSizeToCopy;
    //         }
    //     }

    //     if(hipStreamSynchronize(*context->getCudaStream()) != 0)
    //         throw std::runtime_error("concat cuda: luckCase2 failed!");
    // }
    // else {      // general (slower) case

        const int threadsPerBlock = MAX_NUM_THREADS / 2;
        const int blocksPerGrid = (output.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;
        const int sharedMem = 256;

        // prepare arrays of pointers on buffers and shapes
        std::vector<const void*> hInBuffers(numOfInArrs);
        std::vector<const Nd4jLong*> hInShapeInfo(numOfInArrs);

        for(int i = 0; i < numOfInArrs; ++i) {
            hInBuffers[i]   = inArrs[i]->specialBuffer();
            hInShapeInfo[i] = inArrs[i]->specialShapeInfo();
        }

        PointersManager manager(context, "helpers::concat");

        void* dInBuffers   = manager.replicatePointer(hInBuffers.data(),    hInBuffers.size() * sizeof(void*));
        void* dInShapeInfo = manager.replicatePointer(hInShapeInfo.data(),  hInShapeInfo.size() * sizeof(Nd4jLong*));

        BUILD_SINGLE_SELECTOR(inArrs[0]->dataType(), concatCudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, context->getCudaStream(), dInBuffers, dInShapeInfo, output.specialBuffer(), output.specialShapeInfo(), axis), LIBND4J_TYPES);

        manager.synchronize();
    // }

    NDArray::registerSpecialUse({&output}, inArrs);
}

}
}
}