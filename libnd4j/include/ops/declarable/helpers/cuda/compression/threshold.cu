#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
//

#include <ops/declarable/helpers/threshold.h>
#include <loops/type_conversions.h>
#include <helpers/PointersManager.h>
#include <vector>

namespace sd {
    namespace ops {
        namespace helpers {
            void prescanArrayRecursive(int** g_scanBlockSums, int *dZ, int *dX, int numElements, int level) {
                auto stream = LaunchContext::defaultContext()->getCudaStream();


                int blockSize = 512; // max size of the thread blocks
                int numBlocks = sd::math::nd4j_max<int>(1, static_cast<int>(ceil(static_cast<float>(numElements) / (2.f * blockSize))));
                int numThreads;

                if (numBlocks > 1)
                    numThreads = blockSize;
                else if (sd::isPowerOfTwo(numElements))
                    numThreads = numElements / 2;
                else
                    numThreads = sd::floorPow2(numElements);

                numThreads = sd::math::nd4j_max<int>(1, numThreads);

                int numEltsPerBlock = numThreads * 2;



                // if this is a non-power-of-2 array, the last block will be non-full
                // compute the smallest power of 2 able to compute its scan.
                int numEltsLastBlock =
                        numElements - (numBlocks-1) * numEltsPerBlock;
                int numThreadsLastBlock = sd::math::nd4j_max<int>(1, numEltsLastBlock / 2);
                int np2LastBlock = 0;
                int sharedMemLastBlock = 0;

                if (numEltsLastBlock != numEltsPerBlock) {
                    np2LastBlock = 1;

                    if(!isPowerOfTwo(numEltsLastBlock))
                        numThreadsLastBlock = floorPow2(numEltsLastBlock);

                    unsigned int extraSpace = (2 * numThreadsLastBlock) / NUM_BANKS;
                    sharedMemLastBlock = sizeof(int) * (2 * numThreadsLastBlock + extraSpace);
                }

                // padding space is used to avoid shared memory bank conflicts
                int extraSpace = numEltsPerBlock / NUM_BANKS;
                int sharedMemSize = sizeof(int) * (numEltsPerBlock + extraSpace);

                // setup execution parameters
                // if NP2, we process the last block separately
                dim3 grid(sd::math::nd4j_max<int>(1, numBlocks - np2LastBlock), 1, 1);
                dim3 threads(numThreads, 1, 1);
                dim3 gridOnes(1, 1, 1);
                dim3 threadsOnes(numThreadsLastBlock, 1, 1);

                if (sharedMemSize < 2048)
                    sharedMemSize = 2048;

                if (sharedMemLastBlock < 2048)
                    sharedMemLastBlock = 2048;

                // execute the scan
                if (numBlocks > 1) {
                    sd::prescanLauncher<true, false>(grid, threads, sharedMemSize, stream, dZ, dX, g_scanBlockSums[level], numThreads * 2, 0, 0);
                    if (np2LastBlock) {
                        sd::prescanLauncher<true, true>(gridOnes, threadsOnes, sharedMemLastBlock, stream, dZ, dX, g_scanBlockSums[level], numEltsLastBlock, numBlocks - 1, numElements - numEltsLastBlock);
                    }

                    // After scanning all the sub-blocks, we are mostly done.  But now we
                    // need to take all of the last values of the sub-blocks and scan those.
                    // This will give us a new value that must be sdded to each block to
                    // get the final results.
                    // recursive (CPU) call
                    prescanArrayRecursive(g_scanBlockSums, g_scanBlockSums[level], g_scanBlockSums[level], numBlocks, level+1);

                    sd::uniformAdd<<<grid, threads, 1024, *stream>>>(dZ, g_scanBlockSums[level], numElements - numEltsLastBlock, 0, 0);

                    if (np2LastBlock) {
                        sd::uniformAdd<<<1, numThreadsLastBlock, 1024, *stream>>>(dZ, g_scanBlockSums[level], numEltsLastBlock, numBlocks - 1, numElements - numEltsLastBlock);
                    }
                } else if (isPowerOfTwo(numElements)) {
                    sd::prescanLauncher<false, false>(grid, threads, sharedMemSize, stream, dZ, dX, 0, numThreads * 2, 0, 0);
                } else {
                    sd::prescanLauncher<false, true>(grid, threads, sharedMemSize, stream, dZ, dX, 0, numElements, 0, 0);
                }
            }

            static void encodeThresholdP2Int_(void **prs, int *dx, Nd4jLong N, int *dz) {
                auto stream = LaunchContext::defaultContext()->getCudaStream();

                prescanArrayRecursive(reinterpret_cast<int**>(prs), dz, dx + 1, (int) N, 0);
                sd::DebugHelper::checkErrorCode(stream, "encodeThresholdP2Int(...) failed");
            }

            static void encodeThresholdP3_(void *dx, const Nd4jLong *hXShapeInfo, int *offsets, Nd4jLong N, int *dz){
                auto stream = LaunchContext::defaultContext()->getCudaStream();

                int blockSize = 512;
                int numBlocks = N / blockSize + (N % blockSize ? 1 : 0);

                dim3 launchDims(numBlocks, blockSize, 8192);
                auto xType = sd::ArrayOptions::dataType(hXShapeInfo);
                BUILD_SINGLE_SELECTOR(xType, encoderKernelP3Generic, (launchDims, stream, dx, offsets, N, dz), FLOAT_TYPES);

                sd::DebugHelper::checkErrorCode(stream, "encodeThresholdP3Float(...) failed");
            }


            static NDArray thresholdEstimate_(const NDArray &updates, const float threshold) {
                const int numThreads = 512;
                const int numBlocks = updates.lengthOf() / numThreads + (updates.lengthOf() % numThreads ? 1 : 0);

                auto tmp = NDArrayFactory::create<int>('c', {numBlocks + 1});

                dim3 launchDims(numBlocks, numThreads, 1024);
                auto xType = updates.dataType();

                NDArray::prepareSpecialUse({&tmp}, {&updates});
                BUILD_SINGLE_SELECTOR(xType, encoderKernelP1Generic, (launchDims, LaunchContext::defaultContext()->getCudaStream(), updates.specialBuffer(), updates.lengthOf(), tmp.specialBuffer(), threshold), FLOAT_TYPES);
                NDArray::registerSpecialUse({&tmp}, {&updates});

                return std::move(tmp);
            }

            int32_t thresholdEstimate(const NDArray &updates, const float threshold) {
                return thresholdEstimate_(updates, threshold).e<int>(0);
            }

            void thresholdEncode(NDArray &updates, NDArray &encoded, float threshold) {
                // we need these blocks in order to know, how many "updates" will be processed by each GPU block
                auto blocks = thresholdEstimate_(updates, threshold);

                const int numThreads = 512;
                const int numBlocks = updates.lengthOf() / numThreads + (updates.lengthOf() % numThreads ? 1 : 0);

                const int prefixThreads = 512;
                int numElts = numBlocks;
                int level = 0;

                // here we just calculate number of sumBlock arrays
                do {
                    int numPrefixBlocks = sd::math::nd4j_max<int>(1, sd::math::nd4j_ceil<float, int>((float) numElts / (2.0f * prefixThreads)));
                    if (numPrefixBlocks > 1) {
                        level++;
                    }
                    numElts = numPrefixBlocks;
                } while (numElts > 1);


                std::vector<NDArray> tempArrays(level);
                std::vector<Nd4jPointer> pointers(level);

                level = 0;
                numElts = numBlocks;

                do {
                    int numPrefixBlocks = sd::math::nd4j_max<int>(1, sd::math::nd4j_ceil<float, int>((float) numElts / (2.0f * prefixThreads)));
                    if (numPrefixBlocks > 1) {
                        tempArrays[level] = std::move(NDArrayFactory::create<int>('c', {numPrefixBlocks}));
                        pointers[level] = tempArrays[level].specialBuffer();;
                        level++;
                    }
                    numElts = numPrefixBlocks;
                } while (numElts > 1);

                PointersManager pm(LaunchContext::defaultContext(), "thresholdEncode");
                auto offsets = NDArrayFactory::create<int>('c', {numBlocks});

                // we want to check, if we're hiting external limit on number of encoded elements
                auto numMatches = blocks.e<int>(0);
                if (numMatches > encoded.lengthOf() - 4) {
                    blocks.p(0, encoded.lengthOf() - 4);
                    blocks.syncToDevice();
                }

                NDArray::prepareSpecialUse({}, {&encoded, &updates});

                // filling offsets
                encodeThresholdP2Int_(reinterpret_cast<void **>(pointers.data()),
                                      reinterpret_cast<int*>(blocks.specialBuffer()),
                                      numBlocks,
                                      reinterpret_cast<int*>(offsets.specialBuffer()));

                NDArray::registerSpecialUse({&blocks, &offsets}, {});
                pm.synchronize();


                encodeThresholdP3_(updates.specialBuffer(),
                                   updates.shapeInfo(),
                                   reinterpret_cast<int*>(offsets.specialBuffer()),
                                   updates.lengthOf(),
                                   reinterpret_cast<int*>(encoded.specialBuffer()));

                pm.synchronize();

                NDArray::registerSpecialUse({&encoded, &updates}, {});
            }

            void thresholdDecode(const NDArray &encoded, NDArray &updates) {
                dim3 launchDims(128, 512, 512);
                auto xType = updates.dataType();

                NDArray::prepareSpecialUse({&updates}, {&encoded});
                BUILD_SINGLE_SELECTOR(xType, decoderKernelGeneric, (launchDims, LaunchContext::defaultContext()->getCudaStream(), encoded.specialBuffer(), updates.lengthOf(), updates.specialBuffer()), FLOAT_TYPES);
                NDArray::registerSpecialUse({&updates}, {&encoded});
            }
        }
    }
}
