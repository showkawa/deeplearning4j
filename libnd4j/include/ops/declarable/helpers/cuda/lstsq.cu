#include "hip/hip_runtime.h"
/*
 *  ******************************************************************************
 *  *
 *  *
 *  * This program and the accompanying materials are made available under the
 *  * terms of the Apache License, Version 2.0 which is available at
 *  * https://www.apache.org/licenses/LICENSE-2.0.
 *  *
 *  * See the NOTICE file distributed with this work for additional
 *  * information regarding copyright ownership.
 *  * Unless required by applicable law or agreed to in writing, software
 *  * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 *  * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 *  * License for the specific language governing permissions and limitations
 *  * under the License.
 *  *
 *  * SPDX-License-Identifier: Apache-2.0
 *  *****************************************************************************
 */

//
//  @author GS <sgazeos@gmail.com>
//
#include <system/op_boilerplate.h>
#include <array/NDArray.h>
#include <helpers/MmulHelper.h>
#include <helpers/ShapeUtils.h>
#include <helpers/ConstantTadHelper.h>

#include <ops/declarable/helpers/triangular_solve.h>
#include <ops/declarable/helpers/lup.h>
#include <ops/declarable/helpers/qr.h>
#include <ops/declarable/helpers/lstsq.h>

namespace sd {
namespace ops {
namespace helpers {

    template <typename T>
    static __global__ void fillRegularizerKernel(T* ioMatrixData, const Nd4jLong* ioMatrixShape, const Nd4jLong* ioMatrixTads, const Nd4jLong* ioMatrixOffsets, Nd4jLong batchSize, Nd4jLong rows, T const value) {

        for (auto x = blockIdx.x; x < batchSize; x += gridDim.x) {
            auto z = ioMatrixData + ioMatrixOffsets[x];
            for (auto r = threadIdx.x; r < rows; r += blockDim.x) {
                Nd4jLong pos[] = {r,r};
                auto zIndex = shape::getOffset(ioMatrixTads, pos);
                z[zIndex] = value;
            }
        }

    }

    template <typename T>
    static void fillRegularizer(sd::LaunchContext* context, NDArray& ioMatrix, double const value) {
        auto lastDimsTads = ConstantTadHelper::getInstance().tadForDimensions(ioMatrix.shapeInfo(), {-2, -1});
        auto stream = context->getCudaStream();
        auto rows = ioMatrix.sizeAt(-2);
        //auto cols = ioMatrix.sizeAt(-1);
        fillRegularizerKernel<T><<<256, 256, 128, *stream>>>(ioMatrix.dataBuffer()->specialAsT<T>(), ioMatrix.specialShapeInfo(), lastDimsTads.specialShapeInfo(), lastDimsTads.specialOffsets(), lastDimsTads.numberOfTads(), rows, (T)value);

    }

    template <typename T>
    int leastSquaresSolveFunctor_(sd::LaunchContext* context, NDArray const* leftInput, NDArray const* rightInput, double const l2Regularizer, bool const fast, NDArray* output) {
        if (fast) { // Cholesky decomposition approach
            // Equation for solve A^T * Ax = A^T * b, so
            // 1. Computing A2:
            auto tAtShape = ShapeUtils::evalShapeForMatmul(leftInput->shapeInfo(), leftInput->shapeInfo(), true, false);
            //tAtShape[tAtShape.size() - 2] = output->sizeAt(-2);
            NDArray leftOutput(leftInput->ordering(), tAtShape, output->dataType(), context);
            MmulHelper::matmul(leftInput, leftInput, &leftOutput, true, false); // Computing A2 = A^T * A
            // 2. Computing B' = A^T * b
            auto rightOutput = output->ulike();

            MmulHelper::matmul(leftInput, rightInput, &rightOutput, true, false); // Computing B' = A^T * b
            // 3. Regularization ( indeed A' = A2 - l2Regularizer * I)
            if (l2Regularizer != 0.0) {
                auto regularizer = leftOutput.ulike(); regularizer.nullify();
                fillRegularizer<T>(context, regularizer, (T)l2Regularizer);
                leftOutput += regularizer;
            }

            // 4. Cholesky decomposition -- output matrix is square and lower triangular
            helpers::cholesky(context, &leftOutput, &leftOutput, true); // inplace decomposition
            // 5. Solve two triangular systems:
            auto rightB = rightOutput.ulike(); rightB.nullify();

            helpers::triangularSolveFunctor(context, &leftOutput, &rightOutput, true, false, &rightB);

            helpers::adjointMatrix(context, &leftOutput, true, &leftOutput);
            helpers::triangularSolveFunctor(context, &leftOutput, &rightB, false, false, output);
            // All done
        }
        else { // QR decomposition approach
            // Equation for solve Rx = Q^T * b, where A = Q * R, where Q - orthogonal matrix, and R - upper triangular
            // 1. QR decomposition
            auto qShape = leftInput->getShapeAsVector();
            auto rShape = leftInput->getShapeAsVector();
            qShape[leftInput->rankOf() - 1] = leftInput->sizeAt(-2);

            NDArray Q(leftInput->ordering(), qShape, leftInput->dataType(), context);// = leftInput->ulike();
            NDArray R(leftInput->ordering(), rShape, leftInput->dataType(), context); // = rightInput->ulike();
            helpers::qr(context, leftInput, &Q, &R, true);
            // 2. b` = Q^t * b:
            auto rightOutput = rightInput->ulike();
            MmulHelper::matmul(&Q, rightInput, &rightOutput, true, false);
            // 3. Solve triangular system
            helpers::triangularSolveFunctor(context, &R, &rightOutput, false, false, output);
        }
        return Status::OK();
    }

    int leastSquaresSolveFunctor(sd::LaunchContext* context, NDArray const* leftInput, NDArray const* rightInput, double const l2Regularizer, bool const fast, NDArray* output) {
        BUILD_SINGLE_SELECTOR(leftInput->dataType(), return leastSquaresSolveFunctor_, (context, leftInput, rightInput, l2Regularizer, fast, output), FLOAT_TYPES);
    }

}
}
}
