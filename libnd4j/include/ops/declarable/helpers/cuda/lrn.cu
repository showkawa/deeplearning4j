#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author raver119@gmail.com
//

#include <ops/declarable/helpers/lrn.h>
#include <graph/Status.h>
#include <helpers/ConstantTadHelper.h>

namespace sd {
namespace ops {
namespace helpers {

    template <typename T>
    static _CUDA_G void lrnKernel(void *vx, Nd4jLong  const*xTadShapeInfo, Nd4jLong  const*xTadOffsets, void *vz, Nd4jLong  const*zTadShapeInfo, Nd4jLong  const*zTadOffsets, Nd4jLong numTads, Nd4jLong tadLength, int depth, double bias, double alpha, double beta) {
        extern __shared__ char sharedChar[];
        T* shared = reinterpret_cast<T*>(sharedChar);

        auto xEws = shape::elementWiseStride(xTadShapeInfo);
        auto zEws = shape::elementWiseStride(zTadShapeInfo);

        auto xOrder = shape::order(xTadShapeInfo);
        auto zOrder = shape::order(zTadShapeInfo);

        const T tbias  = static_cast<T>(bias);
        const T tbeta  = static_cast<T>(beta);
        const T talpha = static_cast<T>(alpha);

        // one block of threads processes 1 example within batch
        for (uint i = blockIdx.x; i < numTads; i += gridDim.x) {
            auto x = reinterpret_cast<T*>(vx) + xTadOffsets[i];
            auto z = reinterpret_cast<T*>(vz) + zTadOffsets[i];

            // load everything into shared memory, so we'll operate on shared memory from now on
            shared[threadIdx.x] = x[threadIdx.x * xEws];
            __syncthreads();

            const uint begin = sd::math::nd4j_max<int>(0, threadIdx.x - depth);
            const uint last  = depth + threadIdx.x + 1;
            const uint end   = sd::math::nd4j_min<int>(last, tadLength);

            T prev = 0.;
            for (int s = begin; s < end; s++)
                prev = prev + shared[s] * shared[s];

            z[threadIdx.x * zEws] = shared[threadIdx.x] / sd::math::nd4j_pow<T, T, T>(tbias + alpha * prev, tbeta);
        }
    }

    template <typename X, typename Z>
    static _CUDA_G void lrnBPKernel(void const* vx, Nd4jLong const* xTadShapeInfo, Nd4jLong const* xTadOffsets, void *vz, Nd4jLong const* zTadShapeInfo, Nd4jLong const* zTadOffsets, Nd4jLong numTads, Nd4jLong tadLength, int depth, double bias, double alpha, double beta) {
        extern __shared__ char sharedChar[];
        X* sharedX = reinterpret_cast<X*>(sharedChar);
        Z* sharedY = reinterpret_cast<Z*>(sharedX + blockDim.x);

        auto xEws = shape::elementWiseStride(xTadShapeInfo);
        auto zEws = shape::elementWiseStride(zTadShapeInfo);

        auto xOrder = shape::order(xTadShapeInfo);
        auto zOrder = shape::order(zTadShapeInfo);

        const Z tbias  = static_cast<Z>(bias);
        const Z tbeta  = static_cast<Z>(beta);
        const Z talpha = static_cast<Z>(alpha);
        const Z coeff  = talpha * tbeta;



        for (uint i = blockIdx.x; i < numTads; i += gridDim.x) {
            auto x = reinterpret_cast<X const*>(vx) + xTadOffsets[i];
            auto z = reinterpret_cast<Z*>(vz) + zTadOffsets[i];

            const uint begin = sd::math::nd4j_max<int>(0, threadIdx.x - depth);
            const uint last  = depth + threadIdx.x + 1;
            const uint end   = sd::math::nd4j_min<int>(last, tadLength);

            // load everything into shared memory
            sharedX[threadIdx.x] = x[threadIdx.x * xEws];
            sharedY[threadIdx.x] = 0.f;
            __syncthreads();

            // we're operating in shared memory
            for (int s = begin; s < end; s++)
                sharedY[threadIdx.x] = sharedY[threadIdx.x] + sharedX[s] * sharedX[s];
            __syncthreads();

            Z factor[1024];
            Z init = tbias + talpha * sharedY[threadIdx.x];

            Z prev = 0.f;
            for (uint s = begin; s < end; ++s) {
                factor[s] = sd::math::nd4j_pow<Z, Z, Z>(tbias + talpha * sharedY[s], -tbeta - 1);
                prev = prev + sharedX[s] * factor[s];
            }

            z[threadIdx.x * zEws] = factor[threadIdx.x] * init - 2 * sharedX[threadIdx.x] * coeff * prev;
        }
    }


    template <typename X, typename Z>
    static void lrnBP_(sd::graph::Context& block, const NDArray& input, const NDArray& gradO, NDArray& gradI, const int depth, const float bias, const float alpha, const float beta) {
        auto rank = input.rankOf();
        auto packX = ConstantTadHelper::getInstance().tadForDimensions(input.shapeInfo(), {rank - 1});
        auto packZ = ConstantTadHelper::getInstance().tadForDimensions(gradI.shapeInfo(), {rank - 1});

        const auto tadLength = shape::length(packX.primaryShapeInfo());
        const int numBlocks = sd::math::nd4j_min<Nd4jLong>(1024, packX.numberOfTads());
        const int numThreads = tadLength;

        if (tadLength > 1024 || tadLength < 1)
            throw std::runtime_error("LRN: tadLength > 1024 isn't implemented yet");

        lrnBPKernel<X, Z><<<numBlocks, numThreads, numThreads * sizeof(X) + numThreads * sizeof(Z) + 1024, *block.launchContext()->getCudaStream()>>>(input.specialBuffer(), packX.platformShapeInfo(), packX.platformOffsets(), gradI.specialBuffer(), packZ.platformShapeInfo(), packZ.platformOffsets(), packX.numberOfTads(),  tadLength, depth, bias, alpha, beta);

        gradI.tickWriteDevice();
        gradI *= gradO;
    }

    void lrnBP(sd::graph::Context& block, const NDArray& input, const NDArray& gradO, NDArray& gradI, const int depth, const float bias, const float alpha, const float beta) {
        input.syncToDevice();
        gradO.syncToDevice();

        BUILD_DOUBLE_SELECTOR(input.dataType(), gradO.dataType(), lrnBP_, (block, input, gradO, gradI, depth, bias, alpha, beta), FLOAT_TYPES, FLOAT_TYPES);

        gradI.tickWriteDevice();
    }

    template <typename T>
    static void lrnFunctor_(sd::graph::Context& block, NDArray* input, NDArray* output, int depth, double bias, double alpha, double beta) {
        auto rank = input->rankOf();
        auto packX = ConstantTadHelper::getInstance().tadForDimensions(input->shapeInfo(), {rank - 1});
        auto packZ = ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(), {rank - 1});

        const auto tadLength = shape::length(packX.primaryShapeInfo());
        const int numBlocks = sd::math::nd4j_min<Nd4jLong>(1024, packX.numberOfTads());
        const int numThreads = tadLength;

        if (tadLength > 1024 || tadLength < 1)
            throw std::runtime_error("LRN: tadLength > 1024 isn't implemented yet");

        lrnKernel<T><<<numBlocks, numThreads, numThreads * sizeof(T), *block.launchContext()->getCudaStream()>>>(input->specialBuffer(), packX.platformShapeInfo(), packX.platformOffsets(), output->specialBuffer(), packZ.platformShapeInfo(), packZ.platformOffsets(), packX.numberOfTads(), tadLength, depth, bias, alpha, beta);
    }

    int lrnFunctor(sd::graph::Context& block, NDArray* input, NDArray* output, int depth, double bias, double alpha, double beta) {
        input->syncToDevice();

        BUILD_SINGLE_SELECTOR(input->dataType(), lrnFunctor_, (block, input, output, depth, bias, alpha, beta), FLOAT_TYPES);

        output->tickWriteDevice();

        return Status::OK();
    }
}
}
}
