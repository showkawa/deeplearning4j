#include "hip/hip_runtime.h"
/*
 *  ******************************************************************************
 *  *
 *  *
 *  * This program and the accompanying materials are made available under the
 *  * terms of the Apache License, Version 2.0 which is available at
 *  * https://www.apache.org/licenses/LICENSE-2.0.
 *  *
 *  * See the NOTICE file distributed with this work for additional
 *  * information regarding copyright ownership.
 *  * Unless required by applicable law or agreed to in writing, software
 *  * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 *  * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 *  * License for the specific language governing permissions and limitations
 *  * under the License.
 *  *
 *  * SPDX-License-Identifier: Apache-2.0
 *  *****************************************************************************
 */

//
// @author Yurii Shyrma (iuriish@yahoo.com)
//

#include <ops/declarable/helpers/convolutions.h>
#include <helpers/PointersManager.h>

namespace sd {
namespace ops  {

//////////////////////////////////////////////////////////////////////////
template <typename T>
__global__ static void upsampling2dBPCuda(const void* vx, const Nd4jLong* xShapeInfo, void* vz, const Nd4jLong* zShapeInfo, const bool isNCHW) {

    // x (gradO) has shape [bS, iC, factorH*iH, factorW*iW ] (NCHW) or [bS, factorH*iH, factorW*iW, iC] (NHWC)
    // z (gradI) has shape [bS, iC, iH, iW] (NCHW) or [bS, iH, iW, iC] (NHWC)

    const T* x = reinterpret_cast<const T*>(vx);
          T* z = reinterpret_cast<T*>(vz);

    __shared__ int rank, dimIH;
    __shared__ uint factorH, factorW;
    __shared__ Nd4jLong zLen, *sharedMem;

    if (threadIdx.x == 0) {
        extern __shared__ unsigned char shmem[];
        sharedMem = reinterpret_cast<Nd4jLong*>(shmem);

        dimIH = isNCHW ? 2 : 1;
        zLen  = shape::length(zShapeInfo);
        rank  = 4;

        factorH = xShapeInfo[dimIH + 1] / zShapeInfo[dimIH + 1];
        factorW = xShapeInfo[dimIH + 2] / zShapeInfo[dimIH + 2];
    }
    __syncthreads();

    const auto zInd = threadIdx.x + blockIdx.x * blockDim.x;

    if(zInd >= zLen)
        return;

    auto coords = sharedMem + threadIdx.x * rank;

    shape::index2coords(zInd, zShapeInfo, coords);

    const auto zOffset = shape::getOffset(zShapeInfo, coords);

    z[zOffset] = 0;

    const Nd4jLong zCoord2 = coords[dimIH]     * factorH;
    const Nd4jLong zCoord3 = coords[dimIH + 1] * factorW;

    for(coords[dimIH] = zCoord2; coords[dimIH] < zCoord2 + factorH; ++coords[dimIH])
        for(coords[dimIH + 1] = zCoord3; coords[dimIH + 1] < zCoord3 + factorW; ++coords[dimIH + 1])
            z[zOffset] += x[shape::getOffset(xShapeInfo, coords)];
}

//////////////////////////////////////////////////////////////////////////
template <typename T>
static void upsampling2dBPCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream,
                                       const void* vx, const Nd4jLong* xShapeInfo,
                                             void* vz, const Nd4jLong* zShapeInfo,
                                       const bool isNCHW) {

    upsampling2dBPCuda<T><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(vx, xShapeInfo, vz, zShapeInfo, isNCHW);
}

//////////////////////////////////////////////////////////////////////////
void ConvolutionUtils::upsampling2dBP(sd::graph::Context& block, const NDArray& gradO, NDArray& gradI, const bool isNCHW) {

    PointersManager manager(block.launchContext(), "upsampling2d_bp");

    const int threadsPerBlock = MAX_NUM_THREADS / 2;
    const int blocksPerGrid = (gradI.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;
    const int sharedMem = gradI.rankOf() * sizeof(Nd4jLong) * threadsPerBlock + 128;

    NDArray::prepareSpecialUse({&gradI}, {&gradO});
    BUILD_SINGLE_SELECTOR(gradI.dataType(), upsampling2dBPCudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, block.launchContext()->getCudaStream(), gradO.specialBuffer(), gradO.specialShapeInfo(), gradI.specialBuffer(), gradI.specialShapeInfo(), isNCHW), FLOAT_TYPES);
    NDArray::registerSpecialUse({&gradI}, {&gradO});

    manager.synchronize();
}

}
}