#include "hip/hip_runtime.h"
/*
 *  ******************************************************************************
 *  *
 *  *
 *  * This program and the accompanying materials are made available under the
 *  * terms of the Apache License, Version 2.0 which is available at
 *  * https://www.apache.org/licenses/LICENSE-2.0.
 *  *
 *  * See the NOTICE file distributed with this work for additional
 *  * information regarding copyright ownership.
 *  * Unless required by applicable law or agreed to in writing, software
 *  * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 *  * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 *  * License for the specific language governing permissions and limitations
 *  * under the License.
 *  *
 *  * SPDX-License-Identifier: Apache-2.0
 *  *****************************************************************************
 */

//
//  @author GS <sgazeos@gmail.com>
//

#include <system/op_boilerplate.h>
#include <array/NDArray.h>
#include <execution/Threads.h>
#include <helpers/ConstantTadHelper.h>
#include "../triangular_solve.h"

namespace sd {
    namespace ops {
        namespace helpers {
            /*
             * lower triangular process for system of linear equations
             * x_1 = b_1/a_1,1
             * x_2 = (b_2 - a_2,1 * x_1) / a_2,2
             * x_3 = (b_3 - a_3,1 * x_1 - a_3,2 * x_2) / a_3,3
             * ...
             * x_M = (b_M - a_M,1 * x_1 - ... a_M,M-1 * x_M-1)/ a_M,M
             *
             * output == x
             * a == leftInput
             * b == rightInput
             *
             * */
            template <typename T>
            static _CUDA_HD void lowerTriangularSolve(T const* leftInput, Nd4jLong const* leftInputShape,
                                                        T const* rightInput, Nd4jLong const* rightInputShape,
                                                        bool const unitOnDiag, T* output, const Nd4jLong* outputShape,
                                                        Nd4jLong rows, Nd4jLong cols) {

                for (auto r = 0; r < rows; r++) {
                    for (auto j = 0; j < cols; j++) {
                        Nd4jLong posY[] = {r, j};
                        Nd4jLong posX[] = {r, r};
                        auto xIndex = shape::getOffset(leftInputShape, posX, 0);
                        auto yIndex = shape::getOffset(rightInputShape, posY, 0);
                        auto zIndex = shape::getOffset(outputShape, posY, 0);

                        auto sum = rightInput[yIndex];
                        for (auto c = 0; c < r; c++) {
                            Nd4jLong posZ[] = {c, j};
                            Nd4jLong pos[] = {r, c};
                            auto xcIndex = shape::getOffset(leftInputShape, pos, 0);
                            auto zcIndex = shape::getOffset(outputShape, posZ, 0);
                            sum -= leftInput[xcIndex] * output[zcIndex];
                        }
                        output[zIndex] = unitOnDiag?sum:sum / leftInput[xIndex];
                    }
                }
            }

            /*
             * upper triangular process for system of linear equations
             * x_M = b_M/a_M,M
             * x_M-1 = (b_M-1 - a_M-1,M-2 * x_M) / a_M-1,M-1
             * x_M-2 = (b_M-2 - a_M-2,M-3 * x_M-2 - a_M-2,M-1 * x_M) / a_3,3
             * ...
             * x_1 = (b_1 - a_1,2 * x_2 - ... a_1,M * x_M)/ a_1,1
             *
             * output == x
             * a == leftInput
             * b == rightInput
             *
             * */

            template <typename T>
            static _CUDA_HD void upperTriangularSolve(T const* leftInput, Nd4jLong const* leftInputShape,
                    T const* rightInput, Nd4jLong const* rightInputShape, bool const unitOnDiag, T* output,
                    const Nd4jLong* outputShape, Nd4jLong rows, Nd4jLong cols) {

                for (auto r = rows; r > 0; r--) {
                    for (auto j = 0; j < cols; j++) {
                        Nd4jLong posY[] = {r - 1, j};
                        Nd4jLong posX[] = {r - 1, r - 1};
                        auto xIndex = shape::getOffset(leftInputShape, posX, 0);
                        auto yIndex = shape::getOffset(rightInputShape, posY, 0);
                        auto zIndex = shape::getOffset(outputShape, posY, 0);
                        auto sum = rightInput[yIndex];
                        for (auto c = r; c < rows; c++) {
                            Nd4jLong posZ[] = {c, j};
                            Nd4jLong pos[] = {r - 1, c};
                            auto zcIndex = shape::getOffset(outputShape, posZ, 0);
                            auto xcIndex = shape::getOffset(leftInputShape, pos, 0);
                            sum -= leftInput[xcIndex] * output[zcIndex];
                        }
                        output[zIndex] = unitOnDiag?sum:sum / leftInput[xIndex];
                    }
                }
            }

            template <typename T>
            static __global__ void triangularSolveKernel(T const* leftInput, Nd4jLong const* leftPartShape,
                    T const* rightInput, Nd4jLong const* rightPartShape, bool const lower, bool const unitsOnDiag, T* output,
                    const Nd4jLong* outputShape, const Nd4jLong* tadLeftShape, const Nd4jLong* tadLeftOffset, const Nd4jLong* tadRightShape,
                    const Nd4jLong* tadRightOffset, const Nd4jLong* tadOutputShape, const Nd4jLong* tadOutputOffset, Nd4jLong batchNum) {

                __shared__ Nd4jLong rows;
                __shared__ Nd4jLong cols;

                if (threadIdx.x == 0) {
                    rows = shape::sizeAt(leftPartShape, -2);
                    cols = shape::sizeAt(rightPartShape, -1);
                }
                __syncthreads();

                auto start = blockIdx.x * blockDim.x + threadIdx.x;
                auto stop = batchNum;
                auto increment = blockDim.x * gridDim.x;

                for (auto i = start; i < stop; i += increment) {
                    auto pLeftPart = leftInput + tadLeftOffset[i];
                    auto pRightPart = rightInput + tadRightOffset[i];
                    auto pOutputPart = output + tadOutputOffset[i];
                    if (lower) {
                        lowerTriangularSolve<T>(pLeftPart, tadLeftShape, pRightPart, tadRightShape, unitsOnDiag, pOutputPart, tadOutputShape, rows, cols);
                    } else {
                        upperTriangularSolve<T>(pLeftPart, tadLeftShape, pRightPart, tadRightShape, unitsOnDiag, pOutputPart, tadOutputShape, rows, cols);
                    }
                }
            }

            template <typename T>
            static int triangularSolveFunctor_(sd::LaunchContext * context, NDArray* leftInput, NDArray* rightInput,
                    bool lower, bool unitsOnDiag, NDArray* output) {
                NDArray::prepareSpecialUse({output}, {leftInput, rightInput});
                auto leftTads = ConstantTadHelper::getInstance().tadForDimensions(leftInput->shapeInfo(), {-2, -1});
                auto rightTads = ConstantTadHelper::getInstance().tadForDimensions(rightInput->shapeInfo(), {-2, -1});
                auto outputTads = ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(), {-2, -1});

                auto stream = context->getCudaStream();
                T const* leftBuf = reinterpret_cast<T const*>(leftInput->specialBuffer());
                T const* rightBuf = reinterpret_cast<T const*>(rightInput->specialBuffer());
                T* outputBuf = reinterpret_cast<T*>(output->specialBuffer());
                triangularSolveKernel<T><<<128, 128, 256, *stream>>>(leftBuf, leftInput->specialShapeInfo(),
                        rightBuf, rightInput->specialShapeInfo(), lower, unitsOnDiag, outputBuf, output->specialShapeInfo(),
                        leftTads.specialShapeInfo(), leftTads.specialOffsets(), rightTads.specialShapeInfo(),
                        rightTads.specialOffsets(), outputTads.specialShapeInfo(), outputTads.specialOffsets(),
                        leftTads.numberOfTads());

                NDArray::registerSpecialUse({output}, {leftInput, rightInput});

                return Status::OK();

            }

            ///  triangularSolve2D - 2D implementation of triangularSolveFunctor
            /// \tparam T - type of NDArray output
            /// \param context - launch context pointer
            /// \param leftInput  - T matrix of equation Tx = b
            /// \param rightInput  - b vector of equation Tx = b
            /// \param lower - lower or upper triangular matrix
            /// \param unitsOnDiag - solve for case when only units (1.0) on diagonal is assumed
            /// \param output - output vector (x on equation Tx = b)
            ///
            template <typename T>
            void triangularSolve2D(sd::LaunchContext* context, const NDArray& leftInput, const NDArray& rightInput, bool const lower, bool const unitsOnDiag, NDArray& output) {

                triangularSolveFunctor_<T>(context, const_cast<NDArray*>(&leftInput), const_cast<NDArray*>(&rightInput), lower, unitsOnDiag, &output);

                // leftInput.syncToHost(); rightInput.syncToHost(); output.syncToHost();
                // T const* pLeftPart = (T const*)leftInput.getBuffer();
                // T const* pRightPart = (T const*)rightInput.getBuffer();
                // T* pOutputPart = (T*)output.buffer();
                // auto rows = leftInput.rows();
                // auto cols = leftInput.columns();
                // if (lower) {
                //     lowerTriangularSolve<T>(pLeftPart, leftInput.shapeInfo(), pRightPart, rightInput.shapeInfo(), unitsOnDiag, pOutputPart, output.shapeInfo(), rows, cols);
                // } else {
                //     upperTriangularSolve<T>(pLeftPart, leftInput.shapeInfo(), pRightPart, rightInput.shapeInfo(), unitsOnDiag, pOutputPart, output.shapeInfo(), rows, cols);
                // }
                // output.syncToDevice();
            }
            BUILD_SINGLE_TEMPLATE(template void triangularSolve2D, (sd::LaunchContext* context, NDArray const& leftInput, NDArray const& rightInput, bool const lower, bool const unitsOnDiag, NDArray& output), FLOAT_TYPES);
//            template void triangularSolve2D<float>(sd::LaunchContext* context, NDArray const& leftInput, NDArray const& rightInput, bool const lower, bool const unitsOnDiag, NDArray& output);
//            template void triangularSolve2D<bfloat16>(sd::LaunchContext* context, NDArray const& leftInput, NDArray const& rightInput, bool const lower, bool const unitsOnDiag, NDArray& output);
//            template void triangularSolve2D<float16>(sd::LaunchContext* context, NDArray const& leftInput, NDArray const& rightInput, bool const lower, bool const unitsOnDiag, NDArray& output);
//            template void triangularSolve2D<double>(sd::LaunchContext* context, NDArray const& leftInput, NDArray const& rightInput, bool const lower, bool const unitsOnDiag, NDArray& output);

            int triangularSolveFunctor(sd::LaunchContext * context, NDArray* leftInput, NDArray* rightInput, bool lower, bool unitsOnDiag, NDArray* output) {
                BUILD_SINGLE_SELECTOR(leftInput->dataType(), return triangularSolveFunctor_, (context, leftInput, rightInput, lower, unitsOnDiag, output), FLOAT_NATIVE);
            }

            template <typename T>
            static __global__ void upperAdjointKernel(T const* input, T* output,
                    Nd4jLong batchSize, Nd4jLong rows, Nd4jLong columns,
                    Nd4jLong const* inputTads, Nd4jLong const* inputOffsets, Nd4jLong const* outputTads, Nd4jLong const* outputOffsets) {

                for (auto b = blockIdx.x; b < batchSize; b += gridDim.x) {
                    auto inputPart = input + inputOffsets[b];
                    auto outputPart = output + outputOffsets[b];
                    for (auto r = threadIdx.x; r < rows; r += blockDim.x) {
                        for (auto c = threadIdx.y; c <= r; c += blockDim.y) {
                            Nd4jLong zPos[] = {r, c};
                            Nd4jLong xPos[] = {c, r};
                            auto zIndex = shape::getOffset(outputTads, zPos);
                            auto xIndex = shape::getOffset(inputTads, xPos);
                            outputPart[zIndex] = inputPart[xIndex];
                        }
                    }
                }

            }

            template <typename T>
            static __global__ void lowerAdjointKernel(T const* input, T* output,
                         Nd4jLong batchSize, Nd4jLong rows, Nd4jLong columns,
                         Nd4jLong const* inputTads, Nd4jLong const* inputOffsets, Nd4jLong const* outputTads, Nd4jLong const* outputOffsets) {

                for (auto b = blockIdx.x; b < batchSize; b += gridDim.x) {
                    auto inputPart = input + inputOffsets[b];
                    auto outputPart = output + outputOffsets[b];
                    for (auto r = threadIdx.x; r < rows; r += blockDim.x) {
                        for (auto c = r + threadIdx.y; c < columns; c += blockDim.y) {
                            Nd4jLong zPos[] = {r, c};
                            Nd4jLong xPos[] = {c, r};
                            auto zIndex = shape::getOffset(outputTads, zPos);
                            auto xIndex = shape::getOffset(inputTads, xPos);
                            outputPart[zIndex] = inputPart[xIndex];
                        }
                    }
                }
            }

            template <typename T>
            static void adjointTriangularMatrix_(sd::LaunchContext* context, NDArray const* input, bool const lower,
                    NDArray* output) {

                auto inputTads = ConstantTadHelper::getInstance().tadForDimensions(input->shapeInfo(), {-2, -1});
                auto outputTads = ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(), {-2, -1});
                auto stream = context->getCudaStream();
                auto inputBuf = reinterpret_cast<T const*>(input->specialBuffer());
                auto outputBuf = reinterpret_cast<T*>(output->specialBuffer());
                auto rows = input->sizeAt(-2);
                auto columns = input->sizeAt(-1);

                if (lower) {
                    lowerAdjointKernel<T><<<128, 256, 256, *stream>>>(inputBuf, outputBuf, outputTads.numberOfTads(), rows, columns, inputTads.specialShapeInfo(), inputTads.specialOffsets(), outputTads.specialShapeInfo(), outputTads.specialOffsets());
                } else {
                    upperAdjointKernel<T><<<128, 256, 256, *stream>>>(inputBuf, outputBuf, outputTads.numberOfTads(), rows, columns, inputTads.specialShapeInfo(), inputTads.specialOffsets(), outputTads.specialShapeInfo(), outputTads.specialOffsets());
                }
            }

            void adjointMatrix(sd::LaunchContext* context, NDArray const* input, bool const lower, NDArray* output) {
                BUILD_SINGLE_SELECTOR(input->dataType(), adjointTriangularMatrix_, (context, input, lower, output), FLOAT_NATIVE);
            }

/*
            //////////////////////////////////////////////////////////////////////////
            template <typename T>
            void triangularSolve2D(sd::LaunchContext* context, NDArray const& A, NDArray const& b, bool const lower, bool const unitsOnDiag, NDArray& x) {

                if(A.rankOf() != 2)
                    throw std::runtime_error("triangularSolve2D: input matrix A must be 2D !");

                int temp;

                const bool isBvector = b.isCommonVector(temp);
                const bool isXvector = x.isCommonVector(temp);

                if(A.sizeAt(0) != (isBvector ? b.lengthOf() : b.sizeAt(0)))
                    throw std::runtime_error("triangularSolve2D: A and b must have the same number of rows !");

                if(A.sizeAt(1) != (isXvector ? x.lengthOf() : x.sizeAt(0)))
                    throw std::runtime_error("triangularSolve2D: columns number of array A must be equal to rows number of array x !");

                if(isBvector) {

                    if(lower) {

                        for (int i = 0; i < A.sizeAt(0); ++i) {
                            T sum = b.t<T>(i);
                            for (int j = 0; j < i; ++j)
                                sum -= A.t<T>(i,j) * x.t<T>(j);
                            x.r<T>(i) = unitsOnDiag ? sum : sum / A.t<T>(i,i);
                        }
                    }
                    else {

                        for (int i = A.sizeAt(0) - 1; i >= 0; --i) {
                            T sum = b.t<T>(i);
                            for (int j = i + 1; j < A.sizeAt(1); ++j)
                                sum -= A.t<T>(i,j) * x.t<T>(j);
                            x.r<T>(i) = unitsOnDiag ? sum : sum / A.t<T>(i,i);
                        }
                    }
                }
                else {

                    if(lower) {

                        for (int bCol = 0; bCol < b.sizeAt(1); ++bCol) {
                            for (int i = 0; i < A.sizeAt(0); ++i) {
                                T sum = b.t<T>(i, bCol);
                                for (int j = 0; j < i; ++j)
                                    sum -= A.t<T>(i,j) * x.t<T>(j, bCol);
                                x.r<T>(i, bCol) = unitsOnDiag ? sum : sum / A.t<T>(i,i);
                           }
                        }
                    }
                    else {

                        for (int bCol = 0; bCol < b.sizeAt(1); ++bCol) {
                            for (int i = A.sizeAt(0) - 1; i >= 0; --i) {
                                T sum = b.t<T>(i, bCol);
                                for (int j = i + 1; j < A.sizeAt(1); ++j)
                                    sum -= A.t<T>(i,j) * x.t<T>(j, bCol);
                                x.r<T>(i, bCol) = unitsOnDiag ? sum : sum / A.t<T>(i,i);
                            }
                        }
                    }
                }
            }
            BUILD_SINGLE_TEMPLATE(template void triangularSolve2D, (sd::LaunchContext* context, NDArray const& leftInput, NDArray const& rightInput, bool const lower, bool const unitsOnDiag, NDArray& output), FLOAT_TYPES);
*/


}
}
}
