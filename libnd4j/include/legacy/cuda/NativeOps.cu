#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/


#include <legacy/NativeOpExecutioner.h>
#include <legacy/NativeOps.h>
#include <hip/hip_runtime.h>

#include <system/buffer.h>


#include <loops/transform_any.h>
#include <loops/reduce_bool.h>
#include <loops/reduce_long.h>
#include <loops/scalar.h>
#include <helpers/threshold.h>
#include <ops/specials_cuda.h>
#include <helpers/DebugHelper.h>
#include <execution/AffinityManager.h>

#include <exceptions/datatype_exception.h>
#include <exceptions/cuda_exception.h>
#include <helpers/CudaLaunchHelper.h>
#include <graph/GraphExecutioner.h>
#include <helpers/BlasHelper.h>
#include <graph/GraphHolder.h>
#include <ops/declarable/CustomOperations.h>
#include <helpers/PointersManager.h>


//#include <sys/time.h>

#include <hiprand.h>
#include <graph/Status.h>
#include <helpers/DebugHelper.h>

using namespace sd;

#include <loops/special_kernels.h>
#include <performance/benchmarking/FullBenchmarkSuit.h>
#include <performance/benchmarking/LightBenchmarkSuit.h>

hipDeviceProp_t *deviceProperties;
hipFuncAttributes *funcAttributes = new hipFuncAttributes[64];
int blockLimit = 128;
int maxThreads = 512;
bool allowedP2P = false;
bool supportedP2P = false;
#ifdef __ND4J_EXPERIMENTAL__
bool experimentalSupport = true;
#else
bool experimentalSupport = false;
#endif

int minThreads = 32;

__constant__ char deviceConstantMemory[49152];


// this method just does type conversion in fancy way
int getDeviceId(Nd4jPointer ptrToDeviceId) {
    return (int)(Nd4jLong)ptrToDeviceId;
}

/*
 * Basic CUDA constants here: number of blocks per MP
 */
int getDeviceBlockThreshold(int deviceId) {
	int ccMinor = deviceProperties[deviceId].minor;
	int ccMajor = deviceProperties[deviceId].major;

	int blockThreshold = 8;

	if (ccMajor >= 5)
		blockThreshold = 32;
	else if (ccMajor == 3)
		blockThreshold = 16;
	else if (ccMajor < 3)
		blockThreshold = 8;

	return blockThreshold;
}


/*
 * This message returns shared memory threshold value. default overflow ratio is 0.3
 */
int getDeviceSharedThreshold(int deviceId) {
	int ccMinor = deviceProperties[deviceId].minor;
	int ccMajor = deviceProperties[deviceId].major;

	// please note threshold isn't multiple of 32, and that's NOT a mistake

	int shmemThreshold;
	if (ccMajor == 6 && ccMinor == 0)
		shmemThreshold = 65536;
	else if (ccMajor == 6 && ccMinor == 1)
		shmemThreshold = 49152;
	else if (ccMajor == 5 && ccMinor == 2)
		shmemThreshold = 98304;
	else if (ccMajor == 5)
		shmemThreshold = 65536;
	else if (ccMajor == 3 && ccMinor == 7)
		shmemThreshold = 114688;
	else shmemThreshold = 49152;

	return shmemThreshold / 0.3;
}



sd::buffer::Buffer<Nd4jLong> * createScalarBuffer(hipStream_t stream) {
	auto scalarShapeInfo = shape::createScalarShapeInfo();
	auto buff = sd::buffer::createBuffer(scalarShapeInfo,shape::shapeInfoLength(2), stream);
	sd::buffer::copyDataToGpu(&buff, stream);
	return buff;
}


class ScalarShapeInformation {
private:
	sd::buffer::Buffer<Nd4jLong> *scalarDimension;
	sd::buffer::Buffer<Nd4jLong> *scalarShapeInfo;
//	std::thread::id threadId;

public:
	ScalarShapeInformation(hipStream_t stream) {
		auto scalarDimensionBuff = reinterpret_cast<Nd4jLong *>(malloc(sizeof(Nd4jLong)));

		CHECK_ALLOC(scalarDimensionBuff, "Failed to allocate ShapeInfoBuffer", sizeof(Nd4jLong));

		scalarDimensionBuff[0] = MAX_DIMENSION;
		scalarDimension = sd::buffer::createBuffer(scalarDimensionBuff,1, stream);
		scalarShapeInfo = createScalarBuffer(stream);
//		threadId = std::this_thread::get_id();

	}
	~ScalarShapeInformation() {
		sd::buffer::freeBuffer(&scalarShapeInfo);
		sd::buffer::freeBuffer(&scalarDimension);
	}


	Nd4jLong *getShapeInfoHostPointer() {
		return scalarShapeInfo->data;
	}

	Nd4jLong * getShapeInfoGpuPointer() {
		return scalarShapeInfo->gData;
	}

	Nd4jLong * getDimensionHostPointer() {
		return scalarDimension->data;
	}

	Nd4jLong  * getDimensionGpuPointer() {
		return scalarDimension->gData;
	}

};





template <typename T>
class ScalarInfo {
	sd::buffer::Buffer<T> *scalarData;
	ScalarShapeInformation *shapeInfo;
	T finalResult;
	hipStream_t streamRef;
public:
	ScalarInfo(hipStream_t stream) {
		T *scalarResult = reinterpret_cast<T*>(malloc(sizeof(T)));

		CHECK_ALLOC(scalarResult, "Failed to allocate new scalar buffer", sizeof(T));

		shapeInfo = new ScalarShapeInformation(stream);
		scalarData = sd::buffer::createBuffer(scalarResult,1, stream);
		streamRef = stream;
		sd::buffer::copyDataToGpu(&scalarData, stream);
	}

	T getFinalResultFromDevice() {
		sd::buffer::copyDataFromGpu(&scalarData, streamRef);
		return scalarData->data[0];
	}

	/**
	 * Get the device shape information
	 * representing a scalar
	 */
	 Nd4jLong *getDeviceShapeInfo() {
		return shapeInfo->getShapeInfoGpuPointer();
	}

	/**
	 * Get the dZ pointers
	 */
	 T *getDevicePointer() {
		 return scalarData->gData;
	 }

	 /**
	  * Get the infinite dimension device pointer
	  */
	  Nd4jLong *getDimensionDevicePointer() {
		 return shapeInfo->getDimensionGpuPointer();
	 }

	 ~ScalarInfo() {
		 sd::buffer::freeBuffer(&scalarData);
		 delete shapeInfo;
	 }
};

void execPairwiseTransform( Nd4jPointer *extraPointers,
        								int opNum,
                                        OpaqueDataBuffer *dbX, Nd4jLong const* hXShapeInfo, Nd4jLong const* dXShapeInfo,
                                        OpaqueDataBuffer *dbY, Nd4jLong const* hYShapeInfo, Nd4jLong const* dYShapeInfo,
                                        OpaqueDataBuffer *dbZ, Nd4jLong const* hZShapeInfo, Nd4jLong const* dZShapeInfo,
        								void *extraParams) {
    try {
        InteropDataBuffer::prepareSpecialUse({dbZ}, {dbX, dbY});

        LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
        NativeOpExecutioner::execPairwiseTransform(&lc, opNum, dbX->primary(), hXShapeInfo, dbX->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hXShapeInfo).special(),
                                                        dbY->primary(), hYShapeInfo, dbY->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hYShapeInfo).special(),
                                                        dbZ->primary(), hZShapeInfo, dbZ->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hZShapeInfo).special(), extraParams);

        InteropDataBuffer::registerSpecialUse({dbZ}, {dbX, dbY});
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
    }
}

////////////////////////////////////////////////////////////////////////
void execPairwiseTransformBool(Nd4jPointer *extraPointers,
        								int opNum,
                                        OpaqueDataBuffer *dbX, Nd4jLong const* hXShapeInfo, Nd4jLong const* dXShapeInfo,
                                        OpaqueDataBuffer *dbY, Nd4jLong const* hYShapeInfo, Nd4jLong const* dYShapeInfo,
        								OpaqueDataBuffer *dbZ, Nd4jLong const* hZShapeInfo, Nd4jLong const* dZShapeInfo,
        								void *extraParams) {
    try {
        InteropDataBuffer::prepareSpecialUse({dbZ}, {dbX, dbY});

        LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
        NativeOpExecutioner::execPairwiseBoolTransform(&lc, opNum,
                                                       dbX->primary(), hXShapeInfo, dbX->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hXShapeInfo).special(),
                                                       dbY->primary(), hYShapeInfo, dbY->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hYShapeInfo).special(),
                                                       dbZ->primary(), hZShapeInfo, dbZ->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hZShapeInfo).special(),
                                                       extraParams);

        InteropDataBuffer::registerSpecialUse({dbZ}, {dbX, dbY});
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
    }
}

////////////////////////////////////////////////////////////////////////
void execSummaryStatsScalar(Nd4jPointer *extraPointers,
                                       int opNum,
                                       OpaqueDataBuffer *dbX, Nd4jLong const* hXShapeInfo, Nd4jLong const* dXShapeInfo,
                                       void *extraParams,
                                       OpaqueDataBuffer *dbZ, Nd4jLong const* hZShapeInfo, Nd4jLong const* dZShapeInfo,
                                       bool biasCorrected) {
    try {
        InteropDataBuffer::prepareSpecialUse({dbZ}, {dbX});

        LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
        NativeOpExecutioner::execSummaryStatsScalar(&lc, opNum,
                                                         dbX->primary(), hXShapeInfo, dbX->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hXShapeInfo).special(),
                                                         extraParams,
                                                         dbZ->primary(), hZShapeInfo, dbZ->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hZShapeInfo).special(),
                                                         biasCorrected);

        InteropDataBuffer::registerSpecialUse({dbZ}, {dbX});
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
    }
}

////////////////////////////////////////////////////////////////////////
void execBroadcastBool(Nd4jPointer *extraPointers,
        						int opNum,
                                OpaqueDataBuffer *dbX, Nd4jLong const* hXShapeInfo, Nd4jLong const* dXShapeInfo,
                                OpaqueDataBuffer *dbY, Nd4jLong const* hYShapeInfo, Nd4jLong const* dYShapeInfo,
                                OpaqueDataBuffer *dbZ, Nd4jLong const* hZShapeInfo, Nd4jLong const* dZShapeInfo,
                                void *extraParams,
                                OpaqueDataBuffer *dbDimension, Nd4jLong const* hDimensionShape, Nd4jLong const* dDimensionShape) {
    try {
        InteropDataBuffer::prepareSpecialUse({dbZ}, {dbX, dbY});
        InteropDataBuffer::preparePrimaryUse({}, {dbDimension});

        auto dimension = reinterpret_cast<int *>(dbDimension->primary());
        int dimensionLength = static_cast<int>(shape::length(hDimensionShape));

        auto hTADShapeInfo = reinterpret_cast<Nd4jLong *>(extraPointers[9]);
        auto tadOnlyShapeInfo = reinterpret_cast<Nd4jLong *>(extraPointers[10]);
        auto tadOffsets = reinterpret_cast<Nd4jLong *>(extraPointers[11]);
        auto tadOnlyShapeInfoZ = reinterpret_cast<Nd4jLong *>(extraPointers[12]);
        auto tadOffsetsZ = reinterpret_cast<Nd4jLong *>(extraPointers[13]);

        LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
        NativeOpExecutioner::execBroadcastBool(&lc, opNum,
                dbX->primary(), hXShapeInfo, dbX->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hXShapeInfo).special(),
                dbY->primary(), hYShapeInfo, dbY->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hYShapeInfo).special(),
                dbZ->primary(), hZShapeInfo, dbZ->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hZShapeInfo).special(),
                extraParams,
                dimension, dimensionLength,
                tadOnlyShapeInfo, tadOffsets, tadOnlyShapeInfoZ, tadOffsetsZ);

        InteropDataBuffer::registerSpecialUse({dbZ}, {dbX, dbY});
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
    }
}

/**
 *
 * @param opNum
 * @param dX
 * @param dXShapeInfo
 * @param dY
 * @param dYShapeInfo
 * @param dZ
 * @param dZShapeInfo
 * @param dimension
 * @param dimensionLength
 */
void   execBroadcast(
		Nd4jPointer *extraPointers,
		int opNum,
        OpaqueDataBuffer *dbX, Nd4jLong const* hXShapeInfo, Nd4jLong const* dXShapeInfo,
        OpaqueDataBuffer *dbY, Nd4jLong const* hYShapeInfo, Nd4jLong const* dYShapeInfo,
        OpaqueDataBuffer *dbZ, Nd4jLong const* hZShapeInfo, Nd4jLong const* dZShapeInfo,
        OpaqueDataBuffer *dbDimension, Nd4jLong const* hDimensionShape, Nd4jLong const* dDimensionShape) {
    try {
        InteropDataBuffer::prepareSpecialUse({dbZ}, {dbX, dbY});
        InteropDataBuffer::preparePrimaryUse({}, {dbDimension});

        auto dimension = reinterpret_cast<int *>(dbDimension->primary());
        int dimensionLength = static_cast<int>(shape::length(hDimensionShape));

        hipStream_t *stream = reinterpret_cast<hipStream_t *>(extraPointers[1]);

        auto hTADShapeInfo = reinterpret_cast<Nd4jLong *>(extraPointers[9]);
        auto tadOnlyShapeInfo = reinterpret_cast<Nd4jLong *>(extraPointers[10]);
        auto tadOffsets = reinterpret_cast<Nd4jLong *>(extraPointers[11]);
        auto tadOnlyShapeInfoZ = reinterpret_cast<Nd4jLong *>(extraPointers[12]);
        auto tadOffsetsZ = reinterpret_cast<Nd4jLong *>(extraPointers[13]);

        auto xType = sd::ArrayOptions::dataType(hXShapeInfo);
        auto yType = sd::ArrayOptions::dataType(hYShapeInfo);
        auto zType = sd::ArrayOptions::dataType(hZShapeInfo);

        LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
        NativeOpExecutioner::execBroadcast(&lc, opNum,
                dbX->primary(), hXShapeInfo, dbX->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hXShapeInfo).special(),
                dbY->primary(), hYShapeInfo, dbY->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hYShapeInfo).special(),
                dbZ->primary(), hZShapeInfo, dbZ->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hZShapeInfo).special(),
                dimension, dimensionLength,
                tadOnlyShapeInfo, tadOffsets, tadOnlyShapeInfoZ, tadOffsetsZ);

        InteropDataBuffer::registerSpecialUse({dbZ}, {dbX, dbY});
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
    }
}


/**
 *
 * @param opNum
 * @param dX
 * @param dXShapeInfo
 * @param extraParams
 * @param dZ
 * @param dZShapeInfo
 */
////////////////////////////////////////////////////////////////////////
void execReduceFloat(Nd4jPointer *extraPointers,
							int opNum,
                            OpaqueDataBuffer *dbX, Nd4jLong const* hXShapeInfo, Nd4jLong const* dXShapeInfo,
							void *extraParams,
                            OpaqueDataBuffer *dbZ, Nd4jLong const* hZShapeInfo, Nd4jLong const* dZShapeInfo) {
    try {
        InteropDataBuffer::prepareSpecialUse({dbZ}, {dbX});

        LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
        NativeOpExecutioner::execReduceFloatScalar(&lc, opNum,
                dbX->primary(), hXShapeInfo, dbX->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hXShapeInfo).special(),
                extraParams,
                dbZ->primary(), hZShapeInfo, dbZ->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hZShapeInfo).special());

        InteropDataBuffer::registerSpecialUse({dbZ}, {dbX});
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
    }
}

////////////////////////////////////////////////////////////////////////
void execReduceSame(Nd4jPointer *extraPointers,
                                int opNum,
                                OpaqueDataBuffer *dbX, Nd4jLong const* hXShapeInfo, Nd4jLong const* dXShapeInfo,
                                void *extraParams,
                                OpaqueDataBuffer *dbZ, Nd4jLong const* hZShapeInfo, Nd4jLong const* dZShapeInfo) {
    try {
        InteropDataBuffer::prepareSpecialUse({dbZ}, {dbX});

        LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
        NativeOpExecutioner::execReduceSameScalar(&lc, opNum,
                dbX->primary(), hXShapeInfo, dbX->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hXShapeInfo).special(),
                extraParams,
                dbZ->primary(), hZShapeInfo, dbZ->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hZShapeInfo).special());

        InteropDataBuffer::registerSpecialUse({dbZ}, {dbX});
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
    }
}

////////////////////////////////////////////////////////////////////////
void execReduceSame2(Nd4jPointer *extraPointers,
                            int opNum,
                            OpaqueDataBuffer *dbX, Nd4jLong const*hXShapeInfo, Nd4jLong const*dXShapeInfo,
                            void *extraParams,
                            OpaqueDataBuffer *dbZ, Nd4jLong const*hZShapeInfo, Nd4jLong const*dZShapeInfo,
                            OpaqueDataBuffer *dbDimension, Nd4jLong const*hDimensionShape, Nd4jLong const*dDimensionShape) {
    try {
        InteropDataBuffer::prepareSpecialUse({dbZ}, {dbX});
        InteropDataBuffer::preparePrimaryUse({}, {dbDimension});

        auto dimension = reinterpret_cast<int *>(dbDimension->primary());
        int dimensionLength = static_cast<int>(shape::length(hDimensionShape));

        const auto zLen = shape::length(hZShapeInfo);

        std::vector<int> dimensions(dimension, dimension + dimensionLength);

        const Nd4jLong* zShapeInfoH = hZShapeInfo;

        if(shape::rank(hXShapeInfo) - dimensionLength != shape::rank(hZShapeInfo) && zLen != 1) {
            auto zPack = ConstantShapeHelper::getInstance().createShapeInfoWithNoUnitiesForReduce(hZShapeInfo, dimensions);
            zShapeInfoH = reinterpret_cast<Nd4jLong const*>(zPack.primary());
        }

        std::vector<int> dims = (zLen != 1) ? ShapeUtils::evalDimsForReduceOp(shape::rank(hXShapeInfo), dimensions) : std::vector<int>();
        LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
        NativeOpExecutioner::execReduceSame(&lc, opNum,
                dbX->primary(), hXShapeInfo, dbX->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hXShapeInfo).special(),
                extraParams,
                dbZ->primary(), zShapeInfoH, dbZ->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(zShapeInfoH).special(),
                dims.data(), dims.size());

        InteropDataBuffer::registerSpecialUse({dbZ}, {dbX});
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
    }
}

////////////////////////////////////////////////////////////////////////
void execReduceLong2(Nd4jPointer *extraPointers,
                            int opNum,
                            OpaqueDataBuffer *dbX, Nd4jLong const*hXShapeInfo, Nd4jLong const*dXShapeInfo,
                            void *extraParams,
                            OpaqueDataBuffer *dbZ, Nd4jLong const*hZShapeInfo, Nd4jLong const*dZShapeInfo,
                            OpaqueDataBuffer *dbDimension, Nd4jLong const*hDimensionShape, Nd4jLong const*dDimensionShape) {
    try {
        InteropDataBuffer::prepareSpecialUse({dbZ}, {dbX});
        InteropDataBuffer::preparePrimaryUse({}, {dbDimension});

        auto dimension = reinterpret_cast<int *>(dbDimension->primary());
        int dimensionLength = static_cast<int>(shape::length(hDimensionShape));

        const auto zLen = shape::length(hZShapeInfo);

        std::vector<int> dimensions(dimension, dimension + dimensionLength);

        const Nd4jLong* zShapeInfoH = hZShapeInfo;

        if(shape::rank(hXShapeInfo) - dimensionLength != shape::rank(hZShapeInfo) && zLen != 1) {
            auto zPack = ConstantShapeHelper::getInstance().createShapeInfoWithNoUnitiesForReduce(hZShapeInfo, dimensions);
            zShapeInfoH = reinterpret_cast<Nd4jLong const*>(zPack.primary());
        }

        std::vector<int> dims = (zLen != 1) ? ShapeUtils::evalDimsForReduceOp(shape::rank(hXShapeInfo), dimensions) : std::vector<int>();

        LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
        NativeOpExecutioner::execReduceLong(&lc, opNum,
                dbX->primary(), hXShapeInfo, dbX->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hXShapeInfo).special(),
                extraParams,
                dbZ->primary(), zShapeInfoH, dbZ->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(zShapeInfoH).special(),
                dims.data(), dims.size());

        InteropDataBuffer::registerSpecialUse({dbZ}, {dbX});
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
    }
}

////////////////////////////////////////////////////////////////////////
void   execReduceLong(Nd4jPointer *extraPointers,
                                int opNum,
                                OpaqueDataBuffer *dbX, Nd4jLong const*hXShapeInfo, Nd4jLong const*dXShapeInfo,
                                void *extraParams,
                                OpaqueDataBuffer *dbZ, Nd4jLong const*hZShapeInfo, Nd4jLong const*dZShapeInfo) {
    try {
        InteropDataBuffer::prepareSpecialUse({dbZ}, {dbX});

        auto stream = reinterpret_cast<hipStream_t *>(extraPointers[1]);
        auto hTADShapeInfo = reinterpret_cast<Nd4jLong *>(extraPointers[9]);
        auto dTADShapeInfo = reinterpret_cast<Nd4jLong *>(extraPointers[10]);

        auto reductionPointer = reinterpret_cast<void *>(extraPointers[4]);

        auto xType = sd::ArrayOptions::dataType(hXShapeInfo);
        auto zType = sd::ArrayOptions::dataType(hZShapeInfo);

        if (zType != sd::DataType::INT64)
            throw datatype_exception::build("execReduceLong wrong Z data type", sd::DataType::INT64, zType);

        auto xLength = shape::length(hXShapeInfo);
        auto blockWidth = 256;
        auto numBlocks = CudaLaunchHelper::getReductionBlocks(xLength, blockWidth);
        dim3 launchDims(numBlocks, blockWidth, 32768);

        BUILD_DOUBLE_SELECTOR(xType, zType, functions::reduce::ReduceLongFunction,
                              ::execReduceScalar(launchDims, stream, opNum,
                                      dbX->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hXShapeInfo).special(), hXShapeInfo,
                                      extraParams,
                                      dbZ->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hZShapeInfo).special(), hXShapeInfo,
                                      nullptr, 0, reductionPointer, dTADShapeInfo), LIBND4J_TYPES, LONG_TYPES);

        sd::DebugHelper::checkErrorCode(stream, "execReduceLong(...) failed");

        InteropDataBuffer::registerSpecialUse({dbZ}, {dbX});
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
    }
}

////////////////////////////////////////////////////////////////////////
void execReduceBool2(Nd4jPointer *extraPointers,
                            int opNum,
                            OpaqueDataBuffer *dbX, Nd4jLong const*hXShapeInfo, Nd4jLong const*dXShapeInfo,
                            void *extraParams,
                            OpaqueDataBuffer *dbZ, Nd4jLong const*hZShapeInfo, Nd4jLong const*dZShapeInfo,
                            OpaqueDataBuffer *dbDimension, Nd4jLong const*hDimensionShape, Nd4jLong const*dDimensionShape) {
    try {
        InteropDataBuffer::prepareSpecialUse({dbZ}, {dbX});
        InteropDataBuffer::preparePrimaryUse({}, {dbDimension});

        auto dimension = reinterpret_cast<int *>(dbDimension->primary());
        int dimensionLength = static_cast<int>(shape::length(hDimensionShape));

        const auto zLen = shape::length(hZShapeInfo);

        std::vector<int> dimensions(dimension, dimension + dimensionLength);

        const Nd4jLong* zShapeInfoH = hZShapeInfo;

        if(shape::rank(hXShapeInfo) - dimensionLength != shape::rank(hZShapeInfo) && zLen != 1) {
            auto zPack = ConstantShapeHelper::getInstance().createShapeInfoWithNoUnitiesForReduce(hZShapeInfo, dimensions);
            zShapeInfoH = reinterpret_cast<Nd4jLong const*>(zPack.primary());
        }

        std::vector<int> dims = (zLen != 1) ? ShapeUtils::evalDimsForReduceOp(shape::rank(hXShapeInfo), dimensions) : std::vector<int>();

        LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
        NativeOpExecutioner::execReduceBool(&lc, opNum,
                dbX->primary(), hXShapeInfo, dbX->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hXShapeInfo).special(),
                extraParams,
                dbZ->primary(), zShapeInfoH, dbZ->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(zShapeInfoH).special(),
                dims.data(), dims.size());

        InteropDataBuffer::registerSpecialUse({dbZ}, {dbX});
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
    }
}

////////////////////////////////////////////////////////////////////////
void   execReduceBool(Nd4jPointer *extraPointers,
                                int opNum,
                                OpaqueDataBuffer *dbX, Nd4jLong const* hXShapeInfo, Nd4jLong const* dXShapeInfo,
                                void *extraParams,
                                OpaqueDataBuffer *dbZ, Nd4jLong const* hZShapeInfo, Nd4jLong const* dZShapeInfo) {
    try {
        InteropDataBuffer::prepareSpecialUse({dbZ}, {dbX});

        auto stream = reinterpret_cast<hipStream_t *>(extraPointers[1]);
        auto hTADShapeInfo = reinterpret_cast<Nd4jLong *>(extraPointers[9]);
        auto dTADShapeInfo = reinterpret_cast<Nd4jLong *>(extraPointers[10]);

        auto reductionPointer = reinterpret_cast<void *>(extraPointers[4]);

        auto xType = sd::ArrayOptions::dataType(hXShapeInfo);
        auto zType = sd::ArrayOptions::dataType(hZShapeInfo);

        if (zType != sd::DataType::BOOL)
            throw std::runtime_error("execReduceBool requires Z operand to have BOOL type");

        auto xLength = shape::length(hXShapeInfo);
        auto blockWidth = 256;
        auto numBlocks = CudaLaunchHelper::getReductionBlocks(xLength, blockWidth);
        dim3 launchDims(numBlocks, blockWidth, 32768);

        BUILD_DOUBLE_SELECTOR(xType, zType, functions::reduce::ReduceBoolFunction,
                              ::execReduceScalar(launchDims, stream, opNum,
                                      dbX->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hXShapeInfo).special(), hXShapeInfo,
                                      extraParams,
                                      dbZ->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hZShapeInfo).special(), hZShapeInfo,
                                      nullptr, 0, reductionPointer, dTADShapeInfo), LIBND4J_TYPES, BOOL_TYPES);

        sd::DebugHelper::checkErrorCode(stream, "execReduceBool(...) failed");

        InteropDataBuffer::registerSpecialUse({dbZ}, {dbX});
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
    }
}

/**
 *
 * @param opNum
 * @param dX
 * @param dXShapeInfo
 * @param extraParams
 * @param dZ
 * @param dZShapeInfo
 * @param dimension
 * @param dimensionLength
 */
////////////////////////////////////////////////////////////////////////
void execIndexReduce(Nd4jPointer *extraPointers,
								 int opNum,
								 OpaqueDataBuffer *dbX, Nd4jLong const* hXShapeInfo, Nd4jLong const* dXShapeInfo,
        						 void *extraParams,
        						 OpaqueDataBuffer *dbZ, Nd4jLong const* hZShapeInfo, Nd4jLong const* dZShapeInfo,
        						 OpaqueDataBuffer *dbDimension, Nd4jLong const* hDimensionShape, Nd4jLong const* dDimensionShape) {
    try {
        InteropDataBuffer::prepareSpecialUse({dbZ}, {dbX});
        InteropDataBuffer::preparePrimaryUse({}, {dbDimension});

        auto dimension = reinterpret_cast<int *>(dbDimension->primary());
        int dimensionLength = static_cast<int>(shape::length(hDimensionShape));

        auto tadPack = sd::ConstantTadHelper::getInstance().tadForDimensions(hXShapeInfo,
                                                                                dimension,
                                                                                shape::length(hDimensionShape));

        LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
        NativeOpExecutioner::execIndexReduce(&lc, opNum,
                dbX->primary(), hXShapeInfo, dbX->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hXShapeInfo).special(),
                extraParams,
                dbZ->primary(), hZShapeInfo, dbZ->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hZShapeInfo).special(),
                (int *) dbDimension->special(), dimensionLength,
                tadPack.specialShapeInfo(), tadPack.specialOffsets());

        InteropDataBuffer::registerSpecialUse({dbZ}, {dbX});
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
    }
}

/**
 *
 * @param opNum
 * @param dX
 * @param dXShapeInfo
 * @param extraParams
 * @param dZ
 * @param dZShapeInfo
 */
////////////////////////////////////////////////////////////////////////
void execReduceFloat2(Nd4jPointer *extraPointers,
								int opNum,
                                OpaqueDataBuffer *dbX, Nd4jLong const* hXShapeInfo, Nd4jLong const* dXShapeInfo,
        						void *extraParams,
                                OpaqueDataBuffer *dbZ, Nd4jLong const* hZShapeInfo, Nd4jLong const* dZShapeInfo,
                                OpaqueDataBuffer *dbDimension, Nd4jLong const* hDimensionShape, Nd4jLong const* dDimensionShape) {
    try {
        InteropDataBuffer::prepareSpecialUse({dbZ}, {dbX});
        InteropDataBuffer::preparePrimaryUse({}, {dbDimension});

        auto dimension = reinterpret_cast<int *>(dbDimension->primary());
        int dimensionLength = static_cast<int>(shape::length(hDimensionShape));

        const auto zLen = shape::length(hZShapeInfo);

        std::vector<int> dimensions(dimension, dimension + dimensionLength);

        const Nd4jLong* zShapeInfoH = hZShapeInfo;

        if(shape::rank(hXShapeInfo) - dimensionLength != shape::rank(hZShapeInfo) && zLen != 1) {
            auto zPack = ConstantShapeHelper::getInstance().createShapeInfoWithNoUnitiesForReduce(hZShapeInfo, dimensions);
            zShapeInfoH = reinterpret_cast<Nd4jLong const*>(zPack.primary());
        }

        std::vector<int> dims = (zLen != 1) ? ShapeUtils::evalDimsForReduceOp(shape::rank(hXShapeInfo), dimensions) : std::vector<int>();

        LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
        NativeOpExecutioner::execReduceFloat(&lc, opNum,
                dbX->primary(), hXShapeInfo, dbX->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hXShapeInfo).special(),
                extraParams,
                dbZ->primary(), zShapeInfoH, dbZ->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(zShapeInfoH).special(),
                dims.data(), dims.size());

        InteropDataBuffer::registerSpecialUse({dbZ}, {dbX});
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
    }
}

/**
 *
 * @param opNum
 * @param dX
 * @param dXShapeInfo
 * @param extraParams
 */
////////////////////////////////////////////////////////////////////////
void execIndexReduceScalar(
		Nd4jPointer *extraPointers,
		int opNum,
        OpaqueDataBuffer *dbX, Nd4jLong const* hXShapeInfo, Nd4jLong const* dXShapeInfo,
        void *extraParams,
        OpaqueDataBuffer *dbZ, Nd4jLong const* hZShapeInfo, Nd4jLong const* dZShapeInfo){
    try {
        InteropDataBuffer::prepareSpecialUse({dbZ}, {dbX});

        LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
        NativeOpExecutioner::execIndexReduceScalar(&lc, opNum,
                dbX->primary(), hXShapeInfo, dbX->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hXShapeInfo).special(),
                extraParams,
                dbZ->primary(), hZShapeInfo, dbZ->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hZShapeInfo).special());

        InteropDataBuffer::registerSpecialUse({dbZ}, {dbX});
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
    }
}

////////////////////////////////////////////////////////////////////////
void execTransformSame(Nd4jPointer *extraPointers,int opNum,
                                   OpaqueDataBuffer *dbX, Nd4jLong const* hXShapeInfo, Nd4jLong const* dXShapeInfo,
                                   OpaqueDataBuffer *dbZ, Nd4jLong const* hZShapeInfo, Nd4jLong const* dZShapeInfo,
                                   void *extraParams) {
    try {
        InteropDataBuffer::prepareSpecialUse({dbZ}, {dbX});

        auto tadShapeInfo = reinterpret_cast<Nd4jLong *>(extraPointers != nullptr ? extraPointers[0] : nullptr);
        auto tadOffsets = reinterpret_cast<Nd4jLong *>(extraPointers != nullptr ? extraPointers[1] : nullptr);

        LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
        NativeOpExecutioner::execTransformSame(&lc, opNum,
                dbX->primary(), hXShapeInfo, dbX->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hXShapeInfo).special(),
                dbZ->primary(), hZShapeInfo, dbZ->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hZShapeInfo).special(),
                extraParams,
                tadShapeInfo, tadOffsets);

        InteropDataBuffer::registerSpecialUse({dbZ}, {dbX});
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
    }
}

////////////////////////////////////////////////////////////////////////
void execTransformBool(Nd4jPointer *extraPointers,int opNum,
                                  OpaqueDataBuffer *dbX, Nd4jLong const* hXShapeInfo, Nd4jLong const* dXShapeInfo,
                                  OpaqueDataBuffer *dbZ, Nd4jLong const* hZShapeInfo, Nd4jLong const* dZShapeInfo,
								  void *extraParams) {
    try {
        InteropDataBuffer::prepareSpecialUse({dbZ}, {dbX});

        auto tadShapeInfo = reinterpret_cast<Nd4jLong *>(extraPointers != nullptr ? extraPointers[0] : nullptr);
        auto tadOffsets = reinterpret_cast<Nd4jLong *>(extraPointers != nullptr ? extraPointers[1] : nullptr);

        LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
        NativeOpExecutioner::execTransformBool(&lc, opNum,
                dbX->primary(), hXShapeInfo, dbX->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hXShapeInfo).special(),
                dbZ->primary(), hZShapeInfo, dbZ->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hZShapeInfo).special(),
                extraParams,
                tadShapeInfo, tadOffsets);

        InteropDataBuffer::registerSpecialUse({dbZ}, {dbX});
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
    }
}

////////////////////////////////////////////////////////////////////////
void execTransformAny(Nd4jPointer *extraPointers,int opNum,
                                    OpaqueDataBuffer *dbX, Nd4jLong const* hXShapeInfo, Nd4jLong const* dXShapeInfo,
                                    OpaqueDataBuffer *dbZ, Nd4jLong const* hZShapeInfo, Nd4jLong const* dZShapeInfo,
								    void *extraParams) {
    try {
        InteropDataBuffer::prepareSpecialUse({dbZ}, {dbX});

        auto stream = reinterpret_cast<hipStream_t *>(extraPointers[1]);
        auto streamSpecial = reinterpret_cast<hipStream_t &>(extraPointers[4]);
        LaunchContext lc(stream, streamSpecial, extraPointers[5], extraPointers[3],
                         reinterpret_cast<int *>(extraPointers[6]));

        NativeOpExecutioner::execTransformAny(&lc, opNum,
                dbX->primary(), hXShapeInfo, dbX->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hXShapeInfo).special(),
                dbZ->primary(), hZShapeInfo, dbZ->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hZShapeInfo).special(),
                extraParams,
                nullptr, nullptr);

        InteropDataBuffer::registerSpecialUse({dbZ}, {dbX});
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
    }
}

////////////////////////////////////////////////////////////////////////
void execTransformStrict(Nd4jPointer *extraPointers,int opNum,
                                    OpaqueDataBuffer *dbX, Nd4jLong const* hXShapeInfo, Nd4jLong const* dXShapeInfo,
                                    OpaqueDataBuffer *dbZ, Nd4jLong const* hZShapeInfo, Nd4jLong const* dZShapeInfo,
                                    void *extraParams) {
    try {
        InteropDataBuffer::prepareSpecialUse({dbZ}, {dbX});

        auto tadShapeInfo = reinterpret_cast<Nd4jLong *>(extraPointers != nullptr ? extraPointers[10] : nullptr);
        auto tadOffsets = reinterpret_cast<Nd4jLong *>(extraPointers != nullptr ? extraPointers[11] : nullptr);

        LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
        NativeOpExecutioner::execTransformStrict(&lc, opNum,
                dbX->primary(), hXShapeInfo, dbX->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hXShapeInfo).special(),
                dbZ->primary(), hZShapeInfo, dbZ->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hZShapeInfo).special(),
                extraParams,
                tadShapeInfo, tadOffsets);

        InteropDataBuffer::registerSpecialUse({dbZ}, {dbX});
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
    }
}

////////////////////////////////////////////////////////////////////////
void execTransformFloat(Nd4jPointer *extraPointers,int opNum,
                                    OpaqueDataBuffer *dbX, Nd4jLong const* hXShapeInfo, Nd4jLong const* dXShapeInfo,
                                    OpaqueDataBuffer *dbZ, Nd4jLong const* hZShapeInfo, Nd4jLong const* dZShapeInfo,
                                    void *extraParams) {
    try {
        InteropDataBuffer::prepareSpecialUse({dbZ}, {dbX});

        auto tadShapeInfo = reinterpret_cast<Nd4jLong *>(extraPointers != nullptr ? extraPointers[10] : nullptr);
        auto tadOffsets = reinterpret_cast<Nd4jLong *>(extraPointers != nullptr ? extraPointers[11] : nullptr);

        LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
        NativeOpExecutioner::execTransformFloat(&lc, opNum,
                dbX->primary(), hXShapeInfo, dbX->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hXShapeInfo).special(),
                dbZ->primary(), hZShapeInfo, dbZ->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hZShapeInfo).special(),
                extraParams,
                tadShapeInfo, tadOffsets);

        InteropDataBuffer::registerSpecialUse({dbZ}, {dbX});
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
    }
}

void checkP2P() {
	int curDevice = 0;

	hipGetDevice(&curDevice);

	int devCnt = 0;
	hipGetDeviceCount(&devCnt);

	if (curDevice < 0 && curDevice > devCnt)
		curDevice = 0;

	bool tempSupport = true;

	if (devCnt > 1) {
		for (int dX = 0; dX < devCnt; dX++) {

			for (int dY = 0; dY < devCnt; dY++) {
				if (dX == dY)
					continue;

				int canAccess = 0;
				hipSetDevice(dX);

				hipDeviceCanAccessPeer(&canAccess, dX , dY);

				if (!canAccess) {
                    tempSupport = false;
                    break;
                }
			}
		}

		supportedP2P = tempSupport;

		hipSetDevice(curDevice);
	} else {
		// if we have only 1 device - we say that we support P2P, since all data will be on 1 device
		supportedP2P = true;
	}
}

void enableP2P(bool enable) {
    if (enable == allowedP2P)
        return;

    int curDevice = 0;

    hipGetDevice(&curDevice);

    int devCnt = 0;
    hipGetDeviceCount(&devCnt);

	if (curDevice < 0 && curDevice > devCnt)
		curDevice = 0;

    if (devCnt > 1) {
        for (int dX = 0; dX < devCnt; dX++) {

            for (int dY = 0; dY < devCnt; dY++) {
                if (dX == dY)
                    continue;

                int canAccess = 0;
                hipSetDevice(dX);

                hipDeviceCanAccessPeer(&canAccess, dX , dY);

                if (canAccess) {
                    if (enable) {
                        hipDeviceEnablePeerAccess(dY, 0);
                    } else {
                        hipDeviceDisablePeerAccess(dY);
                    }
                } else {
					if (sd::Environment::getInstance().isVerbose()) printf("Peer access [%i] -> [%i] isn't possible\n", dX, dY);
				}
            }
        }

        hipSetDevice(curDevice);
    }

    allowedP2P = enable;

    hipSetDevice(curDevice);
}

bool isP2PAvailable() {
	return supportedP2P;
}


void initializeDevicesAndFunctions() {
    try {
        int devCnt = 0;
        hipGetDeviceCount(&devCnt);
        deviceProperties = new hipDeviceProp_t[devCnt];
        for (int i = 0; i < devCnt; i++) {
            hipSetDevice(i);
            hipGetDeviceProperties(&deviceProperties[i], i);

            hipDeviceSetLimit(hipLimitStackSize, 4096);
        }

        hipSetDevice(0);

        checkP2P();

        // enabling p2p gpu access if it's supported
        if (supportedP2P && devCnt > 1)
            enableP2P(allowedP2P);
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
    }
}

void initializeFunctions(Nd4jPointer *functions) {
    sd::BlasHelper::getInstance().initializeDeviceFunctions(functions);
	/*
	hipblasSgemv = (CublasSgemv)functions[0];
    hipblasDgemv = (CublasDgemv)functions[1];
    hipblasHgemm = (CublasHgemm)functions[2];
    hipblasSgemm = (CublasSgemm)functions[3];
    hipblasDgemm = (CublasDgemm)functions[4];
    cublasSgemmEx = (CublasSgemmEx)functions[5];
    hipblasHgemmBatched = (CublasHgemmBatched)functions[6];
    hipblasSgemmBatched = (CublasSgemmBatched)functions[7];
    hipblasDgemmBatched = (CublasDgemmBatched)functions[8];
	*/
}


/**
 * This method acquires memory chunk of requested size on host side
 *
 * @param pointer pointer that'll be used for allocation
 * @param memorySize memory size, in bytes
 * @param flags optional parameter
 */
Nd4jPointer mallocHost(Nd4jLong memorySize, int flags) {
	Nd4jPointer pointer;
	// hipHostMallocMapped |hipHostMallocPortable
	auto res = hipHostAlloc(reinterpret_cast<void **>(&pointer), memorySize + 8, hipHostMallocDefault);
	if (res != 0) {
	    sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(res);
	    sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage("hipHostAlloc failed");
    }

	return reinterpret_cast<int8_t*>(pointer);
}

/**
 * This method acquires memory chunk of requested size on specified device
 *
 * @param pointer pointer that'll be used for allocation
 * @param memorySize memory size, in bytes
 * @param ptrToDeviceId pointer to deviceId. For cuda that's just and int, for OpenCL that's pointer to device_id, etc
 * @param flags optional parameter
 */
Nd4jPointer mallocDevice(Nd4jLong memorySize, int deviceId, int flags) {
	Nd4jPointer pointer;
	auto res = hipMalloc(reinterpret_cast<void **>(&pointer), memorySize + 8);
	if (res != 0) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(res);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage("hipMalloc failed");
	}

	return reinterpret_cast<int8_t*>(pointer);
}

/**
 * This method releases previously allocated host memory space
 *
 * @param pointer pointer that'll be freed
 */
int freeHost(Nd4jPointer pointer) {
	auto res = hipHostFree(reinterpret_cast<void *>(pointer));
	if (res != 0) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(res);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage("hipHostFree failed");
    }

	return 1L;
}

/**
 * This method releases previously allocated memory space on device
 *
 * @param pointer pointer that'll be freed
 * @param ptrToDeviceId pointer to deviceId.
 */
int freeDevice(Nd4jPointer pointer, int deviceId) {
	auto res = hipFree(reinterpret_cast<void *>(pointer));

	// we're intentionally skipping
	if (res != 0 && res != 1) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(res);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage("hipFree failed");
	}

	return res == 0 ? 1L : 0L;
}


Nd4jPointer createContext() {
	return 0L;
}

Nd4jPointer createStream() {

    auto stream = new hipStream_t();
    auto dZ = hipStreamCreate(stream);
    if (dZ != 0) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(dZ);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage("hipStreamCreate failed");
    }

    return stream;
}

Nd4jPointer createEvent() {
	Nd4jPointer nativeEvent= (Nd4jPointer) malloc(sizeof(hipEvent_t));

	CHECK_ALLOC(nativeEvent, "Failed to allocate new CUDA event buffer", sizeof(hipEvent_t));

	auto dZ = hipEventCreateWithFlags(reinterpret_cast<hipEvent_t *>(&nativeEvent), hipEventDisableTiming);
	if (dZ != 0) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(dZ);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage("hipEventCreateWithFlags failed");
	}

	return nativeEvent;
}

int registerEvent(Nd4jPointer event, Nd4jPointer stream) {
	auto pEvent = reinterpret_cast<hipEvent_t *>(&event);
	auto pStream = reinterpret_cast<hipStream_t *>(stream);

    auto dZ = hipEventRecord(*pEvent, *pStream);
	if (dZ != 0) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(dZ);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage("hipEventRecord failed");
	}

	return 1;
}

int setDevice(int deviceId) {
	AffinityManager::setCurrentDevice(deviceId);
	return 1;
}

Nd4jLong getDeviceFreeMemoryDefault() {
    size_t memFree = 0;
    size_t memTotal = 0;

    hipMemGetInfo(&memFree, &memTotal);

    return (Nd4jLong) memFree;
}

Nd4jLong getDeviceFreeMemory(int device) {
	int orig = -1;

	hipGetDevice(&orig);

	if (device >= 0 && device != orig) {
		hipSetDevice(device);
	}

	size_t memFree = 0;
	size_t memTotal = 0;

	hipMemGetInfo(&memFree, &memTotal);

	if (device >= 0 && device != orig) {
		hipSetDevice(orig);
	}

	return (Nd4jLong) memFree;
}

Nd4jLong getDeviceTotalMemory(int device) {
	int orig = -1;

	hipGetDevice(&orig);

	if (device >= 0 && device != orig) {
		hipSetDevice(device);
	}
	size_t memFree = 0;
	size_t memTotal = 0;

	hipMemGetInfo(&memFree, &memTotal);

	if (device >= 0 && device != orig) {
		hipSetDevice(orig);
	}

	return (Nd4jLong) memTotal;
}

int memcpySync(Nd4jPointer dst, Nd4jPointer src, Nd4jLong size, int flags, Nd4jPointer reserved) {
    hipMemcpyKind 	kind;

    switch (flags) {
        case 0: {
            kind = hipMemcpyHostToHost;
        }
            break;
        case 1: {
            kind = hipMemcpyHostToDevice;
        }
            break;
        case 2: {
            kind = hipMemcpyDeviceToHost;
        }
            break;
        case 3: {
            kind = hipMemcpyDeviceToDevice;
        }
            break;
        default: {
            sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
            sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage("UNDEFNED MEMCPY");
            return 0;
        }
    }

    auto dZ = hipMemcpy(reinterpret_cast<void *>(dst), const_cast<const void *>(reinterpret_cast<void *>(src)), static_cast<size_t>(size), kind);
    if (dZ != 0) {
        printf("Failed on [%p] -> [%p], size: [%i], direction: [%i], dZ: [%i]\n", src, dst, size, flags, static_cast<int>(dZ));
        fflush(stdout);
        fflush(stderr);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(dZ);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage("hipMemcpy failed");
        return 0;
    }

    return 1;
}

int memcpyAsync(Nd4jPointer dst, Nd4jPointer src, Nd4jLong size, int flags, Nd4jPointer reserved) {
	auto pStream = reinterpret_cast<hipStream_t *>(reserved);

	hipMemcpyKind 	kind;

	//sd::DebugHelper::checkErrorCode(pStream, "Preliminary sync failed");

	switch (flags) {
		case 0: {
				kind = hipMemcpyHostToHost;
			}
			break;
		case 1: {
				kind = hipMemcpyHostToDevice;
			}
			break;
		case 2: {
				kind = hipMemcpyDeviceToHost;
			}
            break;
		case 3: {
			    kind = hipMemcpyDeviceToDevice;
		    }
			break;
		default: {
            sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
            sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage("UNDEFNED MEMCPY");
            return 0;
		}
	}

	auto dZ = hipMemcpyAsync(reinterpret_cast<void *>(dst), const_cast<const void *>(reinterpret_cast<void *>(src)), static_cast<size_t>(size), kind, *pStream);
    //auto dZ = hipMemcpy(reinterpret_cast<void *>(dst), const_cast<const void *>(reinterpret_cast<void *>(src)), static_cast<size_t>(size), kind);
	if (dZ != 0) {
        printf("Failed on [%p] -> [%p], size: [%i], direction: [%i], dZ: [%i]\n", src, dst, size, flags, static_cast<int>(dZ));
        fflush(stdout);
        fflush(stderr);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(dZ);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage("hipMemcpyAsync failed");
        return 0;
	}

	return 1;
}

int memsetSync(Nd4jPointer dst, int value, Nd4jLong size, int flags, Nd4jPointer reserved) {
	auto dZ = hipMemset(reinterpret_cast<void *>(dst), value, static_cast<size_t>(size));
	if (dZ != 0) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(dZ);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage("hipMemset failed");
	}

	return 1;
}

int memsetAsync(Nd4jPointer dst, int value, Nd4jLong size, int flags, Nd4jPointer reserved) {
	auto pStream = reinterpret_cast<hipStream_t *>(reserved);

	auto dZ = hipMemsetAsync(reinterpret_cast<void *>(dst), value, static_cast<size_t>(size), *pStream);
	if (dZ != 0) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(dZ);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage("hipMemsetAsync failed");
	}

	return 1;
}

int destroyEvent(Nd4jPointer event) {
	auto pEvent = reinterpret_cast<hipEvent_t *>(&event);
	auto dZ = hipEventDestroy(*pEvent);
	if (dZ != 0) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(dZ);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage("hipEventDestroy failed");
	}

	return 1;
}

int streamSynchronize(Nd4jPointer stream) {
	auto pStream = reinterpret_cast<hipStream_t *>(stream);

	auto dZ = hipStreamSynchronize(*pStream);
	if (dZ != 0) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(dZ);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage("hipStreamSynchronize failed");
	}

	return 1L;
}

int eventSynchronize(Nd4jPointer event) {
	auto pEvent = reinterpret_cast<hipEvent_t *>(&event);

	auto  dZ = hipEventSynchronize(*pEvent);
	if (dZ != 0) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(dZ);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage("hipEventSynchronize failed");
	}

	return 1L;
}

int getAvailableDevices() {
	int devCnt = 0;
	hipGetDeviceCount(&devCnt);
	return devCnt;
}

void enableDebugMode(bool reallyEnable) {
	sd::Environment::getInstance().setDebug(reallyEnable);
}

void setGridLimit(int gridSize) {
	if (gridSize > 8192)
		gridSize = 8192;
	if (gridSize < 1)
		gridSize = 1;
	blockLimit = gridSize;
}

int ompGetMaxThreads() {
	return maxThreads;
}

int ompGetNumThreads() {
	return maxThreads;
}

void setOmpNumThreads(int threads) {
	if (threads > 1024)
		threads = 1024;
	if (threads < 32)
		threads = 32;
	maxThreads = threads;
}

void enableVerboseMode(bool reallyEnable) {
	sd::Environment::getInstance().setVerbose(reallyEnable);
}

int getDeviceMajor(int device) {
	return deviceProperties[device].major;
}

int getDeviceMinor(int device) {
	return deviceProperties[device].minor;
}


const char * getDeviceName(int device) {
    return deviceProperties[device].name;
}

void specialConcat(
        Nd4jPointer *extraPointers,
        int dimension,
        int numArrays,
        Nd4jPointer *data,
        Nd4jPointer *inputShapeInfo,
        void *dZ,
        Nd4jLong const* dZShapeInfo, Nd4jPointer *tadPointers, Nd4jPointer *offsetPointers) {
    try {
        BUILD_SINGLE_SELECTOR(ArrayOptions::dataType(dZShapeInfo), sd::SpecialMethods,
                              ::concatCpuGeneric(dimension, numArrays, data, inputShapeInfo, dZ, dZShapeInfo),
                              LIBND4J_TYPES);
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
    }
}


/**
 * This method saves
 */
sd::TadPack* tadOnlyShapeInfo(Nd4jLong const* dXShapeInfo, int *dimension, int dimensionLength) {
    try {
        auto pack = new TadPack();
        *pack = sd::ConstantTadHelper::getInstance().tadForDimensions(dXShapeInfo, dimension, dimensionLength);
        return pack;
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
        return nullptr;
    }
}

Nd4jLong const* getPrimaryShapeInfo(sd::TadPack* pack) {
    return pack->primaryShapeInfo();
}
Nd4jLong const* getPrimaryOffsets(sd::TadPack* pack) {
    return pack->primaryOffsets();
}
Nd4jLong const* getSpecialShapeInfo(sd::TadPack* pack) {
    return pack->specialShapeInfo();
}
Nd4jLong const* getSpecialOffsets(sd::TadPack* pack) {
    return pack->specialOffsets();
}
Nd4jLong getNumberOfTads(sd::TadPack* pack) {
    return pack->numberOfTads();
}
int getShapeInfoLength(sd::TadPack* pack) {
    return pack->shapeInfoLength();
}

int memcpyConstantAsync(Nd4jLong dst, Nd4jPointer src, Nd4jLong size, int flags, Nd4jPointer reserved) {
	hipStream_t *pStream = reinterpret_cast<hipStream_t *>(reserved);

	hipMemcpyKind 	kind;

	DEBUG_KERNEL(pStream, -1);

	switch (flags) {
		case 0: {
			kind = hipMemcpyHostToHost;
		}
			break;
		case 1: {
			kind = hipMemcpyHostToDevice;
		}
			break;
		case 2: {
			kind = hipMemcpyDeviceToHost;
		}
		case 3: {
			kind = hipMemcpyDeviceToDevice;
		}
			break;
	}
	auto dZ = hipMemcpyToSymbolAsync(HIP_SYMBOL(deviceConstantMemory), const_cast<const void *>(src), size, dst, kind, *pStream);
	if (dZ != 0) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(dZ);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage("hipMemcpyToSymbolAsync failed");
	}

	return 1;
}

Nd4jPointer getConstantSpace() {
	Nd4jPointer dConstAddr;
	hipError_t dZ = hipGetSymbolAddress(reinterpret_cast<void **>(&dConstAddr), deviceConstantMemory);

	if (dZ != 0) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(dZ);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage("hipGetSymbolAddress failed");
	}

	return dConstAddr;
}

void pullRows(Nd4jPointer *extraPointers,
                         OpaqueDataBuffer *dbX, Nd4jLong const* xShapeInfo, Nd4jLong const* dXShapeInfo,
                         OpaqueDataBuffer *dbZ, Nd4jLong const* zShapeInfo, Nd4jLong const* dZShapeInfo,
						 Nd4jLong n,
						 Nd4jLong *indexes,
						 Nd4jLong const* tadShapeInfo,
						 Nd4jLong const* tadOffsets,
						 Nd4jLong const* zTadShapeInfo,
						 Nd4jLong const* zTadOffsets) {
    try {
        InteropDataBuffer::prepareSpecialUse({dbZ}, {dbX});

        hipStream_t *stream = reinterpret_cast<hipStream_t *>(extraPointers[1]);
        dim3 launchDims(64, 256, 1024);
        auto xType = sd::ArrayOptions::dataType(xShapeInfo);
        BUILD_SINGLE_SELECTOR(xType, pullRowsKernelGeneric,
                              (launchDims, stream, dbX->special(), dbZ->special(), n, indexes, tadShapeInfo, tadOffsets, zTadShapeInfo, zTadOffsets),
                              LIBND4J_TYPES);

        DEBUG_KERNEL(stream, -1);

        InteropDataBuffer::registerSpecialUse({dbZ}, {dbX});
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
    }
}


void average(Nd4jPointer *extras,
						Nd4jPointer *x, Nd4jLong const* xShapeInfo,
						Nd4jPointer *dx, Nd4jLong const* dXShapeInfo,
						void *z, Nd4jLong const* zShapeInfo,
						void *dz, Nd4jLong const* dzShapeInfo,
						int n,
						Nd4jLong length,
						bool propagate) {
    try {
        hipStream_t *stream = reinterpret_cast<hipStream_t *>(extras[1]);
        int mode = getDeviceId(extras[3]);

        auto dX = reinterpret_cast<void **>(dx);

        if (sd::Environment::getInstance().isDebugAndVerbose())
            printf("averageFloat called\n");

        auto xType = sd::ArrayOptions::dataType(xShapeInfo);
        // launching on gpu
        if (mode == 0) {
            dim3 launchDims(256, 256, 4096);
            BUILD_SINGLE_SELECTOR(xType, averagingKernelGeneric, (launchDims, stream, dX, dz, n, length, propagate),
                                  LIBND4J_TYPES);
            sd::DebugHelper::checkErrorCode(stream, "AverageFloat(...) failed");
        } else {
            // launching on host memory
            BUILD_SINGLE_SELECTOR(xType, sd::SpecialMethods, ::averageGeneric(x, z, zShapeInfo, n, length, propagate),
                                  LIBND4J_TYPES);
        }
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
    }
}

void accumulate(Nd4jPointer *extras,
						   Nd4jPointer *x, Nd4jLong const* xShapeInfo,
						   Nd4jPointer *dx, Nd4jLong const* dXShapeInfo,
						   void *z, Nd4jLong const* zShapeInfo,
						   void *dz, Nd4jLong const* dzShapeInfo,
						   int n,
						   Nd4jLong length) {
    try {
        auto stream = reinterpret_cast<hipStream_t *>(extras[1]);
        int mode = getDeviceId(extras[3]);

        auto dX = reinterpret_cast<void **>(dx);

        if (sd::Environment::getInstance().isDebugAndVerbose())
            printf("accumulateFloat called\n");
        auto xType = sd::ArrayOptions::dataType(xShapeInfo);

        // launching on gpu
        if (mode == 0) {
            dim3 launchDims(n, 256, 16384);
            BUILD_SINGLE_SELECTOR(xType, accumulateKernelGeneric, (launchDims, stream, dX, dz, n, length),
                                  LIBND4J_TYPES);
            sd::DebugHelper::checkErrorCode(stream, "AccumulateFloat(...) failed");
        } else {
            // launching on host memory
            BUILD_SINGLE_SELECTOR(xType, sd::SpecialMethods, ::accumulateGeneric(x, z, zShapeInfo, n, length),
                                  LIBND4J_TYPES);
        }
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
    }
}


void shuffle(Nd4jPointer *extras,
						Nd4jPointer *x, Nd4jPointer *xShapeInfo,
						Nd4jPointer *dx, Nd4jPointer *dXShapeInfo,
						Nd4jPointer *z, Nd4jPointer *zShapeInfo,
						Nd4jPointer *dz, Nd4jPointer *dZShapeInfo,
						int N,
						int *shuffleMap,
						Nd4jPointer *tadShapeInfo,
						Nd4jPointer *tadOffsets) {
    try {
        hipStream_t *stream = reinterpret_cast<hipStream_t *>(extras[1]);

        auto dX = reinterpret_cast<void **>(dx);
        auto dZ = reinterpret_cast<void **>(dz);
        auto xShape = reinterpret_cast<Nd4jLong**>(xShapeInfo);
        auto dxShape = reinterpret_cast<Nd4jLong**>(dXShapeInfo);
        auto tadOnlyShapeInfo = reinterpret_cast<Nd4jLong **>(tadShapeInfo);
        auto tadOffset = reinterpret_cast<Nd4jLong **>(tadOffsets);

        auto xType = sd::ArrayOptions::dataType(xShape[0]);
        dim3 launchDims(256, 512, 8192);
        BUILD_SINGLE_SELECTOR(xType, shuffleKernelGeneric,
                              (launchDims, stream, dX, dxShape, dZ, N, shuffleMap, tadOnlyShapeInfo, tadOffset),
                              LIBND4J_TYPES);

        sd::DebugHelper::checkErrorCode(stream, "shuffle(...) failed");
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
    }
}

bool isExperimentalEnabled() {
    return sd::Environment::getInstance().isExperimentalBuild();
}

void setOmpMinThreads(int threads) {
    minThreads = sd::math::nd4j_max<int>(32, threads);
    minThreads = sd::math::nd4j_min<int>(maxThreads, minThreads);
}

int getDevice() {
    return sd::AffinityManager::currentDeviceId();
}

void setElementThreshold(int num) {
    // this is no-op for CUDA
}

void setTADThreshold(int num) {
    // this is no-op for CUDA
}

////////////////////////////////////////////////////////////////////////
void execSummaryStats(Nd4jPointer *extraPointers,
                                 int opNum,
                                 OpaqueDataBuffer *dbX, Nd4jLong const* hXShapeInfo, Nd4jLong const* dXShapeInfo,
                                 void *extraParams,
                                 OpaqueDataBuffer *dbZ, Nd4jLong const* hZShapeInfo, Nd4jLong const* dZShapeInfo,
                                 bool biasCorrected) {
    try {
        InteropDataBuffer::prepareSpecialUse({dbZ}, {dbX});

        LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
        NativeOpExecutioner::execSummaryStats(&lc, opNum,
                dbX->primary(), hXShapeInfo, dbX->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hXShapeInfo).special(),
                extraParams,
                dbZ->primary(), hZShapeInfo, dbZ->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hZShapeInfo).special(),
                biasCorrected);

        InteropDataBuffer::registerSpecialUse({dbZ}, {dbX});
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
    }
}

////////////////////////////////////////////////////////////////////////
void execSummaryStatsTad(Nd4jPointer *extraPointers,
                                 int opNum,
                                 OpaqueDataBuffer *dbX, Nd4jLong const* hXShapeInfo, Nd4jLong const* dXShapeInfo,
                                 void *extraParams,
                                 OpaqueDataBuffer *dbZ, Nd4jLong const* hZShapeInfo, Nd4jLong const* dZShapeInfo,
                                 OpaqueDataBuffer *dbDimension, Nd4jLong const* hDimensionShape, Nd4jLong const* dDimensionShape,
                                 bool biasCorrected,
								 Nd4jLong const* tadShapeInfo, Nd4jLong const* tadOffsets) {
    try {
        InteropDataBuffer::prepareSpecialUse({dbZ}, {dbX, dbDimension});
        InteropDataBuffer::preparePrimaryUse({}, {dbDimension});

        auto dimension = reinterpret_cast<int *>(dbDimension->primary());
        int dimensionLength = static_cast<int>(shape::length(hDimensionShape));

        LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
        NativeOpExecutioner::execSummaryStats(&lc, opNum,
                dbX->primary(), hXShapeInfo, dbX->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hXShapeInfo).special(),
                extraParams,
                dbZ->primary(), hZShapeInfo, dbZ->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hZShapeInfo).special(),
                reinterpret_cast<int *>(dbDimension->special()), dimensionLength,
                tadShapeInfo, tadOffsets,
                biasCorrected);

        InteropDataBuffer::registerSpecialUse({dbZ}, {dbX, dbDimension});
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
    }
}

////////////////////////////////////////////////////////////////////////
void execReduce3(Nd4jPointer *extraPointers,
                            int opNum,
                            OpaqueDataBuffer *dbX, Nd4jLong const* hXShapeInfo, Nd4jLong const* dXShapeInfo,
                            void *extraParams,
                            OpaqueDataBuffer *dbY, Nd4jLong const* hYShapeInfo, Nd4jLong const* dYShapeInfo,
                            OpaqueDataBuffer *dbZ, Nd4jLong const* hZShapeInfo, Nd4jLong const* dZShapeInfo) {
    try {
        InteropDataBuffer::prepareSpecialUse({dbZ}, {dbX, dbY});

        LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
        NativeOpExecutioner::execReduce3(&lc, opNum,
                dbX->primary(), hXShapeInfo, dbX->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hXShapeInfo).special(),
                extraParams,
                dbY->primary(), hYShapeInfo, dbY->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hYShapeInfo).special(),
                dbZ->primary(), hZShapeInfo, dbZ->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hZShapeInfo).special());

        InteropDataBuffer::registerSpecialUse({dbZ}, {dbX, dbY});
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
    }
}

////////////////////////////////////////////////////////////////////////
void execReduce3Tad(Nd4jPointer *extraPointers,
                            int opNum,
                            OpaqueDataBuffer *dbX, Nd4jLong const* hXShapeInfo, Nd4jLong const* dXShapeInfo,
                            void *extraParams,
                            OpaqueDataBuffer *dbY, Nd4jLong const* hYShapeInfo, Nd4jLong const* dYShapeInfo,
                            OpaqueDataBuffer *dbZ, Nd4jLong const* hZShapeInfo, Nd4jLong const* dZShapeInfo,
                            OpaqueDataBuffer *dbDimension, Nd4jLong const* hDimensionShape, Nd4jLong const* dDimensionShape,
                            Nd4jLong const* tadOnlyShapeInfo, Nd4jLong const* tadOffsets,
                            Nd4jLong const* yTadOnlyShapeInfo, Nd4jLong const* yTadOffsets) {
    try {
        InteropDataBuffer::prepareSpecialUse({dbZ}, {dbX, dbY});
        InteropDataBuffer::preparePrimaryUse({}, {dbDimension});

        auto dimension = reinterpret_cast<int *>(dbDimension->primary());
        int dimensionLength = static_cast<int>(shape::length(hDimensionShape));

        auto tadPack = sd::ConstantTadHelper::getInstance().tadForDimensions(hXShapeInfo,
                                                                                dimension,
                                                                                shape::length(hDimensionShape));
        auto tadLength = shape::length(tadPack.primaryShapeInfo());
        auto yLength = shape::length(hYShapeInfo);
        auto xLength = shape::length(hXShapeInfo);

        LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);

        if (tadLength == yLength || tadLength == xLength) {
            // nd4j_printf("== way\n","");
            NativeOpExecutioner::execReduce3(&lc, opNum,
                    dbX->primary(), hXShapeInfo, dbX->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hXShapeInfo).special(),
                    extraParams,
                    dbY->primary(), hYShapeInfo, dbY->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hYShapeInfo).special(),
                    dbZ->primary(), hZShapeInfo, dbZ->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hZShapeInfo).special(),
                    dimension, dimensionLength,
                    tadOnlyShapeInfo, tadOffsets, yTadOnlyShapeInfo, yTadOffsets);
        } else
            NativeOpExecutioner::execReduce3TAD(&lc, opNum,
                    dbX->primary(), hXShapeInfo, dbX->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hXShapeInfo).special(),
                    extraParams,
                    dbY->primary(), hYShapeInfo, dbY->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hYShapeInfo).special(),
                    dbZ->primary(), hZShapeInfo, dbZ->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hZShapeInfo).special(),
                    dimension, dimensionLength,
                    tadOnlyShapeInfo, yTadOffsets, yTadOnlyShapeInfo, yTadOffsets);

        InteropDataBuffer::registerSpecialUse({dbZ}, {dbX, dbY});
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
    }
}

////////////////////////////////////////////////////////////////////////
void execReduce3Scalar(Nd4jPointer *extraPointers,int opNum,
                                    OpaqueDataBuffer *dbX, Nd4jLong const* hXShapeInfo, Nd4jLong const* dXShapeInfo,
                                    void *extraParams,
                                    OpaqueDataBuffer *dbY, Nd4jLong const* hYShapeInfo, Nd4jLong const* dYShapeInfo,
                                    OpaqueDataBuffer *dbZ, Nd4jLong const* hZShapeInfo, Nd4jLong const* dZShapeInfo) {
    try {
        InteropDataBuffer::prepareSpecialUse({dbZ}, {dbX, dbY});

        LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
        NativeOpExecutioner::execReduce3Scalar(&lc, opNum,
                dbX->primary(), hXShapeInfo, dbX->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hXShapeInfo).special(),
                extraParams,
                dbY->primary(), hYShapeInfo, dbY->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hYShapeInfo).special(),
                dbZ->primary(), hZShapeInfo, dbZ->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hZShapeInfo).special());

        InteropDataBuffer::registerSpecialUse({dbZ}, {dbX, dbY});
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
    }
}

////////////////////////////////////////////////////////////////////////
void execScalarBool(Nd4jPointer *extraPointers,
							int opNum,
                            OpaqueDataBuffer *dbX, Nd4jLong const* hXShapeInfo, Nd4jLong const* dXShapeInfo,
                            OpaqueDataBuffer *dbZ, Nd4jLong const* hZShapeInfo, Nd4jLong const* dZShapeInfo,
                            OpaqueDataBuffer *dbScalar, Nd4jLong const* hScalarShapeInfo, Nd4jLong const* dScalarShapeInfo,
							void *extraParams) {
    try {
        InteropDataBuffer::prepareSpecialUse({dbZ}, {dbX, dbScalar});

        LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
        NativeOpExecutioner::execScalarBool(&lc, opNum,
                dbX->primary(), hXShapeInfo, dbX->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hXShapeInfo).special(),
                dbZ->primary(), hZShapeInfo, dbZ->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hZShapeInfo).special(),
                dbScalar->primary(), hScalarShapeInfo, dbScalar->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hScalarShapeInfo).special(),
                extraParams);

        InteropDataBuffer::registerSpecialUse({dbZ}, {dbX, dbScalar});
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
    }
}

////////////////////////////////////////////////////////////////////////
void execScalarBoolTad(Nd4jPointer *extraPointers,
						   int opNum,
						   OpaqueDataBuffer *dbX, Nd4jLong const* hXShapeInfo, Nd4jLong const* dXShapeInfo,
						   OpaqueDataBuffer *dbZ, Nd4jLong const* hZShapeInfo, Nd4jLong const* dZShapeInfo,
						   OpaqueDataBuffer *dbScalars, Nd4jLong const* hScalarShapeInfo, Nd4jLong const* dScalarShapeInfo,
						   void *extraParams,
						   OpaqueDataBuffer *dbDimension, Nd4jLong const* hDimensionShape, Nd4jLong const* dDimensionShape,
                           Nd4jLong const* tadShapeInfo, Nd4jLong const* tadOffsets,
                           Nd4jLong const* tadShapeInfoZ, Nd4jLong const* tadOffsetsZ) {
    try {
        InteropDataBuffer::prepareSpecialUse({dbZ}, {dbX, dbScalars});
        InteropDataBuffer::preparePrimaryUse({}, {dbDimension});

        auto dimension = reinterpret_cast<int *>(dbDimension->primary());
        int dimensionLength = static_cast<int>(shape::length(hDimensionShape));

        LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
        NativeOpExecutioner::execScalarBool(&lc, opNum,
                dbX->primary(), hXShapeInfo, dbX->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hXShapeInfo).special(),
                extraParams,
                dbZ->primary(), hZShapeInfo, dbZ->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hZShapeInfo).special(),
                dbScalars->primary(), hScalarShapeInfo, dbScalars->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hScalarShapeInfo).special(),
                dimension, dimensionLength,
                tadShapeInfo, tadOffsets, tadShapeInfoZ, tadOffsetsZ);

        InteropDataBuffer::registerSpecialUse({dbZ}, {dbX, dbScalars});
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
    }
}

////////////////////////////////////////////////////////////////////////
void execScalar(Nd4jPointer *extraPointers,
						int opNum,
                        OpaqueDataBuffer *dbX, Nd4jLong const* hXShapeInfo, Nd4jLong const* dXShapeInfo,
                        OpaqueDataBuffer *dbZ, Nd4jLong const* hZShapeInfo, Nd4jLong const* dZShapeInfo,
                        OpaqueDataBuffer *dbScalar, Nd4jLong const* hScalarShapeInfo, Nd4jLong const* dScalarShapeInfo,
						void *extraParams) {
    try {
        InteropDataBuffer::prepareSpecialUse({dbZ}, {dbX, dbScalar});

        LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
        NativeOpExecutioner::execScalar(&lc, opNum,
                dbX->primary(), hXShapeInfo, dbX->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hXShapeInfo).special(),
                dbZ->primary(), hZShapeInfo, dbZ->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hZShapeInfo).special(),
                dbScalar->primary(), hScalarShapeInfo, dbScalar->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hScalarShapeInfo).special(),
                        extraParams);

        InteropDataBuffer::registerSpecialUse({dbZ}, {dbX, dbScalar});
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
    }
}

////////////////////////////////////////////////////////////////////////
void execScalarTad(Nd4jPointer *extraPointers,
					 int opNum,
					 OpaqueDataBuffer *dbX, Nd4jLong const* hXShapeInfo, Nd4jLong const* dXShapeInfo,
					 OpaqueDataBuffer *dbZ, Nd4jLong const* hZShapeInfo, Nd4jLong const* dZShapeInfo,
					 OpaqueDataBuffer *dbScalars, Nd4jLong const* hScalarShapeInfo, Nd4jLong const* dScalarShapeInfo,
					 void *extraParams,
					 OpaqueDataBuffer *dbDimension, Nd4jLong const* hDimensionShape, Nd4jLong const* dDimensionShape,
                     Nd4jLong const* tadShapeInfo, Nd4jLong const* tadOffsets,
                     Nd4jLong const* tadShapeInfoZ, Nd4jLong const* tadOffsetsZ) {
    try {
        InteropDataBuffer::prepareSpecialUse({dbZ}, {dbX, dbScalars});
        InteropDataBuffer::preparePrimaryUse({}, {dbDimension});

        auto dimension = reinterpret_cast<int *>(dbDimension->primary());
        int dimensionLength = static_cast<int>(shape::length(hDimensionShape));

        hipStream_t *stream = reinterpret_cast<hipStream_t *>(extraPointers[1]);

        auto xType = sd::ArrayOptions::dataType(hXShapeInfo);
        auto yType = sd::ArrayOptions::dataType(hScalarShapeInfo);
        auto zType = sd::ArrayOptions::dataType(hZShapeInfo);

        if (yType != xType && yType != sd::DataType::BOOL && !isExperimentalEnabled())
            throw sd::datatype_exception::build("execScalar both operands must have same data type", xType, yType);

        dim3 launchDims(256, 256, 16384);

#ifdef __ND4J_EXPERIMENTAL__
        BUILD_PAIRWISE_SELECTOR(xType, yType, zType, functions::scalar::ScalarTransform, ::executeCudaAlongDimension(launchDims, stream, opNum, dX, dXShapeInfo, dZ, dZShapeInfo, dScalars, extraParams, dimension, dimensionLength, tadShapeInfo, tadOffsets, tadShapeInfoZ, tadOffsetsZ), LIBND4J_TYPES, LIBND4J_TYPES);
#else
        BUILD_SINGLE_SELECTOR_THRICE(xType, functions::scalar::ScalarTransform, ::executeCudaAlongDimension(launchDims, stream, opNum, dbX->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hXShapeInfo).special(), dbZ->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hZShapeInfo).special(), dbScalars->special(), extraParams, dimension, dimensionLength, tadShapeInfo, tadOffsets, tadShapeInfoZ, tadOffsetsZ), LIBND4J_TYPES);
#endif

        DEBUG_KERNEL(stream, opNum);

        InteropDataBuffer::registerSpecialUse({dbZ}, {dbX, dbScalars});
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
    }
}

void execAggregate(Nd4jPointer *extraPointers,
								   int opNum,
                                   void **arguments,
                                   int numArguments,
                                   Nd4jLong **shapes,
                                   int numShapes,
                                   int *indexArguments,
                                   int numIndexArguments,
                                   int **intArrays,
                                   int numIntArrays,
                                   void *realArguments,
                                   int numRealArguments,
                                   sd::DataType dtype) {

}

void batchExecutor(Nd4jPointer *extraPointers,
                               int numAggregates,
                               int opNum,
                               int maxArgs,
                               int maxShapes,
                               int maxIntArrays,
                               int maxIntArraySize,
                               int maxIdx,
                               int maxReals,
                               void *ptrToArguments,
                               sd::DataType dtype) {
}

void execAggregateBatch(Nd4jPointer *extraPointers,
									int numAggregates, int opNum,
									int maxArgs, int maxShapes,
									int maxIntArrays, int maxIntArraySize,
									int maxIdx, int maxReals,
									void *ptrToArguments, sd::DataType dtype) {

}

////////////////////////////////////////////////////////////////////////
void execRandom(Nd4jPointer *extraPointers,
						  int opNum,
                          Nd4jPointer stateHost,
                          OpaqueDataBuffer *dbZ, Nd4jLong const* hZShapeInfo, Nd4jLong const* dZShapeInfo,
                          void *extraArguments) {
    try {
        InteropDataBuffer::prepareSpecialUse({dbZ}, {});

        LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
        NativeOpExecutioner::execRandom(&lc, opNum, stateHost,
                dbZ->primary(), hZShapeInfo, dbZ->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hZShapeInfo).special(),
                extraArguments);

        InteropDataBuffer::registerSpecialUse({dbZ}, {});
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
    }
}

////////////////////////////////////////////////////////////////////////
void execRandom2(Nd4jPointer *extraPointers, int opNum, Nd4jPointer stateHost,
                           OpaqueDataBuffer *dbX, Nd4jLong const* hXShapeInfo, Nd4jLong const* dXShapeInfo,
                           OpaqueDataBuffer *dbZ, Nd4jLong const* hZShapeInfo, Nd4jLong const* dZShapeInfo,
						   void *extraArguments) {
    try {
        InteropDataBuffer::prepareSpecialUse({dbZ}, {dbX});

        LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
        NativeOpExecutioner::execRandom(&lc, opNum, stateHost,
                dbX->primary(), hXShapeInfo, dbX->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hXShapeInfo).special(),
                dbZ->primary(), hZShapeInfo, dbZ->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hZShapeInfo).special(),
                extraArguments);

        InteropDataBuffer::registerSpecialUse({dbZ}, {dbX});
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
    }
}

////////////////////////////////////////////////////////////////////////
void execRandom3(Nd4jPointer *extraPointers, int opNum, Nd4jPointer stateHost,
                            OpaqueDataBuffer *dbX, Nd4jLong const* hXShapeInfo, Nd4jLong const* dXShapeInfo,
                            OpaqueDataBuffer *dbY, Nd4jLong const* hYShapeInfo, Nd4jLong const* dYShapeInfo,
                            OpaqueDataBuffer *dbZ, Nd4jLong const* hZShapeInfo, Nd4jLong const* dZShapeInfo,
							void *extraArguments) {
    try {
        InteropDataBuffer::prepareSpecialUse({dbZ}, {dbX, dbY});

        LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
        NativeOpExecutioner::execRandom(&lc, opNum, stateHost,
                dbX->primary(), hXShapeInfo, dbX->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hXShapeInfo).special(),
                dbY->primary(), hYShapeInfo, dbY->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hYShapeInfo).special(),
                dbZ->primary(), hZShapeInfo, dbZ->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hZShapeInfo).special(),
                extraArguments);

        InteropDataBuffer::registerSpecialUse({dbZ}, {dbX, dbY});
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
    }
}


Nd4jPointer initRandom(Nd4jPointer *extraPointers, long seed, long bufferSize, Nd4jPointer ptrToBuffer) {

    unsigned long long *ptrHost = reinterpret_cast<unsigned long long *>(extraPointers[0]);
    hipStream_t *stream = reinterpret_cast<hipStream_t *>(extraPointers[1]);

    // we don't synchronize at random initialization, it's safe to go unsync here
	// hipStreamSynchronize(*stream);

    auto ptrDev = reinterpret_cast<unsigned long long *>(ptrToBuffer);
    auto buffer = new sd::random::RandomBuffer(seed, bufferSize, reinterpret_cast<uint64_t *>(ptrHost), reinterpret_cast<uint64_t *>(ptrDev));
    buffer->propagateToDevice(buffer, *stream);

    sd::DebugHelper::checkErrorCode(stream, "initRandom(...) failed A");

	// we generate sequence in the host memory
    sd::random::Xoroshiro128 generator(buffer);
    generator.refreshBuffer();

	// and copy it to gpu
    hipMemcpyAsync(ptrDev, ptrHost, bufferSize * 8, hipMemcpyHostToDevice, *stream);
    sd::DebugHelper::checkErrorCode(stream, "initRandom(...) failed B");

    return buffer;
}


void destroyRandom(Nd4jPointer ptrBuffer) {

    sd::random::RandomBuffer *buffer = reinterpret_cast<sd::random::RandomBuffer *> (ptrBuffer);

    // FIXME: it's bad thing, but we can't know in advance, which stream(s) where using this generator in practice
    hipDeviceSynchronize();

    delete buffer;
}

void refreshBuffer(Nd4jPointer *extraPointers, long seed, Nd4jPointer ptrRandom) {

    sd::random::RandomBuffer *buffer = reinterpret_cast<sd::random::RandomBuffer *> (ptrRandom);

    unsigned long long *ptrHost = reinterpret_cast<unsigned long long *>(extraPointers[0]);
    hipStream_t *stream = reinterpret_cast<hipStream_t *>(extraPointers[1]);
    hipStreamSynchronize(*stream);

    uint64_t *ptrDev = buffer->getDeviceBuffer();

	// update rng state
    buffer->setSeed(seed);
    buffer->setOffset(0);
    buffer->propagateToDevice(buffer, *stream);

	// refresh buffer on host size
    sd::random::Xoroshiro128 generator(buffer);
    generator.refreshBuffer();

	// copy back to gpu
    hipMemcpyAsync(ptrDev, ptrHost, buffer->getSize() * 8, hipMemcpyHostToDevice, *stream);
}

void reSeedBuffer(Nd4jPointer *extraPointers, long seed, Nd4jPointer ptrRandom) {

    sd::random::RandomBuffer *buffer = reinterpret_cast<sd::random::RandomBuffer *> (ptrRandom);

    hipStream_t *stream = reinterpret_cast<hipStream_t *>(extraPointers[1]);
    hipStreamSynchronize(*stream);

	// update rng state
    buffer->reSeed(seed);
    buffer->setOffset(0);
    buffer->propagateToDevice(buffer, *stream);
}



/**
    * Return the length of a shape buffer
    * based on the pointer
    * @param buffer  the buffer pointer to check
    * @return
    */
int lengthForShapeBufferPointer(Nd4jPointer buffer) {
    auto shapeBuffer = reinterpret_cast<Nd4jLong *>(buffer);
    return shape::shapeInfoLength(shape::rank(shapeBuffer));
}


/**
  * The pointer to get the address for
  *
  * @param address the address to get the pointer
  * @return the pointer for the given address
  */

Nd4jPointer pointerForAddress(Nd4jLong address) {
	return reinterpret_cast<Nd4jPointer >(address);
}

void tear(Nd4jPointer *extras,
                     OpaqueDataBuffer *dbX, Nd4jLong const* xShapeInfo, Nd4jLong const* dXShapeInfo,
					 Nd4jPointer *targets,
					 Nd4jLong const* zShapeInfo,
					 Nd4jLong const* tadShapeInfo,
					 Nd4jLong const* tadOffsets) {
    try {
        InteropDataBuffer::prepareSpecialUse({}, {dbX});

        hipStream_t *stream = reinterpret_cast<hipStream_t *>(extras[1]);
        dim3 launchDims(512, 512, 512);
        auto xType = sd::ArrayOptions::dataType(xShapeInfo);
        BUILD_SINGLE_SELECTOR(xType, tearKernelGeneric,
                              (launchDims, stream, dbX->special(), dXShapeInfo, targets, zShapeInfo, tadShapeInfo, tadOffsets),
                              LIBND4J_TYPES);

        sd::DebugHelper::checkErrorCode(stream, "tearFloat(...) failed");

        InteropDataBuffer::registerSpecialUse({}, {dbX});
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
    }
}


void prescanArrayRecursive(Nd4jPointer *extras, int *dZ, int *dX, int numElements, int level) {

    auto stream = reinterpret_cast<hipStream_t *>(extras[1]);
    auto g_scanBlockSums = reinterpret_cast<int **>(extras[2]);

    int blockSize = 512; // max size of the thread blocks
    int numBlocks = sd::math::nd4j_max<int>(1, static_cast<int>(ceil(static_cast<float>(numElements) / (2.f * blockSize))));
    int numThreads;

    if (numBlocks > 1)
        numThreads = blockSize;
    else if (sd::isPowerOfTwo(numElements))
        numThreads = numElements / 2;
    else
        numThreads = sd::floorPow2(numElements);

    int numEltsPerBlock = numThreads * 2;

    // if this is a non-power-of-2 array, the last block will be non-full
    // compute the smallest power of 2 able to compute its scan.
    int numEltsLastBlock =
            numElements - (numBlocks-1) * numEltsPerBlock;
    int numThreadsLastBlock = sd::math::nd4j_max<int>(1, numEltsLastBlock / 2);
    int np2LastBlock = 0;
    int sharedMemLastBlock = 0;

    if (numEltsLastBlock != numEltsPerBlock) {
        np2LastBlock = 1;

        if(!isPowerOfTwo(numEltsLastBlock))
            numThreadsLastBlock = floorPow2(numEltsLastBlock);

        unsigned int extraSpace = (2 * numThreadsLastBlock) / NUM_BANKS;
        sharedMemLastBlock = sizeof(int) * (2 * numThreadsLastBlock + extraSpace);
    }

    // padding space is used to avoid shared memory bank conflicts
    int extraSpace = numEltsPerBlock / NUM_BANKS;
    int sharedMemSize = sizeof(int) * (numEltsPerBlock + extraSpace);

    // setup execution parameters
    // if NP2, we process the last block separately
    dim3 grid(max(1, numBlocks - np2LastBlock), 1, 1);
    dim3 threads(numThreads, 1, 1);
    dim3 gridOnes(1, 1, 1);
    dim3 threadsOnes(numThreadsLastBlock, 1, 1);

    if (sharedMemSize < 2048)
        sharedMemSize = 2048;

    if (sharedMemLastBlock < 2048)
        sharedMemLastBlock = 2048;

    // execute the scan
    if (numBlocks > 1) {
        sd::prescanLauncher<true, false>(grid, threads, sharedMemSize, stream, dZ, dX, g_scanBlockSums[level], numThreads * 2, 0, 0);
        if (np2LastBlock) {
            sd::prescanLauncher<true, true>(gridOnes, threadsOnes, sharedMemLastBlock, stream, dZ, dX, g_scanBlockSums[level], numEltsLastBlock, numBlocks - 1, numElements - numEltsLastBlock);
        }

        // After scanning all the sub-blocks, we are mostly done.  But now we
        // need to take all of the last values of the sub-blocks and scan those.
        // This will give us a new value that must be sdded to each block to
        // get the final results.
        // recursive (CPU) call
        prescanArrayRecursive(extras, g_scanBlockSums[level], g_scanBlockSums[level], numBlocks, level+1);

        sd::uniformAdd<<<grid, threads, 1024, *stream>>>(dZ, g_scanBlockSums[level], numElements - numEltsLastBlock, 0, 0);

        if (np2LastBlock) {
            sd::uniformAdd<<<1, numThreadsLastBlock, 1024, *stream>>>(dZ, g_scanBlockSums[level], numEltsLastBlock, numBlocks - 1, numElements - numEltsLastBlock);
        }
    } else if (isPowerOfTwo(numElements)) {
        sd::prescanLauncher<false, false>(grid, threads, sharedMemSize, stream, dZ, dX, 0, numThreads * 2, 0, 0);
    } else {
        sd::prescanLauncher<false, true>(grid, threads, sharedMemSize, stream, dZ, dX, 0, numElements, 0, 0);
    }

    sd::DebugHelper::checkErrorCode(stream, "prescanArray(...) failed");
}

////////////////////////////////////////////////////////////////////////
void execReduce3All(Nd4jPointer *extraPointers,
									int opNum,
                                    OpaqueDataBuffer *dbX, Nd4jLong const* hXShapeInfo, Nd4jLong const* dXShapeInfo,
                            		void *extraParamsVals,
                                    OpaqueDataBuffer *dbY, Nd4jLong const* hYShapeInfo, Nd4jLong const* dYShapeInfo,
                                    OpaqueDataBuffer *dbZ, Nd4jLong const* hZShapeInfo, Nd4jLong const* dZShapeInfo,
                                    OpaqueDataBuffer *dbDimension, Nd4jLong const* hDimensionShape, Nd4jLong const* dDimensionShape,
									Nd4jLong const* xTadShapeInfo, Nd4jLong const* xOffsets,
									Nd4jLong const* yTadShapeInfo, Nd4jLong const* yOffsets) {
    try {
        InteropDataBuffer::prepareSpecialUse({dbZ}, {dbX, dbY, dbDimension});
        InteropDataBuffer::preparePrimaryUse({}, {dbDimension});

        auto dimension = reinterpret_cast<int *>(dbDimension->primary());
        int dimensionLength = static_cast<int>(shape::length(hDimensionShape));

        LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
        NativeOpExecutioner::execReduce3All(&lc, opNum,
                dbX->primary(), hXShapeInfo, dbX->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hXShapeInfo).special(),
                extraParamsVals,
                dbY->primary(), hYShapeInfo, dbY->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hYShapeInfo).special(),
                dbZ->primary(), hZShapeInfo, dbZ->special(), ConstantShapeHelper::getInstance().bufferForShapeInfo(hZShapeInfo).special(),
                reinterpret_cast<int *>(dbDimension->special()), dimensionLength,
                xTadShapeInfo, xOffsets, yTadShapeInfo, yOffsets);

        InteropDataBuffer::registerSpecialUse({dbZ}, {dbX, dbY});
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
    }
}


void sort(Nd4jPointer *extraPointers,
					 void *x, Nd4jLong const* xShapeInfo,
					 void *dX, Nd4jLong const* dXShapeInfo,
					 bool descending) {
    try {
        hipStream_t *stream = reinterpret_cast<hipStream_t *>(extraPointers[1]);

        auto xLength = shape::length(xShapeInfo);
        auto xEWS = shape::elementWiseStride(xShapeInfo);
        auto xType = sd::ArrayOptions::dataType(xShapeInfo);


        // check if xLength is a power of 2, and use bitonic sort, if that's the case
        if ((xLength != 0) && ((xLength & (xLength - 1)) == 0) && (xLength <= 1024 * 1024 * 10)) {
            int numThreads = sd::math::nd4j_min<int>(512, xLength);
            int numBlocks = xLength / numThreads;
            if (xLength % numThreads > 0 || numBlocks == 0)
                numBlocks++;

            dim3 launchDims(numBlocks, numThreads, 32768);

            for (int k = 2; k <= xLength; k = 2 * k) {
                for (int j = k >> 1; j > 0; j = j >> 1) {
                    BUILD_SINGLE_SELECTOR(xType, bitonicSortStepGeneric,
                                          (launchDims, stream, dX, dXShapeInfo, j, k, xLength, descending),
                                          LIBND4J_TYPES);
                }
            }
        } else {
            int numThreads = sd::math::nd4j_min<int>(512, xLength);
            int numBlocks = xLength / numThreads;
            if (xLength % numThreads > 0 || numBlocks == 0)
                numBlocks++;

            numBlocks = sd::math::nd4j_min<int>(512, numBlocks);
            dim3 launchDims(numBlocks, numThreads, 32768);

            int max = 2, dg = 0;
            while (max < xLength) {
                max <<= 1;
                dg++;
            }
            max <<= 1;

            for (int window = 2; window < max; window <<= 1) {
                int n = window;
                int rev = 0;
                do {
                    int half = n >> 1;
                    BUILD_SINGLE_SELECTOR(xType, bitonicArbitraryStepGeneric,
                                          (launchDims, stream, dX, dXShapeInfo, n, xLength, rev, descending),
                                          LIBND4J_TYPES);
                    n >>= 1;
                    rev = 1;
                } while (n > 1);
            }
        }

        sd::DebugHelper::checkErrorCode(stream, "sort(...) failed");
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
    }
}


void sortByKey(Nd4jPointer *extraPointers,
               void *x, Nd4jLong const* xShapeInfo,
               void *dX, Nd4jLong const* dXShapeInfo,
               void *y, Nd4jLong const* yShapeInfo,
               void *dy, Nd4jLong const* dyShapeInfo,
               bool descending) {
    try {
        auto stream = reinterpret_cast<hipStream_t *>(extraPointers[1]);

        auto xLength = shape::length(xShapeInfo);
        auto yLength = shape::length(yShapeInfo);
        auto xEWS = shape::elementWiseStride(xShapeInfo);
        auto xType = sd::ArrayOptions::dataType(xShapeInfo);
        auto yType = sd::ArrayOptions::dataType(yShapeInfo);

        if (shape::isEmpty(xShapeInfo) || shape::isEmpty(yShapeInfo))
            return;

        if (xLength != yLength)
            throw std::runtime_error("sortByKey: keys and values must have the same size");


        // check if xLength is a power of 2, and use bitonic sort, if that's the case
        if ((xLength != 0) && ((xLength & (xLength - 1)) == 0) && (xLength <= 1024 * 1024 * 10)) {
            int numThreads = sd::math::nd4j_min<int>(512, xLength);
            int numBlocks = xLength / numThreads;
            if (xLength % numThreads > 0 || numBlocks == 0)
                numBlocks++;

            dim3 launchDims(numBlocks, numThreads, 32768);

            for (int k = 2; k <= xLength; k = 2 * k) {
                for (int j = k >> 1; j > 0; j = j >> 1) {
                    BUILD_DOUBLE_SELECTOR(xType, yType, bitonicSortStepGenericKey,
                                          (launchDims, stream, dX, dXShapeInfo, dy, dyShapeInfo, j, k, xLength, descending),
                                          LIBND4J_TYPES, LIBND4J_TYPES);
                }
            }
        } else {
            int numThreads = sd::math::nd4j_min<int>(512, xLength);
            int numBlocks = xLength / numThreads;
            if (xLength % numThreads > 0 || numBlocks == 0)
                numBlocks++;

            numBlocks = sd::math::nd4j_min<int>(512, numBlocks);
            dim3 launchDims(numBlocks, numThreads, 32768);

            int max = 2, dg = 0;
            while (max < xLength) {
                max <<= 1;
                dg++;
            }
            max <<= 1;

            for (int window = 2; window < max; window <<= 1) {
                int n = window;
                int rev = 0;
                do {
                    int half = n >> 1;
                    BUILD_DOUBLE_SELECTOR(xType, yType, bitonicArbitraryStepGenericKey,
                                          (launchDims, stream, dX, dXShapeInfo, dy, dyShapeInfo, n, xLength, rev, descending),
                                          LIBND4J_TYPES, LIBND4J_TYPES);
                    n >>= 1;
                    rev = 1;
                } while (n > 1);
            }
        }

    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
    }
}

void sortByValue(Nd4jPointer *extraPointers,
                 void *x, Nd4jLong const* xShapeInfo,
                 void *dX, Nd4jLong const* dXShapeInfo,
                 void *y, Nd4jLong const* yShapeInfo,
                 void *dy, Nd4jLong const* dyShapeInfo,
                 bool descending) {
    try {
        auto stream = reinterpret_cast<hipStream_t *>(extraPointers[1]);

        auto xLength = shape::length(xShapeInfo);
        auto yLength = shape::length(yShapeInfo);
        auto xEWS = shape::elementWiseStride(xShapeInfo);
        auto xType = sd::ArrayOptions::dataType(yShapeInfo);
        auto yType = sd::ArrayOptions::dataType(xShapeInfo);

        if (shape::isEmpty(xShapeInfo) || shape::isEmpty(yShapeInfo))
            return;

        if (xLength != yLength)
            throw std::runtime_error("sortByValue: keys and values must have the same size");


        // check if xLength is a power of 2, and use bitonic sort, if that's the case
        if ((xLength != 0) && ((xLength & (xLength - 1)) == 0) && (xLength <= 1024 * 1024 * 10)) {
            int numThreads = sd::math::nd4j_min<int>(512, xLength);
            int numBlocks = xLength / numThreads;
            if (xLength % numThreads > 0 || numBlocks == 0)
                numBlocks++;

            dim3 launchDims(numBlocks, numThreads, 32768);

            for (int k = 2; k <= xLength; k = 2 * k) {
                for (int j = k >> 1; j > 0; j = j >> 1) {
                    BUILD_DOUBLE_SELECTOR(xType, yType, bitonicSortStepGenericKey,
                                          (launchDims, stream, dy, dyShapeInfo, dX, dXShapeInfo, j, k, xLength, descending),
                                          LIBND4J_TYPES, LIBND4J_TYPES);
                }
            }
        } else {
            int numThreads = sd::math::nd4j_min<int>(512, xLength);
            int numBlocks = xLength / numThreads;
            if (xLength % numThreads > 0 || numBlocks == 0)
                numBlocks++;

            numBlocks = sd::math::nd4j_min<int>(512, numBlocks);
            dim3 launchDims(numBlocks, numThreads, 32768);

            int max = 2, dg = 0;
            while (max < xLength) {
                max <<= 1;
                dg++;
            }
            max <<= 1;

            for (int window = 2; window < max; window <<= 1) {
                int n = window;
                int rev = 0;
                do {
                    int half = n >> 1;
                    BUILD_DOUBLE_SELECTOR(xType, yType, bitonicArbitraryStepGenericKey,
                                          (launchDims, stream, dy, dyShapeInfo, dX, dXShapeInfo, n, xLength, rev, descending),
                                          LIBND4J_TYPES, LIBND4J_TYPES);
                    n >>= 1;
                    rev = 1;
                } while (n > 1);
            }
        }
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
    }
}



void sortTadByKey(Nd4jPointer *extraPointers,
                             void *x, Nd4jLong const* xShapeInfo,
                             void *dX, Nd4jLong const* dXShapeInfo,
                             void *y, Nd4jLong const* yShapeInfo,
                             void *dy, Nd4jLong const* dyShapeInfo,
                             int *dimension,
                             int dimensionLength,
                             bool descending) {
    try {
        auto stream = reinterpret_cast<hipStream_t *>(extraPointers[1]);
        auto context = extraPointers[0] == 0 ? LaunchContext::defaultContext()
                                             : reinterpret_cast<LaunchContext *>(extraPointers[0]);
        auto tadPack = sd::ConstantTadHelper::getInstance().tadForDimensions(xShapeInfo, dimension, dimensionLength);
        dim3 launchDims((int) tadPack.numberOfTads(), 256, 2048);
        auto xType = sd::ArrayOptions::dataType(xShapeInfo);
        auto yType = sd::ArrayOptions::dataType(yShapeInfo);
        BUILD_DOUBLE_SELECTOR(xType, yType, oesTadGenericKey,
                              (launchDims, stream, dX, dXShapeInfo, dy, dyShapeInfo, nullptr, dimensionLength, tadPack.platformShapeInfo(), tadPack.platformOffsets(), descending),
                              LIBND4J_TYPES, LIBND4J_TYPES);

        sd::DebugHelper::checkErrorCode(stream, "sortTadKey(...) failed");
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
    }
}

void sortTadByValue(Nd4jPointer *extraPointers,
                               void *x, Nd4jLong const* xShapeInfo,
                               void *dX, Nd4jLong const* dXShapeInfo,
                               void *y, Nd4jLong const* yShapeInfo,
                               void *dy, Nd4jLong const* dyShapeInfo,
                               int *dimension,
                               int dimensionLength,
                               bool descending) {
    try {
        auto stream = reinterpret_cast<hipStream_t *>(extraPointers[1]);
        auto context = extraPointers[0] == 0 ? LaunchContext::defaultContext()
                                             : reinterpret_cast<LaunchContext *>(extraPointers[0]);
        auto tadPack = sd::ConstantTadHelper::getInstance().tadForDimensions(xShapeInfo, dimension, dimensionLength);
        dim3 launchDims((int) tadPack.numberOfTads(), 256, 2048);
        auto xType = sd::ArrayOptions::dataType(yShapeInfo);
        auto yType = sd::ArrayOptions::dataType(xShapeInfo);

        BUILD_DOUBLE_SELECTOR(xType, yType, oesTadGenericKey,
                              (launchDims, stream, dy, dyShapeInfo, dX, dXShapeInfo, nullptr, dimensionLength, tadPack.platformShapeInfo(), tadPack.platformOffsets(), descending),
                              LIBND4J_TYPES, LIBND4J_TYPES);

        sd::DebugHelper::checkErrorCode(stream, "sortTadValue(...) failed");
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
    }
}


void sortTad(Nd4jPointer *extraPointers,
						void *x, Nd4jLong const* xShapeInfo,
						void *dX, Nd4jLong const* dXShapeInfo,
						int *dimension,
						int dimensionLength,
						Nd4jLong const* tadShapeInfo,
						Nd4jLong const* tadOffsets,
						bool descending) {
    try {
        // to be implemented
        auto stream = reinterpret_cast<hipStream_t *>(extraPointers[1]);
        auto context = extraPointers[0] == 0 ? LaunchContext::defaultContext()
                                             : reinterpret_cast<LaunchContext *>(extraPointers[0]);
        auto tadPack = sd::ConstantTadHelper::getInstance().tadForDimensions(xShapeInfo, dimension, dimensionLength);
        dim3 launchDims((int) tadPack.numberOfTads(), 512, 33768);
        auto xType = sd::ArrayOptions::dataType(xShapeInfo);
        BUILD_SINGLE_SELECTOR(xType, oesTadGeneric,
                              (launchDims, stream, dX, dXShapeInfo, nullptr, dimensionLength, tadShapeInfo, tadOffsets, descending),
                              LIBND4J_TYPES);

        sd::DebugHelper::checkErrorCode(stream, "sortTad(...) failed");
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
    }
}

void sortCooIndices(Nd4jPointer *extraPointers, Nd4jLong *indices, void *values, Nd4jLong length, const Nd4jLong *xShapeInfo) {
	throw std::runtime_error("sortCooIndices:: Not implemented yet");
}

void ravelMultiIndex(Nd4jPointer *extraPointers, Nd4jLong *indices, Nd4jLong *flatIndices, Nd4jLong length,  Nd4jLong *shapeInfo, int mode) {
    throw std::runtime_error("ravelMultiIndex:: Not implemented yet");
}

void unravelIndex(Nd4jPointer *extraPointers, Nd4jLong *indices, Nd4jLong *flatIndices, Nd4jLong length,  Nd4jLong *shapeInfo) {
    throw std::runtime_error("unravelIndex:: Not implemented yet");
}


Nd4jLong* mmapFile(Nd4jPointer *extraPointers, const char *fileName, Nd4jLong length) {
	return nullptr;
}

void munmapFile(Nd4jPointer *extraPointers, Nd4jLong* ptrMap, Nd4jLong length) {

}


sd::graph::ResultWrapper* executeFlatGraph(Nd4jPointer *extraPointers, Nd4jPointer flatBufferPointer) {
    try {
        return sd::graph::GraphExecutioner::executeFlatBuffer(flatBufferPointer);
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
        return nullptr;
    }
}

Nd4jLong getResultWrapperSize(sd::graph::ResultWrapper* ptr) {
    return ptr->size();
}
Nd4jPointer getResultWrapperPointer(sd::graph::ResultWrapper* ptr) {
    return ptr->pointer();
}


const char* getAllCustomOps() {
	return sd::ops::OpRegistrator::getInstance().getAllCustomOperations();
}


sd::ShapeList* _calculateOutputShapes(Nd4jPointer* extraPointers, sd::ops::DeclarableOp* op, Nd4jPointer* inputBuffers, Nd4jPointer* inputShapes, int numInputShapes, double* tArgs, int numTArgs, Nd4jLong *iArgs, int numIArgs, bool *bArgs, int numBArgs, int *dArgs, int numDArgs) {
    sd::graph::VariableSpace varSpace;
    Context block(2, &varSpace);
    sd::ShapeList inShapes;

    for (int e = 0; e < numIArgs; e++)
        block.getIArguments()->push_back(iArgs[e]);

    for (int e = 0; e < numTArgs; e++)
        block.getTArguments()->push_back(tArgs[e]);

	for (int e = 0; e < numBArgs; e++)
		block.getBArguments()->push_back(bArgs[e]);

    for (int e = 0; e < numDArgs; e++)
        block.getDArguments()->push_back((sd::DataType) dArgs[e]);

	for (int e = 0; e < numInputShapes; e++) {
		auto shape_ = reinterpret_cast<Nd4jLong *>(inputShapes[e]);

		// we shouldn't copy buffer if that's empty array
		void *buffer_ = sd::ArrayOptions::arrayType(shape_) == ArrayType::EMPTY ? nullptr : inputBuffers[e];
        void *bufferD_ = sd::ArrayOptions::arrayType(shape_) == ArrayType::EMPTY ? nullptr : inputBuffers[e + numInputShapes];

		auto array = new sd::NDArray(buffer_, bufferD_, shape_);

		// block should contain references to proper variable
		varSpace.putVariable(1, e, array);
		block.pickInput(1, e);

		inShapes.push_back(shape_);
	}

    auto shapeList = op->calculateOutputShape(&inShapes, block);

    if (varSpace.launchContext()->getWorkspace() != nullptr)
        shapeList->detach();

    return shapeList;
}

sd::ShapeList* calculateOutputShapes2(Nd4jPointer* extraPointers, Nd4jLong hash, Nd4jPointer* inputBuffers, Nd4jPointer* inputShapes, int numInputShapes, double* tArgs, int numTArgs, Nd4jLong *iArgs, int numIArgs, bool *bArgs, int numBArgs, int *dArgs, int numDArgs) {
    try {
        auto op = sd::ops::OpRegistrator::getInstance().getOperation(hash);

        return _calculateOutputShapes(extraPointers, op, inputBuffers, inputShapes, numInputShapes, tArgs, numTArgs,
                                      iArgs, numIArgs, bArgs, numBArgs, dArgs, numDArgs);
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
        return nullptr;
    }
}

sd::ShapeList* _calculateOutputShapes(Nd4jPointer* extraPointers, sd::ops::DeclarableOp* op, Nd4jPointer* inputShapes, int numInputShapes, double* tArgs, int numTArgs, Nd4jLong *iArgs, int numIArgs) {
    Context block(1);
	sd::ShapeList inShapes;

	for (int e = 0; e < numIArgs; e++)
		block.getIArguments()->push_back(iArgs[e]);

	for (int e = 0; e < numTArgs; e++)
		block.getTArguments()->push_back(tArgs[e]);

	for (int e = 0; e < numInputShapes; e++)
		inShapes.push_back(reinterpret_cast<Nd4jLong *>(inputShapes[e]));

	auto shapeList = op->calculateOutputShape(&inShapes, block);

	return shapeList;
}

sd::ShapeList* calculateOutputShapes(Nd4jPointer* extraPointers, Nd4jLong hash, Nd4jPointer* inputShapes, int numInputShapes, double* tArgs, int numTArgs, Nd4jLong *iArgs, int numIArgs) {
    try {
        auto op = sd::ops::OpRegistrator::getInstance().getOperation(hash);

        return _calculateOutputShapes(extraPointers, op, inputShapes, numInputShapes, tArgs, numTArgs, iArgs, numIArgs);
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
        return nullptr;
    }
}

Nd4jLong getShapeListSize(sd::ShapeList* list) {
    return list->size();
}

Nd4jLong const* getShape(sd::ShapeList* list, Nd4jLong i) {
    return list->at(i);
}

static FORCEINLINE Nd4jStatus realExec(sd::ops::DeclarableOp* op, Nd4jPointer* extraPointers, Nd4jLong hash, Nd4jPointer* inputBuffers, Nd4jPointer* inputShapes, int numInputs, Nd4jPointer* outputBuffers, Nd4jPointer* outputShapes, int numOutputs, double* tArgs, int numTArgs, Nd4jLong *iArgs, int numIArgs, bool* bArgs, int numBArgs, bool isInplace) {
	if (op == nullptr)
		nd4j_printf("Can't find requested operation: [%lld]\n", hash);

	// we're using the same fake nodeId everywhere here

	std::vector<sd::NDArray*> inputs(numInputs);
	std::vector<sd::NDArray*> outputs(numOutputs);
	std::vector<double> ttArgs(numTArgs);
	std::vector<bool> bbArgs(numBArgs);
	std::vector<Nd4jLong> iiArgs(numIArgs);

	// filling block now with inputs
	for (int e = 0; e < numInputs; e++) {
		auto shape = reinterpret_cast<Nd4jLong *>(inputShapes[e]);
		void *buffer = sd::ArrayOptions::arrayType(shape) == ArrayType::EMPTY ? nullptr : inputBuffers[e];
        void *bufferD = sd::ArrayOptions::arrayType(shape) == ArrayType::EMPTY ? nullptr : inputBuffers[e + numInputs];

		inputs[e] = new sd::NDArray(buffer, bufferD, shape);
	}

	// if not inplace - transferring output arrays

	if (!isInplace)
		for (int e = 0; e < numOutputs; e++) {
			// we want to keep original output shape intact
			auto shape = shape::copyShape(reinterpret_cast<Nd4jLong *>(outputShapes[e]));
			void *buffer = sd::ArrayOptions::arrayType(shape) == ArrayType::EMPTY ? nullptr : outputBuffers[e];
            void *bufferD = sd::ArrayOptions::arrayType(shape) == ArrayType::EMPTY ? nullptr : outputBuffers[e + numOutputs];

			// FIXME: revisit this.
			bool canNullify = true;
			for (int i = 0; i < numInputs; i++) {
				void *ibuffer = sd::ArrayOptions::arrayType(shape) == ArrayType::EMPTY ? nullptr : inputBuffers[i];
				if (ibuffer == buffer) {
					canNullify = false;
					break;
				}
			}

			if (canNullify && buffer != nullptr)
				memset((uint8_t *) buffer, '\0', shape::length(shape) * DataTypeUtils::sizeOfElement(ArrayOptions::dataType(shape)));

			auto array = new sd::NDArray(buffer, bufferD, shape);
			outputs[e] = array;
		}

	for (int e = 0; e < numIArgs; e++)
		iiArgs[e] = iArgs[e];

	for (int e = 0; e < numTArgs; e++)
		ttArgs[e] = tArgs[e];

    for (int e = 0; e < numBArgs; e++)
        bbArgs[e] = bArgs[e];


	// hypothetically at this point we have everything filled
	auto dZ = op->execute(inputs, outputs, ttArgs, iiArgs, bbArgs, std::vector<sd::DataType>(), isInplace);
	//auto dZ = op->execute(inputs, ttArgs, iiArgs, isInplace);


	if (!isInplace)
		for (int e = 0; e < numOutputs; e++) {
			//shape::printShapeInfoLinear("JVM output shape", (int *) outputShapes[e]);
			//shape::printShapeInfoLinear("C++ output shape", (int *) outputs[e]->shapeInfo());
			//outputs[e]->printIndexedBuffer("C++ raw output");
			//outputs[e]->printBuffer("C++ indexed output");

			if (outputs[e]->ordering() != shape::order(reinterpret_cast<Nd4jLong *>(outputShapes[e])))
				outputs[e]->streamline(shape::order(reinterpret_cast<Nd4jLong *>(outputShapes[e])));
		}

	for (auto v: inputs)
		delete v;

	for (auto v: outputs)
		delete v;

	return Status::OK();
}


int execCustomOp(Nd4jPointer* extraPointers, Nd4jLong hash, Nd4jPointer* inputBuffers, Nd4jPointer* inputShapes, int numInputs, Nd4jPointer* outputBuffers, Nd4jPointer* outputShapes, int numOutputs, double* tArgs, int numTArgs, Nd4jLong *iArgs, int numIArgs, bool* bArgs, int numBArgs, bool isInplace) {
    try {
        auto op = sd::ops::OpRegistrator::getInstance().getOperation(hash);

        return realExec(op, extraPointers, hash, inputBuffers, inputShapes, numInputs, outputBuffers, outputShapes,
                        numOutputs, tArgs, numTArgs, iArgs, numIArgs, bArgs, numBArgs, isInplace);
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
        return 1;
    }
}

int execCustomOp2(Nd4jPointer* extraPointers, Nd4jLong hash, Nd4jPointer opContext) {
    try {
        auto op = sd::ops::OpRegistrator::getInstance().getOperation(hash);
        auto context = reinterpret_cast<Context *>(opContext);

        auto result = op->execute(context);

        auto res = hipStreamSynchronize(*context->launchContext()->getCudaStream());
        if (res != 0)
            throw sd::cuda_exception::build("customOp execution failed", res);

        for (auto v:context->fastpath_in()) {
            if (!v->isEmpty())
                v->syncToDevice();
        }

        for (auto v:context->fastpath_out()) {
            if (!v->isEmpty())
                v->syncToDevice();
        }

        return result;
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
        return 1;
    }
}

int registerGraph(Nd4jPointer *extraPointers, Nd4jLong graphId, Nd4jPointer flatBufferPointer) {
    try {
        auto graph = sd::graph::GraphExecutioner::importFromFlatPointer(flatBufferPointer);

        sd::graph::GraphHolder::getInstance().registerGraph(graphId, graph);

        return ND4J_STATUS_OK;
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
        return 1;
    }
}


static VariablesSet* executeStoredGraphT(Nd4jPointer *extraPointers, Nd4jLong graphId, Nd4jPointer *inputBuffers, Nd4jPointer *inputShapes, int* inputIndices, int numInputs) {
	auto graph = sd::graph::GraphHolder::getInstance().pullGraph(graphId);
	auto varSpace = graph->getVariableSpace()->clone();

	std::vector<sd::NDArray*> handles;

	for (int e = 0; e < numInputs; e++) {
		auto idx = inputIndices[e];

		// we'll delete this array later, together with cloned VariableSpace
		auto array = new sd::NDArray(inputBuffers[e], reinterpret_cast<Nd4jLong *>(inputShapes[e]));
		handles.emplace_back(array);

		if (varSpace->hasVariable(idx)) {
			auto var = varSpace->getVariable(idx);
			if (var->hasNDArray())
				delete var->getNDArray();

			var->setNDArray(array);
		} else
			varSpace->putVariable(idx, array);
	}

	auto dZ = sd::graph::GraphExecutioner::execute(graph, varSpace);
	auto varSet = new sd::graph::VariablesSet(dZ);

	if (dZ == ND4J_STATUS_OK) {
		// pull back results, and provide them
		auto outputs = graph->fetchOutputs();
		for (int e = 0; e < outputs->size(); e++) {
			// we're only getting variable ID/Index from original grap. values will be taken from cloned workspace
			std::pair<int, int> varId(outputs->at(e)->id(), outputs->at(e)->index());

			auto var = varSpace->getVariable(varId);

			varSet->push_back(var->clone());
		}

		delete outputs;
	}

	delete varSpace;

	return varSet;
}

VariablesSet* executeStoredGraph(Nd4jPointer *extraPointers, Nd4jLong graphId, Nd4jPointer *inputBuffers, Nd4jPointer *inputShapes, int* inputIndices, int numInputs) {
    try {
        return executeStoredGraphT(extraPointers, graphId, inputBuffers, inputShapes, inputIndices, numInputs);
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
        return nullptr;
    }
}

Nd4jLong getVariablesSetSize(sd::graph::VariablesSet* set) {
    return set->size();
}

Nd4jStatus getVariablesSetStatus(sd::graph::VariablesSet* set) {
    return set->status();
}

sd::graph::Variable* getVariable(sd::graph::VariablesSet* set, Nd4jLong i) {
    return set->at(i);
}

int getVariableId(sd::graph::Variable* variable) {
    return variable->id();
}

int getVariableIndex(sd::graph::Variable* variable) {
    return variable->index();
}

const char* getVariableName(sd::graph::Variable* variable) {
    return variable->getName()->c_str();
}

Nd4jLong const* getVariableShape(sd::graph::Variable* variable) {
    return variable->getNDArray()->shapeInfo();
}

void* getVariableBuffer(sd::graph::Variable* variable) {
    return variable->getNDArray()->buffer();
}

int unregisterGraph(Nd4jPointer *extraPointers, Nd4jLong graphId) {
    try {
        sd::graph::GraphHolder::getInstance().dropGraphAny(graphId);

        return ND4J_STATUS_OK;
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
        return 1;
    }
}

void deletePointerArray(Nd4jPointer pointer) {
    Nd4jPointer *ptr = reinterpret_cast<Nd4jPointer *>(pointer);
    delete[] ptr;
}

void deleteCharArray(Nd4jPointer pointer) {
    auto ptr = reinterpret_cast<char *>(pointer);
    delete[] ptr;
}

void deleteIntArray(Nd4jPointer pointer) {
	auto ptr = reinterpret_cast<int *>(pointer);
	delete[] ptr;
}

void deleteLongArray(Nd4jPointer pointer) {
	auto ptr = reinterpret_cast<Nd4jLong *>(pointer);
	delete[] ptr;
}

void deleteVariablesSet(sd::graph::VariablesSet* pointer) {
	delete pointer;
}

void deleteShapeList(Nd4jPointer shapeList) {
    sd::ShapeList* list = reinterpret_cast<sd::ShapeList*>(shapeList);

    //list->destroy();
    delete list;
}

const char* getAllOperations() {
    return sd::OpTracker::getInstance().exportOperations();
}

Nd4jPointer getGraphState(Nd4jLong id) {
    return (Nd4jPointer) new sd::graph::GraphState(id);
}


void deleteGraphState(Nd4jPointer state) {
    auto stateP = reinterpret_cast<sd::graph::GraphState*>(state);
    delete stateP;
}


Nd4jStatus execCustomOpWithScope(Nd4jPointer *extraPointers, sd::graph::GraphState *state, Nd4jLong opHash, Nd4jLong *scopes, int numScopes, Nd4jPointer *inputBuffers, Nd4jPointer *inputShapes, int numInputs, Nd4jPointer *outputBuffers, Nd4jPointer *outputShapes, int numOutputs) {
    /**
     * That's basically exec, with VariableSpace provided in GraphState:
     * depending on operation (i.e. while of if), different logic executors could be used
     */

    auto graph = state->graph();
    auto varSpace = state->variableSpace();

    // Node is dynamically created, and has nothing beyond it: only inputs and outputs
    // this node has id of 0, and inputs are
    Node node(OpType_LOGIC, opHash, 0);

    // mapping inputs
    for (int e = 0; e < numInputs; e++) {
        auto buffer = inputBuffers[e];
        auto shapeInfo = reinterpret_cast<Nd4jLong *>(inputShapes[e]);

        auto array = new sd::NDArray(buffer, shapeInfo, varSpace->launchContext());

        // now we just put array to VarSpace
        varSpace->putVariable(0, e, array);
        node.pickInput(0, e);
    }

    // mapping scopes
    for (int e = 0; e < numScopes; e++) {
        // we should check scope existence in GraphState/Graph
        int scopeId = (int) scopes[e];
        if (!state->hasScope(scopeId)) {
            // nd4j_printf("execCustomOpWithScope: referenced scope [%i] doesn't exist\n", scopeId);
            return Status::THROW();
        }
        node.pickInput(scopeId, 0);
    }

    auto dZ = LogicExecutor::processNode(graph, &node);
    if (dZ != Status::OK())
        return dZ;

    // mapping outputs

    for (int e = 0; e < numOutputs; e++) {
        auto buffer = outputBuffers[e];
        auto shapeInfo = reinterpret_cast<Nd4jLong *>(outputShapes[e]);

        NDArray array(buffer, shapeInfo, varSpace->launchContext());

        // now we just put array to VarSpace to the same ID
        //varSpace->putVariable(0, e, array);

        auto t = varSpace->getVariable(0, e)->getNDArray();
        array.assign(t);
    }

    // removing input variables
    for (int e = 0; e < numInputs; e++) {
        varSpace->dropVariable(0, e);
    }

    // after some bla-bla-bla we should have Graph and Node for current op
    return Status::OK();
}


Nd4jStatus execCustomOpWithScope(Nd4jPointer *extraPointers, Nd4jPointer state, Nd4jLong opHash, Nd4jLong *scopes, int numScopes, Nd4jPointer *inputBuffers, Nd4jPointer *inputShapes, int numInputs, Nd4jPointer *outputBuffers, Nd4jPointer *outputShapes, int numOutputs) {
    try {
        return execCustomOpWithScope(extraPointers, reinterpret_cast<sd::graph::GraphState *>(state), opHash, scopes,
                                     numScopes, inputBuffers, inputShapes, numInputs, outputBuffers, outputShapes,
                                     numOutputs);
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
        return 1;
    }
}

void deleteResultWrapper(Nd4jPointer ptr) {
	// just 0 room for compiler s@!t
	auto p = reinterpret_cast<sd::graph::ResultWrapper *>(ptr);
	delete p;
}

int estimateThreshold(Nd4jPointer *extraPointers, Nd4jPointer dX, Nd4jLong const* dXShapeInfo, int N, float threshold) {
	throw std::runtime_error("estimateThreshold: Not implemented yet");
}

/*
 * TypeDef:
 *     void convertTypes(Nd4jPointer *extras, int srcType, Nd4jPointer dX, long N, int dstType, Nd4jPointer dZ);
 */
void convertTypes(Nd4jPointer *extras, int srcType, Nd4jPointer dX, Nd4jLong N, int dstType, Nd4jPointer dZ) {
    try {
        auto dx = reinterpret_cast<void *>(dX);
        auto dz = reinterpret_cast<void *>(dZ);

        if (srcType == ND4J_FLOAT8) {
            if (dstType == ND4J_FLOAT8) {
                // convertKernel<double, sd::float8>(extras, dx, N, dz);
            } else if (dstType == ND4J_INT8) {
                //sd::TypeCast::convertGenericCuda<sd::float8, sd::int8>(extras, dx, N, dz);
            } else if (dstType == ND4J_UINT8) {
                //sd::TypeCast::convertGenericCuda<sd::float8, sd::uint8>(extras, dx, N, dz);
            } else if (dstType == ND4J_FLOAT16) {
                //sd::TypeCast::convertGenericCuda<sd::float8, float16>(extras, dx, N, dz);
            } else if (dstType == ND4J_INT16) {
                //sd::TypeCast::convertGenericCuda<sd::float8, sd::int16>(extras, dx, N, dz);
            } else if (dstType == ND4J_UINT16) {
                //sd::TypeCast::convertGenericCuda<sd::float8, sd::uint16>(extras, dx, N, dz);
            } else if (dstType == ND4J_FLOAT24) {

            } else if (dstType == ND4J_FLOAT32) {
                //sd::TypeCast::convertGenericCuda<sd::float8, float>(extras, dx, N, dz);
            } else if (dstType == ND4J_DOUBLE) {
                //sd::TypeCast::convertGenericCuda<sd::float8, double>(extras, dx, N, dz);
            } else {
                nd4j_printf("Unsupported types conversion: [%i] -> [%i]\n", srcType, dstType);
            }
        } else if (srcType == ND4J_INT8) {
            if (dstType == ND4J_FLOAT8) {
                //sd::TypeCast::convertGenericCuda<sd::int8, sd::float8>(extras, dx, N, dz);
            } else if (dstType == ND4J_INT8) {
                //convertKernel<sd::int8, sd::int8>(extras, dx, N, dz);
            } else if (dstType == ND4J_UINT8) {
                sd::TypeCast::convertGenericCuda<int8_t, uint8_t>(extras, dx, N, dz);
            } else if (dstType == ND4J_FLOAT16) {
                sd::TypeCast::convertGenericCuda<int8_t, float16>(extras, dx, N, dz);
            } else if (dstType == ND4J_INT16) {
                sd::TypeCast::convertGenericCuda<int8_t, int16_t>(extras, dx, N, dz);
            } else if (dstType == ND4J_UINT16) {
                sd::TypeCast::convertGenericCuda<int8_t, uint16_t>(extras, dx, N, dz);
            } else if (dstType == ND4J_FLOAT24) {
                // TODO: eventually we might want to add it
            } else if (dstType == ND4J_FLOAT32) {
                sd::TypeCast::convertGenericCuda<int8_t, float>(extras, dx, N, dz);
            } else if (dstType == ND4J_DOUBLE) {
                sd::TypeCast::convertGenericCuda<int8_t, double>(extras, dx, N, dz);
            } else {
                nd4j_printf("Unsupported types conversion: [%i] -> [%i]\n", srcType, dstType);
            }
        } else if (srcType == ND4J_UINT8) {
            if (dstType == ND4J_FLOAT8) {
                //sd::TypeCast::convertGenericCuda<uint8_t, sd::float8>(extras, dx, N, dz);
            } else if (dstType == ND4J_INT8) {
                sd::TypeCast::convertGenericCuda<uint8_t, int8_t>(extras, dx, N, dz);
            } else if (dstType == ND4J_UINT8) {
                sd::TypeCast::convertGenericCuda<uint8_t, uint8_t>(extras, dx, N, dz);
            } else if (dstType == ND4J_FLOAT16) {
                sd::TypeCast::convertGenericCuda<uint8_t, float16>(extras, dx, N, dz);
            } else if (dstType == ND4J_INT16) {
                sd::TypeCast::convertGenericCuda<uint8_t, int16_t>(extras, dx, N, dz);
            } else if (dstType == ND4J_UINT16) {
                sd::TypeCast::convertGenericCuda<uint8_t, uint16_t>(extras, dx, N, dz);
            } else if (dstType == ND4J_FLOAT24) {
                // TODO: still might want to add
            } else if (dstType == ND4J_FLOAT32) {
                sd::TypeCast::convertGenericCuda<uint8_t, float>(extras, dx, N, dz);
            } else if (dstType == ND4J_DOUBLE) {
                sd::TypeCast::convertGenericCuda<uint8_t, double>(extras, dx, N, dz);
            } else {
                nd4j_printf("Unsupported types conversion: [%i] -> [%i]\n", srcType, dstType);
            }
        } else if (srcType == ND4J_FLOAT16) {
            if (dstType == ND4J_FLOAT8) {
                //sd::TypeCast::convertGenericCuda<float16, sd::float8>(extras, dx, N, dz);
            } else if (dstType == ND4J_INT8) {
                sd::TypeCast::convertGenericCuda<float16, int8_t>(extras, dx, N, dz);
            } else if (dstType == ND4J_UINT8) {
                sd::TypeCast::convertGenericCuda<float16, uint8_t>(extras, dx, N, dz);
            } else if (dstType == ND4J_FLOAT16) {
                sd::TypeCast::convertGenericCuda<float16, float16>(extras, dx, N, dz);
            } else if (dstType == ND4J_INT16) {
                sd::TypeCast::convertGenericCuda<float16, int16_t>(extras, dx, N, dz);
            } else if (dstType == ND4J_UINT16) {
                sd::TypeCast::convertGenericCuda<float16, uint16_t>(extras, dx, N, dz);
            } else if (dstType == ND4J_FLOAT24) {
                // TODO: .... ^^^
            } else if (dstType == ND4J_FLOAT32) {
                sd::TypeCast::convertGenericCuda<float16, float>(extras, dx, N, dz);
            } else if (dstType == ND4J_DOUBLE) {
                sd::TypeCast::convertGenericCuda<float16, double>(extras, dx, N, dz);
            } else if (dstType == ND4J_THRESHOLD) {
                //sd::convertToThreshold<float16>(nullptr, dx, N, dz);
            } else {
                nd4j_printf("Unsupported types conversion: [%i] -> [%i]\n", srcType, dstType);
            }
        } else if (srcType == ND4J_INT16) {
            if (dstType == ND4J_FLOAT8) {
                //sd::TypeCast::convertGenericCuda<int16_t, sd::float8>(extras, dx, N, dz);
            } else if (dstType == ND4J_INT8) {
                sd::TypeCast::convertGenericCuda<int16_t, int8_t>(extras, dx, N, dz);
            } else if (dstType == ND4J_UINT8) {
                sd::TypeCast::convertGenericCuda<int16_t, uint8_t>(extras, dx, N, dz);
            } else if (dstType == ND4J_FLOAT16) {
                sd::TypeCast::convertGenericCuda<int16_t, float16>(extras, dx, N, dz);
            } else if (dstType == ND4J_INT16) {
                sd::TypeCast::convertGenericCuda<int16_t, int16_t>(extras, dx, N, dz);
            } else if (dstType == ND4J_UINT16) {
                sd::TypeCast::convertGenericCuda<int16_t, uint16_t>(extras, dx, N, dz);
            } else if (dstType == ND4J_FLOAT24) {
                // TODO...
            } else if (dstType == ND4J_FLOAT32) {
                sd::TypeCast::convertGenericCuda<int16_t, float>(extras, dx, N, dz);
            } else if (dstType == ND4J_DOUBLE) {
                sd::TypeCast::convertGenericCuda<int16_t, double>(extras, dx, N, dz);
            } else {
                printf("Unsupported types conversion: [%i] -> [%i]\n", srcType, dstType);
            }
        } else if (srcType == ND4J_FLOAT24) {

        } else if (srcType == ND4J_FLOAT32) {
            if (dstType == ND4J_FLOAT8) {
                //sd::TypeCast::convertGenericCuda<float, sd::float8>(extras, dx, N, dz);
            } else if (dstType == ND4J_INT8) {
                sd::TypeCast::convertGenericCuda<float, int8_t>(extras, dx, N, dz);
            } else if (dstType == ND4J_UINT8) {
                sd::TypeCast::convertGenericCuda<float, uint8_t>(extras, dx, N, dz);
            } else if (dstType == ND4J_FLOAT16) {
                sd::TypeCast::convertGenericCuda<float, float16>(extras, dx, N, dz);
            } else if (dstType == ND4J_INT16) {
                sd::TypeCast::convertGenericCuda<float, int16_t>(extras, dx, N, dz);
            } else if (dstType == ND4J_UINT16) {
                sd::TypeCast::convertGenericCuda<float, uint16_t>(extras, dx, N, dz);
            } else if (dstType == ND4J_FLOAT24) {

            } else if (dstType == ND4J_DOUBLE) {
                sd::TypeCast::convertGenericCuda<float, double>(extras, dx, N, dz);
            } else if (dstType == ND4J_THRESHOLD) {
                //sd::convertToThreshold<float>(nullptr, dx, N, dz);
            } else {
                nd4j_printf("Unsupported types conversion: [%i] -> [%i]\n", srcType, dstType);
            }
        } else if (srcType == ND4J_DOUBLE) {
            if (dstType == ND4J_FLOAT8) {
                //sd::TypeCast::convertGenericCuda<double, sd::float8>(extras, dx, N, dz);
            } else if (dstType == ND4J_INT8) {
                sd::TypeCast::convertGenericCuda<double, int8_t>(extras, dx, N, dz);
            } else if (dstType == ND4J_UINT8) {
                sd::TypeCast::convertGenericCuda<double, uint8_t>(extras, dx, N, dz);
            } else if (dstType == ND4J_FLOAT16) {
                sd::TypeCast::convertGenericCuda<double, float16>(extras, dx, N, dz);
            } else if (dstType == ND4J_INT16) {
                sd::TypeCast::convertGenericCuda<double, int16_t>(extras, dx, N, dz);
            } else if (dstType == ND4J_UINT16) {
                sd::TypeCast::convertGenericCuda<double, uint16_t>(extras, dx, N, dz);
            } else if (dstType == ND4J_FLOAT24) {

            } else if (dstType == ND4J_FLOAT32) {
                sd::TypeCast::convertGenericCuda<double, float>(extras, dx, N, dz);
            } else if (dstType == ND4J_DOUBLE) {
                //
            } else if (dstType == ND4J_THRESHOLD) {
                //sd::convertToThreshold<double>(nullptr, dx, N, dz);
            } else {
                nd4j_printf("Unsupported types conversion: [%i] -> [%i]\n", srcType, dstType);
            }
        } else if (srcType == ND4J_THRESHOLD) {
            if (dstType == ND4J_FLOAT16) {
                //sd::convertFromThreshold<float16>(nullptr, dx, N, dz);
            } else if (dstType == ND4J_FLOAT32) {
                //sd::convertFromThreshold<float>(nullptr, dx, N, dz);
            } else if (dstType == ND4J_DOUBLE) {
                //sd::convertFromThreshold<double>(nullptr, dx, N, dz);
            } else {
                nd4j_printf("Unsupported types conversion: [%i] -> [%i]\n", srcType, dstType);
            }
        } else {
            nd4j_printf("Unsupported types conversion: [%i] -> [%i]\n", srcType, dstType);
        }
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
    }
}

Nd4jPointer createUtf8String(Nd4jPointer *extraPointers, const char *string, int length) {
    auto u = new sd::utf8string(string, length);
    return reinterpret_cast<Nd4jPointer>(u);
}

Nd4jLong getUtf8StringLength(Nd4jPointer *extraPointers, Nd4jPointer ptr) {
    return reinterpret_cast<sd::utf8string*>(ptr)->_length;
}
char* getUtf8StringBuffer(Nd4jPointer *extraPointers, Nd4jPointer ptr) {
    return reinterpret_cast<sd::utf8string*>(ptr)->_buffer;
}

void deleteUtf8String(Nd4jPointer *extraPointers, Nd4jPointer ptr) {
    delete(reinterpret_cast<sd::utf8string*>(ptr));
}

///////////////////////////////////////////////////////////////////
template<typename T, typename I>
__global__ static void scatterUpdateCuda(const int opCode, const int numOfSubArrs,
										      void* vx, const Nd4jLong* xShapeInfo, const Nd4jLong *xOffsets,
										      void* vy, const Nd4jLong *yShapeInfo, const Nd4jLong *yOffsets,
										      const void* vindexes) {

    __shared__ T *x, *y;
    __shared__ Nd4jLong arrLenX, arrLenY;
    auto indexes = reinterpret_cast<const I*>(vindexes);

    for (int e = 0; e < numOfSubArrs; e++ ) {

        const auto xIndex = indexes[e];
        const bool isOwner = xIndex < gridDim.x ? blockIdx.x == xIndex : blockIdx.x == xIndex % gridDim.x;

        if (!isOwner)
            continue;

        if (threadIdx.x == 0) {
            x = reinterpret_cast<T*>(vx) + xOffsets[xIndex];
            y = reinterpret_cast<T*>(vy) + yOffsets[e];
            arrLenX = shape::length(xShapeInfo);
            arrLenY = shape::length(yShapeInfo);
        }
        __syncthreads();

        if (arrLenX != arrLenY)
            return;

        for (Nd4jLong i = threadIdx.x; i < arrLenX; i += blockDim.x) {

            const auto xOffset = shape::getIndexOffset(i, xShapeInfo);
            const auto yOffset = shape::getIndexOffset(i, yShapeInfo);

            switch (opCode) {
                case 0:
                    x[xOffset] += y[yOffset];
                    break;
                case 1:
                    x[xOffset] -= y[yOffset];
                    break;
                case 2:
                    x[xOffset] *= y[yOffset];
                    break;
                case 3:
                    x[xOffset] /= y[yOffset];
                    break;
                case 4:
                    x[xOffset] = y[yOffset] - x[xOffset];
                    break;
                case 5:
                    x[xOffset] = y[yOffset] / x[xOffset];
                    break;
                case 6:
                    x[xOffset] = y[yOffset];
                    break;
                default:
                    continue;
            }
        }
        __syncthreads();
    }
}

template<typename T, typename I>
__host__ static void scatterUpdateCudaLauncher(const hipStream_t* stream, const int opCode, const int numOfSubArrs, void* vx, const Nd4jLong const* xShapeInfo, const Nd4jLong* xOffsets, void* vy, const Nd4jLong *yShapeInfo, const Nd4jLong *yOffsets, const void* indexes) {

    scatterUpdateCuda<T, I><<<512, 256, MAX_NUM_THREADS, *stream>>>(opCode, numOfSubArrs, vx, xShapeInfo, xOffsets, vy, yShapeInfo, yOffsets, indexes);
}


//////////////////////////////////////////////////////////////////////////
void scatterUpdate(Nd4jPointer *extraPointers, int opCode, int numOfSubArrs,
                      			void* hX, Nd4jLong const* hXShapeInfo, Nd4jLong const* hXOffsets,
                      			void* dX, Nd4jLong const* dXShapeInfo, Nd4jLong const* dXOffsets,
                      			void* hY, Nd4jLong const* hYShapeInfo, Nd4jLong const* hYOffsets,
                      			void* dY, Nd4jLong const* dYShapeInfo, Nd4jLong const* dYOffsets,
                      			void* hIindexes, Nd4jLong const* hIndicesShapeInfo, void* dIindexes, Nd4jLong const* dIndicesShapeInfo) {
    try {
        auto stream = reinterpret_cast<hipStream_t *>(extraPointers[1]);

        auto type = ArrayOptions::dataType(hXShapeInfo);
        auto iType = ArrayOptions::dataType(hIndicesShapeInfo);

        BUILD_DOUBLE_SELECTOR(type, iType, scatterUpdateCudaLauncher,
                              (stream, opCode, numOfSubArrs, dX, dXShapeInfo, dXOffsets, dY, dYShapeInfo, dYOffsets, dIindexes),
                              LIBND4J_TYPES, INDEXING_TYPES);

        sd::DebugHelper::checkErrorCode(stream, "scatterUpdate(...) failed");
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
    }
}

void inspectArray(Nd4jPointer *extraPointers, Nd4jPointer buffer, Nd4jLong *shapeInfo, Nd4jPointer specialBuffer, Nd4jLong *specialShapeInfo, Nd4jPointer debugInfo) {
    try {
        LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
        auto p = reinterpret_cast<sd::DebugInfo *>(debugInfo);
        NDArray array(buffer, specialBuffer, shapeInfo, &lc);
        sd::DebugHelper::retrieveDebugStatistics(p, &array);
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
    }
}

void __global__ tryPointerKernel(void* p, int len) {
    auto buf = reinterpret_cast<int8_t*>(p);
    auto tid = threadIdx.x + blockIdx.x * blockDim.x;
    __shared__ int b;
    if (tid < len)
        atomicAdd(&b, buf[tid]);

    __syncthreads();

    if (threadIdx.x ==0 && blockIdx.x == 0)
        printf("Pointer check complete: %i\n", b);
}

void tryPointer(Nd4jPointer extra, Nd4jPointer p, int len) {
    try {
        hipStream_t stream;
        hipStreamCreate(&stream);

        tryPointerKernel <<< 256, 512, len + 64, stream>>> (p, len);
        auto e = hipStreamSynchronize(stream);

        if (e != 0)
            throw sd::cuda_exception::build("tryPointer failed", e);

        hipStreamDestroy(stream);
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
    }
}

int dataTypeFromNpyHeader(void *header) {
    return (int) cnpy::dataTypeFromHeader(reinterpret_cast<char *>(header));
}

OpaqueConstantShapeBuffer* shapeBuffer(int rank, Nd4jLong *shape, Nd4jLong *strides, sd::DataType dtype, char order, Nd4jLong ews, bool empty) {
    try {
        auto buffer = new ConstantShapeBuffer();
        *buffer = sd::ConstantShapeHelper::getInstance().bufferForShapeInfo(
                ShapeDescriptor(dtype, order, shape, strides, rank, ews, empty));
        return buffer;
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
        return nullptr;
    }
}

void deleteConstantShapeBuffer(OpaqueConstantShapeBuffer* ptr) {
    delete ptr;
}

void deleteConstantDataBuffer(OpaqueConstantDataBuffer* ptr) {
  delete ptr;
}

void deleteTadPack(sd::TadPack* ptr) {
    delete ptr;
}

bool isBlasVersionMatches(int major, int minor, int build) {
    auto result = major == Environment::getInstance()._blasMajorVersion && minor == Environment::getInstance()._blasMinorVersion && build == Environment::getInstance()._blasPatchVersion;

    if (!result) {
        nd4j_printf("CUDA/cuBLAS version mismatch. Expected: %i.%i.%i but got %i.%i.%i instead\n", Environment::getInstance()._blasMajorVersion, Environment::getInstance()._blasMinorVersion, Environment::getInstance()._blasPatchVersion, major, minor, build);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(152);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage("CUDA/cuBLAS version mismatch");
    }

    return result;
}

sd::ConstantDataBuffer* constantBufferLong(sd::DataType dtype, Nd4jLong const* data, int length) {
    return sd::ConstantHelper::getInstance().constantBuffer(ConstantDescriptor(data, length), dtype);
}

sd::ConstantDataBuffer* constantBufferDouble(sd::DataType dtype, double *data, int length) {
    return sd::ConstantHelper::getInstance().constantBuffer(ConstantDescriptor(data, length), dtype);
}

sd::ConstantDataBuffer* constantBuffer(sd::DataType dtype, sd::ConstantDescriptor *descriptor) {
    return sd::ConstantHelper::getInstance().constantBuffer(*descriptor, dtype);
}


Nd4jPointer getConstantDataBufferPrimary(sd::ConstantDataBuffer* dbf) {
    return dbf->primary();
}
Nd4jPointer getConstantDataBufferSpecial(sd::ConstantDataBuffer* dbf) {
    return dbf->special();
}
Nd4jLong getConstantDataBufferLength(sd::ConstantDataBuffer* dbf) {
    return dbf->length();
}
Nd4jLong getConstantDataBufferSizeOf(sd::ConstantDataBuffer* dbf) {
    return dbf->sizeOf();
}

Nd4jPointer getConstantShapeBufferPrimary(sd::ConstantShapeBuffer* dbf) {
  return const_cast<Nd4jLong*>(dbf->primary());
}

Nd4jPointer getConstantShapeBufferSpecial(sd::ConstantShapeBuffer* dbf) {
  return const_cast<Nd4jLong*>(dbf->special());
}

sd::graph::Context* createGraphContext(int nodeId) {
    return new sd::graph::Context(nodeId);
}

sd::graph::RandomGenerator* getGraphContextRandomGenerator(sd::graph::Context* ptr) {
    return &ptr->randomGenerator();
}

void markGraphContextInplace(sd::graph::Context* ptr, bool reallyInplace) {
    ptr->markInplace(reallyInplace);
}

void setGraphContextCudaContext(sd::graph::Context* ptr, void *stream, void *reductionPointer, void *allocationPointer) {
    ptr->setCudaContext(stream, reductionPointer, allocationPointer);
}

void setGraphContextInputArray(sd::graph::Context* ptr, int index, void *buffer, void *shapeInfo, void *specialBuffer, void *specialShapeInfo) {
    ptr->setInputArray(index, buffer, shapeInfo, specialBuffer, specialShapeInfo);
}

void setGraphContextOutputArray(sd::graph::Context* ptr, int index, void *buffer, void *shapeInfo, void *specialBuffer, void *specialShapeInfo) {
    ptr->setOutputArray(index, buffer, shapeInfo, specialBuffer, specialShapeInfo);
}

void setGraphContextInputBuffer(OpaqueContext* ptr, int index, OpaqueDataBuffer *buffer, void *shapeInfo, void *specialShapeInfo) {
    ptr->setInputArray(index, buffer, shapeInfo, specialShapeInfo);
}

void setGraphContextOutputBuffer(OpaqueContext* ptr, int index, OpaqueDataBuffer *buffer, void *shapeInfo, void *specialShapeInfo) {
    ptr->setOutputArray(index, buffer, shapeInfo, specialShapeInfo);
}

void setGraphContextTArguments(sd::graph::Context* ptr, double *arguments, int numberOfArguments) {
    ptr->setTArguments(arguments, numberOfArguments);
}

void setGraphContextIArguments(sd::graph::Context* ptr, Nd4jLong *arguments, int numberOfArguments) {
    ptr->setIArguments(arguments, numberOfArguments);
}

void setGraphContextBArguments(sd::graph::Context* ptr, bool *arguments, int numberOfArguments) {
    ptr->setBArguments(arguments, numberOfArguments);
}

void setGraphContextDArguments(OpaqueContext* ptr, int *arguments, int numberOfArguments) {
    std::vector<sd::DataType> dtypes(numberOfArguments);
    for (int e = 0; e < numberOfArguments; e++)
        dtypes[e] = (sd::DataType) arguments[e];

    ptr->setDArguments(dtypes);
}

void deleteGraphContext(sd::graph::Context* ptr) {
    delete ptr;
}


sd::graph::RandomGenerator* createRandomGenerator(Nd4jLong rootSeed, Nd4jLong nodeSeed) {
    try {
        return new sd::graph::RandomGenerator(rootSeed, nodeSeed);
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
        return nullptr;
    }
}

Nd4jLong getRandomGeneratorRootState(sd::graph::RandomGenerator* ptr) {
    return ptr->rootState();
}

Nd4jLong getRandomGeneratorNodeState(sd::graph::RandomGenerator* ptr) {
    return ptr->nodeState();
}

void setRandomGeneratorStates(sd::graph::RandomGenerator* ptr, Nd4jLong rootSeed, Nd4jLong nodeSeed) {
    ptr->setStates(rootSeed, nodeSeed);
}

float getRandomGeneratorRelativeFloat(sd::graph::RandomGenerator* ptr, Nd4jLong index) {
  return ptr->relativeT<float>(index);
}

double getRandomGeneratorRelativeDouble(sd::graph::RandomGenerator* ptr, Nd4jLong index) {
  return ptr->relativeT<double>(index);
}

int getRandomGeneratorRelativeInt(sd::graph::RandomGenerator* ptr, Nd4jLong index) {
    return ptr->relativeInt(index);
}

Nd4jLong getRandomGeneratorRelativeLong(sd::graph::RandomGenerator* ptr, Nd4jLong index) {
    return ptr->relativeLong(index);
}

void deleteRandomGenerator(sd::graph::RandomGenerator* ptr) {
    delete ptr;
}


Nd4jPointer shapeBufferForNumpy(Nd4jPointer npyArray) {
    try {
        cnpy::NpyArray arr = cnpy::loadNpyFromPointer(reinterpret_cast<char *>(npyArray));
        unsigned int shapeSize = arr.shape.size();
        std::vector<Nd4jLong> shape(shapeSize);
        bool _empty = false;
        for (unsigned int i = 0; i < shapeSize; i++) {
            shape[i] = arr.shape[i];

            if (arr.shape[i] == 0)
                _empty = true;
        }

        auto dtype = cnpy::dataTypeFromHeader(reinterpret_cast<char *>(npyArray));

        Nd4jLong *shapeBuffer;
        if (shape.size() == 1 && shape[0] == 0) {
            // scalar case
            shapeBuffer = sd::ShapeBuilders::createScalarShapeInfo(dtype);
        } else if (_empty) {
            if (shapeSize > 0)
                shapeBuffer = sd::ShapeBuilders::emptyShapeInfo(dtype, arr.fortranOrder ? 'f' : 'c', shape);
            else
                shapeBuffer = sd::ShapeBuilders::emptyShapeInfo(dtype);
        } else {
            shapeBuffer = sd::ShapeBuilders::createShapeInfo(dtype, arr.fortranOrder ? 'f' : 'c', shape);
        }
        return (Nd4jPointer)(sd::ConstantShapeHelper::getInstance().createFromExisting(shapeBuffer, true)); // TO DO: this can lead to unpleasant crash sometimes
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
        return nullptr;
    }
}

const char* runLightBenchmarkSuit(bool printOut) {
    try {
        sd::LightBenchmarkSuit suit;
        auto result = suit.runSuit();

        if (printOut)
            nd4j_printf("%s\n", result.data());

        auto chars = new char[result.length() + 1];
        std::memcpy(chars, result.data(), result.length());
        chars[result.length()] = (char) 0x0;

        return chars;
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
        return nullptr;
    }
}

const char* runFullBenchmarkSuit(bool printOut) {
    try {
        sd::FullBenchmarkSuit suit;
        auto result = suit.runSuit();

        if (printOut)
            nd4j_printf("%s\n", result.data());

        auto chars = new char[result.length() + 1];
        std::memcpy(chars, result.data(), result.length());
        chars[result.length()] = (char) 0x0;

        return chars;
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
        return nullptr;
    }
}

Nd4jLong getCachedMemory(int deviceId) {
    return sd::ConstantHelper::getInstance().getCachedAmount(deviceId);
}

sd::LaunchContext* defaultLaunchContext() {
    return LaunchContext::defaultContext();
}

Nd4jPointer lcScalarPointer(OpaqueLaunchContext* lc) {
    return lc->getScalarPointer();
}

Nd4jPointer lcReductionPointer(OpaqueLaunchContext* lc) {
    return lc->getReductionPointer();
}

Nd4jPointer lcAllocationPointer(OpaqueLaunchContext* lc) {
    return lc->getAllocationPointer();
}

Nd4jPointer lcExecutionStream(OpaqueLaunchContext* lc) {
    return lc->getCudaStream();
}

Nd4jPointer lcCopyStream(OpaqueLaunchContext* lc) {
    return lc->getCudaSpecialStream();
}

Nd4jPointer lcBlasHandle(OpaqueLaunchContext* lc) {
    return lc->getCublasHandle();
}

Nd4jPointer lcSolverHandle(OpaqueLaunchContext* lc) {
    return lc->getCusolverHandle();
}

int lastErrorCode() {
    return sd::LaunchContext::defaultContext()->errorReference()->errorCode();
}

const char* lastErrorMessage() {
    return sd::LaunchContext::defaultContext()->errorReference()->errorMessage();
}

void ctxShapeFunctionOverride(OpaqueContext* ptr, bool reallyOverride) {
    ptr->setShapeFunctionOverride(reallyOverride);
}

void ctxPurge(OpaqueContext* ptr) {
    ptr->clearFastPath();
}

int  binaryLevel() {
    return 0;
}

int optimalLevel() {
    return 0;
}

bool isMinimalRequirementsMet() {
    return true;
}

bool isOptimalRequirementsMet() {
    return true;
}

void ctxAllowHelpers(OpaqueContext* ptr, bool reallyAllow) {
    ptr->allowHelpers(reallyAllow);
}

void ctxSetExecutionMode(OpaqueContext* ptr, int execMode) {
    if (execMode < 0 || execMode > 2)
        execMode = 0;

    ptr->setExecutionMode((samediff::ExecutionMode) execMode);
}

OpaqueDataBuffer* dbCreateExternalDataBuffer(Nd4jLong elements, int dataType, Nd4jPointer primary, Nd4jPointer special) {
    auto buffer = dbAllocateDataBuffer(0, dataType, false);

    if (primary != nullptr)
        buffer->setPrimary(primary, elements);

    if (special != nullptr)
        buffer->setSpecial(special, elements);

    return buffer;
}

OpaqueDataBuffer* dbAllocateDataBuffer(Nd4jLong elements, int dataType, bool allocateBoth) {
    return allocateDataBuffer(elements, dataType, allocateBoth);
}

OpaqueDataBuffer* allocateDataBuffer(Nd4jLong elements, int dataType, bool allocateBoth) {
    try {
        auto dtype = DataTypeUtils::fromInt(dataType);
        return new sd::InteropDataBuffer(elements * DataTypeUtils::sizeOf(dtype), dtype, allocateBoth);
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
        return nullptr;
    }
}

Nd4jPointer dbPrimaryBuffer(OpaqueDataBuffer *dataBuffer) {
    return dataBuffer->primary();
}

Nd4jPointer dbSpecialBuffer(OpaqueDataBuffer *dataBuffer) {
    return dataBuffer->special();
}

void deleteDataBuffer(OpaqueDataBuffer *dataBuffer) {
    delete dataBuffer;
}

void dbSetPrimaryBuffer(OpaqueDataBuffer *dataBuffer, Nd4jPointer primaryBuffer, Nd4jLong numBytes) {
    dataBuffer->setPrimary(primaryBuffer, numBytes);
}

void dbSetSpecialBuffer(OpaqueDataBuffer *dataBuffer, Nd4jPointer specialBuffer, Nd4jLong numBytes) {
    dataBuffer->setSpecial(specialBuffer, numBytes);
}

void dbAllocatePrimaryBuffer(OpaqueDataBuffer *dataBuffer) {
    dataBuffer->dataBuffer()->allocatePrimary();
}

void dbAllocateSpecialBuffer(OpaqueDataBuffer *dataBuffer) {
    dataBuffer->dataBuffer()->allocateSpecial();
}

void dbExpandBuffer(OpaqueDataBuffer *dataBuffer, Nd4jLong elements) {
    try {
        dataBuffer->dataBuffer()->expand(elements * DataTypeUtils::sizeOf(dataBuffer->dataBuffer()->getDataType()));
    } catch (std::exception &e) {
        sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
        sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
    }
}

OpaqueDataBuffer* dbCreateView(OpaqueDataBuffer *dataBuffer, Nd4jLong length, Nd4jLong offset) {
    return new InteropDataBuffer(*dataBuffer, length, offset);
}

void dbSyncToSpecial(OpaqueDataBuffer *dataBuffer) {
    dataBuffer->dataBuffer()->syncToSpecial();
}

void dbSyncToPrimary(OpaqueDataBuffer *dataBuffer) {
    dataBuffer->dataBuffer()->syncToPrimary(nullptr);
}

void dbTickHostRead(OpaqueDataBuffer *dataBuffer) {
    dataBuffer->dataBuffer()->readPrimary();
}

void dbTickHostWrite(OpaqueDataBuffer *dataBuffer) {
    dataBuffer->dataBuffer()->writePrimary();
}

void dbTickDeviceRead(OpaqueDataBuffer *dataBuffer) {
    dataBuffer->dataBuffer()->readSpecial();
}

void dbTickDeviceWrite(OpaqueDataBuffer *dataBuffer) {
    dataBuffer->dataBuffer()->writeSpecial();
}

void dbExpand(OpaqueDataBuffer *dataBuffer, Nd4jLong elements) {
    dataBuffer->expand(elements);
}

void dbClose(OpaqueDataBuffer *dataBuffer) {
    dataBuffer->getDataBuffer()->close();
}

int dbDeviceId(OpaqueDataBuffer *dataBuffer) {
    return dataBuffer->deviceId();
}

void dbSetDeviceId(OpaqueDataBuffer *dataBuffer, int deviceId) {
    dataBuffer->setDeviceId(deviceId);
}

int dbLocality(OpaqueDataBuffer *dataBuffer) {
    auto p = dataBuffer->dataBuffer()->isPrimaryActual();
    auto d = dataBuffer->dataBuffer()->isSpecialActual();

    if (p && d)
        return 0;
    else if (p)
        return -1;
    else
        return 1;
}