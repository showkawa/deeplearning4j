#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

#include <legacy/NativeOpExecutioner.h>
#include <hip/hip_runtime.h>
#include <system/op_boilerplate.h>
#include <helpers/DebugHelper.h>
#include <array/DataTypeUtils.h>
#include <exceptions/datatype_exception.h>
#include <exceptions/cuda_exception.h>
#include <helpers/CudaLaunchHelper.h>
#include <helpers/ShapeBuilders.h>
#include <helpers/PointersManager.h>

#include <array/ConstantDataBuffer.h>
#include <array/ShapeDescriptor.h>
#include <helpers/ConstantShapeHelper.h>

#include <loops/transform_float.h>
#include <loops/transform_bool.h>
#include <loops/transform_any.h>
#include <loops/transform_same.h>
#include <loops/transform_strict.h>
#include <loops/reduce_float.h>
#include <loops/reduce_same.h>
#include <loops/reduce_bool.h>
#include <loops/reduce_long.h>
#include <loops/indexreduce.h>
#include <loops/pairwise_transform.h>
#include <loops/pairwise_bool.h>
#include <loops/pairwise_int.h>
#include <loops/broadcasting_bool.h>
#include <loops/broadcasting_int.h>
#include <loops/broadcasting.h>
#include <loops/reduce_float.h>
#include <loops/reduce3.h>
#include <loops/summarystatsreduce.h>
#include <loops/transform_same.h>
#include <loops/random.h>
#include <loops/special_kernels.h>
#include <loops/scalar.h>
#include <loops/scalar_bool.h>
#include <loops/scalar_int.h>

using namespace sd;

/**
* This is utility kernel, that updates given special buffer with proper values in device memory
*/
extern "C" __global__ void prepareShapeBuffer(int *dimension, int *maxDimension, Nd4jLong *specialPointer, int rows, sd::DataType dataType) {
    Nd4jLong tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid > 0)
        return;

    dimension[0] = 0;
    maxDimension[0] = 1;

    specialPointer[0] = 2;
    specialPointer[1] = rows;
    specialPointer[2] = 1;
    specialPointer[3] = 1;
    specialPointer[4] = 1;
    specialPointer[5] = 0;
    specialPointer[6] = 1;
    specialPointer[7] = 99;

    ArrayOptions::setDataType(specialPointer, dataType);

    //printf("special[0]: [%lld]\n", (long long) specialPointer[0]);
    //shape::printShapeInfoLinear("prepareShapeBuffer", specialPointer);
}


////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execPairwiseTransform(sd::LaunchContext  *lc,
                                    int opNum,
                                    void const* hX, Nd4jLong const* hXShapeInfo,
                                    void const* dX, Nd4jLong const* dXShapeInfo,
                                    void const* hY, Nd4jLong const* hYShapeInfo,
                                    void const* dY, Nd4jLong const* dYShapeInfo,
                                    void *hZ, Nd4jLong const* hZShapeInfo,
                                    void *dZ, Nd4jLong const* dZShapeInfo,
                                    void *extraParams) {

    auto stream = lc->getCudaStream();

    auto xType = sd::ArrayOptions::dataType(hXShapeInfo);
    auto yType = sd::ArrayOptions::dataType(hYShapeInfo);
    auto zType = sd::ArrayOptions::dataType(hZShapeInfo);

    if (shape::isEmpty(hXShapeInfo) || shape::isEmpty(hYShapeInfo))
        return;

    if (xType != zType && yType != zType)
        throw std::runtime_error("NativeOpExecutioner::execPairwiseTransform requires Z operand to have either X or Y type");
    if (lc == nullptr)
        throw std::runtime_error("NativeOpExecutioner::execPairwiseTransform: launch context cannot be nullptr !");
    if (stream == nullptr)
        throw std::runtime_error("NativeOpExecutioner::execPairwiseTransform: CUDA stream cannot be nullptr !");

    dim3 launchDims(256, 1024, 8192);

#ifdef __ND4J_EXPERIMENTAL__
    BUILD_PAIRWISE_SELECTOR(xType, yType, zType, functions::pairwise_transforms::PairWiseTransform, ::executeCudaShaped(launchDims, stream, opNum, dX, dXShapeInfo, dY, dYShapeInfo, dZ, dZShapeInfo, extraParams), LIBND4J_TYPES, LIBND4J_TYPES)
#else
    BUILD_SINGLE_SELECTOR_THRICE(xType, functions::pairwise_transforms::PairWiseTransform, ::executeCudaShaped(launchDims, stream, opNum, dX, dXShapeInfo, dY, dYShapeInfo, dZ, dZShapeInfo, extraParams), LIBND4J_TYPES)
#endif

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execPairwiseTransform failed", res);
}

////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execPairwiseBoolTransform( sd::LaunchContext  *lc,
                                                    int opNum,
                                                    void const* hX, Nd4jLong const* hXShapeInfo,
                                                    void const* dX, Nd4jLong const* dXShapeInfo,
                                                    void const* hY, Nd4jLong const* hYShapeInfo,
                                                    void const* dY, Nd4jLong const* dYShapeInfo,
                                                    void *hZ, Nd4jLong const* hZShapeInfo,
                                                    void *dZ, Nd4jLong const* dZShapeInfo,
                                                    void *extraParams) {

	auto stream = lc->getCudaStream();

    auto xType = sd::ArrayOptions::dataType(hXShapeInfo);
    auto yType = sd::ArrayOptions::dataType(hYShapeInfo);
    auto zType = sd::ArrayOptions::dataType(hZShapeInfo);

    if (shape::isEmpty(hXShapeInfo) || shape::isEmpty(hYShapeInfo))
        return;

    if (!DataTypeUtils::isB(zType))
		throw sd::datatype_exception::build("NativeOpExecutioner::execPairwiseBoolTransform wrong Z operand data type", sd::DataType::BOOL, zType);

    if (yType != xType)
        throw sd::datatype_exception::build("NativeOpExecutioner::execPairwiseBoolTransform both operands must have same data type", xType, yType);

    dim3 launchDims(256, 1024, 16384);

    BUILD_DOUBLE_SELECTOR(xType, zType, functions::pairwise_transforms::PairWiseBoolTransform, ::executeCudaShaped(launchDims, stream, opNum, dX, dXShapeInfo, dY, dYShapeInfo, dZ, dZShapeInfo, extraParams), LIBND4J_TYPES, BOOL_TYPES)

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execPairwiseBoolTransform failed", res);
}

////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execPairwiseIntTransform( sd::LaunchContext  *lc,
                                                     int opNum,
                                                     void const* hX, Nd4jLong const* hXShapeInfo,
                                                     void const* dX, Nd4jLong const* dXShapeInfo,
                                                     void const* hY, Nd4jLong const* hYShapeInfo,
                                                     void const* dY, Nd4jLong const* dYShapeInfo,
                                                     void * hZ, Nd4jLong const* hZShapeInfo,
                                                     void * dZ, Nd4jLong const* dZShapeInfo,
                                                     void *extraParams) {

    auto stream = lc->getCudaStream();

    auto xType = sd::ArrayOptions::dataType(hXShapeInfo);
    auto yType = sd::ArrayOptions::dataType(hYShapeInfo);
    auto zType = sd::ArrayOptions::dataType(hZShapeInfo);

    if (shape::isEmpty(hXShapeInfo) || shape::isEmpty(hYShapeInfo))
        return;

    if (!DataTypeUtils::isZ(zType))
        throw sd::datatype_exception::build("NativeOpExecutioner::execPairwiseIntTransform wrong Z operand data type", sd::DataType::BOOL, zType);

    if (yType != xType || zType != xType)
        throw sd::datatype_exception::build("NativeOpExecutioner::execPairwiseIntTransform both operands must have same data type", xType, yType);

    dim3 launchDims(256, 1024, 16384);

    BUILD_SINGLE_SELECTOR(xType, functions::pairwise_transforms::PairWiseIntTransform, ::executeCudaShaped(launchDims, stream, opNum, dX, dXShapeInfo, dY, dYShapeInfo, dZ, dZShapeInfo, extraParams), INTEGER_TYPES)

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execPairwiseIntTransform failed", res);
}

////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execSummaryStatsScalar(sd::LaunchContext  *lc,
                                    int opNum,
                                    void const* hX, Nd4jLong const* hXShapeInfo,
                                    void const* dX, Nd4jLong const* dXShapeInfo,
                                    void *extraParams,
                                    void *hZ, Nd4jLong const* hZShapeInfo,
                                    void *dZ, Nd4jLong const* dZShapeInfo,
                                    bool biasCorrected) {

	auto stream = lc->getCudaStream();
    auto reductionPointer = lc->getReductionPointer();

    dim3 launchDims = dim3(256, CUDA_BLOCK_SIZE, 1024);

	auto xType = sd::ArrayOptions::dataType(hXShapeInfo);
	auto zType = sd::ArrayOptions::dataType(hZShapeInfo);

    BUILD_DOUBLE_SELECTOR(xType, zType, functions::summarystats::SummaryStatsReduce, ::execSummaryStatsReduceScalar(launchDims, stream, opNum, dX, dXShapeInfo, hXShapeInfo, extraParams, dZ, dZShapeInfo, hZShapeInfo, nullptr, nullptr, biasCorrected, reductionPointer), LIBND4J_TYPES, FLOAT_TYPES);

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execSummaryStatsScalar failed", res);
}

////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execBroadcastBool(sd::LaunchContext  *lc,
                            int opNum,
                            void const* hX, Nd4jLong const* hXShapeInfo,
                            void const* dX, Nd4jLong const* dXShapeInfo,
                            void const* hY, Nd4jLong const* hYShapeInfo,
                            void const* dY, Nd4jLong const* dYShapeInfo,
                            void *hZ, Nd4jLong const* hZShapeInfo,
                            void *dZ, Nd4jLong const* dZShapeInfo,
                            void *extraParams,
                            int *dimension, int dimensionLength,
                            Nd4jLong const* tadOnlyShapeInfo,  Nd4jLong const* tadOffsets,
                            Nd4jLong const* tadOnlyShapeInfoZ, Nd4jLong const* tadOffsetsZ) {

	auto stream = lc->getCudaStream();

	auto xType = sd::ArrayOptions::dataType(hXShapeInfo);
	auto yType = sd::ArrayOptions::dataType(hYShapeInfo);
    auto zType = sd::ArrayOptions::dataType(hZShapeInfo);

    if (shape::isEmpty(hXShapeInfo) || shape::isEmpty(hYShapeInfo))
        return;

	if (!DataTypeUtils::isB(zType))
        throw std::runtime_error("NativeOpExecutioner::execBroadcastBool requires Z operand to have BOOL type");

    if (yType != xType)
        throw std::runtime_error("NativeOpExecutioner::execBroadcastBool requires both X & Y operands to have same type");

	if (sd::Environment::getInstance().isDebugAndVerbose())
		printf("F3B opNum:[%i]\n", opNum);

	dim3 launchDims(256, 256, 1024);

	BUILD_DOUBLE_SELECTOR(xType, zType, functions::broadcast::BroadcastBool, ::execBroadcast(launchDims, stream, opNum, dX, dXShapeInfo, dY, dYShapeInfo, dZ, dZShapeInfo, extraParams, dimension, dimensionLength, tadOnlyShapeInfo, tadOffsets, tadOnlyShapeInfoZ, tadOffsetsZ), LIBND4J_TYPES, BOOL_TYPES)

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execBroadcastBool failed", res);
}

////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execBroadcastBool(sd::LaunchContext* lc, const int opNum,
                                        const void *hX, const Nd4jLong *hXShapeInfo,
                                        const void *dX, const Nd4jLong *dXShapeInfo,
                                        const void *hY, const Nd4jLong *hYShapeInfo,
                                        const void *dY, const Nd4jLong *dYShapeInfo,
                                              void *hZ, const Nd4jLong *hZShapeInfo,
                                              void *dZ, const Nd4jLong *dZShapeInfo,
                                              void *extraParams) {

    if (shape::isEmpty(hXShapeInfo) || shape::isEmpty(hYShapeInfo))
        return;

    auto stream = lc->getCudaStream();

    auto xType = sd::ArrayOptions::dataType(hXShapeInfo);
    auto zType = sd::ArrayOptions::dataType(hZShapeInfo);

    dim3 launchDims;

    launchDims.y = MAX_NUM_THREADS / 4; // threadsPerBlock
    launchDims.x = (shape::length(hZShapeInfo) + launchDims.y - 1) / launchDims.y; // blocksPerGrid
    launchDims.z = 1024; // shared memory

    BUILD_DOUBLE_SELECTOR(xType, zType, functions::broadcast::BroadcastBool, ::execBroadcast(launchDims, stream, opNum, dX, dXShapeInfo, dY, dYShapeInfo, dZ, dZShapeInfo, extraParams), LIBND4J_TYPES, BOOL_TYPES);

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execBroadcastBool failed", res);
}


void NativeOpExecutioner::execInverseBroadcastBool(sd::LaunchContext  *lc,
                                                   int opNum,
                                                   void const* hX, Nd4jLong const* hXShapeInfo,
                                                   void const* dX, Nd4jLong const* dXShapeInfo,
                                                   void const* hY, Nd4jLong const* hYShapeInfo,
                                                   void const* dY, Nd4jLong const* dYShapeInfo,
                                                   void* hZ, Nd4jLong const* hZShapeInfo,
                                                   void *dZ, Nd4jLong const* dZShapeInfo,
                                                   void *extraParams,
                                                   int *dimension, int dimensionLength,
                                                   Nd4jLong const* tadOnlyShapeInfo,  Nd4jLong const* tadOffsets,
                                                   Nd4jLong const* tadOnlyShapeInfoZ, Nd4jLong const* tadOffsetsZ) {
    auto stream = lc->getCudaStream();

    auto xType = sd::ArrayOptions::dataType(hXShapeInfo);
    auto yType = sd::ArrayOptions::dataType(hYShapeInfo);
    auto zType = sd::ArrayOptions::dataType(hZShapeInfo);

    if (shape::isEmpty(hXShapeInfo) || shape::isEmpty(hYShapeInfo))
        return;

    if (!DataTypeUtils::isB(zType))
        throw std::runtime_error("NativeOpExecutioner::execBroadcastBool requires Z operand to have BOOL type");

    if (yType != xType)
        throw std::runtime_error("NativeOpExecutioner::execBroadcastBool requires both X & Y operands to have same type");

    dim3 launchDims(256, 256, 1024);

    BUILD_DOUBLE_SELECTOR(xType, zType, functions::broadcast::BroadcastBool, ::execInverseBroadcast(launchDims, stream, opNum, dX, dXShapeInfo, dY, dYShapeInfo, dZ, dZShapeInfo, extraParams, dimension, dimensionLength, tadOnlyShapeInfo, tadOffsets, tadOnlyShapeInfoZ, tadOffsetsZ), LIBND4J_TYPES, BOOL_TYPES)

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execInverseBroadcastBool failed", res);
}

////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execBroadcastInt(sd::LaunchContext  *lc,
                                            int opNum,
                                            void const* hX, Nd4jLong const* hXShapeInfo,
                                            void const* dX, Nd4jLong const* dXShapeInfo,
                                            void const* hY, Nd4jLong const* hYShapeInfo,
                                            void const* dY, Nd4jLong const* dYShapeInfo,
                                            void *hZ, Nd4jLong const* hZShapeInfo,
                                            void *dZ, Nd4jLong const* dZShapeInfo,
                                            int *dimension, int dimensionLength,
                                            Nd4jLong const* tadOnlyShapeInfo, Nd4jLong const* tadOffsets,
                                            Nd4jLong const* tadOnlyShapeInfoZ,Nd4jLong const* tadOffsetsZ) {

    auto stream = lc->getCudaStream();

    auto xType = sd::ArrayOptions::dataType(hXShapeInfo);
    auto yType = sd::ArrayOptions::dataType(hYShapeInfo);
    auto zType = sd::ArrayOptions::dataType(hZShapeInfo);

    if (shape::isEmpty(hXShapeInfo) || shape::isEmpty(hYShapeInfo))
        return;

    if (!DataTypeUtils::isZ(zType))
        throw std::runtime_error("NativeOpExecutioner::execBroadcastInt requires Z operand to have INT type");

    if (yType != xType || zType != xType)
        throw std::runtime_error("NativeOpExecutioner::execBroadcastInt requires both X & Y operands to have same type");

    dim3 launchDims(256, 256, 1024);

    BUILD_SINGLE_SELECTOR(xType, functions::broadcast::BroadcastInt, ::execBroadcast(launchDims, stream, opNum, dX, dXShapeInfo, dY, dYShapeInfo, dZ, dZShapeInfo, dimension, dimensionLength, tadOnlyShapeInfo, tadOffsets, tadOnlyShapeInfoZ, tadOffsetsZ), INTEGER_TYPES)

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execBroadcastBool failed", res);
}

////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execBroadcastInt(sd::LaunchContext* lc, const int opNum,
                                           const void *hX, const Nd4jLong *hXShapeInfo,
                                           const void *dX, const Nd4jLong *dXShapeInfo,
                                           const void *hY, const Nd4jLong *hYShapeInfo,
                                           const void *dY, const Nd4jLong *dYShapeInfo,
                                                 void *hZ, const Nd4jLong *hZShapeInfo,
                                                 void *dZ, const Nd4jLong *dZShapeInfo) {

    auto stream = lc->getCudaStream();

    auto xType = sd::ArrayOptions::dataType(hXShapeInfo);
    auto yType = sd::ArrayOptions::dataType(hYShapeInfo);
    auto zType = sd::ArrayOptions::dataType(hZShapeInfo);

    if (shape::isEmpty(hXShapeInfo) || shape::isEmpty(hYShapeInfo))
        return;

    if (!DataTypeUtils::isZ(zType))
        throw std::runtime_error("NativeOpExecutioner::execBroadcastInt requires Z operand to have INT type");

    if (yType != xType || zType != xType)
        throw std::runtime_error("NativeOpExecutioner::execBroadcastInt requires both X & Y operands to have same type");

    dim3 launchDims;

    launchDims.y = MAX_NUM_THREADS / 4; // threadsPerBlock
    launchDims.x = (shape::length(hZShapeInfo) + launchDims.y - 1) / launchDims.y; // blocksPerGrid
    launchDims.z = 1024; // shared memory

    BUILD_SINGLE_SELECTOR(xType, functions::broadcast::BroadcastInt, ::execBroadcast(launchDims, stream, opNum, dX, dXShapeInfo, dY, dYShapeInfo, dZ, dZShapeInfo), INTEGER_TYPES)

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execBroadcastBool failed", res);
}

void NativeOpExecutioner::execInverseBroadcastInt(sd::LaunchContext  *lc,
                                                   int opNum,
                                                   void const* hX, Nd4jLong const* hXShapeInfo,
                                                   void const* dX, Nd4jLong const* dXShapeInfo,
                                                   void const* hY, Nd4jLong const* hYShapeInfo,
                                                   void const* dY, Nd4jLong const* dYShapeInfo,
                                                   void *hZ, Nd4jLong const* hZShapeInfo,
                                                   void *dZ, Nd4jLong const* dZShapeInfo,
                                                   int *dimension, int dimensionLength,
                                                   Nd4jLong const* tadOnlyShapeInfo, Nd4jLong const* tadOffsets,
                                                   Nd4jLong const* tadOnlyShapeInfoZ,Nd4jLong const* tadOffsetsZ) {
    auto stream = lc->getCudaStream();

    auto xType = sd::ArrayOptions::dataType(hXShapeInfo);
    auto yType = sd::ArrayOptions::dataType(hYShapeInfo);
    auto zType = sd::ArrayOptions::dataType(hZShapeInfo);

    if (shape::isEmpty(hXShapeInfo) || shape::isEmpty(hYShapeInfo))
        return;

    if (!DataTypeUtils::isZ(zType))
        throw std::runtime_error("NativeOpExecutioner::execBroadcastInt requires Z operand to have INT type");

    if (yType != xType || zType != xType)
        throw std::runtime_error("NativeOpExecutioner::execBroadcastInt requires both X & Y operands to have same type");

    if (sd::Environment::getInstance().isDebugAndVerbose())
        printf("F3BI opNum:[%i]\n", opNum);

    dim3 launchDims(256, 256, 1024);

    BUILD_SINGLE_SELECTOR(xType, functions::broadcast::BroadcastInt, ::execInverseBroadcast(launchDims, stream, opNum, dX, dXShapeInfo, dY, dYShapeInfo, dZ, dZShapeInfo, dimension, dimensionLength, tadOnlyShapeInfo, tadOffsets, tadOnlyShapeInfoZ, tadOffsetsZ), INTEGER_TYPES)

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execInverseBroadcastInt failed", res);
}

////////////////////////////////////////////////////////////////////////
/**
 *
 * @param opNum
 * @param dX
 * @param dXShapeInfo
 * @param dY
 * @param dYShapeInfo
 * @param dZ
 * @param dZShapeInfo
 * @param dimension
 * @param dimensionLength
 */
void NativeOpExecutioner::execBroadcast(sd::LaunchContext  *lc,
		                              int opNum,
		                              void const* hX, Nd4jLong const* hXShapeInfo,
		                              void const* dX, Nd4jLong const* dXShapeInfo,
		                              void const* hY, Nd4jLong const* hYShapeInfo,
		                              void const* dY, Nd4jLong const* dYShapeInfo,
		                              void *hZ, Nd4jLong const* hZShapeInfo,
		                              void *dZ, Nd4jLong const* dZShapeInfo,
		                              int *dimension, int dimensionLength,
		                              Nd4jLong const* tadOnlyShapeInfo, Nd4jLong const* tadOffsets,
		                              Nd4jLong const* tadOnlyShapeInfoZ,Nd4jLong const* tadOffsetsZ) {

	auto stream = lc->getCudaStream();

	auto xType = sd::ArrayOptions::dataType(hXShapeInfo);
	auto yType = sd::ArrayOptions::dataType(hYShapeInfo);
    auto zType = sd::ArrayOptions::dataType(hZShapeInfo);

    if (shape::isEmpty(hXShapeInfo) || shape::isEmpty(hYShapeInfo))
        return;

	dim3 launchDims(256, 256, 1024);

#ifdef __ND4J_EXPERIMENTAL__
	BUILD_PAIRWISE_SELECTOR(xType, yType, zType, functions::broadcast::Broadcast, ::execBroadcast(launchDims, stream, opNum, dX, dXShapeInfo, dY, dYShapeInfo, dZ, dZShapeInfo, dimension, dimensionLength, tadOnlyShapeInfo, tadOffsets, tadOnlyShapeInfoZ, tadOffsetsZ), LIBND4J_TYPES, LIBND4J_TYPES);
#else
    BUILD_SINGLE_SELECTOR_THRICE(xType, functions::broadcast::Broadcast, ::execBroadcast(launchDims, stream, opNum, dX, dXShapeInfo, dY, dYShapeInfo, dZ, dZShapeInfo, dimension, dimensionLength, tadOnlyShapeInfo, tadOffsets, tadOnlyShapeInfoZ, tadOffsetsZ), LIBND4J_TYPES);
#endif

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execBroadcast failed", res);
}

////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execBroadcast(sd::LaunchContext  *lc, const int opNum,
                                      const void *hX, const Nd4jLong *hXShapeInfo,
                                      const void *dX, const Nd4jLong *dXShapeInfo,
                                      const void *hY, const Nd4jLong *hYShapeInfo,
                                      const void *dY, const Nd4jLong *dYShapeInfo,
                                            void *hZ, const Nd4jLong *hZShapeInfo,
                                            void *dZ, const Nd4jLong *dZShapeInfo) {

    auto stream = lc->getCudaStream();

    auto xType = sd::ArrayOptions::dataType(hXShapeInfo);
    auto yType = sd::ArrayOptions::dataType(hYShapeInfo);
    auto zType = sd::ArrayOptions::dataType(hZShapeInfo);

    if (shape::isEmpty(hXShapeInfo) || shape::isEmpty(hYShapeInfo))
        return;

    dim3 launchDims;

    launchDims.y = MAX_NUM_THREADS / 4; // threadsPerBlock
    launchDims.x = (shape::length(hZShapeInfo) + launchDims.y - 1) / launchDims.y; // blocksPerGrid
    launchDims.z = 1024; // shared memory

#ifdef __ND4J_EXPERIMENTAL__
    BUILD_PAIRWISE_SELECTOR(xType, yType, zType, functions::broadcast::Broadcast, ::execBroadcast(launchDims, stream, opNum, dX, dXShapeInfo, dY, dYShapeInfo, dZ, dZShapeInfo), LIBND4J_TYPES, LIBND4J_TYPES);
#else
    BUILD_SINGLE_SELECTOR_THRICE(xType, functions::broadcast::Broadcast, ::execBroadcast(launchDims, stream, opNum, dX, dXShapeInfo, dY, dYShapeInfo, dZ, dZShapeInfo), LIBND4J_TYPES);
#endif

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execBroadcast failed", res);
}

void NativeOpExecutioner::execInverseBroadcast(sd::LaunchContext  *lc,
                                               int opNum,
                                               void const* hX, Nd4jLong const* hXShapeInfo,
                                               void const* dX, Nd4jLong const* dXShapeInfo,
                                               void const* hY, Nd4jLong const* hYShapeInfo,
                                               void const* dY, Nd4jLong const* dYShapeInfo,
                                               void *hZ, Nd4jLong const* hZShapeInfo,
                                               void *dZ, Nd4jLong const* dZShapeInfo,
                                               int *dimension, int dimensionLength,
                                               Nd4jLong const* tadOnlyShapeInfo, Nd4jLong const* tadOffsets,
                                               Nd4jLong const* tadOnlyShapeInfoZ,Nd4jLong const* tadOffsetsZ) {

    auto stream = lc->getCudaStream();

    auto xType = sd::ArrayOptions::dataType(hXShapeInfo);
    auto yType = sd::ArrayOptions::dataType(hYShapeInfo);
    auto zType = sd::ArrayOptions::dataType(hZShapeInfo);

    if (shape::isEmpty(hXShapeInfo) || shape::isEmpty(hYShapeInfo))
        return;

    dim3 launchDims(256, 256, 1024);

#ifdef __ND4J_EXPERIMENTAL__
    BUILD_PAIRWISE_SELECTOR(xType, yType, zType, functions::broadcast::Broadcast, ::execInverseBroadcast(launchDims, stream, opNum, dX, dXShapeInfo, dY, dYShapeInfo, dZ, dZShapeInfo, dimension, dimensionLength, tadOnlyShapeInfo, tadOffsets, tadOnlyShapeInfoZ, tadOffsetsZ), LIBND4J_TYPES, LIBND4J_TYPES);
#else
    BUILD_SINGLE_SELECTOR_THRICE(xType, functions::broadcast::Broadcast, ::execInverseBroadcast(launchDims, stream, opNum, dX, dXShapeInfo, dY, dYShapeInfo, dZ, dZShapeInfo, dimension, dimensionLength, tadOnlyShapeInfo, tadOffsets, tadOnlyShapeInfoZ, tadOffsetsZ), LIBND4J_TYPES);
#endif

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execInverseBroadcast failed", res);
}

////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execReduceSame(sd::LaunchContext  *lc,
                            int opNum,
                            void const* hX, Nd4jLong const* hXShapeInfo,
                            void const* dX, Nd4jLong const* dXShapeInfo,
                            void *extraParams,
                            void *hZ, Nd4jLong const* hZShapeInfo,
                            void *dZ, Nd4jLong const* dZShapeInfo,
                            int *dimension, int dimensionLength) {

	auto stream = lc->getCudaStream();
	auto reductionPointer = lc->getReductionPointer();

    if (sd::Environment::getInstance().isDebugAndVerbose())
        printf("SF7 opNum:[%i]\n", opNum);

    auto xType = sd::ArrayOptions::dataType(hXShapeInfo);
    auto zType = sd::ArrayOptions::dataType(hZShapeInfo);

    if (zType != xType)
        throw datatype_exception::build("NativeOpExecutioner::execReduceSame requires both X & Z operands to have same type", xType, zType);

    auto numBlocks = shape::length(hZShapeInfo);
    dim3 launchDims(numBlocks == 0 ? 1 : numBlocks, CUDA_BLOCK_SIZE, 1024);

    BUILD_SINGLE_SELECTOR(xType, functions::reduce::ReduceSameFunction, ::execReduceXD(launchDims, stream, opNum, dX, dXShapeInfo, hXShapeInfo, extraParams, reductionPointer, dZ, dZShapeInfo, hZShapeInfo, dimension), LIBND4J_TYPES);

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execReduceSame failed", res);
}

////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execReduceLong(sd::LaunchContext  *lc,
                            int opNum,
                            void const* hX, Nd4jLong const* hXShapeInfo,
                            void const* dX, Nd4jLong const* dXShapeInfo,
                            void *extraParams,
                            void *hZ, Nd4jLong const* hZShapeInfo,
                            void *dZ, Nd4jLong const* dZShapeInfo,
                            int *dimension,int dimensionLength) {

	auto stream = lc->getCudaStream();
	auto reductionPointer = lc->getReductionPointer();

    if (sd::Environment::getInstance().isDebugAndVerbose())
        printf("LF7 opNum:[%i]\n", opNum);

    auto xType = sd::ArrayOptions::dataType(hXShapeInfo);
    auto zType = sd::ArrayOptions::dataType(hZShapeInfo);

    if (zType != sd::DataType::INT64)
        throw datatype_exception::build("NativeOpExecutioner::execReduceLong wrong Z data type", sd::DataType::INT64, zType);

    auto numBlocks = shape::length(hZShapeInfo);
    dim3 launchDims(numBlocks == 0 ? 1 : numBlocks, CUDA_BLOCK_SIZE, 1024);

    BUILD_DOUBLE_SELECTOR(xType, zType, functions::reduce::ReduceLongFunction, ::execReduceXD(launchDims, stream, opNum, dX, dXShapeInfo, hXShapeInfo, extraParams, reductionPointer, dZ, dZShapeInfo, hZShapeInfo, dimension), LIBND4J_TYPES, LONG_TYPES);

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execReduceLong failed", res);

}

////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execReduceBool(sd::LaunchContext  *lc,
                            int opNum,
                            void const* hX, Nd4jLong const* hXShapeInfo,
                            void const* dX, Nd4jLong const* dXShapeInfo,
                            void *extraParams,
                            void *hZ, Nd4jLong const* hZShapeInfo,
                            void *dZ, Nd4jLong const* dZShapeInfo,
                            int *dimension, int dimensionLength) {

	auto stream = lc->getCudaStream();
	auto reductionPointer = lc->getReductionPointer();

    if (sd::Environment::getInstance().isDebugAndVerbose())
        printf("BF7 opNum:[%i]\n", opNum);

    auto xType = sd::ArrayOptions::dataType(hXShapeInfo);
    auto zType = sd::ArrayOptions::dataType(hZShapeInfo);

    if (zType != sd::DataType::BOOL)
        throw std::runtime_error("NativeOpExecutioner::execReduceBool requires Z operand to have BOOL type");

    auto numBlocks = shape::length(hZShapeInfo);
    dim3 launchDims(numBlocks == 0 ? 1 : numBlocks, CUDA_BLOCK_SIZE, 1024);

    BUILD_DOUBLE_SELECTOR(xType, zType, functions::reduce::ReduceBoolFunction, ::execReduceXD(launchDims, stream, opNum, dX, dXShapeInfo, hXShapeInfo, extraParams, reductionPointer, dZ, dZShapeInfo, hZShapeInfo, dimension), LIBND4J_TYPES, BOOL_TYPES);

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execReduceBool failed", res);
}

////////////////////////////////////////////////////////////////////////
/**
 *
 * @param opNum
 * @param dX
 * @param dXShapeInfo
 * @param extraParams
 * @param dZ
 * @param dZShapeInfo
 */
void  NativeOpExecutioner::execReduceFloat(sd::LaunchContext  *lc,
                                          int opNum,
                                          const void *hX, const Nd4jLong *hXShapeInfo,
                                          const void *dX, const Nd4jLong *dXShapeInfo,
                                          void *extraParams,
                                          void *hZ, const Nd4jLong *hZShapeInfo,
                                          void *dZ, const Nd4jLong *dZShapeInfo,
                                          int *dimension, int dimensionLength) {

    auto stream = lc->getCudaStream();
    auto reductionPointer = lc->getReductionPointer();

    if (sd::Environment::getInstance().isDebugAndVerbose())
        printf("F8 opNum:[%i]\n", opNum);

    auto xType = sd::ArrayOptions::dataType(hXShapeInfo);
    auto zType = sd::ArrayOptions::dataType(hZShapeInfo);

    auto numBlocks = shape::length(hZShapeInfo);
    dim3 launchDims(numBlocks == 0 ? 1 : numBlocks, 256, 32768);

    BUILD_DOUBLE_SELECTOR(xType, zType, functions::reduce::ReduceFloatFunction, ::execReduceXD(launchDims, stream, opNum, dX, dXShapeInfo, hXShapeInfo, extraParams, reductionPointer, dZ, dZShapeInfo, hZShapeInfo, dimension), LIBND4J_TYPES, FLOAT_TYPES);

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execReduceFloat failed", res);
}

////////////////////////////////////////////////////////////////////////
/**
 *
 * @param opNum
 * @param dX
 * @param dXShapeInfo
 * @param extraParams
 * @param dZ
 * @param dZShapeInfo
 * @param dimension
 * @param dimensionLength
 */
void NativeOpExecutioner::execIndexReduce(sd::LaunchContext  *lc,
                                int opNum,
                                void const* hX, Nd4jLong const* hXShapeInfo,
                                void const* dX, Nd4jLong const* dXShapeInfo,
                                void *extraParams,
                                void *hZ, Nd4jLong const* hZShapeInfo,
                                void *dZ, Nd4jLong const* dZShapeInfo,
                                int *dimension, int dimensionLength,
                                Nd4jLong const* tadShapeInfo, Nd4jLong const* tadOffsets) {

	auto stream = lc->getCudaStream();
	auto reductionPointer = lc->getReductionPointer();
	auto allocationPointer = lc->getAllocationPointer();

	if (sd::Environment::getInstance().isDebugAndVerbose())
		printf("F2 opNum:[%i]\n", opNum);

	auto xType = sd::ArrayOptions::dataType(hXShapeInfo);
    auto zType = sd::ArrayOptions::dataType(hZShapeInfo);
	auto numBlocks = shape::length(hZShapeInfo);
	auto tadLength = shape::length(hXShapeInfo) / numBlocks;
    dim3 launchDims(numBlocks == 0 ? 1 : numBlocks, tadLength < CUDA_BLOCK_SIZE ? tadLength : CUDA_BLOCK_SIZE, 1024);

    if (zType != sd::DataType::INT64 && zType != sd::DataType::INT32)
        throw datatype_exception::build("NativeOpExecutioner::execIndexReduce requires Z operand to have INT32/INT64 type", zType);

	auto dz = reinterpret_cast<Nd4jLong*>(dZ);

	BUILD_DOUBLE_SELECTOR(xType, zType, functions::indexreduce::IndexReduce,  ::executeIndexReduce(launchDims, stream, opNum, dX, dXShapeInfo, shape::rank(hXShapeInfo), extraParams, dz, dZShapeInfo, shape::rank(hZShapeInfo), dimension, dimensionLength, 1, allocationPointer, reductionPointer, tadShapeInfo, tadOffsets), LIBND4J_TYPES, INDEXING_TYPES);

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execIndexReduce failed", res);
}



/**
 *
 * @param opNum
 * @param dX
 * @param dXShapeInfo
 * @param extraParams
 */
////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execIndexReduceScalar(sd::LaunchContext  *lc,
											int opNum,
											void const* hX, Nd4jLong const* hXShapeInfo,
        									void const* dX, Nd4jLong const* dXShapeInfo,
        									void *extraParams,
        									void *hZ, Nd4jLong const* hZShapeInfo,
											void *dZ, Nd4jLong const* dZShapeInfo){

	if (sd::Environment::getInstance().isDebug())
		printf("F1 opNum:[%i]\n", opNum);

	auto stream = lc->getCudaStream();
	auto reductionPointer = lc->getReductionPointer();
	auto allocationPointer = lc->getAllocationPointer();

    auto xLength = shape::length(hXShapeInfo);
    auto blockWidth = 256;
    auto numBlocks = CudaLaunchHelper::getReductionBlocks(xLength, blockWidth);
    dim3 launchDims(numBlocks == 0 ? 1 : numBlocks, CUDA_BLOCK_SIZE, 1024);

	if (sd::Environment::getInstance().isDebugAndVerbose() && launchDims.x == 1)
		printf("AF1 opNum:[%i]\n", opNum);

	auto xType = sd::ArrayOptions::dataType(hXShapeInfo);
    auto zType = sd::ArrayOptions::dataType(hZShapeInfo);

    // FIXME: we want Z to be one of integer types
	//if (!DataTypeUtils::isZ(zType))
	//    throw sd::datatype_exception("NativeOpExecutioner::execIndexReduceScalar requires Z operand to have one of integer types")
	if (zType != sd::DataType::INT64 && zType != sd::DataType::INT32)
        throw sd::datatype_exception::build("NativeOpExecutioner::execIndexReduceScalar requires Z operand to have INT32/INT64 data type", zType);

    auto dz = reinterpret_cast<Nd4jLong*>(dZ);

    BUILD_DOUBLE_SELECTOR(xType, zType, functions::indexreduce::IndexReduce, ::executeIndexReduceScalar(launchDims, stream,
                                                                                                opNum,
                                                                                                dX, dXShapeInfo, shape::rank(hXShapeInfo),
                                                                                                extraParams,
                                                                                                dz, dZShapeInfo, 0,
                                                                                                nullptr, 0,
                                                                                                1,
                                                                                                allocationPointer, reductionPointer,
                                                                                                nullptr, nullptr), LIBND4J_TYPES, INDEXING_TYPES);
    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execIndexReduceScalar failed", res);
}


////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execReduceFloatScalar(sd::LaunchContext  *lc,
                                                int opNum,
                                                void const* hX, Nd4jLong const* hXShapeInfo,
                                                void const* dX, Nd4jLong const* dXShapeInfo,
                                                void *extraParams,
                                                void *hZ, Nd4jLong const* hZShapeInfo,
                                                void *dZ, Nd4jLong const* dZShapeInfo) {

    auto stream = lc->getCudaStream();
    auto reductionPointer = lc->getReductionPointer();

    auto xType = sd::ArrayOptions::dataType(hXShapeInfo);
    auto zType = sd::ArrayOptions::dataType(hZShapeInfo);

    auto xLength = shape::length(hXShapeInfo);
    auto blockWidth = 256;
    auto numBlocks = CudaLaunchHelper::getReductionBlocks(xLength, blockWidth);
    dim3 launchDims(numBlocks == 0 ? 1 : numBlocks, CUDA_BLOCK_SIZE, 1024);

    BUILD_DOUBLE_SELECTOR(xType, zType, functions::reduce::ReduceFloatFunction, ::execReduceScalar(launchDims, stream, opNum, dX,dXShapeInfo, hXShapeInfo, extraParams, dZ,dZShapeInfo, hZShapeInfo, nullptr, 0, reductionPointer, nullptr), LIBND4J_TYPES, FLOAT_TYPES);

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execReduceFloatScalar failed", res);
}


////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execReduceBoolScalar(sd::LaunchContext  *lc,
                                        int opNum,
                                        void const* hX, Nd4jLong const* hXShapeInfo,
                                        void const* dX, Nd4jLong const* dXShapeInfo,
                                        void *extraParams,
                                        void *hZ, Nd4jLong const* hZShapeInfo,
                                        void *dZ, Nd4jLong const* dZShapeInfo) {

    auto stream = lc->getCudaStream();
    auto reductionPointer = lc->getReductionPointer();

    auto xType = sd::ArrayOptions::dataType(hXShapeInfo);
    auto zType = sd::ArrayOptions::dataType(hZShapeInfo);

    if (zType != sd::DataType::BOOL)
        throw std::runtime_error("NativeOpExecutioner::execReduceBoolScalar requires Z operand to have BOOL type");

    auto xLength = shape::length(hXShapeInfo);
    auto blockWidth = CUDA_BLOCK_SIZE;
    auto numBlocks = CudaLaunchHelper::getReductionBlocks(xLength, blockWidth);
    dim3 launchDims(numBlocks == 0 ? 1 : numBlocks, blockWidth, 1024);

    BUILD_DOUBLE_SELECTOR(xType, zType, functions::reduce::ReduceBoolFunction, ::execReduceScalar(launchDims, stream, opNum, dX, dXShapeInfo, hXShapeInfo, extraParams, dZ, dZShapeInfo, hZShapeInfo, nullptr, 0, reductionPointer, nullptr), LIBND4J_TYPES, BOOL_TYPES);

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execReduceBoolScalar failed", res);
}

////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execReduceSameScalar(sd::LaunchContext  *lc,
                                        int opNum,
                                        void const* hX, Nd4jLong const* hXShapeInfo,
                                        void const* dX, Nd4jLong const* dXShapeInfo,
                                        void *extraParams,
                                        void *hZ, Nd4jLong const* hZShapeInfo,
                                        void *dZ, Nd4jLong const* dZShapeInfo) {

    auto stream = lc->getCudaStream();
    auto reductionPointer = lc->getReductionPointer();

    auto xType = sd::ArrayOptions::dataType(hXShapeInfo);
    auto zType = sd::ArrayOptions::dataType(hZShapeInfo);

    if (zType != xType)
        throw datatype_exception::build("NativeOpExecutioner::execReduceSameScalar requires both X & Z operands to have same type", xType, zType);

    auto xLength = shape::length(hXShapeInfo);
    auto blockWidth = CUDA_BLOCK_SIZE;
    auto numBlocks = CudaLaunchHelper::getReductionBlocks(xLength, blockWidth);
    dim3 launchDims(numBlocks == 0 ? 1 : numBlocks, blockWidth, 1024);

    BUILD_SINGLE_SELECTOR(xType, functions::reduce::ReduceSameFunction, ::execReduceScalar(launchDims, stream, opNum, dX, dXShapeInfo, hXShapeInfo, extraParams, dZ, dZShapeInfo, hZShapeInfo, nullptr, 0, reductionPointer, nullptr), LIBND4J_TYPES);

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execReduceSameScalar failed", res);
}

////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execReduceLongScalar(sd::LaunchContext  *lc,
                                    int opNum,
                                    void const* hX, Nd4jLong const* hXShapeInfo,
                                    void const* dX, Nd4jLong const* dXShapeInfo,
                                    void *extraParams,
                                    void *hZ, Nd4jLong const* hZShapeInfo,
                                    void *dZ, Nd4jLong const* dZShapeInfo) {

    auto stream = lc->getCudaStream();
    auto reductionPointer = lc->getReductionPointer();

    auto xType = sd::ArrayOptions::dataType(hXShapeInfo);
    auto zType = sd::ArrayOptions::dataType(hZShapeInfo);

    if (zType != sd::DataType::INT64)
        throw datatype_exception::build("NativeOpExecutioner::execReduceLongScalar wrong Z data type", sd::DataType::INT64, zType);

    auto xLength = shape::length(hXShapeInfo);
    auto blockWidth = CUDA_BLOCK_SIZE;
    auto numBlocks = CudaLaunchHelper::getReductionBlocks(xLength, blockWidth);
    dim3 launchDims(numBlocks == 0 ? 1 : numBlocks, blockWidth, 1024);

    BUILD_DOUBLE_SELECTOR(xType, zType, functions::reduce::ReduceLongFunction, ::execReduceScalar(launchDims, stream, opNum, dX, dXShapeInfo, hXShapeInfo, extraParams, dZ, dZShapeInfo, hZShapeInfo, nullptr, 0, reductionPointer, nullptr), LIBND4J_TYPES, LONG_TYPES);

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execReduceLongScalar failed", res);
}

////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execTransformSame(sd::LaunchContext  *lc,
									int opNum,
                                   	void const* hX, Nd4jLong const* hXShapeInfo,
                                   	void const* dX, Nd4jLong const* dXShapeInfo,
                                   	void *hZ, Nd4jLong const* hZShapeInfo,
                                   	void *dZ, Nd4jLong const* dZShapeInfo,
                                   	void *extraParams,
                                   	Nd4jLong const* tadShapeInfo, Nd4jLong const* tadOffsets) {

    auto stream = lc->getCudaStream();

    auto xRank = shape::rank(hXShapeInfo);
    auto zRank = shape::rank(hZShapeInfo);
    auto xType = ArrayOptions::dataType(hXShapeInfo);
    auto zType = ArrayOptions::dataType(hZShapeInfo);

    if (shape::isEmpty(hXShapeInfo)) {
        return;
    }

    if (xType != zType) {
        throw std::runtime_error("NativeOpExecutioner::execTransformSame requires X & Z to have same type");
    }

    dim3 launchDims(512, 512, 16384);
    BUILD_SINGLE_SELECTOR(xType, functions::transform::TransformSame, ::executeTransformShaped(launchDims, stream, opNum, dX, dXShapeInfo, xRank, extraParams, dZ, dZShapeInfo, zRank, nullptr, nullptr, nullptr, nullptr), LIBND4J_TYPES);

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execTransformSame failed", res);
}

////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execTransformBool(sd::LaunchContext  *lc,
                                int opNum,
                                void const* hX, Nd4jLong const* hXShapeInfo,
                                void const* dX, Nd4jLong const* dXShapeInfo,
                                void *hZ, Nd4jLong const* hZShapeInfo,
                                void *dZ, Nd4jLong const* dZShapeInfo,
                                void *extraParams,
                                Nd4jLong const* tadShapeInfo, Nd4jLong const* tadOffsets) {

	auto stream = lc->getCudaStream();

    auto xRank = shape::rank(hXShapeInfo);
    auto zRank = shape::rank(hZShapeInfo);
    auto xType = ArrayOptions::dataType(hXShapeInfo);
    auto zType = ArrayOptions::dataType(hZShapeInfo);

    if (shape::isEmpty(hXShapeInfo)) {
        return;
    }

    if (!DataTypeUtils::isB(zType)) {
        throw std::runtime_error("NativeOpExecutioner::execTransformBool requires Z to have same boolean type");
    }

    dim3 launchDims(512, 512, 16384);
    BUILD_DOUBLE_SELECTOR(xType, zType, functions::transform::TransformBool, ::executeTransformShaped(launchDims, stream, opNum, dX, dXShapeInfo, xRank, extraParams, dZ, dZShapeInfo, zRank, nullptr, nullptr, nullptr, nullptr), LIBND4J_TYPES, BOOL_TYPES);

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execTransformBool failed", res);
}

////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execTransformAny(sd::LaunchContext  *lc,
                                		int opNum,
                                		void const* hX, Nd4jLong const* hXShapeInfo,
                                		void const* dX, Nd4jLong const* dXShapeInfo,
                                		void *hZ, Nd4jLong const* hZShapeInfo,
                                		void *dZ, Nd4jLong const* dZShapeInfo,
                                		void *extraParams,
                                		Nd4jLong const* tadShapeInfo, Nd4jLong const* tadOffsets, bool allowParallelism) {

	auto stream = lc->getCudaStream();

	auto xRank = shape::rank(hXShapeInfo);
	auto zRank = shape::rank(hZShapeInfo);
	auto xType = ArrayOptions::dataType(hXShapeInfo);
	auto zType = ArrayOptions::dataType(hZShapeInfo);

    if (shape::isEmpty(hXShapeInfo))
        return;

    if (opNum == sd::transform::Assign && shape::order(hXShapeInfo) == shape::order(hZShapeInfo) && shape::order(hXShapeInfo) == 'c' && xType == zType && shape::elementWiseStride(hXShapeInfo) == 1 && shape::elementWiseStride(hZShapeInfo) == 1) {
        hipMemcpyAsync(dZ, dX, shape::length(hXShapeInfo) * sd::DataTypeUtils::sizeOfElement(xType), hipMemcpyDeviceToDevice, *stream);
    }
    else {

        dim3 launchDims(512, 512, 2048);
        BUILD_DOUBLE_SELECTOR(xType, zType, functions::transform::TransformAny, ::executeTransformShaped(launchDims, stream, opNum, dX, dXShapeInfo, xRank, extraParams, dZ, dZShapeInfo, zRank, nullptr, nullptr, nullptr, nullptr), LIBND4J_TYPES, LIBND4J_TYPES);
    }

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execTransformAny failed", res);
}

////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execTransformStrict(sd::LaunchContext  *lc,
                                    int opNum,
                                    void const* hX, Nd4jLong const* hXShapeInfo,
                                    void const* dX, Nd4jLong const* dXShapeInfo,
                                    void *hZ, Nd4jLong const* hZShapeInfo,
                                    void *dZ, Nd4jLong const* dZShapeInfo,
                                    void *extraParams,
                                    Nd4jLong const* tadShapeInfo, Nd4jLong const* tadOffsets) {

    auto stream = lc->getCudaStream();

    auto xRank = shape::rank(hXShapeInfo);
    auto zRank = shape::rank(hZShapeInfo);
    auto xType = ArrayOptions::dataType(hXShapeInfo);
    auto zType = ArrayOptions::dataType(hZShapeInfo);

    if (shape::isEmpty(hXShapeInfo)) {
        return;
    }

    if (xType != zType || !DataTypeUtils::isR(xType)) {
        throw datatype_exception::build("NativeOpExecutioner::execTransformStrict requires X & Z to have same floating point type", xType, zType);
    }

    dim3 launchDims(512, 512, 16384);
    BUILD_SINGLE_SELECTOR(xType, functions::transform::TransformStrict, ::executeTransformShaped(launchDims, stream, opNum, dX, dXShapeInfo, xRank, extraParams, dZ, dZShapeInfo, zRank, nullptr, nullptr, nullptr, nullptr), FLOAT_TYPES);

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execTransformStrict failed", res);
}

////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execTransformFloat(sd::LaunchContext  *lc,
                                int opNum,
                                void const* hX, Nd4jLong const* hXShapeInfo,
                                void const* dX, Nd4jLong const* dXShapeInfo,
                                void *hZ, Nd4jLong const* hZShapeInfo,
                                void *dZ, Nd4jLong const* dZShapeInfo,
                                void *extraParams,
                                Nd4jLong const* tadShapeInfo, Nd4jLong const* tadOffsets) {

    auto stream = lc->getCudaStream();
    auto reductionPointer = lc->getReductionPointer();

    auto xRank = shape::rank(hXShapeInfo);
    auto zRank = shape::rank(hZShapeInfo);
    auto xType = ArrayOptions::dataType(hXShapeInfo);
    auto zType = ArrayOptions::dataType(hZShapeInfo);

    if (shape::isEmpty(hXShapeInfo))
        return;

    if (!DataTypeUtils::isR(zType))
        throw datatype_exception::build("NativeOpExecutioner::execTransformFloat requires Z to have floating point type", zType);

    dim3 launchDims(512, 512, 2048);
    BUILD_DOUBLE_SELECTOR(xType, zType, functions::transform::TransformFloat, ::executeTransformShaped(launchDims, stream, opNum, dX, dXShapeInfo, xRank, extraParams, dZ, dZShapeInfo, zRank, nullptr, nullptr, nullptr, nullptr), LIBND4J_TYPES, FLOAT_TYPES);

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execTransformFloat failed", res);
}


////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execSummaryStats(sd::LaunchContext  *lc,
                                int opNum,
                                void const* hX, Nd4jLong const* hXShapeInfo,
                                void const* dX, Nd4jLong const* dXShapeInfo,
                                void *extraParams,
                                void *hZ, Nd4jLong const* hZShapeInfo,
                                void *dZ, Nd4jLong const* dZShapeInfo,
                                bool biasCorrected) {

    auto stream = lc->getCudaStream();
    auto reductionPointer = lc->getReductionPointer();

    dim3 launchDims = dim3(256, CUDA_BLOCK_SIZE, 1024);

	auto xType = sd::ArrayOptions::dataType(hXShapeInfo);
	auto zType = sd::ArrayOptions::dataType(hZShapeInfo);

    if (!DataTypeUtils::isR(zType))
        throw sd::datatype_exception::build("NativeOpExecutioner::execSummaryStats requires Z operand to have floating point data type", zType);

    BUILD_DOUBLE_SELECTOR(xType, zType, functions::summarystats::SummaryStatsReduce, ::execSummaryStatsReduce(launchDims, stream, opNum, dX, dXShapeInfo, hXShapeInfo, extraParams, dZ, dZShapeInfo, hZShapeInfo, nullptr, nullptr, biasCorrected, reductionPointer), LIBND4J_TYPES, FLOAT_TYPES);

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execSummaryStats A failed", res);
}

////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execSummaryStats(sd::LaunchContext  *lc,
                                			int opNum,
                                			void const* hX, Nd4jLong const* hXShapeInfo,
                                			void const* dX, Nd4jLong const* dXShapeInfo,
                                			void *extraParams,
                                			void *hZ, Nd4jLong const* hZShapeInfo,
                                			void *dZ, Nd4jLong const* dZShapeInfo,
                                			int *dimension, int dimensionLength,
                                            Nd4jLong const* tadShapeInfo, Nd4jLong const* tadOffsets,
                                			bool biasCorrected) {
	auto stream = lc->getCudaStream();
	auto reductionPointer = lc->getReductionPointer();

    dim3 launchDims = dim3(256, CUDA_BLOCK_SIZE, 1024);

	auto xType = sd::ArrayOptions::dataType(hXShapeInfo);
	auto zType = sd::ArrayOptions::dataType(hZShapeInfo);

    if (!DataTypeUtils::isR(zType))
        throw sd::datatype_exception::build("NativeOpExecutioner::execSummaryStats requires Z operand to have floating point data type", zType);

    BUILD_DOUBLE_SELECTOR(xType, zType, functions::summarystats::SummaryStatsReduce, ::execSummaryStatsReduce(launchDims, stream, opNum, dX, dXShapeInfo, hXShapeInfo, extraParams, dZ, dZShapeInfo, hZShapeInfo, dimension, dimensionLength, tadShapeInfo, tadOffsets, biasCorrected, reductionPointer), LIBND4J_TYPES, FLOAT_TYPES);

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execSummaryStats B failed", res);
}


////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execReduce3(sd::LaunchContext  *lc,
                            int opNum,
                            void const* hX, Nd4jLong const* hXShapeInfo,
                            void const* dX, Nd4jLong const* dXShapeInfo,
                            void *extraParams,
                            void const* hY, Nd4jLong const* hYShapeInfo,
                            void const* dY, Nd4jLong const* dYShapeInfo,
                            void *hZ, Nd4jLong const* hZShapeInfo,
                            void *dZ, Nd4jLong const* dZShapeInfo) {

	auto stream = lc->getCudaStream();
    auto reductionPointer = lc->getReductionPointer();
	auto allocationPointer = lc->getAllocationPointer();

    auto xType = sd::ArrayOptions::dataType(hXShapeInfo);
    auto yType = sd::ArrayOptions::dataType(hYShapeInfo);
    auto zType = sd::ArrayOptions::dataType(hZShapeInfo);

    auto blockWidth = CUDA_BLOCK_SIZE;
    auto numBlocks = CudaLaunchHelper::getReductionBlocks(shape::length(hXShapeInfo), blockWidth);
    dim3 launchDims(numBlocks == 0 ? 1 : numBlocks, blockWidth, 1024);

    if (xType != yType)
        throw sd::datatype_exception::build("NativeOpExecutioner::execReduce3 requires Y operand to have X type", xType, yType);

    if (!DataTypeUtils::isR(zType))
        throw sd::datatype_exception::build("NativeOpExecutioner::execReduce3 requires Z operand to have floating point data type", zType);

    BUILD_DOUBLE_SELECTOR(xType, zType, functions::reduce3::Reduce3, ::execScalar(launchDims, stream, opNum, dX, dXShapeInfo, dY, dYShapeInfo, extraParams, dZ, dZShapeInfo, allocationPointer, reductionPointer, nullptr), LIBND4J_TYPES, FLOAT_TYPES);

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execReduce3 failed", res);
}

////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execReduce3(sd::LaunchContext  *lc,
                            int opNum,
                            void const* hX, Nd4jLong const* hXShapeInfo,
                            void const* dX, Nd4jLong const* dXShapeInfo,
                            void *extraParams,
                            void const* hY, Nd4jLong const* hYShapeInfo,
                            void const* dY, Nd4jLong const* dYShapeInfo,
                            void *hZ, Nd4jLong const* hZShapeInfo,
                            void *dZ, Nd4jLong const* dZShapeInfo,
                            int *dimension, int dimensionLength,
                            Nd4jLong const*  tadOnlyShapeInfo,  Nd4jLong const* tadOffsets,
                            Nd4jLong const*  yTadOnlyShapeInfo, Nd4jLong const* yTadOffsets) {

    if(shape::isScalar(hZShapeInfo)) {
        NativeOpExecutioner::execReduce3(lc, opNum, hX, hXShapeInfo, dX, dXShapeInfo, extraParams, hY, hYShapeInfo, dY, dYShapeInfo, hZ, hZShapeInfo, dZ, dZShapeInfo);
        return;
    }

    auto stream = lc->getCudaStream();
    auto allocationPointer = lc->getAllocationPointer();

    auto xType = sd::ArrayOptions::dataType(hXShapeInfo);
    auto yType = sd::ArrayOptions::dataType(hYShapeInfo);
    auto zType = sd::ArrayOptions::dataType(hZShapeInfo);

     if (xType != yType)
        throw sd::datatype_exception::build("NativeOpExecutioner::execReduce3 requires Y operand to have X type", xType, yType);

    if (!DataTypeUtils::isR(zType))
        throw sd::datatype_exception::build("NativeOpExecutioner::execReduce3 requires Z operand to have floating point data type", zType);


    auto numBlocks = shape::length(hZShapeInfo);
    dim3 launchDims(numBlocks == 0 ? 1 : numBlocks, CUDA_BLOCK_SIZE, 1024);

    BUILD_DOUBLE_SELECTOR(xType, zType, functions::reduce3::Reduce3, ::exec(launchDims, stream, opNum,
                                                                    dX, dXShapeInfo,
                                                                    dY, dYShapeInfo,
                                                                    extraParams,
                                                                    dZ, dZShapeInfo,
                                                                    dimension, dimensionLength,
                                                                    1,
                                                                    allocationPointer,
                                                                    tadOnlyShapeInfo, tadOffsets,
                                                                    yTadOnlyShapeInfo, yTadOffsets), LIBND4J_TYPES, FLOAT_TYPES);

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execReduce3 B failed", res);
}

////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execReduce3Scalar(sd::LaunchContext  *lc,
								  int opNum,
                                  void const* hX, Nd4jLong const* hXShapeInfo,
                                  void const* dX, Nd4jLong const* dXShapeInfo,
                                  void *extraParams,
                                  void const* hY, Nd4jLong const* hYShapeInfo,
                                  void const* dY, Nd4jLong const* dYShapeInfo,
                                  void *hZ, Nd4jLong const* hZShapeInfo,
                                  void *dZ, Nd4jLong const* dZShapeInfo) {


	auto stream 		   = lc->getCudaStream();
	auto allocationPointer = lc->getAllocationPointer();
	auto reductionPointer  = lc->getReductionPointer();

    auto xType = sd::ArrayOptions::dataType(hXShapeInfo);
    auto yType = sd::ArrayOptions::dataType(hYShapeInfo);
    auto zType = sd::ArrayOptions::dataType(hZShapeInfo);

    auto xLength = shape::length(hXShapeInfo);
    auto blockWidth = CUDA_BLOCK_SIZE;
    auto numBlocks = CudaLaunchHelper::getReductionBlocks(xLength, blockWidth);
    dim3 launchDims(numBlocks == 0 ? 1 : numBlocks, blockWidth, 1024);

    if (xType != yType)
        throw sd::datatype_exception::build("NativeOpExecutioner::execReduce3Scalar requires Y operand to have X type", xType, yType);

    if (!DataTypeUtils::isR(zType))
        throw sd::datatype_exception::build("NativeOpExecutioner::execReduce3Scalar requires Z operand to have floating point data type", zType);

    BUILD_DOUBLE_SELECTOR(xType, zType, functions::reduce3::Reduce3, ::execScalar(launchDims, stream, opNum, dX, dXShapeInfo, dY, dYShapeInfo, extraParams, dZ, dZShapeInfo, allocationPointer, reductionPointer, nullptr), LIBND4J_TYPES, FLOAT_TYPES);

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execReduce3Scalar failed", res);
}


////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execScalarBool(sd::LaunchContext  *lc,
										int opNum,
										void const* hX, Nd4jLong const* hXShapeInfo,
										void const* dX, Nd4jLong const* dXShapeInfo,
										void *hZ, Nd4jLong const* hZShapeInfo,
										void *dZ, Nd4jLong const* dZShapeInfo,
										void const* hScalar, Nd4jLong const* hScalarShapeInfo,
										void const* dScalar, Nd4jLong const* dScalarShapeInfo,
										void *extraParams, bool allowParallelism) {

	auto stream = lc->getCudaStream();

	dim3 launchDims = dim3(256, 512, 8192);

	auto xType = sd::ArrayOptions::dataType(hXShapeInfo);
	auto yType = sd::ArrayOptions::dataType(hScalarShapeInfo);
	auto zType = sd::ArrayOptions::dataType(hZShapeInfo);

    if (shape::isEmpty(hXShapeInfo) || shape::isEmpty(hScalarShapeInfo))
        return;

	if (xType != yType )
		throw std::runtime_error("NativeOpExecutioner::execScalarBool requires X & Y to have same type");

	if (!DataTypeUtils::isB(zType) )
		throw std::runtime_error("NativeOpExecutioner::execScalarBool requires Z operand to have BOOL type");

	BUILD_DOUBLE_SELECTOR(xType, zType, functions::scalar::ScalarBoolTransform, ::executeCudaShaped(launchDims, stream, opNum, dX, dXShapeInfo, dZ, dZShapeInfo, dScalar, extraParams), LIBND4J_TYPES, BOOL_TYPES);

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execScalarBool failed", res);
}

////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execScalarBool(sd::LaunchContext  *lc,
						   				int opNum,
						   				void const* hX, Nd4jLong const* hXShapeInfo,
						   				void const* dX, Nd4jLong const* dXShapeInfo,
                                        void *extraParams,
						   				void *hZ, Nd4jLong const* hZShapeInfo,
						   				void *dZ, Nd4jLong const* dZShapeInfo,
						   				void const* hScalars, Nd4jLong const* hScalarShapeInfo,
						   				void const* dScalars, Nd4jLong const* dScalarShapeInfo,
						   				int *dimension, int dimensionLength,
                           				Nd4jLong const* tadShapeInfo,  Nd4jLong const* tadOffsets,
                           				Nd4jLong const* tadShapeInfoZ, Nd4jLong const* tadOffsetsZ) {

	auto stream = lc->getCudaStream();

	dim3 launchDims(256, 512, 8192);

	auto xType = sd::ArrayOptions::dataType(hXShapeInfo);
	auto yType = sd::ArrayOptions::dataType(hScalarShapeInfo);
	auto zType = sd::ArrayOptions::dataType(hZShapeInfo);

    if (shape::isEmpty(hXShapeInfo) || shape::isEmpty(hScalarShapeInfo))
        return;

	if (xType != yType )
		throw std::runtime_error("NativeOpExecutioner::execScalarBool requires X & Y to have same type");

	if (!DataTypeUtils::isB(zType) )
		throw std::runtime_error("NativeOpExecutioner::execScalarBool requires Z operand to have BOOL type");

	BUILD_DOUBLE_SELECTOR(xType, zType, functions::scalar::ScalarBoolTransform, ::executeCudaAlongDimension(launchDims, stream, opNum, dX, dXShapeInfo, dZ, dZShapeInfo, dScalars, extraParams, dimension, dimensionLength, tadShapeInfo, tadOffsets, tadShapeInfoZ, tadOffsetsZ), LIBND4J_TYPES, BOOL_TYPES);

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execScalarBool B failed", res);
}

////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execScalarInt(sd::LaunchContext  *lc,
                                         int opNum,
                                         void const* hX, Nd4jLong const* hXShapeInfo,
                                         void const* dX, Nd4jLong const* dXShapeInfo,
                                         void *hZ, Nd4jLong const* hZShapeInfo,
                                         void *dZ, Nd4jLong const* dZShapeInfo,
                                         void const* hScalar, Nd4jLong const* hScalarShapeInfo,
                                         void const* dScalar, Nd4jLong const* dScalarShapeInfo,
                                         void *extraParams, bool allowParallelism) {

    auto stream = lc->getCudaStream();

    dim3 launchDims = dim3(256, 512, 8192);

    auto xType = sd::ArrayOptions::dataType(hXShapeInfo);
    auto yType = sd::ArrayOptions::dataType(hScalarShapeInfo);
    auto zType = sd::ArrayOptions::dataType(hZShapeInfo);

    if (shape::isEmpty(hXShapeInfo) || shape::isEmpty(hScalarShapeInfo))
        return;

    if (xType != yType || zType != xType)
        throw std::runtime_error("NativeOpExecutioner::execScalarInt requires X & Y to have same type");

    if (!DataTypeUtils::isZ(zType) )
        throw std::runtime_error("NativeOpExecutioner::execScalarInt requires Z operand to have INT type");

    BUILD_SINGLE_SELECTOR(xType, functions::scalar::ScalarIntTransform, ::executeCudaShaped(launchDims, stream, opNum, dX, dXShapeInfo, dZ, dZShapeInfo, dScalar, extraParams), INTEGER_TYPES);

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execScalarInt failed", res);
}

////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execScalarInt(sd::LaunchContext  *lc,
                                         int opNum,
                                         void const* hX, Nd4jLong const* hXShapeInfo,
                                         void const* dX, Nd4jLong const* dXShapeInfo,
                                         void *extraParams,
                                         void *hZ, Nd4jLong const* hZShapeInfo,
                                         void *dZ, Nd4jLong const* dZShapeInfo,
                                         void const* hScalars, Nd4jLong const* hScalarShapeInfo,
                                         void const* dScalars, Nd4jLong const* dScalarShapeInfo,
                                         int *dimension, int dimensionLength,
                                         Nd4jLong const* tadShapeInfo,  Nd4jLong const* tadOffsets,
                                         Nd4jLong const* tadShapeInfoZ, Nd4jLong const* tadOffsetsZ) {

    auto stream = lc->getCudaStream();

    dim3 launchDims(256, 512, 8192);

    auto xType = sd::ArrayOptions::dataType(hXShapeInfo);
    auto yType = sd::ArrayOptions::dataType(hScalarShapeInfo);
    auto zType = sd::ArrayOptions::dataType(hZShapeInfo);

    if (shape::isEmpty(hXShapeInfo) || shape::isEmpty(hScalarShapeInfo))
        return;

    if (xType != yType || zType != xType)
        throw std::runtime_error("NativeOpExecutioner::execScalarInt requires X & Y to have same type");

    if (!DataTypeUtils::isZ(zType) )
        throw std::runtime_error("NativeOpExecutioner::execScalarInt requires Z operand to have INT type");

    BUILD_SINGLE_SELECTOR(xType, functions::scalar::ScalarIntTransform, ::executeCudaAlongDimension(launchDims, stream, opNum, dX, dXShapeInfo, dZ, dZShapeInfo, dScalars, extraParams, dimension, dimensionLength, tadShapeInfo, tadOffsets, tadShapeInfoZ, tadOffsetsZ), INTEGER_TYPES);

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execScalarInt B failed", res);
}

////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execScalar(sd::LaunchContext  *lc,
									int opNum,
									void const* hX, Nd4jLong const* hXShapeInfo,
									void const* dX, Nd4jLong const* dXShapeInfo,
									void* hZ, Nd4jLong const* hZShapeInfo,
									void* dZ, Nd4jLong const* dZShapeInfo,
									void const* hScalar, Nd4jLong const* hScalarShapeInfo,
									void const* dScalar, Nd4jLong const* dScalarShapeInfo,
									void *extraParams, bool allowParallelism) {

	auto stream = lc->getCudaStream();

	dim3 launchDims(256, 512, 8192);

	auto xType = sd::ArrayOptions::dataType(hXShapeInfo);
	auto yType = sd::ArrayOptions::dataType(hScalarShapeInfo);
	auto zType = sd::ArrayOptions::dataType(hZShapeInfo);

    if (shape::isEmpty(hXShapeInfo) || shape::isEmpty(hScalarShapeInfo))
        return;


#ifdef __ND4J_EXPERIMENTAL__
	BUILD_PAIRWISE_SELECTOR(xType, yType, zType, functions::scalar::ScalarTransform, ::executeCudaShaped(launchDims, stream, opNum, dX, dXShapeInfo, hXShapeInfo, dZ, dZShapeInfo, hZShapeInfo, dScalar, extraParams), LIBND4J_TYPES, LIBND4J_TYPES);
#else
	BUILD_SINGLE_SELECTOR_THRICE(xType, functions::scalar::ScalarTransform, ::executeCudaShaped(launchDims, stream, opNum, dX, dXShapeInfo, hXShapeInfo, dZ, dZShapeInfo, hZShapeInfo, dScalar, extraParams), LIBND4J_TYPES);
#endif

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execScalar failed", res);
}

////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execScalar(sd::LaunchContext  *lc,
					 				int opNum,
					 				void const* hX, Nd4jLong const* hXShapeInfo,
                     				void const* dX, Nd4jLong const* dXShapeInfo,
                                    void *extraParams,
                     				void *hZ, Nd4jLong const* hZShapeInfo,
                     				void *dZ, Nd4jLong const* dZShapeInfo,
                     				void const* hScalars, Nd4jLong const* hScalarShapeInfo,
                     				void const* dScalars, Nd4jLong const* dScalarShapeInfo,
					 				int *dimension, int dimensionLength,
                     				Nd4jLong const* tadShapeInfo,  Nd4jLong const* tadOffsets,
                     				Nd4jLong const* tadShapeInfoZ, Nd4jLong const* tadOffsetsZ) {

    auto stream = lc->getCudaStream();

    auto xType = sd::ArrayOptions::dataType(hXShapeInfo);
    auto yType = sd::ArrayOptions::dataType(hScalarShapeInfo);
    auto zType = sd::ArrayOptions::dataType(hZShapeInfo);

    if (shape::isEmpty(hXShapeInfo) || shape::isEmpty(hScalarShapeInfo))
        return;

	dim3 launchDims(256, 256, 16384);

#ifdef __ND4J_EXPERIMENTAL__
    BUILD_PAIRWISE_SELECTOR(xType, yType, zType, functions::scalar::ScalarTransform, ::executeCudaAlongDimension(launchDims, stream, opNum, dX, dXShapeInfo, dZ, dZShapeInfo, dScalars, extraParams, dimension, dimensionLength, tadShapeInfo, tadOffsets, tadShapeInfoZ, tadOffsetsZ), LIBND4J_TYPES, LIBND4J_TYPES);
#else
	BUILD_SINGLE_SELECTOR_THRICE(xType, functions::scalar::ScalarTransform, ::executeCudaAlongDimension(launchDims, stream, opNum, dX, dXShapeInfo, dZ, dZShapeInfo, dScalars, extraParams, dimension, dimensionLength, tadShapeInfo, tadOffsets, tadShapeInfoZ, tadOffsetsZ), LIBND4J_TYPES);
#endif

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execScalar B failed", res);
}

////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execRandom(sd::LaunchContext  *lc,
						  int opNum,
                          Nd4jPointer stateHost,
                          void *hZ, Nd4jLong const* hZShapeInfo,
                          void *dZ, Nd4jLong const* dZShapeInfo,
                          void *extraArguments) {

    auto stream = lc->getCudaStream();
    auto sizeOf = sizeof(sd::graph::RandomGenerator);
    Nd4jPointer stateDevice;

    hipError_t res = hipMalloc(reinterpret_cast<void **>(&stateDevice), sizeOf);
    checkCudaErrors(hipStreamSynchronize(*stream));
    checkCudaErrors(hipMemcpyAsync(stateDevice, stateHost, sizeOf, hipMemcpyHostToDevice, *stream));

    dim3 launchDims = dim3(512, 512, 32768);
    auto zType = sd::ArrayOptions::dataType(hZShapeInfo);

    auto rng = reinterpret_cast<sd::graph::RandomGenerator*>(stateHost);

    // functions::random::RandomFunction<float>::executeCudaSingle(launchDims, extraPointers, opNum, stateHost, dZ, dZShapeInfo, extraArguments),
    BUILD_SINGLE_SELECTOR(zType, functions::random::RandomFunction, ::executeCudaSingle(launchDims, stream, opNum, stateDevice, dZ, dZShapeInfo, extraArguments), FLOAT_TYPES);

    res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execRandom X failed", res);

    hipFree(stateDevice);

    rng->rewindH(shape::length(hZShapeInfo));
}

////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execRandom(sd::LaunchContext  *lc,
							int opNum,
							Nd4jPointer stateHost,
						   	void const* hX, Nd4jLong const* hXShapeInfo,
						   	void const* dX, Nd4jLong const* dXShapeInfo,
						   	void *hZ, Nd4jLong const* hZShapeInfo,
						   	void *dZ, Nd4jLong const* dZShapeInfo,
						   	void *extraArguments) {

    auto stream = lc->getCudaStream();

    auto sizeOf = sizeof(sd::graph::RandomGenerator);
    Nd4jPointer stateDevice;

    hipError_t res = hipMalloc(reinterpret_cast<void **>(&stateDevice), sizeOf);
    checkCudaErrors(hipStreamSynchronize(*stream));
    checkCudaErrors(hipMemcpyAsync(stateDevice, stateHost, sizeOf, hipMemcpyHostToDevice, *stream));

    auto rng = reinterpret_cast<sd::graph::RandomGenerator*>(stateHost);

    dim3 launchDims = dim3(512, 512, 32768);
    auto xType = sd::ArrayOptions::dataType(hZShapeInfo);
    // functions::random::RandomFunction<float>::executeCudaDouble(launchDims, extraPointers, opNum, stateHost, dX, dXShapeInfo, dZ, dZShapeInfo, extraArguments);
    BUILD_SINGLE_SELECTOR(xType, functions::random::RandomFunction, ::executeCudaDouble(launchDims, stream, opNum, stateDevice, dX, dXShapeInfo, dZ, dZShapeInfo, extraArguments), FLOAT_TYPES);

    res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execRandom XY failed", res);

    hipFree(stateDevice);

    rng->rewindH(shape::length(hZShapeInfo));
}

////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execRandom(sd::LaunchContext  *lc,
							int opNum,
							Nd4jPointer stateHost,
							void const* hX, Nd4jLong const* hXShapeInfo,
							void const* dX, Nd4jLong const* dXShapeInfo,
							void const* hY, Nd4jLong const* hYShapeInfo,
							void const* dY, Nd4jLong const* dYShapeInfo,
							void *hZ, Nd4jLong const* hZShapeInfo,
							void *dZ, Nd4jLong const* dZShapeInfo,
							void *extraArguments) {

    auto stream = lc->getCudaStream();
    auto sizeOf = sizeof(sd::graph::RandomGenerator);
    Nd4jPointer stateDevice;

    hipError_t res = hipMalloc(reinterpret_cast<void **>(&stateDevice), sizeOf);
    checkCudaErrors(hipStreamSynchronize(*stream));
    checkCudaErrors(hipMemcpyAsync(stateDevice, stateHost, sizeOf, hipMemcpyHostToDevice, *stream));

    auto rng = reinterpret_cast<sd::graph::RandomGenerator*>(stateHost);

    dim3 launchDims = dim3(512, 512, 32768);
    auto xType = sd::ArrayOptions::dataType(hZShapeInfo);
    // functions::random::RandomFunction<float>::executeCudaTriple(launchDims, extraPointers, opNum, stateHost, dX, dXShapeInfo, dY, dYShapeInfo, dZ, dZShapeInfo, extraArguments);
    BUILD_SINGLE_SELECTOR(xType, functions::random::RandomFunction, ::executeCudaTriple(launchDims, stream, opNum, stateDevice, dX, dXShapeInfo, dY, dYShapeInfo, dZ, dZShapeInfo, extraArguments), FLOAT_TYPES);

    res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execRandom XYZ failed", res);

    hipFree(stateDevice);

    rng->rewindH(shape::length(hZShapeInfo));
}

////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execReduce3All(sd::LaunchContext  *lc,
									int opNum,
									void const* hX, Nd4jLong const* hXShapeInfo,
                            		void const* dX, Nd4jLong const* dXShapeInfo,
                            		void *extraParamsVals,
									void const* hY, Nd4jLong const* hYShapeInfo,
                            		void const* dY, Nd4jLong const* dYShapeInfo,
                            		void *hZ, Nd4jLong const* hZShapeInfo,
                            		void *dZ, Nd4jLong const* dZShapeInfo,
									int *dimension, int dimensionLength,
									Nd4jLong const* xTadShapeInfo, Nd4jLong const* xOffsets,
									Nd4jLong const* yTadShapeInfo, Nd4jLong const* yOffsets) {

    auto stream = lc->getCudaStream();
    auto allocationPointer = lc->getAllocationPointer();
	auto reductionPointer  = lc->getReductionPointer();

    if (sd::Environment::getInstance().isDebugAndVerbose())
        printf("D119 opNum:[%i]\n", opNum);

    dim3 launchDims(shape::length(hZShapeInfo), CUDA_BLOCK_SIZE / 2, 1024);

    if (sd::Environment::getInstance().isVerbose() && launchDims.x == 1)
        printf("AD119 opNum:[%i]\n", opNum);

    auto xType = sd::ArrayOptions::dataType(hXShapeInfo);
    auto yType = sd::ArrayOptions::dataType(hYShapeInfo);
    auto zType = sd::ArrayOptions::dataType(hZShapeInfo);

    if (yType != xType)
        throw sd::datatype_exception::build("NativeOpExecutioner::execReduce3All both operands must have same data type", xType, yType);

    BUILD_DOUBLE_SELECTOR(xType, zType, functions::reduce3::Reduce3, ::execAll(launchDims, stream, opNum, dX, dXShapeInfo, dY, dYShapeInfo, extraParamsVals, dZ, dZShapeInfo, dimension, dimensionLength, 1, allocationPointer, xTadShapeInfo, xOffsets, yTadShapeInfo, yOffsets), LIBND4J_TYPES, FLOAT_TYPES);

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execReduce3All failed", res);
}


////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execReduce3TAD(sd::LaunchContext  *lc,
                                            int opNum,
                                            void const* hX, Nd4jLong const* hXShapeInfo,
                                            void const* dX, Nd4jLong const* dXShapeInfo,
                                            void *extraParams,
                                            void const* hY, Nd4jLong const* hYShapeInfo,
                                            void const* dY, Nd4jLong const* dYShapeInfo,
                                            void *hZ, Nd4jLong const* hZShapeInfo,
                                            void *dZ, Nd4jLong const* dZShapeInfo,
                                            int *dimension, int dimensionLength,
                                            Nd4jLong const* tadShapeInfo,  Nd4jLong const* tadOffsets,
                                            Nd4jLong const* yTadShapeInfo, Nd4jLong const* yTadOffsets) {

    if(shape::isScalar(hZShapeInfo)) {
        NativeOpExecutioner::execReduce3(lc, opNum, hX, hXShapeInfo, dX, dXShapeInfo, extraParams, hY, hYShapeInfo, dY, dYShapeInfo, hZ, hZShapeInfo, dZ, dZShapeInfo);
        return;
    }

    auto stream = lc->getCudaStream();
    auto allocationPointer = lc->getAllocationPointer();

    auto xType = sd::ArrayOptions::dataType(hXShapeInfo);
    auto yType = sd::ArrayOptions::dataType(hYShapeInfo);
    auto zType = sd::ArrayOptions::dataType(hZShapeInfo);

     if (xType != yType)
        throw sd::datatype_exception::build("NativeOpExecutioner::execReduce3TAD requires Y operand to have X type", xType, yType);

    if (!DataTypeUtils::isR(zType))
        throw sd::datatype_exception::build("NativeOpExecutioner::execReduce3TAD requires Z operand to have floating point data type", zType);

    auto numBlocks = shape::length(hZShapeInfo);
    dim3 launchDims(numBlocks == 0 ? 1 : numBlocks, CUDA_BLOCK_SIZE, 1024);

    BUILD_DOUBLE_SELECTOR(xType, zType, functions::reduce3::Reduce3, ::exec(launchDims, stream, opNum, dX, dXShapeInfo, dY, dYShapeInfo, extraParams, dZ, dZShapeInfo, dimension, dimensionLength, 1, allocationPointer, tadShapeInfo, tadOffsets, yTadShapeInfo, yTadOffsets), LIBND4J_TYPES, FLOAT_TYPES);

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execReduce3TAD failed", res);
}

