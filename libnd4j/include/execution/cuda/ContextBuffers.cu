/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
//

#include <execution/ContextBuffers.h>
#include <exceptions/cuda_exception.h>
#include <helpers/logger.h>
#include <execution/AffinityManager.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>

namespace sd {
    ContextBuffers::ContextBuffers() {
        //nd4j_printf("Creating ContextBuffers for device [%i]\n", AffinityManager::currentDeviceId());
        _deviceId = AffinityManager::currentDeviceId();
    }

    ContextBuffers::ContextBuffers(const ContextBuffers &other) {
        release();

        this->_initialized = other._initialized;
        this->_allocated = other._allocated;
        this->_deviceId = other._deviceId;

        this->_specialStream = other._specialStream;
        this->_execStream = other._execStream;
        this->_allocationPointer = other._allocationPointer;
        this->_reductionPointer = other._reductionPointer;
        this->_scalarPointer = other._scalarPointer;
    }

    ContextBuffers& ContextBuffers::operator=(const ContextBuffers& other) {
        release();

        this->_initialized = other._initialized;
        this->_allocated = other._allocated;
        this->_deviceId = other._deviceId;

        this->_specialStream = other._specialStream;
        this->_execStream = other._execStream;
        this->_allocationPointer = other._allocationPointer;
        this->_reductionPointer = other._reductionPointer;
        this->_scalarPointer = other._scalarPointer;

        return *this;
    }

    ContextBuffers& ContextBuffers::operator=(ContextBuffers&& other) {
        release();

        this->_initialized = other._initialized;
        this->_allocated = other._allocated;
        this->_deviceId = other._deviceId;

        this->_specialStream = other._specialStream;
        this->_execStream = other._execStream;
        this->_allocationPointer = other._allocationPointer;
        this->_reductionPointer = other._reductionPointer;
        this->_scalarPointer = other._scalarPointer;

        return *this;
    }

    void ContextBuffers::release() {
        if (_allocated) {
            //nd4j_printf("Releasing ContextBuffers on device [%i]\n", _deviceId);

            if (_allocationPointer != nullptr)
                hipFree(_allocationPointer);

            if (_scalarPointer != nullptr)
                hipHostFree(_scalarPointer);

            if (_allocationPointer != nullptr)
                hipFree(_reductionPointer);

            auto _cudaStream = reinterpret_cast<hipStream_t*>(_execStream);
            auto _cudaSpecialStream = reinterpret_cast<hipStream_t*>(_specialStream);

            hipStreamSynchronize(*_cudaStream);
            hipStreamSynchronize(*_cudaSpecialStream);

            hipStreamDestroy(*_cudaStream);
            hipStreamDestroy(*_cudaSpecialStream);

            delete _cudaStream;
            delete _cudaSpecialStream;

            //////
            _allocated = false;
            _deviceId = -1;

            this->_specialStream = nullptr;
            this->_execStream = nullptr;
            this->_allocationPointer = nullptr;
            this->_reductionPointer = nullptr;
            this->_scalarPointer = nullptr;
        }

        _initialized = false;
    }

    ContextBuffers::~ContextBuffers() {
        release();
    }

    ContextBuffers::ContextBuffers(void* rPointer, void* sPointer, void* aPointer, bool isOwner) {
        _reductionPointer = rPointer;
        _scalarPointer = sPointer;
        _allocationPointer = aPointer;
        _allocated = isOwner;
    }

    void ContextBuffers::initialize() {
        _deviceId = AffinityManager::currentNativeDeviceId();
        //nd4j_printf("Initializing buffers on deviceId [%i]\n", _deviceId);

        auto res = hipMalloc(reinterpret_cast<void**>(&_reductionPointer), 1024 * 1024 * 8);
        if (res != 0)
            throw cuda_exception::build("_reductionPointer allocation failed", res);

        res = hipHostAlloc(reinterpret_cast<void**>(&_scalarPointer), 16, hipHostMallocDefault);
        if (res != 0)
            throw cuda_exception::build("_scalarPointer allocation failed", res);

        res = hipMalloc(reinterpret_cast<void**>(&_allocationPointer), 1024 * 1024 * 8);
        if (res != 0)
            throw cuda_exception::build("_allocationPointer allocation failed", res);

        _execStream  = new hipStream_t();
        _specialStream = new hipStream_t();
        if (nullptr == _execStream || nullptr == _specialStream)
            throw std::runtime_error("Failed to allocate memory for new CUDA stream");

        res = hipStreamCreate(reinterpret_cast<hipStream_t*>(_execStream));
        if (res != 0)
            throw cuda_exception::build("Failed to create default CUDA stream with launch context", res);

        res = hipStreamCreate(reinterpret_cast<hipStream_t*>(_specialStream));
        if (res != 0)
            throw cuda_exception::build("Failed to create special CUDA stream with launch context", res);

        _allocated = true;
        _initialized = true;
    }

    void* ContextBuffers::reductionBuffer() {
        if (!_initialized)
            initialize();

        return _reductionPointer;
    }

    void* ContextBuffers::scalarBuffer() {
        if (!_initialized)
            initialize();

        return _scalarPointer;
    }

    void* ContextBuffers::allocationBuffer() {
        if (!_initialized)
            initialize();

        return _allocationPointer;
    }

    void ContextBuffers::setReductionBuffer(void* pointer) {
        _reductionPointer = pointer;
    }

    void ContextBuffers::setScalarBuffer(void* pointer) {
        _scalarPointer = pointer;
    }

    void ContextBuffers::setAllocationBuffer(void* pointer) {
        _allocationPointer = pointer;
    }

    void ContextBuffers::triggerOwnership(bool isOwner) {
        _allocated = isOwner;
    }

    int ContextBuffers::deviceId() {
        return _deviceId;
    }

    void* ContextBuffers::execStream() {
        if (!_initialized) {
            //nd4j_printf("execStream not initialized\n", "");
            initialize();
        } else {
            //nd4j_printf("execStream is initialized\n", "");
        }

        return _execStream;
    }

    void* ContextBuffers::specialStream() {
        if (!_initialized) {
            //nd4j_printf("specialStream not initialized\n", "");
            initialize();
        } else {
            //nd4j_printf("specialStream is initialized\n", "");
        }

        return _specialStream;
    }

    bool ContextBuffers::isInitialized() {
        return _initialized;
    }

    sd::ErrorReference* ContextBuffers::errorReference() {
        return &_errorReference;
    }
}

