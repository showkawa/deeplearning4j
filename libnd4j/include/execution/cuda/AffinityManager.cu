/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
//

#include <helpers/logger.h>
#include <execution/AffinityManager.h>
#include <exceptions/cuda_exception.h>
#include <execution/LaunchContext.h>

thread_local int globalThreadToDevice = -1;

namespace sd {
    std::mutex AffinityManager::_currentMutex;
    std::mutex AffinityManager::_numberMutex;
    int AffinityManager::_numberOfDevices = -1;

    int AffinityManager::currentDeviceId() {
        // if there's no affinity set - set it now
        if (globalThreadToDevice < 0) {

            // this block must be thread-local
            _currentMutex.lock();

            globalThreadToDevice = _lastDevice++;

            // we need to check if we've got deviceId >= number of actual devices, and reset to zero otherwise
            if (globalThreadToDevice >= numberOfDevices()) {
                globalThreadToDevice = 0;
                _lastDevice = numberOfDevices() > 1 ? 1 : 0;
            }

            _currentMutex.unlock();

            setCurrentNativeDevice(globalThreadToDevice);
        }

        // if we already know affinity - just return it
        if (globalThreadToDevice >= 0)
            return globalThreadToDevice;

        int dev = 0;
        auto res = hipGetDevice(&dev);

        if (res != 0)
            throw cuda_exception::build("hipGetDevice failed", res);

        return dev;
    }

    int AffinityManager::currentNativeDeviceId() {
        int dev = 0;
        auto res = hipGetDevice(&dev);

        if (res != 0)
            throw cuda_exception::build("hipGetDevice failed", res);

        return dev;
    }

    int AffinityManager::numberOfDevices() {
        _numberMutex.lock();
        // we want to cache number of devices
        if (_numberOfDevices <= 0) {
            int dev = 0;
            auto res = hipGetDeviceCount(&dev);

            if (res != 0)
                throw cuda_exception::build("hipGetDeviceCount failed", res);

            _numberOfDevices = dev;
        }
        _numberMutex.unlock();

        return _numberOfDevices;
    }

    void AffinityManager::setCurrentNativeDevice(int deviceId) {
        auto res = hipSetDevice(deviceId);
        if (res != 0)
            throw cuda_exception::build("setCurrentDevice failed", res);
    }

    void AffinityManager::setCurrentDevice(int deviceId) {
        auto previousDeviceId = globalThreadToDevice;
        if (previousDeviceId >= 0 && LaunchContext::isInitialized()) {
            auto res = hipStreamSynchronize(*LaunchContext::defaultContext()->getCudaStream());
            if (res != 0)
                throw cuda_exception::build("setCurrentDevice -> sync failed", res);

            res = hipStreamSynchronize(*LaunchContext::defaultContext()->getCudaSpecialStream());
            if (res != 0)
                throw cuda_exception::build("setCurrentDevice -> specialSync failed", res);

            if (deviceId != previousDeviceId) {
                // discard existing stuff
                //nd4j_printf("AffinityManager::setCurrentDevice() was invoked, releasing buffers\n", "");
                LaunchContext::releaseBuffers();
            }
        }

        if (deviceId != previousDeviceId) {
            auto res = hipSetDevice(deviceId);
            if (res != 0)
                throw cuda_exception::build("hipSetDevice failed", res);
        }

        // update thread-device affinity
        globalThreadToDevice = deviceId;
    }

    std::atomic<int> AffinityManager::_lastDevice;// = std::atomic<int>(initialV);
}