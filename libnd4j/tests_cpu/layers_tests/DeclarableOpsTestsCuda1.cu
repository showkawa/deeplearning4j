#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/


//
// @author raver119@gmail.com
//

#include "testlayers.h"
#include <ops/declarable/CustomOperations.h>
#include <array/NDArray.h>
#include <ops/ops.h>
#include <helpers/GradCheck.h>
#include <chrono>


using namespace sd;


class DeclarableOpsTestsCuda1 : public testing::Test {
public:

    DeclarableOpsTestsCuda1() {
        printf("\n");
        fflush(stdout);
    }
};


TEST_F(DeclarableOpsTestsCuda1, Test_CHOOSE_SCALAR_LARGE) {
    double inputData[150] = {
            0,  0.51,  0.68,  0.69,  0.86,  0.91,  0.96,  0.97,  0.97,  1.03,  1.13,  1.16,  1.16,  1.17,  1.19,  1.25,  1.25,  1.26,  1.27,  1.28,  1.29,  1.29,  1.29,  1.30,  1.31,  1.32,  1.33,  1.33,  1.35,  1.35,  1.36,  1.37,  1.38,  1.40,  1.41,  1.42,  1.43,  1.44,  1.44,  1.45,  1.45,  1.47,  1.47,  1.51,  1.51,  1.51,  1.52,  1.53,  1.56,  1.57,  1.58,  1.59,  1.61,  1.62,  1.63,  1.63,  1.64,  1.64,  1.66,  1.66,  1.67,  1.67,  1.70,  1.70,  1.70,  1.72,  1.72,  1.72,  1.72,  1.73,  1.74,  1.74,  1.76,  1.76,  1.77,  1.77,  1.80,  1.80,  1.81,  1.82,  1.83,  1.83,  1.84,  1.84,  1.84,  1.85,  1.85,  1.85,  1.86,  1.86,  1.87,  1.88,  1.89,  1.89,  1.89,  1.89,  1.89,  1.91,  1.91,  1.91,  1.92,  1.94,  1.95,  1.97,  1.98,  1.98,  1.98,  1.98,  1.98,  1.99,  2,  2,  2.01,  2.01,  2.02,  2.03,  2.03,  2.03,  2.04,  2.04,  2.05,  2.06,  2.07,  2.08,  2.08,  2.08,  2.08,  2.09,  2.09,  2.10,  2.10,  2.11,  2.11,  2.11,  2.12,  2.12,  2.13,  2.13,  2.14,  2.14,  2.14,  2.14,  2.15,  2.15,  2.16,  2.16,  2.16,  2.16,  2.16,  2.17
    };

    auto precursor = NDArrayFactory::create<double>(inputData,'c',{1,149});
    NDArray x(nullptr, precursor.specialBuffer(), precursor.shapeInfo());

    sd::ops::choose op;
    //greater than test
    auto result = op.evaluate({&x}, {0.0},{3});
    ASSERT_EQ(Status::OK(), result.status());

    auto z = result.at(1);

    ASSERT_EQ(148,z->e<double>(0));
    //ASSERT_TRUE(exp.isSameShape(z));
}

/*
TEST_F(DeclarableOpsTestsCuda1, Test_Reverse_TAD_1) {
    auto x = NDArrayFactory::create<float>('c', {1, 3, 608, 608});
    auto z = x.like();
    x.linspace(1.0f);

    sd::ops::reverse op;
    auto timeStart = std::chrono::system_clock::now();
    auto status = op.execute({&x}, {&z}, {}, {1}, {});
    auto timeEnd = std::chrono::system_clock::now();
    auto outerTime = std::chrono::duration_cast<std::chrono::microseconds> (timeEnd - timeStart).count();
    nd4j_printf("exec time: %lld us\n", outerTime);
    ASSERT_EQ(Status::OK(), status);
}
*/