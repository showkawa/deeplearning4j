#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

 //
 // @author raver119@gmail.com
 //

#include "testlayers.h"
#include <array/NDArray.h>
#include <array/NDArrayFactory.h>
#include <graph/Context.h>
#include <graph/Node.h>
#include <graph/Variable.h>
#include <graph/VariableSpace.h>
#include <ops/specials_cuda.h>
#include <helpers/TAD.h>
#include <helpers/MmulHelper.h>
#include <helpers/PointersManager.h>
#include <hip/hip_runtime.h>
#include <helpers/RandomLauncher.h>
#include <helpers/ConstantShapeHelper.h>
#include <helpers/ConstantTadHelper.h>
#include <array/ShapeDescriptor.h>
#include <array/ConstantDataBuffer.h>
#include <helpers/ShapeUtils.h>
#include <exceptions/cuda_exception.h>

using namespace sd;
using namespace sd::graph;

class CudaBasicsTests1 : public testing::Test {
public:

};


//////////////////////////////////////////////////////////////////////////
static hipError_t allocateDeviceMem(LaunchContext& lc, std::vector<void*>& devicePtrs, const std::vector<std::pair<void*,size_t>>& hostData) {

	if(devicePtrs.size() != hostData.size())
		throw std::invalid_argument("prepareDataForCuda: two input sts::vectors should same sizes !");

	hipError_t cudaResult;

	void* reductionPointer;
    cudaResult = hipMalloc(reinterpret_cast<void **>(&reductionPointer),  1024*1024);			if(cudaResult != 0) return cudaResult;
    int* allocationPointer;
	cudaResult = hipMalloc(reinterpret_cast<void **>(&allocationPointer), 1024*1024);			if(cudaResult != 0) return cudaResult;

	lc.setReductionPointer(reductionPointer);
	lc.setAllocationPointer(allocationPointer);
	hipStream_t stream = *lc.getCudaStream();

	for(int i = 0; i < devicePtrs.size(); ++i) {

		cudaResult = hipMalloc(reinterpret_cast<void **>(&devicePtrs[i]), hostData[i].second); if(cudaResult != 0) return cudaResult;
		hipMemcpyAsync(devicePtrs[i], hostData[i].first, hostData[i].second, hipMemcpyHostToDevice, stream);
	}
	return cudaResult;
}

//////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, TestPairwise_1) {
	// allocating host-side arrays
	auto x = NDArrayFactory::create<double>('c', { 5 }, { 1, 2, 3, 4, 5});
	auto z = NDArrayFactory::create<double>('c', { 5 }, {0,0,0,0,0});

	auto exp = NDArrayFactory::create<double>('c', { 5 }, { 2, 4, 6, 8, 10 });

	// making raw buffers
	Nd4jPointer devBufferPtrX, devBufferPtrZ, devShapePtrX;
	hipError_t res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrX), x.lengthOf() * x.sizeOfT());
	ASSERT_EQ(0, res);
	res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrZ), x.lengthOf() * x.sizeOfT());
	ASSERT_EQ(0, res);
	res = hipMalloc(reinterpret_cast<void **>(&devShapePtrX), shape::shapeInfoByteLength(x.shapeInfo()));
	ASSERT_EQ(0, res);

	Nd4jPointer nativeStream = (Nd4jPointer)malloc(sizeof(hipStream_t));
	CHECK_ALLOC(nativeStream, "Failed to allocate memory for new CUDA stream", sizeof(hipStream_t));
	hipError_t dZ = hipStreamCreate(reinterpret_cast<hipStream_t *>(&nativeStream));
	auto stream = reinterpret_cast<hipStream_t *>(&nativeStream);

    x.dataBuffer()->allocatePrimary();
    x.syncToHost();

	hipMemcpyAsync(devBufferPtrX, x.buffer(), x.lengthOf() * x.sizeOfT(), hipMemcpyHostToDevice, *stream);
	hipMemcpyAsync(devShapePtrX, x.shapeInfo(), shape::shapeInfoByteLength(x.shapeInfo()), hipMemcpyHostToDevice, *stream);
    res = hipStreamSynchronize(*stream);
    ASSERT_EQ(0, res);

	LaunchContext lc(stream, nullptr, nullptr);
	NativeOpExecutioner::execPairwiseTransform(&lc, pairwise::Add, nullptr, x.shapeInfo(), devBufferPtrX, reinterpret_cast<Nd4jLong*>(devShapePtrX), nullptr, x.shapeInfo(), devBufferPtrX, reinterpret_cast<Nd4jLong*>(devShapePtrX), nullptr, z.shapeInfo(), devBufferPtrZ, reinterpret_cast<Nd4jLong*>(devShapePtrX), nullptr);
	res = hipStreamSynchronize(*stream);
	ASSERT_EQ(0, res);

	z.dataBuffer()->allocatePrimary();

	hipMemcpyAsync(z.buffer(), devBufferPtrZ, z.lengthOf() * x.sizeOfT(), hipMemcpyDeviceToHost, *stream);
	res = hipStreamSynchronize(*stream);
	ASSERT_EQ(0, res);

	hipFree(devBufferPtrX);
	hipFree(devBufferPtrZ);
	hipFree(devShapePtrX);

	// needed due to memcpy
    z.tickWriteHost();

	for (int e = 0; e < z.lengthOf(); e++) {
	    //nd4j_printf("step %i\n", e);
		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);
	}
}


////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execIndexReduceScalar_1) {

    NDArray x1('c', {2,2}, {0, 1, 2, 3}, sd::DataType::INT32);
    NDArray x2('c', {2,2}, {0.5, 1.5, -4.5, 3.5}, sd::DataType::BFLOAT16);
    NDArray x3('c', {2,2}, {0, -1, 0, 1}, sd::DataType::BOOL);

    NDArray scalar('c', {}, std::vector<double>{0}, sd::DataType::INT64);

    NDArray exp1('c', {}, std::vector<double>{3}, sd::DataType::INT64);
    NDArray exp2('c', {}, std::vector<double>{2}, sd::DataType::INT64);
    NDArray exp3('c', {}, std::vector<double>{1}, sd::DataType::INT64);

    void *dX1, *dX2, *dX3, *dZ;
    Nd4jLong *dX1ShapeInfo, *dX2ShapeInfo, *dX3ShapeInfo, *dZShapeInfo;

    hipError_t cudaResult;

    cudaResult = hipMalloc(reinterpret_cast<void **>(&dX1), x1.lengthOf() * x1.sizeOfT()); 		   		         	 ASSERT_EQ(0, cudaResult);
    cudaResult = hipMalloc(reinterpret_cast<void **>(&dX2), x2.lengthOf() * x2.sizeOfT()); 		   		         	 ASSERT_EQ(0, cudaResult);
    cudaResult = hipMalloc(reinterpret_cast<void **>(&dX3), x3.lengthOf() * x3.sizeOfT()); 		   		         	 ASSERT_EQ(0, cudaResult);
	cudaResult = hipMalloc(reinterpret_cast<void **>(&dZ), scalar.lengthOf() * scalar.sizeOfT()); 				         ASSERT_EQ(0, cudaResult);
	cudaResult = hipMalloc(reinterpret_cast<void **>(&dX1ShapeInfo), shape::shapeInfoByteLength(x1.shapeInfo()));    ASSERT_EQ(0, cudaResult);
	cudaResult = hipMalloc(reinterpret_cast<void **>(&dX2ShapeInfo), shape::shapeInfoByteLength(x2.shapeInfo()));    ASSERT_EQ(0, cudaResult);
	cudaResult = hipMalloc(reinterpret_cast<void **>(&dX3ShapeInfo), shape::shapeInfoByteLength(x3.shapeInfo()));    ASSERT_EQ(0, cudaResult);
	cudaResult = hipMalloc(reinterpret_cast<void **>(&dZShapeInfo), shape::shapeInfoByteLength(scalar.shapeInfo())); ASSERT_EQ(0, cudaResult);

    hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);
	ASSERT_EQ(0, cudaResult);

	x1.syncToHost();
	x2.syncToHost();
	x3.syncToHost();
	scalar.syncToHost();

	hipMemcpyAsync(dX1, x1.buffer(), x1.lengthOf() * x1.sizeOfT(), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(dX2, x2.buffer(), x2.lengthOf() * x2.sizeOfT(), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(dX3, x3.buffer(), x3.lengthOf() * x3.sizeOfT(), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(dX1ShapeInfo, x1.shapeInfo(), shape::shapeInfoByteLength(x1.shapeInfo()), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(dX2ShapeInfo, x2.shapeInfo(), shape::shapeInfoByteLength(x2.shapeInfo()), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(dX3ShapeInfo, x3.shapeInfo(), shape::shapeInfoByteLength(x3.shapeInfo()), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(dZShapeInfo, scalar.shapeInfo(), shape::shapeInfoByteLength(scalar.shapeInfo()), hipMemcpyHostToDevice, stream);

	void* reductionPointer = nullptr;
	cudaResult = hipMalloc(reinterpret_cast<void **>(&reductionPointer), 1024*1024);
	ASSERT_EQ(0, cudaResult);
	cudaResult = hipMemset(reductionPointer, 0, 1024 * 1024);
    ASSERT_EQ(0, cudaResult);

	LaunchContext lc(&stream, LaunchContext::defaultContext()->getReductionPointer(), LaunchContext::defaultContext()->getScalarPointer(), LaunchContext::defaultContext()->getAllocationPointer());

	/***************************************/

    NativeOpExecutioner::execIndexReduceScalar(&lc,
    											sd::indexreduce::IndexAbsoluteMax,
    											x1.buffer(), x1.shapeInfo(),
    	                                       	dX1, dX1ShapeInfo,
    	                                       	nullptr,
    	                                       	scalar.buffer(), scalar.shapeInfo(),
    	                                       	dZ, dZShapeInfo);

    cudaResult = hipStreamSynchronize(stream);
    ASSERT_EQ(0, cudaResult);

    hipMemcpyAsync(scalar.buffer(), dZ, scalar.lengthOf() * scalar.sizeOfT(), hipMemcpyDeviceToHost, stream);

    cudaResult = hipStreamSynchronize(stream);
    ASSERT_EQ(0, cudaResult);

    scalar.tickWriteHost();

	ASSERT_NEAR(exp1.e<float>(0), scalar.e<float>(0), 1e-5);

    /***************************************/

    NativeOpExecutioner::execIndexReduceScalar(&lc,
    											sd::indexreduce::IndexAbsoluteMax,
    											nullptr, x2.shapeInfo(),
    	                                       	dX2, dX2ShapeInfo,
    	                                       	nullptr,
    	                                       	nullptr, scalar.shapeInfo(),
    	                                       	dZ, dZShapeInfo);

    cudaResult = hipStreamSynchronize(stream);
    ASSERT_EQ(0, cudaResult);

    hipMemcpyAsync(scalar.buffer(), dZ, scalar.lengthOf() * scalar.sizeOfT(), hipMemcpyDeviceToHost, stream);

    cudaResult = hipStreamSynchronize(stream);
    ASSERT_EQ(0, cudaResult);

    ASSERT_NEAR(exp2.e<float>(0), scalar.e<float>(0), 1e-5);

    // *************************************

    NativeOpExecutioner::execIndexReduceScalar(&lc,
    											sd::indexreduce::IndexAbsoluteMax,
    											nullptr, x3.shapeInfo(),
    	                                       	dX3, dX3ShapeInfo,
    	                                       	nullptr,
    	                                       	nullptr, scalar.shapeInfo(),
    	                                       	dZ, dZShapeInfo);

    cudaResult = hipStreamSynchronize(stream);
    ASSERT_EQ(0, cudaResult);

    hipMemcpyAsync(scalar.buffer(), dZ, scalar.lengthOf() * scalar.sizeOfT(), hipMemcpyDeviceToHost, stream);

    cudaResult = hipStreamSynchronize(stream);
    ASSERT_EQ(0, cudaResult);

    ASSERT_NEAR(exp3.e<float>(0), scalar.e<float>(0), 1e-5);

	/***************************************/

	hipFree(dX1); 			hipFree(dX2); 			hipFree(dX3); 			hipFree(dZ);
	hipFree(dX1ShapeInfo); hipFree(dX2ShapeInfo); hipFree(dX3ShapeInfo); hipFree(dZShapeInfo);

	/***************************************/

	cudaResult = hipStreamDestroy(stream);
	ASSERT_EQ(0, cudaResult);

}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execReduce3Scalar_1) {

	 if (!Environment::getInstance().isExperimentalBuild())
        return;

    NDArray x1('c', {2,2}, {1,2,3,4}, sd::DataType::INT32);
    NDArray x2('c', {2,2}, {-1,-2,-3,-4}, sd::DataType::INT32);
    NDArray x3('c', {2,2}, {1.5,1.5,1.5,1.5}, sd::DataType::DOUBLE);
    NDArray x4('c', {2,2}, {1,2,3,4}, sd::DataType::DOUBLE);

    NDArray exp1('c', {}, std::vector<double>{-30.f}, sd::DataType::FLOAT32);
    NDArray exp2('c', {}, std::vector<double>{15.}, sd::DataType::DOUBLE);

	NDArray scalar1('c', {}, std::vector<double>{100.f}, sd::DataType::FLOAT32);
    NDArray scalar2('c', {}, std::vector<double>{100.}, sd::DataType::DOUBLE);

    void *dX1, *dX2, *dX3, *dX4, *dZ1, *dZ2;
    Nd4jLong *dX1ShapeInfo, *dX3ShapeInfo, *dZ1ShapeInfo, *dZ2ShapeInfo;

    hipError_t cudaResult;

    cudaResult = hipMalloc(reinterpret_cast<void **>(&dX1), x1.lengthOf() * x1.sizeOfT()); 		   		         	 	ASSERT_EQ(0, cudaResult);
    cudaResult = hipMalloc(reinterpret_cast<void **>(&dX2), x2.lengthOf() * x2.sizeOfT()); 		   		         	 	ASSERT_EQ(0, cudaResult);
    cudaResult = hipMalloc(reinterpret_cast<void **>(&dX3), x3.lengthOf() * x3.sizeOfT()); 		   		         	 	ASSERT_EQ(0, cudaResult);
    cudaResult = hipMalloc(reinterpret_cast<void **>(&dX4), x4.lengthOf() * x4.sizeOfT()); 		   		         	 	ASSERT_EQ(0, cudaResult);
	cudaResult = hipMalloc(reinterpret_cast<void **>(&dZ1), scalar1.lengthOf() * scalar1.sizeOfT());			         	ASSERT_EQ(0, cudaResult);
	cudaResult = hipMalloc(reinterpret_cast<void **>(&dZ2), scalar2.lengthOf() * scalar2.sizeOfT());			         	ASSERT_EQ(0, cudaResult);
	cudaResult = hipMalloc(reinterpret_cast<void **>(&dX1ShapeInfo), shape::shapeInfoByteLength(x1.shapeInfo()));    	ASSERT_EQ(0, cudaResult);
	cudaResult = hipMalloc(reinterpret_cast<void **>(&dX3ShapeInfo), shape::shapeInfoByteLength(x3.shapeInfo()));    	ASSERT_EQ(0, cudaResult);
	cudaResult = hipMalloc(reinterpret_cast<void **>(&dZ1ShapeInfo), shape::shapeInfoByteLength(scalar1.shapeInfo())); 	ASSERT_EQ(0, cudaResult);
	cudaResult = hipMalloc(reinterpret_cast<void **>(&dZ2ShapeInfo), shape::shapeInfoByteLength(scalar2.shapeInfo())); 	ASSERT_EQ(0, cudaResult);

    hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);
	ASSERT_EQ(0, cudaResult);

	x1.syncToHost();
	x2.syncToHost();
	x3.syncToHost();
	x4.syncToHost();
	scalar1.syncToHost();
	scalar2.syncToHost();

	hipMemcpyAsync(dX1, x1.buffer(), x1.lengthOf() * x1.sizeOfT(), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(dX2, x2.buffer(), x2.lengthOf() * x2.sizeOfT(), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(dX3, x3.buffer(), x3.lengthOf() * x3.sizeOfT(), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(dX4, x4.buffer(), x4.lengthOf() * x4.sizeOfT(), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(dX1ShapeInfo, x1.shapeInfo(), shape::shapeInfoByteLength(x1.shapeInfo()), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(dX3ShapeInfo, x3.shapeInfo(), shape::shapeInfoByteLength(x3.shapeInfo()), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(dZ1ShapeInfo, scalar1.shapeInfo(), shape::shapeInfoByteLength(scalar1.shapeInfo()), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(dZ2ShapeInfo, scalar2.shapeInfo(), shape::shapeInfoByteLength(scalar2.shapeInfo()), hipMemcpyHostToDevice, stream);

	/***************************************/

	void* reductionPointer  = nullptr;
	int*  allocationPointer = nullptr;

	cudaResult = hipMalloc(reinterpret_cast<void **>(&reductionPointer),  1024*1024);		ASSERT_EQ(0, cudaResult);
	cudaResult = hipMalloc(reinterpret_cast<void **>(&allocationPointer), 1024*1024);		ASSERT_EQ(0, cudaResult);

	LaunchContext lc(&stream, reductionPointer, nullptr, allocationPointer);

	/***************************************/

    NativeOpExecutioner::execReduce3Scalar(&lc, sd::reduce3::Dot,nullptr, x1.shapeInfo(),dX1, dX1ShapeInfo, nullptr, nullptr, x2.shapeInfo(),dX2, dX1ShapeInfo,nullptr, scalar1.shapeInfo(),dZ1, dZ1ShapeInfo);

    cudaResult = hipStreamSynchronize(stream);
    ASSERT_EQ(0, cudaResult);

    scalar1.tickWriteHost();
    scalar2.tickWriteHost();

    hipMemcpyAsync(scalar1.buffer(), dZ1, scalar1.lengthOf() * scalar1.sizeOfT(), hipMemcpyDeviceToHost, stream);

    cudaResult = hipStreamSynchronize(stream);
    ASSERT_EQ(0, cudaResult);

	ASSERT_NEAR(exp1.e<float>(0), scalar1.e<float>(0), 1e-5);

    /***************************************/

    NativeOpExecutioner::execReduce3Scalar(&lc, sd::reduce3::Dot,nullptr, x3.shapeInfo(),dX3, dX3ShapeInfo, nullptr, nullptr, x4.shapeInfo(),dX4, dX3ShapeInfo,nullptr, scalar2.shapeInfo(),dZ2, dZ2ShapeInfo);

    cudaResult = hipStreamSynchronize(stream);
    ASSERT_EQ(0, cudaResult);

    hipMemcpyAsync(scalar2.buffer(), dZ2, scalar2.lengthOf() * scalar2.sizeOfT(), hipMemcpyDeviceToHost, stream);

    cudaResult = hipStreamSynchronize(stream);
    ASSERT_EQ(0, cudaResult);

	ASSERT_NEAR(exp2.e<float>(0), scalar2.e<float>(0), 1e-5);

	/***************************************/

	hipFree(dX1); 			hipFree(dX2); hipFree(dX3); 		   hipFree(dX4); 	hipFree(dZ1); 				hipFree(dZ2);
	hipFree(dX1ShapeInfo); 			   hipFree(dX3ShapeInfo); 					hipFree(dZ1ShapeInfo);		hipFree(dZ2ShapeInfo);

	/***************************************/

	cudaResult = hipStreamDestroy(stream);
	ASSERT_EQ(0, cudaResult);
}


////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execReduce3_1) {

    NDArray x('c', {2,2}, {1,2,3,4}, sd::DataType::INT32);
    NDArray y('c', {2,2}, {-1,-2,-3,-4}, sd::DataType::INT32);

    NDArray exp('c', {}, std::vector<double>{-30.f}, sd::DataType::FLOAT32);
    NDArray z('c', {}, std::vector<double>{100.f},  sd::DataType::FLOAT32);

    std::vector<int> dimensions = {0, 1};

    x.syncToHost();
    y.syncToHost();
    z.syncToHost();


    std::vector<std::pair<void*,size_t>> hostData;
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));						// 0 -- dimensions
	std::vector<void*> devicePtrs(hostData.size(), nullptr);

    hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it
	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduce3(&lc, sd::reduce3::Dot,
								nullptr, x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
								nullptr,
								nullptr, y.shapeInfo(), y.specialBuffer(), y.specialShapeInfo(),
								nullptr, z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
								(int*)devicePtrs[0], dimensions.size(),
								nullptr, nullptr, nullptr, nullptr);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) hipFree(devicePtrs[i]);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}


////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execReduce3_2) {

	NDArray x('c', {2,2}, {1.5,1.5,1.5,1.5}, sd::DataType::DOUBLE);
    NDArray y('c', {2,2}, {1,2,3,4}, sd::DataType::DOUBLE);

    NDArray exp('c', {}, std::vector<double>{15.}, sd::DataType::DOUBLE);
    NDArray z('c', {}, std::vector<double>{100.},  sd::DataType::DOUBLE);

    std::vector<int> dimensions = {0, 1};

    // prepare input arrays for prepareDataForCuda function
    std::vector<std::pair<void*,size_t>> hostData;
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));						// 0 -- dimensions
	std::vector<void*> devicePtrs(hostData.size(), nullptr);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it
	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduce3(&lc, sd::reduce3::Dot,
								nullptr, x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
								nullptr,
								nullptr, y.shapeInfo(), y.specialBuffer(), y.specialShapeInfo(),
								nullptr, z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
								(int*)devicePtrs[0], dimensions.size(),
								nullptr, nullptr, nullptr, nullptr);


	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) hipFree(devicePtrs[i]);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execReduce3_3) {

	NDArray x('c', {2,3}, {1,2,3,4,5,6}, sd::DataType::INT32);
    NDArray y('c', {2,3}, {-6,-5,-4,-3,-2,-1}, sd::DataType::INT32);

    NDArray exp('c', {3}, {-18,-20,-18}, sd::DataType::FLOAT32);
    NDArray z('c', {3}, {100,100,100}, sd::DataType::FLOAT32);

    std::vector<int> dimensions = {0};

    // evaluate xTad data
    shape::TAD xTad;
    xTad.init(x.shapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // evaluate yTad data
    shape::TAD yTad;
    yTad.init(y.shapeInfo(), dimensions.data(), dimensions.size());
    yTad.createTadOnlyShapeInfo();
    yTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function
    std::vector<std::pair<void*,size_t>> hostData;
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));						// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));						// 2 -- xTadOffsets
	hostData.emplace_back(yTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(yTad.tadOnlyShapeInfo));// 3 -- yTadShapeInfo
	hostData.emplace_back(yTad.tadOffsets, yTad.numTads * sizeof(Nd4jLong));						// 4-- yTadOffsets
	std::vector<void*> devicePtrs(hostData.size(), nullptr);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it

	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduce3(&lc, sd::reduce3::Dot,
								nullptr, x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
								nullptr,
								nullptr, y.shapeInfo(), y.specialBuffer(), y.specialShapeInfo(),
								nullptr, z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
								(int*)devicePtrs[0], dimensions.size(),
								(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2],
								(Nd4jLong*)devicePtrs[3], (Nd4jLong*)devicePtrs[4]);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
	z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) hipFree(devicePtrs[i]);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execReduce3_4) {

    NDArray x('c', {2,3}, {1,2,3,4,5,6}, sd::DataType::DOUBLE);
    NDArray y('c', {2,3}, {1.5,1.5,1.5,1.5,1.5,1.5}, sd::DataType::DOUBLE);

    NDArray exp('c', {2}, {9,22.5}, sd::DataType::DOUBLE);
    NDArray z('c', {2}, {100,100}, sd::DataType::DOUBLE);

    std::vector<int> dimensions = {1};

    // evaluate xTad data
    shape::TAD xTad;
    xTad.init(x.shapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // evaluate yTad data
    shape::TAD yTad;
    yTad.init(y.shapeInfo(), dimensions.data(), dimensions.size());
    yTad.createTadOnlyShapeInfo();
    yTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function
    std::vector<std::pair<void*,size_t>> hostData;
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));						// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));						// 2 -- xTadOffsets
	hostData.emplace_back(yTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(yTad.tadOnlyShapeInfo));// 3 -- yTadShapeInfo
	hostData.emplace_back(yTad.tadOffsets, yTad.numTads * sizeof(Nd4jLong));						// 4-- yTadOffsets
	std::vector<void*> devicePtrs(hostData.size(), nullptr);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it

	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduce3(&lc, sd::reduce3::Dot,
								nullptr, x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
								nullptr,
								nullptr, y.shapeInfo(), y.specialBuffer(), y.specialShapeInfo(),
								nullptr, z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
								(int*)devicePtrs[0], dimensions.size(),
								(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2],
								(Nd4jLong*)devicePtrs[3], (Nd4jLong*)devicePtrs[4]);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) hipFree(devicePtrs[i]);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execReduce3_5) {

    NDArray x('c', {2,2,3}, {1.5,1.5,1.5,1.5,1.5,1.5,1.5,1.5,1.5,1.5,1.5,1.5}, sd::DataType::FLOAT32);
    NDArray y('c', {2,2,3}, {1,2,3,4,5,6,7,8,9,10,11,12}, sd::DataType::FLOAT32);

    NDArray exp('c', {2,3}, {7.5, 10.5, 13.5, 25.5, 28.5, 31.5}, sd::DataType::FLOAT32);
    NDArray z('c', {2,3}, {100,100,100,100,100,100}, sd::DataType::FLOAT32);

    std::vector<int> dimensions = {1};

    // evaluate xTad data
    shape::TAD xTad;
    xTad.init(x.shapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // evaluate yTad data
    shape::TAD yTad;
    yTad.init(y.shapeInfo(), dimensions.data(), dimensions.size());
    yTad.createTadOnlyShapeInfo();
    yTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function
    std::vector<std::pair<void*,size_t>> hostData;
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));						// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));						// 2 -- xTadOffsets
	hostData.emplace_back(yTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(yTad.tadOnlyShapeInfo));// 3 -- yTadShapeInfo
	hostData.emplace_back(yTad.tadOffsets, yTad.numTads * sizeof(Nd4jLong));						// 4-- yTadOffsets
	std::vector<void*> devicePtrs(hostData.size(), nullptr);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it

	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduce3(&lc, sd::reduce3::Dot,
								nullptr, x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
								nullptr,
								nullptr, y.shapeInfo(), y.specialBuffer(), y.specialShapeInfo(),
								nullptr, z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
								(int*)devicePtrs[0], dimensions.size(),
								(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2],
								(Nd4jLong*)devicePtrs[3], (Nd4jLong*)devicePtrs[4]);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) hipFree(devicePtrs[i]);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execReduce3All_1) {

    NDArray x('c', {2,2}, {1,2,3,4}, sd::DataType::INT32);
    NDArray y('c', {2,3}, {-1,1,-1,1,-1,1}, sd::DataType::INT32);

    NDArray exp('c', {2,3}, {2,-2,2,2,-2,2}, sd::DataType::FLOAT32);
    NDArray z('c', {2,3}, {100,100,100,100,100,100}, sd::DataType::FLOAT32);

    std::vector<int> dimensions = {0};

    // evaluate xTad data
    shape::TAD xTad;
    xTad.init(x.shapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // evaluate yTad data
    shape::TAD yTad;
    yTad.init(y.shapeInfo(), dimensions.data(), dimensions.size());
    yTad.createTadOnlyShapeInfo();
    yTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function
    std::vector<std::pair<void*,size_t>> hostData;
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));						// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));						// 2 -- xTadOffsets
	hostData.emplace_back(yTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(yTad.tadOnlyShapeInfo));// 3 -- yTadShapeInfo
	hostData.emplace_back(yTad.tadOffsets, yTad.numTads * sizeof(Nd4jLong));						// 4 -- yTadOffsets
	std::vector<void*> devicePtrs(hostData.size(), nullptr);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it

	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduce3All(&lc, sd::reduce3::Dot,
										nullptr, x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
										nullptr,
										nullptr, y.shapeInfo(), y.specialBuffer(), y.specialShapeInfo(),
										nullptr, z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
										(int*)devicePtrs[0], dimensions.size(),
										(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2],
										(Nd4jLong*)devicePtrs[3], (Nd4jLong*)devicePtrs[4]);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
	z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) hipFree(devicePtrs[i]);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execReduce3All_2) {

    NDArray x('c', {2,2}, {1,2,3,4}, sd::DataType::DOUBLE);
    NDArray y('c', {2,3}, {1.5,1.5,1.5,1.5,1.5,1.5}, sd::DataType::DOUBLE);

    NDArray exp('c', {2,3}, {6,6,6,9,9,9}, sd::DataType::DOUBLE);
    NDArray z('c', {2,3}, {100,100,100,100,100,100,},sd::DataType::DOUBLE);

    std::vector<int> dimensions = {0};

    // evaluate xTad data
    shape::TAD xTad;
    xTad.init(x.shapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // evaluate yTad data
    shape::TAD yTad;
    yTad.init(y.shapeInfo(), dimensions.data(), dimensions.size());
    yTad.createTadOnlyShapeInfo();
    yTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function
    std::vector<std::pair<void*,size_t>> hostData;
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));						// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));						// 2 -- xTadOffsets
	hostData.emplace_back(yTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(yTad.tadOnlyShapeInfo));// 3 -- yTadShapeInfo
	hostData.emplace_back(yTad.tadOffsets, yTad.numTads * sizeof(Nd4jLong));						// 4-- yTadOffsets
	std::vector<void*> devicePtrs(hostData.size(), nullptr);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it

	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduce3All(&lc, sd::reduce3::Dot,
										nullptr, x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
										nullptr,
										nullptr, y.shapeInfo(), y.specialBuffer(), y.specialShapeInfo(),
										nullptr, z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
										(int*)devicePtrs[0], dimensions.size(),
										(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2],
										(Nd4jLong*)devicePtrs[3], (Nd4jLong*)devicePtrs[4]);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) hipFree(devicePtrs[i]);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execIndexReduce_1) {

    NDArray x('c', {2,3}, {100,100,100,100,100,100}, sd::DataType::DOUBLE);
    x.linspace(-2.); x.syncToDevice();
    NDArray exp('c', {2}, {2, 2}, sd::DataType::INT64);
    NDArray z('c', {2}, {100,100}, sd::DataType::INT64);

    std::vector<int> dimensions = {1};

    // evaluate xTad data
    shape::TAD xTad;
    xTad.init(x.shapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function
    std::vector<std::pair<void*,size_t>> hostData;
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));						// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));						// 2 -- xTadOffsets
	std::vector<void*> devicePtrs(hostData.size(), nullptr);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it

	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execIndexReduce(&lc, sd::indexreduce::IndexMax,
										nullptr, x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
										nullptr,
										nullptr, z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
										(int*)devicePtrs[0], dimensions.size(),
										(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2]);

	cudaResult = hipStreamSynchronize(stream);
	if (cudaResult != 0)
	  throw sd::cuda_exception::build("execIndexReduce failed", cudaResult);

    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i)
		hipFree(devicePtrs[i]);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execIndexReduce_2) {

    NDArray x('c', {2,3,4,5}, {100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,
    						  	100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,
    							100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,
    							100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,
    							100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,
    							100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100}, sd::DataType::FLOAT32);
    x.linspace(-2.f); x.syncToDevice();
    NDArray exp('c', {2,5}, {11,11,11,11,11,11,11,11,11,11}, sd::DataType::INT64);
    NDArray z('c', {2,5}, {100,100,100,100,100,100,100,100,100,100}, sd::DataType::INT64);

    std::vector<int> dimensions = {1,2};

    // evaluate xTad data
    shape::TAD xTad;
    xTad.init(x.shapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function

    std::vector<std::pair<void*,size_t>> hostData;
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));						// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));						// 2 -- xTadOffsets
	std::vector<void*> devicePtrs(hostData.size(), nullptr);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it

	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execIndexReduce(&lc, sd::indexreduce::IndexMax,
										nullptr, x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
										nullptr,
										nullptr, z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
										(int*)devicePtrs[0], dimensions.size(),
										(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2]);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i)
		hipFree(devicePtrs[i]);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execIndexReduce_3) {

    NDArray x('c', {2,3,4,5}, {100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,
    						  	100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,
    							100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,
    							100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,
    							100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,
    							100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100}, sd::DataType::DOUBLE);
    x.linspace(-2.); x.syncToDevice();
    NDArray exp('c', {3}, {39, 39, 39}, sd::DataType::INT64);
    NDArray z('c', {3}, {100,100,100}, sd::DataType::INT64);

    std::vector<int> dimensions = {0,2,3};

    // evaluate xTad data
    shape::TAD xTad;
    xTad.init(x.shapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function
    std::vector<std::pair<void*,size_t>> hostData;
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));						// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));						// 2 -- xTadOffsets
	std::vector<void*> devicePtrs(hostData.size(), nullptr);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it

	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execIndexReduce(&lc, sd::indexreduce::IndexMax,
										nullptr, x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
										nullptr,
										nullptr, z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
										(int*)devicePtrs[0], dimensions.size(),
										(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2]);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i)
		hipFree(devicePtrs[i]);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execScalar_1) {

	if (!Environment::getInstance().isExperimentalBuild())
        return;

    NDArray x('c', {2,3},  {0,1,2,3,4,5}, sd::DataType::INT64);
    NDArray exp('c',{2,3}, {0,0,1,1,2,2}, sd::DataType::INT64);
    NDArray scalar('c',{}, std::vector<double>{2.f}, sd::DataType::FLOAT32);
    NDArray z('c', {2,3}, {100,100,100,100,100,100}, sd::DataType::INT64);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execScalar(&lc, sd::scalar::Divide,
									nullptr, x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
									nullptr, z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
									nullptr, scalar.shapeInfo(), scalar.specialBuffer(), scalar.specialShapeInfo(),
									nullptr);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execScalar_2) {

	if (!Environment::getInstance().isExperimentalBuild())
        return;

    NDArray x('c', {2,3},  {-1,-2,-3,-4,-5,-6}, sd::DataType::INT64);
    NDArray exp('c',{2,3}, {10,10,10,10,10,10}, sd::DataType::FLOAT32);
    NDArray scalar('c',{}, std::vector<double>{10.f}, sd::DataType::FLOAT32);
    NDArray z('c', {2,3}, {100,100,100,100,100,100}, sd::DataType::FLOAT32);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execScalar(&lc, sd::scalar::CopyPws,
									nullptr, x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
									nullptr, z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
									nullptr, scalar.shapeInfo(), scalar.specialBuffer(), scalar.specialShapeInfo(),
									nullptr);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);


	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execScalar_3) {

	if (!Environment::getInstance().isExperimentalBuild())
        return;

    NDArray x('c', {2,3,2},  {0,1,2,3,4,5,6,7,8,9,10,11}, sd::DataType::INT64);
    NDArray scalars('c',{2,2}, {1,2,3,4}, sd::DataType::FLOAT32);
    NDArray exp('c', {2,3,2},  {0,0,2,1,4,2, 2,1,2,2,3,2}, sd::DataType::INT64);
    NDArray z('c', {2,3,2}, {100,100,100,100,100,100,100,100,100,100,100,100}, sd::DataType::INT64);

    std::vector<int> dimensions = {1};

    // evaluate xTad data
    shape::TAD xTad;
    xTad.init(x.shapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function
    std::vector<std::pair<void*,size_t>> hostData;
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));							// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));	// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));							// 2 -- xTadOffsets
	std::vector<void*> devicePtrs(hostData.size(), nullptr);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it

	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execScalar(&lc, sd::scalar::Divide,
									nullptr, x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
									nullptr,
									nullptr, z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
									nullptr, scalars.shapeInfo(), scalars.specialBuffer(), scalars.specialShapeInfo(),
									(int*)devicePtrs[0], dimensions.size(),
									(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2],
									nullptr, nullptr);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i)
		hipFree(devicePtrs[i]);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execScalarBool_1) {

    NDArray x('c', {2,3},  {-1,-2,0,1,2,3}, sd::DataType::BFLOAT16);
    NDArray scalar('c',{}, std::vector<double>{0}, sd::DataType::BFLOAT16);
    NDArray exp('c',{2,3}, {0,0,0,1,1,1}, sd::DataType::BOOL);
    NDArray z('c', {2,3}, {100,100,100,100,100,100,}, sd::DataType::BOOL);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// call cuda kernel which calculates result
	// call cuda kernel which calculates result
	NativeOpExecutioner::execScalarBool(&lc, sd::scalar::GreaterThan,
									nullptr, x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
									nullptr, z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
									nullptr, scalar.shapeInfo(), scalar.specialBuffer(), scalar.specialShapeInfo(),
									nullptr);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execScalarBool_2) {

    NDArray x('c', {2,3},  {0,1,2,3,4,5}, sd::DataType::FLOAT32);
    NDArray scalars('c',{2}, {-1,4}, sd::DataType::FLOAT32);
    NDArray exp('c', {2,3},  {1,1,1,0,0,1}, sd::DataType::BOOL);
    NDArray z('c', {2,3}, {100,100,100,100,100,100}, sd::DataType::BOOL);

    std::vector<int> dimensions = {1};

    // evaluate xTad data
    shape::TAD xTad;
    xTad.init(x.shapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function
    std::vector<std::pair<void*,size_t>> hostData;
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));							// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));	// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));							// 2 -- xTadOffsets
	std::vector<void*> devicePtrs(hostData.size(), nullptr);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it
	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execScalarBool(&lc, sd::scalar::GreaterThan,
									nullptr, x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
									nullptr,
									nullptr, z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
									nullptr, scalars.shapeInfo(), scalars.specialBuffer(), scalars.specialShapeInfo(),
									(int*)devicePtrs[0], dimensions.size(),
									(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2],
									nullptr, nullptr);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i)
		hipFree(devicePtrs[i]);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execBroadcast_1) {

	if (!Environment::getInstance().isExperimentalBuild())
        return;

	NDArray x('c', {2,3,4}, {100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100}, sd::DataType::INT32);
    NDArray y('c', {3},   {10, 20, 30}, sd::DataType::INT64);
    NDArray z('c', {2,3,4}, {100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100}, sd::DataType::INT32);
	NDArray exp('c', {2,3,4}, {10, 11, 12, 13,24, 25, 26, 27,38, 39, 40, 41,22, 23, 24, 25,36, 37, 38, 39,50, 51, 52, 53}, sd::DataType::INT32);
	x.linspace(0); x.syncToDevice();

    std::vector<int> dimensions = {1};

    // evaluate xTad data
    shape::TAD xTad;
    xTad.init(x.shapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function
    std::vector<std::pair<void*,size_t>> hostData;
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));							// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));	// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));							// 2 -- xTadOffsets
	std::vector<void*> devicePtrs(hostData.size(), nullptr);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it
	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execBroadcast(&lc, sd::broadcast::Add,
										nullptr, x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
										nullptr, y.shapeInfo(), y.specialBuffer(), y.specialShapeInfo(),
										nullptr, z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
										(int*)devicePtrs[0], dimensions.size(),
										(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2],
										nullptr, nullptr);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i)
		hipFree(devicePtrs[i]);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execBroadcast_2) {

	if (!Environment::getInstance().isExperimentalBuild())
        return;

	NDArray x('c', {2,3,4}, {100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100}, sd::DataType::INT32);
    NDArray y('c', {2,4},   {10,20,30,40,50,60,70,80}, sd::DataType::FLOAT32);
    NDArray z('c', {2,3,4}, {100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100}, sd::DataType::FLOAT32);
	NDArray exp('c', {2,3,4}, {10., 21., 32., 43., 14., 25., 36., 47., 18., 29., 40., 51., 62., 73., 84., 95., 66., 77., 88., 99., 70., 81., 92., 103}, sd::DataType::FLOAT32);
	x.linspace(0); x.syncToDevice();

    std::vector<int> dimensions = {0,2};

    // evaluate xTad data
    shape::TAD xTad;
    xTad.init(x.shapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function
    std::vector<std::pair<void*,size_t>> hostData;
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));							// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));	// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));							// 2 -- xTadOffsets
	std::vector<void*> devicePtrs(hostData.size(), nullptr);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it
	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execBroadcast(&lc, sd::broadcast::Add,
										nullptr, x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
										nullptr, y.shapeInfo(), y.specialBuffer(), y.specialShapeInfo(),
										nullptr, z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
										(int*)devicePtrs[0], dimensions.size(),
										(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2],
										nullptr, nullptr);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i)
		hipFree(devicePtrs[i]);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execBroadcastBool_1) {

	NDArray x('c', {2,3,4}, {100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100}, sd::DataType::INT32);
    NDArray y('c', {3},   {2, 12, 22}, sd::DataType::INT32);
    NDArray z('c', {2,3,4}, {100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,}, sd::DataType::BOOL);
	NDArray exp('c', {2,3,4}, {0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0}, sd::DataType::BOOL);
	x.linspace(1); x.syncToDevice();

    std::vector<int> dimensions = {1};

    // evaluate xTad data
    shape::TAD xTad;
    xTad.init(x.shapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function
    std::vector<std::pair<void*,size_t>> hostData;
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));							// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));	// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));							// 2 -- xTadOffsets
	std::vector<void*> devicePtrs(hostData.size(), nullptr);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it
	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execBroadcastBool(&lc, sd::broadcast::EqualTo,
										nullptr, x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
										nullptr, y.shapeInfo(), y.specialBuffer(), y.specialShapeInfo(),
										nullptr, z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
                                        nullptr,
										(int*)devicePtrs[0], dimensions.size(),
										(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2],
										nullptr, nullptr);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i)
		hipFree(devicePtrs[i]);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execBroadcastBool_2) {

	NDArray x('c', {2,3,4}, {100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100},sd::DataType::FLOAT32);
    NDArray y('c', {2,4},   {1,10,10,15,20,20,20,24}, sd::DataType::FLOAT32);
    NDArray z('c', {2,3,4}, {100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100}, sd::DataType::BOOL);
	NDArray exp('c', {2,3,4}, {1, 0, 0, 0,0, 0, 0, 0,0, 1, 0, 0,0, 0, 0, 0,0, 0, 0, 0,0, 0, 0, 1}, sd::DataType::BOOL);
	x.linspace(1); x.syncToDevice();

    std::vector<int> dimensions = {0,2};

    // evaluate xTad data
    shape::TAD xTad;
    xTad.init(x.shapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function
    std::vector<std::pair<void*,size_t>> hostData;
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));							// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));	// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));							// 2 -- xTadOffsets

	std::vector<void*> devicePtrs(hostData.size(), nullptr);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it
	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execBroadcastBool(&lc, sd::broadcast::EqualTo,
										nullptr, x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
										nullptr, y.shapeInfo(), y.specialBuffer(), y.specialShapeInfo(),
										nullptr, z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
										nullptr,
										(int*)devicePtrs[0], dimensions.size(),
										(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2],
										nullptr, nullptr);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i)
		hipFree(devicePtrs[i]);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execPairwiseTransform_1) {

	if (!Environment::getInstance().isExperimentalBuild())
        return;

	NDArray x('c', {2,2,2}, {1,5,3,7,2,6,4,8}, sd::DataType::INT32);
    NDArray y('c', {4,2}, {0.1,0.2,0.3,0.4,1.5,0.6,0.7,1.8}, sd::DataType::DOUBLE);
    NDArray z('c', {8}, {100,100,100,100,100,100,100,100}, sd::DataType::INT32);
	NDArray exp('c', {8}, {0,1,2,3,3,5,6,6}, sd::DataType::INT32);
	x.permutei({2,1,0});	// -> {1,2,3,4,5,6,7,8}
    x.syncShape();

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execPairwiseTransform(&lc, sd::pairwise::Subtract,
												nullptr, x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
												nullptr, y.shapeInfo(), y.specialBuffer(), y.specialShapeInfo(),
												nullptr, z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
												nullptr);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execPairwiseBoolTransform_1) {

	NDArray x('c', {2,2,2}, {1,5,3,7,2,6,4,8}, sd::DataType::INT64);
    NDArray y('c', {4,2}, {0,2,0,4,0,6,0,8}, sd::DataType::INT64);
    NDArray z('c', {8}, {100,100,100,100,100,100,100,100}, sd::DataType::BOOL);
	NDArray exp('c', {8}, {0,1,0,1,0,1,0,1}, sd::DataType::BOOL);
	x.permutei({2,1,0});	// -> {1,2,3,4,5,6,7,8}
	x.syncShape();

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execPairwiseBoolTransform(&lc, sd::pairwise::EqualTo,
													nullptr, x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
													nullptr, y.shapeInfo(), y.specialBuffer(), y.specialShapeInfo(),
													nullptr, z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
													nullptr);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}


////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execTransformFloat_1) {

	NDArray x('c', {2,2}, {0, 6.25, 2.25, 12.25}, sd::DataType::DOUBLE);
    NDArray z('c', {4}, {100,100,100,100}, sd::DataType::FLOAT32);
	NDArray exp('c', {4}, {0, 1.5, 2.5, 3.5}, sd::DataType::FLOAT32);
	x.permutei({1,0});
	x.syncShape();

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execTransformFloat(&lc, sd::transform::Sqrt,
		nullptr, x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
		nullptr, z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
		nullptr, nullptr, nullptr);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execTransformFloat_2) {

	NDArray x('c', {1,4}, {0, 4, 9, 16}, sd::DataType::INT64);
    NDArray z('c', {2,2}, {100,100,100,100}, sd::DataType::DOUBLE);
	NDArray exp('c', {2,2}, {0, 2, 3, 4}, sd::DataType::DOUBLE);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execTransformFloat(&lc, sd::transform::Sqrt,
		nullptr, x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
		nullptr, z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
		nullptr, nullptr, nullptr);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execTransformAny_1) {

	NDArray x('c', {2,2}, {0, 6.25, 2.25, 12.25}, sd::DataType::DOUBLE);
    NDArray z('c', {4,1}, {100,100,100,100}, sd::DataType::INT32);
	NDArray exp('c', {4,1}, {0, 2, 6, 12}, sd::DataType::INT32);
	x.permutei({1,0});

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execTransformAny(&lc, sd::transform::Assign,
		nullptr, x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
		nullptr, z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
		nullptr, nullptr, nullptr);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execTransformAny_2) {

	NDArray x('c', {1,4}, {0, 6.25, 2.25, 12.25}, sd::DataType::BFLOAT16);
    NDArray z('c', {2,2}, {100,100,100,100}, sd::DataType::FLOAT32);
	NDArray exp('c', {2,2}, {0, 6.25, 2.25, 12.25}, sd::DataType::FLOAT32);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execTransformAny(&lc, sd::transform::Assign,
		nullptr, x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
		nullptr, z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
		nullptr, nullptr, nullptr);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execTransformStrict_1) {

	NDArray x('c', {2,3}, {0,2,4,1,3,5}, sd::DataType::DOUBLE);
    NDArray z('c', {3,2}, {100,100,100,100,100,100}, sd::DataType::DOUBLE);
	NDArray exp('c', {3,2}, {0, 3, 12, 27, 48, 75}, sd::DataType::DOUBLE);
	x.permutei({1,0});

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execTransformStrict(&lc, sd::transform::CubeDerivative,
		nullptr, x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
		nullptr, z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
		nullptr, nullptr, nullptr);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execTransformStrict_2) {

	NDArray x('c', {6}, {0,1,2,3,4,5}, sd::DataType::FLOAT32);
    NDArray z('c', {3,2}, {100,100,100,100,100,100}, sd::DataType::FLOAT32);
	NDArray exp('c', {3,2}, {0, 3, 12, 27, 48, 75}, sd::DataType::FLOAT32);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execTransformStrict(&lc, sd::transform::CubeDerivative,
		nullptr, x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
		nullptr, z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
		nullptr, nullptr, nullptr);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
	z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execTransformSame_1) {

	NDArray x('c', {2,3}, {0,2.5,4.5,1.5,3.5,5.5}, sd::DataType::DOUBLE);
    NDArray z('c', {1,6}, {100,100,100,100,100,100}, sd::DataType::DOUBLE);
	NDArray exp('c', {1,6}, {0,2.25,6.25,12.25,20.25,30.25}, sd::DataType::DOUBLE);
	x.permutei({1,0});

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execTransformSame(&lc, sd::transform::Square,
		nullptr, x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
		nullptr, z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
		nullptr, nullptr, nullptr);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execTransformSame_2) {

	NDArray x('c', {6}, {0,1,2,3,4,5}, sd::DataType::INT32);
    NDArray z('c', {3,2}, {100,100,100,100,100,100}, sd::DataType::INT32);
	NDArray exp('c', {3,2}, {0,1,4,9,16,25}, sd::DataType::INT32);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execTransformSame(&lc, sd::transform::Square,
		nullptr, x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
		nullptr, z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
		nullptr, nullptr, nullptr);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execTransformBool_1) {

	NDArray x('c', {2,3}, {0,2,4,-1,-3,-5}, sd::DataType::DOUBLE);
    NDArray z('c', {1,6}, {100,100,100,100,100,100}, sd::DataType::BOOL);
	NDArray exp('c', {1,6}, {0,0,1,0,1,0}, sd::DataType::BOOL);
	x.permutei({1,0});

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execTransformBool(&lc, sd::transform::IsPositive,
		nullptr, x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
		nullptr, z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
		nullptr, nullptr, nullptr);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execTransformBool_2) {

	NDArray x('c', {6}, {0,-1,2,-3,4,-5}, sd::DataType::INT32);
    NDArray z('c', {3,2}, {100,100,100,100,100,100}, sd::DataType::BOOL);
	NDArray exp('c', {3,2}, {0,0,1,0,1,0}, sd::DataType::BOOL);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execTransformBool(&lc, sd::transform::IsPositive,
		nullptr, x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
		nullptr, z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
		nullptr, nullptr, nullptr);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execReduceFloat_1) {

    NDArray x('c', {2,3,4}, {-5,-4,-3,-2,-1,0,1,2,3,4,5,6,7,8,9,10,11,12,13,14,15,16,17,18}, sd::DataType::INT32);
    NDArray z('c', {3}, {100,100,100}, sd::DataType::FLOAT32);
    NDArray exp('c', {3}, {2.5, 6.5, 10.5}, sd::DataType::FLOAT32);
    x.permutei({2,1,0});

    std::vector<int> dimensions = {0,2};

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// call cuda kernel which calculates result
	std::vector<int> dims = sd::ShapeUtils::evalDimsForReduceOp(x.rankOf(), dimensions);
    NativeOpExecutioner::execReduceFloat(&lc, sd::reduce::Mean, nullptr, x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(), nullptr, nullptr, z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo(), dims.data(), dims.size());

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream);
	ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execReduceFloat_2) {

    NDArray x('c', {2,3,4}, {-5,-4,-3,-2,-1,0,1,2,3,4,5,6,7,8,9,10,11,12,13,14,15,16,17,18}, sd::DataType::INT32);
    NDArray z('c', {2,4}, {100,100,100,100,100,100,100,100}, sd::DataType::DOUBLE);
    NDArray exp('c', {2,4}, {-1., 0., 1., 2.,11., 12., 13., 14.}, sd::DataType::DOUBLE);

    std::vector<int> dimensions = {1};

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// call cuda kernel which calculates result
	std::vector<int> dims = sd::ShapeUtils::evalDimsForReduceOp(x.rankOf(), dimensions);
    NativeOpExecutioner::execReduceFloat(&lc, sd::reduce::Mean, nullptr, x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(), nullptr, nullptr, z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo(), dims.data(), dims.size());

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream);
	ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execReduceSame_1) {

    NDArray x('c', {2,3,4}, {-5,-4,-3,-2,-1,0,1,2,3,4,5,6,7,8,9,10,11,12,13,14,15,16,17,18}, sd::DataType::INT32);
    NDArray z('c', {3}, {100,100,100}, sd::DataType::INT32);
    NDArray exp('c', {3}, {20, 52, 84}, sd::DataType::INT32);
    x.permutei({2,1,0});

    std::vector<int> dimensions = {0,2};

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// call cuda kernel which calculates result
	std::vector<int> dims = sd::ShapeUtils::evalDimsForReduceOp(x.rankOf(), dimensions);
    NativeOpExecutioner::execReduceSame(&lc, sd::reduce::Sum, nullptr, x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(), nullptr, nullptr, z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo(), dims.data(), dims.size());

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream);
	ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execReduceSame_2) {

    NDArray x('c', {2,3,4}, {-5,-4,-3,-2,-1,0,1,2,3,4,5,6,7,8,9,10,11,12,13,14,15,16,17,18}, sd::DataType::FLOAT32);
    NDArray z('c', {2,4}, {100,100,100,100,100,100,100,100}, sd::DataType::FLOAT32);
    NDArray exp('c', {2,4}, {-3., 0., 3., 6.,33., 36., 39., 42.}, sd::DataType::FLOAT32);

    std::vector<int> dimensions = {1};

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// call cuda kernel which calculates result
	std::vector<int> dims = sd::ShapeUtils::evalDimsForReduceOp(x.rankOf(), dimensions);
    NativeOpExecutioner::execReduceSame(&lc, sd::reduce::Sum, nullptr, x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(), nullptr, nullptr, z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo(), dims.data(), dims.size());

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream);
	ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execReduceBool_1) {

    NDArray x('c', {2,3,4}, {-5,-4,-3,-2,-1,0,1,2,3,4,5,6,-7,-8,-9,-10,-11,-12,-13,-14,-15,-16,-17,-18}, sd::DataType::INT32);
    NDArray z('c', {3}, {100,100,100}, sd::DataType::BOOL);
    NDArray exp('c', {3}, {0, 1, 1}, sd::DataType::BOOL);
    x.permutei({2,1,0});

    std::vector<int> dimensions = {0,2};

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// call cuda kernel which calculates result
	std::vector<int> dims = sd::ShapeUtils::evalDimsForReduceOp(x.rankOf(), dimensions);
    NativeOpExecutioner::execReduceBool(&lc, sd::reduce::IsPositive, nullptr, x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(), nullptr, nullptr, z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo(), dims.data(), dims.size());

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream);
	ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execReduceBool_2) {

    NDArray x('c', {2,3,4}, {-5,-4,-3,-2,-1,0,1,2,3,4,5,6,-7,-8,-9,-10,-11,-12,-13,-14,-15,-16,-17,-18}, sd::DataType::FLOAT32);
    NDArray z('c', {2,4}, {100,100,100,100,100,100,100,100}, sd::DataType::BOOL);
    NDArray exp('c', {2,4}, {1, 1, 1, 1, 0, 0, 0, 0}, sd::DataType::BOOL);

    std::vector<int> dimensions = {1};

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// call cuda kernel which calculates result
	std::vector<int> dims = sd::ShapeUtils::evalDimsForReduceOp(x.rankOf(), dimensions);
    NativeOpExecutioner::execReduceBool(&lc, sd::reduce::IsPositive, nullptr, x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(), nullptr, nullptr, z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo(), dims.data(), dims.size());

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream);
	ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execReduceLong_1) {

    NDArray x('c', {2,3,4}, {-5,0,-3,0,-1,0,1,2,3,4,5,6,7,0,9,10,11,0,13,14,0,16,0,18}, sd::DataType::INT32);
    NDArray z('c', {3}, {100,100,100}, sd::DataType::INT64);
    NDArray exp('c', {3}, {5,6,6}, sd::DataType::INT64);
    x.permutei({2,1,0});

    std::vector<int> dimensions = {0,2};

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// call cuda kernel which calculates result
	std::vector<int> dims = sd::ShapeUtils::evalDimsForReduceOp(x.rankOf(), dimensions);
    NativeOpExecutioner::execReduceLong(&lc, sd::reduce::CountNonZero, nullptr, x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(), nullptr, nullptr, z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo(), dims.data(), dims.size());

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream);
	ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execReduceLong_2) {

    NDArray x('c', {2,3,4}, {-5,0,-3,0,-1,0,1,2,3,4,5,6,7,0,9,10,11,0,13,14,0,16,0,18}, sd::DataType::FLOAT32);
    NDArray z('c', {2,4}, {100,100,100,100,100,100,100,100}, sd::DataType::INT64);
    NDArray exp('c', {2,4}, {3, 1, 3, 2, 2, 1, 2, 3}, sd::DataType::INT64);

    std::vector<int> dimensions = {1};

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// call cuda kernel which calculates result
	std::vector<int> dims = sd::ShapeUtils::evalDimsForReduceOp(x.rankOf(), dimensions);
    NativeOpExecutioner::execReduceLong(&lc, sd::reduce::CountNonZero, nullptr, x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(), nullptr, nullptr, z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo(), dims.data(), dims.size());

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream);
	ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execReduceFloatScalar_1) {

    NDArray x('c', {2,3,4}, {-5,-4,-3,-2,-1,0,1,2,3,4,5,6,7,8,9,10,11,12,13,14,15,16,17,18}, sd::DataType::INT32);
    NDArray z('c', {}, std::vector<double>{100}, sd::DataType::FLOAT32);
    NDArray exp('c', {}, std::vector<double>{6.5}, sd::DataType::FLOAT32);
    x.permutei({2,1,0});

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);
	void* reductionPointer;
    cudaResult = hipMalloc(reinterpret_cast<void **>(&reductionPointer),  1024*1024); ASSERT_EQ(0, cudaResult);
    int* allocationPointer;
	cudaResult = hipMalloc(reinterpret_cast<void **>(&allocationPointer), 1024*1024); ASSERT_EQ(0, cudaResult);
	lc.setReductionPointer(reductionPointer);
	lc.setAllocationPointer(allocationPointer);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduceFloatScalar(&lc, sd::reduce::Mean,
					nullptr, x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
					nullptr,
					nullptr, z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo());

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execReduceFloatScalar_2) {

    NDArray x('c', {2,3,4}, {-5,-4,-3,-2,-1,0,1,2,3,4,5,6,7,8,9,10,11,12,13,14,15,16,17,18}, sd::DataType::INT32);
    NDArray z('c', {}, std::vector<double>{100}, sd::DataType::DOUBLE);
    NDArray exp('c', {}, std::vector<double>{6.5}, sd::DataType::DOUBLE);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);
	void* reductionPointer;
    cudaResult = hipMalloc(reinterpret_cast<void **>(&reductionPointer),  1024*1024); ASSERT_EQ(0, cudaResult);
    int* allocationPointer;
	cudaResult = hipMalloc(reinterpret_cast<void **>(&allocationPointer), 1024*1024); ASSERT_EQ(0, cudaResult);
	lc.setReductionPointer(reductionPointer);
	lc.setAllocationPointer(allocationPointer);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduceFloatScalar(&lc, sd::reduce::Mean,
					nullptr, x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
					nullptr,
					nullptr, z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo());

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execReduceSameScalar_1) {

    NDArray x('c', {2,3,4}, {-5,-4,-3,-2,-1,0,1,2,3,4,5,6,7,8,9,10,11,12,13,14,15,16,17,18}, sd::DataType::INT32);
    NDArray z('c', {}, std::vector<double>{100}, sd::DataType::INT32);
    NDArray exp('c', {}, std::vector<double>{156}, sd::DataType::INT32);
    x.permutei({2,1,0});

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);
	void* reductionPointer;
    cudaResult = hipMalloc(reinterpret_cast<void **>(&reductionPointer),  1024*1024); ASSERT_EQ(0, cudaResult);
    int* allocationPointer;
	cudaResult = hipMalloc(reinterpret_cast<void **>(&allocationPointer), 1024*1024); ASSERT_EQ(0, cudaResult);
	lc.setReductionPointer(reductionPointer);
	lc.setAllocationPointer(allocationPointer);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduceSameScalar(&lc, sd::reduce::Sum,
					nullptr, x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
					nullptr,
					nullptr, z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo());

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execReduceSameScalar_2) {

    NDArray x('c', {2,3,4}, {-5,-4,-3,-2,-1,0,1,2,3,4,5,6,7,8,9,10,11,12,13,14,15,16,17,18}, sd::DataType::DOUBLE);
    NDArray z('c', {}, std::vector<double>{100}, sd::DataType::DOUBLE);
    NDArray exp('c', {}, std::vector<double>{156}, sd::DataType::DOUBLE);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);
	void* reductionPointer;
    cudaResult = hipMalloc(reinterpret_cast<void **>(&reductionPointer),  1024*1024); ASSERT_EQ(0, cudaResult);
    int* allocationPointer;
	cudaResult = hipMalloc(reinterpret_cast<void **>(&allocationPointer), 1024*1024); ASSERT_EQ(0, cudaResult);
	lc.setReductionPointer(reductionPointer);
	lc.setAllocationPointer(allocationPointer);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduceSameScalar(&lc, sd::reduce::Sum,
					nullptr, x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
					nullptr,
					nullptr, z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo());

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execReduceBoolScalar_1) {

    NDArray x('c', {2,3,4}, {-5,-4,-3,-2,-1,0,1,2,3,4,5,6,-7,-8,-9,-10,-11,-12,-13,-14,-15,-16,-17,-18}, sd::DataType::INT32);
    NDArray z('c', {}, std::vector<double>{100}, sd::DataType::BOOL);
    NDArray exp('c', {}, std::vector<double>{1}, sd::DataType::BOOL);
    x.permutei({2,1,0});
    x.syncShape();

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);
	void* reductionPointer;
    cudaResult = hipMalloc(reinterpret_cast<void **>(&reductionPointer),  1024*1024); ASSERT_EQ(0, cudaResult);
    int* allocationPointer;
	cudaResult = hipMalloc(reinterpret_cast<void **>(&allocationPointer), 1024*1024); ASSERT_EQ(0, cudaResult);
	lc.setReductionPointer(reductionPointer);
	lc.setAllocationPointer(allocationPointer);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduceBoolScalar(&lc, sd::reduce::IsPositive,
					nullptr, x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
					nullptr,
					nullptr, z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo());

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execReduceBoolScalar_2) {

    NDArray x('c', {2,3,4}, {-5,-4,-3,-2,-1,0,1,2,3,4,5,6,-7,-8,-9,-10,-11,-12,-13,-14,-15,-16,-17,-18}, sd::DataType::DOUBLE);
    NDArray z('c', {}, std::vector<double>{100}, sd::DataType::BOOL);
    NDArray exp('c', {}, std::vector<double>{1}, sd::DataType::BOOL);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);
	void* reductionPointer;
    cudaResult = hipMalloc(reinterpret_cast<void **>(&reductionPointer),  1024*1024); ASSERT_EQ(0, cudaResult);
    int* allocationPointer;
	cudaResult = hipMalloc(reinterpret_cast<void **>(&allocationPointer), 1024*1024); ASSERT_EQ(0, cudaResult);
	lc.setReductionPointer(reductionPointer);
	lc.setAllocationPointer(allocationPointer);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduceBoolScalar(&lc, sd::reduce::IsPositive,
					nullptr, x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
					nullptr,
					nullptr, z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo());

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execReduceLongScalar_1) {

    NDArray x('c', {2,3,4}, {-5,0,-3,0,-1,0,1,2,3,4,5,6,7,0,9,10,11,0,13,14,0,16,0,18}, sd::DataType::INT32);
    NDArray z('c', {}, std::vector<double>{100}, sd::DataType::INT64);
    NDArray exp('c', {}, std::vector<double>{17}, sd::DataType::INT64);
    x.permutei({2,1,0});
    x.syncShape();

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);
	void* reductionPointer;
    cudaResult = hipMalloc(reinterpret_cast<void **>(&reductionPointer),  1024*1024); ASSERT_EQ(0, cudaResult);
    int* allocationPointer;
	cudaResult = hipMalloc(reinterpret_cast<void **>(&allocationPointer), 1024*1024); ASSERT_EQ(0, cudaResult);
	lc.setReductionPointer(reductionPointer);
	lc.setAllocationPointer(allocationPointer);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduceLongScalar(&lc, sd::reduce::CountNonZero,
					nullptr, x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
					nullptr,
					nullptr, z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo());

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execReduceLongScalar_2) {

    NDArray x('c', {2,3,4}, {-5,0,-3,0,-1,0,1,2,3,4,5,6,7,0,9,10,11,0,13,14,0,16,0,18}, sd::DataType::DOUBLE);
    NDArray z('c', {}, std::vector<double>{100}, sd::DataType::INT64);
    NDArray exp('c', {}, std::vector<double>{17}, sd::DataType::INT64);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);
	void* reductionPointer;
    cudaResult = hipMalloc(reinterpret_cast<void **>(&reductionPointer),  1024*1024); ASSERT_EQ(0, cudaResult);
    int* allocationPointer;
	cudaResult = hipMalloc(reinterpret_cast<void **>(&allocationPointer), 1024*1024); ASSERT_EQ(0, cudaResult);
	lc.setReductionPointer(reductionPointer);
	lc.setAllocationPointer(allocationPointer);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduceLongScalar(&lc, sd::reduce::CountNonZero,
					nullptr, x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
					nullptr,
					nullptr, z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo());

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execReduce3TAD_1) {

    NDArray x('c', {2,2,3}, {-5,-4,-3,-2,-1,0,1,2,3,4,5,6}, sd::DataType::FLOAT32);
    NDArray y('c', {2,2}, {1,2,3,4}, sd::DataType::FLOAT32);
    NDArray exp('c', {3}, {10,20,30}, sd::DataType::DOUBLE);
    NDArray z('c', {3}, {100,100,100}, sd::DataType::DOUBLE);

    std::vector<int> dimensions = {0,1};
    auto packX = ConstantTadHelper::getInstance().tadForDimensions(x.shapeInfo(), dimensions);
    LaunchContext* context = x.getContext();

	x.syncToDevice();
	y.syncToDevice();
	PointersManager pm(context, "execReduce3TAD_1");
	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduce3TAD(context, sd::reduce3::Dot,
								nullptr, x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
								nullptr,
								nullptr, y.shapeInfo(), y.specialBuffer(), y.specialShapeInfo(),
								nullptr, z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
								nullptr, dimensions.size(),
								packX.specialShapeInfo(), packX.specialOffsets(), nullptr, nullptr);
    pm.synchronize();
//	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();
//    z.printIndexedBuffer("OutputReduce3TAD");
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execReduce3TAD_2) {

    NDArray x('c', {2,2,3}, {-5,-4,-3,-2,-1,0,1,2,3,4,5,6}, sd::DataType::INT64);
    NDArray y('c', {2,3}, {1,2,3,4,5,6}, sd::DataType::INT64);
    NDArray exp('c', {2}, {10,73}, sd::DataType::FLOAT32);
    NDArray z('c', {2}, {100,100}, sd::DataType::FLOAT32);

    std::vector<int> dimensions = {0,2};

    // evaluate xTad data
    shape::TAD xTad;
    xTad.init(x.shapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function
    std::vector<std::pair<void*,size_t>> hostData;
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));						// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));						// 2 -- xTadOffsets
	std::vector<void*> devicePtrs(hostData.size(), nullptr);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it

	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduce3TAD(&lc, sd::reduce3::Dot,
								nullptr, x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
								nullptr,
								nullptr, y.shapeInfo(), y.specialBuffer(), y.specialShapeInfo(),
								nullptr, z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
								(int*)devicePtrs[0], dimensions.size(),
								(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2], nullptr, nullptr);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) hipFree(devicePtrs[i]);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execReduce3TAD_3) {

    NDArray x('c', {2,2,3}, {-5,-4,-3,-2,-1,0,1,2,3,4,5,6}, sd::DataType::INT64);
    NDArray y('c', {3}, {1,2,3}, sd::DataType::INT64);
    NDArray exp('c', {2,2}, {-22,-4,14,32}, sd::DataType::FLOAT32);
    NDArray z('c', {2,2}, {100,100,100,100}, sd::DataType::FLOAT32);

    std::vector<int> dimensions = {2};

    // evaluate xTad data
    shape::TAD xTad;
    xTad.init(x.shapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function
    std::vector<std::pair<void*,size_t>> hostData;
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));						// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));						// 2 -- xTadOffsets
	std::vector<void*> devicePtrs(hostData.size(), nullptr);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it

	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduce3TAD(&lc, sd::reduce3::Dot,
								nullptr, x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
								nullptr,
								nullptr, y.shapeInfo(), y.specialBuffer(), y.specialShapeInfo(),
								nullptr, z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
								(int*)devicePtrs[0], dimensions.size(),
								(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2], (Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2]);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) hipFree(devicePtrs[i]);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execReduce3TAD_4) {

    NDArray x('c', {2,2,3}, {-5,-4,-3,-2,-1,0,1,2,3,4,5,6}, sd::DataType::DOUBLE);
    NDArray y('c', {2,2,3}, {10,20,30,40,50,60,70,80,90,100,110,120}, sd::DataType::DOUBLE);
    NDArray exp('c', {}, std::vector<double>{1820}, sd::DataType::FLOAT32);
    NDArray z('c', {}, std::vector<double>{100}, sd::DataType::FLOAT32);

    std::vector<int> dimensions = {0,1,2};

    // evaluate xTad data
    shape::TAD xTad;
    xTad.init(x.shapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function
    std::vector<std::pair<void*,size_t>> hostData;
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));						// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));						// 2 -- xTadOffsets
	std::vector<void*> devicePtrs(hostData.size(), nullptr);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it
	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduce3TAD(&lc, sd::reduce3::Dot,
								nullptr, x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
								nullptr,
								nullptr, y.shapeInfo(), y.specialBuffer(), y.specialShapeInfo(),
								nullptr, z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
								(int*)devicePtrs[0], dimensions.size(),
								(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2], (Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2]);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) hipFree(devicePtrs[i]);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execSummaryStats_1) {
  // FIXME: Yurii, this test should be fixed
    if (1 > 0)
      return;
    	
    NDArray x('c', {2,2,3}, {-5,-4,-3,-2,-1,0,1,2,3,4,5,6}, sd::DataType::INT64);
    NDArray exp('c', {}, std::vector<double>{3.605551}, sd::DataType::FLOAT32);
    NDArray z('c', {}, std::vector<double>{100}, sd::DataType::FLOAT32);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);
	void* reductionPointer;
    cudaResult = hipMalloc(reinterpret_cast<void **>(&reductionPointer), 1024*1024); ASSERT_EQ(0, cudaResult);
	lc.setReductionPointer(reductionPointer);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execSummaryStats(&lc, sd::variance::SummaryStatsStandardDeviation,
								nullptr, x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
								nullptr,
								nullptr, z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
								true);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execSummaryStats_2) {

    NDArray x('c', {2,2,3}, {-5,-4,-3,-20,-1,0,1,2,3,4,5,6}, sd::DataType::DOUBLE);
    NDArray exp('c', {2}, {3.405877, 9.715966}, sd::DataType::FLOAT32);
    NDArray z('c', {2}, {100,100}, sd::DataType::FLOAT32);

    std::vector<int> dimensions = {0,2};

    // evaluate xTad data
    shape::TAD xTad;
    xTad.init(x.shapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function
    std::vector<std::pair<void*,size_t>> hostData;
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));						// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));						// 2 -- xTadOffsets
	std::vector<void*> devicePtrs(hostData.size(), nullptr);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it
	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execSummaryStats(&lc, sd::variance::SummaryStatsStandardDeviation,
								nullptr, x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
								nullptr,
								nullptr, z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
								(int*)devicePtrs[0], dimensions.size(),
								(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2],
								true);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) hipFree(devicePtrs[i]);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}
/*
////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execSummaryStats_3) {

    NDArray x('c', {2,2,3}, {-5,-4,-3,-20,-1,0,1,2,3,4,5,6}, sd::DataType::DOUBLE);
    NDArray exp('c', {2}, {10.606602, 2.121320}, sd::DataType::FLOAT32);
    NDArray z('c', {2}, {100,100}, sd::DataType::FLOAT32);

    std::vector<int> dimensions = {1};

    // evaluate xTad data
    shape::TAD xTad;
    xTad.init(x.shapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function
    std::vector<std::pair<void*,size_t>> hostData;
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));						// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));						// 2 -- xTadOffsets
	std::vector<void*> devicePtrs(hostData.size(), nullptr);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it
	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execSummaryStats(&lc, sd::variance::SummaryStatsStandardDeviation,
								nullptr, x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
								nullptr, 								
								nullptr, z.shapeInfo(), z.specialBuffer(), z.special(),
								(int*)devicePtrs[0], dimensions.size(), 
								(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2],
								true);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) hipFree(devicePtrs[i]);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}
*/

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execSummaryStatsScalar_1) {

    NDArray x('c', {2,2,3}, {-5,-4,-3,-2,-1,0,1,2,3,4,5,6}, sd::DataType::INT64);
    NDArray exp('c', {}, std::vector<double>{3.605551}, sd::DataType::FLOAT32);
    NDArray z('c', {}, std::vector<double>{100}, sd::DataType::FLOAT32);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);
	void* reductionPointer;
    cudaResult = hipMalloc(reinterpret_cast<void **>(&reductionPointer), 1024*1024); ASSERT_EQ(0, cudaResult);
	lc.setReductionPointer(reductionPointer);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execSummaryStatsScalar(&lc, sd::variance::SummaryStatsStandardDeviation,
								nullptr, x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
								nullptr,
								nullptr, z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
								true);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

//////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execRandom_1) {

//    NDArray z('c', {10}, {100,0,0,0,0,0,0,0,0,0}, sd::DataType::DOUBLE);
    NDArray z('c', {10}, {100,0,0,0,0,0,0,0,0,100}, sd::DataType::FLOAT32);
    NDArray exp('c', {10}, {0.050942, -0.183229, -0.093921, 0.075469, 0.257166, -0.254838, 0.342227, -0.682188, -0.004345, 0.464633}, sd::DataType::FLOAT32);

    sd::graph::RandomGenerator gen(119,5);

	hipError_t cudaResult;
    NDArray* array = &z;
    ExtraArguments arguments({0.f, 0.5f});
    auto context = z.getContext();
    PointersManager pm(context, "tests::execRandom_1");
//    z.printIndexedBuffer("Input data");
//    z.syncToDevice();
    NativeOpExecutioner::execRandom(context, random::GaussianDistribution, &gen, array->buffer(), array->shapeInfo(), array->specialBuffer(), array->specialShapeInfo(), array->buffer(), array->shapeInfo(), array->specialBuffer(), array->specialShapeInfo(), array->buffer(), array->shapeInfo(), array->specialBuffer(), array->specialShapeInfo(), arguments.argumentsAsT(array->dataType()));
    pm.synchronize();
    z.tickWriteDevice();
//	z.printIndexedBuffer("Output Gaussian");
//    RandomLauncher::fillGaussian(context, gen, &z,  0.f, 0.5f);
//    pm.synchronize();
//    z.tickWriteDevice();
//    z.printIndexedBuffer("Output Gaussian");

//    hipStream_t stream;
//    cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
//    LaunchContext lc(&stream);
//
//	//	::execRandom(extraPointers, random::GaussianDistribution, &gen, z.buffer(), z.shapeInfo(), z.specialBuffer(), z.special(), &extra);
//	// call cuda kernel which calculates result
//	NativeOpExecutioner::execRandom(&lc, sd::random::GaussianDistribution,
//								&gen,
//								nullptr, z.shapeInfo(), z.specialBuffer(), z.special(),
//								nullptr, z.shapeInfo(), z.specialBuffer(), z.special(),
//								nullptr, z.shapeInfo(), z.specialBuffer(), z.special(),
//								extraArguments.argumentsAsT(z.dataType()));
//
//	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
//	ASSERT_EQ(cudaResult, 0);
//    z.tickWriteDevice();
//    z.syncToHost();
//    z.printIndexedBuffer("Random1");
    ASSERT_EQ(exp, z);
// 	// verify results
// 	for (int e = 0; e < z.lengthOf(); e++)
// 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);
//    hipFree(dExtraArgs);
	// free allocated global device memory
//	hipFree(dGen);
	// delete cuda stream
//	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

//////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execRandom_2) {

    NDArray x('c', {10}, {0.1,0.2,0.3,0.4,0.5,0.6,0.7,0.8,0.9,1}, sd::DataType::DOUBLE);
    NDArray z('c', {2,5}, {100,100,100,100,100,100,100,100,100,100}, sd::DataType::DOUBLE);
    NDArray exp('c', {10}, {0., 0., 0.3, 0., 0.5, 0., 0.7, 0., 0., 1.}, sd::DataType::DOUBLE);

    ExtraArguments extraArguments({0.7});
    sd::graph::RandomGenerator gen(119,5);

//    // prepare input arrays for prepareDataForCuda function
//    std::vector<std::pair<void*,size_t>> hostData;
//	hostData.emplace_back(extraArguments.data(), extraArguments.size() * sizeof(double));		// 0 -- dimensions
//	std::vector<void*> devicePtrs(hostData.size(), nullptr);
//
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
//	hipStream_t stream;
//	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext* lc = x.getContext(); //(&stream);

	// allocate required amount of global device memory and copy host data to it
//	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execRandom(lc, sd::random::DropOut,
								&gen,
								nullptr, x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
								nullptr, z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
								extraArguments.argumentsAsT(z.dataType()));

	cudaResult = hipStreamSynchronize(*lc->getCudaStream()); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();
    z.syncToHost();
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
//	for(int i = 0; i < devicePtrs.size(); ++i) hipFree(devicePtrs[i]);

	// delete cuda stream
//	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

//////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execRandom_3) {

    NDArray z('c', {10}, {100,100,100,100,100,100,100,100,100,100}, sd::DataType::DOUBLE);
    NDArray exp('c', {10}, {2.373649, 2.239791, 1.887353, 2.488636, 2.068904, 2.281399, 1.828228, 2.228222, 2.490847, 1.669537}, sd::DataType::DOUBLE);

    std::vector<double> extraArguments = {1.5, 2.5};
    sd::graph::RandomGenerator gen(119,5);

    // prepare input arrays for prepareDataForCuda function
    std::vector<std::pair<void*,size_t>> hostData;
	hostData.emplace_back(extraArguments.data(), extraArguments.size() * sizeof(double));		// 0 -- dimensions
	std::vector<void*> devicePtrs(hostData.size(), nullptr);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it
	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execRandom(&lc, sd::random::UniformDistribution,
								&gen,
								nullptr, z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
								devicePtrs[0]);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) hipFree(devicePtrs[i]);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

//////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execRandom_4) {

    NDArray z('c', {2,5}, {1,2,3,4,5,6,7,8,9,10}, sd::DataType::FLOAT32);
    NDArray exp('c', {10}, {2.373649, 2.281399, 2.239791, 1.828228, 1.887353, 2.228222, 2.488636, 2.490847, 2.068904, 1.669537}, sd::DataType::FLOAT32);
    z.permutei({1,0});

    ExtraArguments extraArguments({1.5, 2.5});
    sd::graph::RandomGenerator gen(119,5);

//    // prepare input arrays for prepareDataForCuda function
//    std::vector<std::pair<void*,size_t>> hostData;
//	hostData.emplace_back(extraArguments.data(), extraArguments.size() * sizeof(double));		// 0 -- dimensions
//	std::vector<void*> devicePtrs(hostData.size(), nullptr);

	// create cuda stream and LaunchContext
//	hipError_t cudaResult;
//	hipStream_t stream;
//	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
//	LaunchContext lc(&stream);
//
//	// allocate required amount of global device memory and copy host data to it
//	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
    auto context = z.getContext();
    PointersManager pm(context, "execRandom4");
	// call cuda kernel which calculates result
	NativeOpExecutioner::execRandom(context, sd::random::UniformDistribution,
								&gen,
								nullptr, z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
								extraArguments.argumentsAsT(z.dataType()));

//	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();
//    z.printIndexedBuffer("Output Uniform4");
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
//	for(int i = 0; i < devicePtrs.size(); ++i) hipFree(devicePtrs[i]);

	// delete cuda stream
//	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

