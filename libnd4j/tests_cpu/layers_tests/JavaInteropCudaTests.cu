/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
//

#include "testlayers.h"
#include <helpers/PointersManager.h>
#include <array/ExtraArguments.h>
#include <ops/declarable/CustomOperations.h>
#include <array>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

using namespace sd;
using namespace sd::ops;

class JavaInteropCudaTests : public testing::Test {
public:

};

TEST_F(JavaInteropCudaTests, test_DeclarableOp_execution_1) {
    auto x = NDArrayFactory::create<float>('c', {3, 5});
    auto y = NDArrayFactory::create<float>('c', {5}, {1.f, 1.f, 1.f, 1.f, 1.f});
    auto e = NDArrayFactory::create<float>('c', {3, 5});
    x.assign(1.f);
    e.assign(2.f);

    sd::ops::add op;
    Context context(1);

    context.setCudaContext(LaunchContext::defaultContext()->getCudaStream(), LaunchContext::defaultContext()->getReductionPointer(), LaunchContext::defaultContext()->getAllocationPointer());
    context.setInputArray(0, x.buffer(), x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo());
    context.setInputArray(1, y.buffer(), y.shapeInfo(), y.specialBuffer(), y.specialShapeInfo());

    context.setOutputArray(0, x.buffer(), x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo());

    PointersManager pm(LaunchContext::defaultContext(), "test_DeclarableOp_execution_1");
    execCustomOp2(nullptr, op.getOpHash(), &context);

    pm.synchronize();

    ASSERT_EQ(e, x);
}

TEST_F(JavaInteropCudaTests, test_DeclarableOp_execution_2) {
    NDArray x('c', {3, 1, 2}, sd::DataType::FLOAT32);
    NDArray y('c', {2, 2}, sd::DataType::FLOAT32);
    NDArray z('c', {3, 2, 2}, sd::DataType::BOOL);
    NDArray e('c', {3, 2, 2}, sd::DataType::BOOL);

    x.assign(1.f);
    y.assign(2.f);
    e.assign(false);

    sd::ops::equals op;
    Context context(1);

    context.setCudaContext(LaunchContext::defaultContext()->getCudaStream(), LaunchContext::defaultContext()->getReductionPointer(), LaunchContext::defaultContext()->getAllocationPointer());
    context.setInputArray(0, x.buffer(), x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo());
    context.setInputArray(1, y.buffer(), y.shapeInfo(), y.specialBuffer(), y.specialShapeInfo());

    context.setOutputArray(0, z.buffer(), z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo());

    PointersManager pm(LaunchContext::defaultContext(), "test_DeclarableOp_execution_2");
    execCustomOp2(nullptr, op.getOpHash(), &context);

    pm.synchronize();

    ASSERT_EQ(e, z);
}

