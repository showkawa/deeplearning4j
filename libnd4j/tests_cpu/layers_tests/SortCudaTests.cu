/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author raver119@gmail.com
//

#include "testlayers.h"
#include <ops/declarable/CustomOperations.h>
#include <array/NDArray.h>
#include <legacy/NativeOps.h>
#include <helpers/BitwiseUtils.h>

using namespace sd;
using namespace sd::graph;

class SortCudaTests : public testing::Test {
public:

};


TEST_F(SortCudaTests, test_linear_sort_by_key_1) {
    auto k = NDArrayFactory::create<Nd4jLong>('c', {10}, {1, 3, 5, 9, 0, 2, 4, 6, 7, 8});
    auto v = NDArrayFactory::create<double>('c', {10}, {1.5, 3.5, 5.5, 9.5, 0.5, 2.5, 4.5, 6.5, 7.5, 8.5});

    auto ek = NDArrayFactory::create<Nd4jLong>('c', {10}, {0, 1, 2, 3, 4, 5, 6, 7, 8, 9});
    auto ev = NDArrayFactory::create<double>('c', {10}, {0.5, 1.5, 2.5, 3.5, 4.5, 5.5, 6.5, 7.5, 8.5, 9.5});

    Nd4jPointer extras[2] = {nullptr, LaunchContext::defaultContext()->getCudaStream()};

    sortByKey(extras, k.buffer(), k.shapeInfo(), k.specialBuffer(), k.specialShapeInfo(), v.buffer(), v.shapeInfo(), v.specialBuffer(), v.specialShapeInfo(), false);
    k.tickWriteDevice();
    v.tickWriteDevice();

    ASSERT_EQ(ek, k);
    ASSERT_EQ(ev, v);
}

TEST_F(SortCudaTests, test_linear_sort_by_val_1) {
    auto k = NDArrayFactory::create<Nd4jLong>('c', {10}, {1, 3, 5, 9, 0, 2, 4, 6, 7, 8});
    auto v = NDArrayFactory::create<double>('c', {10}, {1.5, 3.5, 5.5, 9.5, 0.5, 2.5, 4.5, 6.5, 7.5, 8.5});

    auto ek = NDArrayFactory::create<Nd4jLong>('c', {10}, {0, 1, 2, 3, 4, 5, 6, 7, 8, 9});
    auto ev = NDArrayFactory::create<double>('c', {10}, {0.5, 1.5, 2.5, 3.5, 4.5, 5.5, 6.5, 7.5, 8.5, 9.5});

    Nd4jPointer extras[2] = {nullptr, LaunchContext::defaultContext()->getCudaStream()};

    sortByValue(extras, k.buffer(), k.shapeInfo(), k.specialBuffer(), k.specialShapeInfo(), v.buffer(), v.shapeInfo(), v.specialBuffer(), v.specialShapeInfo(), false);
    k.tickWriteDevice();
    v.tickWriteDevice();

    ASSERT_EQ(ek, k);
    ASSERT_EQ(ev, v);
}

TEST_F(SortCudaTests, test_linear_sort_by_val_2) {
    auto k = NDArrayFactory::create<int>('c', {6}, {0, 1, 2, 3, 4, 5});
//    auto v = NDArrayFactory::create<double>('c', {6}, {1.5, 3.5, 5.5, 9.5, 0.5, 2.5, 4.5, 6.5, 7.5, 8.5});
    NDArray v = NDArrayFactory::create<double>('c', {6}, {0.9f, .75f, .6f, .95f, .5f, .3f});
    auto ek = NDArrayFactory::create<int>('c', {6}, {3, 0, 1, 2, 4, 5});
    auto ev = NDArrayFactory::create<double>('c', {6}, {0.95, 0.9, 0.75, 0.6, 0.5, 0.3});

    Nd4jPointer extras[2] = {nullptr, LaunchContext::defaultContext()->getCudaStream()};

    sortByValue(extras, k.buffer(), k.shapeInfo(), k.specialBuffer(), k.specialShapeInfo(), v.buffer(), v.shapeInfo(), v.specialBuffer(), v.specialShapeInfo(), true);
    k.tickWriteDevice();
    v.tickWriteDevice();
    // k.printIndexedBuffer("KEYS");
    ASSERT_EQ(ek, k);
    ASSERT_EQ(ev, v);
}

TEST_F(SortCudaTests, test_tad_sort_by_key_1) {
    auto k = NDArrayFactory::create<Nd4jLong>('c', {2, 10}, {1, 3, 5, 9, 0, 2, 4, 6, 7, 8,   1, 3, 5, 9, 0, 2, 4, 6, 7, 8});
    auto v = NDArrayFactory::create<double>('c', {2, 10}, {1.5, 3.5, 5.5, 9.5, 0.5, 2.5, 4.5, 6.5, 7.5, 8.5,   1.5, 3.5, 5.5, 9.5, 0.5, 2.5, 4.5, 6.5, 7.5, 8.5});

    auto ek = NDArrayFactory::create<Nd4jLong>('c', {2, 10}, {0, 1, 2, 3, 4, 5, 6, 7, 8, 9,     0, 1, 2, 3, 4, 5, 6, 7, 8, 9});
    auto ev = NDArrayFactory::create<double>('c', {2, 10}, {0.5, 1.5, 2.5, 3.5, 4.5, 5.5, 6.5, 7.5, 8.5, 9.5,     0.5, 1.5, 2.5, 3.5, 4.5, 5.5, 6.5, 7.5, 8.5, 9.5});

    Nd4jPointer extras[2] = {nullptr, LaunchContext::defaultContext()->getCudaStream()};

    int axis = 1;
    sortTadByKey(extras, k.buffer(), k.shapeInfo(), k.specialBuffer(), k.specialShapeInfo(), v.buffer(), v.shapeInfo(), v.specialBuffer(), v.specialShapeInfo(), &axis, 1, false);
    k.tickWriteDevice();
    v.tickWriteDevice();

    // k.printIndexedBuffer("k");
    // v.printIndexedBuffer("v");

    ASSERT_EQ(ek, k);
    ASSERT_EQ(ev, v);
}

TEST_F(SortCudaTests, test_tad_sort_by_val_1) {
    auto k = NDArrayFactory::create<Nd4jLong>('c', {2, 10}, {1, 3, 5, 9, 0, 2, 4, 6, 7, 8,   1, 3, 5, 9, 0, 2, 4, 6, 7, 8});
    auto v = NDArrayFactory::create<double>('c', {2, 10}, {1.5, 3.5, 5.5, 9.5, 0.5, 2.5, 4.5, 6.5, 7.5, 8.5,   1.5, 3.5, 5.5, 9.5, 0.5, 2.5, 4.5, 6.5, 7.5, 8.5});

    auto ek = NDArrayFactory::create<Nd4jLong>('c', {2, 10}, {0, 1, 2, 3, 4, 5, 6, 7, 8, 9,     0, 1, 2, 3, 4, 5, 6, 7, 8, 9});
    auto ev = NDArrayFactory::create<double>('c', {2, 10}, {0.5, 1.5, 2.5, 3.5, 4.5, 5.5, 6.5, 7.5, 8.5, 9.5,     0.5, 1.5, 2.5, 3.5, 4.5, 5.5, 6.5, 7.5, 8.5, 9.5});

    Nd4jPointer extras[2] = {nullptr, LaunchContext::defaultContext()->getCudaStream()};

    int axis = 1;
    sortTadByValue(extras, k.buffer(), k.shapeInfo(), k.specialBuffer(), k.specialShapeInfo(), v.buffer(), v.shapeInfo(), v.specialBuffer(), v.specialShapeInfo(), &axis, 1, false);
    k.tickWriteDevice();
    v.tickWriteDevice();

    ASSERT_EQ(ek, k);
    ASSERT_EQ(ev, v);
}
