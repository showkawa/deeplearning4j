#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

 //
 // @author raver119@gmail.com
 //

#include "testlayers.h"
#include <array/NDArray.h>
#include <array/NDArrayFactory.h>
#include <graph/Context.h>
#include <graph/Node.h>
#include <graph/Variable.h>
#include <graph/VariableSpace.h>
#include <execution/LaunchContext.h>
#include <ops/specials_cuda.h>
#include <helpers/TAD.h>
#include <ops/declarable/CustomOperations.h>

#include <hip/hip_runtime.h>

using namespace sd;
using namespace sd::graph;

class NDArrayCudaBasicsTests : public testing::Test {
public:

};

//////////////////////////////////////////////////////////////////////////
static hipError_t allocateDeviceMem(LaunchContext& lc, std::vector<void*>& devicePtrs, const std::vector<std::pair<void*,size_t>>& hostData) {

    if(devicePtrs.size() != hostData.size())
        throw std::invalid_argument("prepareDataForCuda: two input sts::vectors should same sizes !");

    hipError_t cudaResult;

    void* reductionPointer;
    cudaResult = hipMalloc(reinterpret_cast<void **>(&reductionPointer),  1024*1024);			if(cudaResult != 0) return cudaResult;
    int* allocationPointer;
    cudaResult = hipMalloc(reinterpret_cast<void **>(&allocationPointer), 1024*1024);			if(cudaResult != 0) return cudaResult;

    lc.setReductionPointer(reductionPointer);
    lc.setAllocationPointer(allocationPointer);
    hipStream_t stream = *lc.getCudaStream();

    for(int i = 0; i < devicePtrs.size(); ++i) {

        cudaResult = hipMalloc(reinterpret_cast<void **>(&devicePtrs[i]), hostData[i].second); if(cudaResult != 0) return cudaResult;
        hipMemcpyAsync(devicePtrs[i], hostData[i].first, hostData[i].second, hipMemcpyHostToDevice, stream);
    }
    return cudaResult;
}

TEST_F(NDArrayCudaBasicsTests, Test_Registration_1) {
    auto x = NDArrayFactory::create<int>('c', {5}, {1, 2, 3, 4, 5});
    auto y = NDArrayFactory::create<int>('c', {5}, {5, 4, 3, 2, 1});

    ASSERT_TRUE(x.isActualOnDeviceSide());
    ASSERT_FALSE(x.isActualOnHostSide());
}

TEST_F(NDArrayCudaBasicsTests, Test_Registration_2) {
    auto x = NDArrayFactory::create<int>('c', {5});
    auto y = NDArrayFactory::create<int>('c', {5});

    ASSERT_TRUE(x.isActualOnDeviceSide());
    ASSERT_FALSE(x.isActualOnHostSide());
}

TEST_F(NDArrayCudaBasicsTests, Test_Registration_3) {
    auto x = NDArrayFactory::create<int>('c', {5}, {1, 2, 3, 4, 5});
    auto y = NDArrayFactory::create<int>('c', {5}, {5, 4, 3, 2, 1});

    ASSERT_TRUE(x.isActualOnDeviceSide());
    ASSERT_FALSE(x.isActualOnHostSide());

    NDArray::registerSpecialUse({&x}, {&y});

    ASSERT_TRUE(x.isActualOnDeviceSide());
    ASSERT_FALSE(x.isActualOnHostSide());

    ASSERT_TRUE(y.isActualOnDeviceSide());
    ASSERT_FALSE(y.isActualOnHostSide());
}

TEST_F(NDArrayCudaBasicsTests, Test_Registration_01) {
    auto x = NDArrayFactory::create_<int>('c', {5}, {1, 2, 3, 4, 5});
    auto y = NDArrayFactory::create_<int>('c', {5}, {5, 4, 3, 2, 1});

    ASSERT_TRUE(x->isActualOnDeviceSide());
    ASSERT_FALSE(x->isActualOnHostSide());
    delete x;
    delete y;
}

TEST_F(NDArrayCudaBasicsTests, Test_Registration_02) {
    auto x = NDArrayFactory::create_<int>('c', {5});
    auto y = NDArrayFactory::create_<int>('c', {5});

    ASSERT_TRUE(x->isActualOnDeviceSide());
    ASSERT_FALSE(x->isActualOnHostSide());
    delete x;
    delete y;
}

TEST_F(NDArrayCudaBasicsTests, Test_Registration_03) {
    auto x = NDArrayFactory::create_<int>('c', {5}, {1, 2, 3, 4, 5});
    auto y = NDArrayFactory::create_<int>('c', {5}, {5, 4, 3, 2, 1});

    ASSERT_TRUE(x->isActualOnDeviceSide());
    ASSERT_FALSE(x->isActualOnHostSide());

    NDArray::registerSpecialUse({y}, {x});
    x->applyTransform(transform::Neg, *y);
    //ASSERT_TRUE(x->isActualOnDeviceSide());
    //ASSERT_FALSE(x->isActualOnHostSide());

    //ASSERT_TRUE(y->isActualOnDeviceSide());
    //ASSERT_TRUE(y->isActualOnHostSide());
    //y->syncToHost();
    // y->printBuffer("Negatives");
    delete x;
    delete y;
}

TEST_F(NDArrayCudaBasicsTests, Test_Cosine_1) {
    auto x = NDArrayFactory::create_<double>('c', {5}, {1, 2, 3, 4, 5});
    auto y = NDArrayFactory::create_<double>('c', {5}, {5, 4, 3, 2, 1});

    ASSERT_TRUE(x->isActualOnDeviceSide());
    ASSERT_FALSE(x->isActualOnHostSide());

    NDArray::registerSpecialUse({y}, {x});
    x->applyTransform(transform::Cosine, *y);
    //ASSERT_TRUE(x->isActualOnDeviceSide());
    //ASSERT_FALSE(x->isActualOnHostSide());

    //ASSERT_TRUE(y->isActualOnDeviceSide());
    //ASSERT_TRUE(y->isActualOnHostSide());
    //y->syncToHost();
    delete x;
    delete y;
}

//////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, TestAdd_1) {
    // allocating host-side arrays
    auto x = NDArrayFactory::create<double>('c', { 5 }, { 1, 2, 3, 4, 5});
    auto y = NDArrayFactory::create<double>('c', { 5 }, { 1, 2, 3, 4, 5});
    auto z = NDArrayFactory::create<double>('c', { 5 }, {10, 10, 10, 10, 10});

    auto exp = NDArrayFactory::create<double>('c', { 5 }, { 2, 4, 6, 8, 10 });

    // making raw buffers
    //Nd4jPointer devBufferPtrX, devBufferPtrZ, devShapePtrX;
    //hipError_t res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrX), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrZ), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devShapePtrX), shape::shapeInfoByteLength(x.shapeInfo()));
    //ASSERT_EQ(0, res);

    Nd4jPointer nativeStream = (Nd4jPointer)malloc(sizeof(hipStream_t));
    CHECK_ALLOC(nativeStream, "Failed to allocate memory for new CUDA stream", sizeof(hipStream_t));
    hipError_t dZ = hipStreamCreate(reinterpret_cast<hipStream_t *>(&nativeStream));
    auto stream = reinterpret_cast<hipStream_t *>(&nativeStream);

    //hipMemcpyAsync(devBufferPtrX, x.buffer(), x.lengthOf() * x.sizeOfT(), hipMemcpyHostToDevice, *stream);
    //hipMemcpyAsync(devShapePtrX, x.shapeInfo(), shape::shapeInfoByteLength(x.shapeInfo()), hipMemcpyHostToDevice, *stream);

    LaunchContext lc(stream, nullptr, nullptr);
    NativeOpExecutioner::execPairwiseTransform(&lc, pairwise::Add, x.buffer(), x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(), y.buffer(), y.shapeInfo(), y.specialBuffer(), y.specialShapeInfo(), z.buffer(), z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo(), nullptr);
    z.tickWriteDevice();
    auto res = hipStreamSynchronize(*stream);
    ASSERT_EQ(0, res);

    for (int e = 0; e < z.lengthOf(); e++)
        ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);
}

//////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, TestAdd_2) {
    // allocating host-side arrays
    NDArray x('c', { 5 }, { 1, 2, 3, 4, 5});
    NDArray y('c', { 5 }, { 1, 2, 3, 4, 5});
    NDArray z('c', { 5 }, sd::DataType::DOUBLE);

    NDArray exp('c', { 5 }, { 2, 4, 6, 8, 10 });

    Nd4jPointer nativeStream = (Nd4jPointer)malloc(sizeof(hipStream_t));
    CHECK_ALLOC(nativeStream, "Failed to allocate memory for new CUDA stream", sizeof(hipStream_t));
    hipError_t dZ = hipStreamCreate(reinterpret_cast<hipStream_t *>(&nativeStream));
    auto stream = reinterpret_cast<hipStream_t *>(&nativeStream);

    LaunchContext lc(stream, *stream, nullptr, nullptr);
    NativeOpExecutioner::execPairwiseTransform(&lc, pairwise::Add, nullptr, x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(), nullptr, y.shapeInfo(), y.specialBuffer(), y.specialShapeInfo(), nullptr, z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo(), nullptr);
    auto res = hipStreamSynchronize(*stream);
    ASSERT_EQ(0, res);

    for (int e = 0; e < z.lengthOf(); e++)
        ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);
}

//////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, TestAdd_3) {
    // allocating host-side arrays
    auto x = NDArrayFactory::create<double>('c', { 5 }, { 1, 2, 3, 4, 5});
    auto y = NDArrayFactory::create<double>('c', { 5 }, { 1, 2, 3, 4, 5});
    auto z = NDArrayFactory::create<double>('c', { 5 }, {10, 10, 10, 10, 10});

    auto exp = NDArrayFactory::create<double>('c', { 5 }, { 2, 4, 6, 8, 10 });

    // making raw buffers
    //Nd4jPointer devBufferPtrX, devBufferPtrZ, devShapePtrX;
    //hipError_t res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrX), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrZ), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devShapePtrX), shape::shapeInfoByteLength(x.shapeInfo()));
    //ASSERT_EQ(0, res);

    Nd4jPointer nativeStream = (Nd4jPointer)malloc(sizeof(hipStream_t));
    CHECK_ALLOC(nativeStream, "Failed to allocate memory for new CUDA stream", sizeof(hipStream_t));
    hipError_t dZ = hipStreamCreate(reinterpret_cast<hipStream_t *>(&nativeStream));
    auto stream = reinterpret_cast<hipStream_t *>(&nativeStream);

    //hipMemcpyAsync(devBufferPtrX, x.buffer(), x.lengthOf() * x.sizeOfT(), hipMemcpyHostToDevice, *stream);
    //hipMemcpyAsync(devShapePtrX, x.shapeInfo(), shape::shapeInfoByteLength(x.shapeInfo()), hipMemcpyHostToDevice, *stream);

    LaunchContext lc(stream, *stream, nullptr, nullptr);
    NativeOpExecutioner::execPairwiseTransform(&lc, pairwise::Add, x.buffer(), x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(), y.buffer(), y.shapeInfo(), y.specialBuffer(), y.specialShapeInfo(), z.buffer(), z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo(), nullptr);
    z.tickWriteDevice();
    auto res = hipStreamSynchronize(*stream);
    ASSERT_EQ(0, res);
    //double* localBuffer = ;
    z.syncToHost();
    hipMemcpy(z.buffer(), z.specialBuffer(), z.lengthOf() * z.sizeOfT(), hipMemcpyDeviceToHost);
    res = hipStreamSynchronize(*stream);
    z.tickWriteHost();
    ASSERT_EQ(0, res);

    //
    // hipFree(devBufferPtrX);
    //hipFree(devBufferPtrZ);
    //hipFree(devShapePtrX);

    for (int e = 0; e < z.lengthOf(); e++) {
        ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);
    }
}

//////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, TestAdd_4) {
    // allocating host-side arrays
    auto x = NDArrayFactory::create<double>('c', { 5 }, { 1, 2, 3, 4, 5});
    auto y = NDArrayFactory::create<double>('c', { 5 }, { 1, 2, 3, 4, 5});
    auto z = NDArrayFactory::create<double>('c', { 5 });

    auto exp = NDArrayFactory::create<double>('c', { 5 }, { 2, 4, 6, 8, 10 });

    // making raw buffers
    //Nd4jPointer devBufferPtrX, devBufferPtrZ, devShapePtrX;
    //hipError_t res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrX), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrZ), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devShapePtrX), shape::shapeInfoByteLength(x.shapeInfo()));
    //ASSERT_EQ(0, res);
    x.applyPairwiseTransform(pairwise::Add, y, z);

    //
    // hipFree(devBufferPtrX);
    //hipFree(devBufferPtrZ);
    //hipFree(devShapePtrX);

    for (int e = 0; e < z.lengthOf(); e++) {
        ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);
    }
}

//////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, TestAdd_5) {
    // allocating host-side arrays
    auto x = NDArrayFactory::create<double>('c', { 5 }, { 1, 2, 3, 4, 5});
    auto y = NDArrayFactory::create<double>('c', { 5 }, { 1, 2, 3, 4, 5});
    //auto z = NDArrayFactory::create<double>('c', { 5 });

    auto exp = NDArrayFactory::create<double>('c', { 5 }, { 2, 4, 6, 8, 10 });

    // making raw buffers
    //Nd4jPointer devBufferPtrX, devBufferPtrZ, devShapePtrX;
    //hipError_t res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrX), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrZ), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devShapePtrX), shape::shapeInfoByteLength(x.shapeInfo()));
    //ASSERT_EQ(0, res);
    x += y;
    //x.applyPairwiseTransform(pairwise::Add, &y, &z, nullptr);
    x.syncToHost();
    //y.printBuffer("3Y = ");
    //z.printBuffer("3Result out");

    //
    // hipFree(devBufferPtrX);
    //hipFree(devBufferPtrZ);
    //hipFree(devShapePtrX);

    for (int e = 0; e < x.lengthOf(); e++) {
        ASSERT_NEAR(exp.e<double>(e), x.e<double>(e), 1e-5);
    }
}

//////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, TestAdd_6) {
    // allocating host-side arrays
    auto x = NDArrayFactory::create<double>('c', { 5 }, { 1, 2, 3, 4, 5});
    auto y = NDArrayFactory::create<double>(2); //.'c', { 5 }, { 1, 2, 3, 4, 5});
    //auto z = NDArrayFactory::create<double>('c', { 5 });

    auto exp = NDArrayFactory::create<double>('c', { 5 }, { 3, 4, 5, 6, 7 });

    // making raw buffers
    //Nd4jPointer devBufferPtrX, devBufferPtrZ, devShapePtrX;
    //hipError_t res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrX), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrZ), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devShapePtrX), shape::shapeInfoByteLength(x.shapeInfo()));
    //ASSERT_EQ(0, res);
    x += y;
    //x.applyPairwiseTransform(pairwise::Add, &y, &z, nullptr);
    x.syncToHost();

    // hipFree(devBufferPtrX);
    //hipFree(devBufferPtrZ);
    //hipFree(devShapePtrX);

    for (int e = 0; e < x.lengthOf(); e++) {
        ASSERT_NEAR(exp.e<double>(e), x.e<double>(e), 1e-5);
    }
}

//////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, TestAdd_7) {
    // allocating host-side arrays
    auto x = NDArrayFactory::create<double>('c', { 5 }, { 1, 2, 3, 4, 5});
    //auto y = NDArrayFactory::create<double>(2); //.'c', { 5 }, { 1, 2, 3, 4, 5});
    //auto z = NDArrayFactory::create<double>('c', { 5 });

    auto exp = NDArrayFactory::create<double>('c', { 5 }, { 3, 4, 5, 6, 7 });

    // making raw buffers
    //Nd4jPointer devBufferPtrX, devBufferPtrZ, devShapePtrX;
    //hipError_t res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrX), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrZ), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devShapePtrX), shape::shapeInfoByteLength(x.shapeInfo()));
    //ASSERT_EQ(0, res);
    x += 2.;
    //x.applyPairwiseTransform(pairwise::Add, &y, &z, nullptr);
    x.syncToHost();

    // hipFree(devBufferPtrX);
    //hipFree(devBufferPtrZ);
    //hipFree(devShapePtrX);

    for (int e = 0; e < x.lengthOf(); e++) {
        ASSERT_NEAR(exp.e<double>(e), x.e<double>(e), 1e-5);
    }
}

//////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, TestMultiply_1) {
    // allocating host-side arrays
    auto x = NDArrayFactory::create<double>('c', { 5 }, { 1, 2, 3, 4, 5});
    auto y = NDArrayFactory::create<double>('c', { 5 }, { 1, 2, 3, 4, 5});
    auto z = NDArrayFactory::create<double>('c', { 5 });

    auto exp = NDArrayFactory::create<double>('c', { 5 }, { 1, 4, 9, 16, 25 });

    // making raw buffers
    //Nd4jPointer devBufferPtrX, devBufferPtrZ, devShapePtrX;
    //hipError_t res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrX), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrZ), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devShapePtrX), shape::shapeInfoByteLength(x.shapeInfo()));
    //ASSERT_EQ(0, res);
    x.applyPairwiseTransform(pairwise::Multiply, y, z);
    // x.printBuffer("3X = ");
    // y.printBuffer("3Y = ");
    // z.printBuffer("3Result out");

    //
    // hipFree(devBufferPtrX);
    //hipFree(devBufferPtrZ);
    //hipFree(devShapePtrX);

    for (int e = 0; e < z.lengthOf(); e++) {
        ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);
    }
}

//////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, TestMultiply_2) {
    // allocating host-side arrays
    auto x = NDArrayFactory::create<double>('c', { 5 }, { 1, 2, 3, 4, 5});
    auto y = NDArrayFactory::create<double>('c', { 5 }, { 1, 2, 3, 4, 5});
    NDArray z('c', { 5 }, sd::DataType::DOUBLE);

    auto exp = NDArrayFactory::create<double>('c', { 5 }, { 1, 4, 9, 16, 25 });

    // making raw buffers
    //Nd4jPointer devBufferPtrX, devBufferPtrZ, devShapePtrX;
    //hipError_t res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrX), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrZ), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devShapePtrX), shape::shapeInfoByteLength(x.shapeInfo()));
    //ASSERT_EQ(0, res);
    x.applyPairwiseTransform(pairwise::Multiply, y, z);

    //
    // hipFree(devBufferPtrX);
    //hipFree(devBufferPtrZ);
    //hipFree(devShapePtrX);

    for (int e = 0; e < z.lengthOf(); e++) {
        ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);
    }
}

//////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, TestMultiply_3) {
    // allocating host-side arrays
    NDArray x('c', { 5 }, { 1, 2, 3, 4, 5}, sd::DataType::DOUBLE);
    NDArray y('c', { 5 }, { 1., 2., 3., 4., 5.}, sd::DataType::DOUBLE);
    auto z = NDArrayFactory::create<double>('c', { 5 });

    auto exp = NDArrayFactory::create<double>('c', { 5 }, { 1, 4, 9, 16, 25 });

    // making raw buffers
    //Nd4jPointer devBufferPtrX, devBufferPtrZ, devShapePtrX;
    //hipError_t res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrX), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrZ), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devShapePtrX), shape::shapeInfoByteLength(x.shapeInfo()));
    //ASSERT_EQ(0, res);
    x.applyPairwiseTransform(pairwise::Multiply, y, z);
    //x.printBuffer("23X = ");
    //y.printBuffer("23Y = ");
    // z.printBuffer("23Result out");

    //
    // hipFree(devBufferPtrX);
    //hipFree(devBufferPtrZ);
    //hipFree(devShapePtrX);

    for (int e = 0; e < z.lengthOf(); e++) {
        ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);
    }
}

//////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, TestMultiply_4) {
    // allocating host-side arrays
    NDArray x('c', { 5 }, { 1, 2, 3, 4, 5}, sd::DataType::DOUBLE);
    NDArray y('c', { 5 }, { 1., 2., 3., 4., 5.}, sd::DataType::DOUBLE);
    //auto z = NDArrayFactory::create<double>('c', { 5 });

    auto exp = NDArrayFactory::create<double>('c', { 5 }, { 1, 4, 9, 16, 25 });

    // making raw buffers
    //Nd4jPointer devBufferPtrX, devBufferPtrZ, devShapePtrX;
    //hipError_t res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrX), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrZ), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devShapePtrX), shape::shapeInfoByteLength(x.shapeInfo()));
    //ASSERT_EQ(0, res);
    //x.applyPairwiseTransform(pairwise::Multiply, &y, &z, nullptr);
    //x.printBuffer("23X = ");
    //y.printBuffer("23Y = ");
    x *= y;
    //x.tickWriteDevice();
    // x.printBuffer("33Result out");

    //
    // hipFree(devBufferPtrX);
    //hipFree(devBufferPtrZ);
    //hipFree(devShapePtrX);

    for (int e = 0; e < x.lengthOf(); e++) {
        ASSERT_NEAR(exp.e<double>(e), x.e<double>(e), 1e-5);
    }
}

//////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, TestPrimitiveNeg_01) {
    // allocating host-side arrays
    auto x = NDArrayFactory::create<int>('c', { 5 }, { 1, 2, 3, 4, 5});
    auto y = NDArrayFactory::create<int>('c', { 5 }, { 1, 2, 3, 4, 5});
    auto exp = NDArrayFactory::create<int>('c', { 5 }, { -1, -2, -3, -4, -5 });

    auto stream = x.getContext()->getCudaStream();//reinterpret_cast<hipStream_t *>(&nativeStream);

    NativeOpExecutioner::execTransformSame(x.getContext(), transform::Neg, x.buffer(), x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(), y.buffer(), y.shapeInfo(), y.specialBuffer(), y.specialShapeInfo(), nullptr, nullptr, nullptr);
    auto res = hipStreamSynchronize(*stream);
    ASSERT_EQ(0, res);
    y.tickWriteDevice();

    // x.printBuffer("X = ");
    // y.printBuffer("Y = ");

    for (int e = 0; e < y.lengthOf(); e++) {
        ASSERT_NEAR(exp.e<int>(e), y.e<int>(e), 1e-5);
    }
}

TEST_F(NDArrayCudaBasicsTests, Test_PrimitiveNeg_2) {
    auto x = NDArrayFactory::create<double>('c', {5}, {1, 2, 3, 4, 5});
    auto y = NDArrayFactory::create<double>('c', {5});

    ASSERT_TRUE(x.isActualOnDeviceSide());
    ASSERT_FALSE(x.isActualOnHostSide());

    x.applyTransform(transform::Neg, y);
    //ASSERT_TRUE(x->isActualOnDeviceSide());
    //ASSERT_FALSE(x->isActualOnHostSide());

    //ASSERT_TRUE(y->isActualOnDeviceSide());
    //ASSERT_TRUE(y->isActualOnHostSide());
    //auto res = hipStreamSynchronize(*y.getContext()->getCudaStream());
    //ASSERT_EQ(0, res);
    // y.printBuffer("Negatives2");
    //delete x;
    //delete y;
}

TEST_F(NDArrayCudaBasicsTests, Test_PrimitiveSqrt_1) { // strict
    auto x = NDArrayFactory::create<double>('c', {5}, {1, 2, 3, 4, 5});
    auto y = NDArrayFactory::create<double>('c', {5});
    auto exp = NDArrayFactory::create<double>({1.000000, 1.414214, 1.732051, 2.000000, 2.236068});
    ASSERT_TRUE(x.isActualOnDeviceSide());
    ASSERT_FALSE(x.isActualOnHostSide());

    x.applyTransform(transform::Sqrt, y);
    //ASSERT_TRUE(x->isActualOnDeviceSide());
    //ASSERT_FALSE(x->isActualOnHostSide());

    //ASSERT_TRUE(y->isActualOnDeviceSide());
    //ASSERT_TRUE(y->isActualOnHostSide());
    //auto res = hipStreamSynchronize(*y.getContext()->getCudaStream());
    //ASSERT_EQ(0, res);
    ASSERT_TRUE(y.equalsTo(exp));
    //y.printBuffer("SQRT output");
    //delete x;
    //delete y;
}

TEST_F(NDArrayCudaBasicsTests, Test_PrimitiveAssign_1) { // strict
    auto x = NDArrayFactory::create<double>('c', {5}, {1, 2, 3, 4, 5});
    auto y = NDArrayFactory::create<double>('c', {5});
    //auto exp = NDArrayFactory::create<double>({1.000000, 1.414214, 1.732051, 2.000000, 2.236068});
    //ASSERT_TRUE(x.isActualOnDeviceSide());
    //ASSERT_TRUE(x.isActualOnHostSide());

    x.applyTransform(transform::Assign, y);
    //ASSERT_TRUE(x->isActualOnDeviceSide());
    //ASSERT_FALSE(x->isActualOnHostSide());

    //ASSERT_TRUE(y->isActualOnDeviceSide());
    //ASSERT_TRUE(y->isActualOnHostSide());
    //auto res = hipStreamSynchronize(*y.getContext()->getCudaStream());
    //ASSERT_EQ(0, res);

    // printf("Assigned to another array\n");
    // y.printBuffer("OUput");
    ASSERT_TRUE(y.equalsTo(x));
    //y.syncToHost();
    //y.printBuffer("IsMax output");
    //delete x;
    //delete y;
}

TEST_F(NDArrayCudaBasicsTests, Test_PrimitiveCosine_1) { // strict
    auto x = NDArrayFactory::create<double>('c', {5}, {1, 2, 3, 4, 5});
    auto y = NDArrayFactory::create<double>('c', {5});
    auto exp = NDArrayFactory::create<double>('c', {5}, {0.540302, -0.416147, -0.989992, -0.653644, 0.283662});

    ASSERT_TRUE(x.isActualOnDeviceSide());
    ASSERT_FALSE(x.isActualOnHostSide());

    x.applyTransform(transform::Cosine, y);
    //ASSERT_TRUE(x->isActualOnDeviceSide());
    //ASSERT_FALSE(x->isActualOnHostSide());

    //ASSERT_TRUE(y->isActualOnDeviceSide());
    //ASSERT_TRUE(y->isActualOnHostSide());
    //auto res = hipStreamSynchronize(*y.getContext()->getCudaStream());
    //ASSERT_EQ(0, res);
    ASSERT_TRUE(exp.isSameShape(y));
    ASSERT_TRUE(exp.dataType() == y.dataType());
    //y.printBuffer("Cosine2");
    //delete x;
    //delete y;
}

TEST_F(NDArrayCudaBasicsTests, Test_PrimitiveCosine_2) {
    auto x = NDArrayFactory::create<double>('c', {5}, {1, 2, 3, 4, 5});
    auto y = NDArrayFactory::create<double>('c', {5});
    auto exp = NDArrayFactory::create<double>('c', {5}, {0.540302, -0.416147, -0.989992, -0.653644, 0.283662});

    ASSERT_TRUE(x.isActualOnDeviceSide());
    ASSERT_FALSE(x.isActualOnHostSide());
    x.applyTransform(transform::Cosine, y);
    //ASSERT_TRUE(x->isActualOnDeviceSide());
    //ASSERT_FALSE(x->isActualOnHostSide());

    //ASSERT_TRUE(y->isActualOnDeviceSide());
    //ASSERT_TRUE(y->isActualOnHostSide());
    //auto res = hipStreamSynchronize(*y.getContext()->getCudaStream());
    //ASSERT_EQ(0, res);
    //exp.syncToHost();
    //y.printBuffer("PrimitiveCosine2");
    //exp.printBuffer("Primitive Cosine exp");
    ASSERT_TRUE(exp.isSameShape(y));
    ASSERT_TRUE(exp.dataType() == y.dataType());
    //for (int e = 0; e < y.lengthOf(); e++) {
    //    ASSERT_NEAR(exp.e<double>(e), y.e<double>(e), 1e-5);
    //}

    ASSERT_TRUE(exp.equalsTo(y));
    //delete x;
    //delete y;
}

TEST_F(NDArrayCudaBasicsTests, Test_PrimitiveCosine_3) {
    auto x = NDArrayFactory::create<double>('c', {5}, {1, 2, 3, 4, 5});
    auto y = NDArrayFactory::create<double>('c', {5});
    auto exp = NDArrayFactory::create<double>({0.540302, -0.416147, -0.989992, -0.653644, 0.283662});

    ASSERT_TRUE(x.isActualOnDeviceSide());
    ASSERT_FALSE(x.isActualOnHostSide());
    x.applyTransform(transform::Cosine, y);
    //ASSERT_TRUE(x->isActualOnDeviceSide());
    //ASSERT_FALSE(x->isActualOnHostSide());

    //ASSERT_TRUE(y->isActualOnDeviceSide());
    //ASSERT_TRUE(y->isActualOnHostSide());
    //auto res = hipStreamSynchronize(*y.getContext()->getCudaStream());
    //ASSERT_EQ(0, res);
    //exp.syncToHost();
//    y.printBuffer("PrimitiveCosine3");
//    exp.printBuffer("Primitive Cosine3 exp");
//    y.printShapeInfo("Y shape");
//    exp.printShapeInfo("Exp Shape");
    ASSERT_TRUE(exp.isSameShape(y));
//
//    for (int e = 0; e < y.lengthOf(); e++) {
//        printf("%lf == %lf\n", exp.e<double>(e), y.e<double>(e));
////        ASSERT_NEAR(exp.e<double>(e), y.e<double>(e), 1e-5);
//    }

    ASSERT_TRUE(exp.equalsTo(y));
    //delete x;
    //delete y;
}

TEST_F(NDArrayCudaBasicsTests, TestRawBroadcast_2) {

    //if (!Environment::getInstance().isExperimentalBuild())
    //    return;

    NDArray x = NDArrayFactory::create<double>('c', {2,3,4});
    NDArray y('c', {2,4},   {10,20,30,40,50,60,70,80}, sd::DataType::DOUBLE);
    NDArray z('c', {2,3,4}, {100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100}, sd::DataType::DOUBLE);
//    NDArray exp('c', {2,3,4}, {10., 21., 32., 43., 14., 25., 36., 47., 18., 29., 40., 51., 62., 73., 84., 95., 66., 77., 88., 99., 70., 81., 92., 103}, sd::DataType::DOUBLE);
    NDArray exp('c', {2,3,4}, {10., 40., 90., 160., 50., 120., 210., 320., 90., 200., 330., 480., 650., 840., 1050., 1280., 850., 1080., 1330., 1600., 1050., 1320., 1610., 1920.}, sd::DataType::DOUBLE);
    x.linspace(1); x.syncToDevice();

    std::vector<int> dimensions = {0,2};

    // evaluate xTad data
    shape::TAD xTad;
    xTad.init(x.shapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function
    std::vector<std::pair<void*,size_t>> hostData;
    hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));							// 0 -- dimensions
    hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));	// 1 -- xTadShapeInfo
    hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));							// 2 -- xTadOffsets
    std::vector<void*> devicePtrs(hostData.size(), nullptr);

    // create cuda stream and LaunchContext
    hipError_t cudaResult;
    hipStream_t stream;
    cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
    LaunchContext lc(&stream);

    // allocate required amount of global device memory and copy host data to it
    cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);

    // call cuda kernel which calculates result
    NativeOpExecutioner::execBroadcast(&lc, sd::broadcast::Multiply,
                                       nullptr, x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
                                       nullptr, y.shapeInfo(), y.specialBuffer(), y.specialShapeInfo(),
                                       nullptr, z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
                                       (int*)devicePtrs[0], dimensions.size(),
                                       (Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2],
                                       nullptr, nullptr);

    cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

    // verify results
    for (int e = 0; e < z.lengthOf(); e++)
        ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

    // free allocated global device memory
    for(int i = 0; i < devicePtrs.size(); ++i)
        hipFree(devicePtrs[i]);

    // delete cuda stream
    cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

TEST_F(NDArrayCudaBasicsTests, TestRawBroadcast_3) {

    //if (!Environment::getInstance().isExperimentalBuild())
    //    return;

    NDArray x('c', {2,3,4}, sd::DataType::DOUBLE);
    NDArray y('c', {2,4},   {10,20,30,40,50,60,70,80}, sd::DataType::DOUBLE);
    NDArray z('c', {2,3,4}, {100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100}, sd::DataType::DOUBLE);
//    NDArray exp('c', {2,3,4}, {10., 21., 32., 43., 14., 25., 36., 47., 18., 29., 40., 51., 62., 73., 84., 95., 66., 77., 88., 99., 70., 81., 92., 103}, sd::DataType::DOUBLE);
    NDArray exp('c', {2,3,4}, {10., 40., 90., 160., 50., 120., 210., 320., 90., 200., 330., 480., 650., 840., 1050., 1280., 850., 1080., 1330., 1600., 1050., 1320., 1610., 1920.}, sd::DataType::DOUBLE);
    x.linspace(1); x.syncToDevice();

    std::vector<int> dimensions = {0,2};

    // evaluate xTad data
    shape::TAD xTad;
    xTad.init(x.shapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function
    std::vector<std::pair<void*,size_t>> hostData;
    hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));							// 0 -- dimensions
    hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));	// 1 -- xTadShapeInfo
    hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));							// 2 -- xTadOffsets
    std::vector<void*> devicePtrs(hostData.size(), nullptr);

    // create cuda stream and LaunchContext
    hipError_t cudaResult;
    //hipStream_t stream;
    //cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
    LaunchContext* pLc = x.getContext();//(&stream);
    hipStream_t* stream = pLc->getCudaStream();
    // allocate required amount of global device memory and copy host data to it
//    cudaResult = allocateDeviceMem(*pLc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
    for(int i = 0; i < devicePtrs.size(); ++i) {

        cudaResult = hipMalloc(reinterpret_cast<void **>(&devicePtrs[i]), hostData[i].second); ASSERT_EQ(0, cudaResult);
        hipMemcpyAsync(devicePtrs[i], hostData[i].first, hostData[i].second, hipMemcpyHostToDevice, *stream);
    }

    NDArray::registerSpecialUse({&z}, {&x, &y});
    // call cuda kernel which calculates result
    NativeOpExecutioner::execBroadcast(pLc, sd::broadcast::Multiply,
                                       nullptr, x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
                                       nullptr, y.shapeInfo(), y.specialBuffer(), y.specialShapeInfo(),
                                       nullptr, z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
                                       (int*)devicePtrs[0], dimensions.size(),
                                       (Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2],
                                       nullptr, nullptr);

    //cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    //z.syncToHost();
    // verify results
    for (int e = 0; e < z.lengthOf(); e++)
        ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

    // free allocated global device memory
    for(int i = 0; i < devicePtrs.size(); ++i)
        hipFree(devicePtrs[i]);
    ASSERT_TRUE(exp.equalsTo(z));
    // delete cuda stream
    //cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}


TEST_F(NDArrayCudaBasicsTests, TestBroadcastMultiply_1) {
    // allocating host-side arrays
    NDArray x('c', { 2, 3 }, { 1, 2, 3, 4, 5, 6}, sd::DataType::DOUBLE);
    NDArray y = NDArrayFactory::create<double>(3.); //'c', { 3 }, { 2., 3., 4.}, sd::DataType::DOUBLE);
    //auto z = NDArrayFactory::create<double>('c', { 5 });

    auto exp = NDArrayFactory::create<double>('c', { 2, 3 }, { 3, 6, 9, 12, 15, 18 });

    // making raw buffers
    //Nd4jPointer devBufferPtrX, devBufferPtrZ, devShapePtrX;
    //hipError_t res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrX), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrZ), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devShapePtrX), shape::shapeInfoByteLength(x.shapeInfo()));
    //ASSERT_EQ(0, res);
    //x.applyPairwiseTransform(pairwise::Multiply, &y, &z, nullptr);
    x *= y;
    //x.syncToHost();

    //
    // hipFree(devBufferPtrX);
    //hipFree(devBufferPtrZ);
    //hipFree(devShapePtrX);
    ASSERT_TRUE(exp.equalsTo(x));
//    for (int e = 0; e < x.lengthOf(); e++) {
//        ASSERT_NEAR(exp.e<double>(e), x.e<double>(e), 1e-5);
//    }
}

TEST_F(NDArrayCudaBasicsTests, TestBroadcastMultiply_01) {
    // allocating host-side arrays
    NDArray x('c', { 2, 3 }, { 1, 2, 3, 4, 5, 6}, sd::DataType::DOUBLE);
    NDArray y = NDArrayFactory::create<double>(3.); //'c', { 3 }, { 2., 3., 4.}, sd::DataType::DOUBLE);
    auto z = NDArrayFactory::create<double>('c', { 2, 3 });

    auto exp = NDArrayFactory::create<double>('c', { 2, 3 }, { 3, 6, 9, 12, 15, 18 });

    // making raw buffers
    //Nd4jPointer devBufferPtrX, devBufferPtrZ, devShapePtrX;
    //hipError_t res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrX), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrZ), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devShapePtrX), shape::shapeInfoByteLength(x.shapeInfo()));
    //ASSERT_EQ(0, res);
    //x.applyPairwiseTransform(pairwise::Multiply, &y, &z, nullptr);
    //x.printBuffer("23X = ");
    //y.printBuffer("23Y = ");
    x.applyTrueBroadcast(BroadcastOpsTuple::Multiply(), y, z);// *= y;
    // z.printBuffer("53Result out");

    //
    // hipFree(devBufferPtrX);
    //hipFree(devBufferPtrZ);
    //hipFree(devShapePtrX);
    ASSERT_TRUE(exp.equalsTo(z));

//    for (int e = 0; e < x.lengthOf(); e++) {
//        ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);
//    }
}

TEST_F(NDArrayCudaBasicsTests, TestBroadcastMultiply_02) {
    // allocating host-side arrays
    auto x = NDArrayFactory::create<double>('c', { 2, 3 }, { 1, 2, 3, 4, 5, 6}); //, sd::DataType::DOUBLE);
    auto y = NDArrayFactory::create<double>('c', {2,3}, {3, 3, 3, 3, 3, 3}); //'c', { 3 }, { 2., 3., 4.}, sd::DataType::DOUBLE);
    auto z = NDArrayFactory::create<double>('c', { 2, 3 });

    auto exp = NDArrayFactory::create<double>('c', { 2, 3 }, { 3, 6, 9, 12, 15, 18 });
    //if (x.isActualOnHostSide() && !x.isActualOnDeviceSide())
    // making raw buffers
    //Nd4jPointer devBufferPtrX, devBufferPtrZ, devShapePtrX;
    //hipError_t res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrX), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrZ), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devShapePtrX), shape::shapeInfoByteLength(x.shapeInfo()));
    //ASSERT_EQ(0, res);
    //x.applyPairwiseTransform(pairwise::Multiply, &y, &z, nullptr);
    //x.printBuffer("23X = ");
    //y.printBuffer("23Y = ");
    x.applyTrueBroadcast(BroadcastOpsTuple::Multiply(), y, z);// *= y;

    // z.printBuffer("52Result out");

    //
    // hipFree(devBufferPtrX);
    //hipFree(devBufferPtrZ);
    //hipFree(devShapePtrX);
    ASSERT_TRUE(exp.equalsTo(z));

//    for (int e = 0; e < x.lengthOf(); e++) {
//        ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);
//    }
}

TEST_F(NDArrayCudaBasicsTests, TestBroadcastMultiply_002) {
    // allocating host-side arrays
    auto x = NDArrayFactory::create<double>('c', { 2, 3 }, { 1, 2, 3, 4, 5, 6}); //, sd::DataType::DOUBLE);
    auto y = NDArrayFactory::create<double>('c', {2, 3}, {2., 3., 3., 3., 3., 3.}); //'c', { 3 }, { 2., 3., 4.}, sd::DataType::DOUBLE);
    auto z = NDArrayFactory::create<double>('c', { 2, 3 });

    auto exp = NDArrayFactory::create<double>('c', { 2, 3 }, { 2, 6, 9, 12, 15, 18 });
    //if (x.isActualOnHostSide() && !x.isActualOnDeviceSide())
    // making raw buffers
    //Nd4jPointer devBufferPtrX, devBufferPtrZ, devShapePtrX;
    //hipError_t res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrX), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrZ), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devShapePtrX), shape::shapeInfoByteLength(x.shapeInfo()));
    //ASSERT_EQ(0, res);
    //x.applyPairwiseTransform(pairwise::Multiply, &y, &z, nullptr);
    //x.printBuffer("23X = ");
    //y.printBuffer("23Y = ");
    x.applyPairwiseTransform(pairwise::Multiply, y, z);// *= y;

    // z.printBuffer("51Result out");

    //
    // hipFree(devBufferPtrX);
    //hipFree(devBufferPtrZ);
    //hipFree(devShapePtrX);
    ASSERT_TRUE(exp.equalsTo(z));

//    for (int e = 0; e < x.lengthOf(); e++) {
//        ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);
//    }
}

////////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, TestBroadcastRaw_1) {

    //if (!Environment::getInstance().isExperimentalBuild())
    //    return;

    NDArray x('c', {2,3,4}, {100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100}, sd::DataType::INT32);
    NDArray y('c', {3},   {10, 20, 30}, sd::DataType::INT64);
    NDArray z('c', {2,3,4}, {100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100}, sd::DataType::INT32);
    NDArray exp('c', {2,3,4}, {10, 11, 12, 13,24, 25, 26, 27,38, 39, 40, 41,22, 23, 24, 25,36, 37, 38, 39,50, 51, 52, 53}, sd::DataType::INT32);
    //real output [10, 11, 12, 13, 4, 5, 6, 7, 28, 29, 30, 31, 22, 23, 24, 25, 16, 17, 18, 19, 40, 41, 42, 43]
    x.linspace(0); x.syncToDevice();

    std::vector<int> dimensions = {1};

    // evaluate xTad data
    shape::TAD xTad;
    xTad.init(x.shapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function
    std::vector<std::pair<void*,size_t>> hostData;
    hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(Nd4jLong));							// 0 -- dimensions
    hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));	// 1 -- xTadShapeInfo
    hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));							// 2 -- xTadOffsets
    std::vector<void*> devicePtrs(hostData.size(), nullptr);

    // create cuda stream and LaunchContext
    hipError_t cudaResult;
    hipStream_t* stream = x.getContext()->getCudaStream();
    LaunchContext* pLc = x.getContext();

    // allocate required amount of global device memory and copy host data to it
    //cudaResult = allocateDeviceMem(*pLc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
    for(size_t i = 0; i < devicePtrs.size(); ++i) {
        cudaResult = hipMalloc(&devicePtrs[i], hostData[i].second); //if(cudaResult != 0) return cudaResult;
        ASSERT_EQ(cudaResult, 0);
        hipMemcpy(devicePtrs[i], hostData[i].first, hostData[i].second, hipMemcpyHostToDevice);
    }

    // call cuda kernel which calculates result
    NativeOpExecutioner::execBroadcast(pLc, sd::broadcast::Add,
                                       nullptr, x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
                                       nullptr, y.shapeInfo(), y.specialBuffer(), y.specialShapeInfo(),
                                       nullptr, z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
                                       (int*)devicePtrs[0], dimensions.size(),
                                       (Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2],
                                       nullptr, nullptr);

    cudaResult = hipStreamSynchronize(*stream); ASSERT_EQ(0, cudaResult);

    // x.printIndexedBuffer(" X");
    // y.printIndexedBuffer("+Y");
    // z.printBuffer("ADD broadcasted output");
    // verify results
   // for (int e = 0; e < z.lengthOf(); e++)
   //     ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

    // free allocated global device memory
    for(int i = 0; i < devicePtrs.size(); ++i)
        hipFree(devicePtrs[i]);

    // delete cuda stream
    //cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

TEST_F(NDArrayCudaBasicsTests, TestBroadcastMultiply) {
    // allocating host-side arrays
    NDArray x('c', { 2, 3 }, { 1, 2, 3, 4, 5, 6}, sd::DataType::DOUBLE);
    NDArray y('c', { 3 }, { 2., 3., 4.}, sd::DataType::DOUBLE);
    //auto z = NDArrayFactory::create<double>('c', { 5 });

    auto exp = NDArrayFactory::create<double>('c', { 2, 3 }, { 2, 6, 12, 8, 15, 24 });

    // making raw buffers
    //Nd4jPointer devBufferPtrX, devBufferPtrZ, devShapePtrX;
    //hipError_t res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrX), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrZ), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devShapePtrX), shape::shapeInfoByteLength(x.shapeInfo()));
    //ASSERT_EQ(0, res);
    //x.applyPairwiseTransform(pairwise::Multiply, &y, &z, nullptr);
    //x.printBuffer("23X = ");
    //y.printBuffer("23Y = ");
    x *= y;

    //
    // hipFree(devBufferPtrX);
    //hipFree(devBufferPtrZ);
    //hipFree(devShapePtrX);

    //for (int e = 0; e < x.lengthOf(); e++) {
    //    ASSERT_NEAR(exp.e<double>(e), x.e<double>(e), 1e-5);
    //}
}


TEST_F(NDArrayCudaBasicsTests, TestBroadcastMultiply_2) {
    // allocating host-side arrays
    NDArray x('c', { 2, 3 }, { 1, 2, 3, 4, 5, 6}, sd::DataType::DOUBLE);
    NDArray y('c', { 3 }, { 2., 3., 4.}, sd::DataType::DOUBLE);
    //auto z = NDArrayFactory::create<double>('c', { 5 });

    auto exp = NDArrayFactory::create<double>('c', { 2, 3 }, { 11,12, 13,14, 15, 16 });
    auto expZ = NDArrayFactory::create<double>('c', { 2, 3 }, { 2, 6, 12, 8, 15, 24 });

    // making raw buffers
    //Nd4jPointer devBufferPtrX, devBufferPtrZ, devShapePtrX;
    //hipError_t res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrX), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrZ), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devShapePtrX), shape::shapeInfoByteLength(x.shapeInfo()));
    //ASSERT_EQ(0, res);
    //x.applyPairwiseTransform(pairwise::Multiply, &y, &z, nullptr);
    //x.printBuffer("23X = ");
    //y.printBuffer("23Y = ");
    //void NDArray::applyTrueBroadcast(sd::BroadcastOpsTuple op, const NDArray* other, NDArray* target, const bool checkTargetShape, ExtraArguments *extraArgs)
    x.applyTrueBroadcast(BroadcastOpsTuple::Multiply(), y, exp);

    //
    // hipFree(devBufferPtrX);
    //hipFree(devBufferPtrZ);
    //hipFree(devShapePtrX);

    //for (int e = 0; e < x.lengthOf(); e++) {
    //    ASSERT_NEAR(exp.e<double>(e), x.e<double>(e), 1e-5);
    //}
    ASSERT_TRUE(exp.equalsTo(expZ));

}


//////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, TestReduceSum_1) {
    // allocating host-side arrays
    auto x = NDArrayFactory::create<double>('c', { 5 }, { 1, 2, 3, 4, 5});
    auto y = NDArrayFactory::create<double>(15);
    auto exp = NDArrayFactory::create<double>(15);

    auto stream = x.getContext()->getCudaStream();//reinterpret_cast<hipStream_t *>(&nativeStream);

    NativeOpExecutioner::execReduceSameScalar(x.getContext(), reduce::Sum, x.buffer(), x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(), nullptr, y.buffer(), y.shapeInfo(), y.specialBuffer(), y.specialShapeInfo());
    auto res = hipStreamSynchronize(*stream);
    ASSERT_EQ(0, res);
    y.syncToHost();

    ASSERT_NEAR(y.e<double>(0), 15, 1e-5);
}

//////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, TestDup1) {

    NDArray array('c', {2,3}, {1,2,3,4,5,6});
    auto arrC = array.dup('c');
    auto arrF = array.dup('f');
    // arrC->printBuffer("arrC");

    // arrF->printBuffer("arrF");
    //arrC->printShapeInfo("C shape");
    //arrF->printShapeInfo("F shape");

    ASSERT_TRUE(array.equalsTo(arrF));
    ASSERT_TRUE(array.equalsTo(arrC));

    ASSERT_TRUE(arrF.equalsTo(arrC));
}

//////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, equalsTo_1) {

    NDArray x('c', {2,5}, {1,2,3,4,5,6,7,8,9,10}, sd::DataType::DOUBLE);
    NDArray y('c', {2,5}, {1,2,3,4,5,6,7,8,9,10}, sd::DataType::DOUBLE);

    ASSERT_TRUE(x.equalsTo(y));

    x.permutei({1,0});
    y.permutei({1,0});

    ASSERT_TRUE(x.equalsTo(y));
}

//////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, equalsTo_2) {

    NDArray x('c', {2,5}, {1,2,3,4,5,6,7,8,10,10}, sd::DataType::DOUBLE);
    NDArray y('c', {2,5}, {1,2,5,4,5,6,7,8,9,10}, sd::DataType::DOUBLE);

    ASSERT_FALSE(x.equalsTo(y));

    x.permutei({1,0});
    y.permutei({1,0});

    ASSERT_FALSE(x.equalsTo(y));
}

//////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, equalsTo_3) {

    NDArray x('c', {2,5}, {1,2,3,4,5,6,7,8,9,10}, sd::DataType::DOUBLE);
    NDArray y('c', {2,5}, {1.f,2.f,3.f,4.f,5.f,6.f,7.f,8.f,9.f,10.f}, sd::DataType::FLOAT32);

    ASSERT_FALSE(x.equalsTo(y));

    x.permutei({1,0});
    y.permutei({1,0});

    ASSERT_FALSE(x.equalsTo(y));
}

////////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, applyReduce3_1) {

    NDArray x('c', {2,3,4}, {-10,-9,-8,-7,-6,-5,-4,-3,-2,-1,0,1,2,3,4,5,6,7,8,9,10,11,12,13}, sd::DataType::INT32);
    NDArray x2('c', {2,3,4}, {-10,-9,-8,-7,-6,-5,-4,-3,-2,-1,0,1,2,3,4,5,6,7,8,9,10,11,12,13}, sd::DataType::INT32);
    NDArray y('c', {2,3,4}, {-2,3,-4,5,-2,3,-4,5,-2,3,-4,5,-2,3,-4,5,-2,3,-4,5,-2,3,-4,5}, sd::DataType::INT32);
    NDArray k('c', {2,3}, {-2,3,-4,5,-2,3}, sd::DataType::INT32);
    NDArray k2('c', {3,2}, {-2,3,-4,5,-2,3}, sd::DataType::INT32);

    NDArray exp1('c', {3}, {4.f, 20.f, 36.f}, sd::DataType::FLOAT32);
    NDArray exp2('c', {2,3}, {-10.f, -2.f, 6.f,14.f, 22.f, 30.f}, sd::DataType::FLOAT32);
    NDArray exp3('c', {4}, {38.f, 41.f, 44.f, 47.f}, sd::DataType::FLOAT32);
    NDArray exp4('c', {4}, {114.f, 117.f, 120.f, 123.f}, sd::DataType::FLOAT32);


    NDArray z = x.applyReduce3(sd::reduce3::Dot, y, {0,2});
    ASSERT_TRUE(z.equalsTo(&exp1));

    z = x.applyReduce3(sd::reduce3::Dot, k, {0,1});
    ASSERT_TRUE(z.equalsTo(&exp3));

    x.permutei({0,2,1});
    y.permutei({0,2,1});

    z = y.applyReduce3(sd::reduce3::Dot, x, {1});
    ASSERT_TRUE(z.equalsTo(&exp2));

    x2.permutei({1,0,2});

    z = x2.applyReduce3(sd::reduce3::Dot, k2, {0,1});
    ASSERT_TRUE(z.equalsTo(&exp4));
}

////////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, applyReduce3_2) {

    NDArray x('c', {2,3,4}, {-10,-9,-8.5,-7,-6,-5,-4,-3,-2,-1,0,1,2,3,4,5,6,7,8,9,10,11,12,13}, sd::DataType::DOUBLE);
    NDArray x2('c', {2,3,4}, {-10,-9,-8,-7,-6,-5,-4,-3,-2,-1,0.5,1,2,3,4,5,6,7,8,9,10,11,12,13}, sd::DataType::DOUBLE);
    NDArray y('c', {2,3,4}, {-2,3,-4,5,-2,3,-4,5,-2,3,-4,5,-2.5,3,-4,5,-2,3,-4,5,-2,3,-4,5}, sd::DataType::DOUBLE);
    NDArray k('c', {2,3}, {-2,3,-4,5.5,-2,3}, sd::DataType::DOUBLE);
    NDArray k2('c', {3,2}, {-2,3,-4,5,-2,3.5}, sd::DataType::DOUBLE);

    NDArray exp1('c', {3}, {5., 20., 36.}, sd::DataType::DOUBLE);
    NDArray exp2('c', {2,3}, {-8., -2., 6., 13., 22., 30.}, sd::DataType::DOUBLE);
    NDArray exp3('c', {4}, {39., 42.5, 47., 49.5}, sd::DataType::DOUBLE);
    NDArray exp4('c', {4}, {119., 122.5, 125., 129.5}, sd::DataType::DOUBLE);

    NDArray z = x.applyReduce3(sd::reduce3::Dot, y, {0,2});
    ASSERT_TRUE(z.equalsTo(&exp1));

    z = x.applyReduce3(sd::reduce3::Dot, k, {0,1});
    ASSERT_TRUE(z.equalsTo(&exp3));

    x.permutei({0,2,1});
    y.permutei({0,2,1});

    z = y.applyReduce3(sd::reduce3::Dot, x, {1});
    ASSERT_TRUE(z.equalsTo(&exp2));

    x2.permutei({1,0,2});

    z = x2.applyReduce3(sd::reduce3::Dot, k2, {0,1});
    ASSERT_TRUE(z.equalsTo(&exp4));
}

////////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, applyReduce3_3) {

    NDArray x1('c', {2,2,2}, {1,2,3,4,5,6,7,8}, sd::DataType::INT32);
    NDArray x2('c', {2,2,2}, {-1,-2,-3,-4,-5,-6,-7,-8}, sd::DataType::INT32);
    NDArray x3('c', {3,2}, {1.5,1.5,1.5,1.5,1.5,1.5}, sd::DataType::DOUBLE);
    NDArray x4('c', {3,2}, {1,2,3,4,5,6}, sd::DataType::DOUBLE);

    NDArray exp1('c', {}, std::vector<double>{-204}, sd::DataType::FLOAT32);
    NDArray exp2('c', {}, std::vector<double>{31.5}, sd::DataType::DOUBLE);


    auto z = x1.applyReduce3(reduce3::Dot, x2);
    ASSERT_TRUE(z.equalsTo(&exp1));

    z = x3.applyReduce3(reduce3::Dot, x4);
    ASSERT_TRUE(z.equalsTo(&exp2));

    x1.permutei({2,1,0});
    x2.permutei({2,1,0});
    x3.permutei({1,0});
    x4.permutei({1,0});

    z = x1.applyReduce3(reduce3::Dot, x2);
    ASSERT_TRUE(z.equalsTo(&exp1));

    z = x3.applyReduce3(reduce3::Dot, x4);
    ASSERT_TRUE(z.equalsTo(&exp2));
}

////////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, applyAllReduce3_1) {

    NDArray x1('c', {2,3,2}, {1,2,3,4,5,6,7,8,-1,-2,-3,-4,}, sd::DataType::INT32);
    NDArray x2('c', {2,2,2}, {-1,-2,-3,-4,-5,-6,-7,-8}, sd::DataType::INT32);
    NDArray x3('c', {3,2}, {1.5,1.5,1.5,1.5,1.5,1.5}, sd::DataType::DOUBLE);
    NDArray x4('c', {3,2}, {1,2,3,4,5,6}, sd::DataType::DOUBLE);

    NDArray exp1('c', {3,2}, {-88.f, -124.f, 6.f, -2.f, 22.f, 14.f}, sd::DataType::FLOAT32);
    NDArray exp2('c', {6,4}, {-36.f, -44.f, -52.f, -60.f,-42.f, -52.f, -62.f, -72.f, 2.f, 0.f, -2.f,
                              -4.f, 6.f, 4.f, 2.f, 0.f, 10.f, 8.f, 6.f, 4.f, 14.f, 12.f, 10.f, 8.f},
            sd::DataType::FLOAT32);
    NDArray exp3('c', {1,1}, std::vector<double>{31.5}, sd::DataType::DOUBLE);
    NDArray exp4('c', {3,3}, {4.5, 10.5, 16.5,4.5, 10.5, 16.5,4.5, 10.5, 16.5}, sd::DataType::DOUBLE);

    auto z = x1.applyAllReduce3(reduce3::Dot, x2, {0,2});
    ASSERT_TRUE(z.equalsTo(&exp1));

    z = x1.applyAllReduce3(reduce3::Dot, x2, {0});
    ASSERT_TRUE(z.equalsTo(&exp2));

    z = x3.applyAllReduce3(reduce3::Dot, x4, {0,1});
    ASSERT_TRUE(z.equalsTo(&exp3));

    z = x3.applyAllReduce3(reduce3::Dot, x4, {1});
    ASSERT_TRUE(z.equalsTo(&exp4));

    x1.permutei({2,1,0});
    x2.permutei({2,1,0});
    x3.permutei({1,0});
    x4.permutei({1,0});

    z = x1.applyAllReduce3(reduce3::Dot, x2, {0,2});
    ASSERT_TRUE(z.equalsTo(&exp1));

    z = x3.applyAllReduce3(reduce3::Dot, x4, {0});
    ASSERT_TRUE(z.equalsTo(&exp4));
}

//////////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, applyIndexReduce_test1) {

    NDArray x('c', {2,3}, {0, 10, 1, 2, 2.5,-4}, sd::DataType::DOUBLE);

    NDArray scalar('c', {}, std::vector<double>{100}, sd::DataType::INT64);
    NDArray vec1('c', {2}, {100,100}, sd::DataType::INT64);
    NDArray vec2('c', {3}, {100,100,100}, sd::DataType::INT64);

    NDArray exp1('c', {}, std::vector<double>{1}, sd::DataType::INT64);
    NDArray exp2('c', {2}, {1,1}, sd::DataType::INT64);
    NDArray exp3('c', {3}, {1,0,0}, sd::DataType::INT64);

    NDArray exp4('c', {}, std::vector<double>{2}, sd::DataType::INT64);
    NDArray exp5('c', {2}, {1,1}, sd::DataType::INT64);
    NDArray exp6('c', {3}, {1,0,0}, sd::DataType::INT64);

    x.applyIndexReduce(sd::indexreduce::IndexMax, scalar, {0,1});
    ASSERT_TRUE(scalar.equalsTo(&exp1));

    x.applyIndexReduce(sd::indexreduce::IndexMax, vec1, {1});
    ASSERT_TRUE(vec1.equalsTo(&exp2));

    x.applyIndexReduce(sd::indexreduce::IndexMax, vec2, {0});
    ASSERT_TRUE(vec2.equalsTo(&exp3));

    x.permutei({1,0});

    x.applyIndexReduce(sd::indexreduce::IndexMax, scalar, {0,1});
    ASSERT_TRUE(scalar.equalsTo(&exp4));

    x.applyIndexReduce(sd::indexreduce::IndexMax, vec1, {0});
    ASSERT_TRUE(vec1.equalsTo(&exp5));

    x.applyIndexReduce(sd::indexreduce::IndexMax, vec2, {1});
    ASSERT_TRUE(vec2.equalsTo(&exp6));
}


//////////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, applyIndexReduce_test2) {

    NDArray x('c', {2,3}, {0, 10, 1, 2, 2.5,-4}, sd::DataType::DOUBLE);

    NDArray exp1('c', {}, std::vector<double>{1}, sd::DataType::INT64);
    NDArray exp2('c', {2}, {1,1}, sd::DataType::INT64);
    NDArray exp3('c', {3}, {1,0,0}, sd::DataType::INT64);

    NDArray exp4('c', {}, std::vector<double>{2}, sd::DataType::INT64);
    NDArray exp5('c', {2}, {1,1}, sd::DataType::INT64);
    NDArray exp6('c', {3}, {1,0,0}, sd::DataType::INT64);

    auto z = x.applyIndexReduce(sd::indexreduce::IndexMax, {0,1});
    ASSERT_TRUE(z.equalsTo(&exp1));

    z = x.applyIndexReduce(sd::indexreduce::IndexMax, {1});
    ASSERT_TRUE(z.equalsTo(&exp2));

    z = x.applyIndexReduce(sd::indexreduce::IndexMax, {0});
    ASSERT_TRUE(z.equalsTo(&exp3));

    x.permutei({1,0});

    z = x.applyIndexReduce(sd::indexreduce::IndexMax, {0,1});
    ASSERT_TRUE(z.equalsTo(&exp4));

    z = x.applyIndexReduce(sd::indexreduce::IndexMax, {0});
    ASSERT_TRUE(z.equalsTo(&exp5));

    z = x.applyIndexReduce(sd::indexreduce::IndexMax, {1});
    ASSERT_TRUE(z.equalsTo(&exp6));
}

////////////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, reduceAlongDimension_float_test1) {

    NDArray x('c', {2,3,2}, {1,2,3,4,5,6,7,8,-1,-2,-3,-4,}, sd::DataType::INT32);

    NDArray z1('c', {}, std::vector<double>{100}, sd::DataType::DOUBLE);
    NDArray z2('c', {2,2}, {100,100,100,100}, sd::DataType::FLOAT32);
    NDArray z3('c', {3}, {100,100,100}, sd::DataType::DOUBLE);
    NDArray z4('c', {3,2}, {100,100,100,100,100,100}, sd::DataType::FLOAT32);
    NDArray z5('c', {2}, {100,100}, sd::DataType::FLOAT32);

    NDArray exp1('c', {}, std::vector<double>{2.166667}, sd::DataType::DOUBLE);
    NDArray exp2('c', {2,2}, {3.f,4.f,1.f,0.666667f}, sd::DataType::FLOAT32);
    NDArray exp3('c', {3}, {4.5,1,1}, sd::DataType::DOUBLE);
    NDArray exp4('c', {3,2}, {4,5,1,1,1,1}, sd::DataType::FLOAT32);
    NDArray exp5('c', {2}, {3.5f,0.833333f}, sd::DataType::FLOAT32);

    x.reduceAlongDimension(sd::reduce::Mean, z1, {0,1,2});
    ASSERT_TRUE(z1.equalsTo(&exp1));

    x.reduceAlongDimension(sd::reduce::Mean, z2, {1});
    ASSERT_TRUE(z2.equalsTo(&exp2));

    x.reduceAlongDimension(sd::reduce::Mean, z3, {0,2});
    ASSERT_TRUE(z3.equalsTo(&exp3));

    x.permutei({1,0,2});    // 3x2x2

    x.reduceAlongDimension(sd::reduce::Mean, z1, {0,1,2});
    ASSERT_TRUE(z1.equalsTo(&exp1));

    x.reduceAlongDimension(sd::reduce::Mean, z4, {1});
    ASSERT_TRUE(z4.equalsTo(&exp4));

    x.reduceAlongDimension(sd::reduce::Mean, z5, {0,2});
    ASSERT_TRUE(z5.equalsTo(&exp5));
}

////////////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, reduceAlongDimension_float_test2) {

    NDArray x('c', {2,3,2}, {1,2,3,4,5,6,7,8,-1,-2,-3,-4,}, sd::DataType::DOUBLE);

    NDArray exp1('c', {}, std::vector<double>{2.166667}, sd::DataType::DOUBLE);
    NDArray exp2('c', {2,2}, {3,4,1,0.666667}, sd::DataType::DOUBLE);
    NDArray exp3('c', {3}, {4.5,1,1}, sd::DataType::DOUBLE);
    NDArray exp4('c', {3,2}, {4,5,1,1,1,1}, sd::DataType::DOUBLE);
    NDArray exp5('c', {2}, {3.5,0.833333}, sd::DataType::DOUBLE);

    NDArray z1 = x.reduceAlongDimension(sd::reduce::Mean, {0,1,2});
    ASSERT_TRUE(z1.equalsTo(&exp1));

    NDArray z2 = x.reduceAlongDimension(sd::reduce::Mean, {1});
    ASSERT_TRUE(z2.equalsTo(&exp2));

    NDArray z3 = x.reduceAlongDimension(sd::reduce::Mean, {0,2});
    ASSERT_TRUE(z3.equalsTo(&exp3));

    x.permutei({1,0,2});    // 3x2x2

    NDArray z4 = x.reduceAlongDimension(sd::reduce::Mean, {0,1,2});
    ASSERT_TRUE(z4.equalsTo(&exp1));

    NDArray z5 = x.reduceAlongDimension(sd::reduce::Mean, {1});
    ASSERT_TRUE(z5.equalsTo(&exp4));

    NDArray z6 = x.reduceAlongDimension(sd::reduce::Mean, {0,2});
    ASSERT_TRUE(z6.equalsTo(&exp5));
}

//////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, EqualityTest1) {
    auto arrayA = NDArrayFactory::create_<float>('f', {3, 5});
    auto arrayB = NDArrayFactory::create_<float>('f', {3, 5});
    auto arrayC = NDArrayFactory::create_<float>('f', {3, 5});

    auto arrayD = NDArrayFactory::create_<float>('f', {2, 4});
    auto arrayE = NDArrayFactory::create_<float>('f', {1, 15});

    for (int i = 0; i < arrayA->rows(); i++) {
        for (int k = 0; k < arrayA->columns(); k++) {
            arrayA->p(i, k, (float) i);
        }
    }

    for (int i = 0; i < arrayB->rows(); i++) {
        for (int k = 0; k < arrayB->columns(); k++) {
            arrayB->p(i, k, (float) i);
        }
    }

    for (int i = 0; i < arrayC->rows(); i++) {
        for (int k = 0; k < arrayC->columns(); k++) {
            arrayC->p(i, k, (float) i+1);
        }
    }

    ASSERT_TRUE(arrayA->equalsTo(arrayB, 1e-5));

    ASSERT_FALSE(arrayC->equalsTo(arrayB, 1e-5));

    ASSERT_FALSE(arrayD->equalsTo(arrayB, 1e-5));

    ASSERT_FALSE(arrayE->equalsTo(arrayB, 1e-5));

    delete arrayA;
    delete arrayB;
    delete arrayC;
    delete arrayD;
    delete arrayE;
}

////////////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, reduceAlongDimension_same_test1) {

    NDArray x('c', {2,3,2}, {1.5f,2.f,3.f,4.f,5.f,6.f,7.5f,8.f,-1.f,-2.f,-3.5f,-4.f}, sd::DataType::FLOAT32);

    NDArray z1('c', {}, std::vector<double>{100}, sd::DataType::FLOAT32);
    NDArray z2('c', {2,2}, {100,100,100,100}, sd::DataType::FLOAT32);
    NDArray z3('c', {3}, {100,100,100}, sd::DataType::FLOAT32);
    NDArray z4('c', {3,2}, {100,100,100,100,100,100}, sd::DataType::FLOAT32);
    NDArray z5('c', {2}, {100,100}, sd::DataType::FLOAT32);

    NDArray exp1('c', {}, std::vector<double>{26.5f}, sd::DataType::FLOAT32);
    NDArray exp2('c', {2,2}, {9.5f,12.f,3.f,2.f}, sd::DataType::FLOAT32);
    NDArray exp3('c', {3}, {19.f,4.f,3.5f}, sd::DataType::FLOAT32);
    NDArray exp4('c', {3,2}, {9.f,10.f,2.f,2.f,1.5f,2.f}, sd::DataType::FLOAT32);
    NDArray exp5('c', {2}, {21.5f,5.f}, sd::DataType::FLOAT32);

    x.reduceAlongDimension(sd::reduce::Sum, z1, {0,1,2});
    ASSERT_TRUE(z1.equalsTo(&exp1));

    x.reduceAlongDimension(sd::reduce::Sum, z2, {1});
    ASSERT_TRUE(z2.equalsTo(&exp2));

    x.reduceAlongDimension(sd::reduce::Sum, z3, {0,2});
    ASSERT_TRUE(z3.equalsTo(&exp3));

    x.permutei({1,0,2});    // 3x2x2

    x.reduceAlongDimension(sd::reduce::Sum, z1, {0,1,2});
    ASSERT_TRUE(z1.equalsTo(&exp1));

    x.reduceAlongDimension(sd::reduce::Sum, z4, {1});
    ASSERT_TRUE(z4.equalsTo(&exp4));

    x.reduceAlongDimension(sd::reduce::Sum, z5, {0,2});
    ASSERT_TRUE(z5.equalsTo(&exp5));
}

////////////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, reduceAlongDimension_same_test2) {

    NDArray x('c', {2,3,2}, {1.5,2,3,4,5,6,7.5,8,-1,-2,-3.5,-4,}, sd::DataType::INT64);

    NDArray exp1('c', {}, std::vector<double>{26}, sd::DataType::INT64);
    NDArray exp2('c', {2,2}, {9,12,3,2}, sd::DataType::INT64);
    NDArray exp3('c', {3}, {18,4,4}, sd::DataType::INT64);
    NDArray exp4('c', {3,2}, {8,10,2,2,2,2}, sd::DataType::INT64);
    NDArray exp5('c', {2}, {21,5}, sd::DataType::INT64);

    NDArray z1 = x.reduceAlongDimension(sd::reduce::Sum, {0,1,2});
    ASSERT_TRUE(z1.equalsTo(&exp1));

    NDArray z2 = x.reduceAlongDimension(sd::reduce::Sum, {1});
    ASSERT_TRUE(z2.equalsTo(&exp2));

    NDArray z3 = x.reduceAlongDimension(sd::reduce::Sum, {0,2});
    ASSERT_TRUE(z3.equalsTo(&exp3));

    x.permutei({1,0,2});    // 3x2x2

    NDArray z4 = x.reduceAlongDimension(sd::reduce::Sum, {0,1,2});
    ASSERT_TRUE(z4.equalsTo(&exp1));

    NDArray z5 = x.reduceAlongDimension(sd::reduce::Sum, {1});
    ASSERT_TRUE(z5.equalsTo(&exp4));

    NDArray z6 = x.reduceAlongDimension(sd::reduce::Sum, {0,2});
    ASSERT_TRUE(z6.equalsTo(&exp5));
}

////////////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, reduceAlongDimension_bool_test1) {

    NDArray x('c', {2,3,2}, {0.5,2,3,-4,5,6,-7.5,8,-1,-0.5,-3.5,4}, sd::DataType::DOUBLE);

    NDArray z1('c', {}, std::vector<double>{true}, sd::DataType::BOOL);
    NDArray z2('c', {2,2}, {true,true,true,true}, sd::DataType::BOOL);
    NDArray z3('c', {3}, {true,true,true}, sd::DataType::BOOL);
    NDArray z4('c', {3,2}, {true,true,true,true,true,true}, sd::DataType::BOOL);
    NDArray z5('c', {2}, {true,true}, sd::DataType::BOOL);

    NDArray exp1('c', {}, std::vector<double>{true}, sd::DataType::BOOL);
    NDArray exp2('c', {2,2}, {true,true,false,true}, sd::DataType::BOOL);
    NDArray exp3('c', {3}, {true,true,true}, sd::DataType::BOOL);
    NDArray exp4('c', {3,2}, {true,true,true,false,true,true}, sd::DataType::BOOL);
    NDArray exp5('c', {2}, {true,true}, sd::DataType::BOOL);

    x.reduceAlongDimension(sd::reduce::IsPositive, z1, {0,1,2});
    ASSERT_TRUE(z1.equalsTo(&exp1));

    x.reduceAlongDimension(sd::reduce::IsPositive, z2, {1});
    ASSERT_TRUE(z2.equalsTo(&exp2));

    x.reduceAlongDimension(sd::reduce::IsPositive, z3, {0,2});
    ASSERT_TRUE(z3.equalsTo(&exp3));

    x.permutei({1,0,2});    // 3x2x2

    x.reduceAlongDimension(sd::reduce::IsPositive, z1, {0,1,2});
    ASSERT_TRUE(z1.equalsTo(&exp1));

    x.reduceAlongDimension(sd::reduce::IsPositive, z4, {1});
    ASSERT_TRUE(z4.equalsTo(&exp4));

    x.reduceAlongDimension(sd::reduce::IsPositive, z5, {0,2});
    ASSERT_TRUE(z5.equalsTo(&exp5));
}

////////////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, reduceAlongDimension_bool_test2) {

    NDArray x('c', {2,3,2}, {0.5,2,3,-4,5,6,-7.5,8,-1,-0.5,-3.5,4}, sd::DataType::INT32);

    NDArray exp1('c', {}, std::vector<double>{1}, sd::DataType::BOOL);
    NDArray exp2('c', {2,2}, {1,1,0,1}, sd::DataType::BOOL);
    NDArray exp3('c', {3}, {1,1,1}, sd::DataType::BOOL);
    NDArray exp4('c', {3,2}, {0,1,1,0,1,1}, sd::DataType::BOOL);
    NDArray exp5('c', {2}, {1,1}, sd::DataType::BOOL);

    NDArray z1 = x.reduceAlongDimension(sd::reduce::IsPositive, {0,1,2});
    ASSERT_TRUE(z1.equalsTo(&exp1));

    NDArray z2 = x.reduceAlongDimension(sd::reduce::IsPositive, {1});
    ASSERT_TRUE(z2.equalsTo(&exp2));

    NDArray z3 = x.reduceAlongDimension(sd::reduce::IsPositive, {0,2});
    ASSERT_TRUE(z3.equalsTo(&exp3));

    x.permutei({1,0,2});    // 3x2x2

    NDArray z4 = x.reduceAlongDimension(sd::reduce::IsPositive, {0,1,2});
    ASSERT_TRUE(z4.equalsTo(&exp1));

    NDArray z5 = x.reduceAlongDimension(sd::reduce::IsPositive, {1});
    ASSERT_TRUE(z5.equalsTo(&exp4));

    NDArray z6 = x.reduceAlongDimension(sd::reduce::IsPositive, {0,2});
    ASSERT_TRUE(z6.equalsTo(&exp5));
}

////////////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, reduceAlongDimension_long_test1) {

    NDArray x('c', {2,3,2}, {0.5f,2.f,3.f,-0.f,5.f,6.f,-7.5f,0.f,-1.f,-0.5f,-3.5f,4.f}, sd::DataType::FLOAT32);

    NDArray z1('c', {}, std::vector<double>{100}, sd::DataType::INT64);
    NDArray z2('c', {2,2}, {100,100,100,100}, sd::DataType::INT64);
    NDArray z3('c', {3}, {100,100,100}, sd::DataType::INT64);
    NDArray z4('c', {3,2}, {100,100,100,100,100,100}, sd::DataType::INT64);
    NDArray z5('c', {2}, {100,100}, sd::DataType::INT64);

    NDArray exp1('c', {}, std::vector<double>{2}, sd::DataType::INT64);
    NDArray exp2('c', {2,2}, {0,1,0,1}, sd::DataType::INT64);
    NDArray exp3('c', {3}, {1,1,0}, sd::DataType::INT64);
    NDArray exp4('c', {3,2}, {0,1,0,1,0,0}, sd::DataType::INT64);
    NDArray exp5('c', {2}, {1,1}, sd::DataType::INT64);

    x.reduceAlongDimension(sd::reduce::CountZero, z1, {0,1,2});
    ASSERT_TRUE(z1.equalsTo(&exp1));

    x.reduceAlongDimension(sd::reduce::CountZero, z2, {1});
    ASSERT_TRUE(z2.equalsTo(&exp2));

    x.reduceAlongDimension(sd::reduce::CountZero, z3, {0,2});
    ASSERT_TRUE(z3.equalsTo(&exp3));

    x.permutei({1,0,2});    // 3x2x2

    x.reduceAlongDimension(sd::reduce::CountZero, z1, {0,1,2});
    ASSERT_TRUE(z1.equalsTo(&exp1));

    x.reduceAlongDimension(sd::reduce::CountZero, z4, {1});
    ASSERT_TRUE(z4.equalsTo(&exp4));

    x.reduceAlongDimension(sd::reduce::CountZero, z5, {0,2});
    ASSERT_TRUE(z5.equalsTo(&exp5));
}

////////////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, reduceAlongDimension_long_test2) {

    NDArray x('c', {2,3,2}, {0.5,2,3,-0,5,6,-7.5,0,-1,-0.5,-3.5,4}, sd::DataType::INT32);

    NDArray exp1('c', {}, std::vector<double>{4}, sd::DataType::INT64);
    NDArray exp2('c', {2,2}, {1,1,0,2}, sd::DataType::INT64);
    NDArray exp3('c', {3}, {2,2,0}, sd::DataType::INT64);
    NDArray exp4('c', {3,2}, {1,1,0,2,0,0}, sd::DataType::INT64);
    NDArray exp5('c', {2}, {2,2}, sd::DataType::INT64);

    NDArray z1 = x.reduceAlongDimension(sd::reduce::CountZero, {0,1,2});
    ASSERT_TRUE(z1.equalsTo(&exp1));

    NDArray z2 = x.reduceAlongDimension(sd::reduce::CountZero, {1});
    ASSERT_TRUE(z2.equalsTo(&exp2));

    NDArray z3 = x.reduceAlongDimension(sd::reduce::CountZero, {0,2});
    ASSERT_TRUE(z3.equalsTo(&exp3));

    x.permutei({1,0,2});    // 3x2x2

    NDArray z4 = x.reduceAlongDimension(sd::reduce::CountZero, {0,1,2});
    ASSERT_TRUE(z4.equalsTo(&exp1));

    NDArray z5 = x.reduceAlongDimension(sd::reduce::CountZero, {1});
    ASSERT_TRUE(z5.equalsTo(&exp4));

    NDArray z6 = x.reduceAlongDimension(sd::reduce::CountZero, {0,2});
    ASSERT_TRUE(z6.equalsTo(&exp5));
}

TEST_F(NDArrayCudaBasicsTests, BroadcastOpsTest1) {

    auto x = NDArrayFactory::create<float>('c', {5, 5});
    auto z = NDArrayFactory::create<float>('c', {5, 5});
    auto row = NDArrayFactory::linspace(1.0f, 5.0f, 5);
    NDArray expRow('c', {1, 5,}, {1,2,3,4,5}, sd::DataType::FLOAT32);
    NDArray exp('c', {5,5}, {1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5}, sd::DataType::FLOAT32);

    ASSERT_TRUE(row->equalsTo(&expRow));

    x.applyBroadcast(broadcast::Add, {1}, *row, z);
    x += *row;

    ASSERT_TRUE(x.equalsTo(z));
    //ASSERT_TRUE(z.equalsTo(&exp));

    delete row;
}

TEST_F(NDArrayCudaBasicsTests, BroadcastOpsTest2) {

    auto x = NDArrayFactory::create<float>('c', {5, 5});
    //auto z = NDArrayFactory::create<float>('c', {5, 5});
    auto row = NDArrayFactory::linspace(1.0f, 5.0f, 5);
    NDArray expRow('c', {1, 5,}, {1,2,3,4,5}, sd::DataType::FLOAT32);
    NDArray exp('c', {5,5}, {1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5}, sd::DataType::FLOAT32);

    ASSERT_TRUE(row->equalsTo(&expRow));
    x.applyBroadcast(broadcast::Add, {1}, *row, x);
    ASSERT_TRUE(x.equalsTo(&exp));
}

//////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, TestBroadcast_1) {

    NDArray exp('c', {2, 3, 2, 2}, {1., 1., 1., 1., 2., 2., 2., 2., 3., 3., 3., 3., 1., 1., 1., 1., 2., 2., 2., 2., 3., 3., 3., 3.}, sd::DataType::DOUBLE);

    auto input = NDArrayFactory::create<double>('c',{ 2, 3, 2, 2});
    auto bias = NDArrayFactory::create<double>('c', {1, 3});

    bias.linspace(1);
    input.applyBroadcast(broadcast::Add, {1}, bias, input);
    ASSERT_TRUE(exp.equalsTo(&input));
}

TEST_F(NDArrayCudaBasicsTests, TestFloat16_1) {
    auto x = NDArrayFactory::create<float>({1,2,3,4,5,7,8,9});
    auto y = NDArrayFactory::create<float>({1,2,3,4,5,7,8,9});
    ASSERT_TRUE(x.equalsTo(&y));
}

TEST_F(NDArrayCudaBasicsTests, TestFloat16_2) {
    auto x = NDArrayFactory::create<float16>('c', {9}, {1,2,3,4,5,6,7,8,9});
    auto y = NDArrayFactory::create<float16>('c', {9}, {1,2,3,4,5,6,7,8,9});
    ASSERT_TRUE(x.equalsTo(y));
    //for (int e = 0; e < x.lengthOf(); e++)
    //    ASSERT_NEAR(x.e<float16>(e), y.e<float16>(e), 1.e-5f);
}

TEST_F(NDArrayCudaBasicsTests, TestFloat16_3) {
    auto x = NDArrayFactory::create<bfloat16>({1,2,3,4,5,7,8,9});
    auto y = NDArrayFactory::create<bfloat16>({1,2,3,4,5,7,8,9});
    ASSERT_TRUE(x.equalsTo(&y));
}

TEST_F(NDArrayCudaBasicsTests, TestFloat_4) {
    auto x = NDArrayFactory::create<float>({1,2,3,4,5,7,8,9});
    auto y = NDArrayFactory::create<float>({2,4,5,5,6,7,8,9});
    ASSERT_FALSE(x.equalsTo(&y));
}

TEST_F(NDArrayCudaBasicsTests, TestFloat_5) {
    auto x = NDArrayFactory::create<float>('c', {3,3}, {1,2,3,4,5,6,7,8,9});
    auto y = NDArrayFactory::create<float>('c', {3,3}, {2,4,5,5,6,7,8,9, 10});
    ASSERT_FALSE(x.equalsTo(&y));
}

TEST_F(NDArrayCudaBasicsTests, TestFloat_6) {
    auto x = NDArrayFactory::create<float>('f', {3,3}, {1,2,3,4,5,6,7,8,9});
    auto y = NDArrayFactory::create<float>('f', {3,3}, {2,4,5,5,6,7,8,9,10});
    ASSERT_FALSE(x.equalsTo(&y));
}

//////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, Operator_Plus_Test_05)
{
    auto x = NDArrayFactory::create<float>('c', {8, 8, 8});
    auto y = NDArrayFactory::create<float>('c', {1, 8, 8});
    auto expected = NDArrayFactory::create<float>('c', {8, 8, 8});
    NDArray res2  = NDArrayFactory::create<float>(expected.ordering(), expected.getShapeAsVector());
    x = 1.;
    y = 2.;
    expected = 3.;
    res2 = 0.f;

    x.applyTrueBroadcast(BroadcastOpsTuple::Add(), y, res2);// *= y;

    ASSERT_TRUE(expected.isSameShape(&res2));
    ASSERT_TRUE(expected.equalsTo(&res2));
}

//////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, Operator_Plus_Test_5)
{
    auto x = NDArrayFactory::create<float>('c', {8, 8, 8});
    auto y = NDArrayFactory::create<float>('c', {8, 1, 8});
    auto expected = NDArrayFactory::create<float>('c', {8, 8, 8});
    NDArray res2(expected);
    x = 1.;
    y = 2.;
    expected = 3.;
    //x.printBuffer("X=");
    //y.printBuffer("Y=");
    //expected.printBuffer("EXPECTED");
    auto result = x + y;
    //result.printBuffer("1 + 2 =");
    //res2.assign(x + y);

    //x.applyTrueBroadcast(BroadcastOpsTuple::Add(), &y, &res2);
    //res2.printBuffer("Z=");
    //x.applyTrueBroadcast(BroadcastOpsTuple::Add(), &y, &res2);// *= y;
//    x += y;
    //x.printBuffer("OutputX");
    //res2.syncToHost();
    //res2.printBuffer("OUputZ");
    //x.printIndexedBuffer("OUtputX");
    ASSERT_TRUE(expected.isSameShape(&result));
    ASSERT_TRUE(expected.equalsTo(&result));
}

//////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, Operator_Plus_Test_51)
{
    auto x = NDArrayFactory::create<float>('c', {8, 8, 8});
    auto y = NDArrayFactory::create<float>('c', {8, 8});
    auto expected = NDArrayFactory::create<float>('c', {8, 8, 8});
    NDArray res2(expected);
    x = 1.;
    y = 2.;
    expected = 3.;
    //x.printBuffer("X=");
    //y.printBuffer("Y=");
    //expected.printBuffer("EXPECTED");
    auto result = x + y;
    //result.printBuffer("1 + 2 =");
    //res2.assign(x + y);

    //x.applyTrueBroadcast(BroadcastOpsTuple::Add(), &y, &res2);
    //res2.printBuffer("Z=");
    //x.applyTrueBroadcast(BroadcastOpsTuple::Add(), &y, &res2);// *= y;
//    x += y;
    //x.printBuffer("OutputX");
    //res2.syncToHost();
    //res2.printBuffer("OUputZ");
    //x.printIndexedBuffer("OUtputX");
    ASSERT_TRUE(expected.isSameShape(&result));
    ASSERT_TRUE(expected.equalsTo(&result));
}

TEST_F(NDArrayCudaBasicsTests, Tile_Test_2_1)
{
    auto x = NDArrayFactory::create<float>('c', {2, 1, 2});
    x = 10.;
    auto y = x.tile({1,2,1});
    auto exp = NDArrayFactory::create<float>('c', {2, 2, 2});
    exp = 10.;

    // y.printShapeInfo("Output SHAPE");
    // y.printBuffer("Output TILE");
    // exp.printBuffer("Expect TILE");
    ASSERT_TRUE(exp.equalsTo(y));
}

TEST_F(NDArrayCudaBasicsTests, Tile_Test_2_2)
{
    auto x = NDArrayFactory::create<float>('f', {2, 1, 2});
    x = 10.;
    auto y = x.tile({1,2,1});
    auto exp = NDArrayFactory::create<float>('f', {2, 2, 2});
    exp = 10.;
    ASSERT_TRUE(exp.equalsTo(y));
}

TEST_F(NDArrayCudaBasicsTests, Tile_Test_2_3)
{
    auto x = NDArrayFactory::create<float>('f', {2, 1, 2});
    x = 10.;
    x.p(1,0,1, 20);
    x.syncToDevice();
    auto y = x.tile({1,2,1});
    auto exp = NDArrayFactory::create<float>('f', {2, 2, 2});
    exp = 10.;
    exp.p(1,0,1, 20.);
    exp.p(1, 1, 1, 20.);
    exp.syncToDevice();
    ASSERT_TRUE(exp.equalsTo(y));
}

//////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, Operator_Plus_Test_2)
{
    double expBuff[] = {2., 3, 3., 4., 4., 5, 5., 6., 6., 7, 7., 8.};
    NDArray a('c', {4,4}, {1,2,3,4,5,6,7,8,9,2,3,2,1,0,4,7}, sd::DataType::FLOAT32);
    auto x = NDArrayFactory::create<double>('c', {3, 2, 1});
    auto y = NDArrayFactory::create<double>('c',    {1, 2});
    auto expected = NDArrayFactory::create<double>(expBuff, 'c', {3, 2, 2});

    x.linspace(1);
    y.linspace(1);
    auto result = x + y;

    ASSERT_TRUE(expected.isSameShape(&result));
    ASSERT_TRUE(expected.equalsTo(&result));
}

//////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, assign_2)
{
    NDArray x('c', {4}, {1.5f,2.5f,3.5f,4.5f}, sd::DataType::FLOAT32);
    NDArray y('c', {4}, sd::DataType::INT32);
    NDArray expected('c', {4}, {1,2,3,4}, sd::DataType::INT32);

    y.assign(x);
    // y.printBuffer("ASSIGN VECTOR");

    ASSERT_TRUE(expected.equalsTo(&y));
}

//////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, subarray_1)
{
    NDArray x('c', {2,3,4}, {1,2,3,4,5,6,7,8,9,10,11,12,13,14,15,16,17,18,19,20,21,22,23,24}, sd::DataType::FLOAT32);
    NDArray y('f', {2,3,4}, {1,2,3,4,5,6,7,8,9,10,11,12,13,14,15,16,17,18,19,20,21,22,23,24}, sd::DataType::FLOAT32);

    Nd4jLong shapeExpX0[] = {1, 2, 12, 8192, 1, 99};
    float    buffExpX0[]  = {1.f, 13.f};
    Nd4jLong shapeExpX1[] = {1, 2, 12, 8192, 1, 99};
    float    buffExpX1[]  = {2.f, 14.f};
    Nd4jLong shapeExpX2[] = {3, 2, 1, 1, 12, 4, 1, 8192, 1, 99};
    float    buffExpX2[]  = {1.f, 13.f};
    Nd4jLong shapeExpX3[] = {2, 2, 4, 12, 1, 8192, 1, 99};
    float    buffExpX3[]  = {9.f, 10.f, 11.f, 12.f, 21.f, 22.f, 23.f, 24.f};
    Nd4jLong shapeExpX4[] = {3, 2, 1, 4, 12, 4, 1, 8192, 1, 99};
    float    buffExpX4[]  = {9.f, 10.f, 11.f, 12.f, 21.f, 22.f, 23.f, 24.f};
    Nd4jLong shapeExpX5[] = {2, 2, 3, 12, 4, 8192, 1, 99};
    float    buffExpX5[]  = {4.f, 8.f, 12.f, 16.f, 20.f, 24.f};

    Nd4jLong shapeExpY0[] = {1, 2, 1, 8192, 1, 99};
    float    buffExpY0[]  = {1.f, 2.f};
    Nd4jLong shapeExpY1[] = {1, 2, 1, 8192, 1, 99};
    float    buffExpY1[]  = {7.f, 8.f};
    Nd4jLong shapeExpY2[] = {3, 2, 1, 1, 1, 2, 6, 8192, 1, 102};
    float    buffExpY2[]  = {1.f, 2.f};
    Nd4jLong shapeExpY3[] = {2, 2, 4, 1, 6, 8192, 1, 99};
    float    buffExpY3[]  = {5.f, 11.f, 17.f, 23.f, 6.f, 12.f, 18.f, 24.f};
    Nd4jLong shapeExpY4[] = {3, 2, 1, 4, 1, 2, 6, 8192, 1, 102};
    float    buffExpY4[]  = {5.f, 11.f, 17.f, 23.f, 6.f, 12.f, 18.f, 24.f};
    Nd4jLong shapeExpY5[] = {2, 2, 3, 1, 2, 8192, 1, 99};
    float    buffExpY5[]  = {19.f, 21.f, 23.f, 20.f, 22.f, 24.f};


    NDArray x0 = x(0, {1,2});
    NDArray xExp(buffExpX0, shapeExpX0);

    ASSERT_TRUE(xExp.isSameShape(x0));
    ASSERT_TRUE(xExp.equalsTo(x0));
//    for(int i = 0; i < shape::shapeInfoLength(x0.rankOf()); ++i)
//        ASSERT_TRUE(x0.shapeInfo()[i] == shapeExpX0[i]);
//    for(int i = 0; i < x0.lengthOf(); ++i)
//        ASSERT_TRUE(x0.e<float>(i) == buffExpX0[i]);

    NDArray x1 = x(1, {1,2});
    NDArray x1Exp(buffExpX1, shapeExpX1);
    ASSERT_TRUE(x1Exp.isSameShape(x1));
    ASSERT_TRUE(x1Exp.equalsTo(x1));

//    for(int i = 0; i < shape::shapeInfoLength(x1.rankOf()); ++i)
//        ASSERT_TRUE(x1.shapeInfo()[i] == shapeExpX1[i]);
//    for(int i = 0; i < x1.lengthOf(); ++i)
//        ASSERT_TRUE(x1.e<float>(i) == buffExpX1[i]);

    NDArray x2 = x(0, {1,2}, true);
    NDArray x2Exp(buffExpX2, shapeExpX2);
    ASSERT_TRUE(x2Exp.isSameShape(x2));
//    x2.printBuffer("X2");
//    x2Exp.printBuffer("X2 EXPECT");
    ASSERT_TRUE(x2Exp.equalsTo(x2));
//    for(int i = 0; i < shape::shapeInfoLength(x2.rankOf()); ++i)
//        ASSERT_TRUE(x2.shapeInfo()[i] == shapeExpX2[i]);
//    for(int i = 0; i < x2.lengthOf(); ++i)
//        ASSERT_TRUE(x2.e<float>(i) == buffExpX2[i]);

    NDArray x3 = x(2, {1});
    NDArray x3Exp(buffExpX3, shapeExpX3);
    ASSERT_TRUE(x3Exp.isSameShape(x3));
    ASSERT_TRUE(x3Exp.equalsTo(x3));
//    for(int i = 0; i < shape::shapeInfoLength(x3.rankOf()); ++i)
//        ASSERT_TRUE(x3.shapeInfo()[i] == shapeExpX3[i]);
//    for(int i = 0; i < x3.lengthOf(); ++i)
//        ASSERT_TRUE(x3.e<float>(i) == buffExpX3[i]);

    NDArray x4 = x(2, {1}, true);
    NDArray x4Exp(buffExpX4, shapeExpX4);
    ASSERT_TRUE(x4Exp.isSameShape(x4));
    ASSERT_TRUE(x4Exp.equalsTo(x4));
//    for(int i = 0; i < shape::shapeInfoLength(x4.rankOf()); ++i)
//        ASSERT_TRUE(x4.shapeInfo()[i] == shapeExpX4[i]);
//    for(int i = 0; i < x4.lengthOf(); ++i)
//        ASSERT_TRUE(x4.e<float>(i) == buffExpX4[i]);

    NDArray x5 = x(3, {2});
    NDArray x5Exp(buffExpX5, shapeExpX5);
    ASSERT_TRUE(x5Exp.isSameShape(x5));
    ASSERT_TRUE(x5Exp.equalsTo(x5));

//    for(int i = 0; i < shape::shapeInfoLength(x5.rankOf()); ++i)
//        ASSERT_TRUE(x5.shapeInfo()[i] == shapeExpX5[i]);
//    for(int i = 0; i < x5.lengthOf(); ++i)
//        ASSERT_TRUE(x5.e<float>(i) == buffExpX5[i]);

    // ******************* //
    NDArray y0 = y(0, {1,2});
    NDArray y0Exp(buffExpY0, shapeExpY0);
    ASSERT_TRUE(y0Exp.isSameShape(y0));
    ASSERT_TRUE(y0Exp.equalsTo(y0));
//    for(int i = 0; i < shape::shapeInfoLength(y0.rankOf()); ++i)
//        ASSERT_TRUE(y0.shapeInfo()[i] == shapeExpY0[i]);
//    for(int i = 0; i < y0.lengthOf(); ++i)
//        ASSERT_TRUE(y0.e<float>(i) == buffExpY0[i]);

    NDArray y1 = y(1, {1,2});
    NDArray y1Exp(buffExpY1, shapeExpY1);
    ASSERT_TRUE(y1Exp.isSameShape(y1));
    ASSERT_TRUE(y1Exp.equalsTo(y1));
//    for(int i = 0; i < shape::shapeInfoLength(y1.rankOf()); ++i)
//        ASSERT_TRUE(y1.shapeInfo()[i] == shapeExpY1[i]);
//    for(int i = 0; i < y1.lengthOf(); ++i)
//        ASSERT_TRUE(y1.e<float>(i) == buffExpY1[i]);

    NDArray y2 = y(0, {1,2}, true);
    NDArray y2Exp(buffExpY2, shapeExpY2);
    ASSERT_TRUE(y2Exp.isSameShape(y2));
    ASSERT_TRUE(y2Exp.equalsTo(y2));
//    for(int i = 0; i < shape::shapeInfoLength(y2.rankOf()); ++i)
//        ASSERT_TRUE(y2.shapeInfo()[i] == shapeExpY2[i]);
//    for(int i = 0; i < y2.lengthOf(); ++i)
//        ASSERT_TRUE(y2.e<float>(i) == buffExpY2[i]);

    NDArray y3 = y(2, {1});
    NDArray y3Exp(buffExpY3, shapeExpY3);
    ASSERT_TRUE(y3Exp.isSameShape(y3));
    ASSERT_TRUE(y3Exp.equalsTo(y3));
//    for(int i = 0; i < shape::shapeInfoLength(y3.rankOf()); ++i)
//        ASSERT_TRUE(y3.shapeInfo()[i] == shapeExpY3[i]);
//    for(int i = 0; i < y3.lengthOf(); ++i)
//        ASSERT_TRUE(y3.e<float>(i) == buffExpY3[i]);

    NDArray y4 = y(2, {1}, true);
    NDArray y4Exp = NDArrayFactory::create<float>('f', {2,1,4}, {5, 6, 11, 12, 17, 18, 23, 24});
    ASSERT_TRUE(y4Exp.isSameShape(y4));
    ASSERT_TRUE(y4Exp.equalsTo(y4));
//    for(int i = 0; i < shape::shapeInfoLength(y4.rankOf()); ++i)
//        ASSERT_TRUE(y4.shapeInfo()[i] == shapeExpY4[i]);
//    for(int i = 0; i < y4.lengthOf(); ++i)
//        ASSERT_TRUE(y4.e<float>(i) == buffExpY4[i]);

    NDArray y5 = y(3, {2});
    NDArray y5Exp(buffExpY5, shapeExpY5);
    ASSERT_TRUE(y5Exp.isSameShape(y5));
    ASSERT_TRUE(y5Exp.equalsTo(y5));
//    for(int i = 0; i < shape::shapeInfoLength(y5.rankOf()); ++i)
//        ASSERT_TRUE(y5.shapeInfo()[i] == shapeExpY5[i]);
//    for(int i = 0; i < y5.lengthOf(); ++i)
//        ASSERT_TRUE(y5.e<float>(i) == buffExpY5[i]);

}
//////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, Test_diagonal_1) {

    auto x = NDArrayFactory::create<float>('c', {2, 3}, {1, 2, 3, 4, 5, 6});
    auto exp = NDArrayFactory::create<float>('c', {2, 1}, {1, 5});

    auto diag = x.diagonal('c');
    //diag.syncToDevice();
    for (Nd4jLong e = 0; e < exp.lengthOf(); ++e) {
        printf("VAL[%ld] = %f\n", e, diag.e<float>(e)); //, exp.e<float>(e), 1.e-5);
    }

    for (Nd4jLong e = 0; e < exp.lengthOf(); ++e) {
        ASSERT_NEAR(diag.e<float>(e), exp.e<float>(e), 1.e-5);
    }
    double eps(1.e-5);
    NDArray tmp(sd::DataType::FLOAT32, x.getContext()); // scalar = 0

    ExtraArguments extras({eps});
    NativeOpExecutioner::execReduce3Scalar(diag.getContext(), reduce3::EqualsWithEps, diag.buffer(),
            diag.shapeInfo(), diag.specialBuffer(), diag.specialShapeInfo(), extras.argumentsAsT(sd::DataType::FLOAT32),
            exp.buffer(), exp.shapeInfo(), exp.specialBuffer(), exp.specialShapeInfo(),
            tmp.buffer(), tmp.shapeInfo(), tmp.specialBuffer(), tmp.specialShapeInfo());
    hipStream_t* stream = x.getContext()->getCudaStream();
    auto res = hipStreamSynchronize(*stream);
    // tmp.printBuffer("Compare result is (expected 0)");
    ASSERT_TRUE(exp.isSameShape(diag));
    ASSERT_TRUE(exp.equalsTo(diag));
}

TEST_F(NDArrayCudaBasicsTests, Test_PermuteEquality_02) {
    auto x = NDArrayFactory::linspace<float>(1.f, 60.f, 60); //('c', {1, 60});
    //x.linspace(1);
    auto exp = NDArrayFactory::create<float>('c', {3, 4, 5}, {1.0f, 2.0f, 3.0f, 4.0f, 5.0f, 6.0f, 7.0f, 8.0f, 9.0f, 10.0f, 11.0f, 12.0f, 13.0f, 14.0f, 15.0f, 16.0f, 17.0f, 18.0f, 19.0f, 20.0f, 21.0f, 22.0f, 23.0f, 24.0f, 25.0f, 26.0f, 27.0f, 28.0f, 29.0f, 30.0f, 31.0f, 32.0f, 33.0f, 34.0f, 35.0f, 36.0f, 37.0f, 38.0f, 39.0f, 40.0f, 41.0f, 42.0f, 43.0f, 44.0f, 45.0f, 46.0f, 47.0f, 48.0f, 49.0f, 50.0f, 51.0f, 52.0f, 53.0f, 54.0f, 55.0f, 56.0f, 57.0f, 58.0f, 59.0f, 60.0});
    x->reshapei('c', {3, 4, 5});

    x->permutei({0, 1, 2});
    x->streamline();

//    x.printShapeInfo("{0, 1, 2} shape");
//    x.printBuffer("{0, 1, 2} data");

    ASSERT_TRUE(exp.isSameShape(x));
    ASSERT_TRUE(exp.equalsTo(x));
    delete x;
}

TEST_F(NDArrayCudaBasicsTests, Test_PermuteEquality_0) {
    auto x = NDArrayFactory::create<float>('c', {1, 60});
    x.linspace(1);
    auto exp = NDArrayFactory::create<float>('c', {3, 4, 5}, {1.0f, 2.0f, 3.0f, 4.0f, 5.0f, 6.0f, 7.0f, 8.0f, 9.0f, 10.0f, 11.0f, 12.0f, 13.0f, 14.0f, 15.0f, 16.0f, 17.0f, 18.0f, 19.0f, 20.0f, 21.0f, 22.0f, 23.0f, 24.0f, 25.0f, 26.0f, 27.0f, 28.0f, 29.0f, 30.0f, 31.0f, 32.0f, 33.0f, 34.0f, 35.0f, 36.0f, 37.0f, 38.0f, 39.0f, 40.0f, 41.0f, 42.0f, 43.0f, 44.0f, 45.0f, 46.0f, 47.0f, 48.0f, 49.0f, 50.0f, 51.0f, 52.0f, 53.0f, 54.0f, 55.0f, 56.0f, 57.0f, 58.0f, 59.0f, 60.0});
    x.reshapei('c', {3, 4, 5});

    x.permutei({0, 1, 2});
    x.streamline();

//    x.printShapeInfo("{0, 1, 2} shape");
//    x.printBuffer("{0, 1, 2} data");

    ASSERT_TRUE(exp.isSameShape(&x));
    ASSERT_TRUE(exp.equalsTo(&x));
}
TEST_F(NDArrayCudaBasicsTests, Test_PermuteEquality_1) {
    auto x = NDArrayFactory::create<float>('c', {1, 60});
    x.linspace(1);
    auto exp = NDArrayFactory::create<float>('c', {3, 4, 5}, {1.0f, 2.0f, 3.0f, 4.0f, 5.0f, 6.0f, 7.0f, 8.0f, 9.0f, 10.0f, 11.0f, 12.0f, 13.0f, 14.0f, 15.0f, 16.0f, 17.0f, 18.0f, 19.0f, 20.0f, 21.0f, 22.0f, 23.0f, 24.0f, 25.0f, 26.0f, 27.0f, 28.0f, 29.0f, 30.0f, 31.0f, 32.0f, 33.0f, 34.0f, 35.0f, 36.0f, 37.0f, 38.0f, 39.0f, 40.0f, 41.0f, 42.0f, 43.0f, 44.0f, 45.0f, 46.0f, 47.0f, 48.0f, 49.0f, 50.0f, 51.0f, 52.0f, 53.0f, 54.0f, 55.0f, 56.0f, 57.0f, 58.0f, 59.0f, 60.0});
    x.reshapei('c', {3, 4, 5});

    x.permutei({0, 1, 2});
    x.streamline();

//    x.printShapeInfo("{0, 1, 2} shape");
//    x.printBuffer("{0, 1, 2} data");

    ASSERT_TRUE(exp.isSameShape(&x));
    ASSERT_TRUE(exp.equalsTo(&x));
}
TEST_F(NDArrayCudaBasicsTests, Test_PermuteEquality_2) {
    //auto x = NDArrayFactory::create<float>('c', {1, 60});
    auto xx = NDArrayFactory::linspace<float>(1.f, 60.f, 60); //('c', {1, 60});
//    auto x = *xx;
    //x.linspace(1);
//    auto exp = NDArrayFactory::create<float>('c', {3, 4, 5}, {1.0f, 2.0f, 3.0f, 4.0f, 5.0f, 6.0f, 7.0f, 8.0f, 9.0f, 10.0f, 11.0f, 12.0f, 13.0f, 14.0f, 15.0f, 16.0f, 17.0f, 18.0f, 19.0f, 20.0f, 21.0f, 22.0f, 23.0f, 24.0f, 25.0f, 26.0f, 27.0f, 28.0f, 29.0f, 30.0f, 31.0f, 32.0f, 33.0f, 34.0f, 35.0f, 36.0f, 37.0f, 38.0f, 39.0f, 40.0f, 41.0f, 42.0f, 43.0f, 44.0f, 45.0f, 46.0f, 47.0f, 48.0f, 49.0f, 50.0f, 51.0f, 52.0f, 53.0f, 54.0f, 55.0f, 56.0f, 57.0f, 58.0f, 59.0f, 60.0});
//    x.reshapei('c', {3, 4, 5});

//    x.permutei({0, 1, 2});
//    x.streamline();

//    x.printShapeInfo("{0, 1, 2} shape");
//    x.printBuffer("{0, 1, 2} data");

//    ASSERT_TRUE(exp.isSameShape(&x));
//    ASSERT_TRUE(exp.equalsTo(&x));
    delete xx;
}
TEST_F(NDArrayCudaBasicsTests, Test_PermuteEquality_3) {
    auto x = NDArrayFactory::create<float>('c', {1, 60});
    //x.linspace(1);
    for (int l = 0; l < x.lengthOf(); l++)
        x.p(l, float(l + 1.f));
    auto exp = NDArrayFactory::create<float>('c', {3, 4, 5}, {1.0f, 2.0f, 3.0f, 4.0f, 5.0f, 6.0f, 7.0f, 8.0f, 9.0f, 10.0f, 11.0f, 12.0f, 13.0f, 14.0f, 15.0f, 16.0f, 17.0f, 18.0f, 19.0f, 20.0f, 21.0f, 22.0f, 23.0f, 24.0f, 25.0f, 26.0f, 27.0f, 28.0f, 29.0f, 30.0f, 31.0f, 32.0f, 33.0f, 34.0f, 35.0f, 36.0f, 37.0f, 38.0f, 39.0f, 40.0f, 41.0f, 42.0f, 43.0f, 44.0f, 45.0f, 46.0f, 47.0f, 48.0f, 49.0f, 50.0f, 51.0f, 52.0f, 53.0f, 54.0f, 55.0f, 56.0f, 57.0f, 58.0f, 59.0f, 60.0});
    x.reshapei('c', {3, 4, 5});

    x.permutei({0, 1, 2});
    x.streamline();

//    x.printShapeInfo("{0, 1, 2} shape");
//    x.printBuffer("{0, 1, 2} data");

    ASSERT_TRUE(exp.isSameShape(&x));
    ASSERT_TRUE(exp.equalsTo(&x));
}

TEST_F(NDArrayCudaBasicsTests, Test_Empty_1) {
    auto x = NDArrayFactory::empty<float>();
    ASSERT_TRUE(x.isActualOnHostSide());
    ASSERT_TRUE(x.isEmpty());
}

TEST_F(NDArrayCudaBasicsTests, Test_Empty_2) {
    auto x = NDArrayFactory::empty_<float>();

    ASSERT_TRUE(x->isEmpty());
    delete x;
}

TEST_F(NDArrayCudaBasicsTests, Test_Empty_3) {
    auto x = NDArrayFactory::empty(sd::DataType::FLOAT32);

    ASSERT_TRUE(x.isEmpty());
}

TEST_F(NDArrayCudaBasicsTests, Test_Empty_4) {
    auto x = NDArrayFactory::empty_(sd::DataType::FLOAT32);

    ASSERT_TRUE(x->isEmpty());
    delete x;
}