#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
//

#include "testlayers.h"
#include <ops/declarable/CustomOperations.h>
#include <array/NDArray.h>
#include <ops/ops.h>
#include <helpers/GradCheck.h>
#include <helpers/RandomLauncher.h>
#include <exceptions/cuda_exception.h>


using namespace sd;


class AtomicTests : public testing::Test {
public:
    AtomicTests() {
        //
    }
};

template <typename T>
static _CUDA_G void multiplyKernel(void *vbuffer, uint64_t length, void *vresult) {
    auto buffer = reinterpret_cast<T*>(vbuffer);
    auto result = reinterpret_cast<T*>(vresult);

    auto tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (auto e = tid; e < length; e += gridDim.x * blockDim.x) {
        auto rem = e % 4;
        auto i = (e - rem) / 4;

        sd::math::atomics::nd4j_atomicMul<T>(&result[i], buffer[e]);
    }
}

template <typename T>
static void multiplyLauncher(void *vbuffer, uint64_t length, void *vresult) {
    multiplyKernel<T><<<256, 256, 1024, *sd::LaunchContext::defaultContext()->getCudaStream()>>>(vbuffer, length, vresult);
    auto err = hipStreamSynchronize(*sd::LaunchContext::defaultContext()->getCudaStream());
    if (err != 0)
        throw sd::cuda_exception::build("multiply failed", err);
}

template <typename T>
static _CUDA_G void sumKernel(void *vbuffer, uint64_t length, void *vresult) {
    auto buffer = reinterpret_cast<T*>(vbuffer);
    auto result = reinterpret_cast<T*>(vresult);

    auto tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (auto e = tid; e < length; e += gridDim.x * blockDim.x) {
        auto rem = e % 4;
        auto i = (e - rem) / 4;

        sd::math::atomics::nd4j_atomicAdd<T>(&result[i], buffer[e]);
    }
}

template <typename T>
static void sumLauncher(void *vbuffer, uint64_t length, void *vresult) {
    sumKernel<T><<<256, 256, 1024, *sd::LaunchContext::defaultContext()->getCudaStream()>>>(vbuffer, length, vresult);
    auto err = hipStreamSynchronize(*sd::LaunchContext::defaultContext()->getCudaStream());
    if (err != 0)
        throw sd::cuda_exception::build("sum failed", err);
}

template <typename T>
static _CUDA_G void subKernel(void *vbuffer, uint64_t length, void *vresult) {
    auto buffer = reinterpret_cast<T*>(vbuffer);
    auto result = reinterpret_cast<T*>(vresult);

    auto tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (auto e = tid; e < length; e += gridDim.x * blockDim.x) {
        auto rem = e % 4;
        auto i = (e - rem) / 4;

        sd::math::atomics::nd4j_atomicSub<T>(&result[i], buffer[e]);
    }
}

template <typename T>
static void subLauncher(void *vbuffer, uint64_t length, void *vresult) {
    subKernel<T><<<256, 256, 1024, *sd::LaunchContext::defaultContext()->getCudaStream()>>>(vbuffer, length, vresult);
    auto err = hipStreamSynchronize(*sd::LaunchContext::defaultContext()->getCudaStream());
    if (err != 0)
        throw sd::cuda_exception::build("sub failed", err);
}

template <typename T>
static _CUDA_G void divKernel(void *vbuffer, uint64_t length, void *vresult) {
    auto buffer = reinterpret_cast<T*>(vbuffer);
    auto result = reinterpret_cast<T*>(vresult);

    auto tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (auto e = tid; e < length; e += gridDim.x * blockDim.x) {
        auto rem = e % 4;
        auto i = (e - rem) / 4;

        sd::math::atomics::nd4j_atomicDiv<T>(&result[i], buffer[e]);
    }
}

template <typename T>
static void divLauncher(void *vbuffer, uint64_t length, void *vresult) {
    divKernel<T><<<256, 256, 1024, *sd::LaunchContext::defaultContext()->getCudaStream()>>>(vbuffer, length, vresult);
    auto err = hipStreamSynchronize(*sd::LaunchContext::defaultContext()->getCudaStream());
    if (err != 0)
        throw sd::cuda_exception::build("div failed", err);
}

static void multiplyHost(NDArray &input, NDArray &output) {
    BUILD_SINGLE_SELECTOR(input.dataType(), multiplyLauncher, (input.specialBuffer(), input.lengthOf(), output.specialBuffer()), NUMERIC_TYPES);
}

static void sumHost(NDArray &input, NDArray &output) {
    BUILD_SINGLE_SELECTOR(input.dataType(), sumLauncher, (input.specialBuffer(), input.lengthOf(), output.specialBuffer()), NUMERIC_TYPES);
}

static void subHost(NDArray &input, NDArray &output) {
    BUILD_SINGLE_SELECTOR(input.dataType(), subLauncher, (input.specialBuffer(), input.lengthOf(), output.specialBuffer()), FLOAT_TYPES);
}

static void divHost(NDArray &input, NDArray &output) {
    BUILD_SINGLE_SELECTOR(input.dataType(), divLauncher, (input.specialBuffer(), input.lengthOf(), output.specialBuffer()), FLOAT_TYPES);
}

TEST_F(AtomicTests, test_multiply) {
    std::vector<sd::DataType> dtypes = {sd::DataType::FLOAT32, sd::DataType::DOUBLE, sd::DataType::INT16, sd::DataType::HALF};

    for (auto t:dtypes) {
        nd4j_printf("Trying data type [%s]\n", DataTypeUtils::asString(t).c_str());
        NDArray input('c', {4, 25}, t);
        NDArray output('c', {input.lengthOf() / 4}, t);
        NDArray exp = output.ulike();

        input.assign(2);
        output.assign(2);
        exp.assign(32);

        multiplyHost(input, output);
        ASSERT_EQ(exp, output);
    }
}

TEST_F(AtomicTests, test_multiply_2) {
    std::vector<sd::DataType> dtypes = {sd::DataType::FLOAT32, sd::DataType::DOUBLE, sd::DataType::HALF, sd::DataType::BFLOAT16};

    for (auto t:dtypes) {
        nd4j_printf("Trying data type [%s]\n", DataTypeUtils::asString(t).c_str());
        NDArray input('c', {4, 25}, t);
        NDArray output('c', {input.lengthOf() / 4}, t);
        NDArray exp = output.ulike();

        input.assign(1.5);
        output.assign(2);
        exp.assign(10.125);

        multiplyHost(input, output);
//        output.printBuffer("multiply 2");
        ASSERT_EQ(exp, output);
    }
}

TEST_F(AtomicTests, test_sum) {
    std::vector<sd::DataType> dtypes = {sd::DataType::FLOAT32, sd::DataType::DOUBLE, sd::DataType::BFLOAT16, sd::DataType::HALF, sd::DataType::INT16};

    for (auto t:dtypes) {
        nd4j_printf("Trying data type [%s]\n", DataTypeUtils::asString(t).c_str());
        NDArray input('c', {4, 25}, t);
        NDArray output('c', {input.lengthOf() / 4}, t);
        NDArray exp = output.ulike();

        input.assign(1);
        output.assign(1);
        exp.assign(5);

        sumHost(input, output);
//        output.printIndexedBuffer("Sum");
        ASSERT_EQ(exp, output);
    }
}

TEST_F(AtomicTests, test_sub) {
    std::vector<sd::DataType> dtypes = {sd::DataType::FLOAT32, sd::DataType::DOUBLE, sd::DataType::HALF};

    for (auto t:dtypes) {
        nd4j_printf("Trying data type [%s]\n", DataTypeUtils::asString(t).c_str());
        NDArray input('c', {4, 25}, t);
        NDArray output('c', {input.lengthOf() / 4}, t);
        NDArray exp = output.ulike();

        input.assign(1);
        output.assign(5);
        exp.assign(1);

        subHost(input, output);
//        output.printBuffer("Sub");

        ASSERT_EQ(exp, output);
    }
}

TEST_F(AtomicTests, test_div) {
    std::vector<sd::DataType> dtypes = {sd::DataType::FLOAT32, sd::DataType::DOUBLE, sd::DataType::BFLOAT16, sd::DataType::HALF};

    for (auto t:dtypes) {
        nd4j_printf("Trying data type [%s]\n", DataTypeUtils::asString(t).c_str());
        NDArray input('c', {4, 25}, t);
        NDArray output('c', {input.lengthOf() / 4}, t);
        NDArray exp = output.ulike();

        input.assign(2);
        output.assign(32);
        exp.assign(2);

        divHost(input, output);
//        output.printBuffer("Div");
        ASSERT_EQ(exp, output);
    }
}