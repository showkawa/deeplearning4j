#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
//

#include "testlayers.h"
#include <array/ExtraArguments.h>
#include <array>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

using namespace sd;

class LambdaTests : public testing::Test {
public:

    LambdaTests() {
        printf("\n");
        fflush(stdout);
    }
};

template <typename Lambda>
__global__ void runLambda(double *input, double *output, Nd4jLong length, Lambda lambda) {
    auto tid = blockIdx.x * blockDim.x + threadIdx.x;
    for (Nd4jLong e = tid; e < length; e += gridDim.x * blockDim.x) {
        output[e] = lambda(input[e]);
    }
}

void launcher(hipStream_t *stream, double *input, double *output, Nd4jLong length) {
    //auto f = [] __host__ __device__ (double x) -> double {
    //        return x + 1.;
    //};
    auto f = LAMBDA_D(x) {
        return x+1.;
    };


    runLambda<<<128, 128, 128, *stream>>>(input, output, length, f);
}


// TEST_F(LambdaTests, test_basic_1) {
//     auto x = NDArrayFactory::create<double>('c', {5});
//     auto e = NDArrayFactory::create<double>('c', {5}, {1., 1., 1., 1., 1.});



//     //x.applyLambda<double>(f, nullptr);
//     launcher(LaunchContext::defaultContext()->getCudaStream(), (double *)x.specialBuffer(), (double *)x.specialBuffer(), x.lengthOf());
//     auto res = hipStreamSynchronize(*LaunchContext::defaultContext()->getCudaStream());
//     ASSERT_EQ(0, res);

//     ASSERT_EQ(e, x);
// }

// void test(NDArray &x) {
//     auto f = LAMBDA_D(x) {
//         return x+1.;
//     };

//     x.applyLambda(f, x);
// }

// template <typename T>
// void test2(NDArray &x) {
//     auto f = LAMBDA_T(x) {
//         return x+1.;
//     };

//     x.applyLambda(f, x);
// }

// void testPairwise(NDArray &x, NDArray &y) {
//     auto f = LAMBDA_DD(x, y) {
//         return x + y +1.;
//     };

//     x.applyPairwiseLambda(y, f, x);
// }

// void testTriplewise(NDArray &i, NDArray &j, NDArray &k) {
//     auto f = LAMBDA_DDD(i, j, k) {
//         return i + j + k + 2.;
//     };

//     i.applyTriplewiseLambda(j, k, f, i);
// }

// void testIndexed(NDArray &x) {
//     auto f = ILAMBDA_D(x) {
//         return _idx + 1.;
//     };

//     x.applyIndexedLambda(f, x);
// }

// void testIndexedPairwise(NDArray &x, NDArray &y) {
//     auto f = ILAMBDA_DD(x, y) {
//         return _idx + x + y +1.;
//     };

//     x.applyIndexedPairwiseLambda(y, f, x);
// }

// TEST_F(LambdaTests, test_basic_2) {
//     auto x = NDArrayFactory::create<double>('c', {5});
//     auto e = NDArrayFactory::create<double>('c', {5}, {1., 1., 1., 1., 1.});

//     test(x);

//     ASSERT_EQ(e, x);
// }

// TEST_F(LambdaTests, test_basic_3) {
//     auto x = NDArrayFactory::create<float>('c', {5});
//     auto e = NDArrayFactory::create<float>('c', {5}, {1.f, 1.f, 1.f, 1.f, 1.f});

//     test(x);

//     ASSERT_EQ(e, x);
// }

// TEST_F(LambdaTests, test_basic_4) {
//     auto x = NDArrayFactory::create<float>('c', {5});
//     auto e = NDArrayFactory::create<float>('c', {5}, {1.f, 1.f, 1.f, 1.f, 1.f});

//     test2<float>(x);

//     ASSERT_EQ(e, x);
// }

// TEST_F(LambdaTests, test_basic_5) {
//     auto x = NDArrayFactory::create<double>('c', {5}, {1., 1., 1., 1., 1.});
//     auto y = NDArrayFactory::create<double>('c', {5}, {2., 2., 2., 2., 2.});
//     auto e = NDArrayFactory::create<double>('c', {5}, {4., 4., 4., 4., 4.});

//     testPairwise(x, y);

//     ASSERT_EQ(e, x);
// }

// TEST_F(LambdaTests, test_basic_6) {
//     auto x = NDArrayFactory::create<double>('c', {5});
//     auto e = NDArrayFactory::create<double>('c', {5}, {1., 2., 3., 4., 5.});

//     testIndexed(x);

//     ASSERT_EQ(e, x);
// }

// TEST_F(LambdaTests, test_basic_7) {
//     auto w = NDArrayFactory::create<double>('c', {5}, {0., 0., 0., 0., 0.});
//     auto x = NDArrayFactory::create<double>('c', {5}, {1., 1., 1., 1., 1.});
//     auto y = NDArrayFactory::create<double>('c', {5}, {2., 2., 2., 2., 2.});
//     auto e = NDArrayFactory::create<double>('c', {5}, {5., 5., 5., 5., 5.});

//     testTriplewise(w, x, y);

//     ASSERT_EQ(e, w);
// }

// TEST_F(LambdaTests, test_basic_8) {
//     auto x = NDArrayFactory::create<double>('c', {5}, {1., 1., 1., 1., 1.});
//     auto y = NDArrayFactory::create<double>('c', {5}, {2., 2., 2., 2., 2.});
//     auto e = NDArrayFactory::create<double>('c', {5}, {4., 5., 6., 7., 8.});

//     testIndexedPairwise(x, y);

//     ASSERT_EQ(e, x);
// }


// template <typename T>
// void testPairwiseMy(NDArray &x, NDArray &y, NDArray &z) {

//     auto f = LAMBDA_TT(x, y){
//         return sd::math::nd4j_max<T>(x, (T)0.f)
//               - x * y
//               + sd::math::nd4j_log<T,T>((T)1.f
//                 + sd::math::nd4j_exp<T,T>(-sd::math::nd4j_abs(x)));
//     };

//     x.applyPairwiseLambda(y, f, z);
// }

// ///////////////////////////////////////////////////////////////////
// TEST_F(LambdaTests, test_basic_9) {

//     NDArray labels('c', {2,3,4},{0,1,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,0,1,1,0,1,0});
//     NDArray logits('c', {2,3,4}, sd::DataType::DOUBLE);
//     NDArray output('c', {2,3,4}, sd::DataType::DOUBLE);
//     NDArray expected('c', {2,3,4}, {0.744397, 0.598139, 0.554355, 0.913015, 0.474077, 1.037488, 0.403186, 1.171101, 0.341154, 1.313262, 0.287335, 1.463282, 0.241008, 1.620417, 0.201413, 1.783901, 0.167786, 1.952978, 2.039387, 0.126928, 0.115520, 2.305083, 0.095545, 2.486836});

//     logits.linspace(0.1, 0.1);

//     NDArray::prepareSpecialUse({&output}, {&logits, &labels});
//     testPairwiseMy<double>(logits, labels, output);
//     NDArray::registerSpecialUse({&output}, {&logits, &labels});

//     // output.printBuffer(nullptr, -1, true);
//     ASSERT_TRUE(expected.equalsTo(output));
// }
