/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/


//
// @author raver119@gmail.com
//

#include "testlayers.h"
#include <initializer_list>
#include <array/NDArrayFactory.h>
#include <ops/declarable/PlatformHelper.h>
#include <ops/declarable/CustomOperations.h>
#include <execution/Engine.h>

#ifdef HAVE_CUDNN

#include <ops/declarable/platform/cudnn/cudnnUtils.h>

#endif

using namespace sd;

class CuDnnTests : public testing::Test {
public:

};

static void printer(std::initializer_list<sd::ops::platforms::PlatformHelper*> helpers) {

    for (auto v:helpers) {
        nd4j_printf("Initialized [%s]\n", v->name().c_str());
    }
}


TEST_F(CuDnnTests, helpers_includer) {
    // we need this block, to make sure all helpers are still available within binary, and not optimized out by linker
#ifdef HAVE_CUDNN
    sd::ops::platforms::PLATFORM_conv2d_ENGINE_CUDA conv2d;
    sd::ops::platforms::PLATFORM_conv2d_bp_ENGINE_CUDA conv2d_bp;
    sd::ops::platforms::PLATFORM_conv3dnew_ENGINE_CUDA conv3dnew;
    sd::ops::platforms::PLATFORM_conv3dnew_bp_ENGINE_CUDA conv3dnew_bp;
    sd::ops::platforms::PLATFORM_depthwise_conv2d_ENGINE_CUDA depthwise_conv2d;
    sd::ops::platforms::PLATFORM_depthwise_conv2d_bp_ENGINE_CUDA depthwise_conv2d_bp;
    sd::ops::platforms::PLATFORM_batchnorm_ENGINE_CUDA batchnorm;
    sd::ops::platforms::PLATFORM_batchnorm_bp_ENGINE_CUDA batchnorm_bp;
    sd::ops::platforms::PLATFORM_avgpool2d_ENGINE_CUDA avgpool2d;
    sd::ops::platforms::PLATFORM_avgpool2d_bp_ENGINE_CUDA avgpool2d_bp;
    sd::ops::platforms::PLATFORM_maxpool2d_ENGINE_CUDA maxpool2d;
    sd::ops::platforms::PLATFORM_maxpool2d_bp_ENGINE_CUDA maxpool2d_bp;
    sd::ops::platforms::PLATFORM_avgpool3dnew_ENGINE_CUDA avgpool3dnew;
    sd::ops::platforms::PLATFORM_avgpool3dnew_bp_ENGINE_CUDA avgpool3dnew_bp;
    sd::ops::platforms::PLATFORM_maxpool3dnew_ENGINE_CUDA maxpool3dnew;
    sd::ops::platforms::PLATFORM_maxpool3dnew_bp_ENGINE_CUDA maxpool3dnew_bp;



    printer({&conv2d});
    printer({&conv2d_bp});
    printer({&conv3dnew});
    printer({&conv3dnew_bp});
    printer({&depthwise_conv2d});
    printer({&depthwise_conv2d_bp});
    printer({&batchnorm});
    printer({&batchnorm_bp});
    printer({&avgpool2d});
    printer({&avgpool2d_bp});
    printer({&maxpool2d});
    printer({&maxpool2d_bp});
    printer({&avgpool3dnew});
    printer({&avgpool3dnew_bp});
    printer({&maxpool3dnew});
    printer({&maxpool3dnew_bp});
#endif
}


TEST_F(CuDnnTests, mixed_helpers_test_1) {
#if defined(HAVE_CUDNN) && defined (HAVE_MKLDNN)
    nd4j_printf("Mixed platforms test\n", "");


    int bS=2, iH=4,iW=3,  iC=4,oC=3,  kH=3,kW=2,  sH=1,sW=1,  pH=0,pW=0,  dH=1,dW=1;
    int       oH=2,oW=2;
    int paddingMode = 0;             // 1-SAME, 0-VALID;
    int dataFormat  = 0;             // 1-NHWC, 0-NCHW

    auto input    = NDArrayFactory::create<float>('c', {bS, iC, iH, iW});
    auto weights  = NDArrayFactory::create<float>('c', {oC, iC, kH, kW});
    auto bias     = NDArrayFactory::create<float>('c', {oC}, {1,2,3});

    auto expOutput = NDArrayFactory::create<float>('c', {bS, oC, oH, oW}, {61.f,   61.f,  61.f,   61.f, 177.2f,  177.2f, 177.2f,  177.2f, 293.4f,  293.4f, 293.4f,  293.4f,  61.f,   61.f,  61.f,   61.f, 177.2f,  177.2f, 177.2f,  177.2f, 293.4f,  293.4f, 293.4f,  293.4f});
    auto zCUDA = expOutput.like();
    auto zMKL = expOutput.like();

    input = 2.;
    weights.linspace(0.1, 0.1);
    weights.permutei({2,3,1,0});

    input.syncToHost();
    weights.syncToHost();
    bias.syncToHost();

    sd::ops::conv2d op;

    // cuDNN part
    Context cuda(1);
    cuda.setTargetEngine(samediff::Engine::ENGINE_CUDA);
    cuda.setInputArray(0, &input);
    cuda.setInputArray(1, &weights);
    cuda.setInputArray(2, &bias);
    cuda.setOutputArray(0, &zCUDA);
    cuda.setIArguments({kH,kW,  sH,sW,  pH,pW,  dH,dW, paddingMode, dataFormat});
    auto statusCUDA = op.execute(&cuda);

    ASSERT_EQ(Status::OK(), statusCUDA);
    ASSERT_EQ(expOutput, zCUDA);

    // MKL-DNN part
    Context mkl(1);
    mkl.setTargetEngine(samediff::Engine::ENGINE_CPU);
    mkl.setInputArray(0, &input);
    mkl.setInputArray(1, &weights);
    mkl.setInputArray(2, &bias);
    mkl.setOutputArray(0, &zMKL);
    mkl.setIArguments({kH,kW,  sH,sW,  pH,pW,  dH,dW, paddingMode, dataFormat});
    auto statusMKL = op.execute(&mkl);

    zMKL.tickWriteHost();

    ASSERT_EQ(Status::OK(), statusMKL);
    ASSERT_EQ(expOutput, zMKL);
#endif
}