/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
//

#include "testlayers.h"
#include <array/NDArray.h>
#include <array/NDArrayFactory.h>
#include <graph/Context.h>
#include <graph/Node.h>
#include <graph/Variable.h>
#include <graph/VariableSpace.h>
#include <execution/LaunchContext.h>
#include <ops/specials_cuda.h>
#include <helpers/TAD.h>

#include <hip/hip_runtime.h>

using namespace sd;
using namespace sd::graph;

class NDArrayConstructorsTests : public testing::Test {
public:

};

TEST_F(NDArrayConstructorsTests, test_constructor_1) {
    auto x = NDArrayFactory::empty_<float>();

    ASSERT_TRUE(x->buffer() == nullptr);
    ASSERT_TRUE(x->specialBuffer() == nullptr);

    ASSERT_FALSE(x->shapeInfo() == nullptr);
    ASSERT_FALSE(x->specialShapeInfo() == nullptr);

    ASSERT_TRUE(x->isActualOnDeviceSide());
    ASSERT_TRUE(x->isActualOnHostSide());

    delete x;
}

TEST_F(NDArrayConstructorsTests, test_constructor_2) {
    auto x = NDArrayFactory::vector<float>(5, 1.0f);


    ASSERT_FALSE(x->buffer() == nullptr);
    ASSERT_FALSE(x->specialBuffer() == nullptr);

    ASSERT_FALSE(x->shapeInfo() == nullptr);
    ASSERT_FALSE(x->specialShapeInfo() == nullptr);

    ASSERT_TRUE(x->isActualOnDeviceSide());
    ASSERT_FALSE(x->isActualOnHostSide());

    delete x;
}

TEST_F(NDArrayConstructorsTests, test_constructor_3) {
    auto x = NDArrayFactory::create<float>('c',{5, 5});

    ASSERT_TRUE(x.buffer() == nullptr);
    ASSERT_FALSE(x.specialBuffer() == nullptr);

    ASSERT_FALSE(x.shapeInfo() == nullptr);
    ASSERT_FALSE(x.specialShapeInfo() == nullptr);

    ASSERT_TRUE(x.isActualOnDeviceSide());
    ASSERT_FALSE(x.isActualOnHostSide());
}

TEST_F(NDArrayConstructorsTests, test_constructor_4) {
    auto x = NDArrayFactory::create(sd::DataType::FLOAT32, 1.0f);

    ASSERT_FALSE(x.buffer() == nullptr);
    ASSERT_FALSE(x.specialBuffer() == nullptr);

    ASSERT_FALSE(x.shapeInfo() == nullptr);
    ASSERT_FALSE(x.specialShapeInfo() == nullptr);

    ASSERT_TRUE(x.isActualOnDeviceSide());
    ASSERT_TRUE(x.isActualOnHostSide());
}

TEST_F(NDArrayConstructorsTests, test_constructor_5) {
    auto x = NDArrayFactory::create<double>('c',{2, 2}, {1, 2, 3, 4});

    ASSERT_TRUE(x.buffer() == nullptr);
    ASSERT_FALSE(x.specialBuffer() == nullptr);

    ASSERT_FALSE(x.shapeInfo() == nullptr);
    ASSERT_FALSE(x.specialShapeInfo() == nullptr);

    ASSERT_TRUE(x.isActualOnDeviceSide());
    ASSERT_FALSE(x.isActualOnHostSide());
}

TEST_F(NDArrayConstructorsTests, test_constructor_6) {
    auto x = NDArrayFactory::create<double>('c', {2, 2}, {1, 2, 3, 4});
    NDArray y(x);

    ASSERT_TRUE(y.buffer() == nullptr);
    ASSERT_FALSE(y.specialBuffer() == nullptr);

    ASSERT_FALSE(y.shapeInfo() == nullptr);
    ASSERT_FALSE(y.specialShapeInfo() == nullptr);

    ASSERT_TRUE(y.isActualOnDeviceSide());
    ASSERT_FALSE(y.isActualOnHostSide());
}

TEST_F(NDArrayConstructorsTests, test_constructor_7) {
    auto x = NDArrayFactory::create<float>(1.0f);

    ASSERT_FALSE(x.buffer() == nullptr);
    ASSERT_FALSE(x.specialBuffer() == nullptr);

    ASSERT_FALSE(x.shapeInfo() == nullptr);
    ASSERT_FALSE(x.specialShapeInfo() == nullptr);

    ASSERT_TRUE(x.isActualOnDeviceSide());
    ASSERT_TRUE(x.isActualOnHostSide());
}

TEST_F(NDArrayConstructorsTests, test_constructor_8) {
    auto x = NDArrayFactory::create_<double>('c',{2, 2}, {1, 2, 3, 4});

    ASSERT_TRUE(x->buffer() == nullptr);
    ASSERT_FALSE(x->specialBuffer() == nullptr);

    ASSERT_FALSE(x->shapeInfo() == nullptr);
    ASSERT_FALSE(x->specialShapeInfo() == nullptr);

    ASSERT_TRUE(x->isActualOnDeviceSide());
    ASSERT_FALSE(x->isActualOnHostSide());

    delete x;
}

TEST_F(NDArrayConstructorsTests, test_constructor_9) {
    auto x = NDArrayFactory::create_<double>('c',{2, 2});

    ASSERT_TRUE(x->buffer() == nullptr);
    ASSERT_FALSE(x->specialBuffer() == nullptr);

    ASSERT_FALSE(x->shapeInfo() == nullptr);
    ASSERT_FALSE(x->specialShapeInfo() == nullptr);

    ASSERT_TRUE(x->isActualOnDeviceSide());
    ASSERT_FALSE(x->isActualOnHostSide());

    delete x;
}

TEST_F(NDArrayConstructorsTests, test_linspace_1) {
    auto x = NDArrayFactory::linspace<float>(1.0f, 10.0f, 20);

    ASSERT_FALSE(x->buffer() == nullptr);
    ASSERT_FALSE(x->specialBuffer() == nullptr);

    ASSERT_FALSE(x->shapeInfo() == nullptr);
    ASSERT_FALSE(x->specialShapeInfo() == nullptr);

    ASSERT_TRUE(x->isActualOnDeviceSide());
    ASSERT_TRUE(x->isActualOnHostSide());

    delete x;
}

TEST_F(NDArrayConstructorsTests, test_constructor_10) {

    NDArray scalar1(sd::DataType::DOUBLE); // scalar1 = 0
    NDArray scalar2('c', {}, std::vector<double>{0});

    ASSERT_TRUE(scalar1.isActualOnDeviceSide());
    ASSERT_TRUE(!scalar1.isActualOnHostSide());
    ASSERT_TRUE(scalar2.isActualOnDeviceSide());
    ASSERT_TRUE(scalar2.isActualOnHostSide());
    
    ASSERT_TRUE(scalar2.equalsTo(scalar1));
    
    ASSERT_TRUE(scalar1.isActualOnDeviceSide());
    ASSERT_TRUE(!scalar1.isActualOnHostSide());
    ASSERT_TRUE(scalar2.isActualOnDeviceSide());
    ASSERT_TRUE(scalar2.isActualOnHostSide());

    ASSERT_TRUE(scalar1.buffer() == nullptr);
    ASSERT_TRUE(scalar1.specialBuffer() != nullptr);
    ASSERT_TRUE(scalar1.shapeInfo() != nullptr);
    ASSERT_TRUE(scalar1.specialShapeInfo() != nullptr);
    ASSERT_TRUE(scalar1.lengthOf() == 1);
}