/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
//

#include "testlayers.h"
#include <array/NDArray.h>
#include <graph/Context.h>
#include <graph/Node.h>
#include <graph/Variable.h>
#include <graph/VariableSpace.h>
#include <ops/declarable/CustomOperations.h>
#include <ops/declarable/helpers/convolutions.h>
#include <ops/declarable/helpers/col2im.h>
#include <helpers/RandomLauncher.h>

using namespace sd;
using namespace sd::graph;
using namespace sd::memory;

class DataBufferTestsCuda : public testing::Test {
public:

};

/*
TEST_F(DataBufferTestsCuda, test_alloc_limit_1) {
    auto deviceId = AffinityManager::currentDeviceId();

    auto odLimit = MemoryCounter::getInstance().deviceLimit(deviceId);

    auto opLimit = MemoryCounter::getInstance().groupLimit(MemoryType::HOST);
    auto osLimit = MemoryCounter::getInstance().groupLimit(MemoryType::DEVICE);

    auto odUse = MemoryCounter::getInstance().allocatedDevice(deviceId);

    auto opUse = MemoryCounter::getInstance().allocatedGroup(MemoryType::HOST);
    auto osUse = MemoryCounter::getInstance().allocatedGroup(MemoryType::DEVICE);

    auto limitSize = odUse + 150000000;
    auto allocSize = 100000000;

    MemoryCounter::getInstance().setDeviceLimit(deviceId, odLimit + limitSize);
    MemoryCounter::getInstance().setGroupLimit(MemoryType::HOST, opLimit + limitSize);
    MemoryCounter::getInstance().setGroupLimit(MemoryType::DEVICE, osLimit + limitSize);

    DataBuffer buffer(allocSize, DataType::INT32, nullptr, true);

    // separately testing per-device limits and group limits
    ASSERT_EQ(odUse + allocSize, MemoryCounter::getInstance().allocatedDevice(deviceId));
    ASSERT_EQ(opUse + allocSize, MemoryCounter::getInstance().allocatedGroup(MemoryType::HOST));
    ASSERT_EQ(osUse + allocSize, MemoryCounter::getInstance().allocatedGroup(MemoryType::DEVICE));

    // setting smaller limits, to make sure next allocation fails with OOM exception
    MemoryCounter::getInstance().setDeviceLimit(deviceId, allocSize - 100);
    MemoryCounter::getInstance().setGroupLimit(MemoryType::DEVICE, allocSize - 100);


    // this allocation should fail, since we're allocating too much
    try {
        DataBuffer bufferFailed(allocSize + 1, DataType::INT32);
        ASSERT_TRUE(false);
    } catch (allocation_exception &e) {
        // we expect exception here
    }

    //

    // restore original limits, so subsequent tests do not fail
    MemoryCounter::getInstance().setDeviceLimit(deviceId, odLimit);
    MemoryCounter::getInstance().setGroupLimit(MemoryType::HOST, opLimit);
    MemoryCounter::getInstance().setGroupLimit(MemoryType::DEVICE, osLimit);
}
 */